#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <cutlass/arch/arch.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gett.hpp"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include <cassert>

using namespace cute;

template <typename ElementAB, typename ElementC, typename ElementSF,
          typename ElementAccumulator, typename LayoutSFA, typename LayoutSFB,
          typename ScaleConfig>
__global__ void __get_group_gemm_starts(
    ElementAB** a_offsets, ElementAB** b_offsets, ElementC** out_offsets,
    ElementSF** a_scales_offsets, ElementSF** b_scales_offsets,
    ElementAccumulator** alpha_offsets, LayoutSFA* layout_sfa_base_as_int,
    LayoutSFB* layout_sfb_base_as_int, ElementAB* a_base_as_int,
    ElementAB* b_base_as_int, ElementC* out_base_as_int,
    ElementSF* a_scales_base_as_int, ElementSF* b_scales_base_as_int,
    ElementAccumulator* alphas_base_as_int, const int32_t* expert_offsets,
    const int32_t* sf_offsets, const int32_t* problem_sizes_as_shapes,
    const int K, const int N) {
  int64_t expert_id = threadIdx.x;
  if (expert_id >= gridDim.x * blockDim.x) {
    return;
  }
  // Originally int32_t but upcasting to int64_t to avoid overflow
  // during offset calculations
  int64_t expert_offset = static_cast<int64_t>(expert_offsets[expert_id]);
  int64_t sf_offset = static_cast<int64_t>(sf_offsets[expert_id]);
  // size for block in block scale.
  int64_t group_size = 16;
  int64_t m = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3]);
  int64_t n = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3 + 1]);
  int64_t k = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3 + 2]);
  assert((m >= 0 && n == N && k == K && k % 2 == 0) &&
         "unexpected problem sizes");

  int64_t half_k = static_cast<int64_t>(k / 2);
  int64_t group_k = static_cast<int64_t>(k / group_size);
  // Shape of A as uint8/byte = [M, K // 2]
  // Shape of B as uint8/byte = [E, N, K // 2]
  a_offsets[expert_id] = a_base_as_int + expert_offset * half_k;

  b_offsets[expert_id] = b_base_as_int + expert_id * n * half_k;
  // Shape of C = [M, N]
  out_offsets[expert_id] = out_base_as_int + expert_offset * n;
  // Shape of a_scale = [sum(sf_sizes), K // group_size]
  a_scales_offsets[expert_id] = a_scales_base_as_int + sf_offset * group_k;

  assert((reinterpret_cast<uintptr_t>(a_scales_offsets[expert_id]) % 128) ==
             0 &&
         "TMA requires 128-byte alignment");

  // Shape of B scale = [E, N, K // group_size]
  b_scales_offsets[expert_id] = b_scales_base_as_int + expert_id * n * group_k;
  assert((reinterpret_cast<uintptr_t>(b_scales_offsets[expert_id]) % 128) ==
             0 &&
         "TMA requires 128-byte alignment");
  // Shape of alpha = [E]
  alpha_offsets[expert_id] = alphas_base_as_int + expert_id;

  LayoutSFA* layout_sfa_ptr = layout_sfa_base_as_int + expert_id;
  LayoutSFB* layout_sfb_ptr = layout_sfb_base_as_int + expert_id;

  *layout_sfa_ptr = ScaleConfig::tile_atom_to_shape_SFA(cute::make_shape(
      static_cast<int>(m), static_cast<int>(n), static_cast<int>(k), 1));
  *layout_sfb_ptr = ScaleConfig::tile_atom_to_shape_SFB(cute::make_shape(
      static_cast<int>(m), static_cast<int>(n), static_cast<int>(k), 1));
}

#define __CALL_GET_STARTS_KERNEL_BLOCKSCALE(ELEMENT_AB_TYPE, SF_TYPE,         \
                                            TENSOR_C_TYPE, C_TYPE, LayoutSFA, \
                                            LayoutSFB, ScaleConfig)           \
  else if (out_tensors.dtype() == TENSOR_C_TYPE) {                            \
    __get_group_gemm_starts<ELEMENT_AB_TYPE, C_TYPE, SF_TYPE, float,          \
                            LayoutSFA, LayoutSFB, ScaleConfig>                \
        <<<1, num_experts, 0, stream>>>(                                      \
            static_cast<ELEMENT_AB_TYPE**>(a_starts.data_ptr()),              \
            static_cast<ELEMENT_AB_TYPE**>(b_starts.data_ptr()),              \
            static_cast<C_TYPE**>(out_starts.data_ptr()),                     \
            static_cast<SF_TYPE**>(a_scales_starts.data_ptr()),               \
            static_cast<SF_TYPE**>(b_scales_starts.data_ptr()),               \
            static_cast<float**>(alpha_starts.data_ptr()),                    \
            reinterpret_cast<LayoutSFA*>(layout_sfa.data_ptr()),              \
            reinterpret_cast<LayoutSFB*>(layout_sfb.data_ptr()),              \
            static_cast<ELEMENT_AB_TYPE*>(a_tensors.data_ptr()),              \
            static_cast<ELEMENT_AB_TYPE*>(b_tensors.data_ptr()),              \
            static_cast<C_TYPE*>(out_tensors.data_ptr()),                     \
            static_cast<SF_TYPE*>(a_scales.data_ptr()),                       \
            static_cast<SF_TYPE*>(b_scales.data_ptr()),                       \
            static_cast<float*>(alphas.data_ptr()),                           \
            static_cast<int32_t*>(expert_offsets.data_ptr()),                 \
            static_cast<int32_t*>(sf_offsets.data_ptr()),                     \
            static_cast<int32_t*>(problem_sizes.data_ptr()), K, N);           \
  }

template <typename LayoutSFA, typename LayoutSFB, typename ScaleConfig>
void run_get_group_gemm_starts(
    const torch::Tensor& a_starts, const torch::Tensor& b_starts,
    const torch::Tensor& out_starts, const torch::Tensor& a_scales_starts,
    const torch::Tensor& b_scales_starts, const torch::Tensor& alpha_starts,
    const torch::Tensor& layout_sfa, const torch::Tensor& layout_sfb,
    /*these are used for their base addresses*/
    torch::Tensor const& a_tensors, torch::Tensor const& b_tensors,
    torch::Tensor const& out_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& alphas,
    torch::Tensor const& expert_offsets, torch::Tensor const& sf_offsets,
    torch::Tensor const& problem_sizes, int M, int N, int K) {
  int num_experts = (int)expert_offsets.size(0);
  auto stream = at::cuda::getCurrentCUDAStream(a_tensors.device().index());

  TORCH_CHECK(out_tensors.size(1) == N,
              "Output tensor shape doesn't match expected shape");
  TORCH_CHECK(K / 2 == b_tensors.size(2),
              "b_tensors(dim = 2) and a_tensors(dim = 1) trailing"
              " dimension must match");
  if (false) {
  }
  //(ELEMENT_AB_TYPE, BS_TYPE, TENSOR_C_TYPE, C_TYPE, LayoutSFA, LayoutSFB,
  // ScaleConfig)
  __CALL_GET_STARTS_KERNEL_BLOCKSCALE(
      cutlass::float_e2m1_t, cutlass::float_ue4m3_t, torch::kBFloat16,
      cutlass::bfloat16_t, LayoutSFA, LayoutSFB, ScaleConfig)
  __CALL_GET_STARTS_KERNEL_BLOCKSCALE(cutlass::float_e2m1_t,
                                      cutlass::float_ue4m3_t, torch::kFloat16,
                                      half, LayoutSFA, LayoutSFB, ScaleConfig)
  else {
    TORCH_CHECK(false, "Invalid output type (must be float16 or bfloat16)");
  }
}

template <typename OutType>
void run_fp4_blockwise_scaled_group_mm(
    torch::Tensor& output, const torch::Tensor& a, const torch::Tensor& b,
    const torch::Tensor& a_blockscale, const torch::Tensor& b_blockscales,
    const torch::Tensor& alphas, const torch::Tensor& problem_sizes,
    const torch::Tensor& expert_offsets, const torch::Tensor& sf_offsets, int M,
    int N, int K) {
  using ProblemShape =
      cutlass::gemm::GroupProblemShape<Shape<int32_t, int32_t, int32_t>>;
  using ElementType = cutlass::float_e2m1_t;
  using ElementSFType = cutlass::float_ue4m3_t;
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;

  using ElementC = OutType;
  using ElementD = ElementC;
  using ElementAccumulator = float;
  // Layout definitions
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using LayoutD = LayoutC;

  // Alignment constraints
  static constexpr int AlignmentA = 32;
  static constexpr int AlignmentB = 32;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  // Architecture definitions
  using ArchTag = cutlass::arch::Sm100;
  using EpilogueOperatorClass =
      cutlass::arch::OpClassTensorOp;  // Epilogue Operator class tag
  using MainloopOperatorClass =
      cutlass::arch::OpClassBlockScaledTensorOp;  // Mainloop Operator class tag
  using StageCountType =
      cutlass::gemm::collective::StageCountAuto;  // Stage count maximized based
                                                  // on the tile size

  using ClusterShape = Shape<_1, _1, _1>;
  struct MMA1SMConfig {
    using MmaTileShape = Shape<_128, _128, _128>;
    using KernelSchedule = cutlass::gemm::
        KernelPtrArrayTmaWarpSpecialized1SmNvf4Sm100;  // Kernel to launch
    using EpilogueSchedule =
        cutlass::epilogue::PtrArrayTmaWarpSpecialized1Sm;  // Epilogue to launch
  };

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, EpilogueOperatorClass, typename MMA1SMConfig::MmaTileShape,
          ClusterShape, Shape<_128, _64>, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutC*, AlignmentC, ElementD,
          LayoutC*, AlignmentD,
          typename MMA1SMConfig::EpilogueSchedule>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, MainloopOperatorClass, ElementA, LayoutA*, AlignmentA,
          ElementB, LayoutB*, AlignmentB, ElementAccumulator,
          typename MMA1SMConfig::MmaTileShape, ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          typename MMA1SMConfig::KernelSchedule>::CollectiveOp;

  using GemmKernel =
      cutlass::gemm::kernel::GemmUniversal<ProblemShape, CollectiveMainloop,
                                           CollectiveEpilogue>;

  using Gemm1SM = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using Gemm = Gemm1SM;
  using StrideA = typename Gemm::GemmKernel::InternalStrideA;
  using StrideB = typename Gemm::GemmKernel::InternalStrideB;
  using StrideC = typename Gemm::GemmKernel::InternalStrideC;
  using StrideD = typename Gemm::GemmKernel::InternalStrideD;

  using LayoutSFA =
      typename Gemm::GemmKernel::CollectiveMainloop::InternalLayoutSFA;
  using LayoutSFB =
      typename Gemm::GemmKernel::CollectiveMainloop::InternalLayoutSFB;
  using ScaleConfig =
      typename Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  using UnderlyingProblemShape = ProblemShape::UnderlyingProblemShape;
  int num_experts = static_cast<int>(expert_offsets.size(0));
  auto options_int =
      torch::TensorOptions().dtype(torch::kInt64).device(a.device());

  torch::Tensor a_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor b_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor out_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor a_scales_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor b_scales_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor alpha_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor layout_sfa = torch::empty({num_experts, 5}, options_int);
  torch::Tensor layout_sfb = torch::empty({num_experts, 5}, options_int);
  torch::Tensor c_strides1 =
      torch::full({num_experts}, output.stride(0), options_int);
  torch::Tensor a_strides1 =
      torch::full({num_experts}, a.stride(0) * 2, options_int);
  torch::Tensor b_strides1 =
      torch::full({num_experts}, b.stride(1) * 2, options_int);

  run_get_group_gemm_starts<LayoutSFA, LayoutSFB, ScaleConfig>(
      a_ptrs, b_ptrs, out_ptrs, a_scales_ptrs, b_scales_ptrs, alpha_ptrs,
      layout_sfa, layout_sfb, a, b, output, a_blockscale, b_blockscales, alphas,
      expert_offsets, sf_offsets, problem_sizes, M, N, K);

  // Create an instance of the GEMM
  Gemm gemm_op;

  // Initialize problem_sizes_as_shapes correctly
  UnderlyingProblemShape* problem_sizes_as_shapes =
      static_cast<UnderlyingProblemShape*>(problem_sizes.data_ptr());

  // Set the Scheduler info
  cutlass::KernelHardwareInfo hw_info;
  using RasterOrderOptions = typename cutlass::gemm::kernel::detail::
      PersistentTileSchedulerSm100GroupParams<
          typename ProblemShape::UnderlyingProblemShape>::RasterOrderOptions;
  typename Gemm::GemmKernel::TileSchedulerArguments scheduler;
  scheduler.raster_order = RasterOrderOptions::AlongM;
  hw_info.device_id = a.get_device();
  static std::unordered_map<int, int> cached_sm_counts;
  if (cached_sm_counts.find(hw_info.device_id) == cached_sm_counts.end()) {
    cached_sm_counts[hw_info.device_id] =
        cutlass::KernelHardwareInfo::query_device_multiprocessor_count(
            hw_info.device_id);
  }
  hw_info.sm_count = min(cached_sm_counts[hw_info.device_id], INT_MAX);

  // Mainloop Arguments
  typename GemmKernel::MainloopArguments mainloop_args{
      static_cast<const ElementType**>(a_ptrs.data_ptr()),
      static_cast<StrideA*>(a_strides1.data_ptr()),
      static_cast<const ElementType**>(b_ptrs.data_ptr()),
      static_cast<StrideB*>(b_strides1.data_ptr()),
      static_cast<const ElementSFType**>(a_scales_ptrs.data_ptr()),
      reinterpret_cast<LayoutSFA*>(layout_sfa.data_ptr()),
      static_cast<const ElementSFType**>(b_scales_ptrs.data_ptr()),
      reinterpret_cast<LayoutSFB*>(layout_sfb.data_ptr())};

  // Epilogue Arguments
  typename GemmKernel::EpilogueArguments epilogue_args{
      {},  // epilogue.thread
      nullptr,
      static_cast<StrideC*>(c_strides1.data_ptr()),
      static_cast<ElementD**>(out_ptrs.data_ptr()),
      static_cast<StrideC*>(c_strides1.data_ptr())};
  auto& fusion_args = epilogue_args.thread;
  fusion_args.alpha_ptr_array =
      reinterpret_cast<float**>(alpha_ptrs.data_ptr());
  fusion_args.dAlpha = {_0{}, _0{}, 1};

  // Gemm Arguments
  typename GemmKernel::Arguments args{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {num_experts, problem_sizes_as_shapes, nullptr},
      mainloop_args,
      epilogue_args,
      hw_info,
      scheduler};

  size_t workspace_size = Gemm::get_workspace_size(args);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto can_implement_status = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement_status == cutlass::Status::kSuccess,
              "Failed to implement GEMM");

  // Run the GEMM
  auto status = gemm_op.initialize(args, workspace.data_ptr());
  TORCH_CHECK(status == cutlass::Status::kSuccess, "Failed to initialize GEMM");

  status = gemm_op.run(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess, "Failed to run GEMM");
}

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;
constexpr auto SF_DTYPE = at::ScalarType::Float8_e4m3fn;

#define CHECK_TYPE(x, st, m) \
  TORCH_CHECK(x.scalar_type() == st, ": Inconsistency of Tensor type:", m)
#define CHECK_TH_CUDA(x, m) \
  TORCH_CHECK(x.is_cuda(), m, ": must be a CUDA tensor.")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, ": must be contiguous.")
#define CHECK_INPUT(x, st, m) \
  CHECK_TH_CUDA(x, m);        \
  CHECK_CONTIGUOUS(x, m);     \
  CHECK_TYPE(x, st, m)

void cutlass_fp4_group_mm(
    torch::Tensor& output, const torch::Tensor& a, const torch::Tensor& b,
    const torch::Tensor& a_blockscale, const torch::Tensor& b_blockscales,
    const torch::Tensor& alphas, const torch::Tensor& problem_sizes,
    const torch::Tensor& expert_offsets, const torch::Tensor& sf_offsets) {
#if defined ENABLE_NVFP4 && ENABLE_NVFP4
  // Input validation
  CHECK_INPUT(a, FLOAT4_E2M1X2, "a");
  CHECK_INPUT(b, FLOAT4_E2M1X2, "b");
  CHECK_INPUT(a_blockscale, SF_DTYPE, "a_blockscale");
  CHECK_INPUT(b_blockscales, SF_DTYPE, "b_blockscales");
  CHECK_INPUT(alphas, at::ScalarType::Float, "alphas");

  TORCH_CHECK(a_blockscale.dim() == 2,
              "expected a_blockscale to be of shape [num_experts, rounded_m,"
              " k // group_size], observed rank: ",
              a_blockscale.dim())
  TORCH_CHECK(b_blockscales.dim() == 3,
              "expected b_blockscale to be of shape: "
              " [num_experts, n, k // group_size], observed rank: ",
              b_blockscales.dim())
  TORCH_CHECK(problem_sizes.dim() == 2, "problem_sizes must be  a 2D tensor");
  TORCH_CHECK(problem_sizes.size(1) == 3,
              "problem_sizes must have the shape (num_experts, 3)");
  TORCH_CHECK(problem_sizes.size(0) == expert_offsets.size(0),
              "Number of experts in problem_sizes must match expert_offsets");
  TORCH_CHECK(problem_sizes.dtype() == torch::kInt32,
              "problem_sizes must be int32.");

  int M = static_cast<int>(a.size(0));
  int N = static_cast<int>(b.size(1));
  int E = static_cast<int>(b.size(0));
  int K = static_cast<int>(2 * b.size(2));

  if (output.scalar_type() == torch::kBFloat16) {
    run_fp4_blockwise_scaled_group_mm<cutlass::bfloat16_t>(
        output, a, b, a_blockscale, b_blockscales, alphas, problem_sizes,
        expert_offsets, sf_offsets, M, N, K);
  } else {
    run_fp4_blockwise_scaled_group_mm<cutlass::half_t>(
        output, a, b, a_blockscale, b_blockscales, alphas, problem_sizes,
        expert_offsets, sf_offsets, M, N, K);
  }
#else
  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_fp4_group_mm kernel, vLLM must "
      "be compiled with ENABLE_NVFP4 for SM100+ and CUDA "
      "12.8 or above.");
#endif
}
