#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>
#include "core/math.hpp"
#include "cuda_compat.h"
#include "dispatch_utils.h"

#include "quantization/fp8/common.cuh"

namespace vllm {

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

// Activation and gating kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          typename fp8_type>
__global__ void act_and_mul_quant_kernel(
    fp8_type* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const float* scale, const int d) {
  const int32_t blocks_per_token = gridDim.y;

  const int32_t elems_per_128bit_load = (128 / 8) / sizeof(scalar_t);

  // We don't expect the hidden dimension to exceed 32 bits so int32 should
  // be safe here.
  const int32_t tgt_elems_per_block = div_ceil(d, blocks_per_token);
  const int32_t elems_per_block =
      round_to_next_multiple_of(tgt_elems_per_block, elems_per_128bit_load);
  const int32_t block_start = blockIdx.y * elems_per_block;
  int32_t block_end = block_start + elems_per_block;
  block_end = block_end > d ? d : block_end;

  // token_idx is 64 bit to prevent 32 bit overflow when the number of tokens
  // is very large
  const int64_t token_idx = blockIdx.x;
  const scalar_t* __restrict__ x_ptr = input + token_idx * 2 * d;
  const scalar_t* __restrict__ y_ptr = input + token_idx * 2 * d + d;
  fp8_type* __restrict__ out_ptr = out + token_idx * d;

  // 128-bit vectorized code
  const int32_t vec_loop_end =
      round_to_previous_multiple_of(elems_per_128bit_load, block_end);
  const int32_t vec_end_idx = vec_loop_end / elems_per_128bit_load;
  const int32_t vec_start_idx = block_start / elems_per_128bit_load;

  const int4* __restrict__ x_128bit_ptr = reinterpret_cast<const int4*>(x_ptr);
  const int4* __restrict__ y_128bit_ptr = reinterpret_cast<const int4*>(y_ptr);
  int2* __restrict__ out_128bit_ptr = reinterpret_cast<int2*>(out_ptr);

  float inverted_scale = 1 / *scale;
#pragma unroll
  for (int32_t vec_idx = vec_start_idx + threadIdx.x; vec_idx < vec_end_idx;
       vec_idx += blockDim.x) {
    const int4 x_128bit = VLLM_LDG(&x_128bit_ptr[vec_idx]);
    const int4 y_128bit = VLLM_LDG(&y_128bit_ptr[vec_idx]);
    using scalar_128bit_vec_t = std::array<scalar_t, elems_per_128bit_load>;
    using scalar_64bit_vec_t = std::array<fp8_type, elems_per_128bit_load>;

    scalar_64bit_vec_t out_vec;
    const auto x_vec = reinterpret_cast<scalar_128bit_vec_t const&>(x_128bit);
    const auto y_vec = reinterpret_cast<scalar_128bit_vec_t const&>(y_128bit);

#pragma unroll
    for (int i = 0; i < elems_per_128bit_load; i++) {
      out_vec[i] = scaled_fp8_conversion<true, fp8_type>(
          ACT_FN(x_vec[i]) * y_vec[i], inverted_scale);
    }

    out_128bit_ptr[vec_idx] = reinterpret_cast<const int2&>(out_vec);
  }

  // Scalar cleanup code
  if (block_end > vec_loop_end) {
    for (int64_t idx = vec_loop_end + threadIdx.x; idx < block_end;
         idx += blockDim.x) {
      const scalar_t x = VLLM_LDG(&x_ptr[idx]);
      const scalar_t y = VLLM_LDG(&y_ptr[idx]);
      out_ptr[idx] =
          scaled_fp8_conversion<true, fp8_type>(ACT_FN(x) * y, inverted_scale);
    }
  }
}
}  // namespace vllm

// Launch activation, gating, and quantize kernel.
#define LAUNCH_ACTIVATION_GATE_KERNEL(KERNEL)                               \
  int d = input.size(-1) / 2;                                               \
  int64_t num_tokens = input.numel() / input.size(-1);                      \
  dim3 grid(num_tokens, num_tokens > 16 ? num_tokens > 32 ? 1 : 2 : 4);     \
  dim3 block(std::min(d, 512));                                             \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));         \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();             \
  VLLM_DISPATCH_FLOATING_TYPES(                                             \
      input.scalar_type(), "act_and_mul_kernel", [&] {                      \
        VLLM_DISPATCH_FP8_TYPES(                                            \
            out.scalar_type(), "fused_add_rms_norm_kernel_fp8_type", [&] {  \
              vllm::act_and_mul_quant_kernel<scalar_t, KERNEL<scalar_t>,    \
                                             fp8_t>                         \
                  <<<grid, block, 0, stream>>>(out.data_ptr<fp8_t>(),       \
                                               input.data_ptr<scalar_t>(),  \
                                               scale.data_ptr<float>(), d); \
            });                                                             \
      });

void silu_and_mul_quant(torch::Tensor& out,    // [..., d]
                        torch::Tensor& input,  // [..., 2 * d]
                        torch::Tensor& scale) {
  TORCH_CHECK(out.dtype() == torch::kFloat8_e4m3fn ||
              out.dtype() == torch::kFloat8_e4m3fnuz);
  TORCH_CHECK(input.dtype() == torch::kFloat16 ||
              input.dtype() == torch::kBFloat16);
  TORCH_CHECK(input.size(-1) % 2 == 0);
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::silu_kernel);
}
