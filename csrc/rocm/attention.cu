#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, The vLLM team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_fp8.h>
#include <hip/hip_bf16.h>
#include "cuda_compat.h"

#include <algorithm>
#include "../attention/dtype_fp8.cuh"
#include "../quantization/fp8/amd/quant_utils.cuh"

#if defined(__HIPCC__) && (defined(__gfx90a__) || defined(__gfx942__))
  #define __HIP__MI300_MI250__
#endif

#if defined(NDEBUG)
  #undef NDEBUG
  #include <assert.h>
  #define UNREACHABLE_CODE assert(false);
  #define NDEBUG
#else
  #define UNREACHABLE_CODE assert(false);
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#if defined(__HIP__MI300_MI250__)  // TODO: Add NAVI support

  #define GCN_MFMA_INSTR1 __builtin_amdgcn_mfma_f32_16x16x4f32
  #define GCN_MFMA_INSTR __builtin_amdgcn_mfma_f32_4x4x4f16

using floatx4 = __attribute__((__vector_size__(4 * sizeof(float)))) float;
using float16x4 =
    __attribute__((__vector_size__(4 * sizeof(_Float16)))) _Float16;
typedef float16x4 _Half4;
using float16x2 =
    __attribute__((__vector_size__(2 * sizeof(_Float16)))) _Float16;
typedef float16x2 _Half2;
typedef struct _Half8 {
  _Half4 xy[2];
} _Half8;

using bit16_t = uint16_t;
using bit16x4 = __attribute__((__vector_size__(4 * sizeof(uint16_t)))) uint16_t;
typedef bit16x4 _B16x4;
typedef struct _B16x8 {
  _B16x4 xy[2];
} _B16x8;

using _B8x8 = uint2;
using _B8x4 = int32_t;  // used in builtins
using bit8_t = uint8_t;

typedef struct _B8x16 {
  _B8x8 xy[2];
} _B8x16;

template <typename T, int absz, int cbid, int blgp>
__device__ __forceinline__ floatx4 gcn_mfma4x4x4_instr(const _B16x4& inpA,
                                                       const _B16x4& inpB,
                                                       const floatx4& inpC) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return __builtin_amdgcn_mfma_f32_4x4x4f16(inpA, inpB, inpC, absz, cbid,
                                              blgp);
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __builtin_amdgcn_mfma_f32_4x4x4bf16_1k(inpA, inpB, inpC, absz, cbid,
                                                  blgp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T, int absz, int cbid, int blgp>
__device__ __forceinline__ floatx4 gcn_mfma16x16x16_instr(const _B16x4& inpA,
                                                          const _B16x4& inpB,
                                                          const floatx4& inpC) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return __builtin_amdgcn_mfma_f32_16x16x16f16(inpA, inpB, inpC, absz, cbid,
                                                 blgp);
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __builtin_amdgcn_mfma_f32_16x16x16bf16_1k(inpA, inpB, inpC, absz,
                                                     cbid, blgp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ float to_float(const T& inp) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return (float)inp;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __bfloat162float(inp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ T from_float(const float& inp) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return (_Float16)inp;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __float2bfloat16(inp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ _B16x4 from_floatx4(const floatx4& inp) {
  [[maybe_unused]] union tmpcvt {
    uint16_t u;
    _Float16 f;
    __hip_bfloat16 b;
  } t16;
  _B16x4 ret;
  if constexpr (std::is_same<T, _Float16>::value) {
    union h2cvt {
      __half2 h2[2];
      _B16x4 b16x4;
    } u;
    u.h2[0] = __float22half2_rn(make_float2(inp[0], inp[1]));
    u.h2[1] = __float22half2_rn(make_float2(inp[2], inp[3]));
    return u.b16x4;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    for (int i = 0; i < 4; i++) {
      union fcvt {
        uint32_t u32;
        float f32;
      } u;
      u.f32 = inp[i];
      u.u32 += 0x7fff + ((u.u32 >> 16) & 1);  // BF16 RNE with no nan/inf check
      ret[i] = uint16_t(u.u32 >> 16);
    }
    return ret;
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ _B16x4 addx4(const _B16x4& inp1,
                                        const _B16x4& inp2) {
  [[maybe_unused]] union tmpcvt {
    uint16_t u;
    _Float16 f;
    __hip_bfloat16 b;
  } t1, t2, res;
  _B16x4 ret;
  if constexpr (std::is_same<T, _Float16>::value) {
    union h2cvt {
      _B16x4 b16x4;
      __half2 h2[2];
    } u1, u2, s;
    u1.b16x4 = inp1;
    u2.b16x4 = inp2;
    s.h2[0] = u1.h2[0] + u2.h2[0];
    s.h2[1] = u1.h2[1] + u2.h2[1];
    return s.b16x4;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    for (int i = 0; i < 4; i++) {
      union fcvt {
        float f32;
        uint32_t i32;
      } u1, u2, s;
      u1.i32 = uint32_t(inp1[i]) << 16;
      u2.i32 = uint32_t(inp2[i]) << 16;
      s.f32 = u1.f32 + u2.f32;
      ret[i] = uint16_t(s.i32 >> 16);
    }
    return ret;
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

__device__ __forceinline__ floatx4 to_float_fp8x4(const _B8x4& inp) {
  // From MI300+ platforms, we have v_cvt_pk_f32_fp8 instruction
  // to convert 2 packed fp8 to 2 packed fp32 values.
  // However, in MI200 platforms, we only have v_cvt_f32_fp8
  // to convert fp8 values individually. So we added
  // #else case for fewer instructions (# inst=2) in MI300+,
  // and fallback to
  // #if case for other platforms (# inst=4).
  #if defined(__gfx90a__)
  float4 f32x4 = vllm::fp8::vec_conversion<float4, uint32_t>(
      *reinterpret_cast<const uint32_t*>(&inp));
  return *reinterpret_cast<floatx4*>(&f32x4);
  #else  // MI3xx+ optimized builtins
  const auto f0 = __builtin_amdgcn_cvt_pk_f32_fp8(inp, false);
  const auto f1 = __builtin_amdgcn_cvt_pk_f32_fp8(inp, true);
  floatx4 ret;
  ret[0] = f0[0];
  ret[1] = f0[1];
  ret[2] = f1[0];
  ret[3] = f1[1];
  return ret;
  #endif
}

template <typename T>
__device__ __forceinline__ _B16x4 from_floatx4_rtz(const floatx4& inp) {
  _B16x4 ret;
  if constexpr (std::is_same<T, _Float16>::value) {
    union h2cvt {
      _Half2 h2[2];
      _B16x4 b16x4;
    } u;
    u.h2[0] = __builtin_amdgcn_cvt_pkrtz(inp[0], inp[1]);
    u.h2[1] = __builtin_amdgcn_cvt_pkrtz(inp[2], inp[3]);
    return u.b16x4;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    for (int i = 0; i < 4; i++) {
      union fcvt {
        uint32_t i32;
        float f32;
      } u;
      u.f32 = inp[i];
      ret[i] = uint16_t(u.i32 >> 16);
    }
    return ret;
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ _B16x8 convert_b8x8_custom(const _B8x8 input) {
  union {
    _B8x8 b8x8;
    _B8x4 b8x4[2];
  } tmp;
  tmp.b8x8 = input;
  _B16x8 ret;
  for (int i = 0; i < 2; i++) {
    ret.xy[i] = from_floatx4_rtz<T>(to_float_fp8x4(tmp.b8x4[i]));
  }
  return ret;
}

// grid (num_seqs, num_partitions,num_kv_heads)
// block (256)
// clang-format off
template <typename scalar_t, typename cache_t,
          vllm::Fp8KVCacheDataType KV_DTYPE, typename OUTT, int BLOCK_SIZE,
          int HEAD_SIZE, int NUM_THREADS, bool ALIBI_ENABLED, int GQA_RATIO>
__global__
__launch_bounds__(NUM_THREADS, 5) void paged_attention_ll4mi_QKV_mfma16_kernel(
    const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,    // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,    // [num_blocks, num_kv_heads, head_size, block_size]
    const int num_kv_heads,   
    const float scale,    
    const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ context_lens,   // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes, // [num_heads]
    const int q_stride,
    const int kv_block_stride,
    const int kv_head_stride,
    float* __restrict__ exp_sums,           // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,         // [num_seqs, num_heads, max_num_partitions]
    scalar_t* __restrict__ out,             // [num_seqs, num_heads, max_num_partitions, head_size]
    OUTT* __restrict__ final_out,           // [num_seqs, num_heads, head_size]
    int max_ctx_blocks, const float* k_scale, const float* v_scale) {
  // clang-format on
  constexpr int NWARPS = NUM_THREADS / WARP_SIZE;
  const int warpid = threadIdx.x / WARP_SIZE;
  const int laneid = threadIdx.x % WARP_SIZE;
  const int lane4id = laneid % 4;
  const int lane16id = laneid % 16;
  const int rowid = laneid / 16;

  const int seq_idx = blockIdx.x;
  const int partition_idx = blockIdx.y;

  constexpr int T_PAR_SIZE = 256;  // token partition size set to 256

  const int max_num_partitions = gridDim.y;

  const int context_len = context_lens[seq_idx];

  const int partition_start_token_idx =
      partition_idx * T_PAR_SIZE;  // partition_size;
  // exit if partition is out of context for seq
  if (partition_start_token_idx >= context_len) {
    return;
  }

  constexpr int GQA_RATIO4 = DIVIDE_ROUND_UP(GQA_RATIO, 4);

  [[maybe_unused]] __shared__ float shared_qk_max[NWARPS][16 + 1];
  [[maybe_unused]] __shared__ float shared_exp_sum[NWARPS][16 + 1];
  // shared_logits is used for multiple purposes
  __shared__ _B16x4 shared_logits[NWARPS][4][16][4];

  // for QK mfma16x16, layout is QHead/Tokenx16 across every 16 lanes, 16 Bytes
  // HeadElements in each lane, 4x16B HeadElements across 4 rows of warp
  constexpr int ROWS_PER_WARP =
      WARP_SIZE / 16;  // rows refers to 16 lanes; refer DDP (Data Parallel
                       // Processing) terminology
  constexpr int CONTIGUOUS_KV_ELEMS_16B_LOAD =
      16 / sizeof(cache_t);  // 8 for 16 bit cache type, 16 for 8 bit types
  constexpr int QKHE_PER_FETCH =
      CONTIGUOUS_KV_ELEMS_16B_LOAD *
      ROWS_PER_WARP;  // each fetch across a warp fetches these many elements
  constexpr int QK_SIZE_RATIO =
      sizeof(scalar_t) /
      sizeof(cache_t);  // 1 for 16bit types, 2 for 8bit types
  constexpr int QKHELOOP = HEAD_SIZE / QKHE_PER_FETCH;  // 4xQKHE_16B across
                                                        // warp

  _B16x8 Qlocal[QKHELOOP]
               [QK_SIZE_RATIO];  // note that 16 contiguous elements of Q should
                                 // be fetched per lane for 8 bit cache types :
                                 // QK_SIZE_RATIO changes for this

  constexpr int CONTIGUOUS_SCALAR_ELEMS_16B = 16 / sizeof(scalar_t);

  constexpr int TOKENS_PER_WARP =
      T_PAR_SIZE /
      NWARPS;  // sub partition of tokens per warp for qk calculation
  constexpr int TLOOP =
      TOKENS_PER_WARP /
      16;  // each mfma16x16x16 instruction processes 16 tokens

  // can be interpreted as B8x16 for 8 bit types
  _B16x8 Klocal[TLOOP][QKHELOOP];

  const int wg_start_head_idx = blockIdx.z * GQA_RATIO;
  const int wg_start_kv_head_idx = blockIdx.z;
  const int total_num_heads = gridDim.z * GQA_RATIO;

  // for QK mfma, tokens in multiples of TOKENS_PER_WARP are spread across warps
  // each mfma takes QH16xT16x16HE across warp
  // repeat mfmas across QKHELOOP dimension
  // output layout from QKmfma : QH16xT4x4 16 qheads across 16 lanes, 16 tokens
  // across 4 rows x 4 tokens per lane

  const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
  const int last_ctx_block = num_context_blocks - 1;

  const int* block_table_seq = block_tables + seq_idx * max_num_blocks_per_seq;

  int kphysical_block_number[TLOOP];

  // fetch k physical block numbers
  for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
    const int klocal_token_idx =
        TOKENS_PER_WARP * warpid + token_depth * 16 + lane16id;
    const int kglobal_token_idx = partition_start_token_idx + klocal_token_idx;
    const int kblock_idx = (kglobal_token_idx < context_len)
                               ? kglobal_token_idx / BLOCK_SIZE
                               : last_ctx_block;
    kphysical_block_number[token_depth] = block_table_seq[kblock_idx];
  }

  // fetch Q in shared across warps and then write to registers
  const int local_qhead_idx = 4 * warpid + rowid;
  const int global_qhead_idx = wg_start_head_idx + local_qhead_idx;
  const int64_t seq_idx64 = static_cast<int64_t>(seq_idx);
  const scalar_t* q_ptr =
      q + seq_idx64 * q_stride + global_qhead_idx * HEAD_SIZE;

  const int qhead_element = lane16id * CONTIGUOUS_SCALAR_ELEMS_16B;
  if ((local_qhead_idx < GQA_RATIO) && (qhead_element < HEAD_SIZE)) {
    const scalar_t* q_fetch_ptr = q_ptr + qhead_element;
    const _B16x8* q_fetch_ptr_16B =
        reinterpret_cast<const _B16x8*>(q_fetch_ptr);
    _B16x8 tmp = *q_fetch_ptr_16B;
    if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
      const int offset1 =
          lane16id /
          4;  // 16 contiguous chunks of head elems are spread across 4x4lanes
      shared_logits[offset1][lane4id][local_qhead_idx][0] = tmp.xy[0];
      shared_logits[offset1][lane4id][local_qhead_idx][1] = tmp.xy[1];
    } else {
      for (int i = 0; i < 2; i++) {
        const int head_elem = lane16id * 2 + i;  // element id in _B16x4 terms
        const int offset3 = head_elem % 4;
        const int offset2 = (head_elem / 4) % 4;
        const int offset1 = head_elem / 4 / 4;
        shared_logits[offset1][offset2][local_qhead_idx][offset3] = tmp.xy[i];
      }
    }
  }
  __syncthreads();
  for (int qkhe_depth = 0; qkhe_depth < QKHELOOP; qkhe_depth++) {
    for (int qkratio = 0; qkratio < QK_SIZE_RATIO; qkratio++) {
      for (int i = 0; i < 2; i++) {
        Qlocal[qkhe_depth][qkratio].xy[i] =
            shared_logits[qkhe_depth][rowid][lane16id % GQA_RATIO]
                         [2 * qkratio + i];
      }
    }
  }

  constexpr int KX =
      16 / sizeof(cache_t);  // vLLM defines x as 16 Bytes of kv cache elements
  const cache_t* k_ptr = k_cache + wg_start_kv_head_idx * kv_head_stride;

  const int row_head_elem = rowid * CONTIGUOUS_KV_ELEMS_16B_LOAD;
  // fetch K values
  for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
    const int64_t kblock_number =
        static_cast<int64_t>(kphysical_block_number[token_depth]);
    const cache_t* k_ptr2 = k_ptr + kblock_number * kv_block_stride;
    const int klocal_token_idx =
        TOKENS_PER_WARP * warpid + token_depth * 16 + lane16id;
    [[maybe_unused]] const int kglobal_token_idx =
        partition_start_token_idx + klocal_token_idx;
    const int kphysical_block_offset = klocal_token_idx % BLOCK_SIZE;
    const cache_t* k_ptr3 = k_ptr2 + kphysical_block_offset * KX;

    for (int qkhe_depth = 0; qkhe_depth < QKHELOOP; qkhe_depth++) {
      const int head_elem = row_head_elem + qkhe_depth * QKHE_PER_FETCH;
      const int offset1 = head_elem / KX;
      const int offset2 = head_elem % KX;
      const cache_t* k_fetch_ptr = k_ptr3 + offset1 * BLOCK_SIZE * KX + offset2;
      const _B16x8* k_fetch_ptr_16B =
          reinterpret_cast<const _B16x8*>(k_fetch_ptr);
      Klocal[token_depth][qkhe_depth] = *k_fetch_ptr_16B;
    }
  }

  float alibi_slope;
  if constexpr (ALIBI_ENABLED) {
    const int alibi_head_idx = wg_start_head_idx + lane16id;
    alibi_slope = (lane16id < GQA_RATIO) ? alibi_slopes[alibi_head_idx] : 0.f;
  }

  constexpr int VTOKENS_PER_LANE =
      TOKENS_PER_WARP / ROWS_PER_WARP;  // 64/4 = 16 contiguous vtokens per lane
  constexpr int VBLOCKS_PER_LANE =
      1;  // assumes block size >=16, each lane can correspond to 1 block only
  constexpr int VTLOOP = NWARPS;  // corresponds to tokens across warps
  constexpr int VTLANELOOP = DIVIDE_ROUND_UP(
      VTOKENS_PER_LANE,
      CONTIGUOUS_KV_ELEMS_16B_LOAD);  // optimized for 16B fetches; assumes
                                      // minimum block size is 16
  constexpr int VHELOOP = HEAD_SIZE / 16 / NWARPS;

  int vphysical_block_number[VTLOOP][VBLOCKS_PER_LANE];

  // fetch v physical block numbers
  for (int vtoken_depth = 0; vtoken_depth < VTLOOP; vtoken_depth++) {
    for (int vblock_depth = 0; vblock_depth < VBLOCKS_PER_LANE;
         vblock_depth++) {
      const int vlocal_token_idx =
          vtoken_depth * VTOKENS_PER_LANE * ROWS_PER_WARP +
          rowid * VTOKENS_PER_LANE + vblock_depth * BLOCK_SIZE;
      // Safe to use an int32_t here assuming we are working with < 2 billion
      // tokens
      const int vglobal_token_idx =
          partition_start_token_idx + vlocal_token_idx;
      const int vblock_idx = (vglobal_token_idx < context_len)
                                 ? vglobal_token_idx / BLOCK_SIZE
                                 : last_ctx_block;
      vphysical_block_number[vtoken_depth][vblock_depth] =
          block_table_seq[vblock_idx];
    }
  }

  _B16x8 Vlocal[VTLOOP][VHELOOP][VTLANELOOP];  // this could be B8x16 too

  const cache_t* v_ptr = v_cache + wg_start_kv_head_idx * kv_head_stride +
                         ((rowid * VTOKENS_PER_LANE) % BLOCK_SIZE);

  // v fetches are 16head elems across lanes x 16 tokens per lane
  for (int vhe_depth = 0; vhe_depth < VHELOOP; vhe_depth++) {
    const int vhead_elem = vhe_depth * NWARPS * 16 + warpid * 16 + lane16id;
    const cache_t* v_ptr2 = v_ptr + vhead_elem * BLOCK_SIZE;

    for (int vtoken_depth = 0; vtoken_depth < VTLOOP; vtoken_depth++) {
      for (int vfetch_depth = 0; vfetch_depth < VTLANELOOP; vfetch_depth++) {
        const int vblock_depth = 0;
        const int64_t vblock_number = static_cast<int64_t>(
            vphysical_block_number[vtoken_depth][vblock_depth]);
        const cache_t* v_ptr3 = v_ptr2 + (vblock_number * kv_block_stride);

        const cache_t* v_fetch_ptr =
            v_ptr3 + vfetch_depth * CONTIGUOUS_KV_ELEMS_16B_LOAD;
        const _B16x8* v_fetch_ptr_16B =
            reinterpret_cast<const _B16x8*>(v_fetch_ptr);
        Vlocal[vtoken_depth][vhe_depth][vfetch_depth] = *v_fetch_ptr_16B;
      }
    }
  }

  // calculate post qk mfma scale
  float scale2 = scale;
  if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) {
    // multiply by k_scale if fp8 kv cache
    scale2 *= *k_scale;
  }

  floatx4 d_out[TLOOP];
  // qk mfma
  for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
    d_out[token_depth] = {0};
    for (int qkhe_depth = 0; qkhe_depth < QKHELOOP; qkhe_depth++) {
      if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
        for (int qkratio = 0; qkratio < QK_SIZE_RATIO; qkratio++) {
          for (int i = 0; i < 2; i++) {
            d_out[token_depth] = gcn_mfma16x16x16_instr<scalar_t, 0, 0, 0>(
                Klocal[token_depth][qkhe_depth].xy[i],
                Qlocal[qkhe_depth][qkratio].xy[i], d_out[token_depth]);
          }
        }
      } else {  // kv cache dtype fp8
        auto Ktmp = Klocal[token_depth][qkhe_depth];
        _B8x16 Ktmp8x16 = *reinterpret_cast<_B8x16*>(&Ktmp);
        for (int qkratio = 0; qkratio < QK_SIZE_RATIO; qkratio++) {
          _B8x8 Ktmp8x8 = Ktmp8x16.xy[qkratio];
          _B16x8 Klocaltmp = convert_b8x8_custom<scalar_t>(Ktmp8x8);
          for (int i = 0; i < 2; i++) {
            d_out[token_depth] = gcn_mfma16x16x16_instr<scalar_t, 0, 0, 0>(
                Klocaltmp.xy[i], Qlocal[qkhe_depth][qkratio].xy[i],
                d_out[token_depth]);
          }
        }
      }
    }
    d_out[token_depth] *= scale2;
  }

  const int qkout_token_idx =
      partition_start_token_idx + TOKENS_PER_WARP * warpid + rowid * 4;

  // apply alibi
  if constexpr (ALIBI_ENABLED) {
    for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
      const int local_token_idx = qkout_token_idx + token_depth * 16;
      const int alibi_offset = local_token_idx - context_len + 1;
      for (int i = 0; i < 4; i++) {
        d_out[token_depth][i] += alibi_slope * (alibi_offset + i);
      }
    }
  }

  // calculate qk_max and exp_sum per warp and write to shared memory
  float qk_max = -FLT_MAX;
  float exp_sum = 0.0f;

  for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
    const int local_token_idx = qkout_token_idx + token_depth * 16;
    for (int i = 0; i < 4; i++) {
      const float tmp = (local_token_idx + i < context_len)
                            ? d_out[token_depth][i]
                            : -FLT_MAX;
      qk_max = fmaxf(qk_max, tmp);
    }
  }

  for (int mask = WARP_SIZE / 2; mask >= 16; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor(qk_max, mask));
  }

  for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
    const int local_token_idx = qkout_token_idx + token_depth * 16;
    for (int i = 0; i < 4; i++) {
      const float tmp = (local_token_idx + i < context_len)
                            ? __expf(d_out[token_depth][i] - qk_max)
                            : 0.0f;
      d_out[token_depth][i] = tmp;
      exp_sum += tmp;
    }
  }

  for (int mask = WARP_SIZE / 2; mask >= 16; mask /= 2) {
    exp_sum += __shfl_xor(exp_sum, mask);
  }

  __syncthreads();  // sync before writing to shared mem

  float* shared_mem = reinterpret_cast<float*>(shared_logits);
  if (laneid < 16) {
    const int qk_max_offset = warpid * 16 + lane16id;
    shared_mem[qk_max_offset] = qk_max;
    const int exp_sum_offset = NWARPS * 16 + qk_max_offset;
    shared_mem[exp_sum_offset] = exp_sum;
  }

  __syncthreads();

  // calculate partition qk_max and exp_sum
  float partition_qk_max = -FLT_MAX;
  float warp_qk_max_exp[NWARPS];
  float partition_exp_sum = 0.0f;

  for (int w = 0; w < NWARPS; w++) {
    warp_qk_max_exp[w] = shared_mem[w * 16 + lane16id];
    partition_qk_max = fmaxf(partition_qk_max, warp_qk_max_exp[w]);
  }

  for (int w = 0; w < NWARPS; w++) {
    warp_qk_max_exp[w] = __expf(warp_qk_max_exp[w] - partition_qk_max);
    partition_exp_sum +=
        shared_mem[NWARPS * 16 + w * 16 + lane16id] * warp_qk_max_exp[w];
  }

  const float inv_sum_scale =
      __fdividef(1.f, partition_exp_sum + 1e-6f) * warp_qk_max_exp[warpid];

  __syncthreads();

  // disable rtz conversion due to its impact on accuracy.
  constexpr bool LOGITS_RTZ_CONVERSION = false;

  // write logits to shared mem
  for (int token_depth = 0; token_depth < TLOOP; token_depth++) {
    d_out[token_depth] *= inv_sum_scale;
    if constexpr (LOGITS_RTZ_CONVERSION) {
      // use rtz conversion for better performance, with negligible impact on
      // accuracy
      shared_logits[warpid][token_depth][lane16id][rowid] =
          from_floatx4_rtz<scalar_t>(d_out[token_depth]);
    } else {
      shared_logits[warpid][token_depth][lane16id][rowid] =
          from_floatx4<scalar_t>(d_out[token_depth]);
    }
  }

  // write out partition max_logits and exp_sum
  if (threadIdx.x < GQA_RATIO) {
    const int qhead_idx = lane16id;
    const int64_t offset = static_cast<int64_t>(seq_idx) *
                               static_cast<int64_t>(total_num_heads) *
                               static_cast<int64_t>(max_num_partitions) +
                           (static_cast<int64_t>(wg_start_head_idx) +
                            static_cast<int64_t>(qhead_idx)) *
                               static_cast<int64_t>(max_num_partitions) +
                           static_cast<int64_t>(partition_idx);
    max_logits[offset] = partition_qk_max;
    exp_sums[offset] = partition_exp_sum;
  }

  __syncthreads();

  constexpr int ELEMS8_ELEMS4_RATIO = 8 / 4;
  constexpr int ELEMS16_ELEMS8_RATIO = 16 / 8;

  _B16x4 outelems[VHELOOP];
  // Softmax V mfma
  // v layout: 16he across lanes x 16 tokens per lane
  for (int vhe_depth = 0; vhe_depth < VHELOOP; vhe_depth++) {
    floatx4 tmp_out = {0};

    for (int vtoken_depth = 0; vtoken_depth < VTLOOP; vtoken_depth++) {
      if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
        for (int vfetch_depth = 0; vfetch_depth < VTLANELOOP; vfetch_depth++) {
          for (int i = 0; i < ELEMS8_ELEMS4_RATIO; i++) {
            const int offset = rowid * VTLANELOOP * ELEMS8_ELEMS4_RATIO +
                               vfetch_depth * ELEMS8_ELEMS4_RATIO + i;
            const int offset1 = offset % ROWS_PER_WARP;
            const int offset2 = offset / ROWS_PER_WARP;
            // output format is 16 qheads across 16 lanes, 16 head elems spread
            // across 4 rows
            tmp_out = gcn_mfma16x16x16_instr<scalar_t, 0, 0, 0>(
                Vlocal[vtoken_depth][vhe_depth][vfetch_depth].xy[i],
                shared_logits[vtoken_depth][offset2][lane16id][offset1],
                tmp_out);
          }
        }
        // KV cache fp8
      } else {
        for (int vfetch_depth = 0; vfetch_depth < VTLANELOOP; vfetch_depth++) {
          _B16x8 Vtmp = Vlocal[vtoken_depth][vhe_depth][vfetch_depth];
          // reinterpret V format as 16 elements of 8bits
          _B8x16 Vtmp8x16 = *reinterpret_cast<_B8x16*>(&Vtmp);
          for (int j = 0; j < ELEMS16_ELEMS8_RATIO; j++) {
            _B8x8 Vtmp8x8 = Vtmp8x16.xy[j];
            _B16x8 Vlocaltmp = convert_b8x8_custom<scalar_t>(Vtmp8x8);
            for (int i = 0; i < ELEMS8_ELEMS4_RATIO; i++) {
              const int offset =
                  rowid * ELEMS16_ELEMS8_RATIO * ELEMS8_ELEMS4_RATIO +
                  j * ELEMS8_ELEMS4_RATIO + i;
              const int offset1 = offset % ROWS_PER_WARP;
              const int offset2 = offset / ROWS_PER_WARP;
              // output format is 16 qheads across 16 lanes, 16 head elems
              // spread across 4 rows
              tmp_out = gcn_mfma16x16x16_instr<scalar_t, 0, 0, 0>(
                  Vlocaltmp.xy[i],
                  shared_logits[vtoken_depth][offset2][lane16id][offset1],
                  tmp_out);
            }
          }
        }
      }
    }
    // apply post Softmax V mfma v_scale
    if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) {
      tmp_out *= *v_scale;
    }
    outelems[vhe_depth] = from_floatx4<scalar_t>(tmp_out);
  }

  __syncthreads();

  // store Softmax-V mfma output to shared mem
  for (int vhe_depth = 0; vhe_depth < VHELOOP; vhe_depth++) {
    // lane16 id head dimension; rowid head element dimension
    shared_logits[warpid][vhe_depth][lane16id][rowid] = outelems[vhe_depth];
  }

  __syncthreads();

  // write to tmp_out with coalesced writes after reading from shared mem
  if (warpid == 0) {
    _B16x8 vout[GQA_RATIO4];
    // each lane writes out 16Bytes of tmp_out along head elem dimension
    const int head_elem_idx = lane16id * 8;
    if (head_elem_idx < HEAD_SIZE) {
      for (int h = 0; h < GQA_RATIO4; h++) {
        const int local_head_idx = 4 * h + rowid;
        const int offset1 = (head_elem_idx / 16) % 4;
        const int offset2 = head_elem_idx / 16 / NWARPS;
        const int offset3 = (head_elem_idx / 4) % 4;
        for (int i = 0; i < 2; i++) {
          vout[h].xy[i] =
              shared_logits[offset1][offset2][local_head_idx][offset3 + i];
        }
      }

      const int64_t hsz_maxp_mult =
          static_cast<int64_t>(HEAD_SIZE * max_num_partitions);
      scalar_t* out_ptr = out + seq_idx * total_num_heads * hsz_maxp_mult +
                          partition_idx * HEAD_SIZE;
      for (int h = 0; h < GQA_RATIO4; h++) {
        const int local_head_idx = 4 * h + rowid;
        if (local_head_idx < GQA_RATIO) {
          const int64_t out_head_idx =
              static_cast<int64_t>(wg_start_head_idx + local_head_idx);
          scalar_t* out_ptr2 = out_ptr + out_head_idx * hsz_maxp_mult;
          scalar_t* out_ptr3 = out_ptr2 + head_elem_idx;
          _B16x8* out_ptr_B16x8 = reinterpret_cast<_B16x8*>(out_ptr3);
          *out_ptr_B16x8 = vout[h];
        }
      }
    }
  }
}

// grid (num_seqs, num_partitions, num_kv_heads)
// block (256 : partition size)
// each WG handles 1 partition per sequence
// clang-format off
template <typename scalar_t, typename cache_t,
          vllm::Fp8KVCacheDataType KV_DTYPE, typename OUTT, int BLOCK_SIZE,
          int HEAD_SIZE, int NUM_THREADS, bool ALIBI_ENABLED,
          int GQA_RATIO>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_QKV_mfma4_kernel(
    const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,    // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,    // [num_blocks, num_kv_heads, head_size, block_size]
    const int num_kv_heads,
    const float scale,
    const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ context_lens,   // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes, // [num_heads]
    const int q_stride,
    const int kv_block_stride,
    const int kv_head_stride,
    float* __restrict__ exp_sums,           // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,         // [num_seqs, num_heads, max_num_partitions]
    scalar_t* __restrict__ out,             // [num_seqs, num_heads, max_num_partitions, head_size]
    OUTT* __restrict__ final_out,           // [num_seqs, num_heads, head_size]
    int max_ctx_blocks, const float* k_scale, const float* v_scale) {
  // clang-format on
  constexpr int NWARPS = NUM_THREADS / WARP_SIZE;
  const int warpid = threadIdx.x / WARP_SIZE;
  const int laneid = threadIdx.x % WARP_SIZE;
  const int lane4id = laneid % 4;

  const int seq_idx = blockIdx.x;
  const int partition_idx = blockIdx.y;
  const int partition_size = blockDim.x;
  const int max_num_partitions = gridDim.y;

  const int context_len = context_lens[seq_idx];
  const int partition_start_token_idx = partition_idx * partition_size;
  // exit if partition is out of context for seq
  if (partition_start_token_idx >= context_len) {
    return;
  }
  // every 4 lanes fetch 4 different qheads
  // qhloop = num loops over qhead dimension
  constexpr int QHLOOP = DIVIDE_ROUND_UP(GQA_RATIO, 4);
  constexpr int GQA_RATIO4 = 4 * QHLOOP;
  __shared__ float shared_qk_max[NWARPS][GQA_RATIO4 + 1];
  __shared__ float shared_exp_sum[NWARPS][GQA_RATIO4 + 1];
  _B16x8 Qlocal[QHLOOP];
  constexpr int x = 16 / sizeof(scalar_t);
  // kheloop = num loops over head_size for 16Bytes of Q/dequantized K elements
  constexpr int KHELOOP = HEAD_SIZE / x;
  _B16x8 Klocal[KHELOOP];
  _B8x8 Klocalb8[KHELOOP];
  // for SoftMax-V Gemm, V head_size dimension is distributed across warp
  // vheloop = num loops to cover v head size dimension
  constexpr int VHELOOP = HEAD_SIZE / WARP_SIZE;
  // softmax out has warp_size tokens across warp
  // vtloop = num loops to cover warp_size(64) tokens with 16Bytes of
  // dequantized V elements
  constexpr int VTLOOP = WARP_SIZE / 8;
  // num vblocks to cover warp_size(64) v elements
  constexpr int VBLOCKS = 8 * VTLOOP / BLOCK_SIZE;
  int vphysical_blocks[VBLOCKS];
  _B16x8 Vlocal[VHELOOP][VTLOOP];
  _B8x8 Vlocalb8[VHELOOP][VTLOOP];
  floatx4 d_out[QHLOOP];
  float qk_max[QHLOOP];

  __shared__ _B16x4 vout_shared[QHLOOP][VHELOOP][WARP_SIZE][NWARPS + 1];

  for (int h = 0; h < QHLOOP; h++) {
    d_out[h] = {0};
    qk_max[h] = -FLT_MAX;
  }

  const int wg_start_head_idx = blockIdx.z * GQA_RATIO;
  const int wg_start_kv_head_idx = blockIdx.z;

  const int warp_start_token_idx =
      partition_start_token_idx + warpid * WARP_SIZE;

  if (warp_start_token_idx >= context_len) {  // warp out of context
  #pragma unroll
    for (int h = 0; h < GQA_RATIO4; h++) {
      shared_qk_max[warpid][h] = -FLT_MAX;
      shared_exp_sum[warpid][h] = 0.0f;
    }
  } else {  // warp within context

    const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
    const int last_ctx_block = num_context_blocks - 1;

    const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
    // token id within partition
    const int local_token_idx = threadIdx.x;
    // token id within sequence
    const int global_token_idx = partition_start_token_idx + local_token_idx;

    // fetch block number for k
    const int block_idx = (global_token_idx < context_len)
                              ? global_token_idx / BLOCK_SIZE
                              : last_ctx_block;

    // fetch k physical block number
    //  int32 physical_block_number leads to overflow when multiplied with
    //  kv_block_stride
    const int64_t physical_block_number =
        static_cast<int64_t>(block_table[block_idx]);

    // fetch vphysical block numbers up front
    const int warp_start_block_idx = warp_start_token_idx / BLOCK_SIZE;
    for (int b = 0; b < VBLOCKS; b++) {
      const int vblock_idx = warp_start_block_idx + b;
      const int vblock_idx_ctx =
          (vblock_idx <= last_ctx_block) ? vblock_idx : last_ctx_block;
      vphysical_blocks[b] = block_table[vblock_idx_ctx];
    }

    // fetch q elements
    // every 4 lanes fetch 8 elems, so warp fetches 8*16 = 128 elems
    const scalar_t* q_ptr =
        q + seq_idx * q_stride + wg_start_head_idx * HEAD_SIZE;
    const _B16x8* q_ptrh8 = reinterpret_cast<const _B16x8*>(q_ptr);
    const int qhead_elemh8 = laneid / 4;

    for (int h = 0; h < QHLOOP - 1; h++) {
      const int qhead_idx = h * 4 + lane4id;
      Qlocal[h] = q_ptrh8[qhead_idx * HEAD_SIZE / 8 + qhead_elemh8];
    }
    const int final_qhead_idx = 4 * (QHLOOP - 1) + lane4id;
    if (final_qhead_idx < GQA_RATIO) {
      Qlocal[QHLOOP - 1] =
          q_ptrh8[final_qhead_idx * HEAD_SIZE / 8 + qhead_elemh8];
    } else {
      Qlocal[QHLOOP - 1].xy[0] = {0};
      Qlocal[QHLOOP - 1].xy[1] = {0};
    }

    // fetch k elements
    const cache_t* k_ptr = k_cache + physical_block_number * kv_block_stride +
                           wg_start_kv_head_idx * kv_head_stride;

    // physical_block_offset is already cast in terms of _B16x8
    const int physical_block_offset = local_token_idx % BLOCK_SIZE;

    // each K fetch is for 8 elements of cache_t which are later dequantized to
    // scalar_t for fp8
    if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
      const _B16x8* k_ptrh8 = reinterpret_cast<const _B16x8*>(k_ptr);
      for (int d = 0; d < KHELOOP; d++) {
        Klocal[d] = k_ptrh8[d * BLOCK_SIZE + physical_block_offset];
      }
    } else {
      // vllm defines X as 16 Bytes of elements of cache_t
      constexpr int X = 16 / sizeof(cache_t);
      const cache_t* k_ptr2 = k_ptr + physical_block_offset * X;
      for (int d = 0; d < KHELOOP; d++) {
        const int head_elem = d * 8;
        const int offset1 = head_elem / X;
        const int offset2 = head_elem % X;
        const cache_t* k_ptr3 = k_ptr2 + offset1 * BLOCK_SIZE * X + offset2;
        Klocalb8[d] = *reinterpret_cast<const _B8x8*>(k_ptr3);
      }
    }

    // optional alibi fetch
    float alibi_slope[QHLOOP];
    if constexpr (ALIBI_ENABLED) {
      for (int h = 0; h < QHLOOP; h++) {
        const int qhead_idx = h * 4 + lane4id;
        alibi_slope[h] = (qhead_idx < GQA_RATIO)
                             ? alibi_slopes[wg_start_head_idx + qhead_idx]
                             : 0.f;
      }
    }

    const cache_t* v_ptr = v_cache + wg_start_kv_head_idx * kv_head_stride;
    // fetch vcache in kv cache auto case
    if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
      const _B16x8* v_ptrh8 = reinterpret_cast<const _B16x8*>(v_ptr);
      // iterate over each v block
      for (int b = 0; b < VBLOCKS; b++) {
        // int32 physical_block_number leads to overflow when multiplied with
        // kv_block_stride
        const int64_t vphysical_block_number =
            static_cast<int64_t>(vphysical_blocks[b]);
        const _B16x8* v_ptrh8b =
            v_ptrh8 + (vphysical_block_number * kv_block_stride) / 8;
        // iterate over each head elem (within head_size)
        for (int h = 0; h < VHELOOP; h++) {
          const int head_size_elem = h * WARP_SIZE + laneid;
          const _B16x8* v_ptrh8be = v_ptrh8b + head_size_elem * BLOCK_SIZE / 8;
          // iterate over all velems within block
          for (int d = 0; d < BLOCK_SIZE / 8; d++) {
            Vlocal[h][b * BLOCK_SIZE / 8 + d] = v_ptrh8be[d];
          }
        }
      }
    }  // if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto)
    // fetch vcache in fp8 case
    else {  // if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto)
      const _B8x8* v_ptrh8 = reinterpret_cast<const _B8x8*>(v_ptr);
      // iterate over each v block
      for (int b = 0; b < VBLOCKS; b++) {
        // int32 physical_block_number leads to overflow when multiplied with
        // kv_block_stride
        const int64_t vphysical_block_number =
            static_cast<int64_t>(vphysical_blocks[b]);
        const _B8x8* v_ptrh8b =
            v_ptrh8 + (vphysical_block_number * kv_block_stride) / 8;
        // iterate over each head elem (within head_size)
        for (int h = 0; h < VHELOOP; h++) {
          const int head_size_elem = h * WARP_SIZE + laneid;
          const _B8x8* v_ptrh8be = v_ptrh8b + head_size_elem * BLOCK_SIZE / 8;
          // iterate over all velems within block
          for (int d = 0; d < BLOCK_SIZE / 8; d++) {
            Vlocalb8[h][b * BLOCK_SIZE / 8 + d] = v_ptrh8be[d];
          }
        }
      }
    }

  #define QK_mfma(x)                                             \
    if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) { \
      Klocal[x] = convert_b8x8_custom<scalar_t>(Klocalb8[x]);    \
    }                                                            \
    for (int h = 0; h < QHLOOP; h++) {                           \
      d_out[h] = gcn_mfma4x4x4_instr<scalar_t, 4, x, 0>(         \
          Qlocal[h].xy[0], Klocal[x].xy[0], d_out[h]);           \
      d_out[h] = gcn_mfma4x4x4_instr<scalar_t, 4, x, 0>(         \
          Qlocal[h].xy[1], Klocal[x].xy[1], d_out[h]);           \
    }
    // QK mfma with Q mfma block broadcast
    // Q values across head_size dimension stored across lanes
    // K values across head_size dimension are stored depthwise within lane
    // Q broadcast with absz, cbid of mfma instruction
    QK_mfma(0);
    QK_mfma(1);
    QK_mfma(2);
    QK_mfma(3);
    QK_mfma(4);
    QK_mfma(5);
    QK_mfma(6);
    QK_mfma(7);
    // below only needed for head size 128
    if constexpr (KHELOOP > 8) {
      QK_mfma(8);
      QK_mfma(9);
      QK_mfma(10);
      QK_mfma(11);
      QK_mfma(12);
      QK_mfma(13);
      QK_mfma(14);
      QK_mfma(15);
    }
  #undef QK_mfma

    float scale2 = scale;
    if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) {
      // post mfma scaling for fp8
      scale2 *= *k_scale;
    }

    for (int h = 0; h < QHLOOP; h++) {
      d_out[h] *= scale2;
    }

    // transpose d_out so that 4 token ids are in each lane, and 4 heads are
    // across 4 lanes
    for (int h = 0; h < QHLOOP; h++) {
      floatx4 tmp = {0};
      for (int i = 0; i < 4; i++) {
        const float B = (lane4id == i) ? 1.0f : 0.0f;
        tmp = __builtin_amdgcn_mfma_f32_4x4x1f32(d_out[h][i], B, tmp, 0, 0, 0);
      }
      d_out[h] = tmp;
    }

    const int lane4_token_idx = 4 * (global_token_idx >> 2);

    if constexpr (ALIBI_ENABLED) {
      const int alibi_offset = lane4_token_idx - context_len + 1;
      for (int h = 0; h < QHLOOP; h++) {
        for (int i = 0; i < 4; i++) {
          d_out[h][i] += alibi_slope[h] * (alibi_offset + i);
        }
      }
    }

    const int bpermute_mask = 4 * (16 * ((laneid >> 2) % 4) + lane4id);

    for (int h = 0; h < QHLOOP; h++) {
      qk_max[h] = -FLT_MAX;
      for (int i = 0; i < 4; i++) {
        qk_max[h] = (lane4_token_idx + i < context_len)
                        ? fmaxf(qk_max[h], d_out[h][i])
                        : qk_max[h];
      }

      // for (int mask = WARP_SIZE / 2; mask >= 4; mask /= 2) {
      //   qk_max[h] = fmaxf(qk_max[h], __shfl_xor(qk_max[h], mask));
      // }
      // faster version of above code with dpp
      asm("v_nop\n v_nop\n v_max_f32_dpp %0, %1, %2 row_ror:4"
          : "=v"(qk_max[h])
          : "v"(qk_max[h]), "v"(qk_max[h]));
      asm("v_nop\n v_nop\n v_max_f32_dpp %0, %1, %2 row_ror:8"
          : "=v"(qk_max[h])
          : "v"(qk_max[h]), "v"(qk_max[h]));

      auto tmp = __builtin_amdgcn_ds_bpermute(
          bpermute_mask, *reinterpret_cast<int*>(&qk_max[h]));
      qk_max[h] = *reinterpret_cast<float*>(&tmp);
      asm("v_nop\n v_nop\n v_max_f32_dpp %0, %1, %2 row_ror:4"
          : "=v"(qk_max[h])
          : "v"(qk_max[h]), "v"(qk_max[h]));
      asm("v_nop\n v_nop\n v_max_f32_dpp %0, %1, %2 row_ror:8"
          : "=v"(qk_max[h])
          : "v"(qk_max[h]), "v"(qk_max[h]));
    }

    float exp_sum[QHLOOP];
    for (int h = 0; h < QHLOOP; h++) {
      exp_sum[h] = 0.0f;
      for (int i = 0; i < 4; i++) {
        d_out[h][i] = (lane4_token_idx + i < context_len)
                          ? __expf(d_out[h][i] - qk_max[h])
                          : 0.0f;
        exp_sum[h] += d_out[h][i];
      }
      // for (int mask = WARP_SIZE / 2; mask >= 4; mask /= 2) {
      //   exp_sum[h] += __shfl_xor(exp_sum[h], mask);
      // }
      // faster version of above code with dpp
      asm("v_nop\n v_nop\n v_add_f32_dpp %0, %1, %2 row_ror:4"
          : "=v"(exp_sum[h])
          : "v"(exp_sum[h]), "v"(exp_sum[h]));
      asm("v_nop\n v_nop\n v_add_f32_dpp %0, %1, %2 row_ror:8"
          : "=v"(exp_sum[h])
          : "v"(exp_sum[h]), "v"(exp_sum[h]));

      auto tmp = __builtin_amdgcn_ds_bpermute(
          bpermute_mask, *reinterpret_cast<int*>(&exp_sum[h]));
      exp_sum[h] = *reinterpret_cast<float*>(&tmp);
      asm("v_nop\n v_nop\n v_add_f32_dpp %0, %1, %2 row_ror:4"
          : "=v"(exp_sum[h])
          : "v"(exp_sum[h]), "v"(exp_sum[h]));
      asm("v_nop\n v_nop\n v_add_f32_dpp %0, %1, %2 row_ror:8"
          : "=v"(exp_sum[h])
          : "v"(exp_sum[h]), "v"(exp_sum[h]));
    }

    if (laneid < 4) {
      for (int h = 0; h < QHLOOP; h++) {
        const int head_idx = 4 * h + lane4id;
        shared_qk_max[warpid][head_idx] = qk_max[h];
        shared_exp_sum[warpid][head_idx] = exp_sum[h];
      }
    }
  }  // warp within context

  __syncthreads();

  const int num_heads = gridDim.z * GQA_RATIO;
  float* max_logits_ptr =
      max_logits + seq_idx * num_heads * max_num_partitions + partition_idx;
  float* exp_sums_ptr =
      exp_sums + seq_idx * num_heads * max_num_partitions + partition_idx;
  // calculate qk_max and exp_sums for partition
  for (int h = 0; h < QHLOOP; h++) {
    float global_qk_max = -FLT_MAX;
    float warp_qk_max[NWARPS];
    const int head_idx = 4 * h + lane4id;
    for (int w = 0; w < NWARPS; w++) {
      warp_qk_max[w] = shared_qk_max[w][head_idx];
      global_qk_max = fmaxf(global_qk_max, warp_qk_max[w]);
    }
    float global_exp_sum = 0.0f;
    for (int w = 0; w < NWARPS; w++) {
      global_exp_sum +=
          shared_exp_sum[w][head_idx] * __expf(warp_qk_max[w] - global_qk_max);
    }
    if (head_idx < GQA_RATIO) {
      max_logits_ptr[(wg_start_head_idx + head_idx) * max_num_partitions] =
          global_qk_max;
      exp_sums_ptr[(wg_start_head_idx + head_idx) * max_num_partitions] =
          global_exp_sum;
    }
    const float global_inv_sum_scale = __fdividef(1.f, global_exp_sum + 1e-6f) *
                                       __expf(qk_max[h] - global_qk_max);
    d_out[h] *= global_inv_sum_scale;
  }
  constexpr bool LOGITS_RTZ_CONVERSION = false;
  // logits[h] -> every 4 lanes hold 4 heads, each lane holds 4 tokens, there
  // are 4x16 tokens across warp
  _B16x4 logits[QHLOOP];
  for (int h = 0; h < QHLOOP; h++) {
    if constexpr (LOGITS_RTZ_CONVERSION) {
      // use rtz for faster performance with no perceivable accuracy loss
      logits[h] = from_floatx4_rtz<scalar_t>(d_out[h]);
    } else {
      logits[h] = from_floatx4<scalar_t>(d_out[h]);
    }
  }

  if (warp_start_token_idx >= context_len) {  // warp out of context
    for (int qh = 0; qh < QHLOOP; qh++) {
      for (int vh = 0; vh < VHELOOP; vh++) {
        vout_shared[qh][vh][laneid][warpid] = {0};
      }
    }
  } else {  // warp in context
  #define SV_mfma(x)                                                  \
    if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) {      \
      Vlocal[vh][x] = convert_b8x8_custom<scalar_t>(Vlocalb8[vh][x]); \
    }                                                                 \
    for (int qh = 0; qh < QHLOOP; qh++) {                             \
      acc[qh] = gcn_mfma4x4x4_instr<scalar_t, 4, 2 * x, 0>(           \
          logits[qh], Vlocal[vh][x].xy[0], acc[qh]);                  \
      acc[qh] = gcn_mfma4x4x4_instr<scalar_t, 4, 2 * x + 1, 0>(       \
          logits[qh], Vlocal[vh][x].xy[1], acc[qh]);                  \
    }

    for (int vh = 0; vh < VHELOOP; vh++) {
      floatx4 acc[QHLOOP];
      for (int qh = 0; qh < QHLOOP; qh++) {
        acc[qh] = {0};
      }
      // SoftMax-V calculation
      // logits -> token dimension is distributed across lanes
      // Vlocal -> token dimension is depthwise within lane
      // uses mfma instruction block broadcast for logits
      SV_mfma(0);
      SV_mfma(1);
      SV_mfma(2);
      SV_mfma(3);
      SV_mfma(4);
      SV_mfma(5);
      SV_mfma(6);
      SV_mfma(7);

      for (int qh = 0; qh < QHLOOP; qh++) {
        if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) {
          // post mfma v scale for fp8
          acc[qh] *= *v_scale;
        }
        vout_shared[qh][vh][laneid][warpid] = from_floatx4<scalar_t>(acc[qh]);
      }
    }

  #undef SV_mfma
  }  // warp in context

  __syncthreads();

  // final write to tmp_out after vout accumulation
  if (warpid == 0) {
    _B16x4 vout[QHLOOP][VHELOOP];
    // iterate across heads
    for (int qh = 0; qh < QHLOOP; qh++) {
      // iterate over each v head elem (within head_size)
      for (int vh = 0; vh < VHELOOP; vh++) {
        vout[qh][vh] = {0};
        for (int w = 0; w < NWARPS; w++) {
          vout[qh][vh] =
              addx4<scalar_t>(vout[qh][vh], vout_shared[qh][vh][laneid][w]);
        }
      }
    }

    scalar_t* out_ptr = out +
                        seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
                        partition_idx * HEAD_SIZE;
    const int out_num_partitions = max_num_partitions;
    bit16_t* out_ptr_b16 = reinterpret_cast<bit16_t*>(out_ptr);
    for (int qh = 0; qh < QHLOOP; qh++) {
      for (int vh = 0; vh < VHELOOP; vh++) {
        const int head_size_elem = vh * WARP_SIZE + laneid;
        for (int i = 0; i < 4; i++) {
          const int head_idx = 4 * qh + i;
          if (head_idx < GQA_RATIO) {
            out_ptr_b16[(wg_start_head_idx + head_idx) * out_num_partitions *
                            HEAD_SIZE +
                        head_size_elem] = vout[qh][vh][i];
          }
        }
      }
    }
  }  // warpid == 0
}

// Grid: (num_heads, num_seqs).
template <typename scalar_t, typename OUTT, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE, int NPAR_LOOPS>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_reduce_kernel(
    OUTT* __restrict__ out,                // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,    // [num_seqs, num_heads,
                                           // max_num_partitions]
    const float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                           // max_num_partitions]
    const scalar_t* __restrict__ tmp_out,  // [num_seqs, num_heads,
                                           // max_num_partitions, head_size]
    const int* __restrict__ context_lens,  // [num_seqs]
    const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int context_len = context_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(context_len, PARTITION_SIZE);
  [[maybe_unused]] constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warpid = threadIdx.x / WARP_SIZE;
  [[maybe_unused]] const int laneid = threadIdx.x % WARP_SIZE;

  __shared__ float shared_global_exp_sum;
  // max num partitions supported is warp_size * NPAR_LOOPS
  __shared__ float shared_exp_sums[NPAR_LOOPS * WARP_SIZE];

  if (warpid == 0) {
    const float* max_logits_ptr = max_logits +
                                  seq_idx * num_heads * max_num_partitions +
                                  head_idx * max_num_partitions;

    // valid partition is the last valid partition in case threadid > num
    // partitions
    int valid_partition[NPAR_LOOPS];
    float reg_max_logit[NPAR_LOOPS];
    const int last_valid_partition = num_partitions - 1;

  #pragma unroll
    for (int i = 0; i < NPAR_LOOPS; i++) {
      const int partition_no = i * WARP_SIZE + threadIdx.x;
      valid_partition[i] =
          (partition_no < num_partitions) ? partition_no : last_valid_partition;
    }
  #pragma unroll
    for (int i = 0; i < NPAR_LOOPS; i++) {
      reg_max_logit[i] = max_logits_ptr[valid_partition[i]];
    }
    float max_logit = reg_max_logit[0];
  #pragma unroll
    for (int i = 1; i < NPAR_LOOPS; i++) {
      max_logit = fmaxf(max_logit, reg_max_logit[i]);
    }

  #pragma unroll
    for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
      max_logit = fmaxf(max_logit, __shfl_xor(max_logit, mask));
    }

    const float* exp_sums_ptr = exp_sums +
                                seq_idx * num_heads * max_num_partitions +
                                head_idx * max_num_partitions;

    float rescaled_exp_sum[NPAR_LOOPS];
  #pragma unroll
    for (int i = 0; i < NPAR_LOOPS; i++) {
      rescaled_exp_sum[i] = exp_sums_ptr[valid_partition[i]];
    }
  #pragma unroll
    for (int i = 0; i < NPAR_LOOPS; i++) {
      const int partition_no = i * WARP_SIZE + threadIdx.x;
      rescaled_exp_sum[i] *= (partition_no < num_partitions)
                                 ? expf(reg_max_logit[i] - max_logit)
                                 : 0.0f;
    }
    float global_exp_sum = rescaled_exp_sum[0];
  #pragma unroll
    for (int i = 1; i < NPAR_LOOPS; i++) {
      global_exp_sum += rescaled_exp_sum[i];
    }
  #pragma unroll
    for (int i = 0; i < NPAR_LOOPS; i++) {
      const int partition_no = i * WARP_SIZE + threadIdx.x;
      shared_exp_sums[partition_no] = rescaled_exp_sum[i];
    }

  #pragma unroll
    for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
      global_exp_sum += __shfl_xor(global_exp_sum, mask);
    }
    if (threadIdx.x == 0) {
      shared_global_exp_sum = global_exp_sum;
    }
  }  // warpid == 0
  const scalar_t* tmp_out_ptr =
      tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
      head_idx * max_num_partitions * HEAD_SIZE + threadIdx.x;
  constexpr int MAX_NPAR = 64;
  scalar_t tmps[MAX_NPAR];
  const float dzero = 0.0f;
  #pragma unroll
  for (int j = 0; j < MAX_NPAR; j++) {
    tmps[j] = from_float<scalar_t>(dzero);
  }
  const int last_partition_offset = (num_partitions - 1) * HEAD_SIZE;
  const int num_partition_offset = (num_partitions)*HEAD_SIZE;
  int idx = 0;

  constexpr int JCHUNK = 16;

  #pragma unroll
  for (int j = 0; j < JCHUNK * HEAD_SIZE; j += HEAD_SIZE) {
    // lastj is last valid partition
    const int lastj_offset =
        (j < num_partition_offset) ? j : last_partition_offset;
    tmps[idx] = tmp_out_ptr[lastj_offset];
    idx++;
  }
  __syncthreads();

  if (num_partitions > JCHUNK) {
  #pragma unroll
    for (int j = JCHUNK * HEAD_SIZE; j < 2 * JCHUNK * HEAD_SIZE;
         j += HEAD_SIZE) {
      const int lastj_offset =
          (j < num_partition_offset) ? j : last_partition_offset;
      tmps[idx] = tmp_out_ptr[lastj_offset];
      idx++;
    }

    if (num_partitions > 2 * JCHUNK) {
  #pragma unroll
      for (int j = 2 * JCHUNK * HEAD_SIZE; j < MAX_NPAR * HEAD_SIZE;
           j += HEAD_SIZE) {
        const int lastj_offset =
            (j < num_partition_offset) ? j : last_partition_offset;
        tmps[idx] = tmp_out_ptr[lastj_offset];
        idx++;
      }
    }
  }  // num_partitions > JCHUNK

  // Aggregate tmp_out to out.
  float acc = 0.0f;
  #pragma unroll
  for (int j = 0; j < JCHUNK; j++) {
    acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j];
  }
  if (num_partitions > JCHUNK) {
  #pragma unroll
    for (int j = JCHUNK; j < 2 * JCHUNK; j++) {
      acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j];
    }
    if (num_partitions > 2 * JCHUNK) {
  #pragma unroll
      for (int j = 2 * JCHUNK; j < MAX_NPAR; j++) {
        acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j];
      }
    }
  }

  for (int p = 1; p < NPAR_LOOPS; p++) {
    if (num_partitions > p * MAX_NPAR) {
      idx = 0;
  #pragma unroll
      for (int j = p * MAX_NPAR * HEAD_SIZE; j < (p + 1) * MAX_NPAR * HEAD_SIZE;
           j += HEAD_SIZE) {
        // lastj is last valid partition
        const int lastj_offset =
            (j < num_partition_offset) ? j : last_partition_offset;
        tmps[idx] = tmp_out_ptr[lastj_offset];
        idx++;
      }

  #pragma unroll
      for (int j = 0; j < MAX_NPAR; j++) {
        acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j + p * MAX_NPAR];
      }
    }
  }

  const float inv_global_exp_sum =
      __fdividef(1.0f, shared_global_exp_sum + 1e-6f);
  acc *= inv_global_exp_sum;

  OUTT* out_ptr = out + static_cast<int64_t>(seq_idx) * num_heads * HEAD_SIZE +
                  static_cast<int64_t>(head_idx) * HEAD_SIZE;
  if constexpr (std::is_same<OUTT, bit8_t>::value) {
    out_ptr[threadIdx.x] =
        __hip_cvt_float_to_fp8(acc, vllm::fp8::fp8_type::__default_saturation,
                               vllm::fp8::fp8_type::__default_interpret);
  } else {
    out_ptr[threadIdx.x] = from_float<scalar_t>(acc);
  }
}

#else  // !defined(__HIP__MI300_MI250__) TODO: Add NAVI support

// clang-format off
template <typename scalar_t, typename cache_t,
          vllm::Fp8KVCacheDataType KV_DTYPE, typename OUTT, int BLOCK_SIZE,
          int HEAD_SIZE, int NUM_THREADS, bool ALIBI_ENABLED,
          int GQA_RATIO>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_QKV_mfma16_kernel(
    const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,    // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,    // [num_blocks, num_kv_heads, head_size, block_size]
    const int num_kv_heads,
    const float scale,
    const int* __restrict__ block_tables,    // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ context_lens,    // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride,
    const int kv_block_stride,
    const int kv_head_stride,
    float* __restrict__ exp_sums,             // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,           // [num_seqs, num_heads, max_num_partitions]
    scalar_t* __restrict__ out,               // [num_seqs, num_heads, max_num_partitions, head_size]
    OUTT* __restrict__ final_out,             // [num_seqs, num_heads, head_size]
    int max_ctx_blocks, const float* k_scale, const float* v_scale) {
  UNREACHABLE_CODE
}

template <typename scalar_t, typename cache_t,
          vllm::Fp8KVCacheDataType KV_DTYPE, typename OUTT, int BLOCK_SIZE,
          int HEAD_SIZE, int NUM_THREADS, bool ALIBI_ENABLED,
          int GQA_RATIO>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_QKV_mfma4_kernel(
    const scalar_t* __restrict__ q,          // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,     // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,     // [num_blocks, num_kv_heads, head_size, block_size]
    const int num_kv_heads,
    const float scale,
    const int* __restrict__ block_tables,    // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ context_lens,    // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride,
    const int kv_block_stride,
    const int kv_head_stride,
    float* __restrict__ exp_sums,            // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,          // [num_seqs, num_heads, max_num_partitions]
    scalar_t* __restrict__ out,              // [num_seqs, num_heads, max_num_partitions, head_size]
    OUTT* __restrict__ final_out,            // [num_seqs, num_heads, head_size]
    int max_ctx_blocks, const float* k_scale, const float* v_scale) {
  UNREACHABLE_CODE
}

// Grid: (num_heads, num_seqs).
template <typename scalar_t, typename OUTT, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE, int NPAR_LOOPS>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_reduce_kernel(
    OUTT* __restrict__ out,                // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,    // [num_seqs, num_heads, max_num_partitions]
    const float* __restrict__ max_logits,  // [num_seqs, num_heads, max_num_partitions]
    const scalar_t* __restrict__ tmp_out,  // [num_seqs, num_heads, max_num_partitions, head_size]
    const int* __restrict__ context_lens,  // [num_seqs]
    const int max_num_partitions) {
  UNREACHABLE_CODE
}
// clang-format on

#endif  // defined(__HIP__MI300_MI250__) TODO: Add NAVI support

#define LAUNCH_CUSTOM_ATTENTION_MFMA16(GQA_RATIO)                             \
  paged_attention_ll4mi_QKV_mfma16_kernel<T, KVT, KV_DTYPE, OUTT, BLOCK_SIZE, \
                                          HEAD_SIZE, NTHR, ALIBI_ENABLED,     \
                                          GQA_RATIO>                          \
      <<<grid, block, 0, stream>>>(                                           \
          query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, scale,     \
          block_tables_ptr, context_lens_ptr, max_num_blocks_per_seq,         \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,        \
          exp_sums_ptr, max_logits_ptr, tmp_out_ptr, out_ptr, max_ctx_blocks, \
          k_scale_ptr, v_scale_ptr);

#define LAUNCH_CUSTOM_ATTENTION_MFMA4(GQA_RATIO)                              \
  paged_attention_ll4mi_QKV_mfma4_kernel<T, KVT, KV_DTYPE, OUTT, BLOCK_SIZE,  \
                                         HEAD_SIZE, NTHR, ALIBI_ENABLED,      \
                                         GQA_RATIO>                           \
      <<<grid, block, 0, stream>>>(                                           \
          query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, scale,     \
          block_tables_ptr, context_lens_ptr, max_num_blocks_per_seq,         \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,        \
          exp_sums_ptr, max_logits_ptr, tmp_out_ptr, out_ptr, max_ctx_blocks, \
          k_scale_ptr, v_scale_ptr);

#define LAUNCH_CUSTOM_REDUCTION(NPAR_LOOPS)                          \
  paged_attention_ll4mi_reduce_kernel<T, OUTT, HEAD_SIZE, HEAD_SIZE, \
                                      PARTITION_SIZE, NPAR_LOOPS>    \
      <<<reduce_grid, reduce_block, 0, stream>>>(                    \
          out_ptr, exp_sums_ptr, max_logits_ptr, tmp_out_ptr,        \
          context_lens_ptr, max_num_partitions);

template <typename T, typename KVT, vllm::Fp8KVCacheDataType KV_DTYPE,
          int BLOCK_SIZE, int HEAD_SIZE, typename OUTT, int PARTITION_SIZE_OLD,
          bool ALIBI_ENABLED>
void paged_attention_custom_launcher(
    torch::Tensor& out, torch::Tensor& exp_sums, torch::Tensor& max_logits,
    torch::Tensor& tmp_out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, const int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& context_lens,
    int max_context_len, const std::optional<torch::Tensor>& alibi_slopes,
    torch::Tensor& k_scale, torch::Tensor& v_scale) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  float* exp_sums_ptr = reinterpret_cast<float*>(exp_sums.data_ptr());
  float* max_logits_ptr = reinterpret_cast<float*>(max_logits.data_ptr());
  T* tmp_out_ptr = reinterpret_cast<T*>(tmp_out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  KVT* key_cache_ptr = reinterpret_cast<KVT*>(key_cache.data_ptr());
  KVT* value_cache_ptr = reinterpret_cast<KVT*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* context_lens_ptr = context_lens.data_ptr<int>();
  const float* k_scale_ptr = reinterpret_cast<const float*>(k_scale.data_ptr());
  const float* v_scale_ptr = reinterpret_cast<const float*>(v_scale.data_ptr());
  OUTT* out_ptr = reinterpret_cast<OUTT*>(out.data_ptr());

  const int max_ctx_blocks = DIVIDE_ROUND_UP(max_context_len, BLOCK_SIZE);

  // partition size is fixed at 256 since both mfma4 and mfma16 kernels support
  // it mfma4 kernel also supports partition size 512
  constexpr int PARTITION_SIZE = 256;
  const int max_num_partitions =
      DIVIDE_ROUND_UP(max_context_len, PARTITION_SIZE);
  const int gqa_ratio = num_heads / num_kv_heads;
  assert(num_heads % num_kv_heads == 0);
  assert(head_size == HEAD_SIZE);

  constexpr int NTHR = 256;
  dim3 grid(num_seqs, max_num_partitions, num_kv_heads);
  dim3 block(NTHR);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // mfma4 kernel is faster than mfma16 for gqa_ratio <= 4
  switch (gqa_ratio) {
    case 1:
      LAUNCH_CUSTOM_ATTENTION_MFMA4(1);
      break;
    case 2:
      LAUNCH_CUSTOM_ATTENTION_MFMA4(2);
      break;
    case 3:
      LAUNCH_CUSTOM_ATTENTION_MFMA4(3);
      break;
    case 4:
      LAUNCH_CUSTOM_ATTENTION_MFMA4(4);
      break;
    case 5:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(5);
      break;
    case 6:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(6);
      break;
    case 7:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(7);
      break;
    case 8:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(8);
      break;
    case 9:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(9);
      break;
    case 10:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(10);
      break;
    case 11:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(11);
      break;
    case 12:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(12);
      break;
    case 13:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(13);
      break;
    case 14:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(14);
      break;
    case 15:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(15);
      break;
    case 16:
      LAUNCH_CUSTOM_ATTENTION_MFMA16(16);
      break;
    default:
      TORCH_CHECK(false, "Unsupported gqa ratio: ", gqa_ratio);
      break;
  }

  dim3 reduce_grid(num_heads, num_seqs);
  dim3 reduce_block(head_size);
  const int npar_loops = DIVIDE_ROUND_UP(max_num_partitions, WARP_SIZE);
  // reduction kernel supports upto 8 NPAR_loops * 64 (warp_size) * 256
  // (partition size) = 128K context length
  switch (npar_loops) {
    case 1:
      LAUNCH_CUSTOM_REDUCTION(1);
      break;
    case 2:
      LAUNCH_CUSTOM_REDUCTION(2);
      break;
    case 3:
      LAUNCH_CUSTOM_REDUCTION(3);
      break;
    case 4:
      LAUNCH_CUSTOM_REDUCTION(4);
      break;
    case 5:
      LAUNCH_CUSTOM_REDUCTION(5);
      break;
    case 6:
      LAUNCH_CUSTOM_REDUCTION(6);
      break;
    case 7:
      LAUNCH_CUSTOM_REDUCTION(7);
      break;
    case 8:
      LAUNCH_CUSTOM_REDUCTION(8);
      break;
    default:
      TORCH_CHECK(false, "Unsupported npar_loops: ", npar_loops);
      break;
  }
}

#define CALL_CUSTOM_LAUNCHER(T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE, PSIZE,  \
                             ALIBI_ENABLED)                                 \
  paged_attention_custom_launcher<T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE, T, \
                                  PSIZE, ALIBI_ENABLED>(                    \
      out, exp_sums, max_logits, tmp_out, query, key_cache, value_cache,    \
      num_kv_heads, scale, block_tables, context_lens, max_context_len,     \
      alibi_slopes, k_scale, v_scale);

#define CALL_CUSTOM_LAUNCHER_ALIBI(T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE,      \
                                   PSIZE)                                      \
  if (alibi_slopes) {                                                          \
    CALL_CUSTOM_LAUNCHER(T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE, PSIZE, true);  \
  } else {                                                                     \
    CALL_CUSTOM_LAUNCHER(T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE, PSIZE, false); \
  }

#define CALL_CUSTOM_LAUNCHER_BLK(T, KVT, KV_DTYPE, HEAD_SIZE)           \
  switch (block_size) {                                                 \
    case 16:                                                            \
      CALL_CUSTOM_LAUNCHER_ALIBI(T, KVT, KV_DTYPE, 16, HEAD_SIZE, 256); \
      break;                                                            \
    case 32:                                                            \
      CALL_CUSTOM_LAUNCHER_ALIBI(T, KVT, KV_DTYPE, 32, HEAD_SIZE, 256); \
      break;                                                            \
    default:                                                            \
      TORCH_CHECK(false, "Unsupported block size: ", block_size);       \
      break;                                                            \
  }

#define CALL_CUSTOM_LAUNCHER_BLK_HEAD(T, KVT, KV_DTYPE)         \
  switch (head_size) {                                          \
    case 64:                                                    \
      CALL_CUSTOM_LAUNCHER_BLK(T, KVT, KV_DTYPE, 64);           \
      break;                                                    \
    case 128:                                                   \
      CALL_CUSTOM_LAUNCHER_BLK(T, KVT, KV_DTYPE, 128);          \
      break;                                                    \
    default:                                                    \
      TORCH_CHECK(false, "Unsupported head size: ", head_size); \
      break;                                                    \
  }

// clang-format off
void paged_attention(
    torch::Tensor& out,         // [num_seqs, num_heads, head_size]
    torch::Tensor& exp_sums,    // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor& max_logits,  // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor& tmp_out,     // [num_seqs, num_heads, max_num_partitions, head_size]
    torch::Tensor& query,       // [num_seqs, num_heads, head_size]
    torch::Tensor& key_cache,   // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor& value_cache, // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads, 
    double scale,
    torch::Tensor& block_tables, // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& context_lens, // [num_seqs]
    int64_t block_size, int64_t max_context_len,
    const std::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, torch::Tensor& k_scale,
    torch::Tensor& v_scale) {
  // clang-format on
  const int head_size = query.size(2);
  if (kv_cache_dtype == "auto") {
    if (query.dtype() == at::ScalarType::Half) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(_Float16, _Float16,
                                    vllm::Fp8KVCacheDataType::kAuto);
    } else if (query.dtype() == at::ScalarType::BFloat16) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(__hip_bfloat16, __hip_bfloat16,
                                    vllm::Fp8KVCacheDataType::kAuto);
    } else {
      TORCH_CHECK(false, "Unsupported data type: ", query.dtype());
    }
  } else if (kv_cache_dtype == "fp8" || kv_cache_dtype == "fp8_e4m3") {
    if (query.dtype() == at::ScalarType::Half) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(_Float16, uint8_t,
                                    vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (query.dtype() == at::ScalarType::BFloat16) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(__hip_bfloat16, uint8_t,
                                    vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else {
      TORCH_CHECK(false, "Unsupported data type: ", query.dtype());
    }
  } else {
    TORCH_CHECK(false, "Unsupported KV cache dtype: ", kv_cache_dtype);
  }
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
