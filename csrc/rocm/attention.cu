#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, The vLLM team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_bf16.h>
#include "cuda_compat.h"

#include <algorithm>
#include "../attention/dtype_fp8.cuh"
#include "../quantization/fp8/amd/quant_utils.cuh"

#if defined(__HIPCC__) && (defined(__gfx90a__) || defined(__gfx940__) || \
                           defined(__gfx941__) || defined(__gfx942__))
  #define __HIP__MI300_MI250__
#endif

#if defined(NDEBUG)
  #undef NDEBUG
  #include <assert.h>
  #define UNREACHABLE_CODE assert(false);
  #define NDEBUG
#else
  #define UNREACHABLE_CODE assert(false);
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#if defined(__HIP__MI300_MI250__)  // TODO: Add NAVI support

  #define GCN_MFMA_INSTR1 __builtin_amdgcn_mfma_f32_16x16x4f32
  #define GCN_MFMA_INSTR __builtin_amdgcn_mfma_f32_4x4x4f16

using floatx4 = __attribute__((__vector_size__(4 * sizeof(float)))) float;
using float16x4 =
    __attribute__((__vector_size__(4 * sizeof(_Float16)))) _Float16;
typedef float16x4 _Half4;
typedef struct _Half8 {
  _Half4 xy[2];
} _Half8;

using bit16_t = uint16_t;
using bit16x4 = __attribute__((__vector_size__(4 * sizeof(uint16_t)))) uint16_t;
typedef bit16x4 _B16x4;
typedef struct _B16x8 {
  _B16x4 xy[2];
} _B16x8;

using _B8x8 = uint2;

////// Non temporal load stores ///////

template <typename T>
__device__ __forceinline__ T load(T* addr) {
  return addr[0];
}

template <typename T>
__device__ __forceinline__ void store(T value, T* addr) {
  addr[0] = value;
}

template <typename T, int absz, int cbid, int blgp>
__device__ __forceinline__ floatx4 gcn_mfma_instr(const _B16x4& inpA,
                                                  const _B16x4& inpB,
                                                  const floatx4& inpC) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return __builtin_amdgcn_mfma_f32_4x4x4f16(inpA, inpB, inpC, absz, cbid,
                                              blgp);
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __builtin_amdgcn_mfma_f32_4x4x4bf16_1k(inpA, inpB, inpC, absz, cbid,
                                                  blgp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ float to_float(const T& inp) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return (float)inp;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __bfloat162float(inp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ T from_float(const float& inp) {
  if constexpr (std::is_same<T, _Float16>::value) {
    return (_Float16)inp;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    return __float2bfloat16(inp);
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ _B16x4 from_floatx4(const floatx4& inp) {
  union tmpcvt {
    uint16_t u;
    _Float16 f;
    __hip_bfloat16 b;
  } t16;
  _B16x4 ret;
  if constexpr (std::is_same<T, _Float16>::value) {
  #pragma unroll
    for (int i = 0; i < 4; i++) {
      t16.f = (_Float16)inp[i];
      ret[i] = t16.u;
    }
    return ret;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
  #pragma unroll
    for (int i = 0; i < 4; i++) {
      t16.b = __float2bfloat16(inp[i]);
      ret[i] = t16.u;
    }
    return ret;
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T>
__device__ __forceinline__ _B16x4 addx4(const _B16x4& inp1,
                                        const _B16x4& inp2) {
  union tmpcvt {
    uint16_t u;
    _Float16 f;
    __hip_bfloat16 b;
  } t1, t2, res;
  _B16x4 ret;
  if constexpr (std::is_same<T, _Float16>::value) {
  #pragma unroll
    for (int i = 0; i < 4; i++) {
      t1.u = inp1[i];
      t2.u = inp2[i];
      res.f = t1.f + t2.f;
      ret[i] = res.u;
    }
    return ret;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
  #pragma unroll
    for (int i = 0; i < 4; i++) {
      t1.u = inp1[i];
      t2.u = inp2[i];
      res.b = t1.b + t2.b;
      ret[i] = res.u;
    }
    return ret;
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

template <typename T, vllm::Fp8KVCacheDataType KV_DTYPE>
__device__ __forceinline__ _B16x8 scaled_convert_b8x8(const _B8x8 input,
                                                      const float scale) {
  union alignas(16) {
    uint4 u4;
    _B16x8 u16x8;
    vllm::bf16_8_t b16x8;
  } tmp;
  if constexpr (std::is_same<T, _Float16>::value) {
    tmp.u4 = vllm::fp8::scaled_convert<uint4, _B8x8, KV_DTYPE>(input, scale);
    return tmp.u16x8;
  } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
    tmp.b16x8 = vllm::fp8::scaled_convert<vllm::bf16_8_t, _B8x8, KV_DTYPE>(
        input, scale);
    return tmp.u16x8;
  } else {
    static_assert(false, "unsupported 16b dtype");
  }
}

///////////////////////////////////////

// grid (num_seqs, num_partitions,num_heads/gqa_ratio)
// block (partition size)
template <typename scalar_t, typename cache_t,
          vllm::Fp8KVCacheDataType KV_DTYPE, int BLOCK_SIZE, int HEAD_SIZE,
          int NUM_THREADS,
          int GQA_RATIO>
__global__ __launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_QKV_kernel(
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads, const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ context_lens,  // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                     // max_num_partitions]
    scalar_t* __restrict__ out,  // [num_seqs, num_heads, max_num_partitions,
                                 // head_size]
    scalar_t* __restrict__ final_out,  // [num_seqs, num_heads, head_size]
    int max_ctx_blocks, float k_scale, float v_scale) {
  constexpr int NWARPS = NUM_THREADS / WARP_SIZE;
  const int warpid = threadIdx.x / WARP_SIZE;
  const int laneid = threadIdx.x % WARP_SIZE;
  const int lane4id = laneid % 4;

  const int seq_idx = blockIdx.x;
  const int partition_idx = blockIdx.y;
  const int partition_size = blockDim.x;
  const int max_num_partitions = gridDim.y;

  const int context_len = context_lens[seq_idx];
  const int partition_start_token_idx = partition_idx * partition_size;
  // exit if partition is out of context for seq
  if (partition_start_token_idx >= context_len) {
    return;
  }
  constexpr int QHLOOP =
      DIVIDE_ROUND_UP(GQA_RATIO, 4);  // each 4 lanes fetch 4 different qheads,
                                      // total qheads =8, so qhloop is 2
  constexpr int GQA_RATIO4 = 4 * QHLOOP;
  __shared__ float shared_qk_max[NWARPS][GQA_RATIO4 + 1];
  __shared__ float shared_exp_sum[NWARPS][GQA_RATIO4 + 1];
  _B16x8 Qlocal[QHLOOP];
  constexpr int x = 16 / sizeof(scalar_t);
  constexpr int KHELOOP = HEAD_SIZE / x;
  _B16x8 Klocal[KHELOOP];
  _B8x8 Klocalb8[KHELOOP];
  constexpr int VHELOOP =
      HEAD_SIZE /
      WARP_SIZE;  // v head_size dimension is distributed across lanes
  constexpr int VTLOOP = 8;  // 16 separate 4xtokens across warp -> 16/2
                             // 8xtokens
  _B16x8 Vlocal[VHELOOP][VTLOOP];
  _B8x8 Vlocalb8[VHELOOP][VTLOOP];
  floatx4 dout[QHLOOP];
  float qk_max[QHLOOP];
  #pragma unroll
  for (int h = 0; h < QHLOOP; h++) {
    dout[h] = {0};
    qk_max[h] = -FLT_MAX;
  }

  const int wg_start_head_idx = blockIdx.z * GQA_RATIO;
  const int wg_start_kv_head_idx = blockIdx.z;

  const int warp_start_token_idx =
      partition_start_token_idx + warpid * WARP_SIZE;

  if (warp_start_token_idx >= context_len) {  // warp out of context
  #pragma unroll
    for (int h = 0; h < GQA_RATIO4; h++) {
      shared_qk_max[warpid][h] = -FLT_MAX;
      shared_exp_sum[warpid][h] = 0.0f;
    }
  } else {  // warp within context

    const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
    const int last_ctx_block = num_context_blocks - 1;

    const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;

    const int local_token_idx = threadIdx.x;
    const int global_token_idx = partition_start_token_idx + local_token_idx;

    const int block_idx = (global_token_idx < context_len)
                              ? global_token_idx / BLOCK_SIZE
                              : last_ctx_block;
    // fetch block number for q and k
    // int32 physical_block_number leads to overflow when multiplied with
    // kv_block_stride
    const int64_t physical_block_number =
        static_cast<int64_t>(block_table[block_idx]);

    // fetch vphysical block numbers up front
    constexpr int VBLOCKS = 8 * VTLOOP / BLOCK_SIZE;
    int vphysical_blocks[VBLOCKS];

    const int warp_start_block_idx = warp_start_token_idx / BLOCK_SIZE;
  #pragma unroll
    for (int b = 0; b < VBLOCKS; b++) {
      const int vblock_idx = warp_start_block_idx + b;
      const int vblock_idx_ctx =
          (vblock_idx <= last_ctx_block) ? vblock_idx : last_ctx_block;
      vphysical_blocks[b] = block_table[vblock_idx_ctx];
    }

    // each 4 lanes fetch 8 helems, so warp fetches 8*16 = 128 helems
    const scalar_t* q_ptr =
        q + seq_idx * q_stride + wg_start_head_idx * HEAD_SIZE;
    const _B16x8* q_ptrh8 = reinterpret_cast<const _B16x8*>(q_ptr);
    const int qhead_elemh8 = laneid / 4;
  #pragma unroll
    for (int h = 0; h < QHLOOP - 1; h++) {
      const int qhead_idx = h * 4 + lane4id;
      Qlocal[h] = q_ptrh8[qhead_idx * HEAD_SIZE / 8 + qhead_elemh8];
    }
    const int final_qhead_idx = 4 * (QHLOOP - 1) + lane4id;
    if (final_qhead_idx < GQA_RATIO) {
      Qlocal[QHLOOP - 1] =
          q_ptrh8[final_qhead_idx * HEAD_SIZE / 8 + qhead_elemh8];
    } else {
      Qlocal[QHLOOP - 1].xy[0] = {0};
      Qlocal[QHLOOP - 1].xy[1] = {0};
    }

    const cache_t* k_ptr = k_cache + physical_block_number * kv_block_stride +
                           wg_start_kv_head_idx * kv_head_stride;

    const int physical_block_offset =
        local_token_idx % BLOCK_SIZE;  // since x=half8, physical_block_offset
                                       // is already cast as _H8
    if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
      const _B16x8* k_ptrh8 = reinterpret_cast<const _B16x8*>(k_ptr);
  #pragma unroll
      for (int d = 0; d < KHELOOP; d++) {
        Klocal[d] = k_ptrh8[d * BLOCK_SIZE + physical_block_offset];
      }
    } else {
      constexpr int X = 16 / sizeof(cache_t);
      const cache_t* k_ptr2 = k_ptr + physical_block_offset * X;
  #pragma unroll
      for (int d = 0; d < KHELOOP; d++) {
        const int head_elem = d * 8;
        const int offset1 = head_elem / X;
        const int offset2 = head_elem % X;
        const cache_t* k_ptr3 = k_ptr2 + offset1 * BLOCK_SIZE * X + offset2;
        Klocalb8[d] = *reinterpret_cast<const _B8x8*>(k_ptr3);
      }
    }

    float alibi_slope[QHLOOP];
    if (alibi_slopes != nullptr) {
  #pragma unroll
      for (int h = 0; h < QHLOOP; h++) {
        const int qhead_idx = h * 4 + lane4id;
        alibi_slope[h] = (qhead_idx < GQA_RATIO)
                             ? alibi_slopes[wg_start_head_idx + qhead_idx]
                             : 0.f;
      }
    }

    const cache_t* v_ptr = v_cache + wg_start_kv_head_idx * kv_head_stride;
    if constexpr (KV_DTYPE == vllm::Fp8KVCacheDataType::kAuto) {
      const _B16x8* v_ptrh8 = reinterpret_cast<const _B16x8*>(v_ptr);
      // iterate over each v block
  #pragma unroll
      for (int b = 0; b < VBLOCKS; b++) {
        // int32 physical_block_number leads to overflow when multiplied with
        // kv_block_stride
        const int64_t vphysical_block_number =
            static_cast<int64_t>(vphysical_blocks[b]);
        const _B16x8* v_ptrh8b =
            v_ptrh8 + (vphysical_block_number * kv_block_stride) / 8;
        // iterate over each head elem (within head_size)
  #pragma unroll
        for (int h = 0; h < VHELOOP; h++) {
          const int head_size_elem = h * WARP_SIZE + laneid;
          const _B16x8* v_ptrh8be = v_ptrh8b + head_size_elem * BLOCK_SIZE / 8;
          // iterate over all velems within block
  #pragma unroll
          for (int d = 0; d < BLOCK_SIZE / 8; d++) {
            Vlocal[h][b * BLOCK_SIZE / 8 + d] = v_ptrh8be[d];
          }
        }
      }
    } else {
      const _B8x8* v_ptrh8 = reinterpret_cast<const _B8x8*>(v_ptr);
      // iterate over each v block
  #pragma unroll
      for (int b = 0; b < VBLOCKS; b++) {
        // int32 physical_block_number leads to overflow when multiplied with
        // kv_block_stride
        const int64_t vphysical_block_number =
            static_cast<int64_t>(vphysical_blocks[b]);
        const _B8x8* v_ptrh8b =
            v_ptrh8 + (vphysical_block_number * kv_block_stride) / 8;
        // iterate over each head elem (within head_size)
  #pragma unroll
        for (int h = 0; h < VHELOOP; h++) {
          const int head_size_elem = h * WARP_SIZE + laneid;
          const _B8x8* v_ptrh8be = v_ptrh8b + head_size_elem * BLOCK_SIZE / 8;
          // iterate over all velems within block
  #pragma unroll
          for (int d = 0; d < BLOCK_SIZE / 8; d++) {
            // Vlocalb8[h][b * BLOCK_SIZE / 8 + d] = v_ptrh8be[d];
            const _B8x8 Vlocalb8 = v_ptrh8be[d];
            Vlocal[h][b * BLOCK_SIZE / 8 + d] =
                scaled_convert_b8x8<scalar_t, KV_DTYPE>(Vlocalb8, v_scale);
          }
        }
      }
    }

    if constexpr (KV_DTYPE != vllm::Fp8KVCacheDataType::kAuto) {
  #pragma unroll
      for (int d = 0; d < KHELOOP; d++) {
        Klocal[d] =
            scaled_convert_b8x8<scalar_t, KV_DTYPE>(Klocalb8[d], k_scale);
      }
    }

  #pragma unroll
    for (int h = 0; h < QHLOOP; h++) {
      dout[h] = gcn_mfma_instr<scalar_t, 4, 0, 0>(Qlocal[h].xy[0],
                                                  Klocal[0].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 0, 0>(Qlocal[h].xy[1],
                                                  Klocal[0].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 1, 0>(Qlocal[h].xy[0],
                                                  Klocal[1].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 1, 0>(Qlocal[h].xy[1],
                                                  Klocal[1].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 2, 0>(Qlocal[h].xy[0],
                                                  Klocal[2].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 2, 0>(Qlocal[h].xy[1],
                                                  Klocal[2].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 3, 0>(Qlocal[h].xy[0],
                                                  Klocal[3].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 3, 0>(Qlocal[h].xy[1],
                                                  Klocal[3].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 4, 0>(Qlocal[h].xy[0],
                                                  Klocal[4].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 4, 0>(Qlocal[h].xy[1],
                                                  Klocal[4].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 5, 0>(Qlocal[h].xy[0],
                                                  Klocal[5].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 5, 0>(Qlocal[h].xy[1],
                                                  Klocal[5].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 6, 0>(Qlocal[h].xy[0],
                                                  Klocal[6].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 6, 0>(Qlocal[h].xy[1],
                                                  Klocal[6].xy[1], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 7, 0>(Qlocal[h].xy[0],
                                                  Klocal[7].xy[0], dout[h]);
      dout[h] = gcn_mfma_instr<scalar_t, 4, 7, 0>(Qlocal[h].xy[1],
                                                  Klocal[7].xy[1], dout[h]);
      if constexpr (KHELOOP > 8) {
        dout[h] = gcn_mfma_instr<scalar_t, 4, 8, 0>(Qlocal[h].xy[0],
                                                    Klocal[8].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 8, 0>(Qlocal[h].xy[1],
                                                    Klocal[8].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 9, 0>(Qlocal[h].xy[0],
                                                    Klocal[9].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 9, 0>(Qlocal[h].xy[1],
                                                    Klocal[9].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 10, 0>(Qlocal[h].xy[0],
                                                     Klocal[10].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 10, 0>(Qlocal[h].xy[1],
                                                     Klocal[10].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 11, 0>(Qlocal[h].xy[0],
                                                     Klocal[11].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 11, 0>(Qlocal[h].xy[1],
                                                     Klocal[11].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 12, 0>(Qlocal[h].xy[0],
                                                     Klocal[12].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 12, 0>(Qlocal[h].xy[1],
                                                     Klocal[12].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 13, 0>(Qlocal[h].xy[0],
                                                     Klocal[13].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 13, 0>(Qlocal[h].xy[1],
                                                     Klocal[13].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 14, 0>(Qlocal[h].xy[0],
                                                     Klocal[14].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 14, 0>(Qlocal[h].xy[1],
                                                     Klocal[14].xy[1], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 15, 0>(Qlocal[h].xy[0],
                                                     Klocal[15].xy[0], dout[h]);
        dout[h] = gcn_mfma_instr<scalar_t, 4, 15, 0>(Qlocal[h].xy[1],
                                                     Klocal[15].xy[1], dout[h]);
      }  // KHELOOP>8
      dout[h] *= scale;
    }
  // transpose dout so that 4 token ids are in each lane, and 4 heads are across
  // 4 lanes
  #pragma unroll
    for (int h = 0; h < QHLOOP; h++) {
      floatx4 tmp = {0};
  #pragma unroll
      for (int i = 0; i < 4; i++) {
        const float B = (lane4id == i) ? 1.0f : 0.0f;
        // const float A = (global_token_idx < context_len) ? dout[h][i] : 0.0f;
        tmp = __builtin_amdgcn_mfma_f32_4x4x1f32(dout[h][i], B, tmp, 0, 0, 0);
        // tmp = __builtin_amdgcn_mfma_f32_4x4x1f32(A, B, tmp, 0, 0, 0);
      }
      dout[h] = tmp;
    }

    const int lane4_token_idx = 4 * (global_token_idx >> 2);
    const int alibi_offset = lane4_token_idx - context_len + 1;
    if (alibi_slopes != nullptr) {
  #pragma unroll
      for (int h = 0; h < QHLOOP; h++) {
  #pragma unroll
        for (int i = 0; i < 4; i++) {
          dout[h][i] += alibi_slope[h] * (alibi_offset + i);
        }
      }
    }

  #pragma unroll
    for (int h = 0; h < QHLOOP; h++) {
      qk_max[h] = -FLT_MAX;
  #pragma unroll
      for (int i = 0; i < 4; i++) {
        qk_max[h] = (lane4_token_idx + i < context_len)
                        ? fmaxf(qk_max[h], dout[h][i])
                        : qk_max[h];
      }
  #pragma unroll
      for (int mask = WARP_SIZE / 2; mask >= 4; mask /= 2) {
        qk_max[h] = fmaxf(qk_max[h], __shfl_xor(qk_max[h], mask));
      }
    }

    float exp_sum[QHLOOP];
  #pragma unroll
    for (int h = 0; h < QHLOOP; h++) {
      exp_sum[h] = 0.0f;
  #pragma unroll
      for (int i = 0; i < 4; i++) {
        dout[h][i] = (lane4_token_idx + i < context_len)
                         ? __expf(dout[h][i] - qk_max[h])
                         : 0.0f;
        exp_sum[h] += dout[h][i];
      }
  #pragma unroll
      for (int mask = WARP_SIZE / 2; mask >= 4; mask /= 2) {
        exp_sum[h] += __shfl_xor(exp_sum[h], mask);
      }
    }

  #pragma unroll
    for (int h = 0; h < QHLOOP; h++) {
      const int head_idx = 4 * h + lane4id;
      shared_qk_max[warpid][head_idx] = qk_max[h];
      shared_exp_sum[warpid][head_idx] = exp_sum[h];
    }
  }  // warp within context

  __syncthreads();

  const int num_heads = gridDim.z * GQA_RATIO;
  float* max_logits_ptr =
      max_logits + seq_idx * num_heads * max_num_partitions + partition_idx;
  float* exp_sums_ptr =
      exp_sums + seq_idx * num_heads * max_num_partitions + partition_idx;
  #pragma unroll
  for (int h = 0; h < QHLOOP; h++) {
    float global_qk_max = -FLT_MAX;
    float warp_qk_max[NWARPS];
    const int head_idx = 4 * h + lane4id;
  #pragma unroll
    for (int w = 0; w < NWARPS; w++) {
      warp_qk_max[w] = shared_qk_max[w][head_idx];
      global_qk_max = fmaxf(global_qk_max, warp_qk_max[w]);
    }
    float global_exp_sum = 0.0f;
  #pragma unroll
    for (int w = 0; w < NWARPS; w++) {
      global_exp_sum +=
          shared_exp_sum[w][head_idx] * __expf(warp_qk_max[w] - global_qk_max);
    }
    if (head_idx < GQA_RATIO) {
      max_logits_ptr[(wg_start_head_idx + head_idx) * max_num_partitions] =
          global_qk_max;
      exp_sums_ptr[(wg_start_head_idx + head_idx) * max_num_partitions] =
          global_exp_sum;
    }
    const float global_inv_sum_scale = __fdividef(1.f, global_exp_sum + 1e-6f) *
                                       __expf(qk_max[h] - global_qk_max);
    dout[h] *= global_inv_sum_scale;
  }
  // logits[h] -> every 4 lanes hold 4 heads, each lane holds 4 tokens, there
  // are 4x16 tokens across warp
  _B16x4 logits[QHLOOP];
  #pragma unroll
  for (int h = 0; h < QHLOOP; h++) {
    logits[h] = from_floatx4<scalar_t>(dout[h]);
  }

  __shared__ _B16x4 vout_shared[QHLOOP][VHELOOP][WARP_SIZE][NWARPS + 1];

  if (warp_start_token_idx >= context_len) {  // warp out of context
  #pragma unroll
    for (int qh = 0; qh < QHLOOP; qh++) {
  #pragma unroll
      for (int vh = 0; vh < VHELOOP; vh++) {
        vout_shared[qh][vh][laneid][warpid] = {0};
      }
    }
  } else {  // warp in context
  // iterate across heads
  #pragma unroll
    for (int qh = 0; qh < QHLOOP; qh++) {
  // iterate over each v head elem (within head_size)
  #pragma unroll
      for (int vh = 0; vh < VHELOOP; vh++) {
        floatx4 acc = {0};
        // iterate over tokens
        acc = gcn_mfma_instr<scalar_t, 4, 0, 0>(logits[qh], Vlocal[vh][0].xy[0],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 1, 0>(logits[qh], Vlocal[vh][0].xy[1],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 2, 0>(logits[qh], Vlocal[vh][1].xy[0],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 3, 0>(logits[qh], Vlocal[vh][1].xy[1],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 4, 0>(logits[qh], Vlocal[vh][2].xy[0],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 5, 0>(logits[qh], Vlocal[vh][2].xy[1],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 6, 0>(logits[qh], Vlocal[vh][3].xy[0],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 7, 0>(logits[qh], Vlocal[vh][3].xy[1],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 8, 0>(logits[qh], Vlocal[vh][4].xy[0],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 9, 0>(logits[qh], Vlocal[vh][4].xy[1],
                                                acc);
        acc = gcn_mfma_instr<scalar_t, 4, 10, 0>(logits[qh],
                                                 Vlocal[vh][5].xy[0], acc);
        acc = gcn_mfma_instr<scalar_t, 4, 11, 0>(logits[qh],
                                                 Vlocal[vh][5].xy[1], acc);
        acc = gcn_mfma_instr<scalar_t, 4, 12, 0>(logits[qh],
                                                 Vlocal[vh][6].xy[0], acc);
        acc = gcn_mfma_instr<scalar_t, 4, 13, 0>(logits[qh],
                                                 Vlocal[vh][6].xy[1], acc);
        acc = gcn_mfma_instr<scalar_t, 4, 14, 0>(logits[qh],
                                                 Vlocal[vh][7].xy[0], acc);
        acc = gcn_mfma_instr<scalar_t, 4, 15, 0>(logits[qh],
                                                 Vlocal[vh][7].xy[1], acc);
        vout_shared[qh][vh][laneid][warpid] = from_floatx4<scalar_t>(acc);
      }
    }
  }  // warp in context

  __syncthreads();

  if (warpid == 0) {
    _B16x4 vout[QHLOOP][VHELOOP];
    // iterate across heads
    scalar_t* out_ptr;
    int out_num_partitions;
    if (context_len > partition_size) {
      out_num_partitions = max_num_partitions;
      out_ptr = out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
                partition_idx * HEAD_SIZE;
    } else {
      out_num_partitions = 1;
      out_ptr = final_out + seq_idx * num_heads * HEAD_SIZE;
    }
  #pragma unroll
    for (int qh = 0; qh < QHLOOP; qh++) {
  // iterate over each v head elem (within head_size)
  #pragma unroll
      for (int vh = 0; vh < VHELOOP; vh++) {
        vout[qh][vh] = {0};
  #pragma unroll
        for (int w = 0; w < NWARPS; w++) {
          vout[qh][vh] =
              addx4<scalar_t>(vout[qh][vh], vout_shared[qh][vh][laneid][w]);
        }
        const int head_size_elem = vh * WARP_SIZE + laneid;
        bit16_t* out_ptr_b16 = reinterpret_cast<bit16_t*>(out_ptr);
  #pragma unroll
        for (int i = 0; i < 4; i++) {
          const int head_idx = 4 * qh + i;
          if (head_idx < GQA_RATIO) {
            out_ptr_b16[(wg_start_head_idx + head_idx) * out_num_partitions *
                            HEAD_SIZE +
                        head_size_elem] = vout[qh][vh][i];
          }
        }
      }
    }
  }
}

// Grid: (num_heads, num_seqs).
template <typename scalar_t, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_reduce_kernel(
    scalar_t* __restrict__ out,            // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,    // [num_seqs, num_heads,
                                           // max_num_partitions]
    const float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                           // max_num_partitions]
    const scalar_t* __restrict__ tmp_out,  // [num_seqs, num_heads,
                                           // max_num_partitions, head_size]
    const int* __restrict__ context_lens,  // [num_seqs]
    const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int context_len = context_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(context_len, PARTITION_SIZE);
  if (num_partitions == 1) {
    // if num_partitions==1, main kernel will write to out directly, no work in
    // reduction kernel
    return;
  }

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warpid = threadIdx.x / WARP_SIZE;
  const int laneid = threadIdx.x % WARP_SIZE;

  __shared__ float shared_global_exp_sum;
  __shared__ float shared_exp_sums[2 * WARP_SIZE];

  if (warpid == 0) {
    const float* max_logits_ptr = max_logits +
                                  seq_idx * num_heads * max_num_partitions +
                                  head_idx * max_num_partitions;

    // valid partition is the last valid partition in case threadid > num
    // partitions
    const int valid_partition =
        (threadIdx.x < num_partitions) ? threadIdx.x : num_partitions - 1;
    const int valid_partition2 = (WARP_SIZE + threadIdx.x < num_partitions)
                                     ? WARP_SIZE + threadIdx.x
                                     : num_partitions - 1;
    float reg_max_logit = max_logits_ptr[valid_partition];
    float reg_max_logit2 = max_logits_ptr[valid_partition2];
    float max_logit = fmaxf(reg_max_logit, reg_max_logit2);

  #pragma unroll
    for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
      max_logit = fmaxf(max_logit, __shfl_xor(max_logit, mask));
    }

    const float* exp_sums_ptr = exp_sums +
                                seq_idx * num_heads * max_num_partitions +
                                head_idx * max_num_partitions;

    float global_exp_sum = 0.0f;
    float rescaled_exp_sum = exp_sums_ptr[valid_partition];
    float rescaled_exp_sum2 = exp_sums_ptr[valid_partition2];
    rescaled_exp_sum *=
        (threadIdx.x < num_partitions) ? expf(reg_max_logit - max_logit) : 0.0f;
    rescaled_exp_sum2 *= (threadIdx.x + WARP_SIZE < num_partitions)
                             ? expf(reg_max_logit2 - max_logit)
                             : 0.0f;
    global_exp_sum += rescaled_exp_sum + rescaled_exp_sum2;
    shared_exp_sums[threadIdx.x] = rescaled_exp_sum;
    shared_exp_sums[threadIdx.x + WARP_SIZE] = rescaled_exp_sum2;

  #pragma unroll
    for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
      global_exp_sum += __shfl_xor(global_exp_sum, mask);
    }
    if (threadIdx.x == 0) {
      shared_global_exp_sum = global_exp_sum;
    }
  }  // warpid == 0
  const scalar_t* tmp_out_ptr =
      tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
      head_idx * max_num_partitions * HEAD_SIZE + threadIdx.x;
  constexpr int MAX_NPAR = 64;
  scalar_t tmps[MAX_NPAR];
  const float dzero = 0.0f;
  #pragma unroll
  for (int j = 0; j < MAX_NPAR; j++) {
    tmps[j] = from_float<scalar_t>(dzero);
  }
  const int last_partition_offset = (num_partitions - 1) * HEAD_SIZE;
  const int num_partition_offset = (num_partitions)*HEAD_SIZE;
  int idx = 0;

  constexpr int JCHUNK = 16;

  #pragma unroll
  for (int j = 0; j < JCHUNK * HEAD_SIZE; j += HEAD_SIZE) {
    // lastj is last valid partition
    const int lastj_offset =
        (j < num_partition_offset) ? j : last_partition_offset;
    tmps[idx] = tmp_out_ptr[lastj_offset];
    idx++;
  }
  __syncthreads();

  if (num_partitions > JCHUNK) {
  #pragma unroll
    for (int j = JCHUNK * HEAD_SIZE; j < 2 * JCHUNK * HEAD_SIZE;
         j += HEAD_SIZE) {
      const int lastj_offset =
          (j < num_partition_offset) ? j : last_partition_offset;
      tmps[idx] = tmp_out_ptr[lastj_offset];
      idx++;
    }

    if (num_partitions > 2 * JCHUNK) {
  #pragma unroll
      for (int j = 2 * JCHUNK * HEAD_SIZE; j < MAX_NPAR * HEAD_SIZE;
           j += HEAD_SIZE) {
        const int lastj_offset =
            (j < num_partition_offset) ? j : last_partition_offset;
        tmps[idx] = tmp_out_ptr[lastj_offset];
        idx++;
      }
    }
  }  // num_partitions > JCHUNK

  // Aggregate tmp_out to out.
  float acc = 0.0f;
  #pragma unroll
  for (int j = 0; j < JCHUNK; j++) {
    acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j];
  }
  if (num_partitions > JCHUNK) {
  #pragma unroll
    for (int j = JCHUNK; j < 2 * JCHUNK; j++) {
      acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j];
    }
    if (num_partitions > 2 * JCHUNK) {
  #pragma unroll
      for (int j = 2 * JCHUNK; j < MAX_NPAR; j++) {
        acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j];
      }
    }
  }

  if (num_partitions > MAX_NPAR) {
    idx = 0;
  #pragma unroll
    for (int j = MAX_NPAR * HEAD_SIZE; j < 2 * MAX_NPAR * HEAD_SIZE;
         j += HEAD_SIZE) {
      // lastj is last valid partition
      const int lastj_offset =
          (j < num_partition_offset) ? j : last_partition_offset;
      tmps[idx] = tmp_out_ptr[lastj_offset];
      idx++;
    }

  #pragma unroll
    for (int j = 0; j < MAX_NPAR; j++) {
      acc += to_float<scalar_t>(tmps[j]) * shared_exp_sums[j + MAX_NPAR];
    }
  }

  const float inv_global_exp_sum =
      __fdividef(1.0f, shared_global_exp_sum + 1e-6f);
  acc *= inv_global_exp_sum;
  scalar_t* out_ptr =
      out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
  out_ptr[threadIdx.x] = from_float<scalar_t>(acc);
}

#else  // !defined(__HIP__MI300_MI250__) TODO: Add NAVI support

template <typename scalar_t, typename cache_t,
          vllm::Fp8KVCacheDataType KV_DTYPE, int BLOCK_SIZE, int HEAD_SIZE,
          int NUM_THREADS,
          int GQA_RATIO>
__global__ __launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_QKV_kernel(
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads, const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ context_lens,  // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                     // max_num_partitions]
    scalar_t* __restrict__ out,  // [num_seqs, num_heads, max_num_partitions,
                                 // head_size]
    scalar_t* __restrict__ final_out,  // [num_seqs, num_heads, head_size]
    int max_ctx_blocks, float k_scale, float v_scale) {
  UNREACHABLE_CODE
}

// Grid: (num_heads, num_seqs).
template <typename scalar_t, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE>
__global__
__launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_reduce_kernel(
    scalar_t* __restrict__ out,            // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,    // [num_seqs, num_heads,
                                           // max_num_partitions]
    const float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                           // max_num_partitions]
    const scalar_t* __restrict__ tmp_out,  // [num_seqs, num_heads,
                                           // max_num_partitions, head_size]
    const int* __restrict__ context_lens,  // [num_seqs]
    const int max_num_partitions){UNREACHABLE_CODE}

#endif  // defined(__HIP__MI300_MI250__) TODO: Add NAVI support

#define LAUNCH_CUSTOM_ATTENTION(GQA_RATIO)                                    \
  paged_attention_ll4mi_QKV_kernel<T, KVT, KV_DTYPE, BLOCK_SIZE, HEAD_SIZE,   \
                                   NTHR, GQA_RATIO>                           \
      <<<grid, block, 0, stream>>>(                                           \
          query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, scale,     \
          block_tables_ptr, context_lens_ptr, max_num_blocks_per_seq,         \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,        \
          exp_sums_ptr, max_logits_ptr, tmp_out_ptr, out_ptr, max_ctx_blocks, \
          k_scale, v_scale);

template <typename T, typename KVT, vllm::Fp8KVCacheDataType KV_DTYPE,
          int BLOCK_SIZE, int HEAD_SIZE, int PARTITION_SIZE = 512>
void paged_attention_custom_launcher(
    torch::Tensor& out, torch::Tensor& exp_sums, torch::Tensor& max_logits,
    torch::Tensor& tmp_out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, const int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& context_lens,
    int max_context_len, const c10::optional<torch::Tensor>& alibi_slopes,
    float k_scale, float v_scale) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  float* exp_sums_ptr = reinterpret_cast<float*>(exp_sums.data_ptr());
  float* max_logits_ptr = reinterpret_cast<float*>(max_logits.data_ptr());
  T* tmp_out_ptr = reinterpret_cast<T*>(tmp_out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  KVT* key_cache_ptr = reinterpret_cast<KVT*>(key_cache.data_ptr());
  KVT* value_cache_ptr = reinterpret_cast<KVT*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* context_lens_ptr = context_lens.data_ptr<int>();

  const int max_ctx_blocks = DIVIDE_ROUND_UP(max_context_len, BLOCK_SIZE);
  const int max_num_partitions =
      DIVIDE_ROUND_UP(max_context_len, PARTITION_SIZE);
  const int gqa_ratio = num_heads / num_kv_heads;
  assert(num_heads % num_kv_heads == 0);
  assert(head_size == HEAD_SIZE);
  assert(max_num_partitions <= 128);

  constexpr int NTHR = PARTITION_SIZE;
  dim3 grid(num_seqs, max_num_partitions, num_kv_heads);
  dim3 block(NTHR);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (gqa_ratio) {
    case 1:
      LAUNCH_CUSTOM_ATTENTION(1);
      break;
    case 2:
      LAUNCH_CUSTOM_ATTENTION(2);
      break;
    case 3:
      LAUNCH_CUSTOM_ATTENTION(3);
      break;
    case 4:
      LAUNCH_CUSTOM_ATTENTION(4);
      break;
    case 5:
      LAUNCH_CUSTOM_ATTENTION(5);
      break;
    case 6:
      LAUNCH_CUSTOM_ATTENTION(6);
      break;
    case 7:
      LAUNCH_CUSTOM_ATTENTION(7);
      break;
    case 8:
      LAUNCH_CUSTOM_ATTENTION(8);
      break;
    case 9:
      LAUNCH_CUSTOM_ATTENTION(9);
      break;
    case 10:
      LAUNCH_CUSTOM_ATTENTION(10);
      break;
    case 11:
      LAUNCH_CUSTOM_ATTENTION(11);
      break;
    case 12:
      LAUNCH_CUSTOM_ATTENTION(12);
      break;
    case 13:
      LAUNCH_CUSTOM_ATTENTION(13);
      break;
    case 14:
      LAUNCH_CUSTOM_ATTENTION(14);
      break;
    case 15:
      LAUNCH_CUSTOM_ATTENTION(15);
      break;
    case 16:
      LAUNCH_CUSTOM_ATTENTION(16);
      break;
    default:
      TORCH_CHECK(false, "Unsupported gqa ratio: ", gqa_ratio);
      break;
  }
  // dim3 grid2(num_heads,num_seqs,head_size/HEAD_ELEMS_PER_WG);
  // dim3 block2(1024);
  //  LAUNCH_CUSTOM_ATTENTION2;

  // reduction kernel is only required if max_context_len > partition size,
  // otherwise main kernel writes directly to final output
  //  note there are cases with graphing where max_context_len is the max
  //  supported by graphing, not the actual max among all the sequences: in that
  //  case reduction kernel will still run but return immediately
  if (max_context_len > PARTITION_SIZE) {
    dim3 reduce_grid(num_heads, num_seqs);
    dim3 reduce_block(head_size);
    paged_attention_ll4mi_reduce_kernel<T, HEAD_SIZE, HEAD_SIZE, PARTITION_SIZE>
        <<<reduce_grid, reduce_block, 0, stream>>>(
            out_ptr, exp_sums_ptr, max_logits_ptr, tmp_out_ptr,
            context_lens_ptr, max_num_partitions);
  }
}

#define CALL_CUSTOM_LAUNCHER(T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE)       \
  paged_attention_custom_launcher<T, KVT, KV_DTYPE, BLK_SIZE, HEAD_SIZE>( \
      out, exp_sums, max_logits, tmp_out, query, key_cache, value_cache,  \
      num_kv_heads, scale, block_tables, context_lens, max_context_len,   \
      alibi_slopes, k_scale, v_scale);

#define CALL_CUSTOM_LAUNCHER_BLK(T, KVT, KV_DTYPE, HEAD_SIZE)     \
  switch (block_size) {                                           \
    case 16:                                                      \
      CALL_CUSTOM_LAUNCHER(T, KVT, KV_DTYPE, 16, HEAD_SIZE);      \
      break;                                                      \
    case 32:                                                      \
      CALL_CUSTOM_LAUNCHER(T, KVT, KV_DTYPE, 32, HEAD_SIZE);      \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

#define CALL_CUSTOM_LAUNCHER_BLK_HEAD(T, KVT, KV_DTYPE)         \
  switch (head_size) {                                          \
    case 64:                                                    \
      CALL_CUSTOM_LAUNCHER_BLK(T, KVT, KV_DTYPE, 64);           \
      break;                                                    \
    case 128:                                                   \
      CALL_CUSTOM_LAUNCHER_BLK(T, KVT, KV_DTYPE, 128);          \
      break;                                                    \
    default:                                                    \
      TORCH_CHECK(false, "Unsupported head size: ", head_size); \
      break;                                                    \
  }

void paged_attention(
    torch::Tensor& out,         // [num_seqs, num_heads, head_size]
    torch::Tensor& exp_sums,    // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor& max_logits,  // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor&
        tmp_out,  // [num_seqs, num_heads, max_num_partitions, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,  // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads, double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& context_lens,  // [num_seqs]
    int64_t block_size, int64_t max_context_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, double k_scale, double v_scale) {
  const int head_size = query.size(2);
  if (kv_cache_dtype == "auto") {
    if (query.dtype() == at::ScalarType::Half) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(_Float16, _Float16,
                                    vllm::Fp8KVCacheDataType::kAuto);
    } else if (query.dtype() == at::ScalarType::BFloat16) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(__hip_bfloat16, __hip_bfloat16,
                                    vllm::Fp8KVCacheDataType::kAuto);
    } else {
      TORCH_CHECK(false, "Unsupported data type: ", query.dtype());
    }
  } else if (kv_cache_dtype == "fp8" || kv_cache_dtype == "fp8_e4m3") {
    if (query.dtype() == at::ScalarType::Half) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(_Float16, uint8_t,
                                    vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (query.dtype() == at::ScalarType::BFloat16) {
      CALL_CUSTOM_LAUNCHER_BLK_HEAD(__hip_bfloat16, uint8_t,
                                    vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else {
      TORCH_CHECK(false, "Unsupported data type: ", query.dtype());
    }
  } else {
    TORCH_CHECK(false, "Unsupported KV cache dtype: ", kv_cache_dtype);
  }
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP