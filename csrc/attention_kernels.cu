#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "attention_utils.h"
#include "cuda_primitives.h"
#include "reduction_utils.h"

#include <algorithm>

#define WARP_SIZE 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))

namespace cacheflow {

// Grid: (num_heads, num_seqs).
template<
  typename scalar_t,
  int HEAD_SIZE,
  int BLOCK_SIZE,
  int NUM_THREADS>
__global__ void single_query_cached_kv_attention_kernel(
  scalar_t* __restrict__ out,             // [num_seqs, num_heads, head_size]
  const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
  const scalar_t* __restrict__ k_cache,   // [num_blocks, num_heads, head_size/x, block_size, x]
  const scalar_t* __restrict__ v_cache,   // [num_blocks, num_heads, head_size, block_size]
  const float scale,
  const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq]
  const int* __restrict__ context_lens,   // [num_seqs]
  const int max_num_blocks_per_seq,
  const int q_stride) {
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP = (BLOCK_SIZE + WARP_SIZE - 1) / WARP_SIZE;
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;

  const int head_idx = blockIdx.x;
  const int num_heads = gridDim.x;
  const int seq_idx = blockIdx.y;

  // A vector type to store a part of a key or a query.
  // The vector size is configured in such a way that the threads in a thread group
  // fetch or compute 16 bytes at a time.
  // For example, if the size of a thread group is 4 and the data type is half,
  // then the vector size is 16 / (4 * sizeof(half)) == 2.
  constexpr int VEC_SIZE = MAX(16 / (THREAD_GROUP_SIZE * sizeof(scalar_t)), 1);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;

  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

  // Load the query to registers.
  // Each thread in a thread group has a different part of the query.
  // For example, if the the thread group size is 4, then the first thread in the group
  // has 0, 4, 8, ... th vectors of the query, and the second thread has 1, 5, 9, ...
  // th vectors of the query, and so on.
  // NOTE(woosuk): Because q is split from a qkv tensor, it may not be contiguous.
  const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  Q_vec q_vecs[NUM_VECS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < NUM_VECS_PER_THREAD; i++) {
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    q_vecs[i] = *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }

  // Memory planning.
  extern __shared__ char shared_mem[];
  // NOTE(woosuk): We use FP32 logits and accumulation.
  float *logits = reinterpret_cast<float*>(shared_mem);
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // x == THREAD_GROUP_SIZE * VEC_SIZE
  // Each thread group fetches x elements from the key at a time.
  constexpr int x = 16 / sizeof(scalar_t);
  float qk_max = -FLT_MAX;

  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  const int context_len = context_lens[seq_idx];
  const int num_blocks = (context_len + BLOCK_SIZE - 1) / BLOCK_SIZE;

  // Iterate over the key blocks.
  // Each warp fetches a block of keys for each iteration.
  // Each thread group in a warp fetches a key from the block, and computes
  // dot product with the query.
  for (int block_idx = warp_idx; block_idx < num_blocks; block_idx += NUM_WARPS) {
    const int physical_block_number = block_table[block_idx];

    // Load a key to registers.
    // Each thread in a thread group has a different part of the key.
    // For example, if the the thread group size is 4, then the first thread in the group
    // has 0, 4, 8, ... th vectors of the key, and the second thread has 1, 5, 9, ... th
    // vectors of the key, and so on.
    for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      const int physical_block_offset = (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      K_vec k_vecs[NUM_VECS_PER_THREAD];

#pragma unroll
      for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        const scalar_t* k_ptr = k_cache + physical_block_number * num_heads * HEAD_SIZE * BLOCK_SIZE
                                        + head_idx * HEAD_SIZE * BLOCK_SIZE
                                        + physical_block_offset * x;
        const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
        const int offset1 = (vec_idx * VEC_SIZE) / x;
        const int offset2 = (vec_idx * VEC_SIZE) % x;
        k_vecs[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
      }

      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
      const float qk = scale * Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(q_vecs, k_vecs);
      const bool mask = token_idx >= context_len;
    
      if (thread_group_offset == 0) {
        // Store the partial reductions to shared memory.
        // NOTE(woosuk): It is required to zero out the masked logits.
        logits[token_idx] = mask ? 0.f : qk;
        // Update the max value.
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }

  // Perform reduction across the threads in the same warp to get the
  // max qk value for each "warp" (not across the thread block yet).
  // The 0-th thread of each thread group already has its max qk value.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = qk_max;
  }
  __syncthreads();

  // TODO(woosuk): Refactor this part.
  // Get the max qk value for the sequence.
  qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
      qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }
  // Broadcast the max qk value to all threads.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  // Get the sum of the exp values.
  float exp_sum = 0.f;
  for (int i = thread_idx; i < context_len; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }
  exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);

  // Compute softmax.
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < context_len; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();

  // Each thread will fetch 16 bytes from the value cache at a time.
  constexpr int V_VEC_SIZE = MIN(16 / sizeof(scalar_t), BLOCK_SIZE);
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename FloatVec<V_vec>::Type;

  constexpr int NUM_V_VECS_PER_ROW = BLOCK_SIZE / V_VEC_SIZE;
  constexpr int NUM_ROWS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_ROW;
  constexpr int NUM_ROWS_PER_THREAD = (HEAD_SIZE + NUM_ROWS_PER_ITER - 1) / NUM_ROWS_PER_ITER;

  float accs[NUM_ROWS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    accs[i] = 0.f;
  }

  for (int block_idx = warp_idx; block_idx < num_blocks; block_idx += NUM_WARPS) {
    const int physical_block_number = block_table[block_idx];
    const int physical_block_offset = (lane % NUM_V_VECS_PER_ROW) * V_VEC_SIZE;
    const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
    L_vec logits_vec = *reinterpret_cast<L_vec*>(logits + token_idx);

    const scalar_t* v_ptr = v_cache + physical_block_number * num_heads * HEAD_SIZE * BLOCK_SIZE
                                    + head_idx * HEAD_SIZE * BLOCK_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE) {
        const int offset = row_idx * BLOCK_SIZE + physical_block_offset;
        V_vec v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
        accs[i] += dot(logits_vec, cast_to_float(v_vec));
      }
    }
  }

  // Perform reduction within each warp.
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    float acc = accs[i];
#pragma unroll
    for (int mask = NUM_V_VECS_PER_ROW / 2; mask >= 1; mask /= 2) {
      acc += __shfl_xor_sync(uint32_t(-1), acc, mask);
    }
    accs[i] = acc;
  }

  // NOTE(woosuk): A barrier is required because the shared memory space for logits
  // is reused for the output.
  __syncthreads();

  // Perform reduction across warps.
  float* out_smem = reinterpret_cast<float*>(shared_mem);
#pragma unroll
  for (int i = NUM_WARPS; i > 1; i /= 2) {
    int mid = i / 2;
    // Upper warps write to shared memory.
    if (warp_idx >= mid && warp_idx < i) {
      float* dst = &out_smem[(warp_idx - mid) * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          dst[row_idx] = accs[i];
        }
      }
    }
    __syncthreads();

    // Lower warps update the output.
    if (warp_idx < mid) {
      const float* src = &out_smem[warp_idx * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          accs[i] += src[row_idx];
        }
      }
    }
    __syncthreads();
  }

  // Write the final output.
  if (warp_idx == 0) {
    scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
        convert_from_float(*(out_ptr + row_idx), accs[i]);
      }
    }
  }
}

} // namespace cacheflow

#define LAUNCH_ATTENTION_KERNEL(T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS)                        \
  cacheflow::single_query_cached_kv_attention_kernel<T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>   \
  <<<grid, block, shared_mem_size, stream>>>(                                                 \
    out_ptr,                                                                                  \
    query_ptr,                                                                                \
    key_cache_ptr,                                                                            \
    value_cache_ptr,                                                                          \
    scale,                                                                                    \
    block_tables_ptr,                                                                         \
    context_lens_ptr,                                                                         \
    max_num_blocks_per_seq,                                                                   \
    query_stride);

// TODO(woosuk): Tune NUM_THREADS.
template<
  typename T,
  int BLOCK_SIZE,
  int NUM_THREADS = 128>
void single_query_cached_kv_attention_launcher(
  torch::Tensor& out,
  torch::Tensor& query,
  torch::Tensor& key_cache,
  torch::Tensor& value_cache,
  float scale,
  torch::Tensor& block_tables,
  torch::Tensor& context_lens,
  int max_context_len) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int query_stride = query.stride(0);

  int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  T* key_cache_ptr = reinterpret_cast<T*>(key_cache.data_ptr());
  T* value_cache_ptr = reinterpret_cast<T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* context_lens_ptr = context_lens.data_ptr<int>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int padded_max_context_len = ((max_context_len + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_context_len * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);
  int shared_mem_size = std::max(logits_size, outputs_size);

  dim3 grid(num_heads, num_seqs);
  dim3 block(NUM_THREADS);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (head_size) {
    case 32:
      LAUNCH_ATTENTION_KERNEL(T, 32, BLOCK_SIZE, NUM_THREADS);
      break;
    case 64:
      LAUNCH_ATTENTION_KERNEL(T, 64, BLOCK_SIZE, NUM_THREADS);
      break;
    case 80:
      LAUNCH_ATTENTION_KERNEL(T, 80, BLOCK_SIZE, NUM_THREADS);
      break;
    case 96:
      LAUNCH_ATTENTION_KERNEL(T, 96, BLOCK_SIZE, NUM_THREADS);
      break;
    case 128:
      LAUNCH_ATTENTION_KERNEL(T, 128, BLOCK_SIZE, NUM_THREADS);
      break;
    case 160:
      LAUNCH_ATTENTION_KERNEL(T, 160, BLOCK_SIZE, NUM_THREADS);
      break;
    case 192:
      LAUNCH_ATTENTION_KERNEL(T, 192, BLOCK_SIZE, NUM_THREADS);
      break;
    case 256:
      LAUNCH_ATTENTION_KERNEL(T, 256, BLOCK_SIZE, NUM_THREADS);
      break;
    default:
      assert(false);
      break;
  }
}

#define CALL_KERNEL_LAUNCHER(T, BLOCK_SIZE)                         \
  single_query_cached_kv_attention_launcher<T, BLOCK_SIZE>(         \ 
        out,                                                        \
        query,                                                      \
        key_cache,                                                  \
        value_cache,                                                \
        scale,                                                      \
        block_tables,                                               \
        context_lens,                                               \
        max_context_len);

void single_query_cached_kv_attention(
  torch::Tensor& out,             // [num_seqs, num_heads, head_size]
  torch::Tensor& query,           // [num_seqs, num_heads, head_size]
  torch::Tensor& key_cache,       // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,     // [num_blocks, num_heads, head_size, block_size]
  float scale,
  torch::Tensor& block_tables,    // [num_seqs, max_num_blocks_per_seq]
  torch::Tensor& context_lens,    // [num_seqs]
  int block_size,
  int max_context_len) {
  // TODO(woosuk): Support BF16.
  if (query.element_size() == 2) {
    // Half.
    if (block_size == 1) {
      CALL_KERNEL_LAUNCHER(uint16_t, 1);
    } else if (block_size == 2) {
      CALL_KERNEL_LAUNCHER(uint16_t, 2);
    } else if (block_size == 4) {
      CALL_KERNEL_LAUNCHER(uint16_t, 4);
    } else if (block_size == 8) {
      CALL_KERNEL_LAUNCHER(uint16_t, 8);
    } else if (block_size == 16) {
      CALL_KERNEL_LAUNCHER(uint16_t, 16);
    } else if (block_size == 32) {
      CALL_KERNEL_LAUNCHER(uint16_t, 32);
    } else if (block_size == 64) {
      CALL_KERNEL_LAUNCHER(uint16_t, 64);
    } else if (block_size == 128) {
      CALL_KERNEL_LAUNCHER(uint16_t, 128);
    } else if (block_size == 256) {
      CALL_KERNEL_LAUNCHER(uint16_t, 256);
    } else {
      assert(false);
    }
  } else {
    // Float.
    assert(false);
  }
}

// namespace cacheflow {

// // Grid: (num_heads, num_query_tokens).
// template<
//   typename scalar_t,
//   int HEAD_SIZE,
//   int BLOCK_SIZE,
//   int NUM_THREADS>
// __device__ void multi_query_cached_kv_attention_kernel_unoptimized_(
//   scalar_t* __restrict__ out,             // [num_seqs, num_heads, head_size]
//   const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
//   const int seq_start_idx,
//   const int seq_len,
//   const scalar_t* __restrict__ k_cache,   // [num_blocks, num_heads, head_size/x, block_size, x]
//   const scalar_t* __restrict__ v_cache,   // [num_blocks, num_heads, head_size, block_size]
//   const float scale,
//   const int* __restrict__ block_table,   // [num_seqs, max_num_blocks_per_seq]
//   const int context_len,
//   const int max_num_blocks_per_seq,
//   const int q_stride) {
//   constexpr int THREAD_GROUP_SIZE = WARP_SIZE / BLOCK_SIZE;
//   constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
//   const int thread_idx = threadIdx.x;
//   const int warp_idx = thread_idx / WARP_SIZE;
//   const int lane = thread_idx % WARP_SIZE;

//   const int head_idx = blockIdx.x;
//   const int num_heads = gridDim.x;
//   const int seq_idx = blockIdx.y;

//   // A vector type to store a part of a key or a query.
//   // The vector size is configured in such a way that the threads in a thread group
//   // fetch or comput 16 bytes at a time.
//   // For example, if the size of a thread group is 4 and the data type is half,
//   // then the vector size is 16 / (4 * sizeof(half)) == 2.
//   constexpr int VEC_SIZE = 16 / (THREAD_GROUP_SIZE * sizeof(scalar_t));
//   using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
//   using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;

//   constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
//   constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

//   const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
//   const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

//   // Load the query to registers.
//   // Each thread in a thread group has a different part of the query.
//   // For example, if the the thread group size is 4, then the first thread in the group
//   // has 0, 4, 8, ... th vectors of the query, and the second thread has 1, 5, 9, ...
//   // th vectors of the query, and so on.
//   // NOTE(woosuk): Because q is split from a qkv tensor, it may not be contiguous.
//   const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
//   Q_vec q_vecs[NUM_VECS_PER_THREAD];
// #pragma unroll
//   for (int i = 0; i < NUM_VECS_PER_THREAD; i++) {
//     const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
//     q_vecs[i] = *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
//   }

//   // Memory planning.
//   extern __shared__ char shared_mem[];
//   // NOTE(woosuk): We use FP32 logits and accumulation.
//   float *logits = reinterpret_cast<float*>(shared_mem);
//   // Workspace for reduction.
//   __shared__ float red_smem[2 * NUM_WARPS];

//   // x == THREAD_GROUP_SIZE * VEC_SIZE
//   // Each thread group fetches x elements from the key at a time.
//   constexpr int x = 16 / sizeof(scalar_t);
//   float qk_max = -FLT_MAX;

//   const int num_blocks = (context_len + BLOCK_SIZE - 1) / BLOCK_SIZE;
//   const int mask_boundary = context_len - seq_len + 1 + (seq_idx - seq_start_idx);

//   // Iterate over the key blocks.
//   // Each warp fetches a block of keys for each iteration.
//   // Each thread group in a warp fetches a key from the block, and computes
//   // dot product with the query.
//   for (int block_idx = warp_idx; block_idx < num_blocks; block_idx += NUM_WARPS) {
//     const int physical_block_number = block_table[block_idx];
//     const int physical_block_offset = thread_group_idx % BLOCK_SIZE;
//     const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;

//     // Load a key to registers.
//     // Each thread in a thread group has a different part of the key.
//     // For example, if the the thread group size is 4, then the first thread in the group
//     // has 0, 4, 8, ... th vectors of the key, and the second thread has 1, 5, 9, ... th
//     // vectors of the key, and so on.
//     K_vec k_vecs[NUM_VECS_PER_THREAD];
// #pragma unroll
//     for (int i = 0; i < NUM_VECS_PER_THREAD; i++) {
//       const scalar_t* k_ptr = k_cache + physical_block_number * num_heads * HEAD_SIZE * BLOCK_SIZE
//                                       + head_idx * HEAD_SIZE * BLOCK_SIZE
//                                       + physical_block_offset * x;
//       const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
//       const int offset1 = (vec_idx * VEC_SIZE) / x;
//       const int offset2 = (vec_idx * VEC_SIZE) % x;
//       k_vecs[i] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
//     }

//     // Compute dot product.
//     // This includes a reduction across the threads in the same thread group.
//     const float qk = scale * Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(q_vecs, k_vecs);
//     const bool mask = token_idx >= mask_boundary;

//     if (thread_group_offset == 0) {
//       // Store the partial reductions to shared memory.
//       // NOTE(woosuk): It is required to zero out the masked logits.
//       logits[token_idx] = mask ? 0.f : qk;
//       // Update the max value.
//       qk_max = mask ? qk_max : fmaxf(qk_max, qk);
//     }
//   }

//   // Perform reduction across the threads in the same warp to get the
//   // max qk value for each "warp" (not across the thread block yet).
//   // The 0-th thread of each thread group already has its max qk value.
// #pragma unroll
//   for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
//     qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
//   }
//   if (lane == 0) {
//     red_smem[warp_idx] = qk_max;
//   }
//   __syncthreads();

//   // TODO(woosuk): Refactor this part.
//   // Get the max qk value for the sequence.
//   qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
// #pragma unroll
//   for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
//       qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
//   }
//   // Broadcast the max qk value to all threads.
//   qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

//   // Get the sum of the exp values.
//   float exp_sum = 0.f;
//   for (int i = thread_idx; i < mask_boundary; i += NUM_THREADS) {
//     float val = __expf(logits[i] - qk_max);
//     logits[i] = val;
//     exp_sum += val;
//   }
//   exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);

//   // Compute softmax.
//   const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
//   for (int i = thread_idx; i < context_len; i += NUM_THREADS) {
//     logits[i] *= inv_sum;
//   }
//   __syncthreads();

//   // Each thread will fetch 16 bytes from the value cache at a time.
//   constexpr int V_VEC_SIZE = 16 / sizeof(scalar_t);
//   using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
//   using L_vec = typename FloatVec<V_vec>::Type;

//   constexpr int NUM_V_VECS_PER_ROW = BLOCK_SIZE / V_VEC_SIZE;
//   constexpr int NUM_ROWS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_ROW;
//   constexpr int NUM_ROWS_PER_THREAD = (HEAD_SIZE + NUM_ROWS_PER_ITER - 1) / NUM_ROWS_PER_ITER;

//   float accs[NUM_ROWS_PER_THREAD];
// #pragma unroll
//   for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
//     accs[i] = 0.f;
//   }

//   for (int block_idx = warp_idx; block_idx < num_blocks; block_idx += NUM_WARPS) {
//     const int physical_block_number = block_table[block_idx];
//     const int physical_block_offset = (lane % NUM_V_VECS_PER_ROW) * V_VEC_SIZE;
//     const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
//     L_vec logits_vec = *reinterpret_cast<L_vec*>(logits + token_idx);

//     const scalar_t* v_ptr = v_cache + physical_block_number * num_heads * HEAD_SIZE * BLOCK_SIZE
//                                     + head_idx * HEAD_SIZE * BLOCK_SIZE;
// #pragma unroll
//     for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
//       const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
//       if (row_idx < HEAD_SIZE) {
//         const int offset = row_idx * BLOCK_SIZE + physical_block_offset;
//         V_vec v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
//         accs[i] += dot(logits_vec, cast_to_float(v_vec));
//       }
//     }
//   }

//   // Perform reduction within each warp.
// #pragma unroll
//   for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
//     float acc = accs[i];
// #pragma unroll
//     for (int mask = NUM_V_VECS_PER_ROW / 2; mask >= 1; mask /= 2) {
//       acc += __shfl_xor_sync(uint32_t(-1), acc, mask);
//     }
//     accs[i] = acc;
//   }

//   // NOTE(woosuk): A barrier is required because the shared memory space for logits
//   // is reused for the output.
//   __syncthreads();

//   // Perform reduction across warps.
//   float* out_smem = reinterpret_cast<float*>(shared_mem);
// #pragma unroll
//   for (int i = NUM_WARPS; i > 1; i /= 2) {
//     int mid = i / 2;
//     // Upper warps write to shared memory.
//     if (warp_idx >= mid && warp_idx < i) {
//       float* dst = &out_smem[(warp_idx - mid) * HEAD_SIZE];
// #pragma unroll
//       for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
//         const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
//         if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
//           dst[row_idx] = accs[i];
//         }
//       }
//     }
//     __syncthreads();

//     // Lower warps update the output.
//     if (warp_idx < mid) {
//       const float* src = &out_smem[warp_idx * HEAD_SIZE];
// #pragma unroll
//       for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
//         const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
//         if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
//           accs[i] += src[row_idx];
//         }
//       }
//     }
//     __syncthreads();
//   }

//   // Write the final output.
//   if (warp_idx == 0) {
//     scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
// #pragma unroll
//     for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
//       const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
//       if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
//         convert_from_float(*(out_ptr + row_idx), accs[i]);
//       }
//     }
//   }
// }


// // Grid: (num_heads, num_query_tokens).
// template<
//   typename scalar_t,
//   int HEAD_SIZE,
//   int BLOCK_SIZE,
//   int NUM_THREADS>
// __global__ void multi_query_cached_kv_attention_kernel(
//   const int* cu_query_lens,               // [num_prompts+1]
//   const int* seq_prompt_mapping,          // [num_seqs] mapping from seq_idx to prompt_idx
//   scalar_t* __restrict__ out,             // [num_seqs, num_heads, head_size]
//   const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
//   const scalar_t* __restrict__ k_cache,   // [num_blocks, num_heads, head_size/x, block_size, x]
//   const scalar_t* __restrict__ v_cache,   // [num_blocks, num_heads, head_size, block_size]
//   const float scale,
//   const int* __restrict__ block_tables,   // [num_prompts, max_num_blocks_per_seq]
//   const int* __restrict__ context_lens,   // [num_prompts]
//   const int max_num_blocks_per_seq,
//   const int q_stride) {
//     const int seq_idx = blockIdx.y;
//     const int prompt_idx = seq_prompt_mapping[seq_idx];
//     const int seq_start_idx = cu_query_lens[prompt_idx];
//     const int seq_len = cu_query_lens[prompt_idx + 1] - seq_start_idx;
//     const int* block_table = block_tables + prompt_idx * max_num_blocks_per_seq;
//     const int context_len = context_lens[prompt_idx];
//     multi_query_cached_kv_attention_kernel_unoptimized_<
//         scalar_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>(
//           out,
//           q,
//           seq_start_idx,
//           seq_len,
//           k_cache,
//           v_cache,
//           scale,
//           block_table,
//           context_len,
//           max_num_blocks_per_seq,
//           q_stride);
// }

// } // namespace cacheflow

// #define LAUNCH_MULTI_ATTENTION_KERNEL(T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS)                  \
//   cacheflow::multi_query_cached_kv_attention_kernel<T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>    \
//   <<<grid, block, shared_mem_size, stream>>>(                                                 \
//     cu_query_lens_ptr,                                                                        \
//     seq_prompt_mapping_ptr,                                                                   \
//     out_ptr,                                                                                  \
//     query_ptr,                                                                                \
//     key_cache_ptr,                                                                            \
//     value_cache_ptr,                                                                          \
//     scale,                                                                                    \
//     block_tables_ptr,                                                                         \
//     context_lens_ptr,                                                                         \
//     max_num_blocks_per_seq,                                                                   \
//     query_stride);


// // TODO(woosuk): Tune NUM_THREADS.
// template<
//   typename T,
//   int BLOCK_SIZE,
//   int NUM_THREADS = 128>
// void multi_query_cached_kv_attention_launcher(
//   torch::Tensor& cu_query_lens,
//   torch::Tensor& seq_prompt_mapping,
//   torch::Tensor& out,
//   torch::Tensor& query,
//   torch::Tensor& key_cache,
//   torch::Tensor& value_cache,
//   float scale,
//   torch::Tensor& block_tables,
//   torch::Tensor& context_lens,
//   int max_context_len) {
//   int num_seqs = query.size(0);
//   int num_heads = query.size(1);
//   int head_size = query.size(2);
//   int max_num_blocks_per_seq = block_tables.size(1);
//   int query_stride = query.stride(0);

//   int* cu_query_lens_ptr = cu_query_lens.data_ptr<int>();
//   int* seq_prompt_mapping_ptr = seq_prompt_mapping.data_ptr<int>();
//   T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
//   T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
//   T* key_cache_ptr = reinterpret_cast<T*>(key_cache.data_ptr());
//   T* value_cache_ptr = reinterpret_cast<T*>(value_cache.data_ptr());
//   int* block_tables_ptr = block_tables.data_ptr<int>();
//   int* context_lens_ptr = context_lens.data_ptr<int>();

//   constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
//   int padded_max_context_len = ((max_context_len + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
//   int logits_size = padded_max_context_len * sizeof(float);
//   int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);
//   int shared_mem_size = std::max(logits_size, outputs_size);

//   dim3 grid(num_heads, num_seqs);
//   dim3 block(NUM_THREADS);
//   const hipStream_t stream = at::cuda::getCurrentCUDAStream();
//   switch (head_size) {
//     case 32:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 32, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 64:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 64, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 80:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 80, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 96:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 96, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 128:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 128, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 160:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 160, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 192:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 192, BLOCK_SIZE, NUM_THREADS);
//       break;
//     case 256:
//       LAUNCH_MULTI_ATTENTION_KERNEL(T, 256, BLOCK_SIZE, NUM_THREADS);
//       break;
//     default:
//       assert(false);
//       break;
//   }
// }

// void multi_query_cached_kv_attention(
//   torch::Tensor& cu_query_lens,
//   torch::Tensor& out,
//   torch::Tensor& query,
//   torch::Tensor& key_cache,
//   torch::Tensor& value_cache,
//   float scale,
//   torch::Tensor& block_tables,
//   torch::Tensor& context_lens,
//   int block_size,
//   int max_context_len) {

//   torch::Tensor query_lens = cu_query_lens.to(torch::kCPU);
  
//   int num_queries = query_lens.size(0) - 1;
//   const int* query_lens_ptr = query_lens.data_ptr<int>();
//   int num_seqs = query.size(0);

//   torch::Tensor cpu_tensor = torch::empty({num_seqs}, torch::dtype(torch::kInt32));
//   auto accessor = cpu_tensor.accessor<int32_t, 1>();
//   for (int i = 0, query_cursor = 0; i < num_seqs; ++i) {
//     if (i >= query_lens_ptr[query_cursor + 1]) {
//       ++query_cursor; 
//     }
//     accessor[i] = query_cursor;
//   }

//   // TODO(suquark): This can be slow, as it to(torch::kCPU) and to(torch::kCUDA)
//   // implicitly synchronizes the CPU and GPU. And we can avoid this issue by giving
//   // the mapping as an input parameter. Let's do this optimization in a later PR.
//   torch::Tensor seq_prompt_mapping = cpu_tensor.to(torch::kCUDA);

//   // TODO(woosuk): Support BF16.
//   if (query.element_size() == 2) {
//     // Half.
//     if (block_size == 8) {
//       multi_query_cached_kv_attention_launcher<uint16_t, 8>(
//         cu_query_lens,
//         seq_prompt_mapping,
//         out,
//         query,
//         key_cache,
//         value_cache,
//         scale,
//         block_tables,
//         context_lens,
//         max_context_len);
//     } else if (block_size == 16) {
//       multi_query_cached_kv_attention_launcher<uint16_t, 16>(
//         cu_query_lens,
//         seq_prompt_mapping,
//         out,
//         query,
//         key_cache,
//         value_cache,
//         scale,
//         block_tables,
//         context_lens,
//         max_context_len);
//     } else if (block_size == 32) {
//       multi_query_cached_kv_attention_launcher<uint16_t, 32>(
//         cu_query_lens,
//         seq_prompt_mapping,
//         out,
//         query,
//         key_cache,
//         value_cache,
//         scale,
//         block_tables,
//         context_lens,
//         max_context_len);
//     } else {
//       assert(false);
//     }
//   } else if (query.element_size() == 4) {
//     // Float.
//     if (block_size == 8) {
//       multi_query_cached_kv_attention_launcher<float, 8>(
//         cu_query_lens,
//         seq_prompt_mapping,
//         out,
//         query,
//         key_cache,
//         value_cache,
//         scale,
//         block_tables,
//         context_lens,
//         max_context_len);
//     } else if (block_size == 16) {
//       multi_query_cached_kv_attention_launcher<float, 16>(
//         cu_query_lens,
//         seq_prompt_mapping,
//         out,
//         query,
//         key_cache,
//         value_cache,
//         scale,
//         block_tables,
//         context_lens,
//         max_context_len);
//     } else if (block_size == 32) {
//       multi_query_cached_kv_attention_launcher<float, 32>(
//         cu_query_lens,
//         seq_prompt_mapping,
//         out,
//         query,
//         key_cache,
//         value_cache,
//         scale,
//         block_tables,
//         context_lens,
//         max_context_len);
//     } else {
//       assert(false);
//     }
//   } else {
//     assert(false);
//   }
// }

#undef WARP_SIZE
#undef MAX
#undef MIN
