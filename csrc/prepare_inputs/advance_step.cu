#include "hip/hip_runtime.h"
/*
 * The goal of this GPU kernel is to advance input tensors on the GPU directly
 * PR: https://github.com/vllm-project/vllm/pull/6338
 * Current restrictions:
 *     1. Specialized for DraftModelRunner
 *     2. Supports flash_attn only
 */

#include "advance_step.cuh"

namespace prepare_inputs {

//
template <int const num_threads>
__global__ void advance_step_kernel(int num_seqs, int num_queries,
                                    int block_size, long* input_tokens_ptr,
                                    long const* sampled_token_ids_ptr,
                                    long* input_positions_ptr,
                                    int* seq_lens_ptr, long* slot_mapping_ptr,
                                    int const* block_tables_ptr,
                                    int64_t const block_tables_stride) {
  int num_query_blocks = div_ceil(num_queries, num_threads);

  if (blockIdx.x >= num_query_blocks) {
    return;
  }

  int cur_query_id = blockIdx.x * num_threads + threadIdx.x;

  if (cur_query_id >= num_queries) {
    return;
  }

  // Update input_tokens
  input_tokens_ptr[cur_query_id] = sampled_token_ids_ptr[cur_query_id];

  int seq_len = seq_lens_ptr[cur_query_id];
  int next_seq_len = seq_len + 1;
  int next_input_pos = next_seq_len - 1;

  // Update seq_lens
  seq_lens_ptr[cur_query_id] = next_seq_len;
  // Update input_positions
  input_positions_ptr[cur_query_id] = next_input_pos;

  int const* seq_block_tables_ptr =
      block_tables_ptr + block_tables_stride * cur_query_id;

  int block_index = next_input_pos / block_size;
  int block_offset = next_input_pos % block_size;

  int slot_num = seq_block_tables_ptr[block_index] * block_size + block_offset;
  // Update slot_mapping
  slot_mapping_ptr[cur_query_id] = slot_num;
}

inline void verify_tensor(std::string const& name, torch::Tensor& t,
                          int64_t const size_0, int64_t const size_1,
                          c10::ScalarType const type) {
  bool size_0_cond = true;
  if (size_0 != -1) {
    size_0_cond = t.size(0) == size_0;
  }

  bool size_1_cond = true;
  if (size_1 != -1) {
    size_1_cond = t.size(1) == size_1;
  }

  bool is_contiguous = t.is_contiguous();
  bool same_type = t.dtype() == type;

  bool pass = size_0_cond && size_1_cond && is_contiguous && same_type;
  if (!pass) {
    TORCH_CHECK(false, "tensor: name = ", name, ", shape = ", t.sizes(),
                " is_cont = ", t.is_contiguous(), ", type = ", t.dtype(),
                " is not as expected: shape = [", size_0, ", ", size_1,
                "], type = ", type);
  }
}

void advance_step(int num_seqs, int num_queries, int block_size,
                  torch::Tensor& input_tokens,       // type: long
                  torch::Tensor& sampled_token_ids,  // type: long
                  torch::Tensor& input_positions,    // type: long
                  torch::Tensor& seq_lens,           // type: int
                  torch::Tensor& slot_mapping,       // type: long
                  torch::Tensor& block_tables) {     // type: int

  if (logging) {
    printf("advance_step:\n");
    printf("  num_seqs = %d\n", num_seqs);
    printf("  num_queries = %d\n", num_queries);
    printf("  block_size = %d\n", block_size);
  }
  // Verify all tensors
  verify_tensor("input_tokens", input_tokens, num_seqs, -1, at::kLong);
  verify_tensor("sampled_token_ids", sampled_token_ids, num_queries, 1,
                at::kLong);
  verify_tensor("input_positions", input_positions, num_seqs, -1, at::kLong);
  verify_tensor("seq_lens", seq_lens, num_seqs, -1, at::kInt);
  verify_tensor("slot_mapping", slot_mapping, num_seqs, -1, at::kLong);
  verify_tensor("block_tables", block_tables, num_seqs, -1, at::kInt);

  int dev = sampled_token_ids.get_device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(dev);

  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  advance_step_kernel<max_threads><<<blocks, max_threads, 0, stream>>>(
      num_seqs, num_queries, block_size,
      reinterpret_cast<long*>(input_tokens.data_ptr()),
      reinterpret_cast<long const*>(sampled_token_ids.data_ptr()),
      reinterpret_cast<long*>(input_positions.data_ptr()),
      reinterpret_cast<int*>(seq_lens.data_ptr()),
      reinterpret_cast<long*>(slot_mapping.data_ptr()),
      reinterpret_cast<int const*>(block_tables.data_ptr()),
      block_tables.stride(0));
}

}  // namespace prepare_inputs

void advance_step(int64_t num_seqs, int64_t num_queries, int64_t block_size,
                  torch::Tensor& input_tokens, torch::Tensor& sampled_token_ids,
                  torch::Tensor& input_positions, torch::Tensor& seq_lens,
                  torch::Tensor& slot_mapping, torch::Tensor& block_tables) {
  prepare_inputs::advance_step(num_seqs, num_queries, block_size, input_tokens,
                               sampled_token_ids, input_positions, seq_lens,
                               slot_mapping, block_tables);
}