#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#define CEILDIV(x,y) (((x) + (y) - 1) / (y))

namespace vllm {

namespace {
__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row, int32_t col) {
    // don't worry about overflow because num_experts is relatively small
    return row * total_col + col;
}
}

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(scalar_t *__restrict__ topk_ids, 
                                int32_t *sorted_token_ids, 
                                int32_t *expert_ids, 
                                int32_t *total_tokens_post_pad,
                                int32_t num_experts, 
                                int32_t block_size, 
                                size_t numel) {
    const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
    const size_t start_idx = threadIdx.x * tokens_per_thread;

    extern __shared__ int32_t shared_mem[];

    int32_t* tokens_cnts = shared_mem; // 2d tensor with shape (num_experts + 1, num_experts)
    int32_t* cumsum = shared_mem + (num_experts + 1) * num_experts; // 1d tensor with shape (num_experts + 1)

    for (int i = 0; i < num_experts; ++i) {
        tokens_cnts[index(num_experts, threadIdx.x + 1, i)] = 0;
    }

    /**
    * In the first step we compute token_cnts[thread_index + 1][expert_index],
    * which counts how many tokens in the token shard of thread_index are assigned
    * to expert expert_index.
    */
    for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
        ++tokens_cnts[index(num_experts, threadIdx.x + 1, topk_ids[i])]; 
    }

    __syncthreads();

    // For each expert we accumulate the token counts from the different threads.
    tokens_cnts[index(num_experts, 0, threadIdx.x)] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
        tokens_cnts[index(num_experts, i, threadIdx.x)] += tokens_cnts[index(num_experts, i-1, threadIdx.x)];
    }

    __syncthreads();
    
    // We accumulate the token counts of all experts in thread 0.
    if (threadIdx.x == 0) {
        cumsum[0] = 0;
        for (int i = 1; i <= num_experts; ++i) {
            cumsum[i] = cumsum[i-1] + CEILDIV(tokens_cnts[index(num_experts, blockDim.x, i - 1)], block_size) * block_size;
        }
        *total_tokens_post_pad = cumsum[num_experts];
    }

    __syncthreads();

    /**
    * For each expert, each thread processes the tokens of the corresponding blocks
    * and stores the corresponding expert_id for each block.
    */
    for (int i = cumsum[threadIdx.x];i < cumsum[threadIdx.x + 1];i += block_size) {
        expert_ids[i / block_size] = threadIdx.x;
    }
    
    /**
    * Each thread processes a token shard, calculating the index of each token after
    * sorting by expert number. Given the example topk_ids = [0,1,2,1,2,3,0,3,4] and
    * block_size = 4, then the output would be [0, 6, *, *, 1, 3, *, *, 2, 4, *, *, 5, 7, *, *, 8, *, *, *],
    * where * represents a padding value(preset in python).
    */
    for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
        int32_t expert_id = topk_ids[i];
        /** The cumsum[expert_id] stores the starting index of the tokens that the
        * expert with expert_id needs to process, and tokens_cnts[threadIdx.x][expert_id]
        * stores the indices of the tokens processed by the expert with expert_id within
        * the current thread's token shard.
        */
        int32_t rank_post_pad = tokens_cnts[index(num_experts, threadIdx.x, expert_id)] + cumsum[expert_id];
        sorted_token_ids[rank_post_pad] = i;
        ++tokens_cnts[index(num_experts, threadIdx.x, expert_id)];
    }
}
}

void moe_align_block_size(
    torch::Tensor topk_ids,
    int num_experts,
    int block_size,
    torch::Tensor sorted_token_ids,
    torch::Tensor experts_ids,
    torch::Tensor num_tokens_post_pad) {
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    VLLM_DISPATCH_INTEGRAL_TYPES(
        topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
        // calc needed amount of shared mem for `tokens_cnts` and `cumsum` tensors
        const int32_t shared_mem = ((num_experts + 1) * num_experts + (num_experts + 1)) * sizeof(int32_t);

        // set dynamic shared mem
        auto kernel = vllm::moe_align_block_size_kernel<scalar_t>;
        AT_CUDA_CHECK(
            VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize((void *)kernel, shared_mem));
        kernel<<<1, num_experts, shared_mem, stream>>>(
            topk_ids.data_ptr<scalar_t>(),
            sorted_token_ids.data_ptr<int32_t>(), 
            experts_ids.data_ptr<int32_t>(), 
            num_tokens_post_pad.data_ptr<int32_t>(), 
            num_experts,
            block_size,
            topk_ids.numel());
    });
}
