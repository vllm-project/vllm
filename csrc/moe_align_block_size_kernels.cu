#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include "cuda_compat.h"
#include "dispatch_utils.h"

const static size_t NUM_MAX_EXPERTS = 64;
#define CEILDIV(x,y) (((x) + (y) - 1) / (y))

namespace vllm {
template <typename scalar_t>
__global__ void moe_align_block_size_kernel(scalar_t *__restrict__ topk_ids, 
                                int32_t *sorted_token_ids, 
                                int32_t *expert_ids, 
                                int32_t *total_tokens_post_pad,
                                int32_t num_experts, 
                                int32_t block_size, 
                                size_t numel) {
    const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
    const size_t start_idx = threadIdx.x * tokens_per_thread;
    __shared__ int32_t tokens_cnts[NUM_MAX_EXPERTS + 1][NUM_MAX_EXPERTS];
    __shared__ int32_t cumsum[NUM_MAX_EXPERTS + 1];
    for (int i = 0; i < num_experts; ++i) {
        tokens_cnts[threadIdx.x + 1][i] = 0;
    }

    /**
    * In the first step we compute token_cnts[thread_index + 1][expert_index],
    * which counts how many tokens in the token shard of thread_index are assigned
    * to expert expert_index.
    */
    for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
        ++tokens_cnts[threadIdx.x + 1][topk_ids[i]]; 
    }

    __syncthreads();

    // For each expert we accumulate the token counts from the different threads.
    tokens_cnts[0][threadIdx.x] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
        tokens_cnts[i][threadIdx.x] += tokens_cnts[i-1][threadIdx.x];
    }

    __syncthreads();
    
    // We accumulate the token counts of all experts in thread 0.
    if (threadIdx.x == 0) {
        cumsum[0] = 0;
        for (int i = 1; i <= num_experts; ++i) {
            cumsum[i] = cumsum[i-1] + CEILDIV(tokens_cnts[blockDim.x][i - 1], block_size) * block_size;
        }
        *total_tokens_post_pad = cumsum[num_experts];
    }

    __syncthreads();

    /**
    * For each expert, each thread processes the tokens of the corresponding blocks
    * and stores the corresponding expert_id for each block.
    */
    for (int i = cumsum[threadIdx.x];i < cumsum[threadIdx.x + 1];i += block_size) {
        expert_ids[i / block_size] = threadIdx.x;
    }
    
    /**
    * Each thread processes a token shard, calculating the index of each token after
    * sorting by expert number. Given the example topk_ids = [0,1,2,1,2,3,0,3,4] and
    * block_size = 4, then the output would be [0, 6, *, *, 1, 3, *, *, 2, 4, *, *, 5, 7, *, *, 8, *, *, *],
    * where * represents a padding value(preset in python).
    */
    for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
        int32_t expert_id = topk_ids[i];
        /** The cumsum[expert_id] stores the starting index of the tokens that the
        * expert with expert_id needs to process, and tokens_cnts[threadIdx.x][expert_id]
        * stores the indices of the tokens processed by the expert with expert_id within
        * the current thread's token shard.
        */
        int32_t rank_post_pad = tokens_cnts[threadIdx.x][expert_id] + cumsum[expert_id];
        sorted_token_ids[rank_post_pad] = i;
        ++tokens_cnts[threadIdx.x][expert_id];
    }
}
}

void moe_align_block_size(
    torch::Tensor topk_ids,
    int num_experts,
    int block_size,
    torch::Tensor sorted_token_ids,
    torch::Tensor experts_ids,
    torch::Tensor num_tokens_post_pad) {
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    assert(num_experts <= NUM_MAX_EXPERTS);
    VLLM_DISPATCH_INTEGRAL_TYPES(
        topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
        vllm::moe_align_block_size_kernel<scalar_t><<<1, num_experts, 0, stream>>>(
            topk_ids.data_ptr<scalar_t>(), 
            sorted_token_ids.data_ptr<int32_t>(), 
            experts_ids.data_ptr<int32_t>(), 
            num_tokens_post_pad.data_ptr<int32_t>(), 
            num_experts,
            block_size,
            topk_ids.numel());
    });
}
