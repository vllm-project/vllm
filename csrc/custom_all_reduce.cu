#include "hip/hip_runtime.h"
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

#include "custom_all_reduce.cuh"

// fake pointer type
using fptr_t = uint64_t;
static_assert(sizeof(void *) == sizeof(fptr_t));

fptr_t init_custom_ar(torch::Tensor &meta, torch::Tensor &rank_data,
                      const std::vector<std::string> &handles,
                      const std::vector<int64_t> &offsets, int rank,
                      bool full_nvlink) {
  int world_size = offsets.size();
  if (world_size > 8)
    throw std::invalid_argument("world size > 8 is not supported");
  if (world_size % 2 != 0)
    throw std::invalid_argument("Odd num gpus is not supported for now");
  if (world_size != handles.size())
    throw std::invalid_argument(
        "handles length should equal to offsets length");
  if (rank < 0 || rank >= world_size)
    throw std::invalid_argument("invalid rank passed in");

  hipIpcMemHandle_t ipc_handles[8];
  for (int i = 0; i < world_size; i++) {
    std::memcpy(&ipc_handles[i], handles[i].data(), sizeof(hipIpcMemHandle_t));
  }
  return (fptr_t) new vllm::CustomAllreduce(
      reinterpret_cast<vllm::Metadata *>(meta.data_ptr()), rank_data.data_ptr(),
      rank_data.numel(), ipc_handles, offsets, rank, full_nvlink);
}

/**
 * Make sure tensor t's data lies completely within ((char)t.data_ptr()) +
 * t.numel() * t.element_size(). This is slightly weaker than t.is_contiguous()
 * because it allows transpose of contiguous slice (i.e. slicing the first
 * dimension). Currently, we require this because stride information is not
 * passed into the kernels and we treat input tensors as flat.
 *
 * Examples
 * A = torch.zeros(3, 3, 3)
 * 1. A: OK
 * 2. A[1:]: OK
 * 3. A.permute(2, 0, 1): OK
 * 4. A[1:].permute(2, 0, 1): OK
 * 5. A[None].expand(2, -1, -1, -1): Not OK
 * 6. A[:, 1:, 1:]: Not OK
 */
bool _is_weak_contiguous(torch::Tensor &t) {
  return t.is_contiguous() ||
         (t.storage().nbytes() - t.storage_offset() * t.element_size() ==
          t.numel() * t.element_size());
}

bool should_custom_ar(torch::Tensor &inp, int max_size, int world_size,
                      bool full_nvlink) {
  auto inp_size = inp.numel() * inp.element_size();
  // custom allreduce requires input byte size to be multiples of 16
  if (inp_size % 16 != 0) return false;
  if (!_is_weak_contiguous(inp)) return false;
  if (world_size == 2 || full_nvlink) return inp_size <= max_size;
  // 4 PCIE GPUs use 2 stage allreduce, and is only faster than NCCL when size
  // <= 512k
  return world_size <= 4 && inp_size <= 512 * 1024;
}

void _all_reduce(fptr_t _fa, torch::Tensor &inp, torch::Tensor &out,
                 hipStream_t stream) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce *>(_fa);
  TORCH_CHECK(_is_weak_contiguous(out));
  switch (out.scalar_type()) {
    case at::ScalarType::Float: {
      fa->allreduce<float>(stream, reinterpret_cast<float *>(inp.data_ptr()),
                           reinterpret_cast<float *>(out.data_ptr()),
                           out.numel());
      break;
    }
    case at::ScalarType::Half: {
      fa->allreduce<half>(stream, reinterpret_cast<half *>(inp.data_ptr()),
                          reinterpret_cast<half *>(out.data_ptr()),
                          out.numel());
      break;
    }
#if (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
    case at::ScalarType::BFloat16: {
      fa->allreduce<hip_bfloat16>(
          stream, reinterpret_cast<hip_bfloat16 *>(inp.data_ptr()),
          reinterpret_cast<hip_bfloat16 *>(out.data_ptr()), out.numel());
      break;
    }
#endif
    default:
      throw std::runtime_error(
          "custom allreduce only supports float32, float16 and bfloat16");
  }
}

void all_reduce_reg(fptr_t _fa, torch::Tensor &inp, torch::Tensor &out) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(inp));
  auto stream = c10::cuda::getCurrentCUDAStream().stream();
  TORCH_CHECK_EQ(inp.scalar_type(), out.scalar_type());
  TORCH_CHECK_EQ(inp.numel(), out.numel());
  _all_reduce(_fa, inp, out, stream);
}

void all_reduce_unreg(fptr_t _fa, torch::Tensor &inp, torch::Tensor &reg_buffer,
                      torch::Tensor &out) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(inp));
  auto stream = c10::cuda::getCurrentCUDAStream().stream();

  auto input_size = inp.numel() * inp.element_size();
  TORCH_CHECK_EQ(inp.scalar_type(), out.scalar_type());
  TORCH_CHECK_EQ(inp.numel(), out.numel());
  TORCH_CHECK(input_size <= reg_buffer.numel() * reg_buffer.element_size(),
              "registered buffer is too small to contain the input");
  AT_CUDA_CHECK(hipMemcpyAsync(reg_buffer.data_ptr(), inp.data_ptr(),
                                input_size, hipMemcpyDeviceToDevice, stream));
  _all_reduce(_fa, reg_buffer, out, stream);
}

void dispose(fptr_t _fa) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce *>(_fa);
  delete fa;
}

int meta_size() { return sizeof(vllm::Metadata); }

void register_buffer(fptr_t _fa, torch::Tensor &t,
                     const std::vector<std::string> &handles,
                     const std::vector<int64_t> &offsets) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce *>(_fa);
  fa->register_buffer(handles, offsets, t.data_ptr());
}

std::pair<std::vector<uint8_t>, std::vector<int64_t>> get_graph_buffer_ipc_meta(
    fptr_t _fa) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce *>(_fa);
  return fa->get_graph_buffer_ipc_meta();
}

void register_graph_buffers(fptr_t _fa, const std::vector<std::string> &handles,
                            const std::vector<std::vector<int64_t>> &offsets) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce *>(_fa);
  fa->register_graph_buffers(handles, offsets);
}
