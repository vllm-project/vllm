#include "hip/hip_runtime.h"
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/all.h>

#include "custom_all_reduce.cuh"

// Fake pointer type, must match fptr_t type in ops.h.
// We use this type alias to indicate when pointers are passed in as int64_t.
using fptr_t = int64_t;
static_assert(sizeof(void*) == sizeof(fptr_t));

fptr_t init_custom_ar(const std::vector<fptr_t>& fake_ipc_ptrs,
                      torch::Tensor& rank_data, int64_t rank,
                      bool full_nvlink) {
  int world_size = fake_ipc_ptrs.size();
  if (world_size > 8)
    throw std::invalid_argument("world size > 8 is not supported");
  if (world_size % 2 != 0)
    throw std::invalid_argument("Odd num gpus is not supported for now");
  if (rank < 0 || rank >= world_size)
    throw std::invalid_argument("invalid rank passed in");

  vllm::Signal* ipc_ptrs[8];
  for (int i = 0; i < world_size; i++) {
    ipc_ptrs[i] = reinterpret_cast<vllm::Signal*>(fake_ipc_ptrs[i]);
  }
  return (fptr_t) new vllm::CustomAllreduce(ipc_ptrs, rank_data.data_ptr(),
                                            rank_data.numel(), rank, world_size,
                                            full_nvlink);
}

/**
 * Make sure tensor t's data lies completely within ((char)t.data_ptr()) +
 * t.numel() * t.element_size(). This is slightly weaker than t.is_contiguous()
 * because it allows transpose of contiguous slice (i.e. slicing the first
 * dimension). Currently, we require this because stride information is not
 * passed into the kernels and we treat input tensors as flat.
 *
 * Examples
 * A = torch.zeros(3, 3, 3)
 * 1. A: OK
 * 2. A[1:]: OK
 * 3. A.permute(2, 0, 1): OK
 * 4. A[1:].permute(2, 0, 1): OK
 * 5. A[None].expand(2, -1, -1, -1): Not OK
 * 6. A[:, 1:, 1:]: Not OK
 */
bool _is_weak_contiguous(torch::Tensor& t) {
  return t.is_contiguous() ||
         (t.storage().nbytes() - t.storage_offset() * t.element_size() ==
          t.numel() * t.element_size());
}

/**
 * Performs an out-of-place allreduce and stores result in out.
 *
 * If _reg_buffer is null, assumes inp.data_ptr() is already IPC-registered.
 * Otherwise, _reg_buffer is assumed to be IPC-registered and inp is first
 * copied into _reg_buffer.
 */
void all_reduce(fptr_t _fa, torch::Tensor& inp, torch::Tensor& out,
                fptr_t _reg_buffer, int64_t reg_buffer_sz_bytes) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(inp));
  auto stream = c10::cuda::getCurrentCUDAStream().stream();

  TORCH_CHECK_EQ(inp.scalar_type(), out.scalar_type());
  TORCH_CHECK_EQ(inp.numel(), out.numel());
  TORCH_CHECK(_is_weak_contiguous(out));
  TORCH_CHECK(_is_weak_contiguous(inp));
  auto input_size = inp.numel() * inp.element_size();
  auto reg_buffer = reinterpret_cast<void*>(_reg_buffer);
  if (reg_buffer) {
    TORCH_CHECK_LE(input_size, reg_buffer_sz_bytes);
    AT_CUDA_CHECK(hipMemcpyAsync(reg_buffer, inp.data_ptr(), input_size,
                                  hipMemcpyDeviceToDevice, stream));
  } else {
    reg_buffer = inp.data_ptr();
  }
  switch (out.scalar_type()) {
    case at::ScalarType::Float: {
      fa->allreduce<float>(stream, reinterpret_cast<float*>(reg_buffer),
                           reinterpret_cast<float*>(out.data_ptr()),
                           out.numel());
      break;
    }
    case at::ScalarType::Half: {
      fa->allreduce<half>(stream, reinterpret_cast<half*>(reg_buffer),
                          reinterpret_cast<half*>(out.data_ptr()), out.numel());
      break;
    }
#if (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
    case at::ScalarType::BFloat16: {
      fa->allreduce<hip_bfloat16>(
          stream, reinterpret_cast<hip_bfloat16*>(reg_buffer),
          reinterpret_cast<hip_bfloat16*>(out.data_ptr()), out.numel());
      break;
    }
#endif
    default:
      throw std::runtime_error(
          "custom allreduce only supports float32, float16 and bfloat16");
  }
}

void dispose(fptr_t _fa) {
  delete reinterpret_cast<vllm::CustomAllreduce*>(_fa);
}

int64_t meta_size() { return sizeof(vllm::Signal); }

void register_buffer(fptr_t _fa, const std::vector<fptr_t>& fake_ipc_ptrs) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  TORCH_CHECK(fake_ipc_ptrs.size() == fa->world_size_);
  void* ipc_ptrs[8];
  for (int i = 0; i < fake_ipc_ptrs.size(); i++) {
    ipc_ptrs[i] = reinterpret_cast<void*>(fake_ipc_ptrs[i]);
  }
  fa->register_buffer(ipc_ptrs);
}

// Use vector<int64_t> to represent byte data for python binding compatibility.
std::tuple<std::vector<int64_t>, std::vector<int64_t>>
get_graph_buffer_ipc_meta(fptr_t _fa) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  auto [handle, offsets] = fa->get_graph_buffer_ipc_meta();
  std::vector<int64_t> bytes(handle.begin(), handle.end());
  return std::make_tuple(bytes, offsets);
}

// Use vector<int64_t> to represent byte data for python binding compatibility.
void register_graph_buffers(fptr_t _fa,
                            const std::vector<std::vector<int64_t>>& handles,
                            const std::vector<std::vector<int64_t>>& offsets) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  std::vector<std::string> bytes;
  bytes.reserve(handles.size());
  for (int i = 0; i < handles.size(); i++) {
    bytes.emplace_back(handles[i].begin(), handles[i].end());
  }
  bytes.reserve(handles.size());
  fa->register_graph_buffers(bytes, offsets);
}
