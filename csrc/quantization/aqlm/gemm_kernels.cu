#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Adapted from https://github.com/Vahe1994/AQLM
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>
#include <cstdlib>

namespace vllm {
namespace aqlm {

__global__ void Code1x16MatVec(
    const int4* __restrict__ A, const int4* __restrict__ B,
    int4* __restrict__ C, const int4* __restrict__ codebook, const int prob_m,
    const int prob_k,
    const int4 codebook_a_sizes,  // cumulative sizes of A spanning each
                                  // codebook, at most 3 long.
    const int codebook_stride     // as int4.
) {
  int a_gl_stride = prob_k / 8 / 8;
  int a_gl_rd = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
  bool pred = a_gl_rd < prob_m;

  if (pred) {
    // advance to the correct codebook, this easy because we only multiply one
    // column of the codebook.
    auto codebook_size = &codebook_a_sizes.x;
    while (a_gl_rd >= *codebook_size) {
      codebook += codebook_stride;
      ++codebook_size;
    }
  }

  int b_gl_rd = 0;
  int c_gl_wr = a_gl_rd;
  a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % 32;
  int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % 32;

  __shared__ int4 sh_b[32 * 9];
  float res = 0;

  int iters = (prob_k / 8 + 8 * 32 - 1) / (8 * 32);
  while (iters--) {
    // We pad shared memory to avoid bank conflicts during reads
    __syncthreads();
    for (int i = threadIdx.x; i < 32 * 8; i += blockDim.x) {
      if (b_gl_rd + i < prob_k / 8) sh_b[9 * (i / 8) + i % 8] = B[b_gl_rd + i];
    }
    __syncthreads();
    b_gl_rd += 32 * 8;

    int b_sh_rd = 9 * (threadIdx.x % 32);
    if (pred && a_gl_rd < a_gl_end) {
      const uint16_t* enc = reinterpret_cast<const uint16_t*>(&A[a_gl_rd]);
#pragma unroll
      for (int i = 0; i < 8; i++) {
        uint32_t dec[4];
        // We bypass the L1 cache to avoid massive amounts of memory streaming
        // that doesn't actually help us; this brings > 2x speedup.
        asm volatile("ld.cg.global.v4.u32 {%0, %1, %2, %3}, [%4];"
                     : "=r"(dec[0]), "=r"(dec[1]), "=r"(dec[2]), "=r"(dec[3])
                     : "l"((void*)&codebook[enc[i]]));
        half2* a = reinterpret_cast<half2*>(&dec);
        half2* b = reinterpret_cast<half2*>(&sh_b[b_sh_rd]);
        half2 res2 = {};
#pragma unroll
        for (int j = 0; j < 4; j++) res2 = __hfma2(a[j], b[j], res2);
        res += __half2float(res2.x) + __half2float(res2.y);
        b_sh_rd++;
      }
      a_gl_rd += 32;
    }
  }

  if (pred) {
#pragma unroll
    for (int i = 16; i > 0; i /= 2) res += __shfl_down_sync(0xffffffff, res, i);
    if (threadIdx.x % 32 == 0)
      reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
  }
}

__global__ void Code2x8MatVec(
    const int4* __restrict__ A, const int4* __restrict__ B,
    int4* __restrict__ C, const int4* __restrict__ codebook, int prob_m,
    int prob_k,
    const int4 codebook_a_sizes,  // cumulative sizes of A spanning each
                                  // codebook, at most 3 long.
    const int codebook_stride     // as int4.

) {
  int a_gl_stride = prob_k / 8 / 8;
  int a_gl_rd = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
  bool pred = a_gl_rd < prob_m;

  if (pred) {
    // advance to the correct codebook, this easy because we only multiply one
    // column of the codebook.
    auto codebook_size = &codebook_a_sizes.x;
    while (a_gl_rd >= *codebook_size) {
      codebook += codebook_stride;
      ++codebook_size;
    }
  }

  int b_gl_rd = 0;
  int c_gl_wr = a_gl_rd;
  a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % 32;
  int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % 32;
  int lane = threadIdx.x % 8;

  extern __shared__ int4 sh[];
  int4* sh_b = sh;
  int4* sh_code = sh_b + 32 * 9;
  int4* sh_code0 = sh_code;
  int4* sh_code1 = sh_code + 256 * 8;

  for (int i = threadIdx.x; i < 2 * 256; i += blockDim.x) {
    int4 dec = codebook[i];
#pragma unroll
    for (int j = 0; j < 8; j++) sh_code[8 * i + (j + lane) % 8] = dec;
  }
  __syncthreads();

  float res = 0;

  int iters = (prob_k / 8 + 8 * 32 - 1) / (8 * 32);
  while (iters--) {
    // We pad shared memory to avoid bank conflicts during reads
    __syncthreads();
    for (int i = threadIdx.x; i < 32 * 8; i += blockDim.x) {
      if (b_gl_rd + i < prob_k / 8) sh_b[9 * (i / 8) + i % 8] = B[b_gl_rd + i];
    }
    __syncthreads();
    b_gl_rd += 32 * 8;

    int b_sh_rd = 9 * (threadIdx.x % 32);
    if (pred && a_gl_rd < a_gl_end) {
      const uint8_t* enc = reinterpret_cast<const uint8_t*>(&A[a_gl_rd]);
#pragma unroll
      for (int i = 0; i < 8; i++) {
        half2* a0 =
            reinterpret_cast<half2*>(&sh_code0[8 * enc[2 * i + 0] + lane]);
        half2* a1 =
            reinterpret_cast<half2*>(&sh_code1[8 * enc[2 * i + 1] + lane]);
        half2* b = reinterpret_cast<half2*>(&sh_b[b_sh_rd]);
        half2 res2 = {};
#pragma unroll
        for (int j = 0; j < 4; j++)
          res2 = __hfma2(__hadd2(a0[j], a1[j]), b[j], res2);
        res += __half2float(res2.x) + __half2float(res2.y);
        b_sh_rd++;
      }
      a_gl_rd += 32;
    }
  }

  if (pred) {
#pragma unroll
    for (int i = 16; i > 0; i /= 2) res += __shfl_down_sync(0xffffffff, res, i);
    if (threadIdx.x % 32 == 0)
      reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
  }
}

__global__ void Code1x16Dequant(
    const int4* __restrict__ A, int4* __restrict__ C,
    const int4* __restrict__ codebook, int prob_m, int prob_k,
    const int4 codebook_a_sizes,  // cumulative sizes of A spanning each
                                  // codebook, at most 3 long, sums to m.
    const int codebook_stride     // as int4
) {
  int a_gl_stride = prob_k / 8 / 8;
  int a_gl_rd = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
  bool pred = a_gl_rd < prob_m;

  if (pred) {
    // advance to the correct codebook, this easy because we only multiply one
    // column of the codebook.
    auto codebook_size = &codebook_a_sizes.x;
    while (a_gl_rd >= *codebook_size) {
      codebook += codebook_stride;
      ++codebook_size;
    }
  }

  a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % 32;
  int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % 32;

  int c_gl_stride = prob_k / 8;
  int c_gl_wr = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
  c_gl_wr = c_gl_stride * c_gl_wr + (threadIdx.x % 32) * 8;

  int iters = (prob_k / 8 - 1) / (8 * 32) + 1;
  while (iters--) {
    if (pred && a_gl_rd < a_gl_end) {
      const uint16_t* enc = reinterpret_cast<const uint16_t*>(&A[a_gl_rd]);
#pragma unroll
      for (int i = 0; i < 8; i++) {
        int4 chunk;
        auto dec = reinterpret_cast<uint32_t*>(&chunk);
        // We bypass the L1 cache to avoid massive amounts of memory streaming
        // that doesn't actually help us; this brings > 2x speedup.
        asm volatile("ld.cg.global.v4.u32 {%0, %1, %2, %3}, [%4];"
                     : "=r"(dec[0]), "=r"(dec[1]), "=r"(dec[2]), "=r"(dec[3])
                     : "l"((void*)&codebook[enc[i]]));

        C[a_gl_rd * 8 + i] = chunk;
      }
    }
    a_gl_rd += 32;
  }
}

__global__ void Code2x8Dequant(
    const int4* __restrict__ A, int4* __restrict__ C,
    const int4* __restrict__ codebook, int prob_m, int prob_k,
    const int4
        codebook_a_sizes,  // cumulative sizes of A spanning each codebook, at
                           // most 3 long, corresponds to cols.
    const int codebook_stride  // as int4
) {
  int a_gl_stride = prob_k / 8 / 8;
  int a_gl_rd = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
  bool pred = a_gl_rd < prob_m;

  if (pred) {
    // advance to the correct codebook, this easy because we only multiply one
    // column of the codebook.
    auto codebook_size = &codebook_a_sizes.x;
    while (a_gl_rd >= *codebook_size) {
      codebook += codebook_stride;
      ++codebook_size;
    }
  }

  a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % 32;
  int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % 32;
  int lane = threadIdx.x % 8;

  int c_gl_stride = prob_k / 8;
  int c_gl_wr = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
  c_gl_wr = c_gl_stride * c_gl_wr + (threadIdx.x % 32) * 8;

  extern __shared__ int4 sh[];
  int4* sh_code = sh;
  int4* sh_code0 = sh_code;
  int4* sh_code1 = sh_code + 256 * 8;

  for (int i = threadIdx.x; i < 2 * 256; i += blockDim.x) {
    int4 dec = codebook[i];
#pragma unroll
    for (int j = 0; j < 8; j++) sh_code[8 * i + (j + lane) % 8] = dec;
  }
  __syncthreads();

  float res = 0;

  int iters = (prob_k / 8 - 1) / (8 * 32) + 1;
  while (iters--) {
    if (pred && a_gl_rd < a_gl_end) {
      const uint8_t* enc = reinterpret_cast<const uint8_t*>(&A[a_gl_rd]);
#pragma unroll
      for (int i = 0; i < 8; i++) {
        int4 chunk;
        half2* a0 =
            reinterpret_cast<half2*>(&sh_code0[8 * enc[2 * i + 0] + lane]);
        half2* a1 =
            reinterpret_cast<half2*>(&sh_code1[8 * enc[2 * i + 1] + lane]);
#pragma unroll
        for (int j = 0; j < 4; j++)
          reinterpret_cast<half2*>(&chunk)[j] = __hadd2(a0[j], a1[j]);
        C[a_gl_rd * 8 + i] = chunk;
      }
    }
    a_gl_rd += 32;
  }
}

inline int ceildiv(int a, int b) { return (a + b - 1) / b; }

const int THREAD_M = 16;

void code1x16_matvec_cuda(const void* __restrict__ A,
                          const void* __restrict__ B, void* __restrict__ C,
                          const void* __restrict__ codebook, int prob_m,
                          int prob_k, const int4 codebook_a_sizes,
                          const int codebook_stride) {
  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  Code1x16MatVec<<<blocks, threads, 16 * 32 * 9, stream>>>(
      (const int4*)A, (const int4*)B, (int4*)C, (const int4*)codebook, prob_m,
      prob_k, codebook_a_sizes, codebook_stride);
}

void code2x8_matvec_cuda(const void* __restrict__ A, const void* __restrict__ B,
                         void* __restrict__ C,
                         const void* __restrict__ codebook, int prob_m,
                         int prob_k, const int4 codebook_a_sizes,
                         const int codebook_stride) {
  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  int shared = 16 * (2 * 256 * 8 + 32 * 9);
  hipFuncSetAttribute(reinterpret_cast<const void*>(Code2x8MatVec),
                       hipFuncAttributeMaxDynamicSharedMemorySize, shared);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  Code2x8MatVec<<<blocks, threads, shared, stream>>>(
      (const int4*)A, (const int4*)B, (int4*)C, (const int4*)codebook, prob_m,
      prob_k, codebook_a_sizes, codebook_stride);
}

void code1x16_dequant_cuda(
    const void* __restrict__ A, void* __restrict__ C,
    const void* __restrict__ codebook, int prob_m, int prob_k,
    const int4 codebook_a_sizes,  // cumulative sizes of A spanning each
                                  // codebook, at most 3 long.
    const int codebook_stride     // as int4.
) {
  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  Code1x16Dequant<<<blocks, threads, 0, stream>>>(
      (const int4*)A, (int4*)C, (const int4*)codebook, prob_m, prob_k,
      codebook_a_sizes,  // cumulative sizes of A spanning each codebook, at
                         // most 3 long.
      codebook_stride    // as int4.
  );
}

// Dequantizes the code and codebook into weights.
void code2x8_dequant_cuda(
    const void* __restrict__ A, void* __restrict__ C,
    const void* __restrict__ codebook, int prob_m, int prob_k,
    const int4
        codebook_a_sizes,  // cumulative sizes of A spanning each codebook, at
                           // most 3 long, corresponds to cols.
    const int codebook_stride  // as int4
) {
  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  int shared = 16 * (2 * 256 * 8 + 32 * 9);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  hipFuncSetAttribute(reinterpret_cast<const void*>(Code2x8Dequant),
                       hipFuncAttributeMaxDynamicSharedMemorySize, shared);
  Code2x8Dequant<<<blocks, threads, shared, stream>>>(
      (const int4*)A, (int4*)C, (const int4*)codebook, prob_m, prob_k,
      codebook_a_sizes, codebook_stride);
}

int codebook_stride(const torch::Tensor& codebooks) {
  return codebooks.stride(0) * codebooks.element_size() / sizeof(int4);
}

void code1x16_matvec(
    const torch::Tensor& A, const torch::Tensor& B, torch::Tensor& C,
    const torch::Tensor& codebook,
    const int4 codebook_a_sizes  // cumulative sizes of A spanning each
                                 // codebook, at most 3 long.
) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
  int prob_m = C.size(0);
  int prob_k = B.size(0);

  code1x16_matvec_cuda(A.data_ptr(), B.data_ptr(), C.data_ptr(),
                       codebook.data_ptr(), prob_m, prob_k, codebook_a_sizes,
                       codebook_stride(codebook));
}

torch::Tensor code1x16_matmat(const torch::Tensor& input,
                              const torch::Tensor& codes,
                              const torch::Tensor& codebooks,
                              const torch::Tensor& scales,
                              const int4 codebook_a_sizes,
                              const std::optional<torch::Tensor>& bias) {
  auto input_sizes = input.sizes();
  auto out_features = codes.size(0) * codebooks.size(2);
  auto flat_input = input.reshape({-1, input.size(-1)});
  auto flat_output = torch::empty(
      {flat_input.size(0), out_features},
      torch::TensorOptions().dtype(input.dtype()).device(input.device()));

  for (int i = 0; i < flat_input.size(0); ++i) {
    auto input_vec = flat_input.index({i});
    auto output_vec = flat_output.index({i});
    code1x16_matvec(codes.squeeze(2), input_vec, output_vec, codebooks,
                    codebook_a_sizes);
  }
  flat_output *= scales.flatten().unsqueeze(0);

  if (bias.has_value()) {
    flat_output += bias->unsqueeze(0);
  }

  auto output_sizes = input_sizes.vec();
  output_sizes.pop_back();
  output_sizes.push_back(-1);
  auto output = flat_output.reshape(output_sizes);
  return output;
}

void code2x8_matvec(const torch::Tensor& A, const torch::Tensor& B,
                    torch::Tensor& C, const torch::Tensor& codebook,
                    const int4 codebook_a_sizes) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
  int prob_m = C.size(0);
  int prob_k = B.size(0);
  code2x8_matvec_cuda(A.data_ptr(), B.data_ptr(), C.data_ptr(),
                      codebook.data_ptr(), prob_m, prob_k, codebook_a_sizes,
                      2 * codebook_stride(codebook));
}

torch::Tensor code2x8_matmat(const torch::Tensor& input,
                             const torch::Tensor& codes,
                             const torch::Tensor& codebooks,
                             const torch::Tensor& scales,
                             const int4 codebook_a_sizes,
                             const std::optional<torch::Tensor>& bias) {
  auto input_sizes = input.sizes();
  auto out_features = codes.size(0) * codebooks.size(2);
  auto flat_input = input.reshape({-1, input.size(-1)});
  auto flat_output = torch::empty(
      {flat_input.size(0), out_features},
      torch::TensorOptions().dtype(input.dtype()).device(input.device()));

  for (int i = 0; i < flat_input.size(0); ++i) {
    auto input_vec = flat_input.index({i});
    auto output_vec = flat_output.index({i});
    code2x8_matvec(codes.squeeze(2), input_vec, output_vec, codebooks,
                   codebook_a_sizes);
  }
  flat_output *= scales.flatten().unsqueeze(0);
  if (bias.has_value()) {
    flat_output += bias->unsqueeze(0);
  }

  auto output_sizes = input_sizes.vec();
  output_sizes.pop_back();
  output_sizes.push_back(-1);
  auto output = flat_output.reshape(output_sizes);
  return output;
}

// Accumulate the partition sizes.
int4 accumulate_sizes(const torch::Tensor& codebook_partition_sizes) {
  int4 cumulative_sizes;
  auto cumulative_size = &cumulative_sizes.x;
  int i = 0;
  int last = 0;
  assert(codebook_partition_sizes.size(0) <= 4);
  for (; i < codebook_partition_sizes.size(0); ++i, ++cumulative_size) {
    *cumulative_size = codebook_partition_sizes[i].item<int>() + last;
    last = *cumulative_size;
  }
  // fill in the rest with unreachable.
  for (; i < 4; ++i, ++cumulative_size) {
    *cumulative_size = last * 10;
  }
  return cumulative_sizes;
}

}  // namespace aqlm
}  // namespace vllm

torch::Tensor aqlm_gemm(const torch::Tensor& input, const torch::Tensor& codes,
                        const torch::Tensor& codebooks,
                        const torch::Tensor& scales,
                        const torch::Tensor& codebook_partition_sizes,
                        const std::optional<torch::Tensor>& bias) {
  int4 cumulative_sizes =
      vllm::aqlm::accumulate_sizes(codebook_partition_sizes);

  int const nbooks = codebooks.size(0) / codebook_partition_sizes.size(0);
  int const entries = codebooks.size(1);

  if (nbooks == 1 && entries == (1 << 16)) {
    return vllm::aqlm::code1x16_matmat(input, codes, codebooks, scales,
                                       cumulative_sizes, bias);
  }
  if (nbooks == 2 && entries == (1 << 8)) {
    return vllm::aqlm::code2x8_matmat(input, codes, codebooks, scales,
                                      cumulative_sizes, bias);
  }

  TORCH_CHECK(false, "AQLM with ", nbooks, " codebooks and ", entries,
              " entries is not currently supported.")
  return {};
}

torch::Tensor aqlm_dequant(const torch::Tensor& codes,
                           const torch::Tensor& codebooks,
                           const torch::Tensor& codebook_partition_sizes) {
  int4 cumulative_sizes =
      vllm::aqlm::accumulate_sizes(codebook_partition_sizes);

  int const nbooks = codebooks.size(0) / codebook_partition_sizes.size(0);
  int const entries = codebooks.size(1);

  const at::cuda::OptionalCUDAGuard device_guard(device_of(codes));
  int rows = codes.size(1);
  int cols = codes.size(0);

  auto in_features = codes.size(1) * 8;
  auto out_features = codes.size(0);

  assert(out_features = codebook_partition_sizes.sum().item<int>());

  auto weights = torch::empty({out_features, in_features},
                              torch::TensorOptions()
                                  .dtype(codebooks.dtype())
                                  .device(codebooks.device()));

  if (nbooks == 1 && entries == (1 << 16)) {
    vllm::aqlm::code1x16_dequant_cuda(codes.data_ptr(), weights.data_ptr(),
                                      codebooks.data_ptr(), out_features,
                                      in_features, cumulative_sizes,
                                      vllm::aqlm::codebook_stride(codebooks));

    // if you wanted to flip to scaling the weights, (though it's 30%-ish slower
    // and not consistent with gemv implementation.) weights *=
    // scales.index({"...", 0, 0});

    return weights;
  }

  if (nbooks == 2 && entries == (1 << 8)) {
    vllm::aqlm::code2x8_dequant_cuda(codes.data_ptr(), weights.data_ptr(),
                                     codebooks.data_ptr(), out_features,
                                     in_features, cumulative_sizes,
                                     vllm::aqlm::codebook_stride(codebooks));

    // if you wanted to flip to scaling the weights, (though it's 30%-ish slower
    // and not consistent with gemv implementation) weights *=
    // scales.index({"...", 0, 0});

    return weights;
  }

  TORCH_CHECK(false, "AQLM with ", nbooks, " codebooks and ", entries,
              " entries is not currently supported.")
  return {};
}
