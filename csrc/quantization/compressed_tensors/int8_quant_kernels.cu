#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <cmath>

#include "../../dispatch_utils.h"

static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static const float i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static const float i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());
  // round
  float dst = std::nearbyint(x);
  // saturate
  dst = std::clamp(dst, i8_min, i8_max);
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ out,
    const scale_type* scale_ptr, const int hidden_size) {
  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;
  scale_type scale = *scale_ptr;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] =
        float_to_int8_rn(((float)input[token_idx * hidden_size + i]) / scale);
  }
}
}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);

  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        vllm::static_scaled_int8_quant_kernel<scalar_t, float>
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),
                                         out.data_ptr<int8_t>(),
                                         scale.data_ptr<float>(), hidden_size);
      });
}
