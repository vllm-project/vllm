#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>

#include <cmath>

#include "../../dispatch_utils.h"
#include "../vectorization_utils.cuh"

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
  #include <cub/util_type.cuh>
#else
  #include <hipcub/hipcub.hpp>
  #include <hipcub/util_type.hpp>
#endif

static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static constexpr auto i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());

  // To match the rounding mode of CUDA, we use nearbyint.
  // It uses the current rounding mode, which is always FE_TONEAREST on HIP.
  // If that changes in the future, we may need to set the rounding mode
  // explicitly, either at runtime or compile time.
  float dst = std::nearbyint(x);

  // saturate

  // See https://github.com/pytorch/pytorch/issues/127666
  // See https://github.com/llvm/llvm-project/issues/95183
  // hip-clang std::clamp __glibcxx_assert_fail host function when building on
  // Arch/gcc14. The following replaces std::clamp usage with similar logic
  // dst = std::clamp(dst, i8_min, i8_max);
  dst = (dst < i8_min) ? i8_min : (dst > i8_max) ? i8_max : dst;
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

static inline __device__ int32_t float_to_int32_rn(float x) {
#ifdef USE_ROCM
  // int32_max is not exactly representable as float.
  // Therefore, we need to be careful and manually return int32_max on overflow.
  // For symmetry, we also do the same for int32_min, even though it is exactly
  // representable as float and the conversion should be exact.
  static constexpr auto i32_min = std::numeric_limits<int32_t>::min();
  static constexpr auto i32_min_f = static_cast<float>(i32_min);
  static constexpr auto i32_max = std::numeric_limits<int32_t>::max();
  static constexpr auto i32_max_f = static_cast<float>(i32_max);

  // To match the rounding mode of CUDA, we use nearbyint.
  // It uses the current rounding mode, which is always FE_TONEAREST on HIP.
  // If that changes in the future, we may need to set the rounding mode
  // explicitly, either at runtime or compile time.
  float dst = std::nearbyint(x);

  // saturate on the higher end.
  if (dst >= i32_max_f) {
    return i32_max;
  }
  // saturate on the lower end.
  if (dst <= i32_min_f) {
    return i32_min;
  }

  return static_cast<int32_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s32.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int32_t&>(dst);
#endif
}

static inline __device__ int8_t int32_to_int8(int32_t x) {
#ifdef USE_ROCM
  static constexpr auto i8_min =
      static_cast<int32_t>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<int32_t>(std::numeric_limits<int8_t>::max());

  // saturate

  // See https://github.com/pytorch/pytorch/issues/127666
  // See https://github.com/llvm/llvm-project/issues/95183
  // hip-clang std::clamp __glibcxx_assert_fail host function when building on
  // Arch/gcc14. The following replaces std::clamp usage with similar logic
  // int32_t dst = std::clamp(x, i8_min, i8_max);
  int32_t dst = (x < i8_min) ? i8_min : (x > i8_max) ? i8_max : x;
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.sat.s8.s32 %0, %1;" : "=r"(dst) : "r"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_t>
__global__ void static_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    const scale_t* scale_ptr, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;
  const float scale = *scale_ptr;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        dst = float_to_int8_rn(static_cast<float>(src) / scale);
      });
}

template <typename scalar_t, typename scale_t, typename azp_t>
__global__ void static_scaled_int8_azp_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    const scale_t* scale_ptr, const azp_t* azp_ptr, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;
  const float scale = *scale_ptr;
  const azp_t azp = *azp_ptr;
  const float inv_s = 1.0f / scale;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        const auto v = static_cast<float>(src) * inv_s;
        dst = int32_to_int8(float_to_int32_rn(v) + azp);
      });
}

template <typename scalar_t, typename scale_t>
__global__ void dynamic_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    scale_t* scale_out, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  // calculate for absmax
  float thread_max = 0.f;
  for (int i = tid; i < hidden_size; i += stride) {
    const auto v = fabsf(static_cast<float>(row_in[i]));
    thread_max = fmaxf(thread_max, v);
  }
  using BlockReduce = hipcub::BlockReduce<float, 256>;
  __shared__ typename BlockReduce::TempStorage tmp;
  float block_max = BlockReduce(tmp).Reduce(thread_max, hipcub::Max{}, blockDim.x);
  __shared__ float absmax;
  if (tid == 0) {
    absmax = block_max;
    scale_out[blockIdx.x] = absmax / 127.f;
  }
  __syncthreads();

  float inv_s = (absmax == 0.f) ? 0.f : 127.f / absmax;

  // 2. quantize
  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        dst = float_to_int8_rn(static_cast<float>(src) * inv_s);
      });
}

// MinMax structure to hold min and max values in one go
struct MinMax {
  float min, max;

  __host__ __device__ MinMax()
      : min(std::numeric_limits<float>::max()),
        max(std::numeric_limits<float>::lowest()) {}

  __host__ __device__ explicit MinMax(float v) : min(v), max(v) {}

  // add a value to the MinMax
  __host__ __device__ MinMax& operator+=(float v) {
    min = fminf(min, v);
    max = fmaxf(max, v);
    return *this;
  }

  // merge two MinMax objects
  __host__ __device__ MinMax& operator&=(const MinMax& other) {
    min = fminf(min, other.min);
    max = fmaxf(max, other.max);
    return *this;
  }
};

__host__ __device__ inline MinMax operator+(MinMax a, float v) {
  return a += v;
}
__host__ __device__ inline MinMax operator&(MinMax a, const MinMax& b) {
  return a &= b;
}

template <typename scalar_t, typename scale_t, typename azp_t>
__global__ void dynamic_scaled_int8_azp_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    scale_t* scale_out, azp_t* azp_out, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  // 1. calculate min & max
  MinMax thread_mm;
  for (int i = tid; i < hidden_size; i += stride) {
    thread_mm += static_cast<float>(row_in[i]);
  }

  using BlockReduce = hipcub::BlockReduce<MinMax, 256>;
  __shared__ typename BlockReduce::TempStorage tmp;

  MinMax mm = BlockReduce(tmp).Reduce(
      thread_mm,
      [] __device__(MinMax a, const MinMax& b) {
        a &= b;
        return a;
      },
      blockDim.x);

  __shared__ float scale_sh;
  __shared__ azp_t azp_sh;
  if (tid == 0) {
    float s = (mm.max - mm.min) / 255.f;
    float zp = nearbyintf(-128.f - mm.min / s);  // round-to-even
    scale_sh = s;
    azp_sh = azp_t(zp);
    scale_out[blockIdx.x] = s;
    azp_out[blockIdx.x] = azp_sh;
  }
  __syncthreads();

  const float inv_s = 1.f / scale_sh;
  const azp_t azp = azp_sh;

  // 2. quantize
  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        const auto v = static_cast<float>(src) * inv_s;
        dst = int32_to_int8(float_to_int32_rn(v) + azp);
      });
}

}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale,
                              std::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);
  TORCH_CHECK(!azp || azp->numel() == 1);

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 256));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::static_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), hidden_size);
        } else {
          vllm::static_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}

void dynamic_scaled_int8_quant(
    torch::Tensor& out,          // [..., hidden_size]
    torch::Tensor const& input,  // [..., hidden_size]
    torch::Tensor& scales, std::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scales.is_contiguous());
  TORCH_CHECK(!azp || azp->is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 256));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), hidden_size);
        } else {
          vllm::dynamic_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}
