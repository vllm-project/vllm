#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <cmath>

#include "../../dispatch_utils.h"
#include "../../reduction_utils.cuh"

static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static const float i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static const float i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());
  // round
  float dst = std::nearbyint(x);
  // saturate
  dst = std::clamp(dst, i8_min, i8_max);
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type const* scale_ptr, const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  scale_type const scale = *scale_ptr;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] = float_to_int8_rn(
        static_cast<float>(input[token_idx * hidden_size + i]) / scale);
  }
}

template <typename scalar_t, typename scale_type>
__global__ void dynamic_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  float absmax_val = 0.0f;
  float const zero = 0.0f;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float val = static_cast<float>(input[token_idx * hidden_size + i]);
    val = val > zero ? val : -val;
    absmax_val = val > absmax_val ? val : absmax_val;
  }

  float const block_absmax_val_maybe = blockReduceMax(absmax_val);
  __shared__ float block_absmax_val;
  if (tid == 0) {
    block_absmax_val = block_absmax_val_maybe;
    scale[token_idx] = block_absmax_val / 127.0f;
  }
  __syncthreads();

  float const tmp_scale = 127.0f / block_absmax_val;
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] = float_to_int8_rn(
        static_cast<float>(input[token_idx * hidden_size + i]) * tmp_scale);
  }
}

}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        vllm::static_scaled_int8_quant_kernel<scalar_t, float>
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),
                                         out.data_ptr<int8_t>(),
                                         scale.data_ptr<float>(), hidden_size);
      });
}

void dynamic_scaled_int8_quant(
    torch::Tensor& out,          // [..., hidden_size]
    torch::Tensor const& input,  // [..., hidden_size]
    torch::Tensor& scales) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
        vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float>
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),
                                         out.data_ptr<int8_t>(),
                                         scales.data_ptr<float>(), hidden_size);
      });
}
