#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <cmath>

#include "../../dispatch_utils.h"

#ifndef USE_ROCM
  #include <cub/util_type.cuh>
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/util_type.hpp>
  #include <hipcub/hipcub.hpp>
#endif

static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static constexpr auto i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());

  // To match the rounding mode of CUDA, we use nearbyint.
  // It uses the current rounding mode, which is always FE_TONEAREST on HIP.
  // If that changes in the future, we may need to set the rounding mode
  // explicitly, either at runtime or compile time.
  float dst = std::nearbyint(x);

  // saturate
  dst = std::clamp(dst, i8_min, i8_max);
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

static inline __device__ int32_t float_to_int32_rn(float x) {
#ifdef USE_ROCM
  // int32_max is not exactly representable as float.
  // Therefore, we need to be careful and manually return int32_max on overflow.
  // For symmetry, we also do the same for int32_min, even though it is exactly
  // representable as float and the conversion should be exact.
  static constexpr auto i32_min = std::numeric_limits<int32_t>::min();
  static constexpr auto i32_min_f = static_cast<float>(i32_min);
  static constexpr auto i32_max = std::numeric_limits<int32_t>::max();
  static constexpr auto i32_max_f = static_cast<float>(i32_max);

  // To match the rounding mode of CUDA, we use nearbyint.
  // It uses the current rounding mode, which is always FE_TONEAREST on HIP.
  // If that changes in the future, we may need to set the rounding mode
  // explicitly, either at runtime or compile time.
  float dst = std::nearbyint(x);

  // saturate on the higher end.
  if (dst >= i32_max_f) {
    return i32_max;
  }
  // saturate on the lower end.
  if (dst <= i32_min_f) {
    return i32_min;
  }

  return static_cast<int32_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s32.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int32_t&>(dst);
#endif
}

static inline __device__ int8_t int32_to_int8(int32_t x) {
#ifdef USE_ROCM
  static constexpr auto i8_min =
      static_cast<int32_t>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<int32_t>(std::numeric_limits<int8_t>::max());

  // saturate
  int32_t dst = std::clamp(x, i8_min, i8_max);
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.sat.s8.s32 %0, %1;" : "=r"(dst) : "r"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type const* scale_ptr, const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  scale_type const scale = *scale_ptr;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] = float_to_int8_rn(
        static_cast<float>(input[token_idx * hidden_size + i]) / scale);
  }
}

template <typename scalar_t, typename scale_type, typename azp_type>
__global__ void static_scaled_int8_azp_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type const* scale_ptr, azp_type const* azp_ptr,
    const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  scale_type const scale = *scale_ptr;
  azp_type const azp = *azp_ptr;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    auto const val = static_cast<float>(input[token_idx * hidden_size + i]);
    auto const quant_val = int32_to_int8(float_to_int32_rn(val / scale) + azp);
    out[token_idx * hidden_size + i] = quant_val;
  }
}

template <typename scalar_t, typename scale_type>
__global__ void dynamic_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  float absmax_val = 0.0f;
  float const zero = 0.0f;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float val = static_cast<float>(input[token_idx * hidden_size + i]);
    val = val > zero ? val : -val;
    absmax_val = val > absmax_val ? val : absmax_val;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float block_absmax_val;
  if (tid == 0) {
    block_absmax_val = block_absmax_val_maybe;
    scale[token_idx] = block_absmax_val / 127.0f;
  }
  __syncthreads();

  float const tmp_scale = 127.0f / block_absmax_val;
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] = float_to_int8_rn(
        static_cast<float>(input[token_idx * hidden_size + i]) * tmp_scale);
  }
}

template <typename scalar_t, typename scale_type, typename azp_type>
__global__ void dynamic_scaled_int8_azp_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, azp_type* azp, const int hidden_size) {
  int const token_idx = blockIdx.x;

  // Scan for the min and max value for this token
  float max_val = std::numeric_limits<float>::min();
  float min_val = std::numeric_limits<float>::max();
  for (int i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    auto val = static_cast<float>(input[token_idx * hidden_size + i]);
    max_val = std::max(max_val, val);
    min_val = std::min(min_val, val);
  }

  // Reduce the max and min values across the block
  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  max_val = BlockReduce(reduceStorage).Reduce(max_val, hipcub::Max{}, blockDim.x);
  __syncthreads();  // Make sure min doesn't mess with max shared memory
  min_val = BlockReduce(reduceStorage).Reduce(min_val, hipcub::Min{}, blockDim.x);

  __shared__ scale_type scale_sh;
  __shared__ azp_type azp_sh;

  // Compute the scale and zero point and store them, only on the first thread
  if (threadIdx.x == 0) {
    float const scale_val = (max_val - min_val) / 255.0f;
    // Use rounding to even (same as torch.round)
    auto const azp_float = std::nearbyint(-128.0f - min_val / scale_val);
    auto const azp_val = static_cast<azp_type>(azp_float);

    // Store the scale and azp into shared and global
    scale[token_idx] = scale_sh = scale_val;
    azp[token_idx] = azp_sh = azp_val;
  }

  // Wait for the scale and azp to be computed
  __syncthreads();

  float const scale_val = scale_sh;
  azp_type const azp_val = azp_sh;

  // Quantize the values
  for (int i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    auto const val = static_cast<float>(input[token_idx * hidden_size + i]);
    auto const quant_val =
        int32_to_int8(float_to_int32_rn(val / scale_val) + azp_val);
    out[token_idx * hidden_size + i] = quant_val;
  }
}

}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale,
                              c10::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);
  TORCH_CHECK(!azp || azp->numel() == 1);

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::static_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), hidden_size);
        } else {
          vllm::static_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}

void dynamic_scaled_int8_quant(
    torch::Tensor& out,          // [..., hidden_size]
    torch::Tensor const& input,  // [..., hidden_size]
    torch::Tensor& scales, c10::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scales.is_contiguous());
  TORCH_CHECK(!azp || azp->is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), hidden_size);
        } else {
          vllm::dynamic_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}
