#include "hip/hip_runtime.h"
#include <stddef.h>
#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>

// clang-format will break include orders
// clang-format off
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/util/device_memory.h"

#include "cutlass/cutlass.h"
#include "cutlass/gemm_coord.h"
#include "cutlass/arch/mma_sm75.h"
#include "cutlass/arch/arch.h"
#include "cutlass/arch/mma.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"

#include "cutlass/epilogue/threadblock/fusion/visitors.hpp"
#include "cutlass/gemm/kernel/default_gemm_universal_with_visitor.h"

#include "broadcast_load_epilogue_c2x.hpp"
#include "common.hpp"
// clang-format on

using namespace cute;

/*
   This file defines quantized GEMM operations using the CUTLASS 2.x API, for
   NVIDIA GPUs with SM versions prior to sm90 (Hopper).

   Epilogue functions can be defined to post-process the output before it is
   written to GPU memory.
   Epilogues must contain a public type named EVTCompute of type Sm80EVT,
   as well as a static prepare_args function that constructs an
   EVTCompute::Arguments struct.
*/

namespace {

// Wrappers for the GEMM kernel that is used to guard against compilation on
// architectures that will never use the kernel. The purpose of this is to
// reduce the size of the compiled binary.
// __CUDA_ARCH__ is not defined in host code, so this lets us smuggle the ifdef
// into code that will be executed on the device where it is defined.
template <typename Kernel>
struct enable_sm75_to_sm80 : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE static void invoke(Args&&... args) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
    Kernel::invoke(std::forward<Args>(args)...);
#endif
  }
};

template <typename Kernel>
struct enable_sm80_to_sm89 : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE static void invoke(Args&&... args) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 890
    Kernel::invoke(std::forward<Args>(args)...);
#endif
  }
};

template <typename Kernel>
struct enable_sm89_to_sm90 : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE static void invoke(Args&&... args) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 890 && __CUDA_ARCH__ < 900
    Kernel::invoke(std::forward<Args>(args)...);
#endif
  }
};

/*
 * This class provides the common ScaleA and ScaleB descriptors for the
 * ScaledEpilogue and ScaledEpilogueBias classes.
 */
template <typename ElementD, typename OutputTileThreadMap>
struct ScaledEpilogueBase {
 protected:
  using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;

  using ScaleA = cutlass::epilogue::threadblock::VisitorColOrScalarBroadcast<
      OutputTileThreadMap, float, Stride<Int<1>, Int<0>, Int<0>>>;

  using ScaleB = cutlass::epilogue::threadblock::VisitorRowOrScalarBroadcast<
      OutputTileThreadMap, float, Stride<Int<0>, Int<1>, Int<0>>>;
};

/*
 This epilogue function defines a quantized GEMM operation similar to
 torch._scaled_mm.

 A and B may be both either int8 or fp8_e4m3. A can be quantized per-tensor or
 per-row. B can be quantized per-tensor or per-column.
 Any combination of per-tensor and per-row or column is supported.
 A and B must have symmetric quantization (zero point == 0).

 So the GEMM operation is D = (a_scales * A) (b_scales * B), where the
 scales are applied elementwise with numpy-style broadcasting.

 ScaleA and ScaleB define the epilogue functions that apply the scales for
 the A and B operands respectively. These scales may be either per-tensor or
 per row or column.
*/
template <typename ElementD, typename OutputTileThreadMap>
struct ScaledEpilogue
    : private ScaledEpilogueBase<ElementD, OutputTileThreadMap> {
 private:
  using SUPER = ScaledEpilogueBase<ElementD, OutputTileThreadMap>;
  using Accum = typename SUPER::Accum;
  using ScaleA = typename SUPER::ScaleA;
  using ScaleB = typename SUPER::ScaleB;

  using Compute0 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::threadblock::Sm80EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, ElementD, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

 public:
  using EVTCompute =
      cutlass::epilogue::threadblock::Sm80EVT<Compute1, ScaleA, EVTCompute0>;
  using ArgumentType = typename EVTCompute::Arguments;

  static ArgumentType prepare_args(torch::Tensor const& a_scales,
                                   torch::Tensor const& b_scales) {
    using ScaleAArgs = typename ScaleA::Arguments;
    using ScaleBArgs = typename ScaleB::Arguments;

    ScaleBArgs b_args{b_scales.data_ptr<float>(), b_scales.numel() != 1, {}};
    ScaleAArgs a_args{a_scales.data_ptr<float>(), a_scales.numel() != 1, {}};

    typename EVTCompute0::Arguments evt0_compute_args{b_args};

    typename EVTCompute::Arguments evt_compute_args{a_args, evt0_compute_args};
    return evt_compute_args;
  }
};

template <typename ElementD, typename OutputTileThreadMap>
struct ScaledEpilogueBias
    : private ScaledEpilogueBase<ElementD, OutputTileThreadMap> {
 private:
  using SUPER = ScaledEpilogueBase<ElementD, OutputTileThreadMap>;
  using Accum = typename SUPER::Accum;
  using ScaleA = typename SUPER::ScaleA;
  using ScaleB = typename SUPER::ScaleB;

  using Compute0 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::threadblock::Sm80EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiply_add, ElementD, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using Bias = cutlass::epilogue::threadblock::VisitorRowBroadcast<
      OutputTileThreadMap, ElementD, Stride<Int<0>, Int<1>, Int<0>>>;

 public:
  using EVTCompute = cutlass::epilogue::threadblock::Sm80EVT<Compute1, ScaleA,
                                                             EVTCompute0, Bias>;
  using ArgumentType = typename EVTCompute::Arguments;

  static ArgumentType prepare_args(torch::Tensor const& a_scales,
                                   torch::Tensor const& b_scales,
                                   torch::Tensor const& bias) {
    using ScaleAArgs = typename ScaleA::Arguments;
    using ScaleBArgs = typename ScaleB::Arguments;
    using BiasArgs = typename Bias::Arguments;

    ScaleBArgs b_args{b_scales.data_ptr<float>(), b_scales.numel() != 1, {}};
    ScaleAArgs a_args{a_scales.data_ptr<float>(), a_scales.numel() != 1, {}};
    BiasArgs bias_args{static_cast<ElementD*>(bias.data_ptr()), {}};

    typename EVTCompute0::Arguments evt0_compute_args{b_args};

    typename EVTCompute::Arguments evt_compute_args{a_args, evt0_compute_args,
                                                    bias_args};
    return evt_compute_args;
  }
};

template <typename Arch, template <typename> typename ArchGuard,
          typename ElementAB_, typename ElementD_,
          template <typename, typename> typename Epilogue_, typename TileShape,
          typename WarpShape, typename InstructionShape, int32_t MainLoopStages>
struct cutlass_2x_gemm {
  using ElementAB = ElementAB_;
  using ElementD = ElementD_;

  using ElementAcc =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>, int32_t,
                                float>::type;

  using Operator =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>,
                                cutlass::arch::OpMultiplyAddSaturate,
                                cutlass::arch::OpMultiplyAdd>::type;

  using OutputTileThreadMap =
      cutlass::epilogue::threadblock::OutputTileThreadLayout<
          TileShape, WarpShape, float, 4, 1 /* epilogue stages */
          >;

  using Epilogue = Epilogue_<ElementD, OutputTileThreadMap>;
  using EVTCompute = typename Epilogue::EVTCompute;

  using D = cutlass::epilogue::threadblock::VisitorAuxStore<
      OutputTileThreadMap, ElementD, cutlass::FloatRoundStyle::round_to_nearest,
      Stride<int64_t, Int<1>, Int<0>>>;

  using EVTD = cutlass::epilogue::threadblock::Sm80EVT<D, EVTCompute>;

  // clang-format off
  using RowMajor = typename cutlass::layout::RowMajor;
  using ColumnMajor = typename cutlass::layout::ColumnMajor;
  using KernelType =
    ArchGuard<typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
      ElementAB, RowMajor, cutlass::ComplexTransform::kNone, 16,
      ElementAB, ColumnMajor, cutlass::ComplexTransform::kNone, 16,
      float, cutlass::layout::RowMajor, 4,
      ElementAcc, float, cutlass::arch::OpClassTensorOp,
      Arch,
      TileShape, WarpShape, InstructionShape,
      EVTD,
      cutlass::gemm::threadblock::ThreadblockSwizzleStreamK,
      MainLoopStages, Operator,
      1 /* epilogue stages */
      >::GemmKernel>;
  // clang-format on

  using Op = cutlass::gemm::device::GemmUniversalAdapter<KernelType>;
};

template <typename Gemm, typename... EpilogueArgs>
void cutlass_gemm_caller(torch::Tensor& out, torch::Tensor const& a,
                         torch::Tensor const& b,
                         EpilogueArgs&&... epilogue_params) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);
  cutlass::gemm::GemmCoord problem_size{m, n, k};

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  using StrideC = Stride<int64_t, Int<1>, Int<0>>;
  StrideC c_stride{ldc, Int<1>{}, Int<0>{}};

  auto a_ptr = static_cast<ElementAB const*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB const*>(b.data_ptr());
  auto c_ptr = static_cast<ElementD*>(out.data_ptr());

  typename Gemm::D::Arguments d_args{c_ptr, c_stride};

  using Epilogue = typename Gemm::Epilogue;
  auto evt_args =
      Epilogue::prepare_args(std::forward<EpilogueArgs>(epilogue_params)...);

  typename Gemm::EVTD::Arguments epilogue_args{
      evt_args,
      d_args,
  };

  typename Gemm::Op::Arguments args{
      cutlass::gemm::GemmUniversalMode::kGemmSplitKParallel,  // universal mode
      problem_size,                                           // problem size
      1,                                                      // batch count
      epilogue_args,
      a_ptr,
      b_ptr,
      nullptr,
      nullptr,
      0,
      0,
      0,
      0,
      lda,
      ldb,
      ldc,
      ldc};

  // Launch the CUTLASS GEMM kernel.
  typename Gemm::Op gemm_op;
  size_t workspace_size = gemm_op.get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  CUTLASS_CHECK(gemm_op.can_implement(args));
  cutlass::Status status = gemm_op(args, workspace.get(), stream);
  CUTLASS_CHECK(status);
}

template <typename Gemm, typename FallbackGemm, typename... EpilogueArgs>
void fallback_cutlass_gemm_caller(torch::Tensor& out, torch::Tensor const& a,
                                  torch::Tensor const& b,
                                  EpilogueArgs&&... args) {
  // In some cases, the GPU isn't able to accommodate the
  // shared memory requirements of the Gemm. In such cases, use
  // the FallbackGemm instead.
  static const int max_shared_mem_per_block_opt_in =
      get_cuda_max_shared_memory_per_block_opt_in(0);

  size_t const gemm_shared_mem_size =
      sizeof(typename Gemm::KernelType::SharedStorage);
  size_t const fallback_gemm_shared_mem_size =
      sizeof(typename FallbackGemm::KernelType::SharedStorage);

  if (gemm_shared_mem_size <= max_shared_mem_per_block_opt_in) {
    return cutlass_gemm_caller<Gemm>(out, a, b,
                                     std::forward<EpilogueArgs>(args)...);
  } else {
    TORCH_CHECK(fallback_gemm_shared_mem_size <=
                max_shared_mem_per_block_opt_in);
    return cutlass_gemm_caller<FallbackGemm>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  }
}

template <typename InType, typename OutType,
          template <typename, typename> typename Epilogue>
struct sm80_config_default {
  // This config is used in 2 cases,
  //  - M in (128, inf)
  //  - M in (64, 128] and N >= 8192
  // Shared Memory required by this Gemm - 81920 bytes
  static_assert(std::is_same<InType, int8_t>());
  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;
  using Cutlass2xGemm =
      cutlass_2x_gemm<cutlass::arch::Sm80, enable_sm80_to_sm89, InType, OutType,
                      Epilogue, TileShape, WarpShape, InstructionShape, 5>;
};

template <typename InType, typename OutType,
          template <typename, typename> typename Epilogue>
struct sm80_config_M64 {
  // This config is used in 2 cases,
  // - M in (32, 64]
  // - M in (64, 128] and N < 8192
  // Shared Memory required by this Gemm - 122880 bytes
  static_assert(std::is_same<InType, int8_t>());
  using TileShape = typename cutlass::gemm::GemmShape<64, 128, 128>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;
  using Cutlass2xGemm =
      cutlass_2x_gemm<cutlass::arch::Sm80, enable_sm80_to_sm89, InType, OutType,
                      Epilogue, TileShape, WarpShape, InstructionShape, 5>;
};

template <typename InType, typename OutType,
          template <typename, typename> typename Epilogue>
struct sm80_config_M32 {
  // M in (16, 32]
  // Shared Memory required by this Gemm - 61440 bytes
  static_assert(std::is_same<InType, int8_t>());
  using TileShape = typename cutlass::gemm::GemmShape<32, 64, 128>;
  using WarpShape = typename cutlass::gemm::GemmShape<32, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;
  using Cutlass2xGemm =
      cutlass_2x_gemm<cutlass::arch::Sm80, enable_sm80_to_sm89, InType, OutType,
                      Epilogue, TileShape, WarpShape, InstructionShape, 5>;
};

template <typename InType, typename OutType,
          template <typename, typename> typename Epilogue>
struct sm80_config_M16 {
  // M in [1, 16]
  // Shared Memory required by this Gemm - 51200 bytes
  static_assert(std::is_same<InType, int8_t>());
  using TileShape = typename cutlass::gemm::GemmShape<16, 64, 128>;
  using WarpShape = typename cutlass::gemm::GemmShape<16, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;
  using Cutlass2xGemm =
      cutlass_2x_gemm<cutlass::arch::Sm80, enable_sm80_to_sm89, InType, OutType,
                      Epilogue, TileShape, WarpShape, InstructionShape, 5>;
};

}  // namespace

template <typename InType, typename OutType,
          template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm80_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                torch::Tensor const& b,
                                EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, int8_t>());
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  using Cutlass2xGemmDefault =
      typename sm80_config_default<InType, OutType, Epilogue>::Cutlass2xGemm;
  using Cutlass2xGemmM128BigN =
      typename sm80_config_default<InType, OutType, Epilogue>::Cutlass2xGemm;
  using Cutlass2xGemmM128SmallN =
      typename sm80_config_M64<InType, OutType, Epilogue>::Cutlass2xGemm;
  using Cutlass2xGemmM64 =
      typename sm80_config_M64<InType, OutType, Epilogue>::Cutlass2xGemm;
  using Cutlass2xGemmM32 =
      typename sm80_config_M32<InType, OutType, Epilogue>::Cutlass2xGemm;
  using Cutlass2xGemmM16 =
      typename sm80_config_M16<InType, OutType, Epilogue>::Cutlass2xGemm;

  // Due to shared memory requirements, some Gemms may fail to run on some
  // GPUs. As the name indicates, the Fallback Gemm is used as an alternative
  // in such cases.
  // sm80_config_M16 has the least shared-memory requirement. However,
  // based on some profiling, we select sm80_config_M32 as a better alternative
  // performance wise.
  using FallbackGemm =
      typename sm80_config_M32<InType, OutType, Epilogue>::Cutlass2xGemm;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(16), next_pow_2(m));  // next power of 2
  if (mp2 <= 16) {
    // M in [1, 16]
    return fallback_cutlass_gemm_caller<Cutlass2xGemmM16, FallbackGemm>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 32) {
    // M in (16, 32]
    return fallback_cutlass_gemm_caller<Cutlass2xGemmM32, FallbackGemm>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 64) {
    // M in (32, 64]
    return fallback_cutlass_gemm_caller<Cutlass2xGemmM64, FallbackGemm>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 128) {
    // M in (64, 128]
    uint32_t const n = out.size(1);
    bool const small_n = n < 8192;
    if (small_n) {
      return fallback_cutlass_gemm_caller<Cutlass2xGemmM128SmallN,
                                          FallbackGemm>(
          out, a, b, std::forward<EpilogueArgs>(args)...);
    } else {
      return fallback_cutlass_gemm_caller<Cutlass2xGemmM128BigN, FallbackGemm>(
          out, a, b, std::forward<EpilogueArgs>(args)...);
    }
  } else {
    // M in (128, inf)
    return fallback_cutlass_gemm_caller<Cutlass2xGemmDefault, FallbackGemm>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  }
}

template <template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm75_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<8, 8, 16>;

  if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm75, enable_sm75_to_sm80, int8_t, cutlass::bfloat16_t,
        Epilogue, TileShape, WarpShape, InstructionShape, 2>>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm75, enable_sm75_to_sm80, int8_t, cutlass::half_t,
        Epilogue, TileShape, WarpShape, InstructionShape, 2>>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  }
}

void cutlass_scaled_mm_sm75(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_mm_sm75_epilogue<ScaledEpilogueBias>(
        out, a, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_mm_sm75_epilogue<ScaledEpilogue>(out, a, b, a_scales,
                                                           b_scales);
  }
}

template <template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm80_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_sm80_dispatch<int8_t, cutlass::bfloat16_t, Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_sm80_dispatch<int8_t, cutlass::half_t, Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  }
}

void cutlass_scaled_mm_sm80(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_mm_sm80_epilogue<ScaledEpilogueBias>(
        out, a, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_mm_sm80_epilogue<ScaledEpilogue>(out, a, b, a_scales,
                                                           b_scales);
  }
}

template <template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm89_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, enable_sm89_to_sm90, int8_t, cutlass::bfloat16_t,
          Epilogue, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      assert(out.dtype() == torch::kFloat16);
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, enable_sm89_to_sm90, int8_t, cutlass::half_t,
          Epilogue, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_caller<
          cutlass_2x_gemm<cutlass::arch::Sm89, enable_sm89_to_sm90,
                          cutlass::float_e4m3_t, cutlass::bfloat16_t, Epilogue,
                          TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_caller<
          cutlass_2x_gemm<cutlass::arch::Sm89, enable_sm89_to_sm90,
                          cutlass::float_e4m3_t, cutlass::half_t, Epilogue,
                          TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    }
  }
}

void cutlass_scaled_mm_sm89(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_mm_sm89_epilogue<ScaledEpilogueBias>(
        out, a, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_mm_sm89_epilogue<ScaledEpilogue>(out, a, b, a_scales,
                                                           b_scales);
  }
}
