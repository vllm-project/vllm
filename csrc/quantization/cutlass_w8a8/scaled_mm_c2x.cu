#include <stddef.h>
#include <torch/all.h>
#include "cutlass/cutlass.h"

#include "scaled_mm_c2x.cuh"
#include "scaled_mm_c2x_sm75_dispatch.cuh"
#include "scaled_mm_c2x_sm80_dispatch.cuh"
#include "scaled_mm_c2x_sm89_fp8_dispatch.cuh"
#include "scaled_mm_c2x_sm89_int8_dispatch.cuh"

/*
   This file defines quantized GEMM operations using the CUTLASS 2.x API, for
   NVIDIA GPUs with SM versions prior to sm90 (Hopper).
*/

template <template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm75_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  if (out.dtype() == torch::kBFloat16) {
    return vllm::cutlass_gemm_sm75_dispatch<int8_t, cutlass::bfloat16_t,
                                            Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return vllm::cutlass_gemm_sm75_dispatch<int8_t, cutlass::half_t, Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  }
}

void cutlass_scaled_mm_sm75(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_mm_sm75_epilogue<vllm::ScaledEpilogueBias>(
        out, a, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_mm_sm75_epilogue<vllm::ScaledEpilogue>(
        out, a, b, a_scales, b_scales);
  }
}

template <template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm80_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  if (out.dtype() == torch::kBFloat16) {
    return vllm::cutlass_gemm_sm80_dispatch<int8_t, cutlass::bfloat16_t,
                                            Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return vllm::cutlass_gemm_sm80_dispatch<int8_t, cutlass::half_t, Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  }
}

void cutlass_scaled_mm_sm80(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_mm_sm80_epilogue<vllm::ScaledEpilogueBias>(
        out, a, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_mm_sm80_epilogue<vllm::ScaledEpilogue>(
        out, a, b, a_scales, b_scales);
  }
}

template <template <typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm89_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    if (out.dtype() == torch::kBFloat16) {
      return vllm::cutlass_gemm_sm89_int8_dispatch<int8_t, cutlass::bfloat16_t,
                                                   Epilogue>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      assert(out.dtype() == torch::kFloat16);
      return vllm::cutlass_gemm_sm89_int8_dispatch<int8_t, cutlass::half_t,
                                                   Epilogue>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kBFloat16) {
      return vllm::cutlass_gemm_sm89_fp8_dispatch<
          cutlass::float_e4m3_t, cutlass::bfloat16_t, Epilogue>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return vllm::cutlass_gemm_sm89_fp8_dispatch<cutlass::float_e4m3_t,
                                                  cutlass::half_t, Epilogue>(
          out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
    }
  }
}

void cutlass_scaled_mm_sm89(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_mm_sm89_epilogue<vllm::ScaledEpilogueBias>(
        out, a, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_mm_sm89_epilogue<vllm::ScaledEpilogue>(
        out, a, b, a_scales, b_scales);
  }
}
