#include "hip/hip_runtime.h"
// clang-format will break include orders
// clang-format off
#include <cudaTypedefs.h>

#if defined CUDA_VERSION && CUDA_VERSION >= 12000

#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>

#include <iostream>
#include <sstream>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/util/device_memory.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"

#include "broadcast_load_epilogue_c3x.hpp"
#include "common.hpp"
// clang-format on

using namespace cute;

/*
   This defines a quantized GEMM operation with dequantized output, similar to
   torch._scaled_mm. It is defined using the CUTLASS 3.x API, and is used for
   NVIDIA GPUs with sm90a (Hopper) or later.

   A and B may be both either int8 or fp8_e4m3. A can be quantized per-tensor or
   per-row. B can be quantized per-tensor or per-column.
   Any combination of per-tensor and per-row or column is supported.
   A and B must have symmetric quantization (zero point == 0).

   So the GEMM operation is D = (a_scales * A) (b_scales * B), where the
   scales are applied elementwise with numpy-style broadcasting.

   ScaleA and ScaleB define the epilogue functions that apply the scales for
   the A and B operands respectively. These scales may be either per-tensor or
   per row or column.
*/

namespace {

uint32_t next_pow_2(uint32_t const num) {
  if (num <= 1) return num;
  return 1 << (CHAR_BIT * sizeof(num) - __builtin_clz(num - 1));
}

// A wrapper for the GEMM kernel that is used to guard against compilation on
// architectures that will never use the kernel. The purpose of this is to
// reduce the size of the compiled binary.
// __CUDA_ARCH__ is not defined in host code, so this lets us smuggle the ifdef
// into code that will be executed on the device where it is defined.
template <typename Kernel>
struct enable_sm90_or_later : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE void operator()(Args&&... args) {
  #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 900
    Kernel::operator()(std::forward<Args>(args)...);
  #endif
  }
};

template <typename ElementAB_, typename ElementD_, typename TileShape,
          typename ClusterShape, typename KernelSchedule,
          typename EpilogueSchedule>
struct cutlass_3x_gemm {
  using ElementAB = ElementAB_;
  using ElementD = ElementD_;
  using ElementAcc =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>, int32_t,
                                float>::type;

  using EpilogueDescriptor =
      cutlass::epilogue::collective::detail::EpilogueDescriptor<
          TileShape, cutlass::epilogue::collective::EpilogueTileAuto, ElementD,
          ElementD, EpilogueSchedule>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using ScaleA = cutlass::epilogue::fusion::Sm90ColOrScalarBroadcast<
      0 /*Stages*/, typename EpilogueDescriptor::TileShape, float,
      Stride<Int<1>, Int<0>, Int<0>>>;

  using ScaleBDescriptor =
      cutlass::epilogue::collective::detail::RowBroadcastDescriptor<
          EpilogueDescriptor, float>;

  using ScaleB = cutlass::epilogue::fusion::Sm90RowOrScalarBroadcast<
      ScaleBDescriptor::Stages, typename EpilogueDescriptor::TileShape,
      typename ScaleBDescriptor::Element, Stride<Int<0>, Int<1>, Int<0>>>;

  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::fusion::Sm90EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies, ElementD, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute1 =
      cutlass::epilogue::fusion::Sm90EVT<Compute1, ScaleA, EVTCompute0>;

  using StrideD = Stride<int64_t, Int<1>, Int<0>>;
  using ElementC = void;
  using StrideC = StrideD;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp, TileShape,
          ClusterShape, cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAcc, float, ElementC, StrideC, 4, ElementD, StrideD, 4,
          EpilogueSchedule, EVTCompute1>::CollectiveOp;

  static constexpr size_t CEStorageSize =
      sizeof(typename CollectiveEpilogue::SharedStorage);
  using Stages = typename cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(CEStorageSize)>;

  // clang-format off
  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp, 
          ElementAB, cutlass::layout::RowMajor, 16, 
          ElementAB, cutlass::layout::ColumnMajor, 16, 
          ElementAcc, TileShape, ClusterShape,
          Stages,
          KernelSchedule>::CollectiveOp;
  // clang-format on

  using KernelType = enable_sm90_or_later<cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue,
      cutlass::gemm::PersistentScheduler>>;

  struct GemmKernel : public KernelType {};
};

template <typename Gemm>
void cutlass_scaled_mm_dq_dispatcher(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     torch::Tensor const& a_scales,
                                     torch::Tensor const& b_scales) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  using StrideA = Stride<int64_t, Int<1>, Int<0>>;
  using StrideB = Stride<int64_t, Int<1>, Int<0>>;
  using StrideC = typename Gemm::StrideC;

  StrideA a_stride{lda, Int<1>{}, Int<0>{}};
  StrideB b_stride{ldb, Int<1>{}, Int<0>{}};
  StrideC c_stride{ldc, Int<1>{}, Int<0>{}};

  using GemmKernel = typename Gemm::GemmKernel;
  typename GemmKernel::ProblemShape prob_shape{m, n, k, 1};

  auto a_ptr = static_cast<ElementAB*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB*>(b.data_ptr());
  typename GemmKernel::MainloopArguments mainloop_args{a_ptr, a_stride, b_ptr,
                                                       b_stride};

  auto c_ptr = static_cast<ElementD*>(out.data_ptr());
  typename GemmKernel::EpilogueArguments epilogue_args{
      {}, c_ptr, c_stride, c_ptr, c_stride};

  typename GemmKernel::Arguments args{cutlass::gemm::GemmUniversalMode::kGemm,
                                      prob_shape, mainloop_args, epilogue_args};

  using ScaleA_Args = typename Gemm::ScaleA::Arguments;
  using ScaleB_Args = typename Gemm::ScaleB::Arguments;

  ScaleA_Args a_args{a_scales.data_ptr<float>(), a_scales.numel() != 1, {}};
  ScaleB_Args b_args{b_scales.data_ptr<float>(), b_scales.numel() != 1, {}};

  args.epilogue.thread = {a_args, {b_args}};

  // Launch the CUTLASS GEMM kernel.
  using GemmOp = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  GemmOp gemm_op;
  CUTLASS_CHECK(gemm_op.can_implement(args));

  size_t workspace_size = gemm_op.get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  cutlass::Status status = gemm_op.run(args, workspace.get(), stream);
  CUTLASS_CHECK(status);
}

template <typename InType, typename OutType, int32_t M>
struct sm90_fp8_config {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_2, _1, _1>;

  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, TileShape, ClusterShape, KernelSchedule,
                      EpilogueSchedule>;
};

template <typename InType, typename OutType>
struct sm90_fp8_config<InType, OutType, 128> {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _128, _128>;
  using ClusterShape = Shape<_2, _1, _1>;

  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, TileShape, ClusterShape, KernelSchedule,
                      EpilogueSchedule>;
};

template <typename InType, typename OutType>
struct sm90_fp8_config<InType, OutType, 64> {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _64, _128>;
  using ClusterShape = Shape<_1, _8, _1>;

  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, TileShape, ClusterShape, KernelSchedule,
                      EpilogueSchedule>;
};

}  // namespace

template <typename InType, typename OutType>
void cutlass_scaled_mm_dq_sm90_fp8_dispatch(torch::Tensor& out,
                                            torch::Tensor const& a,
                                            torch::Tensor const& b,
                                            torch::Tensor const& a_scales,
                                            torch::Tensor const& b_scales) {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  using Cutlass3xGemmDefault =
      typename sm90_fp8_config<InType, OutType, 0>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_fp8_config<InType, OutType, 64>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_fp8_config<InType, OutType, 128>::Cutlass3xGemm;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(64), next_pow_2(m));  // next power of 2

  if (mp2 <= 64) {
    // m in [1, 64]
    return cutlass_scaled_mm_dq_dispatcher<Cutlass3xGemmM64>(
        out, a, b, a_scales, b_scales);
  } else if (mp2 <= 128) {
    // m in (64, 128]
    return cutlass_scaled_mm_dq_dispatcher<Cutlass3xGemmM128>(
        out, a, b, a_scales, b_scales);
  } else {
    // m in (128, inf)
    return cutlass_scaled_mm_dq_dispatcher<Cutlass3xGemmDefault>(
        out, a, b, a_scales, b_scales);
  }
}

void cutlass_scaled_mm_dq_sm90(torch::Tensor& out, torch::Tensor const& a,
                               torch::Tensor const& b,
                               torch::Tensor const& a_scales,
                               torch::Tensor const& b_scales) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    using TileShape = Shape<_128, _128, _128>;
    using ClusterShape = Shape<_1, _2, _1>;
    using KernelSchedule =
        typename cutlass::gemm::KernelTmaWarpSpecializedPingpong;
    using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_scaled_mm_dq_dispatcher<
          cutlass_3x_gemm<int8_t, cutlass::bfloat16_t, TileShape, ClusterShape,
                          KernelSchedule, EpilogueSchedule>>(
          out, a, b, a_scales, b_scales);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);

      return cutlass_scaled_mm_dq_dispatcher<
          cutlass_3x_gemm<int8_t, cutlass::half_t, TileShape, ClusterShape,
                          KernelSchedule, EpilogueSchedule>>(
          out, a, b, a_scales, b_scales);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_scaled_mm_dq_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                                    cutlass::bfloat16_t>(
          out, a, b, a_scales, b_scales);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_scaled_mm_dq_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                                    cutlass::half_t>(
          out, a, b, a_scales, b_scales);
    }
  }
}

#endif
