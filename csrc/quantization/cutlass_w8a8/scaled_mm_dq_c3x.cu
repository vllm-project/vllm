#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>

#include <iostream>
#include <sstream>
#include <vector>

// clang-format will break include orders
// clang-format off
#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"

#include "common.hpp"
// clang-format on

using namespace cute;

/*
   This defines a quantized GEMM operation with dequantized output, similar to
   torch._scaled_mm. It is defined using the CUTLASS 3.x API, and is used for
   NVIDIA GPUs with sm90a (Hopper) or later.

   A and B may be both either int8 or fp8_e4m3. A can be quantized per-tensor or
   per-row. B can be quantized per-tensor or per-column.
   Any combination of per-tensor and per-row or column is supported.
   A and B must have symmetric quantization (zero point == 0).

   So the GEMM operation is D = (a_scales * A) (b_scales * B), where the
   scales are applied elementwise with numpy-style broadcasting.

   ScaleA and ScaleB define the epilogue functions that apply the scales for
   the A and B operands respectively. These scales may be either per-tensor or
   per row or column.
*/

namespace {

template <typename ElementAB_, typename ElementD_, typename TileShape,
          typename ClusterShape, typename KernelSchedule,
          typename EpilogueSchedule>
struct cutlass_3x_gemm {
  using ElementAB = ElementAB_;
  using ElementD = ElementD_;
  using ElementAcc =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>, int32_t,
                                float>::type;

  using EpilogueDescriptor =
      cutlass::epilogue::collective::detail::EpilogueDescriptor<
          TileShape, cutlass::epilogue::collective::EpilogueTileAuto, ElementD,
          ElementD, EpilogueSchedule>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using ScaleA = cutlass::epilogue::fusion::Sm90ColBroadcast<
      0 /*Stages*/, typename EpilogueDescriptor::TileShape, float,
      Stride<Int<1>, Int<0>, Int<0>>>;

  using ScaleBDescriptor =
      cutlass::epilogue::collective::detail::RowBroadcastDescriptor<
          EpilogueDescriptor, float>;

  using ScaleB = cutlass::epilogue::fusion::Sm90RowBroadcast<
      ScaleBDescriptor::Stages, typename EpilogueDescriptor::TileShape,
      typename ScaleBDescriptor::Element, Stride<Int<0>, Int<1>, Int<0>>>;

  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::fusion::Sm90EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies, ElementD, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute1 =
      cutlass::epilogue::fusion::Sm90EVT<Compute1, ScaleA, EVTCompute0>;

  using StrideD = Stride<int64_t, Int<1>, Int<0>>;
  using ElementC = void;
  using StrideC = StrideD;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp, TileShape,
          ClusterShape, cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAcc, float, ElementC, StrideC, 4, ElementD, StrideD, 4,
          EpilogueSchedule, EVTCompute1>::CollectiveOp;

  static constexpr size_t CEStorageSize =
      sizeof(typename CollectiveEpilogue::SharedStorage);
  using Stages = typename cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(CEStorageSize)>;

  // clang-format off
  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp, 
          ElementAB, cutlass::layout::RowMajor, 16, 
          ElementAB, cutlass::layout::ColumnMajor, 16, 
          ElementAcc, TileShape, ClusterShape,
          Stages,
          KernelSchedule>::CollectiveOp;
  // clang-format on

  using KernelType = cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue,
      cutlass::gemm::PersistentScheduler>;

  struct GemmKernel : public KernelType {};
};

template <typename Gemm>
void cutlass_scaled_mm_dq_dispatcher(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     torch::Tensor const& a_scales,
                                     torch::Tensor const& b_scales) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  using StrideA = Stride<int64_t, Int<1>, Int<0>>;
  using StrideB = Stride<int64_t, Int<1>, Int<0>>;
  using StrideC = typename Gemm::StrideC;

  StrideA a_stride{lda, Int<1>{}, Int<0>{}};
  StrideB b_stride{ldb, Int<1>{}, Int<0>{}};
  StrideC c_stride{ldc, Int<1>{}, Int<0>{}};

  using GemmKernel = typename Gemm::GemmKernel;
  typename GemmKernel::ProblemShape prob_shape{m, n, k, 1};

  auto a_ptr = static_cast<ElementAB*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB*>(b.data_ptr());
  typename GemmKernel::MainloopArguments mainloop_args{a_ptr, a_stride, b_ptr,
                                                       b_stride};

  auto c_ptr = static_cast<ElementD*>(out.data_ptr());
  typename GemmKernel::EpilogueArguments epilogue_args{
      {}, c_ptr, c_stride, c_ptr, c_stride};

  typename GemmKernel::Arguments args{cutlass::gemm::GemmUniversalMode::kGemm,
                                      prob_shape, mainloop_args, epilogue_args};

  using ScaleA_Args = typename Gemm::ScaleA::Arguments;
  using ScaleB_Args = typename Gemm::ScaleB::Arguments;
  ScaleA_Args a_args = a_scales.numel() == 1
                           ? ScaleA_Args{nullptr, a_scales.item<float>(), {}}
                           : ScaleA_Args{a_scales.data_ptr<float>(), {}, {}};

  ScaleB_Args b_args = b_scales.numel() == 1
                           ? ScaleB_Args{nullptr, b_scales.item<float>(), {}}
                           : ScaleB_Args{b_scales.data_ptr<float>(), {}, {}};

  args.epilogue.thread = {a_args, {b_args}};

  // Launch the CUTLASS GEMM kernel.
  using GemmOp = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  GemmOp gemm_op;
  CUTLASS_CHECK(gemm_op.can_implement(args));

  size_t workspace_size = gemm_op.get_workspace_size(args);
  TORCH_CHECK(workspace_size == 0);

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());
  cutlass::Status status = gemm_op.run(args, stream);
  CUTLASS_CHECK(status);
}
}  // namespace

void cutlass_scaled_mm_dq_sm90(torch::Tensor& out, torch::Tensor const& a,
                               torch::Tensor const& b,
                               torch::Tensor const& a_scales,
                               torch::Tensor const& b_scales) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    using TileShape = Shape<_128, _128, _128>;
    using ClusterShape = Shape<_1, _2, _1>;
    using KernelSchedule =
        typename cutlass::gemm::KernelTmaWarpSpecializedPingpong;
    using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_scaled_mm_dq_dispatcher<
          cutlass_3x_gemm<int8_t, cutlass::bfloat16_t, TileShape, ClusterShape,
                          KernelSchedule, EpilogueSchedule>>(
          out, a, b, a_scales, b_scales);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);

      return cutlass_scaled_mm_dq_dispatcher<
          cutlass_3x_gemm<int8_t, cutlass::half_t, TileShape, ClusterShape,
                          KernelSchedule, EpilogueSchedule>>(
          out, a, b, a_scales, b_scales);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    using TileShape = Shape<_128, _128, _128>;
    using ClusterShape = Shape<_1, _2, _1>;
    using KernelSchedule =
        typename cutlass::gemm::KernelCpAsyncWarpSpecializedCooperative;
    using EpilogueSchedule =
        typename cutlass::epilogue::TmaWarpSpecializedCooperative;

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_scaled_mm_dq_dispatcher<
          cutlass_3x_gemm<cutlass::float_e4m3_t, cutlass::bfloat16_t, TileShape,
                          ClusterShape, KernelSchedule, EpilogueSchedule>>(
          out, a, b, a_scales, b_scales);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);

      return cutlass_scaled_mm_dq_dispatcher<
          cutlass_3x_gemm<cutlass::float_e4m3_t, cutlass::half_t, TileShape,
                          ClusterShape, KernelSchedule, EpilogueSchedule>>(
          out, a, b, a_scales, b_scales);
    }
  }
}
