#include "hip/hip_runtime.h"
#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#include <iostream>

constexpr uint64_t THREADS_PER_EXPERT = 512;

__global__ void compute_problem_sizes(const int* __restrict__ topk_ids,
                                      int32_t* problem_sizes1,
                                      int32_t* problem_sizes2,
                                      int32_t* atomic_buffer,
                                      const int topk_length, const int n,
                                      const int k) {
  int expert_id = blockIdx.x;

  int occurrences = 0;
  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    occurrences += (topk_ids[i] == expert_id);
  }
  atomicAdd(&atomic_buffer[expert_id], occurrences);
  __syncthreads();

  if (threadIdx.x == 0) {
    int final_occurrences = atomic_buffer[expert_id];
    problem_sizes1[expert_id * 3] = final_occurrences;
    problem_sizes1[expert_id * 3 + 1] = 2 * n;
    problem_sizes1[expert_id * 3 + 2] = k;
    problem_sizes2[expert_id * 3] = final_occurrences;
    problem_sizes2[expert_id * 3 + 1] = k;
    problem_sizes2[expert_id * 3 + 2] = n;
  }
}

__global__ void compute_expert_offsets(
    const int32_t* __restrict__ problem_sizes1, int32_t* expert_offsets,
    int32_t* atomic_buffer, const int num_experts) {
  int32_t tot_offset = 0;
  expert_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    tot_offset += problem_sizes1[i * 3];
    expert_offsets[i + 1] = tot_offset;
  }
}

__global__ void compute_arg_sorts(const int* __restrict__ topk_ids,
                                  const int32_t* __restrict__ expert_offsets,
                                  int32_t* input_permutation,
                                  int32_t* output_permutation,
                                  int32_t* atomic_buffer, const int topk_length,
                                  const int topk) {
  int const blk_expert_id = blockIdx.x;
  int const num_experts = gridDim.x;
  int32_t const num_tokens = expert_offsets[num_experts];

  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    int const expert_id = topk_ids[i];
    if (expert_id == -1 && blockIdx.x == 0) {
      // output_permutation is used to re-order the moe outputs. It is
      // used as c2 = c2[c_map], where c2 is a torch.tensor that is the
      // output of the cutlass kernels and c_map is the output_permutation.
      // c2 is initialized to zeros, therefore by setting the output_permutation
      // to num_tokens, we are guaranteed to fill the moe outputs to zero
      // for "invalid" topk_ids.
      output_permutation[i] = num_tokens;
    } else if (expert_id == blk_expert_id) {
      int start = atomicAdd(&atomic_buffer[expert_id], 1);
      input_permutation[start] = i / topk;
      output_permutation[i] = start;
    }
  }
}

void get_cutlass_moe_mm_data_caller(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation, torch::Tensor& output_permutation,
    const int64_t num_experts, const int64_t n, const int64_t k) {
  auto stream = at::cuda::getCurrentCUDAStream(topk_ids.device().index());
  auto options_int32 =
      torch::TensorOptions().dtype(torch::kInt32).device(topk_ids.device());
  torch::Tensor atomic_buffer = torch::zeros(num_experts, options_int32);

  int num_threads = min(THREADS_PER_EXPERT, topk_ids.numel());
  compute_problem_sizes<<<num_experts, num_threads, 0, stream>>>(
      static_cast<const int32_t*>(topk_ids.data_ptr()),
      static_cast<int32_t*>(problem_sizes1.data_ptr()),
      static_cast<int32_t*>(problem_sizes2.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()), topk_ids.numel(), n, k);
  compute_expert_offsets<<<1, 1, 0, stream>>>(
      static_cast<const int32_t*>(problem_sizes1.data_ptr()),
      static_cast<int32_t*>(expert_offsets.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()), num_experts);
  compute_arg_sorts<<<num_experts, num_threads, 0, stream>>>(
      static_cast<const int32_t*>(topk_ids.data_ptr()),
      static_cast<const int32_t*>(expert_offsets.data_ptr()),
      static_cast<int32_t*>(input_permutation.data_ptr()),
      static_cast<int32_t*>(output_permutation.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()), topk_ids.numel(),
      topk_ids.size(1));
}
