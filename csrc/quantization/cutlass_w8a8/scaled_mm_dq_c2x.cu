#include <stddef.h>
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>

// clang-format will break include orders
// clang-format off
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/util/device_memory.h"

#include "cutlass/cutlass.h"
#include "cutlass/gemm_coord.h"
#include "cutlass/arch/mma_sm75.h"
#include "cutlass/arch/arch.h"
#include "cutlass/arch/mma.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"

#include "cutlass/epilogue/threadblock/fusion/visitors.hpp"
#include "cutlass/gemm/kernel/default_gemm_universal_with_visitor.h"

#include "broadcast_load_epilogue_c2x.hpp"
#include "common.hpp"
// clang-format on

using namespace cute;

/*
   This defines a quantized GEMM operation with dequantized output, similar to
   torch._scaled_mm. It is defined using the CUTLASS 2.x API, and is used for
   NVIDIA GPUs with SM versions prior to sm90 (Hopper).

   A and B may be both either int8 or fp8_e4m3. A can be quantized per-tensor or
   per-row. B can be quantized per-tensor or per-column.
   Any combination of per-tensor and per-row or column is supported.
   A and B must have symmetric quantization (zero point == 0).

   So the GEMM operation is D = (a_scales * A) (b_scales * B), where the
   scales are applied elementwise with numpy-style broadcasting.

   ScaleA and ScaleB define the epilogue functions that apply the scales for
   the A and B operands respectively. These scales may be either per-tensor or
   per row or column.
*/

namespace {

// Wrappers for the GEMM kernel that is used to guard against compilation on
// architectures that will never use the kernel. The purpose of this is to
// reduce the size of the compiled binary.
// __CUDA_ARCH__ is not defined in host code, so this lets us smuggle the ifdef
// into code that will be executed on the device where it is defined.
template <typename Kernel>
struct enable_sm75_to_sm80 : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE static void invoke(Args&&... args) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
    Kernel::invoke(std::forward<Args>(args)...);
#endif
  }
};

template <typename Kernel>
struct enable_sm80_to_sm89 : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE static void invoke(Args&&... args) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 890
    Kernel::invoke(std::forward<Args>(args)...);
#endif
  }
};

template <typename Kernel>
struct enable_sm89_to_sm90 : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE static void invoke(Args&&... args) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 890 && __CUDA_ARCH__ < 900
    Kernel::invoke(std::forward<Args>(args)...);
#endif
  }
};

template <typename Arch, template <typename> typename ArchGuard,
          typename ElementAB_, typename ElementD_, typename TileShape,
          typename WarpShape, typename InstructionShape, int32_t MainLoopStages>
struct cutlass_2x_gemm {
  using ElementAB = ElementAB_;
  using ElementD = ElementD_;

  using ElementAcc =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>, int32_t,
                                float>::type;

  using Operator =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>,
                                cutlass::arch::OpMultiplyAddSaturate,
                                cutlass::arch::OpMultiplyAdd>::type;

  using OutputTileThreadMap =
      cutlass::epilogue::threadblock::OutputTileThreadLayout<
          TileShape, WarpShape, float, 4, 1 /* epilogue stages */
          >;

  using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;

  using ScaleA = cutlass::epilogue::threadblock::VisitorColOrScalarBroadcast<
      OutputTileThreadMap, float, Stride<Int<1>, Int<0>, Int<0>>>;

  using ScaleB = cutlass::epilogue::threadblock::VisitorRowOrScalarBroadcast<
      OutputTileThreadMap, float, Stride<Int<0>, Int<1>, Int<0>>>;

  using Compute0 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::threadblock::Sm80EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, ElementD, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute1 =
      cutlass::epilogue::threadblock::Sm80EVT<Compute1, ScaleA, EVTCompute0>;

  using D = cutlass::epilogue::threadblock::VisitorAuxStore<
      OutputTileThreadMap, ElementD, cutlass::FloatRoundStyle::round_to_nearest,
      Stride<int64_t, Int<1>, Int<0>>>;

  using EVTD = cutlass::epilogue::threadblock::Sm80EVT<D, EVTCompute1>;

  // clang-format off
  using RowMajor = typename cutlass::layout::RowMajor;
  using ColumnMajor = typename cutlass::layout::ColumnMajor;
  using KernelType = 
    ArchGuard<typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
      ElementAB, RowMajor, cutlass::ComplexTransform::kNone, 16, 
      ElementAB, ColumnMajor, cutlass::ComplexTransform::kNone, 16, 
      float, cutlass::layout::RowMajor, 4,
      ElementAcc, float, cutlass::arch::OpClassTensorOp, 
      Arch, 
      TileShape, WarpShape, InstructionShape,
      EVTD,
      cutlass::gemm::threadblock::ThreadblockSwizzleStreamK,
      MainLoopStages, Operator,
      1 /* epilogue stages */
      >::GemmKernel>;
  // clang-format on

  using Op = cutlass::gemm::device::GemmUniversalAdapter<KernelType>;
};

template <typename Gemm>
void cutlass_scaled_mm_dq_dispatcher(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     torch::Tensor const& a_scales,
                                     torch::Tensor const& b_scales) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);
  cutlass::gemm::GemmCoord problem_size{m, n, k};

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  using StrideC = Stride<int64_t, Int<1>, Int<0>>;
  StrideC c_stride{ldc, Int<1>{}, Int<0>{}};

  auto a_ptr = static_cast<ElementAB const*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB const*>(b.data_ptr());
  auto c_ptr = static_cast<ElementD*>(out.data_ptr());

  auto a_scales_ptr = a_scales.data_ptr<float>();
  auto b_scales_ptr = b_scales.data_ptr<float>();

  using ScaleAArgs = typename Gemm::ScaleA::Arguments;
  using ScaleBArgs = typename Gemm::ScaleB::Arguments;

  ScaleBArgs b_args{b_scales.data_ptr<float>(), b_scales.numel() != 1, {}};
  ScaleAArgs a_args{a_scales.data_ptr<float>(), a_scales.numel() != 1, {}};

  typename Gemm::EVTCompute0::Arguments evt0_compute_args{b_args};

  typename Gemm::EVTCompute1::Arguments evt1_compute_args{a_args,
                                                          evt0_compute_args};
  typename Gemm::D::Arguments d_args{c_ptr, c_stride};

  typename Gemm::EVTD::Arguments epilogue_args{
      evt1_compute_args,
      d_args,
  };

  typename Gemm::Op::Arguments args{
      cutlass::gemm::GemmUniversalMode::kGemmSplitKParallel,  // universal mode
      problem_size,                                           // problem size
      1,                                                      // batch count
      epilogue_args,
      a_ptr,
      b_ptr,
      nullptr,
      nullptr,
      0,
      0,
      0,
      0,
      lda,
      ldb,
      ldc,
      ldc};

  // Launch the CUTLASS GEMM kernel.
  typename Gemm::Op gemm_op;
  size_t workspace_size = gemm_op.get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  CUTLASS_CHECK(gemm_op.can_implement(args));
  cutlass::Status status = gemm_op(args, workspace.get(), stream);
  CUTLASS_CHECK(status);
}

}  // namespace

void cutlass_scaled_mm_dq_sm75(torch::Tensor& out, torch::Tensor const& a,
                               torch::Tensor const& b,
                               torch::Tensor const& a_scales,
                               torch::Tensor const& b_scales) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<8, 8, 16>;

  if (out.dtype() == torch::kBFloat16) {
    return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
        cutlass::arch::Sm75, enable_sm75_to_sm80, int8_t, cutlass::bfloat16_t,
        TileShape, WarpShape, InstructionShape, 2>>(out, a, b, a_scales,
                                                    b_scales);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
        cutlass::arch::Sm75, enable_sm75_to_sm80, int8_t, cutlass::half_t,
        TileShape, WarpShape, InstructionShape, 2>>(out, a, b, a_scales,
                                                    b_scales);
  }
}

void cutlass_scaled_mm_dq_sm80(torch::Tensor& out, torch::Tensor const& a,
                               torch::Tensor const& b,
                               torch::Tensor const& a_scales,
                               torch::Tensor const& b_scales) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  if (out.dtype() == torch::kBFloat16) {
    return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
        cutlass::arch::Sm80, enable_sm80_to_sm89, int8_t, cutlass::bfloat16_t,
        TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                    b_scales);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
        cutlass::arch::Sm80, enable_sm80_to_sm89, int8_t, cutlass::half_t,
        TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                    b_scales);
  }
}

void cutlass_scaled_mm_dq_sm89(torch::Tensor& out, torch::Tensor const& a,
                               torch::Tensor const& b,
                               torch::Tensor const& a_scales,
                               torch::Tensor const& b_scales) {
  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
          cutlass::arch::Sm89, enable_sm89_to_sm90, int8_t, cutlass::bfloat16_t,
          TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                      b_scales);
    } else {
      assert(out.dtype() == torch::kFloat16);
      return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
          cutlass::arch::Sm89, enable_sm89_to_sm90, int8_t, cutlass::half_t,
          TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                      b_scales);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
          cutlass::arch::Sm89, enable_sm89_to_sm90, cutlass::float_e4m3_t,
          cutlass::bfloat16_t, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, a_scales, b_scales);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_scaled_mm_dq_dispatcher<cutlass_2x_gemm<
          cutlass::arch::Sm89, enable_sm89_to_sm90, cutlass::float_e4m3_t,
          cutlass::half_t, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, a_scales, b_scales);
    }
  }
}
