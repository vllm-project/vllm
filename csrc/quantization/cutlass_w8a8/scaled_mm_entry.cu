#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

void cutlass_scaled_mm_sm75(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm80(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm89(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
void cutlass_scaled_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);
#endif

void cutlass_scaled_mm_azp_sm75(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_azp_sm80(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_azp_sm89(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);

#if defined CUDA_VERSION && CUDA_VERSION >= 12000
void cutlass_scaled_mm_azp_sm90(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);
#endif

bool cutlass_scaled_mm_supports_fp8(int64_t cuda_device_capability) {
  // CUTLASS FP8 kernels need at least
  //   CUDA 12.0 on SM90 systems (Hopper)
  //   CUDA 12.4 on SM89 systems (Lovelace)

#if defined CUDA_VERSION
  if (cuda_device_capability >= 90) {
    return CUDA_VERSION >= 12000;
  } else if (cuda_device_capability >= 89) {
    return CUDA_VERSION >= 12040;
  }
#endif

  return false;
}

int32_t get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}

void cutlass_scaled_mm(torch::Tensor& c, torch::Tensor const& a,
                       torch::Tensor const& b, torch::Tensor const& a_scales,
                       torch::Tensor const& b_scales,
                       c10::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = get_sm_version_num();
  // Hopper

  // Guard against compilation issues for sm90 kernels
#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
  if (version_num >= 90) {
    cutlass_scaled_mm_sm90(c, a, b, a_scales, b_scales, bias);
    return;
  }
#endif

#if defined ENABLE_SCALED_MM_C2X && ENABLE_SCALED_MM_C2X
  if (version_num == 89) {
    // Ada Lovelace
    cutlass_scaled_mm_sm89(c, a, b, a_scales, b_scales, bias);
    return;
  }

  if (version_num >= 80) {
    // Ampere
    cutlass_scaled_mm_sm80(c, a, b, a_scales, b_scales, bias);
    return;
  }

  if (version_num >= 75) {
    // Turing
    cutlass_scaled_mm_sm75(c, a, b, a_scales, b_scales, bias);
    return;
  }
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}

void cutlass_scaled_mm_azp(torch::Tensor& c, torch::Tensor const& a,
                           torch::Tensor const& b,
                           torch::Tensor const& a_scales,
                           torch::Tensor const& b_scales,
                           torch::Tensor const& azp_adj,
                           c10::optional<torch::Tensor> const& azp,
                           c10::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  // bias, azp, azp_adj are all 1d
  // bias and azp_adj have n elements, azp has m elements
  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous());
  }
  if (azp) {
    TORCH_CHECK(azp->numel() == a.size(0) && azp->is_contiguous());
  }
  TORCH_CHECK(azp_adj.numel() == b.size(1) && azp_adj.is_contiguous());

  // azp & bias types
  TORCH_CHECK(azp_adj.dtype() == torch::kInt32);
  TORCH_CHECK(!azp || azp->dtype() == torch::kInt32);
  TORCH_CHECK(!bias || bias->dtype() == c.dtype(),
              "currently bias dtype must match output dtype ", c.dtype());

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));

  int32_t version_num = get_sm_version_num();

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
  if (version_num >= 90) {
    cutlass_scaled_mm_azp_sm90(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }
#endif

#if defined ENABLE_SCALED_MM_C2X && ENABLE_SCALED_MM_C2X
  if (version_num == 89) {
    // Ada Lovelace
    cutlass_scaled_mm_azp_sm89(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }

  if (version_num >= 80) {
    // Ampere
    cutlass_scaled_mm_azp_sm80(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }

  // Turing
  TORCH_CHECK(version_num >= 75);
  cutlass_scaled_mm_azp_sm75(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
  return;
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm_azp for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}