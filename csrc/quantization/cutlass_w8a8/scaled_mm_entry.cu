#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

void cutlass_scaled_mm_sm75(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm80(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm89(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

#if defined CUDA_VERSION && CUDA_VERSION >= 12000
void cutlass_scaled_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);
#endif

bool cutlass_scaled_mm_supports_fp8(int64_t cuda_device_capability) {
  // CUTLASS FP8 kernels need at least
  //   CUDA 12.0 on SM90 systems (Hopper)
  //   CUDA 12.4 on SM89 systems (Lovelace)

#if defined CUDA_VERSION
  if (cuda_device_capability >= 90) {
    return CUDA_VERSION >= 12000;
  } else if (cuda_device_capability >= 89) {
    return CUDA_VERSION >= 12040;
  }
#endif

  return false;
}

void cutlass_scaled_mm(torch::Tensor& c, torch::Tensor const& a,
                       torch::Tensor const& b, torch::Tensor const& a_scales,
                       torch::Tensor const& b_scales,
                       c10::optional<torch::Tensor> const& bias) {
  int32_t major_capability;
  int32_t minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;

  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));

  if (version_num >= 90) {
    // Hopper

    // Guard against compilation issues for sm90 kernels
#if defined CUDA_VERSION && CUDA_VERSION >= 12000
    cutlass_scaled_mm_sm90(c, a, b, a_scales, b_scales, bias);
#else
    cutlass_scaled_mm_sm80(c, a, b, a_scales, b_scales, bias);
#endif
  } else if (version_num == 89) {
    // Ada Lovelace
    cutlass_scaled_mm_sm89(c, a, b, a_scales, b_scales, bias);
  } else if (version_num >= 80) {
    // Ampere
    cutlass_scaled_mm_sm80(c, a, b, a_scales, b_scales, bias);
  } else {
    // Turing
    TORCH_CHECK(version_num >= 75);
    cutlass_scaled_mm_sm75(c, a, b, a_scales, b_scales, bias);
  }
}