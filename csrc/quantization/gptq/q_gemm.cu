#include "hip/hip_runtime.h"
/*
Adapted from https://github.com/turboderp/exllamav2 and https://github.com/qwopqwop200/GPTQ-for-LLaMa
*/

#include <cstdint>
#include <cstdio>

#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "compat.cuh"
#include "matrix_view.cuh"
#include "qdq_2.cuh"
#include "qdq_3.cuh"
#include "qdq_4.cuh"
#include "qdq_8.cuh"

namespace vllm {
namespace gptq {

#define BLOCK_KN_SIZE 128
#define BLOCK_M_SIZE_MAX 8
#define MAX_GROUPS_IN_BLOCK (BLOCK_KN_SIZE / 32)
#define MAX_Q_GEMM_ROWS 50
#define MAX_Q_GEMM_ROWS_8BIT 24
#define MAX_ALT_GEMM_ROWS 8
#define THREADS_X 32
#define THREADS_Y 32
#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

#if defined(USE_ROCM)
#include <hipblas/hipblas.h>
__host__ __forceinline__ hipblasStatus_t __compat_hipblasHgemm(hipblasHandle_t    handle,
                                                               hipblasOperation_t transA,
                                                               hipblasOperation_t transB,
                                                               int                m,
                                                               int                n,
                                                               int                k,
                                                               const half*        alpha,
                                                               const half*        AP,
                                                               int                lda,
                                                               const half*        BP,
                                                               int                ldb,
                                                               const half*        beta,
                                                               half*              CP,
                                                               int                ldc) {
    return hipblasHgemm(handle, transA, transB, m, n, k,
                        reinterpret_cast<const hipblasHalf *>(alpha),
                        reinterpret_cast<const hipblasHalf *>(AP), lda,
                        reinterpret_cast<const hipblasHalf *>(BP), ldb,
                        reinterpret_cast<const hipblasHalf *>(beta),
                        reinterpret_cast<hipblasHalf *>(CP), ldc);
}
#define hipblasHgemm __compat_hipblasHgemm

// Previous version of PyTorch were converting to rocBLAS instead of hipBLAS.
#define rocblas_operation_none HIPBLAS_OP_N
#define rocblas_hgemm __compat_hipblasHgemm
#endif

__forceinline__ __device__ half2 dot22_8(half2(&dq)[4], const half* a_ptr, const half2 g_result)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    return __hadd2(result, g_result);
}

__forceinline__ __device__ float dot22_8_f(half2(&dq)[4], const half* a_ptr)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    return __half2float(__low2half(result)) + __half2float(__high2half(result));
}

__forceinline__ __device__ half2 dot22_8(half2(&dq)[4], const half* a_ptr, const half2 g_result, const half qs_h)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    return __hfma2(result, __halves2half2(qs_h, qs_h), g_result);
}

__forceinline__ __device__ half2 dot22_16(half2(&dq)[8], const half* a_ptr, const half2 g_result, const half qs_h)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 8; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    return __hfma2(result, __halves2half2(qs_h, qs_h), g_result);
}

__forceinline__ __device__ half2 dot22_32(half2(&dq)[16], const half* a_ptr, const half2 g_result, const half qs_h)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 16; i += 1) result = __hfma2(dq[i], *a2_ptr++, result);
    return __hfma2(result, __halves2half2(qs_h, qs_h), g_result);
}

__forceinline__ __device__ float dot22_8_f(half2(&dq)[4], const half* a_ptr, const float g_result, const float qs_f)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    float result_f = __half2float(__low2half(result)) + __half2float(__high2half(result));
    return fma(result_f, qs_f, g_result);
}

__forceinline__ __device__ float dot22_16_f(half2(&dq)[8], const half* a_ptr, const float g_result, const float qs_f)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 8; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    float result_f = __half2float(__low2half(result)) + __half2float(__high2half(result));
    return fma(result_f, qs_f, g_result);
}

__forceinline__ __device__ float dot22_32_f(half2(&dq)[16], const half* a_ptr, const float g_result, const float qs_f)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 16; i += 1) result = __hfma2(dq[i], *a2_ptr++, result);
    float result_f = __half2float(__low2half(result)) + __half2float(__high2half(result));
    return fma(result_f, qs_f, g_result);
}

__forceinline__ __device__ half dot22_8_h(half2(&dq)[4], const half* a_ptr, const half g_result, const half qs_h)
{
    // Use FP32 accumulator to avoid potential overflow since unscaled weights are in the range -128..127

    float result = {};
    #pragma unroll
    for (int i = 0; i < 4; i++)
    {
        half2 w01 = dq[i];
        float w0 = __low2float(w01);
        float w1 = __high2float(w01);
        float x0 = __half2float(*a_ptr++);
        float x1 = __half2float(*a_ptr++);
        result = fma(w0, x0, result);
        result = fma(w1, x1, result);
    }
    float qs = __half2float(qs_h);
    result *= qs;
    half result_h = __float2half_rn(result);
    return __hadd(result_h, g_result);
}

__forceinline__ __device__ half dot22_16_h(half2(&dq)[8], const half* a_ptr, const half g_result, const half qs_h)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 8; i++) result = __hfma2(dq[i], *a2_ptr++, result);
    half result_h = __hadd(__low2half(result), __high2half(result));
    return __hfma(result_h, qs_h, g_result);
}

__forceinline__ __device__ half dot22_32_h(half2(&dq)[16], const half* a_ptr, const half g_result, const half qs_h)
{
    half2 result = {};
    const half2* a2_ptr = (const half2*)a_ptr;
    #pragma unroll
    for (int i = 0; i < 16; i += 1) result = __hfma2(dq[i], *a2_ptr++, result);
    half result_h = __hadd(__low2half(result), __high2half(result));
    return __hfma(result_h, qs_h, g_result);
}


typedef void (*fp_gemm_half_q_half_gptq_kernel)
(
    const half*,
    const uint32_t*,
    const uint32_t*,
    const half*,
    half*,
    const int,
    const int,
    const int,
    const int,
    const int*
);


template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_4bit_kernel
(
    const half* __restrict__ a,
    const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    half* __restrict__ c,
    const int size_m,
    const int size_n,
    const int size_k,
    const int groups,
    const int* __restrict__ b_q_perm
)
{
    MatrixView_half a_(a, size_m, size_k);
    MatrixView_half_rw c_(c, size_m, size_n);
    MatrixView_q4_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int t = threadIdx.x;

    // Block
    int offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
    int offset_m = blockIdx.y * m_count;
    int offset_k = blockIdx.z * BLOCK_KN_SIZE;

    int end_n = min(offset_n + BLOCK_KN_SIZE * 4, size_n);
    int end_m = min(offset_m + m_count, size_m);
    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    int n = offset_n + t * 4;

    // Preload block_a
    __shared__ half block_a[m_count][BLOCK_KN_SIZE];

    if (offset_k + t < end_k)
    {
        for (int m = 0; m < m_count; ++m)
        {
            const half* a_ptr = a_.item_ptr(offset_m + m, 0);
            half* block_a_ptr = block_a[m];

            half a0;
            if (b_q_perm) a0 = a_ptr[b_q_perm[offset_k + t]];
            else a0 = a_ptr[offset_k + t];
            block_a_ptr[t] = a0;
        }
    }

    // Zero output
    if (n >= size_n) return;

    if (blockIdx.z == 0)
    {
        for (int m = 0; m < m_count; m++)
            *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
    }

    __syncthreads();

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // a, b offset
    int qk = offset_k / (32 / 4);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
    const half* a_ptr = &block_a[0][0];
    int a_stride = BLOCK_KN_SIZE;

    // Initial group
    int zeros[4];
    float scales[4];
    half2 z1z16[4][2];
    half2 y1y16[4][2];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_f(scales, group, n);
    dequant_4bit_8_prep_zero(zeros[0] + 1, z1z16[0], y1y16[0]);
    dequant_4bit_8_prep_zero(zeros[1] + 1, z1z16[1], y1y16[1]);
    dequant_4bit_8_prep_zero(zeros[2] + 1, z1z16[2], y1y16[2]);
    dequant_4bit_8_prep_zero(zeros[3] + 1, z1z16[3], y1y16[3]);

    // Column result
    float block_c[m_count][4] = {};

    // Dequantize and multiply
    int k = offset_k;
    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_f(scales, group, n);
            dequant_4bit_8_prep_zero(zeros[0] + 1, z1z16[0], y1y16[0]);
            dequant_4bit_8_prep_zero(zeros[1] + 1, z1z16[1], y1y16[1]);
            dequant_4bit_8_prep_zero(zeros[2] + 1, z1z16[2], y1y16[2]);
            dequant_4bit_8_prep_zero(zeros[3] + 1, z1z16[3], y1y16[3]);
        }

        #pragma unroll
        for (int j = 0; j < 4; j++)
        {
            const int4* b_ptr4 = (int4*) b_ptr;
            int4 load_int4 = *b_ptr4;

            half2 dq[4][4];
            dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n, false);
            dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n, false);
            dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n, false);
            dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n, false);

            #pragma unroll
            for (int m = 0; m < m_count; m++)
            {
                block_c[m][0] = fma(dot22_8_f(dq[0], a_ptr + m * a_stride), scales[0], block_c[m][0]);
                block_c[m][1] = fma(dot22_8_f(dq[1], a_ptr + m * a_stride), scales[1], block_c[m][1]);
                block_c[m][2] = fma(dot22_8_f(dq[2], a_ptr + m * a_stride), scales[2], block_c[m][2]);
                block_c[m][3] = fma(dot22_8_f(dq[3], a_ptr + m * a_stride), scales[3], block_c[m][3]);
            }

            b_ptr += size_n;
            a_ptr += 8;
        }

        k += 32;
    }

    for (int m = 0; m < m_count; m++)
    {
        half2 *out = (half2*) c_.item_ptr(offset_m + m, n);
        half2 result01 = __halves2half2(__float2half_rn(block_c[m][0]), __float2half_rn(block_c[m][1]));
        half2 result23 = __halves2half2(__float2half_rn(block_c[m][2]), __float2half_rn(block_c[m][3]));
        atomicAdd(out    , result01);
        atomicAdd(out + 1, result23);
    }
}

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_2bit_kernel
(
    const half* __restrict__ a,
    const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    half* __restrict__ c,
    const int size_m,
    const int size_n,
    const int size_k,
    const int groups,
    const int* __restrict__ b_q_perm
)
{
    MatrixView_half a_(a, size_m, size_k);
    MatrixView_half_rw c_(c, size_m, size_n);
    MatrixView_q2_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int t = threadIdx.x;

    // Block
    int offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
    int offset_m = blockIdx.y * m_count;
    int offset_k = blockIdx.z * BLOCK_KN_SIZE;

    int end_n = min(offset_n + BLOCK_KN_SIZE * 4, size_n);
    int end_m = min(offset_m + m_count, size_m);
    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    int n = offset_n + t * 4;

    // Preload block_a
    __shared__ half block_a[m_count][BLOCK_KN_SIZE];

    if (offset_k + t < end_k)
    {
        for (int m = 0; m < m_count; ++m)
        {
            const half* a_ptr = a_.item_ptr(offset_m + m, 0);
            half* block_a_ptr = block_a[m];

            half a0;
            if (b_q_perm) a0 = a_ptr[b_q_perm[offset_k + t]];
            else a0 = a_ptr[offset_k + t];
            block_a_ptr[t] = a0;
        }
    }

    // Zero output
    if (n >= size_n) return;

    if (blockIdx.z == 0)
    {
        for (int m = 0; m < m_count; m++)
            *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
    }

    __syncthreads();

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // a, b offset
    int qk = offset_k / (32 / 2);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
    const half* a_ptr = &block_a[0][0];
    int a_stride = BLOCK_KN_SIZE;

    // Initial group
    int zeros[4];
    half scales[4];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4(scales, group, n);
    // Column result
    half block_c[m_count][4] = {};

    // Dequantize and multiply
    int k = offset_k;
    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4(scales, group, n);
        }

        #pragma unroll
        for (int j = 0; j < 1; j++)
        {
            const int4* b_ptr4 = (int4*) b_ptr;
            int4 load_int4 = *b_ptr4;

            half2 dq[4][8];
            dequant_2bit_16(load_int4.x, dq[0], size_n, zeros[0] + 1);
            dequant_2bit_16(load_int4.y, dq[1], size_n, zeros[1] + 1);
            dequant_2bit_16(load_int4.z, dq[2], size_n, zeros[2] + 1);
            dequant_2bit_16(load_int4.w, dq[3], size_n, zeros[3] + 1);

            #pragma unroll
            for (int m = 0; m < m_count; m++)
            {
                block_c[m][0] = dot22_16_h(dq[0], a_ptr + m * a_stride, block_c[m][0], scales[0]);
                block_c[m][1] = dot22_16_h(dq[1], a_ptr + m * a_stride, block_c[m][1], scales[1]);
                block_c[m][2] = dot22_16_h(dq[2], a_ptr + m * a_stride, block_c[m][2], scales[2]);
                block_c[m][3] = dot22_16_h(dq[3], a_ptr + m * a_stride, block_c[m][3], scales[3]);
            }

            b_ptr += size_n;
            a_ptr += 16;
        }

        k += 16;
    }

    for (int m = 0; m < m_count; m++)
    {
        half2 *out = (half2*) c_.item_ptr(offset_m + m, n);
        half2 result01 = __halves2half2(block_c[m][0], block_c[m][1]);
        half2 result23 = __halves2half2(block_c[m][2], block_c[m][3]);
        atomicAdd(out    , result01);
        atomicAdd(out + 1, result23);
    }
}

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_3bit_kernel
(
    const half* __restrict__ a,
    const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    half* __restrict__ c,
    const int size_m,
    const int size_n,
    const int size_k,
    const int groups,
    const int* __restrict__ b_q_perm
)
{
    MatrixView_half a_(a, size_m, size_k);
    MatrixView_half_rw c_(c, size_m, size_n);
    MatrixView_q3_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int t = threadIdx.x;

    // Block
    int offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
    int offset_m = blockIdx.y * m_count;
    int offset_k = blockIdx.z * BLOCK_KN_SIZE;

    int end_n = min(offset_n + BLOCK_KN_SIZE * 4, size_n);
    int end_m = min(offset_m + m_count, size_m);
    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    int n = offset_n + t * 4;

    // Preload block_a
    __shared__ half block_a[m_count][BLOCK_KN_SIZE];

    if (offset_k + t < end_k)
    {
        for (int m = 0; m < m_count; ++m)
        {
            const half* a_ptr = a_.item_ptr(offset_m + m, 0);
            half* block_a_ptr = block_a[m];

            half a0;
            if (b_q_perm) a0 = a_ptr[b_q_perm[offset_k + t]];
            else a0 = a_ptr[offset_k + t];
            block_a_ptr[t] = a0;
        }
    }

    // Zero output
    if (n >= size_n) return;

    if (blockIdx.z == 0)
    {
        for (int m = 0; m < m_count; m++)
            *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
    }

    __syncthreads();

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // a, b offset
    int qk = offset_k / 32 * 3;

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
    const half* a_ptr = &block_a[0][0];
    int a_stride = BLOCK_KN_SIZE;

    // Initial group
    int zeros[4];
    half scales[4];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4(scales, group, n);
    // Column result
    half block_c[m_count][4] = {};

    // Dequantize and multiply
    int k = offset_k;
    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4(scales, group, n);
        }

        #pragma unroll
        for (int j = 0; j < 1; j++)
        {
            int4 load_int4[3];
            load_int4[0] = *((int4*) b_ptr); b_ptr += size_n;
            load_int4[1] = *((int4*) b_ptr); b_ptr += size_n;
            load_int4[2] = *((int4*) b_ptr); b_ptr += size_n;

            half2 dq[4][16];
            dequant_3bit_32(load_int4[0].x, load_int4[1].x, load_int4[2].x, dq[0], size_n, zeros[0] + 1);
            dequant_3bit_32(load_int4[0].y, load_int4[1].y, load_int4[2].y, dq[1], size_n, zeros[1] + 1);
            dequant_3bit_32(load_int4[0].z, load_int4[1].z, load_int4[2].z, dq[2], size_n, zeros[2] + 1);
            dequant_3bit_32(load_int4[0].w, load_int4[1].w, load_int4[2].w, dq[3], size_n, zeros[3] + 1);

            #pragma unroll
            for (int m = 0; m < m_count; m++)
            {
                block_c[m][0] = dot22_32_h(dq[0], a_ptr + m * a_stride, block_c[m][0], scales[0]);
                block_c[m][1] = dot22_32_h(dq[1], a_ptr + m * a_stride, block_c[m][1], scales[1]);
                block_c[m][2] = dot22_32_h(dq[2], a_ptr + m * a_stride, block_c[m][2], scales[2]);
                block_c[m][3] = dot22_32_h(dq[3], a_ptr + m * a_stride, block_c[m][3], scales[3]);
            }
            a_ptr += 32;
        }

        k += 32;
    }

    for (int m = 0; m < m_count; m++)
    {
        half2 *out = (half2*) c_.item_ptr(offset_m + m, n);
        half2 result01 = __halves2half2(block_c[m][0], block_c[m][1]);
        half2 result23 = __halves2half2(block_c[m][2], block_c[m][3]);
        atomicAdd(out    , result01);
        atomicAdd(out + 1, result23);
    }
}

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_8bit_kernel
(
    const half* __restrict__ a,
    const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    half* __restrict__ c,
    const int size_m,
    const int size_n,
    const int size_k,
    const int groups,
    const int* __restrict__ b_q_perm
)
{
    MatrixView_half a_(a, size_m, size_k);
    MatrixView_half_rw c_(c, size_m, size_n);
    MatrixView_q8_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int t = threadIdx.x;

    // Block
    int offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
    int offset_m = blockIdx.y * m_count;
    int offset_k = blockIdx.z * BLOCK_KN_SIZE;

    int end_n = min(offset_n + BLOCK_KN_SIZE * 4, size_n);
    int end_m = min(offset_m + m_count, size_m);
    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    int n = offset_n + t * 4;

    // Preload block_a
    __shared__ half block_a[m_count][BLOCK_KN_SIZE];

    if (offset_k + t < end_k)
    {
        for (int m = 0; m < m_count; ++m)
        {
            const half* a_ptr = a_.item_ptr(offset_m + m, 0);
            half* block_a_ptr = block_a[m];

            half a0;
            if (b_q_perm) a0 = a_ptr[b_q_perm[offset_k + t]];
            else a0 = a_ptr[offset_k + t];
            block_a_ptr[t] = a0;
        }
    }

    // Zero output
    if (n >= size_n) return;

    if (blockIdx.z == 0)
    {
        for (int m = 0; m < m_count; m++)
            *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
    }

    __syncthreads();

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // a, b offset
    int qk = offset_k / (32 / 8);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
    const half* a_ptr = &block_a[0][0];
    int a_stride = BLOCK_KN_SIZE;

    // Initial group
    int zeros[4];
    half scales[4];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4(scales, group, n);
    // Column result
    half block_c[m_count][4] = {};

    // Dequantize and multiply
    int k = offset_k;
    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4(scales, group, n);
        }

        #pragma unroll
        for (int j = 0; j < 4; j++)
        {
            int4 load_int4[2];
            load_int4[0] = *((int4*) b_ptr); b_ptr += size_n;
            load_int4[1] = *((int4*) b_ptr); b_ptr += size_n;

            half2 dq[4][4];
            dequant_8bit_8(load_int4[0].x, load_int4[1].x, dq[0], size_n, zeros[0] + 1);
            dequant_8bit_8(load_int4[0].y, load_int4[1].y, dq[1], size_n, zeros[1] + 1);
            dequant_8bit_8(load_int4[0].z, load_int4[1].z, dq[2], size_n, zeros[2] + 1);
            dequant_8bit_8(load_int4[0].w, load_int4[1].w, dq[3], size_n, zeros[3] + 1);

            for (int m = 0; m < m_count; m++)
            {
                block_c[m][0] = dot22_8_h(dq[0], a_ptr + m * a_stride, block_c[m][0], scales[0]);
                block_c[m][1] = dot22_8_h(dq[1], a_ptr + m * a_stride, block_c[m][1], scales[1]);
                block_c[m][2] = dot22_8_h(dq[2], a_ptr + m * a_stride, block_c[m][2], scales[2]);
                block_c[m][3] = dot22_8_h(dq[3], a_ptr + m * a_stride, block_c[m][3], scales[3]);
            }
            a_ptr += 8;
        }
        k += 32;
    }

    for (int m = 0; m < m_count; m++)
    {
        half2 *out = (half2*) c_.item_ptr(offset_m + m, n);
        half2 result01 = __halves2half2(block_c[m][0], block_c[m][1]);
        half2 result23 = __halves2half2(block_c[m][2], block_c[m][3]);
        atomicAdd(out    , result01);
        atomicAdd(out + 1, result23);
    }
}

fp_gemm_half_q_half_gptq_kernel pick_gemm_half_q_half_gptq_kernel(
    bool first_block, const int m_count, const int bit)
{
    #define SELECT_KERNEL(M_COUNT)                                            \
    if (m_count == M_COUNT) {                                                 \
      if (bit == 2) return gemm_half_q_half_gptq_2bit_kernel<true, M_COUNT>;  \
      if (bit == 3) return gemm_half_q_half_gptq_3bit_kernel<true, M_COUNT>;  \
      if (bit == 4) return gemm_half_q_half_gptq_4bit_kernel<true, M_COUNT>;  \
      if (bit == 8) return gemm_half_q_half_gptq_8bit_kernel<true, M_COUNT>;  \
    }
    #if BLOCK_M_SIZE_MAX >= 1
    SELECT_KERNEL(1);
    #endif
    #if BLOCK_M_SIZE_MAX >= 2
    SELECT_KERNEL(2);
    #endif
    #if BLOCK_M_SIZE_MAX >= 3
    SELECT_KERNEL(3);
    #endif
    #if BLOCK_M_SIZE_MAX >= 4
    SELECT_KERNEL(4);
    #endif
    #if BLOCK_M_SIZE_MAX >= 5
    SELECT_KERNEL(5);
    #endif
    #if BLOCK_M_SIZE_MAX >= 6
    SELECT_KERNEL(6);
    #endif
    #if BLOCK_M_SIZE_MAX >= 7
    SELECT_KERNEL(7);
    #endif
    #if BLOCK_M_SIZE_MAX >= 8
    SELECT_KERNEL(8);
    #endif
    return NULL;
}


void gemm_half_q_half_cuda_part
(
    const half* a,
    const uint32_t* b_q_weight,
    const uint32_t* b_gptq_qzeros,
    const half* b_gptq_scales,
    const int* b_q_perm,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    int m_count,
    int groups,
    int bit
)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = DIVIDE(size_n, BLOCK_KN_SIZE * 4);
    gridDim.y = DIVIDE(size_m, m_count);
    gridDim.z = DIVIDE(size_k, BLOCK_KN_SIZE);

    fp_gemm_half_q_half_gptq_kernel kernel = pick_gemm_half_q_half_gptq_kernel(true, m_count, bit);

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    kernel<<<gridDim, blockDim, 0, stream>>>
    (
        a,
        b_q_weight,
        b_gptq_qzeros,
        b_gptq_scales,
        c,
        size_m,
        size_n,
        size_k,
        groups,
        b_q_perm
    );
}


__global__ void reconstruct_exllama_8bit_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    const int size_k,
    const int size_n,
    const int groups,
    half* __restrict__ b
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q8_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    // Preload remapping table
    __shared__ int perm[BLOCK_KN_SIZE];
    int t = threadIdx.x;

    if (b_q_perm)
    {
        if (offset_k + t < size_k)
            perm[t] = b_q_perm[offset_k + t];
    }

    // Column
    int n = offset_n + t * 4;
    if (n >= size_n) return;

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // b offset
    int qk = offset_k / (32 / 8);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    // Initial zeros/scale
    int zeros[4];
    half2 scales[4];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_h2(scales, group, n);

    __syncthreads();

    int k = offset_k;
    int lk = 0;

    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_h2(scales, group, n);
        }

        for (int p = 0; p < 4; p++)
        {
            int4 load_int4[2];
            load_int4[0] = *((int4*) b_ptr); b_ptr += size_n;
            load_int4[1] = *((int4*) b_ptr); b_ptr += size_n;

            half2 dq[4][4];
            dequant_8bit_8(load_int4[0].x, load_int4[1].x, dq[0], size_n, zeros[0] + 1);
            dequant_8bit_8(load_int4[0].y, load_int4[1].y, dq[1], size_n, zeros[1] + 1);
            dequant_8bit_8(load_int4[0].z, load_int4[1].z, dq[2], size_n, zeros[2] + 1);
            dequant_8bit_8(load_int4[0].w, load_int4[1].w, dq[3], size_n, zeros[3] + 1);

            //half* dqh = (half*)dq;
            if (b_q_perm)
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
            else
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(offset_k + lk++, n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(offset_k + lk++, n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
        }
        k += 32;
    }
}

__global__ void reconstruct_exllama_4bit_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    const int size_k,
    const int size_n,
    const int groups,
    half* __restrict__ b
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q4_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    // Preload remapping table
    __shared__ int perm[BLOCK_KN_SIZE];
    int t = threadIdx.x;

    if (b_q_perm)
    {
        if (offset_k + t < size_k)
            perm[t] = b_q_perm[offset_k + t];
    }

    // Column
    int n = offset_n + t * 4;
    if (n >= size_n) return;

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // b offset
    int qk = offset_k / (32 / 4);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    // Initial zeros/scale
    int zeros[4];
    half2 scales[4];
    half2 z1z16[4][2];
    half2 y1y16[4][2];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_h2(scales, group, n);
    dequant_4bit_8_prep_zero(zeros[0] + 1, z1z16[0], y1y16[0]);
    dequant_4bit_8_prep_zero(zeros[1] + 1, z1z16[1], y1y16[1]);
    dequant_4bit_8_prep_zero(zeros[2] + 1, z1z16[2], y1y16[2]);
    dequant_4bit_8_prep_zero(zeros[3] + 1, z1z16[3], y1y16[3]);

    __syncthreads();

    int k = offset_k;
    int lk = 0;

    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_h2(scales, group, n);
            dequant_4bit_8_prep_zero(zeros[0] + 1, z1z16[0], y1y16[0]);
            dequant_4bit_8_prep_zero(zeros[1] + 1, z1z16[1], y1y16[1]);
            dequant_4bit_8_prep_zero(zeros[2] + 1, z1z16[2], y1y16[2]);
            dequant_4bit_8_prep_zero(zeros[3] + 1, z1z16[3], y1y16[3]);
        }

        for (int p = 0; p < 4; p++)
        {
            half2 dq[4][4];
            const int4* b_ptr4 = (int4*) b_ptr;
            int4 load_int4 = *b_ptr4;

            dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n, false);
            dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n, false);
            dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n, false);
            dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n, false);

            b_ptr += size_n;
            //half* dqh = (half*)dq;
            if (b_q_perm)
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
            else
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(offset_k + lk++, n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(offset_k + lk++, n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
        }
        k += 32;
    }
}

__global__ void reconstruct_exllama_3bit_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    const int size_k,
    const int size_n,
    const int groups,
    half* __restrict__ b
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q3_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    // Preload remapping table
    __shared__ int perm[BLOCK_KN_SIZE];
    int t = threadIdx.x;

    if (b_q_perm)
    {
        if (offset_k + t < size_k)
            perm[t] = b_q_perm[offset_k + t];
    }

    // Column
    int n = offset_n + t * 4;
    if (n >= size_n) return;

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // b offset
    int qk = offset_k / 32* 3;

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    // Initial zeros/scale
    int zeros[4];
    half2 scales[4];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_h2(scales, group, n);

    __syncthreads();

    int k = offset_k;
    int lk = 0;

    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_h2(scales, group, n);
        }

        for (int p = 0; p < 1; p++)
        {
            int4 load_int4[3];
            load_int4[0] = *((int4*) b_ptr); b_ptr += size_n;
            load_int4[1] = *((int4*) b_ptr); b_ptr += size_n;
            load_int4[2] = *((int4*) b_ptr); b_ptr += size_n;

            half2 dq[4][16];
            dequant_3bit_32(load_int4[0].x, load_int4[1].x, load_int4[2].x, dq[0], size_n, zeros[0] + 1);
            dequant_3bit_32(load_int4[0].y, load_int4[1].y, load_int4[2].y, dq[1], size_n, zeros[1] + 1);
            dequant_3bit_32(load_int4[0].z, load_int4[1].z, load_int4[2].z, dq[2], size_n, zeros[2] + 1);
            dequant_3bit_32(load_int4[0].w, load_int4[1].w, load_int4[2].w, dq[3], size_n, zeros[3] + 1);

            if (b_q_perm)
            {
                for (int j = 0; j < 16; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
            else
            {
                for (int j = 0; j < 16; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(offset_k + lk++, n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(offset_k + lk++, n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
        }
        k += 32;
    }
}

__global__ void reconstruct_exllama_2bit_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    const int size_k,
    const int size_n,
    const int groups,
    half* __restrict__ b
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q2_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    // Preload remapping table
    __shared__ int perm[BLOCK_KN_SIZE];
    int t = threadIdx.x;

    if (b_q_perm)
    {
        if (offset_k + t < size_k)
            perm[t] = b_q_perm[offset_k + t];
    }

    // Column
    int n = offset_n + t * 4;
    if (n >= size_n) return;

    // Find initial group
    int groupsize = size_k / groups;
    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // b offset
    int qk = offset_k / (32 / 2);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    // Initial zeros/scale
    int zeros[4];
    half2 scales[4];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_h2(scales, group, n);

    __syncthreads();

    int k = offset_k;
    int lk = 0;

    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_h2(scales, group, n);
        }

        for (int p = 0; p < 2; p++)
        {
            const int4* b_ptr4 = (int4*) b_ptr;
            int4 load_int4 = *b_ptr4;

            half2 dq[4][8];
            dequant_2bit_16(load_int4.x, dq[0], size_n, zeros[0] + 1);
            dequant_2bit_16(load_int4.y, dq[1], size_n, zeros[1] + 1);
            dequant_2bit_16(load_int4.z, dq[2], size_n, zeros[2] + 1);
            dequant_2bit_16(load_int4.w, dq[3], size_n, zeros[3] + 1);

            b_ptr += size_n;
            //half* dqh = (half*)dq;
            if (b_q_perm)
            {
                for (int j = 0; j < 8; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
            else
            {
                for (int j = 0; j < 8; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(offset_k + lk++, n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(offset_k + lk++, n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
        }
        k += 32;
    }
}

void reconstruct_exllama
(
    const uint32_t* b_q_weight,
    const uint32_t* b_gptq_qzeros,
    const half* b_gptq_scales,
    const int* b_q_perm,
    half* out,
    int height,
    int width,
    int groups,
    int bit
)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    gridDim.y = DIVIDE(height, BLOCK_KN_SIZE);
    gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);

    auto reconstruct_exllama_kernel = reconstruct_exllama_4bit_kernel;
    if (bit == 2) {
        reconstruct_exllama_kernel = reconstruct_exllama_2bit_kernel;
    } else if (bit == 3) {
        reconstruct_exllama_kernel = reconstruct_exllama_3bit_kernel;
    } else if (bit == 8) {
        reconstruct_exllama_kernel = reconstruct_exllama_8bit_kernel;
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    reconstruct_exllama_kernel<<<gridDim, blockDim, 0, stream>>>
    (
        b_q_weight,
        b_q_perm,
        b_gptq_qzeros,
        b_gptq_scales,
        height,
        width,
        groups,
        out
    );
}


__global__ void gemm_half_q_half_alt_4bit_kernel(
    const half2* __restrict__ vec,
    const uint32_t* __restrict__ mat,
    half* __restrict__ mul,
    const half* __restrict__ scales,
    const uint32_t* __restrict__ zeros,
    const int* __restrict__ g_idx,
    int batch,
    int height,
    int width
)
{
    int zero_width = width / 8;
    int vec_height = height * 4;
    const int blockwidth2 = BLOCK_KN_SIZE / 2;
    int b = blockIdx.y * BLOCK_M_SIZE_MAX;
    int b_end = min(BLOCK_M_SIZE_MAX, batch - b);
    int h = BLOCK_KN_SIZE * blockIdx.z / 8;
    int h_end = min(BLOCK_KN_SIZE / 8, height - h) * 4;
    int w = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;

    __shared__ half2 blockvec[BLOCK_M_SIZE_MAX][blockwidth2];
    if (threadIdx.x < h_end) {
        for (int m = 0; m < b_end; ++m) {
          blockvec[m][threadIdx.x] =
              vec[(m + b) * vec_height + blockIdx.z * BLOCK_KN_SIZE / 2 +
                  threadIdx.x];
        }
    }

    __shared__ half2 deq2[256][8];
    int val = threadIdx.x / 8;
    int off = threadIdx.x % 8;
    for (; val < 256; val += BLOCK_KN_SIZE / 8) {
        deq2[val][off] = __halves2half2(
            __int2half_rn(val & 0xF), __int2half_rn(val >> 4)
        );
    }

    if (blockIdx.z == 0)
    {
        for (int m = 0; m < b_end; m++)
            mul[(b + m) * width + w] = __int2half_rn(0);
    }
    __syncthreads();

    int i = width * h + w;
    int g_h = h * 8;
    int k = 0;
    int z_w = w / 8;
    int z_mod = (w % 8) * 4;
    half2 res2;
    half res[BLOCK_M_SIZE_MAX] = {};

    unsigned int tmp;
    while (k < h_end) {
        tmp = mat[i];
        half2 scales_tmp[4];
        half2 zeros_tmp[4];
        for (int tmp_k = 0; tmp_k < 4; tmp_k++) {
            int g = g_idx[g_h + (k + tmp_k) * 2];
            int g2 = g_idx[g_h + (k + tmp_k) * 2 + 1];
            half scale_f = scales[g * width + w];
            half scale_f2 = scales[g2 * width + w];
            half2 scale = __halves2half2(scale_f, scale_f2);
            half2 zero = __halves2half2(
                __hmul(scale_f, __int2half_rn(-((zeros[g * zero_width + z_w] >> z_mod) & 0xF) - 1)),
                __hmul(scale_f2, __int2half_rn(-((zeros[g2 * zero_width + z_w] >> z_mod) & 0xF) - 1))
            );
            scales_tmp[tmp_k] = scale;
            zeros_tmp[tmp_k] = zero;
        }
        for (int m = 0; m < b_end; m++) {
#ifndef USE_ROCM
            res2 = {};
#else
            res2.x = __half_as_ushort(__float2half(0));
            res2.y = __half_as_ushort(__float2half(0));
#endif
            res2 = __hfma2(__hfma2(deq2[(tmp >>  0) & 0xff][off], scales_tmp[0], zeros_tmp[0]), blockvec[m][k + 0], res2);
            res2 = __hfma2(__hfma2(deq2[(tmp >>  8) & 0xff][off], scales_tmp[1], zeros_tmp[1]), blockvec[m][k + 1], res2);
            res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xff][off], scales_tmp[2], zeros_tmp[2]), blockvec[m][k + 2], res2);
            res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xff][off], scales_tmp[3], zeros_tmp[3]), blockvec[m][k + 3], res2);
#ifndef USE_ROCM
            res[m] = __hadd(res[m], __hadd(res2.x, res2.y));
#else
            res[m] = __hadd(res[m], __hadd(__ushort_as_half(res2.x), __ushort_as_half(res2.y)));
#endif
        }
        i += width;
        k += 4;
    }
    for (int m = 0; m < b_end; m++) {
        atomicAdd(&mul[(b + m) * width + w], res[m]);
    }
}


__global__ void gemm_half_q_half_alt_8bit_kernel(
    const half2* __restrict__ vec,
    const uint32_t* __restrict__ mat,
    half* __restrict__ mul,
    const half* __restrict__ scales,
    const uint32_t* __restrict__ zeros,
    const int* __restrict__ g_idx,
    int batch,
    int height,
    int width
)
{
    int zero_width = width / 4;
    int vec_height = height * 2;
    const int blockwidth2 = BLOCK_KN_SIZE / 2;
    int b = blockIdx.y * BLOCK_M_SIZE_MAX;
    int b_end = min(BLOCK_M_SIZE_MAX, batch - b);
    int h = BLOCK_KN_SIZE * blockIdx.z / 4;
    int h_end = min(BLOCK_KN_SIZE / 4, height - h) * 2;
    int w = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;

    __shared__ half2 blockvec[BLOCK_M_SIZE_MAX][blockwidth2];
    if (threadIdx.x < h_end) {
        for (int m = 0; m < b_end; ++m) {
          blockvec[m][threadIdx.x] =
              vec[(m + b) * vec_height + blockIdx.z * BLOCK_KN_SIZE / 2 +
                  threadIdx.x];
        }
    }


    if (blockIdx.z == 0)
    {
        for (int m = 0; m < b_end; m++)
            mul[(b + m) * width + w] = __int2half_rn(0);
    }
    __syncthreads();

    int i = width * h + w;
    int g_h = h * 4;
    int k = 0;
    int z_w = w / 4;
    int z_mod = (w % 4) * 8;
    half2 res2;
    half res[BLOCK_M_SIZE_MAX] = {};

    unsigned int tmp;
    while (k < h_end) {
        tmp = mat[i];
        half2 scales_tmp[2];
        half2 zeros_tmp[2];
        for (int tmp_k = 0; tmp_k < 2; tmp_k++) {
            int g = g_idx[g_h + (k + tmp_k) * 2];
            int g2 = g_idx[g_h + (k + tmp_k) * 2 + 1];
            half scale_f = scales[g * width + w];
            half scale_f2 = scales[g2 * width + w];
            half2 scale = __halves2half2(scale_f, scale_f2);
            half2 zero = __halves2half2(
                __hmul(scale_f, __int2half_rn(-((zeros[g * zero_width + z_w] >> z_mod) & 0xff) - 1)),
                __hmul(scale_f2, __int2half_rn(-((zeros[g2 * zero_width + z_w] >> z_mod) & 0xff) - 1))
            );
            scales_tmp[tmp_k] = scale;
            zeros_tmp[tmp_k] = zero;
        }
        for (int m = 0; m < b_end; m++) {
#ifndef USE_ROCM
            res2 = {};
#else
            res2.x = __half_as_ushort(__float2half(0));
            res2.y = __half_as_ushort(__float2half(0));
#endif
            half2 v12 = __halves2half2(__int2half_rn(tmp & 0xFF), __int2half_rn((tmp >> 8) & 0xFF));
            res2 = __hfma2(__hfma2(v12, scales_tmp[0], zeros_tmp[0]), blockvec[m][k + 0], res2);
            half2 v34 = __halves2half2(__int2half_rn((tmp >> 16) & 0xFF), __int2half_rn((tmp >> 24) & 0xFF));
            res2 = __hfma2(__hfma2(v34, scales_tmp[1], zeros_tmp[1]), blockvec[m][k + 1], res2);
#ifndef USE_ROCM
            res[m] = __hadd(res[m], __hadd(res2.x, res2.y));
#else
            res[m] = __hadd(res[m], __hadd(__ushort_as_half(res2.x), __ushort_as_half(res2.y)));
#endif
        }
        i += width;
        k += 2;
    }
    for (int m = 0; m < b_end; m++) {
        atomicAdd(&mul[(b + m) * width + w], res[m]);
    }
}

void gemm_half_q_half_alt
(
    const half* a,
    const uint32_t* b_q_weight,
    const uint32_t* b_gptq_qzeros,
    const half* b_gptq_scales,
    const int* b_g_idx,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    int bit
)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = DIVIDE(size_n, BLOCK_KN_SIZE);
    gridDim.y = DIVIDE(size_m, BLOCK_M_SIZE_MAX);
    gridDim.z = DIVIDE(size_k, BLOCK_KN_SIZE);

    auto kernel = gemm_half_q_half_alt_4bit_kernel;
    if (bit == 8) {
        kernel = gemm_half_q_half_alt_8bit_kernel;
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    kernel<<<gridDim, blockDim, 0, stream>>>
    (
        (const half2*) a,
        b_q_weight,
        c,
        b_gptq_scales,
        b_gptq_qzeros,
        b_g_idx,
        size_m,
        size_k / 32 * bit,
        size_n
    );
}

template<class T, int bit>
__global__ void reconstruct_gptq_kernel
(
    const uint32_t* __restrict__ w,
    const half* __restrict__ w_scales,
    const uint32_t* __restrict__ w_zeros,
    const int* __restrict__ g_idx,
    const int height,
    const int width,
    const int group,
    half* __restrict__ out
)
{
    // Start of block

    int column = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;
    int row = blockIdx.y * 32 / bit;
    if (column >= width) return;

    // Views

    MatrixView_half_rw out_(out, height, width);
    MatrixView_half w_scales_(w_scales, group, width);
    T w_zeros_(w_zeros, group, width);

    uint32_t w_read = w[blockIdx.y * width + column];
    half* out_ptr = out_.item_ptr(row, column);

    #pragma unroll
    for (int s = 0; s < 32; s += bit)
    {
        int group = g_idx[row + s / bit];
        half w_scale = w_scales_.item(group, column);
        uint32_t w_zero = w_zeros_.item(group, column) + 1;
        half w_item = __hmul(__int2half_rn((int)((w_read >> s) & ((1 << bit) - 1)) - w_zero), w_scale);
        *out_ptr = w_item; out_ptr += out_.width;
    }
}

__global__ void reconstruct_gptq_3bit_kernel
(
    const uint32_t* __restrict__ w,
    const half* __restrict__ w_scales,
    const uint32_t* __restrict__ w_zeros,
    const int* __restrict__ g_idx,
    const int height,
    const int width,
    const int group,
    half* __restrict__ out
)
{
    // Start of block
    int column = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;
    int row = blockIdx.y * 32;
    if (column >= width) return;

    // Views

    MatrixView_half_rw out_(out, height, width);
    MatrixView_half w_scales_(w_scales, group, width);
    MatrixView_q3_row w_zeros_(w_zeros, group, width);

    uint32_t w1 = w[(blockIdx.y * 3) * width + column];
    uint32_t w2 = w[(blockIdx.y * 3 + 1) * width + column];
    uint32_t w3 = w[(blockIdx.y * 3 + 2) * width + column];
    half* out_ptr = out_.item_ptr(row, column);

    #pragma unroll
    for (int i = 0; i < 32; i += 1)
    {
        int group = g_idx[row + i];
        half w_scale = w_scales_.item(group, column);
        uint32_t w_zero = w_zeros_.item(group, column) + 1;
        int w_item;
        if (i == 10) {
            w_item = (w1 >> 30) | ((w2 << 2) & 0x4);
        } else if (i == 21) {
            w_item = (w2 >> 31) | ((w3 << 1) & 0x6);
        } else if (i < 10) {
            w_item = ((w1 >> (i * 3)) & 0x7);
        } else if (i < 21) {
            w_item = ((w2 >> (i * 3 - 32)) & 0x7);
        } else {
            w_item = ((w3 >> (i * 3 - 64)) & 0x7);
        }
        *out_ptr = __hmul(__int2half_rn(w_item - w_zero), w_scale);
        out_ptr += out_.width;
    }
}

void reconstruct_gptq
(
    const uint32_t* b_q_weight,
    const uint32_t* b_gptq_qzeros,
    const half* b_gptq_scales,
    const int* b_g_idx,
    half* out,
    int height,
    int width,
    int groups,
    int bit
)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    gridDim.y = DIVIDE(height, 32 / bit);
    gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);

    auto kernel = reconstruct_gptq_kernel<MatrixView_q4_row, 4>;
    if (bit == 2) {
        kernel = reconstruct_gptq_kernel<MatrixView_q2_row, 2>;
    } else if (bit == 8) {
        kernel = reconstruct_gptq_kernel<MatrixView_q8_row, 8>;
    } else if (bit == 3) {
        kernel = reconstruct_gptq_3bit_kernel;
        gridDim.y = DIVIDE(height, 32);
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    kernel<<<gridDim, blockDim, 0, stream>>>
    (
        b_q_weight,
        b_gptq_scales,
        b_gptq_qzeros,
        b_g_idx,
        height,
        width,
        groups,
        out
    );
}


void gemm_half_q_half_cuda
(
    hipblasHandle_t cublas_handle,
    const half* a,
    const uint32_t* b_q_weight,
    const uint32_t* b_gptq_qzeros,
    const half* b_gptq_scales,
    const int* b_g_idx,
    half* c,
    half* temp_dq,
    int size_m,
    int size_n,
    int size_k,
    int groups,
    bool use_exllama,
    int bit
)
{
    bool use_reconstruct;
    if (use_exllama) {
        use_reconstruct = ((bit == 8 && size_m > MAX_Q_GEMM_ROWS_8BIT) || (bit != 8 && size_m > MAX_Q_GEMM_ROWS));
    } else {
        // The 2/3-bit kernels are somehow slower than dequant + gemm baseline, so we disabled them for now.
        use_reconstruct = (bit < 4 || size_m > MAX_ALT_GEMM_ROWS);
    }
    if (use_reconstruct) {
        // Reconstruct FP16 matrix, then cuBLAS
        if (use_exllama) {
            reconstruct_exllama(b_q_weight, b_gptq_qzeros, b_gptq_scales, b_g_idx, temp_dq,
                                size_k, size_n, groups, bit);
        }
        else
        {
            reconstruct_gptq(b_q_weight, b_gptq_qzeros, b_gptq_scales, b_g_idx,
                             temp_dq, size_k, size_n, groups, bit);
        }

        const half alpha = __float2half(1.0f);
        const half beta = __float2half(0.0f);
        hipblasHgemm(cublas_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_N,
                    size_n, size_m, size_k,
                    &alpha, temp_dq, size_n,
                            a,       size_k,
                    &beta,  c,       size_n);
    }
    else if (use_exllama)
    {
        // Quantized matmul
        int max_chunks = size_m / BLOCK_M_SIZE_MAX;
        int last_chunk = max_chunks * BLOCK_M_SIZE_MAX;
        int last_chunk_size = size_m - last_chunk;

        if (max_chunks)
        {
            gemm_half_q_half_cuda_part(a, b_q_weight, b_gptq_qzeros, b_gptq_scales, b_g_idx,
                                        c, last_chunk, size_n, size_k, BLOCK_M_SIZE_MAX,
                                        groups, bit);
        }

        if (last_chunk_size)
        {
            gemm_half_q_half_cuda_part(a + last_chunk * size_k, b_q_weight, b_gptq_qzeros,
                                        b_gptq_scales, b_g_idx, c + last_chunk * size_n,
                                        last_chunk_size, size_n, size_k, last_chunk_size,
                                        groups, bit);
        }
    }
    else
    {
        gemm_half_q_half_alt(a, b_q_weight, b_gptq_qzeros, b_gptq_scales, b_g_idx,
                             c, size_m, size_n, size_k, bit);
    }
}

__global__ void shuffle_4bit_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < size_k) { shuffle_4bit_8 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  8; }
}

__global__ void shuffle_8bit_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < size_k) { shuffle_8bit_4 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  4; }
}

__global__ void shuffle_2bit_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < size_k) { shuffle_2bit_16(b_ptr, size_n); b_ptr += 1 * size_n; k += 16;  }
}

__global__ void shuffle_3bit_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < size_k) { shuffle_3bit_32(b_ptr, size_n); b_ptr += 3 * size_n; k += 32;  }
}

__global__ void make_sequential_4bit_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const int* __restrict__ q_perm,
    const int w_width
)
{
    const uint64_t* w2 = (uint64_t*) w;
    uint64_t* w_new2 = (uint64_t*) w_new;
    int w2_stride = w_width >> 1;
    int w2_column = THREADS_X * blockIdx.x + threadIdx.x;
    if (w2_column >= w2_stride) return;
    int w_new2_row = blockIdx.y;
    int q_perm_idx = w_new2_row << 3;
    uint64_t dst = 0;

    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        int source_row = q_perm[q_perm_idx++];

        int w2_row = source_row >> 3;
        int w2_subrow = source_row & 0x07;
        int w2_row_shift = w2_subrow << 2;
        int wnew2_row_shift = i << 2;

        uint64_t src = w2[w2_row * w2_stride + w2_column];
        src >>= w2_row_shift;
        src &= 0x0000000f0000000f;
        src <<= wnew2_row_shift;
        dst |= src;
    }
    w_new2[w_new2_row * w2_stride + w2_column] = dst;
}

__global__ void make_sequential_2bit_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const int* __restrict__ q_perm,
    const int w_width
)
{
    const uint64_t* w2 = (uint64_t*) w;
    uint64_t* w_new2 = (uint64_t*) w_new;
    int w2_stride = w_width >> 1;
    int w2_column = THREADS_X * blockIdx.x + threadIdx.x;
    if (w2_column >= w2_stride) return;
    int w_new2_row = blockIdx.y;
    int q_perm_idx = w_new2_row << 4;
    uint64_t dst = 0;

    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        int source_row = q_perm[q_perm_idx++];

        int w2_row = source_row >> 4;
        int w2_subrow = source_row & 0x0f;
        int w2_row_shift = w2_subrow << 1;
        int wnew2_row_shift = i << 1;

        uint64_t src = w2[w2_row * w2_stride + w2_column];
        src >>= w2_row_shift;
        src &= 0x0000000300000003;
        src <<= wnew2_row_shift;
        dst |= src;
    }
    w_new2[w_new2_row * w2_stride + w2_column] = dst;
}

__global__ void make_sequential_3bit_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const int* __restrict__ q_perm,
    const int w_width
)
{
    int w_column = THREADS_X * blockIdx.x + threadIdx.x;
    if (w_column >= w_width) return;
    int w_new_row = blockIdx.y * 3;
    int q_perm_idx = blockIdx.y << 5;
    uint32_t dst[3] = {0, 0, 0};

    #pragma unroll
    for (int i = 0; i < 32; i++)
    {
        int source_row = q_perm[q_perm_idx++];
        int z_w = (source_row / 32) * 3;
        int z_mod = source_row % 32;
        int z_bit;

        if (z_mod != 10){
            if (z_mod != 21){
                z_bit = z_mod;
                if (z_bit > 21){
                    z_bit *= 3;
                    z_bit -= 64;
                    z_w += 2;
                } else if (z_bit > 10){
                    z_bit *= 3;
                    z_bit -= 32;
                    z_w += 1;
                } else {
                    z_bit *= 3;
                }
            } else {
                z_w += 1;
            }
        }

        uint64_t src;
        if (z_mod == 10) {
            src = (w[z_w * w_width + w_column] >> 30) | ((w[(z_w + 1) * w_width + w_column] << 2) & 0x4);
        } else if (z_mod == 21){
            src = (w[z_w * w_width + w_column] >> 31) | ((w[(z_w + 1) * w_width + w_column] << 1) & 0x6);
        } else {
            src = w[z_w * w_width + w_column];
            src >>= z_bit;
            src &= 0x07;
        }

        z_w = 0;
        if (i != 10){
            if (i != 21){
                z_bit = i;
                if (z_bit > 21){
                    z_bit *= 3;
                    z_bit -= 64;
                    z_w += 2;
                } else if (z_bit > 10){
                    z_bit *= 3;
                    z_bit -= 32;
                    z_w += 1;
                } else {
                    z_bit *= 3;
                }
            } else {
                z_w += 1;
            }
        }
        if (i == 10) {
            dst[z_w] |= (src & 0x03) << 30;
            dst[z_w + 1] |= ((src & 0x4) >> 2);
        } else if (i == 21) {
            dst[z_w] |= (src & 0x01) << 31;
            dst[z_w + 1] |= ((src & 0x6) >> 1);
        } else {
            dst[z_w] |= (src << z_bit);
        }
    }
    w_new[w_new_row * w_width + w_column] = dst[0];
    w_new[(w_new_row + 1) * w_width + w_column] = dst[1];
    w_new[(w_new_row + 2) * w_width + w_column] = dst[2];
}

__global__ void make_sequential_8bit_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const int* __restrict__ q_perm,
    const int w_width
)
{
    const uint64_t* w2 = (uint64_t*) w;
    uint64_t* w_new2 = (uint64_t*) w_new;
    int w2_stride = w_width >> 1;
    int w2_column = THREADS_X * blockIdx.x + threadIdx.x;
    if (w2_column >= w2_stride) return;
    int w_new2_row = blockIdx.y;
    int q_perm_idx = w_new2_row << 2;
    uint64_t dst = 0;

    #pragma unroll
    for (int i = 0; i < 4; i++)
    {
        int source_row = q_perm[q_perm_idx++];

        int w2_row = source_row >> 2;
        int w2_subrow = source_row & 0x03;
        int w2_row_shift = w2_subrow << 3;
        int wnew2_row_shift = i << 3;

        uint64_t src = w2[w2_row * w2_stride + w2_column];
        src >>= w2_row_shift;
        src &= 0x000000ff000000ff;
        src <<= wnew2_row_shift;
        dst |= src;
    }
    w_new2[w_new2_row * w2_stride + w2_column] = dst;
}


void shuffle_exllama_weight
(
    uint32_t* q_weight,
    int* q_perm,
    int height,
    int width,
    int bit
)
{
    if (q_perm)
    {
        uint32_t* new_qweight = NULL;
        hipMalloc(&new_qweight, height / 32 * bit * width * sizeof(uint32_t));

        dim3 blockDim, gridDim;
        blockDim.x = THREADS_X;
        blockDim.y = 1;
        gridDim.x = DIVIDE(width, THREADS_X);
        gridDim.y = height / 32 * bit;

        auto kernel = make_sequential_4bit_kernel;
        if (bit == 2) {
            kernel = make_sequential_2bit_kernel;
        } else if (bit == 3) {
            kernel = make_sequential_3bit_kernel;
            gridDim.y = height / 32;
        } else if (bit == 8) {
            kernel = make_sequential_8bit_kernel;
        }
        const hipStream_t stream = at::cuda::getCurrentCUDAStream();
        kernel<<<gridDim, blockDim, 0, stream>>>
        (
            q_weight,
            new_qweight,
            q_perm,
            width
        );
        // Replace qweights
        hipMemcpyAsync(q_weight, new_qweight, height / 32 * bit * width * sizeof(uint32_t), hipMemcpyDeviceToDevice);
        // Cleanup
        hipDeviceSynchronize();
        hipFree(new_qweight);
    }
    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, THREADS_X);
    gridDim.y = 1;
    auto shuffle_kernel = shuffle_4bit_kernel;
    if (bit == 2) {
        shuffle_kernel = shuffle_2bit_kernel;
    } else if (bit == 3) {
        shuffle_kernel = shuffle_3bit_kernel;
    } else if (bit == 8) {
        shuffle_kernel = shuffle_8bit_kernel;
    }
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    shuffle_kernel<<<gridDim, blockDim, 0, stream>>>(q_weight, height, width);
}

}  // namespace gptq
}  // namespace vllm

torch::Tensor gptq_gemm
(
    torch::Tensor a,
    torch::Tensor b_q_weight,
    torch::Tensor b_gptq_qzeros,
    torch::Tensor b_gptq_scales,
    torch::Tensor b_g_idx,
    bool use_exllama,
    int bit
)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
    auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
    at::Tensor c = torch::empty({a.size(0), b_q_weight.size(1)}, options);
    at::Tensor temp_dq = torch::empty({b_q_weight.size(0) * 32 / bit, b_q_weight.size(1)}, options);

    vllm::gptq::gemm_half_q_half_cuda
    (
        at::cuda::getCurrentCUDABlasHandle(),
        (const half*) a.data_ptr(),
        (const uint32_t*) b_q_weight.data_ptr(),
        (const uint32_t*)b_gptq_qzeros.data_ptr(),
        (const half*) b_gptq_scales.data_ptr(),
        b_g_idx.device().is_meta() ? NULL : (const int*) b_g_idx.data_ptr(),
        (half*) c.data_ptr(),
        (half*) temp_dq.data_ptr(),
        c.size(0),  // m
        c.size(1),  // n
        a.size(1),  // k
        b_gptq_qzeros.size(0),  // group number
        use_exllama,
        bit
    );
    return c;
}

void gptq_shuffle
(
    torch::Tensor q_weight,
    torch::Tensor q_perm,
    int bit
)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(q_weight));
    vllm::gptq::shuffle_exllama_weight(
        (uint32_t*) q_weight.data_ptr(),
        q_perm.device().is_meta() ? NULL : (int*) q_perm.data_ptr(),
        q_weight.size(0) * 32 / bit,
        q_weight.size(1),
        bit
    );
}
