#include "hip/hip_runtime.h"
/*
Adapted from `q_gemm.cu`, which is adapted from
https://github.com/turboderp/exllamav2 and
https://github.com/qwopqwop200/GPTQ-for-LLaMa.

This supports GPTQ v2 format checkpoints (checkpoint_format: 'gptq_v2'),
by removing the v1-specific "zero + 1" logic during dequantization.
Specifically, GPTQ v1 format checkpoints store (zero - 1), and need to + 1 at
runtime during dequantization. GPTQ v2 format checkpoints store the zero point
as is, and doesn't require + 1 at runtime. For more details, please refer to
ModelCloud/GPTQModel:
https://github.com/ModelCloud/GPTQModel/blob/020ac04b74f6263f22491e6a6a034cb4fa5bf181/gptqmodel/utils/model.py#L625
*/

#include <cstdint>
#include <cstdio>

#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "compat.cuh"
#include "matrix_view.cuh"
#include "qdq_2.cuh"
#include "qdq_3.cuh"
#include "qdq_4.cuh"
#include "qdq_8.cuh"

namespace vllm {
namespace gptq {

#define BLOCK_KN_SIZE 128
#define BLOCK_M_SIZE_MAX 8
#define MAX_GROUPS_IN_BLOCK (BLOCK_KN_SIZE / 32)
#define MAX_Q_GEMM_ROWS 50
#define MAX_Q_GEMM_ROWS_8BIT 24
#define MAX_ALT_GEMM_ROWS 8
#define THREADS_X 32
#define THREADS_Y 32
#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

#if defined(USE_ROCM)
  #include <hipblas/hipblas.h>
__host__ __forceinline__ hipblasStatus_t __compat_hipblasHgemm(
    hipblasHandle_t handle, hipblasOperation_t transA,
    hipblasOperation_t transB, int m, int n, int k, const half* alpha,
    const half* AP, int lda, const half* BP, int ldb, const half* beta,
    half* CP, int ldc) {
  return hipblasHgemm(handle, transA, transB, m, n, k,
                      reinterpret_cast<const hipblasHalf*>(alpha),
                      reinterpret_cast<const hipblasHalf*>(AP), lda,
                      reinterpret_cast<const hipblasHalf*>(BP), ldb,
                      reinterpret_cast<const hipblasHalf*>(beta),
                      reinterpret_cast<hipblasHalf*>(CP), ldc);
}
  #define hipblasHgemm __compat_hipblasHgemm

  // Previous version of PyTorch were converting to rocBLAS instead of hipBLAS.
  #define rocblas_operation_none HIPBLAS_OP_N
  #define rocblas_hgemm __compat_hipblasHgemm
#endif

__forceinline__ __device__ half2 dot22_8(half2 (&dq)[4], const half* a_ptr,
                                         const half2 g_result) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  return __hadd2(result, g_result);
}

__forceinline__ __device__ float dot22_8_f(half2 (&dq)[4], const half* a_ptr) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  return __half2float(__low2half(result)) + __half2float(__high2half(result));
}

__forceinline__ __device__ half2 dot22_8(half2 (&dq)[4], const half* a_ptr,
                                         const half2 g_result,
                                         const half qs_h) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  return __hfma2(result, __halves2half2(qs_h, qs_h), g_result);
}

__forceinline__ __device__ half2 dot22_16(half2 (&dq)[8], const half* a_ptr,
                                          const half2 g_result,
                                          const half qs_h) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 8; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  return __hfma2(result, __halves2half2(qs_h, qs_h), g_result);
}

__forceinline__ __device__ half2 dot22_32(half2 (&dq)[16], const half* a_ptr,
                                          const half2 g_result,
                                          const half qs_h) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 16; i += 1) result = __hfma2(dq[i], *a2_ptr++, result);
  return __hfma2(result, __halves2half2(qs_h, qs_h), g_result);
}

__forceinline__ __device__ float dot22_8_f(half2 (&dq)[4], const half* a_ptr,
                                           const float g_result,
                                           const float qs_f) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  float result_f =
      __half2float(__low2half(result)) + __half2float(__high2half(result));
  return fma(result_f, qs_f, g_result);
}

__forceinline__ __device__ float dot22_16_f(half2 (&dq)[8], const half* a_ptr,
                                            const float g_result,
                                            const float qs_f) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 8; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  float result_f =
      __half2float(__low2half(result)) + __half2float(__high2half(result));
  return fma(result_f, qs_f, g_result);
}

__forceinline__ __device__ float dot22_32_f(half2 (&dq)[16], const half* a_ptr,
                                            const float g_result,
                                            const float qs_f) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 16; i += 1) result = __hfma2(dq[i], *a2_ptr++, result);
  float result_f =
      __half2float(__low2half(result)) + __half2float(__high2half(result));
  return fma(result_f, qs_f, g_result);
}

__forceinline__ __device__ half dot22_8_h(half2 (&dq)[4], const half* a_ptr,
                                          const half g_result,
                                          const half qs_h) {
  // Use FP32 accumulator to avoid potential overflow since unscaled weights are
  // in the range -128..127

  float result = {};
#pragma unroll
  for (int i = 0; i < 4; i++) {
    half2 w01 = dq[i];
    float w0 = __low2float(w01);
    float w1 = __high2float(w01);
    float x0 = __half2float(*a_ptr++);
    float x1 = __half2float(*a_ptr++);
    result = fma(w0, x0, result);
    result = fma(w1, x1, result);
  }
  float qs = __half2float(qs_h);
  result *= qs;
  half result_h = __float2half_rn(result);
  return __hadd(result_h, g_result);
}

__forceinline__ __device__ half dot22_16_h(half2 (&dq)[8], const half* a_ptr,
                                           const half g_result,
                                           const half qs_h) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 8; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  half result_h = __hadd(__low2half(result), __high2half(result));
  return __hfma(result_h, qs_h, g_result);
}

__forceinline__ __device__ half dot22_32_h(half2 (&dq)[16], const half* a_ptr,
                                           const half g_result,
                                           const half qs_h) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 16; i += 1) result = __hfma2(dq[i], *a2_ptr++, result);
  half result_h = __hadd(__low2half(result), __high2half(result));
  return __hfma(result_h, qs_h, g_result);
}

typedef void (*fp_gemm_half_q_half_gptq_kernel)(const half*, const uint32_t*,
                                                const uint32_t*, const half*,
                                                half*, const int, const int,
                                                const int, const int,
                                                const int*);

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_4bit_kernel_v2(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups,
    const int* __restrict__ b_q_perm) {
  MatrixView_half a_(a, size_m, size_k);
  MatrixView_half_rw c_(c, size_m, size_n);
  MatrixView_q4_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto t = threadIdx.x;

  // Block
  auto offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
  auto offset_m = blockIdx.y * m_count;
  auto offset_k = blockIdx.z * BLOCK_KN_SIZE;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  int n = offset_n + t * 4;

  // Preload block_a
  __shared__ half block_a[m_count][BLOCK_KN_SIZE];

  if (offset_k + t < end_k) {
    for (int m = 0; m < m_count; ++m) {
      const half* a_ptr = a_.item_ptr(offset_m + m, 0);
      half* block_a_ptr = block_a[m];

      half a0;
      if (b_q_perm)
        a0 = a_ptr[b_q_perm[offset_k + t]];
      else
        a0 = a_ptr[offset_k + t];
      block_a_ptr[t] = a0;
    }
  }

  // Zero output
  if (n >= size_n) return;

  if (blockIdx.z == 0) {
    for (int m = 0; m < m_count; m++)
      *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
  }

  __syncthreads();

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // a, b offset
  int qk = offset_k / (32 / 4);

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
  const half* a_ptr = &block_a[0][0];
  int a_stride = BLOCK_KN_SIZE;

  // Initial group
  int zeros[4];
  float scales[4];
  half2 z1z16[4][2];
  half2 y1y16[4][2];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4_f(scales, group, n);
  dequant_4bit_8_prep_zero(zeros[0], z1z16[0], y1y16[0]);
  dequant_4bit_8_prep_zero(zeros[1], z1z16[1], y1y16[1]);
  dequant_4bit_8_prep_zero(zeros[2], z1z16[2], y1y16[2]);
  dequant_4bit_8_prep_zero(zeros[3], z1z16[3], y1y16[3]);

  // Column result
  float block_c[m_count][4] = {};

  // Dequantize and multiply
  int k = offset_k;
  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4_f(scales, group, n);
      dequant_4bit_8_prep_zero(zeros[0], z1z16[0], y1y16[0]);
      dequant_4bit_8_prep_zero(zeros[1], z1z16[1], y1y16[1]);
      dequant_4bit_8_prep_zero(zeros[2], z1z16[2], y1y16[2]);
      dequant_4bit_8_prep_zero(zeros[3], z1z16[3], y1y16[3]);
    }

#pragma unroll
    for (int j = 0; j < 4; j++) {
      const int4* b_ptr4 = (int4*)b_ptr;
      int4 load_int4 = *b_ptr4;

      half2 dq[4][4];
      dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n,
                          false);
      dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n,
                          false);
      dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n,
                          false);
      dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n,
                          false);

#pragma unroll
      for (int m = 0; m < m_count; m++) {
        block_c[m][0] = fma(dot22_8_f(dq[0], a_ptr + m * a_stride), scales[0],
                            block_c[m][0]);
        block_c[m][1] = fma(dot22_8_f(dq[1], a_ptr + m * a_stride), scales[1],
                            block_c[m][1]);
        block_c[m][2] = fma(dot22_8_f(dq[2], a_ptr + m * a_stride), scales[2],
                            block_c[m][2]);
        block_c[m][3] = fma(dot22_8_f(dq[3], a_ptr + m * a_stride), scales[3],
                            block_c[m][3]);
      }

      b_ptr += size_n;
      a_ptr += 8;
    }

    k += 32;
  }

  for (int m = 0; m < m_count; m++) {
    half2* out = (half2*)c_.item_ptr(offset_m + m, n);
    half2 result01 = __halves2half2(__float2half_rn(block_c[m][0]),
                                    __float2half_rn(block_c[m][1]));
    half2 result23 = __halves2half2(__float2half_rn(block_c[m][2]),
                                    __float2half_rn(block_c[m][3]));
    atomicAdd(out, result01);
    atomicAdd(out + 1, result23);
  }
}

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_2bit_kernel_v2(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups,
    const int* __restrict__ b_q_perm) {
  MatrixView_half a_(a, size_m, size_k);
  MatrixView_half_rw c_(c, size_m, size_n);
  MatrixView_q2_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto t = threadIdx.x;

  // Block
  auto offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
  auto offset_m = blockIdx.y * m_count;
  auto offset_k = blockIdx.z * BLOCK_KN_SIZE;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  int n = offset_n + t * 4;

  // Preload block_a
  __shared__ half block_a[m_count][BLOCK_KN_SIZE];

  if (offset_k + t < end_k) {
    for (int m = 0; m < m_count; ++m) {
      const half* a_ptr = a_.item_ptr(offset_m + m, 0);
      half* block_a_ptr = block_a[m];

      half a0;
      if (b_q_perm)
        a0 = a_ptr[b_q_perm[offset_k + t]];
      else
        a0 = a_ptr[offset_k + t];
      block_a_ptr[t] = a0;
    }
  }

  // Zero output
  if (n >= size_n) return;

  if (blockIdx.z == 0) {
    for (int m = 0; m < m_count; m++)
      *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
  }

  __syncthreads();

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // a, b offset
  int qk = offset_k / (32 / 2);

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
  const half* a_ptr = &block_a[0][0];
  int a_stride = BLOCK_KN_SIZE;

  // Initial group
  int zeros[4];
  half scales[4];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4(scales, group, n);
  // Column result
  half block_c[m_count][4] = {};

  // Dequantize and multiply
  int k = offset_k;
  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4(scales, group, n);
    }

#pragma unroll
    for (int j = 0; j < 1; j++) {
      const int4* b_ptr4 = (int4*)b_ptr;
      int4 load_int4 = *b_ptr4;

      half2 dq[4][8];
      dequant_2bit_16(load_int4.x, dq[0], size_n, zeros[0]);
      dequant_2bit_16(load_int4.y, dq[1], size_n, zeros[1]);
      dequant_2bit_16(load_int4.z, dq[2], size_n, zeros[2]);
      dequant_2bit_16(load_int4.w, dq[3], size_n, zeros[3]);

#pragma unroll
      for (int m = 0; m < m_count; m++) {
        block_c[m][0] =
            dot22_16_h(dq[0], a_ptr + m * a_stride, block_c[m][0], scales[0]);
        block_c[m][1] =
            dot22_16_h(dq[1], a_ptr + m * a_stride, block_c[m][1], scales[1]);
        block_c[m][2] =
            dot22_16_h(dq[2], a_ptr + m * a_stride, block_c[m][2], scales[2]);
        block_c[m][3] =
            dot22_16_h(dq[3], a_ptr + m * a_stride, block_c[m][3], scales[3]);
      }

      b_ptr += size_n;
      a_ptr += 16;
    }

    k += 16;
  }

  for (int m = 0; m < m_count; m++) {
    half2* out = (half2*)c_.item_ptr(offset_m + m, n);
    half2 result01 = __halves2half2(block_c[m][0], block_c[m][1]);
    half2 result23 = __halves2half2(block_c[m][2], block_c[m][3]);
    atomicAdd(out, result01);
    atomicAdd(out + 1, result23);
  }
}

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_3bit_kernel_v2(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups,
    const int* __restrict__ b_q_perm) {
  MatrixView_half a_(a, size_m, size_k);
  MatrixView_half_rw c_(c, size_m, size_n);
  MatrixView_q3_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto t = threadIdx.x;

  // Block
  auto offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
  auto offset_m = blockIdx.y * m_count;
  auto offset_k = blockIdx.z * BLOCK_KN_SIZE;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  int n = offset_n + t * 4;

  // Preload block_a
  __shared__ half block_a[m_count][BLOCK_KN_SIZE];

  if (offset_k + t < end_k) {
    for (int m = 0; m < m_count; ++m) {
      const half* a_ptr = a_.item_ptr(offset_m + m, 0);
      half* block_a_ptr = block_a[m];

      half a0;
      if (b_q_perm)
        a0 = a_ptr[b_q_perm[offset_k + t]];
      else
        a0 = a_ptr[offset_k + t];
      block_a_ptr[t] = a0;
    }
  }

  // Zero output
  if (n >= size_n) return;

  if (blockIdx.z == 0) {
    for (int m = 0; m < m_count; m++)
      *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
  }

  __syncthreads();

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // a, b offset
  int qk = offset_k / 32 * 3;

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
  const half* a_ptr = &block_a[0][0];
  int a_stride = BLOCK_KN_SIZE;

  // Initial group
  int zeros[4];
  half scales[4];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4(scales, group, n);
  // Column result
  half block_c[m_count][4] = {};

  // Dequantize and multiply
  int k = offset_k;
  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4(scales, group, n);
    }

#pragma unroll
    for (int j = 0; j < 1; j++) {
      int4 load_int4[3];
      load_int4[0] = *((int4*)b_ptr);
      b_ptr += size_n;
      load_int4[1] = *((int4*)b_ptr);
      b_ptr += size_n;
      load_int4[2] = *((int4*)b_ptr);
      b_ptr += size_n;

      half2 dq[4][16];
      dequant_3bit_32(load_int4[0].x, load_int4[1].x, load_int4[2].x, dq[0],
                      size_n, zeros[0]);
      dequant_3bit_32(load_int4[0].y, load_int4[1].y, load_int4[2].y, dq[1],
                      size_n, zeros[1]);
      dequant_3bit_32(load_int4[0].z, load_int4[1].z, load_int4[2].z, dq[2],
                      size_n, zeros[2]);
      dequant_3bit_32(load_int4[0].w, load_int4[1].w, load_int4[2].w, dq[3],
                      size_n, zeros[3]);

#pragma unroll
      for (int m = 0; m < m_count; m++) {
        block_c[m][0] =
            dot22_32_h(dq[0], a_ptr + m * a_stride, block_c[m][0], scales[0]);
        block_c[m][1] =
            dot22_32_h(dq[1], a_ptr + m * a_stride, block_c[m][1], scales[1]);
        block_c[m][2] =
            dot22_32_h(dq[2], a_ptr + m * a_stride, block_c[m][2], scales[2]);
        block_c[m][3] =
            dot22_32_h(dq[3], a_ptr + m * a_stride, block_c[m][3], scales[3]);
      }
      a_ptr += 32;
    }

    k += 32;
  }

  for (int m = 0; m < m_count; m++) {
    half2* out = (half2*)c_.item_ptr(offset_m + m, n);
    half2 result01 = __halves2half2(block_c[m][0], block_c[m][1]);
    half2 result23 = __halves2half2(block_c[m][2], block_c[m][3]);
    atomicAdd(out, result01);
    atomicAdd(out + 1, result23);
  }
}

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_8bit_kernel_v2(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups,
    const int* __restrict__ b_q_perm) {
  MatrixView_half a_(a, size_m, size_k);
  MatrixView_half_rw c_(c, size_m, size_n);
  MatrixView_q8_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto t = threadIdx.x;

  // Block
  auto offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
  auto offset_m = blockIdx.y * m_count;
  auto offset_k = blockIdx.z * BLOCK_KN_SIZE;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  int n = offset_n + t * 4;

  // Preload block_a
  __shared__ half block_a[m_count][BLOCK_KN_SIZE];

  if (offset_k + t < end_k) {
    for (int m = 0; m < m_count; ++m) {
      const half* a_ptr = a_.item_ptr(offset_m + m, 0);
      half* block_a_ptr = block_a[m];

      half a0;
      if (b_q_perm)
        a0 = a_ptr[b_q_perm[offset_k + t]];
      else
        a0 = a_ptr[offset_k + t];
      block_a_ptr[t] = a0;
    }
  }

  // Zero output
  if (n >= size_n) return;

  if (blockIdx.z == 0) {
    for (int m = 0; m < m_count; m++)
      *((uint64_t*)c_.item_ptr(offset_m + m, n)) = 0;
  }

  __syncthreads();

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // a, b offset
  int qk = offset_k / (32 / 8);

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
  const half* a_ptr = &block_a[0][0];
  int a_stride = BLOCK_KN_SIZE;

  // Initial group
  int zeros[4];
  half scales[4];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4(scales, group, n);
  // Column result
  half block_c[m_count][4] = {};

  // Dequantize and multiply
  int k = offset_k;
  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4(scales, group, n);
    }

#pragma unroll
    for (int j = 0; j < 4; j++) {
      int4 load_int4[2];
      load_int4[0] = *((int4*)b_ptr);
      b_ptr += size_n;
      load_int4[1] = *((int4*)b_ptr);
      b_ptr += size_n;

      half2 dq[4][4];
      dequant_8bit_8(load_int4[0].x, load_int4[1].x, dq[0], size_n, zeros[0]);
      dequant_8bit_8(load_int4[0].y, load_int4[1].y, dq[1], size_n, zeros[1]);
      dequant_8bit_8(load_int4[0].z, load_int4[1].z, dq[2], size_n, zeros[2]);
      dequant_8bit_8(load_int4[0].w, load_int4[1].w, dq[3], size_n, zeros[3]);

      for (int m = 0; m < m_count; m++) {
        block_c[m][0] =
            dot22_8_h(dq[0], a_ptr + m * a_stride, block_c[m][0], scales[0]);
        block_c[m][1] =
            dot22_8_h(dq[1], a_ptr + m * a_stride, block_c[m][1], scales[1]);
        block_c[m][2] =
            dot22_8_h(dq[2], a_ptr + m * a_stride, block_c[m][2], scales[2]);
        block_c[m][3] =
            dot22_8_h(dq[3], a_ptr + m * a_stride, block_c[m][3], scales[3]);
      }
      a_ptr += 8;
    }
    k += 32;
  }

  for (int m = 0; m < m_count; m++) {
    half2* out = (half2*)c_.item_ptr(offset_m + m, n);
    half2 result01 = __halves2half2(block_c[m][0], block_c[m][1]);
    half2 result23 = __halves2half2(block_c[m][2], block_c[m][3]);
    atomicAdd(out, result01);
    atomicAdd(out + 1, result23);
  }
}

fp_gemm_half_q_half_gptq_kernel pick_gemm_half_q_half_gptq_kernel_v2(
    bool first_block, const int m_count, const int bit) {
#define SELECT_KERNEL(M_COUNT)                                                \
  if (m_count == M_COUNT) {                                                   \
    if (bit == 2) return gemm_half_q_half_gptq_2bit_kernel_v2<true, M_COUNT>; \
    if (bit == 3) return gemm_half_q_half_gptq_3bit_kernel_v2<true, M_COUNT>; \
    if (bit == 4) return gemm_half_q_half_gptq_4bit_kernel_v2<true, M_COUNT>; \
    if (bit == 8) return gemm_half_q_half_gptq_8bit_kernel_v2<true, M_COUNT>; \
  }
#if BLOCK_M_SIZE_MAX >= 1
  SELECT_KERNEL(1);
#endif
#if BLOCK_M_SIZE_MAX >= 2
  SELECT_KERNEL(2);
#endif
#if BLOCK_M_SIZE_MAX >= 3
  SELECT_KERNEL(3);
#endif
#if BLOCK_M_SIZE_MAX >= 4
  SELECT_KERNEL(4);
#endif
#if BLOCK_M_SIZE_MAX >= 5
  SELECT_KERNEL(5);
#endif
#if BLOCK_M_SIZE_MAX >= 6
  SELECT_KERNEL(6);
#endif
#if BLOCK_M_SIZE_MAX >= 7
  SELECT_KERNEL(7);
#endif
#if BLOCK_M_SIZE_MAX >= 8
  SELECT_KERNEL(8);
#endif
  return NULL;
}

void gemm_half_q_half_cuda_part_v2(const half* a, const uint32_t* b_q_weight,
                                   const uint32_t* b_gptq_qzeros,
                                   const half* b_gptq_scales,
                                   const int* b_q_perm, half* c, int size_m,
                                   int size_n, int size_k, int m_count,
                                   int groups, int bit) {
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_KN_SIZE;
  blockDim.y = 1;
  blockDim.z = 1;
  gridDim.x = DIVIDE(size_n, BLOCK_KN_SIZE * 4);
  gridDim.y = DIVIDE(size_m, m_count);
  gridDim.z = DIVIDE(size_k, BLOCK_KN_SIZE);

  fp_gemm_half_q_half_gptq_kernel kernel =
      pick_gemm_half_q_half_gptq_kernel_v2(true, m_count, bit);

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  kernel<<<gridDim, blockDim, 0, stream>>>(a, b_q_weight, b_gptq_qzeros,
                                           b_gptq_scales, c, size_m, size_n,
                                           size_k, groups, b_q_perm);
}

__global__ void reconstruct_exllama_8bit_kernel_v2(
    const uint32_t* __restrict__ b_q_weight, const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, const int size_k, const int size_n,
    const int groups, half* __restrict__ b) {
  MatrixView_half_rw b_(b, size_k, size_n);
  MatrixView_q8_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto offset_k = BLOCK_KN_SIZE * blockIdx.y;
  auto offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  // Preload remapping table
  __shared__ int perm[BLOCK_KN_SIZE];
  auto t = threadIdx.x;

  if (b_q_perm) {
    if (offset_k + t < size_k) perm[t] = b_q_perm[offset_k + t];
  }

  // Column
  int n = offset_n + t * 4;
  if (n >= size_n) return;

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // b offset
  int qk = offset_k / (32 / 8);

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

  // Initial zeros/scale
  int zeros[4];
  half2 scales[4];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4_h2(scales, group, n);

  __syncthreads();

  int k = offset_k;
  int lk = 0;

  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4_h2(scales, group, n);
    }

    for (int p = 0; p < 4; p++) {
      int4 load_int4[2];
      load_int4[0] = *((int4*)b_ptr);
      b_ptr += size_n;
      load_int4[1] = *((int4*)b_ptr);
      b_ptr += size_n;

      half2 dq[4][4];
      dequant_8bit_8(load_int4[0].x, load_int4[1].x, dq[0], size_n, zeros[0]);
      dequant_8bit_8(load_int4[0].y, load_int4[1].y, dq[1], size_n, zeros[1]);
      dequant_8bit_8(load_int4[0].z, load_int4[1].z, dq[2], size_n, zeros[2]);
      dequant_8bit_8(load_int4[0].w, load_int4[1].w, dq[3], size_n, zeros[3]);

      // half* dqh = (half*)dq;
      if (b_q_perm) {
        for (int j = 0; j < 4; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]),
                  __low2half(dq[2][j]), __low2half(dq[3][j]));
          b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]),
                  __high2half(dq[2][j]), __high2half(dq[3][j]));
        }
      } else {
        for (int j = 0; j < 4; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(offset_k + lk++, n, __low2half(dq[0][j]),
                  __low2half(dq[1][j]), __low2half(dq[2][j]),
                  __low2half(dq[3][j]));
          b_.set4(offset_k + lk++, n, __high2half(dq[0][j]),
                  __high2half(dq[1][j]), __high2half(dq[2][j]),
                  __high2half(dq[3][j]));
        }
      }
    }
    k += 32;
  }
}

__global__ void reconstruct_exllama_4bit_kernel_v2(
    const uint32_t* __restrict__ b_q_weight, const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, const int size_k, const int size_n,
    const int groups, half* __restrict__ b) {
  MatrixView_half_rw b_(b, size_k, size_n);
  MatrixView_q4_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto offset_k = BLOCK_KN_SIZE * blockIdx.y;
  auto offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  // Preload remapping table
  __shared__ int perm[BLOCK_KN_SIZE];
  auto t = threadIdx.x;

  if (b_q_perm) {
    if (offset_k + t < size_k) perm[t] = b_q_perm[offset_k + t];
  }

  // Column
  int n = offset_n + t * 4;
  if (n >= size_n) return;

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // b offset
  int qk = offset_k / (32 / 4);

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

  // Initial zeros/scale
  int zeros[4];
  half2 scales[4];
  half2 z1z16[4][2];
  half2 y1y16[4][2];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4_h2(scales, group, n);
  dequant_4bit_8_prep_zero(zeros[0], z1z16[0], y1y16[0]);
  dequant_4bit_8_prep_zero(zeros[1], z1z16[1], y1y16[1]);
  dequant_4bit_8_prep_zero(zeros[2], z1z16[2], y1y16[2]);
  dequant_4bit_8_prep_zero(zeros[3], z1z16[3], y1y16[3]);

  __syncthreads();

  int k = offset_k;
  int lk = 0;

  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4_h2(scales, group, n);
      dequant_4bit_8_prep_zero(zeros[0], z1z16[0], y1y16[0]);
      dequant_4bit_8_prep_zero(zeros[1], z1z16[1], y1y16[1]);
      dequant_4bit_8_prep_zero(zeros[2], z1z16[2], y1y16[2]);
      dequant_4bit_8_prep_zero(zeros[3], z1z16[3], y1y16[3]);
    }

    for (int p = 0; p < 4; p++) {
      half2 dq[4][4];
      const int4* b_ptr4 = (int4*)b_ptr;
      int4 load_int4 = *b_ptr4;

      dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n,
                          false);
      dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n,
                          false);
      dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n,
                          false);
      dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n,
                          false);

      b_ptr += size_n;
      // half* dqh = (half*)dq;
      if (b_q_perm) {
        for (int j = 0; j < 4; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]),
                  __low2half(dq[2][j]), __low2half(dq[3][j]));
          b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]),
                  __high2half(dq[2][j]), __high2half(dq[3][j]));
        }
      } else {
        for (int j = 0; j < 4; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(offset_k + lk++, n, __low2half(dq[0][j]),
                  __low2half(dq[1][j]), __low2half(dq[2][j]),
                  __low2half(dq[3][j]));
          b_.set4(offset_k + lk++, n, __high2half(dq[0][j]),
                  __high2half(dq[1][j]), __high2half(dq[2][j]),
                  __high2half(dq[3][j]));
        }
      }
    }
    k += 32;
  }
}

__global__ void reconstruct_exllama_3bit_kernel_v2(
    const uint32_t* __restrict__ b_q_weight, const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, const int size_k, const int size_n,
    const int groups, half* __restrict__ b) {
  MatrixView_half_rw b_(b, size_k, size_n);
  MatrixView_q3_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto offset_k = BLOCK_KN_SIZE * blockIdx.y;
  auto offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  // Preload remapping table
  __shared__ int perm[BLOCK_KN_SIZE];
  auto t = threadIdx.x;

  if (b_q_perm) {
    if (offset_k + t < size_k) perm[t] = b_q_perm[offset_k + t];
  }

  // Column
  int n = offset_n + t * 4;
  if (n >= size_n) return;

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // b offset
  int qk = offset_k / 32 * 3;

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

  // Initial zeros/scale
  int zeros[4];
  half2 scales[4];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4_h2(scales, group, n);

  __syncthreads();

  int k = offset_k;
  int lk = 0;

  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4_h2(scales, group, n);
    }

    for (int p = 0; p < 1; p++) {
      int4 load_int4[3];
      load_int4[0] = *((int4*)b_ptr);
      b_ptr += size_n;
      load_int4[1] = *((int4*)b_ptr);
      b_ptr += size_n;
      load_int4[2] = *((int4*)b_ptr);
      b_ptr += size_n;

      half2 dq[4][16];
      dequant_3bit_32(load_int4[0].x, load_int4[1].x, load_int4[2].x, dq[0],
                      size_n, zeros[0]);
      dequant_3bit_32(load_int4[0].y, load_int4[1].y, load_int4[2].y, dq[1],
                      size_n, zeros[1]);
      dequant_3bit_32(load_int4[0].z, load_int4[1].z, load_int4[2].z, dq[2],
                      size_n, zeros[2]);
      dequant_3bit_32(load_int4[0].w, load_int4[1].w, load_int4[2].w, dq[3],
                      size_n, zeros[3]);

      if (b_q_perm) {
        for (int j = 0; j < 16; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]),
                  __low2half(dq[2][j]), __low2half(dq[3][j]));
          b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]),
                  __high2half(dq[2][j]), __high2half(dq[3][j]));
        }
      } else {
        for (int j = 0; j < 16; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(offset_k + lk++, n, __low2half(dq[0][j]),
                  __low2half(dq[1][j]), __low2half(dq[2][j]),
                  __low2half(dq[3][j]));
          b_.set4(offset_k + lk++, n, __high2half(dq[0][j]),
                  __high2half(dq[1][j]), __high2half(dq[2][j]),
                  __high2half(dq[3][j]));
        }
      }
    }
    k += 32;
  }
}

__global__ void reconstruct_exllama_2bit_kernel_v2(
    const uint32_t* __restrict__ b_q_weight, const int* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, const int size_k, const int size_n,
    const int groups, half* __restrict__ b) {
  MatrixView_half_rw b_(b, size_k, size_n);
  MatrixView_q2_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
  MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

  auto offset_k = BLOCK_KN_SIZE * blockIdx.y;
  auto offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

  // Preload remapping table
  __shared__ int perm[BLOCK_KN_SIZE];
  auto t = threadIdx.x;

  if (b_q_perm) {
    if (offset_k + t < size_k) perm[t] = b_q_perm[offset_k + t];
  }

  // Column
  int n = offset_n + t * 4;
  if (n >= size_n) return;

  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;

  // b offset
  int qk = offset_k / (32 / 2);

  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

  // Initial zeros/scale
  int zeros[4];
  half2 scales[4];
  b_gptq_qzeros_.item4(zeros, group, n);
  b_gptq_scales_.item4_h2(scales, group, n);

  __syncthreads();

  int k = offset_k;
  int lk = 0;

  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      b_gptq_qzeros_.item4(zeros, group, n);
      b_gptq_scales_.item4_h2(scales, group, n);
    }

    for (int p = 0; p < 2; p++) {
      const int4* b_ptr4 = (int4*)b_ptr;
      int4 load_int4 = *b_ptr4;

      half2 dq[4][8];
      dequant_2bit_16(load_int4.x, dq[0], size_n, zeros[0]);
      dequant_2bit_16(load_int4.y, dq[1], size_n, zeros[1]);
      dequant_2bit_16(load_int4.z, dq[2], size_n, zeros[2]);
      dequant_2bit_16(load_int4.w, dq[3], size_n, zeros[3]);

      b_ptr += size_n;
      // half* dqh = (half*)dq;
      if (b_q_perm) {
        for (int j = 0; j < 8; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]),
                  __low2half(dq[2][j]), __low2half(dq[3][j]));
          b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]),
                  __high2half(dq[2][j]), __high2half(dq[3][j]));
        }
      } else {
        for (int j = 0; j < 8; j++) {
          for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
          b_.set4(offset_k + lk++, n, __low2half(dq[0][j]),
                  __low2half(dq[1][j]), __low2half(dq[2][j]),
                  __low2half(dq[3][j]));
          b_.set4(offset_k + lk++, n, __high2half(dq[0][j]),
                  __high2half(dq[1][j]), __high2half(dq[2][j]),
                  __high2half(dq[3][j]));
        }
      }
    }
    k += 32;
  }
}

void reconstruct_exllama_v2(const uint32_t* b_q_weight,
                            const uint32_t* b_gptq_qzeros,
                            const half* b_gptq_scales, const int* b_q_perm,
                            half* out, int height, int width, int groups,
                            int bit) {
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_KN_SIZE;
  blockDim.y = 1;
  gridDim.y = DIVIDE(height, BLOCK_KN_SIZE);
  gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);

  auto reconstruct_exllama_kernel = reconstruct_exllama_4bit_kernel_v2;
  if (bit == 2) {
    reconstruct_exllama_kernel = reconstruct_exllama_2bit_kernel_v2;
  } else if (bit == 3) {
    reconstruct_exllama_kernel = reconstruct_exllama_3bit_kernel_v2;
  } else if (bit == 8) {
    reconstruct_exllama_kernel = reconstruct_exllama_8bit_kernel_v2;
  }

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  reconstruct_exllama_kernel<<<gridDim, blockDim, 0, stream>>>(
      b_q_weight, b_q_perm, b_gptq_qzeros, b_gptq_scales, height, width, groups,
      out);
}

__global__ void gemm_half_q_half_alt_4bit_kernel_v2(
    const half2* __restrict__ vec, const uint32_t* __restrict__ mat,
    half* __restrict__ mul, const half* __restrict__ scales,
    const uint32_t* __restrict__ zeros, const int* __restrict__ g_idx,
    int batch, int height, int width) {
  int zero_width = width / 8;
  int vec_height = height * 4;
  const int blockwidth2 = BLOCK_KN_SIZE / 2;
  auto b = blockIdx.y * BLOCK_M_SIZE_MAX;
  int b_end = min(BLOCK_M_SIZE_MAX, batch - b);
  auto h = BLOCK_KN_SIZE * blockIdx.z / 8;
  int h_end = min(BLOCK_KN_SIZE / 8, height - h) * 4;
  auto w = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;

  __shared__ half2 blockvec[BLOCK_M_SIZE_MAX][blockwidth2];
  if (threadIdx.x < h_end) {
    for (int m = 0; m < b_end; ++m) {
      blockvec[m][threadIdx.x] =
          vec[(m + b) * vec_height + blockIdx.z * BLOCK_KN_SIZE / 2 +
              threadIdx.x];
    }
  }

  __shared__ half2 deq2[256][8];
  auto val = threadIdx.x / 8;
  auto off = threadIdx.x % 8;
  for (; val < 256; val += BLOCK_KN_SIZE / 8) {
    deq2[val][off] =
        __halves2half2(__int2half_rn(val & 0xF), __int2half_rn(val >> 4));
  }

  if (blockIdx.z == 0) {
    for (int m = 0; m < b_end; m++) mul[(b + m) * width + w] = __int2half_rn(0);
  }
  __syncthreads();

  int i = width * h + w;
  int g_h = h * 8;
  int k = 0;
  int z_w = w / 8;
  int z_mod = (w % 8) * 4;
  half2 res2;
  half res[BLOCK_M_SIZE_MAX] = {};

  unsigned int tmp;
  while (k < h_end) {
    tmp = mat[i];
    half2 scales_tmp[4];
    half2 zeros_tmp[4];
    for (int tmp_k = 0; tmp_k < 4; tmp_k++) {
      int g = g_idx[g_h + (k + tmp_k) * 2];
      int g2 = g_idx[g_h + (k + tmp_k) * 2 + 1];
      half scale_f = scales[g * width + w];
      half scale_f2 = scales[g2 * width + w];
      half2 scale = __halves2half2(scale_f, scale_f2);
      half2 zero = __halves2half2(
          __hmul(scale_f, __int2half_rn(
                              -((zeros[g * zero_width + z_w] >> z_mod) & 0xF))),
          __hmul(
              scale_f2,
              __int2half_rn(-((zeros[g2 * zero_width + z_w] >> z_mod) & 0xF))));
      scales_tmp[tmp_k] = scale;
      zeros_tmp[tmp_k] = zero;
    }
    for (int m = 0; m < b_end; m++) {
#ifndef USE_ROCM
      res2 = {};
#else
      res2.x = __half_as_ushort(__float2half(0));
      res2.y = __half_as_ushort(__float2half(0));
#endif
      res2 = __hfma2(
          __hfma2(deq2[(tmp >> 0) & 0xff][off], scales_tmp[0], zeros_tmp[0]),
          blockvec[m][k + 0], res2);
      res2 = __hfma2(
          __hfma2(deq2[(tmp >> 8) & 0xff][off], scales_tmp[1], zeros_tmp[1]),
          blockvec[m][k + 1], res2);
      res2 = __hfma2(
          __hfma2(deq2[(tmp >> 16) & 0xff][off], scales_tmp[2], zeros_tmp[2]),
          blockvec[m][k + 2], res2);
      res2 = __hfma2(
          __hfma2(deq2[(tmp >> 24) & 0xff][off], scales_tmp[3], zeros_tmp[3]),
          blockvec[m][k + 3], res2);
#ifndef USE_ROCM
      res[m] = __hadd(res[m], __hadd(res2.x, res2.y));
#else
      res[m] = __hadd(
          res[m], __hadd(__ushort_as_half(res2.x), __ushort_as_half(res2.y)));
#endif
    }
    i += width;
    k += 4;
  }
  for (int m = 0; m < b_end; m++) {
    atomicAdd(&mul[(b + m) * width + w], res[m]);
  }
}

__global__ void gemm_half_q_half_alt_8bit_kernel_v2(
    const half2* __restrict__ vec, const uint32_t* __restrict__ mat,
    half* __restrict__ mul, const half* __restrict__ scales,
    const uint32_t* __restrict__ zeros, const int* __restrict__ g_idx,
    int batch, int height, int width) {
  int zero_width = width / 4;
  int vec_height = height * 2;
  const int blockwidth2 = BLOCK_KN_SIZE / 2;
  auto b = blockIdx.y * BLOCK_M_SIZE_MAX;
  int b_end = min(BLOCK_M_SIZE_MAX, batch - b);
  auto h = BLOCK_KN_SIZE * blockIdx.z / 4;
  int h_end = min(BLOCK_KN_SIZE / 4, height - h) * 2;
  auto w = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;

  __shared__ half2 blockvec[BLOCK_M_SIZE_MAX][blockwidth2];
  if (threadIdx.x < h_end) {
    for (int m = 0; m < b_end; ++m) {
      blockvec[m][threadIdx.x] =
          vec[(m + b) * vec_height + blockIdx.z * BLOCK_KN_SIZE / 2 +
              threadIdx.x];
    }
  }

  if (blockIdx.z == 0) {
    for (int m = 0; m < b_end; m++) mul[(b + m) * width + w] = __int2half_rn(0);
  }
  __syncthreads();

  int i = width * h + w;
  int g_h = h * 4;
  int k = 0;
  int z_w = w / 4;
  int z_mod = (w % 4) * 8;
  half2 res2;
  half res[BLOCK_M_SIZE_MAX] = {};

  unsigned int tmp;
  while (k < h_end) {
    tmp = mat[i];
    half2 scales_tmp[2];
    half2 zeros_tmp[2];
    for (int tmp_k = 0; tmp_k < 2; tmp_k++) {
      int g = g_idx[g_h + (k + tmp_k) * 2];
      int g2 = g_idx[g_h + (k + tmp_k) * 2 + 1];
      half scale_f = scales[g * width + w];
      half scale_f2 = scales[g2 * width + w];
      half2 scale = __halves2half2(scale_f, scale_f2);
      half2 zero = __halves2half2(
          __hmul(scale_f, __int2half_rn(-(
                              (zeros[g * zero_width + z_w] >> z_mod) & 0xff))),
          __hmul(scale_f2,
                 __int2half_rn(
                     -((zeros[g2 * zero_width + z_w] >> z_mod) & 0xff))));
      scales_tmp[tmp_k] = scale;
      zeros_tmp[tmp_k] = zero;
    }
    for (int m = 0; m < b_end; m++) {
#ifndef USE_ROCM
      res2 = {};
#else
      res2.x = __half_as_ushort(__float2half(0));
      res2.y = __half_as_ushort(__float2half(0));
#endif
      half2 v12 = __halves2half2(__int2half_rn(tmp & 0xFF),
                                 __int2half_rn((tmp >> 8) & 0xFF));
      res2 = __hfma2(__hfma2(v12, scales_tmp[0], zeros_tmp[0]),
                     blockvec[m][k + 0], res2);
      half2 v34 = __halves2half2(__int2half_rn((tmp >> 16) & 0xFF),
                                 __int2half_rn((tmp >> 24) & 0xFF));
      res2 = __hfma2(__hfma2(v34, scales_tmp[1], zeros_tmp[1]),
                     blockvec[m][k + 1], res2);
#ifndef USE_ROCM
      res[m] = __hadd(res[m], __hadd(res2.x, res2.y));
#else
      res[m] = __hadd(
          res[m], __hadd(__ushort_as_half(res2.x), __ushort_as_half(res2.y)));
#endif
    }
    i += width;
    k += 2;
  }
  for (int m = 0; m < b_end; m++) {
    atomicAdd(&mul[(b + m) * width + w], res[m]);
  }
}

void gemm_half_q_half_alt_v2(const half* a, const uint32_t* b_q_weight,
                             const uint32_t* b_gptq_qzeros,
                             const half* b_gptq_scales, const int* b_g_idx,
                             half* c, int size_m, int size_n, int size_k,
                             int bit) {
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_KN_SIZE;
  blockDim.y = 1;
  blockDim.z = 1;
  gridDim.x = DIVIDE(size_n, BLOCK_KN_SIZE);
  gridDim.y = DIVIDE(size_m, BLOCK_M_SIZE_MAX);
  gridDim.z = DIVIDE(size_k, BLOCK_KN_SIZE);

  auto kernel = gemm_half_q_half_alt_4bit_kernel_v2;
  if (bit == 8) {
    kernel = gemm_half_q_half_alt_8bit_kernel_v2;
  }

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  kernel<<<gridDim, blockDim, 0, stream>>>(
      (const half2*)a, b_q_weight, c, b_gptq_scales, b_gptq_qzeros, b_g_idx,
      size_m, size_k / 32 * bit, size_n);
}

template <class T, int bit>
__global__ void reconstruct_gptq_kernel_v2(const uint32_t* __restrict__ w,
                                           const half* __restrict__ w_scales,
                                           const uint32_t* __restrict__ w_zeros,
                                           const int* __restrict__ g_idx,
                                           const int height, const int width,
                                           const int group,
                                           half* __restrict__ out) {
  // Start of block

  auto column = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;
  auto row = blockIdx.y * 32 / bit;
  if (column >= width) return;

  // Views

  MatrixView_half_rw out_(out, height, width);
  MatrixView_half w_scales_(w_scales, group, width);
  T w_zeros_(w_zeros, group, width);

  uint32_t w_read = w[blockIdx.y * width + column];
  half* out_ptr = out_.item_ptr(row, column);

#pragma unroll
  for (int s = 0; s < 32; s += bit) {
    int group = g_idx[row + s / bit];
    half w_scale = w_scales_.item(group, column);
    uint32_t w_zero = w_zeros_.item(group, column);
    half w_item =
        __hmul(__int2half_rn((int)((w_read >> s) & ((1 << bit) - 1)) - w_zero),
               w_scale);
    *out_ptr = w_item;
    out_ptr += out_.width;
  }
}

__global__ void reconstruct_gptq_3bit_kernel_v2(
    const uint32_t* __restrict__ w, const half* __restrict__ w_scales,
    const uint32_t* __restrict__ w_zeros, const int* __restrict__ g_idx,
    const int height, const int width, const int group,
    half* __restrict__ out) {
  // Start of block
  auto column = BLOCK_KN_SIZE * blockIdx.x + threadIdx.x;
  auto row = blockIdx.y * 32;
  if (column >= width) return;

  // Views

  MatrixView_half_rw out_(out, height, width);
  MatrixView_half w_scales_(w_scales, group, width);
  MatrixView_q3_row w_zeros_(w_zeros, group, width);

  uint32_t w1 = w[(blockIdx.y * 3) * width + column];
  uint32_t w2 = w[(blockIdx.y * 3 + 1) * width + column];
  uint32_t w3 = w[(blockIdx.y * 3 + 2) * width + column];
  half* out_ptr = out_.item_ptr(row, column);

#pragma unroll
  for (int i = 0; i < 32; i += 1) {
    int group = g_idx[row + i];
    half w_scale = w_scales_.item(group, column);
    uint32_t w_zero = w_zeros_.item(group, column);
    int w_item;
    if (i == 10) {
      w_item = (w1 >> 30) | ((w2 << 2) & 0x4);
    } else if (i == 21) {
      w_item = (w2 >> 31) | ((w3 << 1) & 0x6);
    } else if (i < 10) {
      w_item = ((w1 >> (i * 3)) & 0x7);
    } else if (i < 21) {
      w_item = ((w2 >> (i * 3 - 32)) & 0x7);
    } else {
      w_item = ((w3 >> (i * 3 - 64)) & 0x7);
    }
    *out_ptr = __hmul(__int2half_rn(w_item - w_zero), w_scale);
    out_ptr += out_.width;
  }
}

void reconstruct_gptq_v2(const uint32_t* b_q_weight,
                         const uint32_t* b_gptq_qzeros,
                         const half* b_gptq_scales, const int* b_g_idx,
                         half* out, int height, int width, int groups,
                         int bit) {
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_KN_SIZE;
  blockDim.y = 1;
  gridDim.y = DIVIDE(height, 32 / bit);
  gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);

  auto kernel = reconstruct_gptq_kernel_v2<MatrixView_q4_row, 4>;
  if (bit == 2) {
    kernel = reconstruct_gptq_kernel_v2<MatrixView_q2_row, 2>;
  } else if (bit == 8) {
    kernel = reconstruct_gptq_kernel_v2<MatrixView_q8_row, 8>;
  } else if (bit == 3) {
    kernel = reconstruct_gptq_3bit_kernel_v2;
    gridDim.y = DIVIDE(height, 32);
  }

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  kernel<<<gridDim, blockDim, 0, stream>>>(b_q_weight, b_gptq_scales,
                                           b_gptq_qzeros, b_g_idx, height,
                                           width, groups, out);
}

void gemm_half_q_half_cuda_v2(hipblasHandle_t cublas_handle, const half* a,
                              const uint32_t* b_q_weight,
                              const uint32_t* b_gptq_qzeros,
                              const half* b_gptq_scales, const int* b_g_idx,
                              half* c, half* temp_dq, int size_m, int size_n,
                              int size_k, int groups, bool use_exllama,
                              int bit) {
  bool use_reconstruct;
  if (use_exllama) {
    use_reconstruct = ((bit == 8 && size_m > MAX_Q_GEMM_ROWS_8BIT) ||
                       (bit != 8 && size_m > MAX_Q_GEMM_ROWS));
  } else {
    // The 2/3-bit kernels are somehow slower than dequant + gemm baseline, so
    // we disabled them for now.
    use_reconstruct = (bit < 4 || size_m > MAX_ALT_GEMM_ROWS);
  }
  if (use_reconstruct) {
    // Reconstruct FP16 matrix, then cuBLAS
    if (use_exllama) {
      reconstruct_exllama_v2(b_q_weight, b_gptq_qzeros, b_gptq_scales, b_g_idx,
                             temp_dq, size_k, size_n, groups, bit);
    } else {
      reconstruct_gptq_v2(b_q_weight, b_gptq_qzeros, b_gptq_scales, b_g_idx,
                          temp_dq, size_k, size_n, groups, bit);
    }

    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);
    hipblasHgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size_n, size_m, size_k,
                &alpha, temp_dq, size_n, a, size_k, &beta, c, size_n);
  } else if (use_exllama) {
    // Quantized matmul
    int max_chunks = size_m / BLOCK_M_SIZE_MAX;
    int last_chunk = max_chunks * BLOCK_M_SIZE_MAX;
    int last_chunk_size = size_m - last_chunk;

    if (max_chunks) {
      gemm_half_q_half_cuda_part_v2(a, b_q_weight, b_gptq_qzeros, b_gptq_scales,
                                    b_g_idx, c, last_chunk, size_n, size_k,
                                    BLOCK_M_SIZE_MAX, groups, bit);
    }

    if (last_chunk_size) {
      gemm_half_q_half_cuda_part_v2(
          a + last_chunk * size_k, b_q_weight, b_gptq_qzeros, b_gptq_scales,
          b_g_idx, c + last_chunk * size_n, last_chunk_size, size_n, size_k,
          last_chunk_size, groups, bit);
    }
  } else {
    gemm_half_q_half_alt_v2(a, b_q_weight, b_gptq_qzeros, b_gptq_scales,
                            b_g_idx, c, size_m, size_n, size_k, bit);
  }
}

}  // namespace gptq
}  // namespace vllm

torch::Tensor gptq_gemm_v2(torch::Tensor a, torch::Tensor b_q_weight,
                           torch::Tensor b_gptq_qzeros,
                           torch::Tensor b_gptq_scales, torch::Tensor b_g_idx,
                           bool use_exllama, int64_t bit) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  at::Tensor c = torch::empty({a.size(0), b_q_weight.size(1)}, options);
  at::Tensor temp_dq = torch::empty(
      {b_q_weight.size(0) * 32 / bit, b_q_weight.size(1)}, options);

  vllm::gptq::gemm_half_q_half_cuda_v2(
      at::cuda::getCurrentCUDABlasHandle(), (const half*)a.data_ptr(),
      (const uint32_t*)b_q_weight.data_ptr(),
      (const uint32_t*)b_gptq_qzeros.data_ptr(),
      (const half*)b_gptq_scales.data_ptr(),
      b_g_idx.device().is_meta() ? NULL : (const int*)b_g_idx.data_ptr(),
      (half*)c.data_ptr(), (half*)temp_dq.data_ptr(),
      c.size(0),              // m
      c.size(1),              // n
      a.size(1),              // k
      b_gptq_qzeros.size(0),  // group number
      use_exllama, bit);
  return c;
}