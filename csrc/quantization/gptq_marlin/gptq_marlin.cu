#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Adapted from https://github.com/IST-DASLab/marlin
 */

#include "gptq_marlin.cuh"

template <typename T> inline std::string str(T x) { return std::to_string(x); }

namespace gptq_marlin {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

__global__ void permute_cols_kernel(int4 const *__restrict__ a_int4_ptr,
                                    int const *__restrict__ perm_int_ptr,
                                    int4 *__restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {}

template <const int num_bits,        // number of bits used for weights
          const int threads,         // number of threads in a threadblock
          const int thread_m_blocks, // number of 16x16 blocks in the m
                                     // dimension (batchsize) of the threadblock
          const int thread_n_blocks, // same for n dimension (output)
          const int thread_k_blocks, // same for k dimension (reduction)
          const int stages, // number of stages for the async global->shared
                            // fetch pipeline
          const bool has_act_order,   // whether act_order is enabled
          const int group_blocks = -1 // number of consecutive 16x16 blocks with
                                      // a separate quantization scale
          >
__global__ void
Marlin(const int4 *__restrict__ A, // fp16 input matrix of shape mxk
       const int4 *__restrict__ B, // 4bit quantized weight matrix of shape kxn
       int4 *__restrict__ C,       // fp16 output buffer of shape mxn
       const int4 *__restrict__ scales_ptr, // fp16 quantization scales of shape
                                            // (k/groupsize)xn
       const int *__restrict__ g_idx,       // int32 group indices of shape k
       int num_groups, // number of scale groups per output channel
       int prob_m,     // batch dimension m
       int prob_n,     // output dimension n
       int prob_k,     // reduction dimension k
       int *locks      // extra global storage for barrier synchronization
) {}

} // namespace gptq_marlin

torch::Tensor gptq_marlin_gemm(torch::Tensor &a, torch::Tensor &b_q_weight,
                               torch::Tensor &b_scales, torch::Tensor &g_idx,
                               torch::Tensor &perm, torch::Tensor &workspace,
                               int64_t num_bits, int64_t size_m, int64_t size_n,
                               int64_t size_k, bool is_k_full) {
  TORCH_CHECK_NOT_IMPLEMENTED(false,
                              "marlin_gemm(..) requires CUDA_ARCH >= 8.0");
  return torch::empty({1, 1});
}

#else

// Matrix fragments for tensor core instructions; their precise layout is
// documented here:
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-floating-point-type
using FragA = Vec<half2, 4>;
using FragB = Vec<half2, 2>;
using FragC = Vec<float, 4>;
using FragS = Vec<half2, 1>; // quantization scales

// m16n8k16 tensor core mma instruction with fp16 inputs and fp32
// output/accumulation.
__device__ inline void mma(const FragA &a_frag, const FragB &frag_b,
                           FragC &frag_c) {
  const uint32_t *a = reinterpret_cast<const uint32_t *>(&a_frag);
  const uint32_t *b = reinterpret_cast<const uint32_t *>(&frag_b);
  float *c = reinterpret_cast<float *>(&frag_c);
  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
               : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
               : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]),
                 "r"(b[1]), "f"(c[0]), "f"(c[1]), "f"(c[2]), "f"(c[3]));
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared
// memory, directly in tensor core layout.
__device__ inline void ldsm4(FragA &frag_a, const void *smem_ptr) {
  uint32_t *a = reinterpret_cast<uint32_t *>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
               : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3])
               : "r"(smem));
}

// Lookup-table based 3-input logical operation; explicitly used for
// dequantization as the compiler does not seem to automatically recognize it in
// all cases.
template <int lut> __device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(res)
               : "r"(a), "r"(b), "r"(c), "n"(lut));
  return res;
}

// Constructs destination register by taking bytes from 2 sources (based on mask)
template <int start_byte, int mask>
__device__ inline uint32_t prmt(uint32_t a) {
  uint32_t res;
  asm volatile("prmt.b32 %0, %1, %2, %3;\n"
               : "=r"(res)
               : "r"(a), "n"(start_byte), "n"(mask));
  return res;
}

// Efficiently dequantize an int32 value into a full B-fragment of 4 fp16
// values. We mostly follow the strategy in the link below, with some small
// changes:
// https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
__device__ inline FragB dequant_4bit(int q) {
  const int LO = 0x000f000f;
  const int HI = 0x00f000f0;
  const int EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  int lo = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  int hi = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  // We want signed int4 outputs, hence we fuse the `-8` symmetric zero point
  // directly into `SUB` and `ADD`.
  const int SUB = 0x64086408;
  const int MUL = 0x2c002c00;
  const int ADD = 0xd480d480;
  FragB frag_b;
  frag_b[0] = __hsub2(*reinterpret_cast<half2 *>(&lo),
                      *reinterpret_cast<const half2 *>(&SUB));
  frag_b[1] = __hfma2(*reinterpret_cast<half2 *>(&hi),
                      *reinterpret_cast<const half2 *>(&MUL),
                      *reinterpret_cast<const half2 *>(&ADD));
  return frag_b;
}

__device__ inline FragB dequant_8bit(int q) {
  static constexpr uint32_t mask_for_elt_01 = 0x5250;
  static constexpr uint32_t mask_for_elt_23 = 0x5351;
  static constexpr uint32_t start_byte_for_fp16 = 0x64646464;

  uint32_t lo = prmt<start_byte_for_fp16, mask_for_elt_01>(q);
  uint32_t hi = prmt<start_byte_for_fp16, mask_for_elt_23>(q);

  static constexpr uint32_t I8s_TO_F16s_MAGIC_NUM = 0x64806480;

  FragB frag_b;
  frag_b[0] = __hsub2(*reinterpret_cast<half2 *>(&lo),
                      *reinterpret_cast<const half2 *>(&I8s_TO_F16s_MAGIC_NUM));
  frag_b[1] = __hsub2(*reinterpret_cast<half2 *>(&hi),
                      *reinterpret_cast<const half2 *>(&I8s_TO_F16s_MAGIC_NUM));
  return frag_b;
}

// Multiply dequantized values by the corresponding quantization scale; used
// only for grouped quantization.
__device__ inline void scale(FragB &frag_b, FragS &frag_s, int i) {
  half2 s = __half2half2(reinterpret_cast<__half *>(&frag_s)[i]);
  frag_b[0] = __hmul2(frag_b[0], s);
  frag_b[1] = __hmul2(frag_b[1], s);
}

// Same as above, but for act_order (each K is multiplied individually)
__device__ inline void scale4(FragB &frag_b, FragS &frag_s_1, FragS &frag_s_2,
                              FragS &frag_s_3, FragS &frag_s_4, int i) {
  __half2 s_val_1_2;
  s_val_1_2.x = reinterpret_cast<__half *>(&frag_s_1)[i];
  s_val_1_2.y = reinterpret_cast<__half *>(&frag_s_2)[i];

  __half2 s_val_3_4;
  s_val_3_4.x = reinterpret_cast<__half *>(&frag_s_3)[i];
  s_val_3_4.y = reinterpret_cast<__half *>(&frag_s_4)[i];

  frag_b[0] = __hmul2(frag_b[0], s_val_1_2);
  frag_b[1] = __hmul2(frag_b[1], s_val_3_4);
}

// Given 2 floats multiply by 2 scales (halves)
__device__ inline void scale_float(float *c, FragS &s) {
  __half *s_ptr = reinterpret_cast<__half *>(&s);
  c[0] = __fmul_rn(c[0], __half2float(s_ptr[0]));
  c[1] = __fmul_rn(c[1], __half2float(s_ptr[1]));
}

// Wait until barrier reaches `count`, then lock for current threadblock.
__device__ inline void barrier_acquire(int *lock, int count) {
  if (threadIdx.x == 0) {
    int state = -1;
    do
      // Guarantee that subsequent writes by this threadblock will be visible
      // globally.
      asm volatile("ld.global.acquire.gpu.b32 %0, [%1];\n"
                   : "=r"(state)
                   : "l"(lock));
    while (state != count);
  }
  __syncthreads();
}

// Release barrier and increment visitation count.
__device__ inline void barrier_release(int *lock, bool reset = false) {
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    // Make sure that all writes since acquiring this barrier are visible
    // globally, while releasing the barrier.
    asm volatile("fence.acq_rel.gpu;\n");
    asm volatile("red.relaxed.gpu.global.add.s32 [%0], %1;\n"
                 :
                 : "l"(lock), "r"(val));
  }
}

// For a given "a" of size [M,K] performs a permutation of the K columns based
// on the given "perm" indices.
__global__ void permute_cols_kernel(int4 const *__restrict__ a_int4_ptr,
                                    int const *__restrict__ perm_int_ptr,
                                    int4 *__restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {

  int start_row = block_rows * blockIdx.x;
  int finish_row = start_row + block_rows;
  if (finish_row > size_m) {
    finish_row = size_m;
  }
  int cur_block_rows = finish_row - start_row;

  int row_stride = size_k * sizeof(half) / 16;

  auto permute_row = [&](int row) {
    int iters = size_k / default_threads;
    int rest = size_k % default_threads;

    int offset = row * row_stride;

    half const *a_row_half =
        reinterpret_cast<half const *>(a_int4_ptr + offset);
    half *out_half = reinterpret_cast<half *>(out_int4_ptr + offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      int cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];

      out_half[cur_k] = a_row_half[src_pos];

      base_k += default_threads;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        int cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];

        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int i = 0; i < cur_block_rows; i++) {
    int cur_row = start_row + i;
    if (cur_row < size_m) {
      permute_row(cur_row);
    }
  }
}

template <const int num_bits,        // number of bits used for weights
          const int threads,         // number of threads in a threadblock
          const int thread_m_blocks, // number of 16x16 blocks in the m
                                     // dimension (batchsize) of the threadblock
          const int thread_n_blocks, // same for n dimension (output)
          const int thread_k_blocks, // same for k dimension (reduction)
          const int stages, // number of stages for the async global->shared
                            // fetch pipeline
          const bool has_act_order,   // whether act_order is enabled
          const int group_blocks = -1 // number of consecutive 16x16 blocks with
                                      // a separate quantization scale
          >
__global__ void
Marlin(const int4 *__restrict__ A, // fp16 input matrix of shape mxk
       const int4 *__restrict__ B, // 4bit quantized weight matrix of shape kxn
       int4 *__restrict__ C,       // fp16 output buffer of shape mxn
       const int4 *__restrict__ scales_ptr, // fp16 quantization scales of shape
                                            // (k/groupsize)xn
       const int *__restrict__ g_idx,       // int32 group indices of shape k
       int num_groups, // number of scale groups per output channel
       int prob_m,     // batch dimension m
       int prob_n,     // output dimension n
       int prob_k,     // reduction dimension k
       int *locks      // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the
  // same size, which might involve multiple column "slices" (of width 16 *
  // `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM
  // example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it
  // ensures good utilization of all SMs for many kinds of shape and GPU
  // configurations, while requiring as few slow global cross-threadblock
  // reductions as possible.

  constexpr int pack_factor = 32 / num_bits;

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a
  // better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = div_ceil(k_tiles * n_tiles * parallel, gridDim.x);

  if constexpr (!has_act_order && group_blocks != -1) {
    if (group_blocks >= thread_k_blocks) {
      // Ensure that the number of tiles in each stripe is a multiple of the
      // groupsize; this avoids an annoying special case where a stripe starts
      // in the middle of group.
      iters = (group_blocks / thread_k_blocks) *
              div_ceil(iters, (group_blocks / thread_k_blocks));
    }
  }

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters; // number of threadblock tiles in the current slice
  int slice_count =
      0;         // total number of active threadblocks in the current slice
  int slice_idx; // index of threadblock in current slice; numbered bottom to
                 // top

  // We can easily implement parallel problem execution by just remapping
  // indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for
  // synchronization.
  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel)
      slice_iters = 0;
    if (slice_iters == 0)
      return;
    if (slice_row + slice_iters > k_tiles)
      slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * div_ceil(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = div_ceil(k_tiles - col_off, iters);
      if (col_off > 0)
        slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0)
          slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  // A sizes/strides

  // stride of the A matrix in global memory
  int a_gl_stride = prob_k / 8;
  // stride of an A matrix tile in shared memory
  constexpr int a_sh_stride = 16 * thread_k_blocks / 8;
  // delta between subsequent A tiles in global memory
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 8;
  // between subsequent accesses within a tile
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o);
  // between shared memory writes
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o);
  // between shared memory tile reads
  constexpr int a_sh_rd_delta_o = 2 * ((threads / 32) / (thread_n_blocks / 4));
  // within a shared memory tile
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16;
  // overall size of a tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks);
  // number of shared write iterations for a tile
  constexpr int a_sh_wr_iters = div_ceil(a_sh_stage, a_sh_wr_delta);

  // B sizes/strides
  int b_gl_stride = 16 * prob_n / (pack_factor * 4);
  constexpr int b_sh_stride = ((thread_n_blocks * 16) * 16 / pack_factor) / 4;
  constexpr int b_thread_vecs = num_bits == 4 ? 1 : 2;
  constexpr int b_sh_stride_threads = b_sh_stride / b_thread_vecs;

  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride_threads);
  constexpr int b_sh_wr_delta = threads * b_thread_vecs;
  constexpr int b_sh_rd_delta = threads * b_thread_vecs;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  // Scale sizes/strides without act_order
  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_tb_groups =
      !has_act_order && group_blocks != -1 && group_blocks < thread_k_blocks
          ? thread_k_blocks / group_blocks
          : 1;
  constexpr int s_sh_stage = s_tb_groups * s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  // Scale size/strides with act_order
  constexpr int tb_k = 16 * thread_k_blocks;
  constexpr int g_idx_stage = has_act_order ? (tb_k * sizeof(int)) / 16 : 0;
  // constexpr int act_s_row_stride      = 1;
  // int           act_s_col_stride      = act_s_row_stride * num_groups;
  int act_s_col_stride = 1;
  int act_s_col_warp_stride = act_s_col_stride * 8;
  int tb_n_warps = thread_n_blocks / 4;
  int act_s_col_tb_stride = act_s_col_warp_stride * tb_n_warps;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  int a_sh_rd =
      a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride_threads) +
                (threadIdx.x % b_sh_stride_threads) * b_thread_vecs;
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x * b_thread_vecs;
  int b_sh_rd = threadIdx.x * b_thread_vecs;

  // For act_order
  constexpr int k_iter_size = tb_k / b_sh_wr_iters;
  int slice_k_start = tb_k * slice_row;
  int slice_k_finish = slice_k_start + tb_k * slice_iters;
  int slice_k_start_shared_fetch = slice_k_start;
  int slice_n_offset = act_s_col_tb_stride * slice_col;

  // No act_order
  int s_gl_rd;
  if constexpr (!has_act_order) {
    if constexpr (group_blocks == -1) {
      s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
    } else {
      s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
                s_sh_stride * slice_col + threadIdx.x;
    }
  }
  int s_sh_wr = threadIdx.x;
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  // We use a different scale layout for grouped and column-wise quantization as
  // we scale a `half2` tile in column-major layout in the former and in
  // row-major in the latter case.
  int s_sh_rd;
  if constexpr (group_blocks != -1)
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) / 4;
  else
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) % 4;

  // Precompute which thread should not read memory in which iterations; this is
  // needed if there are more threads than required for a certain tilesize or
  // when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
#pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;

  // To ensure that writing and reading A tiles to/from shared memory, the
  // latter in fragment format, is fully bank conflict free, we need to use a
  // rather fancy XOR-based layout. The key here is that neither reads nor
  // writes of the 16-byte `int4` blocks of 8 consecutive threads involve the
  // same shared memory banks. Further, it seems (based on NSight-Compute) that
  // each warp must also write a consecutive memory segment?
  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main
  // loop unrolls, all shared memory accesses are static, we simply precompute
  // both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
#pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
#pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
#pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at
  // runtime; we break dependencies between subsequent accesses with a tile by
  // maintining multiple pointers (we have enough registers), a tiny
  // optimization.
  const int4 *B_ptr[b_sh_wr_iters];
#pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  int4 *sh_a = sh;
  int4 *sh_b = sh_a + (stages * a_sh_stage);
  int4 *sh_g_idx = sh_b + (stages * b_sh_stage);
  int4 *sh_s = sh_g_idx + (stages * g_idx_stage);

  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2][b_thread_vecs];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];        // No act-order
  FragS act_frag_s[2][4][4]; // For act-order

  // Zero accumulators.
  auto zero_accums = [&]() {
#pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float *>(frag_c)[i] = 0;
  };

  int sh_first_group_id = -1;
  int sh_num_groups = -1;
  constexpr int sh_max_num_groups = 32;

  auto fetch_scales_to_shared = [&](bool is_async, int first_group_id,
                                    int last_group_id) {
    sh_first_group_id = first_group_id;
    sh_num_groups = last_group_id - first_group_id + 1;

    if (sh_num_groups < sh_max_num_groups) {
      sh_num_groups = sh_max_num_groups;
    }

    if (sh_first_group_id + sh_num_groups > num_groups) {
      sh_num_groups = num_groups - sh_first_group_id;
    }

    int row_offset = first_group_id * s_gl_stride;

    if (is_async) {
      for (int i = 0; i < sh_num_groups; i++) {
        if (threadIdx.x < s_sh_stride) {
          cp_async4_pred(&sh_s[(i * s_sh_stride) + threadIdx.x],
                         &scales_ptr[row_offset + (i * s_gl_stride) +
                                     slice_n_offset + threadIdx.x]);
        }
      }
    } else {
      for (int i = 0; i < sh_num_groups; i++) {
        if (threadIdx.x < s_sh_stride) {
          sh_s[(i * s_sh_stride) + threadIdx.x] =
              scales_ptr[row_offset + (i * s_gl_stride) + slice_n_offset +
                         threadIdx.x];
        }
      }
    }
  };
  // Asynchronously fetch the next A, B and s tile from global to the next
  // shared memory pipeline location.
  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4 *sh_a_stage = sh_a + a_sh_stage * pipe;
#pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4 *sh_b_stage = sh_b + b_sh_stage * pipe;
#pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
#pragma unroll
        for (int j = 0; j < b_thread_vecs; j++) {
          cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr + j], B_ptr[i] + j);
        }

        B_ptr[i] += b_gl_rd_delta_o;
      }

      if constexpr (has_act_order) {
        // Fetch g_idx thread-block portion
        int full_pipe = a_off;
        int cur_k = slice_k_start_shared_fetch + tb_k * full_pipe;
        if (cur_k < prob_k && cur_k < slice_k_finish) {
          int4 *sh_g_idx_stage = sh_g_idx + g_idx_stage * pipe;

          int4 const *cur_g_idx_stage_ptr =
              reinterpret_cast<int4 const *>(&g_idx[cur_k]);

          if (threadIdx.x < g_idx_stage) {
            cp_async4_pred(&sh_g_idx_stage[threadIdx.x],
                           &cur_g_idx_stage_ptr[threadIdx.x]);
          }
        }
      } else {
        if constexpr (group_blocks != -1) {
          int4 *sh_s_stage = sh_s + s_sh_stage * pipe;

          if constexpr (group_blocks >= thread_k_blocks) {
            // Only fetch scales if this tile starts a new group
            if (pipe % (group_blocks / thread_k_blocks) == 0) {
              if (s_sh_wr_pred) {
                cp_async4(&sh_s_stage[s_sh_wr], &scales_ptr[s_gl_rd]);
              }
              s_gl_rd += s_gl_rd_delta;
            }
          } else {
            for (int i = 0; i < s_tb_groups; i++) {
              if (s_sh_wr_pred) {
                cp_async4(&sh_s_stage[i * s_sh_stride + s_sh_wr],
                          &scales_ptr[s_gl_rd]);
              }
              s_gl_rd += s_gl_rd_delta;
            }
          }
        }
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that
    // waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe
  // into the current register buffer.
  auto fetch_to_registers = [&](int k, int pipe) {
    int4 *sh_a_stage = sh_a + a_sh_stage * pipe;
#pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4 *sh_b_stage = sh_b + b_sh_stage * pipe;

#pragma unroll
    for (int i = 0; i < b_thread_vecs; i++) {
      frag_b_quant[k % 2][i] = *reinterpret_cast<I4 *>(
          &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd + i]);
    }
  };

  bool is_same_group[stages];
  int same_group_id[stages];

  auto init_same_group = [&](int pipe) {
    if constexpr (!has_act_order) {
      is_same_group[pipe] = false;
      same_group_id[pipe] = 0;
      return;
    }

    int4 *sh_g_idx_stage = sh_g_idx + g_idx_stage * pipe;
    int *sh_g_idx_int_ptr = reinterpret_cast<int *>(sh_g_idx_stage);

    int group_id_1 = sh_g_idx_int_ptr[0];
    int group_id_2 = sh_g_idx_int_ptr[tb_k - 1];

    is_same_group[pipe] = group_id_1 == group_id_2;
    same_group_id[pipe] = group_id_1;
  };

  auto fetch_scales_to_registers = [&](int k, int full_pipe) {
    int pipe = full_pipe % stages;

    if constexpr (!has_act_order) {
      // No act-order case
      if constexpr (group_blocks != -1) {
        if constexpr (group_blocks >= thread_k_blocks) {
          int4 *sh_s_stage =
              sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) *
                                   (pipe / (group_blocks / thread_k_blocks)));
          reinterpret_cast<int4 *>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
        } else {
          int warp_id = threadIdx.x / 32;
          int n_warps = thread_n_blocks / 4;

          int warp_row = warp_id / n_warps;

          int cur_k = warp_row * 16;
          cur_k += k_iter_size * (k % b_sh_wr_iters);

          int k_blocks = cur_k / 16;
          int cur_group_id = k_blocks / group_blocks;

          int4 *sh_s_stage = sh_s + s_sh_stage * pipe;

          reinterpret_cast<int4 *>(&frag_s[k % 2])[0] =
              sh_s_stage[s_sh_rd + cur_group_id * s_sh_stride];
        }
      }

      return;
    }

    // Act-order case

    // Determine K of the "current" thread-block
    int cur_k = slice_k_start + tb_k * full_pipe;
    if (cur_k >= prob_k || cur_k >= slice_k_finish) {
      return;
    }

    // Reset (to current thread-block) since we read g_idx portion from the
    // shared memory
    cur_k = 0;

    // Progress to current iteration
    cur_k += k_iter_size * (k % b_sh_wr_iters);

    // Determine "position" inside the thread-block (based on warp and
    // thread-id)
    int warp_id = threadIdx.x / 32;
    int n_warps =
        thread_n_blocks / 4; // Each warp processes 4 16-size tiles over N

    int warp_row = warp_id / n_warps;
    int warp_col = warp_id % n_warps;

    cur_k += warp_row * 16;

    int th_id = threadIdx.x % 32;
    cur_k += (th_id % 4) * 2; // Due to tensor-core layout for fp16 B matrix

    int s_col_shift =
        /*slice_n_offset +*/ (act_s_col_warp_stride * warp_col) +
        (th_id / 4) * act_s_col_stride;

    if (is_same_group[pipe]) {
      if (k % 2 == 0) {
        *(reinterpret_cast<int4 *>(&(act_frag_s[k % 2][0][0]))) =
            sh_s[(same_group_id[pipe] - sh_first_group_id) * s_sh_stride +
                 s_col_shift];
      } else {
        *(reinterpret_cast<int4 *>(&(act_frag_s[k % 2][0][0]))) =
            *(reinterpret_cast<int4 *>(&(act_frag_s[(k - 1) % 2][0][0])));
      }

      for (int i = 1; i < 4; i++) {
        *(reinterpret_cast<int4 *>(&(act_frag_s[k % 2][i][0]))) =
            *(reinterpret_cast<int4 *>(&(act_frag_s[k % 2][0][0])));
      }
      return;
    }

    int4 *sh_g_idx_stage = sh_g_idx + g_idx_stage * pipe;
    int *sh_g_idx_int_ptr = reinterpret_cast<int *>(sh_g_idx_stage);

    constexpr int k_frag_offsets[4] = {0, 1, 8,
                                       9}; // Tensor core offsets per thread

#pragma unroll
    for (int i = 0; i < 4; i++) {

      int actual_k = cur_k + k_frag_offsets[i];

      int group_id = sh_g_idx_int_ptr[actual_k];
      int rel_group_id = group_id - sh_first_group_id;

      *(reinterpret_cast<int4 *>(&(act_frag_s[k % 2][i][0]))) =
          sh_s[rel_group_id * s_sh_stride + s_col_shift];
    }
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&](int k) {
// We have the m dimension as the inner loop in order to encourage overlapping
// dequantization and matmul operations.
#pragma unroll
    for (int j = 0; j < 4; j++) {
      FragB frag_b0;
      FragB frag_b1;
      if constexpr (num_bits == 4) {
        int b_quant = frag_b_quant[k % 2][0][j];
        int b_quant_shift = b_quant >> 8;

        frag_b0 = dequant_4bit(b_quant);
        frag_b1 = dequant_4bit(b_quant_shift);

      } else {
        int *frag_b_quant_ptr = reinterpret_cast<int *>(frag_b_quant[k % 2]);
        int b_quant_0 = frag_b_quant_ptr[j * 2 + 0];
        int b_quant_1 = frag_b_quant_ptr[j * 2 + 1];

        frag_b0 = dequant_8bit(b_quant_0);
        frag_b1 = dequant_8bit(b_quant_1);
      }

      // Apply scale to frag_b0
      if constexpr (has_act_order) {
        scale4(frag_b0, act_frag_s[k % 2][0][j], act_frag_s[k % 2][1][j],
               act_frag_s[k % 2][2][j], act_frag_s[k % 2][3][j], 0);
      } else {
        if constexpr (group_blocks != -1) {
          scale(frag_b0, frag_s[k % 2][j], 0);
        }
      }

      // Apply scale to frag_b1
      if constexpr (has_act_order) {
        scale4(frag_b1, act_frag_s[k % 2][0][j], act_frag_s[k % 2][1][j],
               act_frag_s[k % 2][2][j], act_frag_s[k % 2][3][j], 1);

      } else {
        if constexpr (group_blocks != -1) {
          scale(frag_b1, frag_s[k % 2][j], 1);
        }
      }

#pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the
  // number of warps while keeping the n dimension of a tile reasonable, we have
  // multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride_threads / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride_threads;
      constexpr int red_sh_stride = b_sh_stride_threads * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride_threads;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride_threads) +
                      (threadIdx.x % b_sh_stride_threads);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any
      // unnecessary read or write iterations, e.g., for two warps we write only
      // once by warp 1 and read only once by warp 0.

#pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
#pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
#pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float *c_rd = reinterpret_cast<float *>(
                    &sh[red_sh_delta * j + red_sh_rd]);
                float *c_wr = reinterpret_cast<float *>(&sh[red_sh_wr]);
#pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC *>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] =
                  reinterpret_cast<int4 *>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
#pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float *c_rd =
                reinterpret_cast<float *>(&sh[red_sh_delta * i + red_sh_rd]);
#pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC *>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we
  // finally have to globally reduce over the results. As the striped partitioning
  // minimizes the number of such reductions and our outputs are usually rather
  // small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&](bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to
    // maximize L2 cache utilization in this step. To do this, we write out
    // results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) +
                    4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
// Interestingly, doing direct global accesses here really seems to mess up the
// compiler and lead to slowdowns, hence we also use async-copies even though
// these fetches are not actually asynchronous.
#pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(&sh[c_sh_wr + c_sh_wr_delta * i],
                         &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                            c_gl_wr_delta_i * (i % 2)],
                         i < (thread_m_blocks - 1) * 4 ||
                             8 * (i / 2) + row < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

#pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
#pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float *>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] +=
                  __half2float(reinterpret_cast<__half *>(&c_red)[j]);
            }
          }
          if (!last) {
            int4 c;
#pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<__half *>(&c)[j] =
                  __float2half(reinterpret_cast<float *>(
                      &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]);
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually
  // reshuffle matrix fragments in this step, the reduction above is performed
  // in fragment layout.
  auto write_result = [&]() {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta =
        c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr =
        (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final
    // global write patterns
    auto write = [&](int idx, float c0, float c1, FragS &s) {
      half2 res = __halves2half2(__float2half(c0), __float2half(c1));

      // For per-column quantization we finally apply the scale here (only for
      // 4-bit)
      if constexpr (!has_act_order && group_blocks == -1 && num_bits == 4) {
        res = __hmul2(res, s[0]);
      }

      ((half2 *)sh)[idx] = res;
    };

    if (threadIdx.x / 32 < thread_n_blocks / 4) {
#pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
#pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0],
                frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2],
                frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0],
                frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2],
                frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

#pragma unroll
    for (int i = 0;
         i < div_ceil(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&]() {

#pragma unroll
    for (int i = 0; i < stages - 1; i++) {
      if (has_act_order && i == 0) {
        int last_g_idx = slice_k_start + stages * tb_k * 2;
        if (last_g_idx >= prob_k) {
          last_g_idx = prob_k - 1;
        }
        fetch_scales_to_shared(true, g_idx[slice_k_start], g_idx[last_g_idx]);
      }
      fetch_to_shared(i, i, i < slice_iters);
    }

    zero_accums();
    wait_for_stage();
    init_same_group(0);
    fetch_to_registers(0, 0);
    fetch_scales_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
    slice_k_start_shared_fetch += tb_k * (stages - 1);
  };
  if (slice_iters) {
    start_pipes();
  }

  // Main loop.
  while (slice_iters) {
    // We unroll over both the global fetch and the register load pipeline to
    // ensure all shared memory accesses are static. Note that both pipelines
    // have even length meaning that the next iteration will always start at
    // index 0.

#pragma unroll
    for (int pipe = 0; pipe < stages;) {
#pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        fetch_scales_to_registers(k + 1, pipe);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe,
                          slice_iters >= stages);
          pipe++;
          wait_for_stage();
          init_same_group(pipe % stages);
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0) {
        break;
      }
    }

    a_gl_rd += a_gl_rd_delta_o * stages;
    slice_k_start += tb_k * stages;
    slice_k_start_shared_fetch += tb_k * stages;

    if constexpr (has_act_order) {
      int first_group_id = g_idx[slice_k_start];
      int last_g_idx = slice_k_start + stages * tb_k * 2;
      if (last_g_idx >= prob_k) {
        last_g_idx = prob_k - 1;
      }
      int last_group_id = g_idx[last_g_idx];
      if (last_group_id >= sh_first_group_id + sh_num_groups) {
        fetch_scales_to_shared(false, first_group_id, last_group_id);
        __syncthreads();
      }
    }

    // Process results and, if necessary, proceed to the next column slice.
    // While this pattern may not be the most readable, other ways of writing
    // the loop seemed to noticeably worse performance after compilation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before
      // write-out
      if constexpr (!has_act_order && group_blocks == -1) {
        if constexpr (num_bits == 8) {
          if (s_sh_wr_pred) {
            cp_async4(&sh_s[s_sh_wr], &scales_ptr[s_gl_rd]);
          }
          cp_async_fence();
        } else {
          if (last) {
            if (s_sh_wr_pred) {
              cp_async4(&sh_s[s_sh_wr], &scales_ptr[s_gl_rd]);
            }
            cp_async_fence();
          }
        }
      }

      thread_block_reduce();
      if constexpr (!has_act_order && group_blocks == -1) {
        if constexpr (num_bits == 8) {
          cp_async_wait<0>();
          __syncthreads();
          if (threadIdx.x / 32 < thread_n_blocks / 4) {
            reinterpret_cast<int4 *>(&frag_s)[0] = sh_s[s_sh_rd + 0];
            reinterpret_cast<int4 *>(&frag_s)[1] = sh_s[s_sh_rd + 4];
          }

        } else {
          if (last) {
            cp_async_wait<0>();
            __syncthreads();
            if (threadIdx.x / 32 < thread_n_blocks / 4) {
              reinterpret_cast<int4 *>(&frag_s)[0] = sh_s[s_sh_rd + 0];
              reinterpret_cast<int4 *>(&frag_s)[1] = sh_s[s_sh_rd + 4];
            }
          }
        }
      }

      // For 8-bit channelwise, we apply the scale before the global reduction
      // that converts the fp32 results to fp16 (so that we avoid possible
      // overflow in fp16)
      if constexpr (!has_act_order && group_blocks == -1 && num_bits == 8) {
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
#pragma unroll
          for (int i = 0; i < thread_m_blocks; i++) {
#pragma unroll
            for (int j = 0; j < 4; j++) {
              scale_float(reinterpret_cast<float *>(&frag_c[i][j][0][0]),
                          frag_s[j / 2][2 * (j % 2) + 0]);
              scale_float(reinterpret_cast<float *>(&frag_c[i][j][0][2]),
                          frag_s[j / 2][2 * (j % 2) + 0]);

              scale_float(reinterpret_cast<float *>(&frag_c[i][j][1][0]),
                          frag_s[j / 2][2 * (j % 2) + 1]);
              scale_float(reinterpret_cast<float *>(&frag_c[i][j][1][2]),
                          frag_s[j / 2][2 * (j % 2) + 1]);
            }
          }
        }
      }

      if (slice_count > 1) { // only globally reduce if there is more than one
                             // block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last) // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
#pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
#pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++)
            B_ptr[i] -= b_gl_stride;
        }

        // Update slice k/n for scales loading
        if constexpr (has_act_order) {
          slice_k_start = tb_k * slice_row;
          slice_k_finish = slice_k_start + tb_k * slice_iters;
          slice_k_start_shared_fetch = slice_k_start;
          slice_n_offset = act_s_col_tb_stride * slice_col;

        } else {
          s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
        }

        start_pipes();
      }
    }
  }
}

#define __CALL_IF(NUM_BITS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, \
                  HAS_ACT_ORDER, GROUP_BLOCKS, NUM_THREADS)                    \
  else if (num_bits == NUM_BITS && thread_m_blocks == THREAD_M_BLOCKS &&       \
           thread_n_blocks == THREAD_N_BLOCKS &&                               \
           thread_k_blocks == THREAD_K_BLOCKS &&                               \
           has_act_order == HAS_ACT_ORDER && group_blocks == GROUP_BLOCKS &&   \
           num_threads == NUM_THREADS) {                                       \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                      \
        Marlin<NUM_BITS), NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS,        \
               THREAD_K_BLOCKS, pipe_stages, HAS_ACT_ORDER, GROUP_BLOCKS>,     \
        hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);          \
    Marlin<NUM_BITS, NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS,            \
           THREAD_K_BLOCKS, pipe_stages, HAS_ACT_ORDER, GROUP_BLOCKS>          \
        <<<blocks, NUM_THREADS, max_shared_mem, stream>>>(                     \
            A_ptr, B_ptr, C_ptr, s_ptr, g_idx_ptr, num_groups, prob_m, prob_n, \
            prob_k, locks);                                                    \
  }

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

typedef struct {
  int max_m_blocks;
  thread_config_t tb_cfg;
} exec_config_t;

thread_config_t thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256}, // Default (max cache usage)
    {64, 128, 128}, // Reduce N, reduce warps
    {128, 64, 128}, // Reduce N more, but increase K

};

int get_scales_cache_size(thread_config_t const &th_config, int prob_m,
                          int prob_n, int prob_k, int num_bits, int group_size,
                          bool has_act_order, bool is_k_full) {
  bool cache_scales_chunk = has_act_order && !is_k_full;

  int tb_n = th_config.thread_n;
  int tb_k = th_config.thread_k;

  // Get max scale groups per thread-block
  int tb_groups;
  if (group_size == -1) {
    tb_groups = 1;
  } else if (group_size == 0) {
    tb_groups = div_ceil(tb_k, 32); // Worst case is 32 group size
  } else {
    tb_groups = div_ceil(tb_k, group_size);
  }

  if (cache_scales_chunk) {
    int load_groups =
        tb_groups * pipe_stages * 2;    // Chunk size is 2x pipeline over dim K
    load_groups = max(load_groups, 32); // We load at least 32 scale groups
    return load_groups * tb_n * 2;

  } else {
    int tb_scales = tb_groups * tb_n * 2;

    return tb_scales * pipe_stages;
  }
}

bool is_valid_cache_size(thread_config_t const &th_config, int max_m_blocks,
                         int prob_m, int prob_n, int prob_k, int num_bits,
                         int scales_cache_size, int max_shared_mem) {
  int pack_factor = 32 / num_bits;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;

  int b_size = (tb_k * tb_n / pack_factor) * 4;

  // Get A size
  int m_blocks = div_ceil(prob_m, 16);
  int tb_max_m = 16;

  while (true) {
    if (m_blocks >= max_m_blocks) {
      tb_max_m *= max_m_blocks;
      break;
    }

    max_m_blocks--;
    if (max_m_blocks == 0) {
      TORCH_CHECK(false, "Unexpected m_blocks = ", m_blocks);
    }
  }

  int a_size = (tb_max_m * tb_k) * 2;

  float pipe_size = (a_size + b_size) * pipe_stages;

  TORCH_CHECK(max_shared_mem / 2 > scales_cache_size); // Sanity

  return pipe_size < 0.95f * (max_shared_mem - scales_cache_size);
}

bool is_valid_config(thread_config_t const &th_config, int max_m_blocks,
                     int prob_m, int prob_n, int prob_k, int num_bits,
                     int group_size, bool has_act_order, bool is_k_full,
                     int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  //  Determine cache for scales
  int scales_cache_size =
      get_scales_cache_size(th_config, prob_m, prob_n, prob_k, num_bits,
                            group_size, has_act_order, is_k_full);

  // Check that pipeline fits into cache
  if (!is_valid_cache_size(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                           num_bits, scales_cache_size, max_shared_mem)) {
    return false;
  }

  return true;
}

exec_config_t determine_thread_config(int prob_m, int prob_n, int prob_k,
                                      int num_bits, int group_size,
                                      bool has_act_order, bool is_k_full,
                                      int max_shared_mem) {
  int max_m_blocks = 4;
  while (max_m_blocks > 0) {
    for (auto th_config : thread_configs) {
      if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                          num_bits, group_size, has_act_order, is_k_full,
                          max_shared_mem)) {
        return exec_config_t{max_m_blocks, th_config};
      }
    }

    printf("WARNING: Marlin kernel is reducing max_m_blocks due to small SM "
           "GPU cache. This may "
           "hurt performance. Consider upgrading your GPU.\n");

    max_m_blocks--; // Process less M blocks per invocation to reduce cache
                    // usage
  }

  return exec_config_t{0, {-1, -1, -1}};
}

#define CALL_IF(NUM_BITS, N_BLOCKS, K_BLOCKS, NUM_THREADS)                     \
  __CALL_IF(NUM_BITS, 1, N_BLOCKS, K_BLOCKS, true, 0, NUM_THREADS)             \
  __CALL_IF(NUM_BITS, 2, N_BLOCKS, K_BLOCKS, true, 0, NUM_THREADS)             \
  __CALL_IF(NUM_BITS, 3, N_BLOCKS, K_BLOCKS, true, 0, NUM_THREADS)             \
  __CALL_IF(NUM_BITS, 4, N_BLOCKS, K_BLOCKS, true, 0, NUM_THREADS)             \
                                                                               \
  __CALL_IF(NUM_BITS, 1, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS)           \
  __CALL_IF(NUM_BITS, 1, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 1, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 1, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS)            \
                                                                               \
  __CALL_IF(NUM_BITS, 2, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS)           \
  __CALL_IF(NUM_BITS, 2, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 2, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 2, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS)            \
                                                                               \
  __CALL_IF(NUM_BITS, 3, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS)           \
  __CALL_IF(NUM_BITS, 3, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 3, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 3, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS)            \
                                                                               \
  __CALL_IF(NUM_BITS, 4, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS)           \
  __CALL_IF(NUM_BITS, 4, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 4, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS)            \
  __CALL_IF(NUM_BITS, 4, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS)

void marlin_mm_f16i4(const void *A, const void *B, void *C, void *s,
                     void *g_idx, void *perm, void *a_tmp, int prob_m,
                     int prob_n, int prob_k, void *workspace, int num_bits,
                     bool has_act_order, bool is_k_full, int num_groups,
                     int group_size, int dev, hipStream_t stream, int thread_k,
                     int thread_n, int sms, int max_par) {
  TORCH_CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  TORCH_CHECK(prob_m > 0 && prob_n > 0 && prob_k > 0, "Invalid MNK = [", prob_m,
              ", ", prob_n, ", ", prob_k, "]");

  int tot_m = prob_m;
  int tot_m_blocks = div_ceil(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1) {
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  }

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  // Set thread config
  exec_config_t exec_cfg;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    exec_cfg =
        exec_config_t{4, thread_config_t{thread_k, thread_n, default_threads}};
  } else {
    // Auto config
    exec_cfg =
        determine_thread_config(prob_m, prob_n, prob_k, num_bits, group_size,
                                has_act_order, is_k_full, max_shared_mem);
  }

  TORCH_CHECK(exec_cfg.max_m_blocks > 0 &&
                  is_valid_config(exec_cfg.tb_cfg, exec_cfg.max_m_blocks,
                                  prob_m, prob_n, prob_k, num_bits, group_size,
                                  has_act_order, is_k_full, max_shared_mem),
              "Invalid thread config: max_m_blocks = ", exec_cfg.max_m_blocks,
              ", thread_k = ", exec_cfg.tb_cfg.thread_k,
              ", thread_n = ", exec_cfg.tb_cfg.thread_n,
              ", num_threads = ", exec_cfg.tb_cfg.num_threads, " for MKN = [",
              prob_m, ", ", prob_k, ", ", prob_n, "] and num_bits = ", num_bits,
              ", group_size = ", group_size,
              ", has_act_order = ", has_act_order, ", is_k_full = ", is_k_full,
              ", max_shared_mem = ", max_shared_mem);

  int num_threads = exec_cfg.tb_cfg.num_threads;
  thread_k = exec_cfg.tb_cfg.thread_k;
  thread_n = exec_cfg.tb_cfg.thread_n;

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;

  int blocks = sms;

  TORCH_CHECK(prob_n % thread_n == 0, "prob_n = ", prob_n,
              " is not divisible by thread_n = ", thread_n);
  TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
              " is not divisible by thread_k = ", thread_k);

  int group_blocks = 0;
  if (has_act_order) {
    if (is_k_full) {
      TORCH_CHECK(group_size != -1);
      group_blocks = group_size / 16;
      TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                  " is not divisible by group_blocks = ", group_blocks);
    } else {
      TORCH_CHECK(group_size == 0);
      group_blocks = 0;
    }

  } else {
    if (group_size == -1) {
      group_blocks = -1;
    } else {
      group_blocks = group_size / 16;
      TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                  " is not divisible by group_blocks = ", group_blocks);
    }
  }

  const int4 *A_ptr = (const int4 *)A;
  const int4 *B_ptr = (const int4 *)B;
  int4 *C_ptr = (int4 *)C;
  const int4 *s_ptr = (const int4 *)s;
  const int *g_idx_ptr = (const int *)g_idx;
  const int *perm_ptr = (const int *)perm;
  int4 *a_tmp_ptr = (int4 *)a_tmp;

  int *locks = (int *)workspace;

  if (has_act_order) {
    // Permute A columns
    int block_rows = div_ceil(prob_m, blocks);
    permute_cols_kernel<<<blocks, default_threads, 0, stream>>>(
        A_ptr, perm_ptr, a_tmp_ptr, prob_m, prob_k, block_rows);
    A_ptr = a_tmp_ptr;
  }

  // If we have a full K, then we can run the non-act-order version of Marlin
  // (since the weight rows are reordered by increasing group ids, and by having
  // a full K, we have full original groups)
  if (is_k_full) {
    has_act_order = false;
  }

  // Main loop
  for (int i = 0; i < tot_m_blocks; i += exec_cfg.max_m_blocks) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > exec_cfg.max_m_blocks) {
      // Note that parallel > 1 currently only works for inputs without any
      // padding
      par = (16 * thread_m_blocks - pad) / (16 * exec_cfg.max_m_blocks);
      if (par > max_par)
        par = max_par;
      prob_m = (16 * exec_cfg.max_m_blocks) * par;
      i += exec_cfg.max_m_blocks * (par - 1);
      thread_m_blocks = exec_cfg.max_m_blocks;
    }

    // Define kernel configurations
    if (false) {
    }
    CALL_IF(4, 32, 2, 256)
    CALL_IF(4, 16, 4, 256)
    CALL_IF(4, 8, 4, 128)
    CALL_IF(4, 4, 8, 128)
    CALL_IF(8, 32, 2, 256)
    CALL_IF(8, 16, 4, 256)
    CALL_IF(8, 8, 4, 128)
    CALL_IF(8, 4, 8, 128)
    else {
      TORCH_CHECK(false, "Unsupported shapes: MNK = [" + str(prob_m) + ", " +
                             str(prob_n) + ", " + str(prob_k) + "]" +
                             ", has_act_order = " + str(has_act_order) +
                             ", num_groups = " + str(num_groups) +
                             ", group_size = " + str(group_size) +
                             ", thread_m_blocks = " + str(thread_m_blocks) +
                             ", thread_n_blocks = " + str(thread_n_blocks) +
                             ", thread_k_blocks = " + str(thread_k_blocks));
    }

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }
}

} // namespace gptq_marlin

torch::Tensor gptq_marlin_gemm(torch::Tensor &a, torch::Tensor &b_q_weight,
                               torch::Tensor &b_scales, torch::Tensor &g_idx,
                               torch::Tensor &perm, torch::Tensor &workspace,
                               int64_t num_bits, int64_t size_m, int64_t size_n,
                               int64_t size_k, bool is_k_full) {
  // Verify num_bits
  TORCH_CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  int pack_factor = 32 / num_bits;

  // Verify A
  TORCH_CHECK(a.size(0) == size_m, "Shape mismatch: a.size(0) = ", a.size(0),
              ", size_m = ", size_m);
  TORCH_CHECK(a.size(1) == size_k, "Shape mismatch: a.size(1) = ", a.size(1),
              ", size_k = ", size_k);

  // Verify B
  TORCH_CHECK(size_k % gptq_marlin::tile_size == 0, "size_k = ", size_k,
              " is not divisible by tile_size = ", gptq_marlin::tile_size);
  TORCH_CHECK((size_k / gptq_marlin::tile_size) == b_q_weight.size(0),
              "Shape mismatch: b_q_weight.size(0) = ", b_q_weight.size(0),
              ", size_k = ", size_k, ", tile_size = ", gptq_marlin::tile_size);
  TORCH_CHECK(b_q_weight.size(1) % gptq_marlin::tile_size == 0,
              "b_q_weight.size(1) = ", b_q_weight.size(1),
              " is not divisible by tile_size = ", gptq_marlin::tile_size);
  int actual_size_n =
      (b_q_weight.size(1) / gptq_marlin::tile_size) * pack_factor;
  TORCH_CHECK(size_n == actual_size_n, "size_n = ", size_n,
              ", actual_size_n = ", actual_size_n);

  // Verify device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  TORCH_CHECK(g_idx.device().is_cuda(), "g_idx is not on GPU");
  TORCH_CHECK(g_idx.is_contiguous(), "g_idx is not contiguous");

  TORCH_CHECK(perm.device().is_cuda(), "perm is not on GPU");
  TORCH_CHECK(perm.is_contiguous(), "perm is not contiguous");

  // Alloc buffers
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c = torch::empty({size_m, size_n}, options);
  torch::Tensor a_tmp = torch::empty({size_m, size_k}, options);

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel (can usually be left as auto -1)
  int sms = -1;

  // Verify g_idx and perm
  TORCH_CHECK((g_idx.size(0) == 0 && perm.size(0) == 0) ||
                  (g_idx.size(0) == size_k && perm.size(0) == size_k),
              "Unexpected g_idx.size(0) = ", g_idx.size(0),
              " and perm.size(0) = ", perm.size(0),
              ", where size_k = ", size_k);

  // Detect groupsize and act_order
  int num_groups = -1;
  int group_size = -1;
  bool has_act_order = g_idx.size(0) != 0;

  int b_rank = b_scales.sizes().size();
  TORCH_CHECK(b_rank == 2, "b_scales rank = ", b_rank, " is not 2");
  TORCH_CHECK(b_scales.size(1) == size_n, "b_scales dim 1 = ", b_scales.size(1),
              " is not size_n = ", size_n);
  num_groups = b_scales.size(0);

  if (has_act_order) {
    if (is_k_full) {
      TORCH_CHECK(num_groups > 1, "For act_order, num_groups must be > 1");
      TORCH_CHECK(size_k % num_groups == 0, "size_k = ", size_k,
                  ", is not divisible by num_groups = ", num_groups);
      group_size = size_k / num_groups;
    } else {
      group_size = 0;
    }

  } else {
    if (num_groups > 1) {
      TORCH_CHECK(
          size_k % num_groups == 0, "size_k = ", size_k,
          ", is not divisible by b_scales.size(0) = ", b_scales.size(0));
      group_size = size_k / num_groups;
    } else {
      group_size = -1;
    }
  }

  // Verify workspace size
  TORCH_CHECK(
      size_n % gptq_marlin::min_thread_n == 0, "size_n = ", size_n,
      ", is not divisible by min_thread_n = ", gptq_marlin::min_thread_n);
  int min_workspace_size =
      (size_n / gptq_marlin::min_thread_n) * gptq_marlin::max_par;
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = ", workspace.numel(),
              " is below min_workspace_size = ", min_workspace_size);

  int dev = a.get_device();
  gptq_marlin::marlin_mm_f16i4(
      a.data_ptr(), b_q_weight.data_ptr(), c.data_ptr(), b_scales.data_ptr(),
      g_idx.data_ptr(), perm.data_ptr(), a_tmp.data_ptr(), size_m, size_n,
      size_k, workspace.data_ptr(), num_bits, has_act_order, is_k_full,
      num_groups, group_size, dev, at::cuda::getCurrentCUDAStream(dev),
      thread_k, thread_n, sms, gptq_marlin::max_par);

  return c;
}

#endif
