#include "hip/hip_runtime.h"
#include "marlin.cuh"

#include "core/registration.h"

namespace marlin {

template <int const num_threads, int const num_bits>
__global__ void awq_marlin_repack_kernel(
    uint32_t const* __restrict__ b_q_weight_ptr, uint32_t* __restrict__ out_ptr,
    int size_k, int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  int start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int tile_n_ints = tile_n_size / pack_factor;

  constexpr int stage_n_threads = tile_n_ints / 4;
  constexpr int stage_k_threads = tile_k_size;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;
    int first_n_packed = first_n / pack_factor;

    int4* sh_ptr = sh + stage_size * pipe;

    if (threadIdx.x < stage_size) {
      int k_id = threadIdx.x / stage_n_threads;
      int n_id = threadIdx.x % stage_n_threads;

      int first_k = k_tile_id * tile_k_size;

      cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                reinterpret_cast<int4 const*>(
                    &(b_q_weight_ptr[(first_k + k_id) * (size_n / pack_factor) +
                                     first_n_packed + (n_id * 4)])));
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    int warp_id = threadIdx.x / 32;
    int th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;
    int cur_n_packed = cur_n / pack_factor;
    int cur_n_pos = cur_n % pack_factor;

    constexpr int sh_stride = tile_n_ints;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4* sh_stage_ptr = sh + stage_size * pipe;
    uint32_t* sh_stage_int_ptr = reinterpret_cast<uint32_t*>(sh_stage_ptr);

    // Undo interleaving
    int cur_n_pos_unpacked;
    if constexpr (num_bits == 4) {
      constexpr int undo_pack[8] = {0, 4, 1, 5, 2, 6, 3, 7};
      cur_n_pos_unpacked = undo_pack[cur_n_pos];
    } else {
      constexpr int undo_pack[4] = {0, 2, 1, 3};
      cur_n_pos_unpacked = undo_pack[cur_n_pos];
    }

    uint32_t vals[8];
#pragma unroll
    for (int i = 0; i < 4; i++) {
      int cur_elem = tc_row + tc_offsets[i];

      int packed_src_0 = sh_stage_int_ptr[cur_n_packed + sh_stride * cur_elem];
      int packed_src_1 = sh_stage_int_ptr[cur_n_packed + (8 / pack_factor) +
                                          sh_stride * cur_elem];

      vals[i] = (packed_src_0 >> (cur_n_pos_unpacked * num_bits)) & mask;
      vals[4 + i] = (packed_src_1 >> (cur_n_pos_unpacked * num_bits)) & mask;
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

}  // namespace marlin

#define CALL_IF(NUM_BITS)                                                   \
  else if (num_bits == NUM_BITS) {                                          \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                   \
        marlin::awq_marlin_repack_kernel<marlin::repack_threads), NUM_BITS>, \
        hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);       \
    marlin::awq_marlin_repack_kernel<marlin::repack_threads, NUM_BITS>      \
        <<<blocks, marlin::repack_threads, max_shared_mem, stream>>>(       \
            b_q_weight_ptr, out_ptr, size_k, size_n);                       \
  }

torch::Tensor awq_marlin_repack(torch::Tensor& b_q_weight, int64_t size_k,
                                int64_t size_n, int64_t num_bits) {
  // Verify compatibility with marlin tile of 16x64
  TORCH_CHECK(size_k % marlin::tile_k_size == 0, "size_k = ", size_k,
              " is not divisible by tile_k_size = ", marlin::tile_k_size);
  TORCH_CHECK(size_n % marlin::tile_n_size == 0, "size_n = ", size_n,
              " is not divisible by tile_n_size = ", marlin::tile_n_size);

  TORCH_CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  int const pack_factor = 32 / num_bits;

  // Verify B
  TORCH_CHECK(b_q_weight.size(0) == size_k,
              "b_q_weight.size(0) = ", b_q_weight.size(0),
              " is not size_k = ", size_k);
  TORCH_CHECK((size_n / pack_factor) == b_q_weight.size(1),
              "Shape mismatch: b_q_weight.size(1) = ", b_q_weight.size(1),
              ", size_n = ", size_n, ", pack_factor = ", pack_factor);

  // Verify device and strides
  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");
  TORCH_CHECK(b_q_weight.dtype() == at::kInt, "b_q_weight type is not kInt");

  // Alloc buffers
  const at::cuda::OptionalCUDAGuard device_guard(device_of(b_q_weight));
  auto options = torch::TensorOptions()
                     .dtype(b_q_weight.dtype())
                     .device(b_q_weight.device());
  torch::Tensor out = torch::empty(
      {size_k / marlin::tile_size, size_n * marlin::tile_size / pack_factor},
      options);

  // Get ptrs
  uint32_t const* b_q_weight_ptr =
      reinterpret_cast<uint32_t const*>(b_q_weight.data_ptr());
  uint32_t* out_ptr = reinterpret_cast<uint32_t*>(out.data_ptr());

  // Get dev info
  int dev = b_q_weight.get_device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(dev);
  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  if (false) {
  }
  CALL_IF(4)
  CALL_IF(8)
  else {
    TORCH_CHECK(false, "Unsupported repack config: num_bits = ", num_bits);
  }

  return out;
}

torch::Tensor awq_marlin_repack_meta(torch::Tensor& b_q_weight,
                                     c10::SymInt size_k, c10::SymInt size_n,
                                     int64_t num_bits) {
  int const pack_factor = 32 / num_bits;
  auto options = torch::TensorOptions()
                     .dtype(b_q_weight.dtype())
                     .device(b_q_weight.device());
  return torch::empty_symint(
      {size_k / marlin::tile_size, size_n * marlin::tile_size / pack_factor},
      options);
}

TORCH_LIBRARY_IMPL_EXPAND(TORCH_EXTENSION_NAME, CUDA, m) {
  m.impl("awq_marlin_repack", &awq_marlin_repack);
}

TORCH_LIBRARY_IMPL_EXPAND(TORCH_EXTENSION_NAME, Meta, m) {
  m.impl("awq_marlin_repack", &awq_marlin_repack_meta);
}