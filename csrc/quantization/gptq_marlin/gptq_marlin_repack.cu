#include "hip/hip_runtime.h"
#include "gptq_marlin.cuh"

namespace gptq_marlin {

static constexpr int repack_stages = 8;

static constexpr int repack_threads = 256;

static constexpr int tile_k_size = tile_size;
static constexpr int tile_n_size = tile_k_size * 4;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

template <int const num_threads, int const num_bits, bool const has_perm>
__global__ void marlin_repack_kernel(
    uint32_t const* __restrict__ b_q_weight_ptr,
    uint32_t const* __restrict__ perm_ptr, uint32_t* __restrict__ out_ptr,
    int size_k, int size_n) {}

}  // namespace gptq_marlin

torch::Tensor gptq_marlin_repack(torch::Tensor& b_q_weight, torch::Tensor& perm,
                                 int64_t size_k, int64_t size_n,
                                 int64_t num_bits) {
  TORCH_CHECK_NOT_IMPLEMENTED(
      false, "marlin_repack_from_gptq(..) requires CUDA_ARCH >= 8.0");
  return torch::empty({1, 1});
}

#else

template <int const num_threads, int const num_bits, bool const has_perm>
__global__ void marlin_repack_kernel(
    uint32_t const* __restrict__ b_q_weight_ptr,
    uint32_t const* __restrict__ perm_ptr, uint32_t* __restrict__ out_ptr,
    int size_k, int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  int start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int perm_size = tile_k_size / 4;

  int4* sh_perm_ptr = sh;
  int4* sh_pipe_ptr = sh_perm_ptr;
  if constexpr (has_perm) {
    sh_pipe_ptr += perm_size;
  }

  constexpr int tile_ints = tile_k_size / pack_factor;

  constexpr int stage_n_threads = tile_n_size / 4;
  constexpr int stage_k_threads = has_perm ? tile_k_size : tile_ints;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto load_perm_to_shared = [&](int k_tile_id) {
    int first_k_int4 = (k_tile_id * tile_k_size) / 4;

    int4 const* perm_int4_ptr = reinterpret_cast<int4 const*>(perm_ptr);

    if (threadIdx.x < perm_size) {
      sh_perm_ptr[threadIdx.x] = perm_int4_ptr[first_k_int4 + threadIdx.x];
    }
    __syncthreads();
  };

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;

    int4* sh_ptr = sh_pipe_ptr + stage_size * pipe;

    if constexpr (has_perm) {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        uint32_t const* sh_perm_int_ptr =
            reinterpret_cast<uint32_t const*>(sh_perm_ptr);

        int src_k = sh_perm_int_ptr[k_id];
        int src_k_packed = src_k / pack_factor;

        cp_async4(
            &sh_ptr[k_id * stage_n_threads + n_id],
            reinterpret_cast<int4 const*>(&(
                b_q_weight_ptr[src_k_packed * size_n + first_n + (n_id * 4)])));
      }

    } else {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        int first_k = k_tile_id * tile_k_size;
        int first_k_packed = first_k / pack_factor;

        cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                  reinterpret_cast<int4 const*>(
                      &(b_q_weight_ptr[(first_k_packed + k_id) * size_n +
                                       first_n + (n_id * 4)])));
      }
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    int warp_id = threadIdx.x / 32;
    int th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;

    constexpr int sh_stride = 64;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4* sh_stage_ptr = sh_pipe_ptr + stage_size * pipe;
    uint32_t* sh_stage_int_ptr = reinterpret_cast<uint32_t*>(sh_stage_ptr);

    uint32_t* sh_perm_int_ptr = reinterpret_cast<uint32_t*>(sh_perm_ptr);

    uint32_t vals[8];

    if constexpr (has_perm) {
      for (int i = 0; i < 4; i++) {
        int k_idx = tc_row + tc_offsets[i];

        uint32_t src_k = sh_perm_int_ptr[k_idx];
        uint32_t src_k_pos = src_k % pack_factor;

        uint32_t b1_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n];
        uint32_t b1_cur_val = (b1_val >> (src_k_pos * num_bits)) & mask;

        uint32_t b2_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n + 8];
        uint32_t b2_cur_val = (b2_val >> (src_k_pos * num_bits)) & mask;

        vals[i] = b1_cur_val;
        vals[4 + i] = b2_cur_val;
      }

    } else {
      uint32_t b1_vals[tile_ints];
      uint32_t b2_vals[tile_ints];

  #pragma unroll
      for (int i = 0; i < tile_ints; i++) {
        b1_vals[i] = sh_stage_int_ptr[cur_n + sh_stride * i];
        b2_vals[i] = sh_stage_int_ptr[cur_n + 8 + sh_stride * i];
      }

  #pragma unroll
      for (int i = 0; i < 4; i++) {
        int cur_elem = tc_row + tc_offsets[i];
        int cur_int = cur_elem / pack_factor;
        int cur_pos = cur_elem % pack_factor;

        vals[i] = (b1_vals[cur_int] >> (cur_pos * num_bits)) & mask;
        vals[4 + i] = (b2_vals[cur_int] >> (cur_pos * num_bits)) & mask;
      }
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
  #pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
  #pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
  #pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
  #pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    if constexpr (has_perm) {
      load_perm_to_shared(k_tile_id);
    }

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
  #pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

}  // namespace gptq_marlin

  #define CALL_IF(NUM_BITS, HAS_PERM)                                          \
    else if (num_bits == NUM_BITS && has_perm == HAS_PERM) {                   \
      hipFuncSetAttribute(reinterpret_cast<const void*>(                                                    \
          gptq_marlin::marlin_repack_kernel<gptq_marlin::repack_threads),       \
                                            NUM_BITS, HAS_PERM>,               \
          hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);        \
      gptq_marlin::marlin_repack_kernel<gptq_marlin::repack_threads, NUM_BITS, \
                                        HAS_PERM>                              \
          <<<blocks, gptq_marlin::repack_threads, max_shared_mem, stream>>>(   \
              b_q_weight_ptr, perm_ptr, out_ptr, size_k, size_n);              \
    }

torch::Tensor gptq_marlin_repack(torch::Tensor& b_q_weight, torch::Tensor& perm,
                                 int64_t size_k, int64_t size_n,
                                 int64_t num_bits) {
  // Verify compatibility with marlin tile of 16x64
  TORCH_CHECK(size_k % gptq_marlin::tile_k_size == 0, "size_k = ", size_k,
              " is not divisible by tile_k_size = ", gptq_marlin::tile_k_size);
  TORCH_CHECK(size_n % gptq_marlin::tile_n_size == 0, "size_n = ", size_n,
              " is not divisible by tile_n_size = ", gptq_marlin::tile_n_size);

  TORCH_CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  int const pack_factor = 32 / num_bits;

  // Verify B
  TORCH_CHECK((size_k / pack_factor) == b_q_weight.size(0),
              "Shape mismatch: b_q_weight.size(0) = ", b_q_weight.size(0),
              ", size_k = ", size_k, ", pack_factor = ", pack_factor);
  TORCH_CHECK(b_q_weight.size(1) == size_n,
              "b_q_weight.size(1) = ", b_q_weight.size(1),
              " is not size_n = ", size_n);

  // Verify device and strides
  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");
  TORCH_CHECK(b_q_weight.dtype() == at::kInt, "b_q_weight type is not kInt");

  TORCH_CHECK(perm.device().is_cuda(), "perm is not on GPU");
  TORCH_CHECK(perm.is_contiguous(), "perm is not contiguous");
  TORCH_CHECK(perm.dtype() == at::kInt, "perm type is not at::kInt");

  // Alloc buffers
  const at::cuda::OptionalCUDAGuard device_guard(device_of(b_q_weight));
  auto options = torch::TensorOptions()
                     .dtype(b_q_weight.dtype())
                     .device(b_q_weight.device());
  torch::Tensor out =
      torch::empty({size_k / gptq_marlin::tile_size,
                    size_n * gptq_marlin::tile_size / pack_factor},
                   options);

  // Detect if there is act_order
  bool has_perm = perm.size(0) != 0;

  // Get ptrs
  uint32_t const* b_q_weight_ptr =
      reinterpret_cast<uint32_t const*>(b_q_weight.data_ptr());
  uint32_t const* perm_ptr = reinterpret_cast<uint32_t const*>(perm.data_ptr());
  uint32_t* out_ptr = reinterpret_cast<uint32_t*>(out.data_ptr());

  // Get dev info
  int dev = b_q_weight.get_device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(dev);
  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  if (false) {
  }
  CALL_IF(4, false)
  CALL_IF(4, true)
  CALL_IF(8, false)
  CALL_IF(8, true)
  else {
    TORCH_CHECK(false, "Unsupported repack config: num_bits = ", num_bits,
                ", has_perm = ", has_perm);
  }

  return out;
}

#endif
