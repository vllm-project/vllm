#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// half-tensor
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/CUDATensorMethods.cuh>

#define BLOCKWIDTH 128
#define BLOCKHEIGHT4 16

namespace vllm {
namespace squeezellm {

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

// 4-bit matvec kernel (LUT-based)
__global__ void NUQ4MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           half2* __restrict__ mul,
    const  __half* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
) {

  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT4 * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];

  __shared__ __half deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 16;
  for (int val = 0; val < 16; val += 1) {
    int lut_index = column_offset + val;
    deq2[val][off] = lookup_table[lut_index];
  }

  __half res;
  half2 res2;
  half2 tmp2;

  int i;
  int k;

  unsigned int tmp1;
  unsigned int lut_index1, lut_index2;

  for (int b = 0; b < batch; ++b){
    i = width * row + col;
    res = __int2half_rd(0);
    k = 0;

    __syncthreads();
    if (threadIdx.x < blockwidth2)
      blockvec[threadIdx.x] = vec[b * vec_height / 2 + (row / BLOCKHEIGHT4) * blockwidth2 + threadIdx.x];
    __syncthreads();

    while (k < blockwidth2) {
      tmp1 = as_unsigned(mat[i]);

      res2 = {};
      tmp2 = {};

      lut_index1 = tmp1 & 0xF;
      lut_index2 = (tmp1 >> 4) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 0], res2);

      lut_index1 = (tmp1 >> 8) & 0xF;
      lut_index2 = (tmp1 >> 12) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 1], res2);

      lut_index1 = (tmp1 >> 16) & 0xF;
      lut_index2 = (tmp1 >> 20) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 2], res2);

      lut_index1 = (tmp1 >> 24) & 0xF;
      lut_index2 = (tmp1 >> 28) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 3], res2);

      res = __hadd(__hadd(res2.x, res2.y), res);

      i += width;
      k += 4;
    }

    // col%2 -> only set one of the two values
    half2 res3 = {};
    if (col % 2 == 0) {
      res3.x = res;
    } else {
      res3.y = res;
    }

    atomicAdd(&mul[b * width / 2 + col / 2], res3);
  }
}

} // namespace squeezellm
} // namespace vllm

// 4-bit matvec kernel (LUT-based)
void squeezellm_gemm(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table
) {
  int height = mat.size(0);
  int width = mat.size(1);

  int batch = vec.size(0);
  int vec_height = vec.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  vllm::squeezellm::NUQ4MatMulKernel<<<blocks, threads>>>(
    (half2*) vec.data<at::Half>(),
    mat.data_ptr<int>(),
    (half2*) mul.data<at::Half>(),
    (__half*) lookup_table.data<at::Half>(),
    height, width, batch, vec_height
  );
}

#undef BLOCKWIDTH
#undef BLOCKHEIGHT4
