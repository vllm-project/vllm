#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// half-tensor
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/CUDATensorMethods.cuh>

#define BLOCKWIDTH 128
#define BLOCKHEIGHT4 16

namespace vllm {
namespace squeezellm {

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

// 4-bit matvec kernel (LUT-based)
__global__ void NUQ4MatMulKernel(
#ifndef USE_ROCM
    const  half2* __restrict__ vec,
#else
    const  __half2* __restrict__ vec,
#endif
    const    int* __restrict__ mat,
#ifndef USE_ROCM
           half2* __restrict__ mul,
#else
          float2* __restrict__ mul,
#endif
    const  __half* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
) {

  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT4 * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;

#ifndef USE_ROCM
  __shared__ half2 blockvec[blockwidth2];
#else
  __shared__ __half2 blockvec[blockwidth2];
#endif

  __shared__ __half deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 16;
  for (int val = 0; val < 16; val += 1) {
    int lut_index = column_offset + val;
    deq2[val][off] = lookup_table[lut_index];
  }

  __half res;
#ifndef USE_ROCM
  half2 res2;
  half2 tmp2;
#else
  __half2 res2;
  __half2 tmp2;
#endif

  int i;
  int k;

  unsigned int tmp1;
  unsigned int lut_index1, lut_index2;

  for (int b = 0; b < batch; ++b){
    i = width * row + col;
    res = __int2half_rd(0);
    k = 0;

    __syncthreads();
    if (threadIdx.x < blockwidth2)
      blockvec[threadIdx.x] = vec[b * vec_height / 2 + (row / BLOCKHEIGHT4) * blockwidth2 + threadIdx.x];
    __syncthreads();

    while (k < blockwidth2) {
      tmp1 = as_unsigned(mat[i]);

#ifndef USE_ROCM
      res2 = {};
      tmp2 = {};
#else
      res2.x = __half_as_ushort(__float2half(0));
      res2.y = __half_as_ushort(__float2half(0));
      tmp2.x = __half_as_ushort(__float2half(0));
      tmp2.y = __half_as_ushort(__float2half(0));
#endif

      lut_index1 = tmp1 & 0xF;
      lut_index2 = (tmp1 >> 4) & 0xF;
#ifndef USE_ROCM
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
#else
      tmp2.x = __half_as_ushort(deq2[lut_index1][off]);
      tmp2.y = __half_as_ushort(deq2[lut_index2][off]);
#endif
      res2 = __hfma2(tmp2, blockvec[k + 0], res2);

      lut_index1 = (tmp1 >> 8) & 0xF;
      lut_index2 = (tmp1 >> 12) & 0xF;
#ifndef USE_ROCM
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
#else
      tmp2.x = __half_as_ushort(deq2[lut_index1][off]);
      tmp2.y = __half_as_ushort(deq2[lut_index2][off]);
#endif
      res2 = __hfma2(tmp2, blockvec[k + 1], res2);

      lut_index1 = (tmp1 >> 16) & 0xF;
      lut_index2 = (tmp1 >> 20) & 0xF;
#ifndef USE_ROCM
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
#else
      tmp2.x = __half_as_ushort(deq2[lut_index1][off]);
      tmp2.y = __half_as_ushort(deq2[lut_index2][off]);
#endif
      res2 = __hfma2(tmp2, blockvec[k + 2], res2);

      lut_index1 = (tmp1 >> 24) & 0xF;
      lut_index2 = (tmp1 >> 28) & 0xF;
#ifndef USE_ROCM
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
#else
      tmp2.x = __half_as_ushort(deq2[lut_index1][off]);
      tmp2.y = __half_as_ushort(deq2[lut_index2][off]);
#endif
      res2 = __hfma2(tmp2, blockvec[k + 3], res2);

#ifndef USE_ROCM
      res = __hadd(__hadd(res2.x, res2.y), res);
#else
      res = __hadd(__hadd(__ushort_as_half(res2.x), __ushort_as_half(res2.y)), res);
#endif

      i += width;
      k += 4;
    }

    // col%2 -> only set one of the two values
#ifndef USE_ROCM
    half2 res3 = {};
    if (col % 2 == 0) {
      res3.x = res;
    } else {
      res3.y = res;
    }
#else
    __half2 res3;
    res3.x = __half_as_ushort(__float2half(0));
    res3.y = __half_as_ushort(__float2half(0));
    if (col % 2 == 0) {
      res3.x = __half_as_ushort(res);
    } else {
      res3.y = __half_as_ushort(res);
    }
#endif

#ifndef USE_ROCM
    atomicAdd(&mul[b * width / 2 + col / 2], res3);
#else
    int tmp_addr = b * width / 2 + col / 2;
    atomicAdd(&(mul[tmp_addr].x), __half2float(__ushort_as_half(res3.x)));
    atomicAdd(&(mul[tmp_addr].y), __half2float(__ushort_as_half(res3.y)));
#endif
  }
}

} // namespace squeezellm
} // namespace vllm

// 4-bit matvec kernel (LUT-based)
void squeezellm_gemm(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table
) {
  int height = mat.size(0);
  int width = mat.size(1);

  int batch = vec.size(0);
  int vec_height = vec.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  vllm::squeezellm::NUQ4MatMulKernel<<<blocks, threads>>>(
#ifndef USE_ROCM
    (half2*) vec.data<at::Half>(),
#else
    (__half2*) vec.data_ptr<at::Half>(),
#endif
    mat.data_ptr<int>(),
#ifndef USE_ROCM
    (half2*) mul.data<at::Half>(),
    (__half*) lookup_table.data<at::Half>(),
#else
    (float2*) mul.data_ptr<float>(),
    (__half*) lookup_table.data_ptr<at::Half>(),
#endif
    height, width, batch, vec_height
  );
}

#undef BLOCKWIDTH
#undef BLOCKHEIGHT4
