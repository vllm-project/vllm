#include "hip/hip_runtime.h"
#include "allspark_utils.cuh"
#include <torch/all.h>
#include "core/registration.h"
#include <hipblas.h>

at::Tensor as_g_workspace;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

torch::Tensor allspark_w8a16_gemm(
    torch::Tensor const& a, torch::Tensor const& b_qweight,
    torch::Tensor const& b_scales, c10::optional<torch::Tensor> const& b_qzeros,
    int64_t n, int64_t group_size, int64_t sm_count, int64_t sm_version,
    int64_t CUBLAS_M_THRESHOLD, bool has_zp, bool n32k16_reorder) {
  TORCH_CHECK_NOT_IMPLEMENTED(
      false, "allspark_w8a16_gemm(..) requires CUDA_ARCH >= 8.0");
  return torch::empty({1, 1});
}

#else
namespace allspark {
/*
 * GemmTile manage data movement from Global Memory to Shared Memory
 * requiring N % 8 == 0， K % 16 == 0 by loading uint
 * BN is obtained by padding the original N to a multiple of 32
 * weight B is rearranged as N32K16 order,
 * i.e. a initial data block of size 32(n)x16(k) is reordered as n8k4n4k4，
 * in order to put data loaded by the same thread of 32x16 data block together
 * continuously (see
 * https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-floating-point-type)
 */
template <typename FType, typename QType, int Mtile, int Ntile, int NStage,
          int BLOCK>
struct GmemTile_W8A16_PerC_MtilexNtilex32_multistage_SM8x_SplitK {
  // element num loaded by a LDG inst.
  static constexpr int LDG_ELEMENT_CNT_A = 8;
  static constexpr int LDG_ELEMENT_CNT_B = 16;
  static constexpr int WARP_SIZE = 32;
  static constexpr int M_SIZE_ONE_LOAD = (BLOCK * LDG_ELEMENT_CNT_A) / 32;
  static constexpr int N_SIZE_ONE_LOAD = (BLOCK * LDG_ELEMENT_CNT_B) / 32;

  __device__ GmemTile_W8A16_PerC_MtilexNtilex32_multistage_SM8x_SplitK(
      const SM8x_GEMM_W8A16_Splitk_Params<FType, QType>& k_params,
      const uint32_t& A_smem_addr, const uint32_t& BQ_smem_addr,
      const uint32_t& A_stage_stride, const uint32_t& BQ_stage_stride)
      : params(k_params),
        A_smem_base_addr(A_smem_addr),
        BQ_smem_base_addr(BQ_smem_addr),
        A_smem_stage_stride(A_stage_stride),
        BQ_smem_stage_stride(BQ_stage_stride) {
    this_block_A_base_ptr = params.A_ptr + blockIdx.x * Mtile * params.K +
                            blockIdx.z * params.SplitK;
    // here B is rearranged as N32K16 order, i.e. 4 continuous N-direction
    // 8(N)x16(K) size data blocks are packed together
    this_block_B_base_ptr = params.B_ptr + blockIdx.y * Ntile * params.K +
                            blockIdx.z * params.SplitK * 4;

    const int lane_id = threadIdx.x % WARP_SIZE;

    // For matrix A, a block load/store Mtile(row) x 32(col) elements in
    // multiple iters, 8x4 warp load/store 8(row) x 32(col) elements per iter
    const int Aldg_row_base_idx = threadIdx.x / 4;
    Aldg_col_idx = (threadIdx.x % 4) * LDG_ELEMENT_CNT_A;
    const int Aldg_base_offset = Aldg_row_base_idx * params.K + Aldg_col_idx;

    // For matrix B, a block load/store elements of (Ntile / 4) row x 128 col
    // elements of N32K16 packing in multiple iters, 4x8 warp load/store 4(row)
    // * 128(col) per iter
    Bldg_col_idx = (threadIdx.x % 8) * LDG_ELEMENT_CNT_B;
    const int Bldg_row_base_idx = threadIdx.x / 8;
    const int Bldg_base_offset =
        Bldg_row_base_idx * params.K * 4 + Bldg_col_idx;

    this_block_A_base_ptr += Aldg_base_offset;
    this_block_B_base_ptr += Bldg_base_offset;

    const int sts_a_base_offset =
        (threadIdx.x / 4) * 32 +
        ((lane_id % 4) ^ ((lane_id / 4) % 4) ^ ((lane_id / 4) / 4)) *
            LDG_ELEMENT_CNT_A;
    const int sts_bq_base_offset =
        Bldg_row_base_idx * 32 * 4 +
        ((threadIdx.x % 8) ^ (((threadIdx.x / 8) % 2) * 4)) * LDG_ELEMENT_CNT_B;

    A_smem_base_addr += sts_a_base_offset * sizeof(FType);
    BQ_smem_base_addr += sts_bq_base_offset * sizeof(uint8_t);

    A_ldg_guard = 0;
    B_ldg_guard = 0;
  #pragma unroll
    for (int i = 0; i < (Mtile + M_SIZE_ONE_LOAD - 1) / M_SIZE_ONE_LOAD; ++i) {
      int m_idx = blockIdx.x * Mtile + Aldg_row_base_idx + i * M_SIZE_ONE_LOAD;
      if (m_idx < params.M) {
        A_ldg_guard |= (1u << i);
      }
    }

    const int N_padded = (params.N + 31) / 32 * 32;
  #pragma unroll
    for (int i = 0; i < (Ntile + N_SIZE_ONE_LOAD - 1) / N_SIZE_ONE_LOAD; ++i) {
      int n_idx = blockIdx.y * Ntile + (Bldg_row_base_idx / 8) * 32 +
                  i * N_SIZE_ONE_LOAD;
      if (n_idx < N_padded) {
        B_ldg_guard |= (1u << i);
      }
    }
  }

  __device__ void ldgsts_first_ktiles(const int& first_k_tile,
                                      const int& k_tiles) {
    // load first k_tile
    // load A
    const int A_src_size = Aldg_col_idx < first_k_tile ? 16 : 0;
  #pragma unroll
    for (int i = 0; i < (Mtile + M_SIZE_ONE_LOAD - 1) / M_SIZE_ONE_LOAD; ++i) {
      cp_async<16>(
          A_smem_base_addr + (i * M_SIZE_ONE_LOAD * 32) * sizeof(FType),
          this_block_A_base_ptr + i * M_SIZE_ONE_LOAD * params.K, A_src_size,
          (A_ldg_guard & (1u << i)) != 0);
    }

    // load B
    const int B_src_size = (Bldg_col_idx / 4) < first_k_tile ? 16 : 0;
  #pragma unroll
    for (int i = 0; i < (Ntile + N_SIZE_ONE_LOAD - 1) / N_SIZE_ONE_LOAD; ++i) {
      cp_async<16>(
          BQ_smem_base_addr + (i * N_SIZE_ONE_LOAD * 32) * sizeof(uint8_t),
          this_block_B_base_ptr + i * N_SIZE_ONE_LOAD * params.K, B_src_size,
          (B_ldg_guard & (1u << i)) != 0);
    }

    cp_async_commit_group();
    this_block_A_base_ptr += first_k_tile;
    this_block_B_base_ptr += (first_k_tile * 4);

    // load second to (N-stage - 1) k_tiles
    for (int stage_idx = 1; stage_idx < NStage - 1; ++stage_idx) {
      if (stage_idx < k_tiles) {
  #pragma unroll
        for (int i = 0; i < (Mtile + M_SIZE_ONE_LOAD - 1) / M_SIZE_ONE_LOAD;
             ++i) {
          cp_async<16>(A_smem_base_addr + stage_idx * A_smem_stage_stride +
                           (i * M_SIZE_ONE_LOAD * 32) * sizeof(FType),
                       this_block_A_base_ptr + i * M_SIZE_ONE_LOAD * params.K,
                       16, (A_ldg_guard & (1u << i)) != 0);
        }

  #pragma unroll
        for (int i = 0; i < (Ntile + N_SIZE_ONE_LOAD - 1) / N_SIZE_ONE_LOAD;
             ++i) {
          cp_async<16>(BQ_smem_base_addr + stage_idx * BQ_smem_stage_stride +
                           (i * N_SIZE_ONE_LOAD * 32) * sizeof(uint8_t),
                       this_block_B_base_ptr + i * N_SIZE_ONE_LOAD * params.K,
                       16, (B_ldg_guard & (1u << i)) != 0);
        }

        this_block_A_base_ptr += 32;
        this_block_B_base_ptr += (32 * 4);
      }
      cp_async_commit_group();
    }
  }

  __device__ void ldgsts(const int& sts_stage_idx) {
    const int a_stage_offset = sts_stage_idx * A_smem_stage_stride;
    const int bq_stage_offset = sts_stage_idx * BQ_smem_stage_stride;
  #pragma unroll
    for (int i = 0; i < (Mtile + M_SIZE_ONE_LOAD - 1) / M_SIZE_ONE_LOAD; ++i) {
      cp_async<16>(A_smem_base_addr + a_stage_offset +
                       (i * M_SIZE_ONE_LOAD * 32) * sizeof(FType),
                   this_block_A_base_ptr + i * M_SIZE_ONE_LOAD * params.K, 16,
                   (A_ldg_guard & (1u << i)) != 0);
    }

  #pragma unroll
    for (int i = 0; i < (Ntile + N_SIZE_ONE_LOAD - 1) / N_SIZE_ONE_LOAD; ++i) {
      cp_async<16>(BQ_smem_base_addr + bq_stage_offset +
                       (i * N_SIZE_ONE_LOAD * 32) * sizeof(uint8_t),
                   this_block_B_base_ptr + i * N_SIZE_ONE_LOAD * params.K, 16,
                   (B_ldg_guard & (1u << i)) != 0);
    }

    cp_async_commit_group();
    this_block_A_base_ptr += 32;
    this_block_B_base_ptr += (32 * 4);
  }

  const FType* this_block_A_base_ptr = nullptr;
  const QType* this_block_B_base_ptr = nullptr;

  int Aldg_col_idx;
  int Bldg_col_idx;

  uint32_t A_ldg_guard;
  uint32_t B_ldg_guard;

  uint32_t A_smem_base_addr, BQ_smem_base_addr;
  const uint32_t A_smem_stage_stride, BQ_smem_stage_stride;

  const SM8x_GEMM_W8A16_Splitk_Params<FType, QType>& params;
};

/*
 * requiring N % 8 == 0
 */
template <typename FType, typename QType, int Mtile, int Ntile, int BLOCK,
          bool EnableFuse, bool has_zp>
struct ComputeTile_W8A16_PerC_MtilexNtilex32_multistage_SM8x_SplitK {
  static constexpr int WARP_SIZE = 32;
  static constexpr int WARP_CNT = BLOCK / WARP_SIZE;
  static constexpr int WARP_NTILE = Ntile / WARP_CNT;
  static constexpr int WARP_NITER = WARP_NTILE / 8;  // hmma16816
  static_assert(WARP_NTILE == 32 or WARP_NTILE == 64,
                "now only support WARP_NTILE = 32 or 64!");

  __device__ ComputeTile_W8A16_PerC_MtilexNtilex32_multistage_SM8x_SplitK(
      const SM8x_GEMM_W8A16_Splitk_Params<FType, QType>& k_params,
      const uint32_t& A_smem_addr, const uint32_t& BQ_smem_addr,
      const uint32_t& A_stage_stride, const uint32_t& BQ_stage_stride)
      : params(k_params),
        A_smem_base_addr(A_smem_addr),
        BQ_smem_base_addr(BQ_smem_addr),
        A_smem_stage_stride(A_stage_stride),
        BQ_smem_stage_stride(BQ_stage_stride) {
    warp_id = threadIdx.x / WARP_SIZE;
    lane_id = threadIdx.x % WARP_SIZE;

    load_a_base_offset[0] =
        (lane_id % 16) * 32 +
        ((lane_id / 16) ^ (lane_id % 4) ^ ((lane_id / 4) % 2)) * 8;
    load_a_base_offset[1] =
        (lane_id % 16) * 32 +
        ((lane_id / 16 + 2) ^ (lane_id % 4) ^ ((lane_id / 4) % 2)) * 8;

    load_b_base_offset[0] =
        (lane_id / 4 + warp_id * (WARP_NTILE / 4)) * 32 * 4 +
        (lane_id % 4) * 16 + ((lane_id / 4) % 2) * 16 * 4;
    load_b_base_offset[1] =
        (lane_id / 4 + warp_id * (WARP_NTILE / 4)) * 32 * 4 +
        (lane_id % 4) * 16 + (((lane_id / 4) % 2) ^ 1) * 16 * 4;

    sts_c_base_offset = warp_id * Mtile * WARP_NTILE +
                        (lane_id / 4) * WARP_NTILE + (lane_id % 4) * 2;

    if (EnableFuse) {
      this_block_C_base_ptr =
          params.C_ptr + blockIdx.x * Mtile * params.N + blockIdx.y * Ntile;
    } else {
      this_block_C_base_ptr =
          params.C_split_ptr + blockIdx.z * params.M * params.N +
          blockIdx.x * Mtile * params.N + blockIdx.y * Ntile;
    }
    int store_thds_in_row = WARP_NTILE / 8;
    store_c_row_base_idx = lane_id / store_thds_in_row;
    store_c_col_idx = warp_id * WARP_NTILE + (lane_id % store_thds_in_row) * 8;
    store_c_base_offset = store_c_row_base_idx * params.N + store_c_col_idx;

  #pragma unroll
    for (int i = 0; i < Mtile / 16; ++i) {
  #pragma unroll
      for (int j = 0; j < WARP_NITER; ++j) {
  #pragma unroll
        for (int k = 0; k < 4; ++k) {
          C_frag[i][j][k] = 0.f;
        }
      }
    }
    params_n_idx =
        blockIdx.y * Ntile + warp_id * WARP_NTILE + (lane_id / 4) * 4;
  }

  __device__ void lds(const int& smem_stage_idx, const int& reg_buf_idx,
                      const int& k_phase_idx) {
    uint32_t A_smem_addr =
        A_smem_base_addr + A_smem_stage_stride * smem_stage_idx;
    uint32_t B_smem_addr =
        BQ_smem_base_addr + BQ_smem_stage_stride * smem_stage_idx;

  #pragma unroll
    for (int i = 0; i < Mtile / 16; ++i) {
      ldsm_4(A_frag[reg_buf_idx][i][0], A_frag[reg_buf_idx][i][1],
             A_frag[reg_buf_idx][i][2], A_frag[reg_buf_idx][i][3],
             A_smem_addr + (load_a_base_offset[k_phase_idx] + i * 16 * 32) *
                               sizeof(FType));
    }
  #pragma unroll
    for (int i = 0; i < WARP_NTILE / 32; ++i) {
      lds128(BQ_frag[reg_buf_idx][4 * i + 0], BQ_frag[reg_buf_idx][4 * i + 1],
             BQ_frag[reg_buf_idx][4 * i + 2], BQ_frag[reg_buf_idx][4 * i + 3],
             B_smem_addr + (load_b_base_offset[k_phase_idx] + i * 32 * 32) *
                               sizeof(uint8_t));
    }

  // dequant B
  #pragma unroll
    for (int i = 0; i < WARP_NITER / 2; ++i) {
      cvt_8bx4_to_16bx4_bias128(BQ_frag[reg_buf_idx][2 * i],
                                BF_frag[reg_buf_idx][2 * i]);
      if (has_zp) {
        BF_frag[reg_buf_idx][2 * i][0] =
            __hsub2(BF_frag[reg_buf_idx][2 * i][0], num2num2(B_zero[i].x));
        BF_frag[reg_buf_idx][2 * i][1] =
            __hsub2(BF_frag[reg_buf_idx][2 * i][1], num2num2(B_zero[i].x));
      }

      BF_frag[reg_buf_idx][2 * i][0] =
          __hmul2(BF_frag[reg_buf_idx][2 * i][0], num2num2(B_scale[i].x));
      BF_frag[reg_buf_idx][2 * i][1] =
          __hmul2(BF_frag[reg_buf_idx][2 * i][1], num2num2(B_scale[i].x));

      cvt_8bx4_to_16bx4_bias128(BQ_frag[reg_buf_idx][2 * i + 1],
                                BF_frag[reg_buf_idx][2 * i + 1]);
      if (has_zp) {
        BF_frag[reg_buf_idx][2 * i + 1][0] =
            __hsub2(BF_frag[reg_buf_idx][2 * i + 1][0], num2num2(B_zero[i].y));
        BF_frag[reg_buf_idx][2 * i + 1][1] =
            __hsub2(BF_frag[reg_buf_idx][2 * i + 1][1], num2num2(B_zero[i].y));
      }

      BF_frag[reg_buf_idx][2 * i + 1][0] =
          __hmul2(BF_frag[reg_buf_idx][2 * i + 1][0], num2num2(B_scale[i].y));
      BF_frag[reg_buf_idx][2 * i + 1][1] =
          __hmul2(BF_frag[reg_buf_idx][2 * i + 1][1], num2num2(B_scale[i].y));
    }
  }

  __device__ void ldg_params() {
    const int N_padded = (params.N + 31) / 32 * 32;
    // load B scale and zero_point
  #pragma unroll
    for (int i = 0; i < WARP_NTILE / 32; ++i) {
      ldg64_ca(B_scale[2 * i + 0], B_scale[2 * i + 1],
               params.B_scale_ptr + params_n_idx + i * 32,
               (params_n_idx + i * 32) < N_padded);
      if (has_zp) {
        ldg64_ca(B_zero[2 * i + 0], B_zero[2 * i + 1],
                 params.B_zero_ptr + params_n_idx + i * 32,
                 (params_n_idx + i * 32) < N_padded);
      }
    }
  }

  __device__ void mma(const int& reg_buf_idx) {
  #pragma unroll
    for (int m_idx = 0; m_idx < Mtile / 16; ++m_idx) {
  #pragma unroll
      for (int n_idx = 0; n_idx < WARP_NITER; ++n_idx) {
        hmma16816_f32<FType>(
            C_frag[m_idx][n_idx], A_frag[reg_buf_idx][m_idx],
            reinterpret_cast<uint32_t(&)[2]>(BF_frag[reg_buf_idx][n_idx]));
      }
    }
  }

  __device__ void fused_splitk_reduce() {
    // need splitk-reduce if enable splitk
    if (gridDim.z > 1) {
      int blk_red_idx = blockIdx.x * gridDim.y + blockIdx.y;
      // Wait for all previous blocks in the splitk direction to accumulate the
      // results into C_tmp
      if (threadIdx.x == 0) {
        uint32_t* red_count_ptr = params.red_count_ptr + blk_red_idx;
        uint32_t count;
        do {
          // make sure the ld.cg inside the do-wile loop
          __threadfence_block();
          asm volatile("ld.global.cg.b32 %0, [%1];"
                       : "=r"(count)
                       : "l"(red_count_ptr));
        } while (count != blockIdx.z);
      }
      __syncthreads();

      int C_tmp_base_offset = blk_red_idx * Mtile * Ntile + threadIdx.x * 4;
      if (blockIdx.z != 0) {
        // expecting that temporary register here reuses the previous A&B frag
        // register
        float temp_frag[Mtile / 16][WARP_NITER][4];
  #pragma unroll
        for (int m_idx = 0; m_idx < Mtile / 16; ++m_idx) {
  #pragma unroll
          for (int n_idx = 0; n_idx < WARP_NITER; ++n_idx) {
            int offset =
                C_tmp_base_offset + (m_idx * WARP_NITER + n_idx) * BLOCK * 4;
            *reinterpret_cast<int4*>(temp_frag[m_idx][n_idx]) =
                *reinterpret_cast<int4*>(params.C_tmp_ptr + offset);
          }
        }
  #pragma unroll
        for (int m_idx = 0; m_idx < Mtile / 16; ++m_idx) {
  #pragma unroll
          for (int n_idx = 0; n_idx < WARP_NITER; ++n_idx) {
  #pragma unroll
            for (int idx = 0; idx < 4; ++idx) {
              C_frag[m_idx][n_idx][idx] += temp_frag[m_idx][n_idx][idx];
            }
          }
        }
      }

      // first splitk - 1 blocks need to write partial results into C_tmp
      if (blockIdx.z != gridDim.z - 1) {
  #pragma unroll
        for (int m_idx = 0; m_idx < Mtile / 16; ++m_idx) {
  #pragma unroll
          for (int n_idx = 0; n_idx < WARP_NITER; ++n_idx) {
            int offset =
                C_tmp_base_offset + (m_idx * WARP_NITER + n_idx) * BLOCK * 4;
            asm volatile(
                "{st.global.cg.v4.b32 [%0], {%1, %2, %3, %4};}\n"
                :
                : "l"(params.C_tmp_ptr + offset), "f"(C_frag[m_idx][n_idx][0]),
                  "f"(C_frag[m_idx][n_idx][1]), "f"(C_frag[m_idx][n_idx][2]),
                  "f"(C_frag[m_idx][n_idx][3]));
          }
        }
        __threadfence();
        __syncthreads();
        if (threadIdx.x == 0) {
          uint32_t* red_count_ptr = params.red_count_ptr + blk_red_idx;
          atomicInc(red_count_ptr, gridDim.z);
        }
      }
    }
  }

  __device__ void stg(char* smem) {
    if (EnableFuse) {
      if (blockIdx.z != gridDim.z - 1) return;
    }
    uint32_t* C_sts_ptr =
        reinterpret_cast<uint32_t*>(smem + sts_c_base_offset * sizeof(FType));
    // C_tile sts
  #pragma unroll
    for (int m_idx = 0; m_idx < Mtile / 16; ++m_idx) {
  #pragma unroll
      for (int n_idx = 0; n_idx < WARP_NITER; ++n_idx) {
  #pragma unroll
        for (int k_idx = 0; k_idx < 2; ++k_idx) {
          FType low16 = static_cast<FType>(C_frag[m_idx][n_idx][k_idx * 2]);
          FType high16 =
              static_cast<FType>(C_frag[m_idx][n_idx][k_idx * 2 + 1]);
          uint32_t tmp = (reinterpret_cast<uint32_t&>(low16) & 0xffff) |
                         (reinterpret_cast<uint32_t&>(high16) << 16);
          int sts_offset =
              m_idx * 16 * (WARP_NTILE / 2) +
              (((lane_id / (32 / WARP_NITER)) + n_idx) % WARP_NITER) * (8 / 2) +
              k_idx * 8 * (WARP_NTILE / 2);
          C_sts_ptr[sts_offset] = tmp;
        }
      }
    }

    __syncthreads();

    FType* C_base_ptr = this_block_C_base_ptr + store_c_base_offset;
    // C_tile lds and stg
    int m_base_idx = store_c_row_base_idx + blockIdx.x * Mtile;
    bool n_guard = (store_c_col_idx + blockIdx.y * Ntile) < params.N;
    if (WARP_NTILE == 32) {
      int lds_c_base_offset = warp_id * Mtile * WARP_NTILE +
                              (lane_id / 4) * WARP_NTILE +
                              ((lane_id % 4 + lane_id / 8) % 4) * 8;
      uint4* C_lds_ptr =
          reinterpret_cast<uint4*>(smem + lds_c_base_offset * sizeof(FType));
  #pragma unroll
      for (int i = 0; i < (Mtile / 16) * (WARP_NITER / 2); ++i) {
        uint4 stg_reg = C_lds_ptr[i * 8 * 4];
        stg128(stg_reg.x, stg_reg.y, stg_reg.z, stg_reg.w,
               C_base_ptr + i * 8 * params.N,
               (m_base_idx + i * 8) < params.M && n_guard);
      }
    } else if (WARP_NTILE == 64) {
      int lds_c_base_offset =
          warp_id * Mtile * WARP_NTILE + (lane_id / 8) * WARP_NTILE;
  #pragma unroll
      for (int i = 0; i < (Mtile / 16) * (WARP_NITER / 2); ++i) {
        int lds_c_offset = lds_c_base_offset + i * 4 * WARP_NTILE +
                           ((lane_id % 8 + lane_id / 8 + (i % 2) * 4) % 8) * 8;
        uint4 stg_reg =
            *reinterpret_cast<uint4*>(smem + lds_c_offset * sizeof(FType));
        stg128(stg_reg.x, stg_reg.y, stg_reg.z, stg_reg.w,
               C_base_ptr + i * 4 * params.N,
               (m_base_idx + i * 4) < params.M && n_guard);
      }
    }
  }

  const SM8x_GEMM_W8A16_Splitk_Params<FType, QType>& params;

  int load_a_base_offset[2];
  int load_b_base_offset[2];
  int sts_c_base_offset;

  int store_c_base_offset;

  int store_c_row_base_idx, store_c_col_idx;
  FType* this_block_C_base_ptr = nullptr;

  int params_n_idx;
  const uint32_t A_smem_base_addr, BQ_smem_base_addr;
  const uint32_t A_smem_stage_stride, BQ_smem_stage_stride;

  int lane_id;
  int warp_id;
  // first 2 denotes double buffer, second dim denotes M direction
  uint32_t A_frag[2][Mtile / 16][4];

  typename HalfType<FType>::T2 B_scale[WARP_NITER / 2];
  typename HalfType<FType>::T2 B_zero[WARP_NITER / 2];
  uint32_t BQ_frag[2][WARP_NITER];
  // first 2 denotes double buffer, second dim denotes N direction, last 2
  // denotes K direction
  typename HalfType<FType>::T2 BF_frag[2][WARP_NITER][2];
  // first dim denotes M direction, second dim denotes N direction
  float C_frag[Mtile / 16][WARP_NITER][4];
};

/*
 *  @brief W8A16 Perchannel Quantization GEMM,
 *         requires N % 8 == 0, K % 16 == 0
 *         accumulator precision: FP32
 *  @tparam FType: DataType for A, B_scale, B_zero, and C, supports half or
 * hip_bfloat16
 *  @tparam QType: DataType for B, support uint8(bias128)
 *  @tparam Mtile: M-dimensional size of the gemm block tile, supports 16, 32,
 * 48 or 64
 *  @tparam Ntile: N-dimensional size of the gemm block tile, supports 128 or
 * 256
 *  @tparam NStage: Num of stages for async copy
 *  @tparam BLOCK: BLOCK size
 *  @tparam EnableFuse: If true, use fused splitk-reduce, otherwise use
 * non-fused splitk-reduce
 *  @tparam has_zp: whether to use zero_point
 *
 *  @fparam params struct consists of following parameters:
 *      @param A_ptr: Matrix A value ptr, A = (M, K)
 *      @param B_ptr: Matrix B value ptr, B = (N32_align, K) (N32K16 special
 * format), N32_align = (N + 32 - 1) / 32 * 32
 *      @param B_scale_ptr: B_scale value ptr, B_scale = (N32_align,) (N32K16
 * special format)
 *      @param B_zero_ptr: B_zero value ptr, B_zero = (N32_align,) (N32K16
 * special format)
 *      @param C_ptr: Matrix C value ptr, C = (M, N)
 *      @param M: dimnesion m
 *      @param N: dimnesion n
 *      @param K: dimnesion k
 *      @param SplitK: split size along K-dimension
 *      @param C_split_ptr: Matrix C_split value ptr, used only in non-fused
 * splitk-reduce
 *      @param C_tmp_ptr: Matrix C_tmp value ptr, used only in fused
 * splitk-reduce
 *      @param red_count_ptr: 1-D red_count value ptr, used only in fused
 * splitk-reduce
 */
template <typename FType, typename QType, int Mtile, int Ntile, int NStage,
          int BLOCK, bool EnableFuse, bool has_zp>
__global__ void __launch_bounds__(BLOCK)
    ampere_hgemm_W8A16_perc_f16_f16_MtilexNtilex32_hmma16816_multistage_AN_BTN32K16_CN_splitk_kernel(
        const SM8x_GEMM_W8A16_Splitk_Params<FType, QType> params) {
  // A smem size = 64 * 32 * 2B/elem * 4(stage) = 16KB
  // B smem size = 128 * 32 * 1B/elem * 4(stage) = 16KB
  constexpr int smem_size_one_stage = Mtile * 32 * 2 + Ntile * 32;
  __shared__ char smem[NStage * smem_size_one_stage];
  char* A_smem = smem;
  char* BQ_smem = smem + Mtile * 32 * 2 * NStage;

  uint32_t A_smem_addr = smem_u32addr(A_smem);
  uint32_t BQ_smem_addr = smem_u32addr(BQ_smem);
  uint32_t A_smem_stage_stride = Mtile * 32 * 2;
  uint32_t BQ_smem_stage_stride = Ntile * 32;

  // initialize the data move process from GM to SMEM for this block
  GmemTile_W8A16_PerC_MtilexNtilex32_multistage_SM8x_SplitK<
      FType, QType, Mtile, Ntile, NStage, BLOCK>
      gmem_tile(params, A_smem_addr, BQ_smem_addr, A_smem_stage_stride,
                BQ_smem_stage_stride);

  int sts_stage_idx = 0;
  int lds_stage_idx = 0;

  int tb_k_slice = blockIdx.z * params.SplitK + params.SplitK <= params.K
                       ? params.SplitK
                       : params.K - blockIdx.z * params.SplitK;
  int k_tiles = (tb_k_slice + 31) / 32;
  int first_k_tile = tb_k_slice - (k_tiles - 1) * 32;

  // load first three tiles to shared memory
  gmem_tile.ldgsts_first_ktiles(first_k_tile, k_tiles);
  sts_stage_idx += (NStage - 2);
  ComputeTile_W8A16_PerC_MtilexNtilex32_multistage_SM8x_SplitK<
      FType, QType, Mtile, Ntile, BLOCK, EnableFuse, has_zp>
      compute_tile(params, A_smem_addr, BQ_smem_addr, A_smem_stage_stride,
                   BQ_smem_stage_stride);
  compute_tile.ldg_params();
  cp_asyc_wait_group<NStage - 2>();
  __syncthreads();

  compute_tile.lds(lds_stage_idx, 0, 0);
  int reg_buf_idx = 1;

  // main loop
  for (; k_tiles > NStage - 1; --k_tiles) {
    // load next A&B tile
    sts_stage_idx = sts_stage_idx < NStage - 1 ? sts_stage_idx + 1 : 0;
    gmem_tile.ldgsts(sts_stage_idx);

  #pragma unroll
    for (int k_phase_idx = 0; k_phase_idx < 2; k_phase_idx++) {
      // dequantize next B tile
      if (k_phase_idx == 1) {
        cp_asyc_wait_group<NStage - 2>();
        __syncthreads();
        lds_stage_idx = lds_stage_idx < NStage - 1 ? lds_stage_idx + 1 : 0;
      }

      compute_tile.lds(lds_stage_idx, reg_buf_idx, (k_phase_idx + 1) % 2);

      compute_tile.mma(reg_buf_idx ^ 1);
      reg_buf_idx ^= 1;
    }
  }

  // last NStage-1 tiles
  for (; k_tiles > 0; --k_tiles) {
    cp_async_commit_group();
  #pragma unroll
    for (int k_phase_idx = 0; k_phase_idx < 2; k_phase_idx++) {
      // dequantize next B tile
      if (k_phase_idx == 1) {
        cp_asyc_wait_group<NStage - 2>();
        __syncthreads();
        lds_stage_idx = lds_stage_idx < NStage - 1 ? lds_stage_idx + 1 : 0;
      }

      compute_tile.lds(lds_stage_idx, reg_buf_idx, (k_phase_idx + 1) % 2);

      compute_tile.mma(reg_buf_idx ^ 1);
      reg_buf_idx ^= 1;
    }
  }

  if (EnableFuse) {
    compute_tile.fused_splitk_reduce();
  }
  compute_tile.stg(smem);
}

  #define __CALL_IF(MTILE, NTILE, NUM_THREADS, ENABLE_FUSE, HAS_ZP)                                     \
    else if (Mtile == MTILE && Ntile == NTILE && BLOCK == NUM_THREADS &&                                \
             enable_fuse == ENABLE_FUSE && has_zp == HAS_ZP) {                                          \
      ampere_hgemm_W8A16_perc_f16_f16_MtilexNtilex32_hmma16816_multistage_AN_BTN32K16_CN_splitk_kernel< \
          FType, QType, MTILE, NTILE, 4, NUM_THREADS, ENABLE_FUSE, HAS_ZP>                              \
          <<<grid, block, 0, stream>>>(params);                                                         \
    }

template <typename FType, typename QType>
void ampere_hgemm_W8A16_perc_f16_f16_MtilexNtilex32_mma16816_multistage_AN_BTN32K16_CN_splitk(
    const FType* A, const QType* B, const FType* B_scale, const FType* B_zero,
    FType* C, const int M, const int N, const int K, void* workspace,
    const int sm_version, const BlockTileSplitkParams& fused_gemm_params,
    hipStream_t stream) {
  int Mtile = fused_gemm_params.Mtile;
  int grid_x = (M + Mtile - 1) / Mtile;
  int Ntile = fused_gemm_params.Ntile;
  int grid_y = (N + Ntile - 1) / Ntile;
  int SplitK = fused_gemm_params.SplitK;
  int grid_z = (K + SplitK - 1) / SplitK;

  int BLOCK = (Ntile == 256) ? 256 : 128;

  dim3 grid(grid_x, grid_y, grid_z);
  dim3 block(BLOCK);

  bool enable_fuse = fused_gemm_params.EnableFuse;
  bool has_zp = B_zero != nullptr;
  if (enable_fuse) {
    float* C_tmp = reinterpret_cast<float*>(workspace);
    uint32_t* red_count = reinterpret_cast<uint32_t*>(
        (char*)workspace + grid_x * Mtile * grid_y * Ntile * sizeof(float));
    CHECK_CUDA(hipMemsetAsync(red_count, 0, grid_x * grid_y * sizeof(uint32_t),
                               stream));
    SM8x_GEMM_W8A16_Splitk_Params<FType, QType> params{
        A, B,      B_scale, B_zero, C,       M,     N,
        K, SplitK, 0,       -1,     nullptr, C_tmp, red_count};

    if (false) {
    }
    // Select the template parameters for kernel launch
    // according to the above settings. Tuning is not supported.
    __CALL_IF(16, 256, 256, true, false)
    __CALL_IF(32, 256, 256, true, false)
    __CALL_IF(48, 256, 256, true, false)
    __CALL_IF(64, 128, 128, true, false)
    __CALL_IF(64, 256, 256, true, false)
    __CALL_IF(16, 256, 256, true, true)
    __CALL_IF(32, 256, 256, true, true)
    __CALL_IF(48, 256, 256, true, true)
    __CALL_IF(64, 128, 128, true, true)
    __CALL_IF(64, 256, 256, true, true)
  } else {
    FType* C_split = reinterpret_cast<FType*>(workspace);
    SM8x_GEMM_W8A16_Splitk_Params<FType, QType> params{
        A, B,      B_scale, B_zero, C,       M,       N,
        K, SplitK, 0,       -1,     C_split, nullptr, nullptr};

    if (false) {
    }
    // Select the template parameters for kernel launch
    // according to the above settings. Tuning is not supported.
    __CALL_IF(16, 256, 256, false, false)
    __CALL_IF(32, 256, 256, false, false)
    __CALL_IF(48, 256, 256, false, false)
    __CALL_IF(64, 128, 128, false, false)
    __CALL_IF(64, 256, 256, false, false)
    __CALL_IF(16, 256, 256, false, true)
    __CALL_IF(32, 256, 256, false, true)
    __CALL_IF(48, 256, 256, false, true)
    __CALL_IF(64, 128, 128, false, true)
    __CALL_IF(64, 256, 256, false, true)

    // SplitK reduce
    f16_gemm_splitk_reduce(C_split, C, M, N, grid_z, stream);
  }
}

size_t allspark_qgemm_w8a16_perc_n32k16_ampere_workspace_size(
    int m, int n, int k, int sm_count,
    BlockTileSplitkParams& fused_gemm_params) {
  // Determine the block tile and splitk strategy
  int m16_times = (m + 16 - 1) / 16;
  int Mtile = m16_times <= 4 ? m16_times * 16 : 64;
  int grid_x = (m + Mtile - 1) / Mtile;
  int Ntile =
      (float(grid_x * ((n + 127) / 128)) / sm_count > 10) || (Mtile < 64) ? 256
                                                                          : 128;
  int grid_y = (n + Ntile - 1) / Ntile;
  int grid_z;

  // split-k
  const float SPLIT_THRESHOLD = 0.8;
  int n_slice;
  for (n_slice = 1; n_slice < k / 256; ++n_slice) {
    int n_block = grid_x * grid_y * n_slice;
    if (n_block >= sm_count * SPLIT_THRESHOLD &&
        (n_block % sm_count == 0 || n_block % sm_count >= sm_count * 0.5)) {
      break;
    }
  }

  int k_slice =
      (k / n_slice) % 32 == 0 ? k / n_slice : k / n_slice / 32 * 32 + 32;
  grid_z = (k + k_slice - 1) / k_slice;
  bool enable_fuse = float(grid_x * grid_y) / sm_count >= 0.5 ? 1 : 0;

  size_t ws_size;
  if (enable_fuse) {
    ws_size = grid_x * Mtile * grid_y * Ntile * sizeof(float)  // For C_tmp
              + grid_x * grid_y * sizeof(uint32_t);            // For red_count
  } else {
    ws_size = grid_z * m * n * sizeof(__half);
  }

  fused_gemm_params.Mtile = Mtile;
  fused_gemm_params.Ntile = Ntile;
  fused_gemm_params.SplitK = k_slice;
  fused_gemm_params.EnableFuse = enable_fuse;
  return ws_size;
}

// restore from N32K16 order to original N-major order
// K % 16 == 0, N % 8 == 0
// each block process 64(k) * 32(n) result elements
template <typename FT, typename QT>
__global__ void restore_N32_K16_dequantize_rhs_w8a16_perc_kernel(
    const QT* qdata, const FT* scales, const FT* zeros, FT* fdata,
    const int N_32align, const int N, const int K) {
  __shared__ FT smem[64 * 32];
  int warp_id = threadIdx.x / 32;
  int lane_id = threadIdx.x % 32;
  const int src_row_idx = blockIdx.x * 8 + lane_id / 4;
  const int src_col_idx =
      blockIdx.y * 64 * 4 + warp_id * 16 * 4 + (lane_id % 4) * 16;
  const int src_offset = src_row_idx * K * 4 + src_col_idx;
  int params_nidx = blockIdx.x * 32 + (lane_id / 4) * 4;

  QT qval_reg[16];
  const QT* pdata = qdata + src_offset;
  if (src_col_idx < (K * 4)) {
    *(reinterpret_cast<uint4*>(qval_reg)) =
        *(reinterpret_cast<const uint4*>(qdata + src_offset));
  }
  FT scale_reg[4];
  *(reinterpret_cast<uint2*>(scale_reg)) =
      *(reinterpret_cast<const uint2*>(scales + params_nidx));
  FT zero_reg[4] = {0};
  if (zeros != nullptr) {
    *(reinterpret_cast<uint2*>(zero_reg)) =
        *(reinterpret_cast<const uint2*>(zeros + params_nidx));
  }
  FT fval_reg[16];

  const int sts_base_offset =
      (warp_id * 16 + (lane_id % 4) * 2) * 32 + lane_id / 4;
  #pragma unroll
  for (int ni = 0; ni < 4; ++ni) {
    cvt_8bx4_to_16bx4_bias128(
        *reinterpret_cast<uint32_t*>(&qval_reg[ni * 4]),
        reinterpret_cast<typename HalfType<FT>::T2*>(&(fval_reg[ni * 4])));
  #pragma unroll
    for (int ki = 0; ki < 4; ++ki) {
      fval_reg[ni * 4 + ki] =
          (fval_reg[ni * 4 + ki] - zero_reg[ni]) * scale_reg[ni];
      int sts_offset = sts_base_offset + ((ki / 2) * 8 + (ki % 2)) * 32 +
                       ((ni + lane_id % 4) % 4) * 8;
      smem[sts_offset] = fval_reg[ni * 4 + ki];
    }
  }
  __syncthreads();

  const int lds_base_offset =
      (threadIdx.x / 4) * 32 + ((threadIdx.x % 4 + threadIdx.x / 8) % 4) * 8;
  #pragma unroll
  for (int i = 0; i < 2; ++i) {
    *reinterpret_cast<uint4*>(fval_reg + i * 8) =
        *reinterpret_cast<uint4*>(smem + lds_base_offset + i * 32 * 32);
  }

  const int dst_row_base_kidx = blockIdx.y * 64 + threadIdx.x / 4;
  const int dst_col_nidx = blockIdx.x * 32 + (threadIdx.x % 4) * 8;
  #pragma unroll
  for (int i = 0; i < 2; ++i) {
    int dst_row_kidx = dst_row_base_kidx + i * 32;
    int dst_offset = dst_row_kidx * N + dst_col_nidx;
    if (dst_row_kidx < K && dst_col_nidx < N) {
      *reinterpret_cast<uint4*>(fdata + dst_offset) =
          *reinterpret_cast<uint4*>(fval_reg + i * 8);
    }
  }
}

template <typename FT, typename QT>
void restore_N32_K16_dequantize_rhs_w8a16(const QT* qdata, const FT* scales,
                                          const FT* zeros, FT* fdata,
                                          const int N_32align, const int N,
                                          const int K, const int GroupSize,
                                          hipStream_t stream) {
  TORCH_CHECK(N % 8 == 0 && K % 16 == 0 && N_32align % 32 == 0,
              "Unsupported shape");
  if (GroupSize == -1) {
    const int BLOCK = 128;
    dim3 grid(N_32align / 32, ((K / 16) + 3) / 4);
    restore_N32_K16_dequantize_rhs_w8a16_perc_kernel<FT, QT>
        <<<grid, BLOCK, 0, stream>>>(qdata, scales, zeros, fdata, N_32align, N,
                                     K);
  }
  // TODO: Support SubChannel
  else {
    TORCH_CHECK(false, "Now only support PerChannel");
  }
}

template <typename FT, typename QT>
void w8a16_gemm_dq_cublas(const FT* in, const QT* rhs_qdata_ptr,
                          const FT* rhs_scales_ptr, const FT* rhs_zeros_ptr,
                          FT* out, void* workspace, const int M,
                          const int N_32align, const int N, const int K,
                          const int group_size, hipStream_t stream,
                          hipblasHandle_t handle) {
  static_assert(
      std::is_same<FT, half>::value || std::is_same<FT, hip_bfloat16>::value,
      "only float16 and bfloat16 is supported");
  // Dequant
  FT* rhs_fdata_ptr = static_cast<FT*>(workspace);
  restore_N32_K16_dequantize_rhs_w8a16(rhs_qdata_ptr, rhs_scales_ptr,
                                       rhs_zeros_ptr, rhs_fdata_ptr, N_32align,
                                       N, K, group_size, stream);
  // cuBLAS GEMM
  int lda = K;
  int ldb = N;
  int ldc = N;
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipDataType cuda_type;
  if (std::is_same<FT, __half>::value) {
    cuda_type = HIP_R_16F;
  } else {
    cuda_type = HIP_R_16BF;
  }
  CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha,
                            rhs_fdata_ptr, cuda_type, ldb, in, cuda_type, lda,
                            &beta, out, cuda_type, ldc, HIP_R_32F,
                            CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

template <typename FType, typename QType>
void allspark_qgemm_w8a16_perc_ampere(
    const FType* A, const QType* B, const FType* B_scale, const FType* B_zero,
    FType* C, const int M, const int N_32align, const int N, const int K,
    void* workspace, const BlockTileSplitkParams& fused_gemm_params,
    const int group_size, int CUBLAS_M_THRESHOLD, const int sm_version,
    hipStream_t stream, hipblasHandle_t handle) {
  if (M > CUBLAS_M_THRESHOLD) {
    w8a16_gemm_dq_cublas<FType, QType>(A, B, B_scale, B_zero, C, workspace, M,
                                       N_32align, N, K, group_size, stream,
                                       handle);
  } else {
    ampere_hgemm_W8A16_perc_f16_f16_MtilexNtilex32_mma16816_multistage_AN_BTN32K16_CN_splitk<
        FType, QType>(A, B, B_scale, B_zero, C, M, N, K, workspace, sm_version,
                      fused_gemm_params, stream);
  }
}

}  // namespace allspark

torch::Tensor allspark_w8a16_gemm(
    torch::Tensor const& a, torch::Tensor const& b_qweight,
    torch::Tensor const& b_scales, c10::optional<torch::Tensor> const& b_qzeros,
    int64_t n, int64_t group_size, int64_t sm_count, int64_t sm_version,
    int64_t CUBLAS_M_THRESHOLD, bool has_zp, bool n32k16_reorder) {
  // Verify device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  TORCH_CHECK(b_qweight.device().is_cuda(), "b_qweight is not on GPU");
  TORCH_CHECK(b_qweight.is_contiguous(), "b_qweight is not contiguous");

  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  if (has_zp) {
    TORCH_CHECK(b_qzeros.value().device().is_cuda(), "b_qzeros is not on GPU");
    TORCH_CHECK(b_qzeros.value().is_contiguous(), "b_qzeros is not contiguous");
  }

  int m = a.size(0);
  int n_32align = (n + 32 - 1) / 32 * 32;
  int k = a.size(1);

  // Verify shape
  TORCH_CHECK(b_qweight.size(0) == n_32align,
              "Shape mismatch: b_qweight.size(0) = ", b_qweight.size(0),
              ", n_32align = ", n_32align);
  TORCH_CHECK(b_qweight.size(1) == k,
              "Shape mismatch: b_qweight.size(1) = ", b_qweight.size(1),
              ", k = ", k);

  TORCH_CHECK(group_size == -1, "Currently only supports group_size = -1");

  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  const void* a_ptr = reinterpret_cast<const void*>(a.data_ptr());
  const uint8_t* b_ptr = reinterpret_cast<const uint8_t*>(b_qweight.data_ptr());
  const void* b_scale_ptr = reinterpret_cast<const void*>(b_scales.data_ptr());
  const void* b_zero_ptr = nullptr;
  if (b_qzeros.has_value()) {
    b_zero_ptr = reinterpret_cast<const void*>(b_qzeros.value().data_ptr());
  }

  auto c_options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c = torch::empty({m, n}, c_options);
  void* c_ptr = reinterpret_cast<void*>(c.data_ptr());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

  allspark::BlockTileSplitkParams fused_gemm_params;

  size_t ws_size = 0;
  if (m > CUBLAS_M_THRESHOLD) {
    ws_size = k * n * 2;  // sizeof(f16)==2
  } else {
    ws_size = allspark::allspark_qgemm_w8a16_perc_n32k16_ampere_workspace_size(
        m, n, k, sm_count, fused_gemm_params);
  }

  auto ws_options = torch::TensorOptions().dtype(at::kChar).device(a.device());
  if (as_g_workspace.numel() <
      ws_size) {  // ws_options: kChar, so numel() is bytes
    as_g_workspace = torch::empty({long(ws_size)}, ws_options);
  }
  void* ws = reinterpret_cast<void*>(as_g_workspace.data_ptr());

  if (a.dtype() == at::ScalarType::Half) {
    allspark::allspark_qgemm_w8a16_perc_ampere<__half, uint8_t>(
        reinterpret_cast<const __half*>(a_ptr), b_ptr,
        reinterpret_cast<const __half*>(b_scale_ptr),
        reinterpret_cast<const __half*>(b_zero_ptr),
        reinterpret_cast<__half*>(c_ptr), m, n_32align, n, k, ws,
        fused_gemm_params, group_size, CUBLAS_M_THRESHOLD, sm_version, stream,
        handle);
  } else if (a.dtype() == at::ScalarType::BFloat16) {
    allspark::allspark_qgemm_w8a16_perc_ampere<__hip_bfloat16, uint8_t>(
        reinterpret_cast<const __hip_bfloat16*>(a_ptr), b_ptr,
        reinterpret_cast<const __hip_bfloat16*>(b_scale_ptr),
        reinterpret_cast<const __hip_bfloat16*>(b_zero_ptr),
        reinterpret_cast<__hip_bfloat16*>(c_ptr), m, n_32align, n, k, ws,
        fused_gemm_params, group_size, CUBLAS_M_THRESHOLD, sm_version, stream,
        handle);
  }

  return c;
}

#endif

TORCH_LIBRARY_IMPL_EXPAND(TORCH_EXTENSION_NAME, CUDA, m) {
  m.impl("allspark_w8a16_gemm", &allspark_w8a16_gemm);
}