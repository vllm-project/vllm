#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include "ggml-common.h"
#include "vecdotq.cuh"
#include "dequantize.cuh"
#include "mmvq.cuh"
#include "mmq.cuh"

// Q8 gemv
static __global__ void quantize_q8_1(const half* __restrict__ x,
                                     void* __restrict__ vy, const int kx,
                                     const int kx_padded) {
  const int ix = blockDim.x * blockIdx.x + threadIdx.x;
  if (ix >= kx_padded) {
    return;
  }
  const int iy = blockDim.y * blockIdx.y + threadIdx.y;
  const int i_padded = iy * kx_padded + ix;

  block_q8_1* y = (block_q8_1*)vy;

  const int ib = i_padded / QK8_1;   // block index
  const int iqs = i_padded % QK8_1;  // quant index

  const float xi = ix < kx ? __half2float(x[iy * kx + ix]) : 0.0f;
  float amax = fabsf(xi);
  float sum = xi;

#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    amax = fmaxf(amax, __shfl_xor_sync(0xffffffff, amax, mask, 32));
    sum += __shfl_xor_sync(0xffffffff, sum, mask, 32);
  }

  const float d = amax / 127;
  const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

  y[ib].qs[iqs] = q;

  if (iqs > 0) {
    return;
  }

  y[ib].ds.x = __float2half(d);
  y[ib].ds.y = __float2half(sum);
}

static void quantize_row_q8_1_cuda(const half* x, void* vy, const int kx,
                                   const int ky, hipStream_t stream) {
  const int64_t kx_padded = (kx + 512 - 1) / 512 * 512;
  const int block_num_x =
      (kx_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
  const dim3 num_blocks(block_num_x, ky, 1);
  const dim3 block_size(CUDA_DEQUANTIZE_BLOCK_SIZE, 1, 1);
  quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, kx, kx_padded);
}

torch::Tensor ggml_dequantize(torch::Tensor W,  // quant weight
                              int8_t type, int64_t m, int64_t n) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(W));
  auto options =
      torch::TensorOptions().dtype(torch::kFloat16).device(W.device());
  at::Tensor DW = torch::empty({m, n}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  const to_fp16_cuda_t to_fp16_cuda = ggml_get_to_fp16_cuda(type);
  to_fp16_cuda((void*)W.data_ptr(), (half*)DW.data_ptr(), m * n, stream);
  return DW;
}

torch::Tensor ggml_mul_mat_vec_a8(torch::Tensor W,  // quant weight
                                  torch::Tensor X,  // input
                                  int8_t type, int64_t row) {
  int col = X.sizes()[1];
  const int padded = (col + 512 - 1) / 512 * 512;
  const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
  auto options =
      torch::TensorOptions().dtype(torch::kFloat16).device(W.device());
  at::Tensor Y = torch::empty({1, row}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  options = torch::TensorOptions().dtype(torch::kInt32).device(W.device());
  at::Tensor quant_X = torch::empty({1, padded / 32 * 9}, options);
  quantize_row_q8_1_cuda((half*)X.data_ptr(), (void*)quant_X.data_ptr(), col, 1,
                         stream);
  switch (type) {
    case 2:
      mul_mat_vec_q4_0_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 3:
      mul_mat_vec_q4_1_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 6:
      mul_mat_vec_q5_0_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 7:
      mul_mat_vec_q5_1_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 8:
      mul_mat_vec_q8_0_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 10:
      mul_mat_vec_q2_K_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 11:
      mul_mat_vec_q3_K_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 12:
      mul_mat_vec_q4_K_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 13:
      mul_mat_vec_q5_K_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 14:
      mul_mat_vec_q6_K_q8_1_cuda((void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                                 (half*)Y.data_ptr(), col, row, stream);
      break;
    case 16:
      mul_mat_vec_iq2_xxs_q8_1_cuda((void*)W.data_ptr(),
                                    (void*)quant_X.data_ptr(),
                                    (half*)Y.data_ptr(), col, row, stream);
      break;
    case 17:
      mul_mat_vec_iq2_xs_q8_1_cuda((void*)W.data_ptr(),
                                   (void*)quant_X.data_ptr(),
                                   (half*)Y.data_ptr(), col, row, stream);
      break;
    case 18:
      mul_mat_vec_iq3_xxs_q8_1_cuda((void*)W.data_ptr(),
                                    (void*)quant_X.data_ptr(),
                                    (half*)Y.data_ptr(), col, row, stream);
      break;
    case 19:
      mul_mat_vec_iq1_s_q8_1_cuda((void*)W.data_ptr(),
                                  (void*)quant_X.data_ptr(),
                                  (half*)Y.data_ptr(), col, row, stream);
      break;
    case 20:
      mul_mat_vec_iq4_nl_q8_1_cuda((void*)W.data_ptr(),
                                   (void*)quant_X.data_ptr(),
                                   (half*)Y.data_ptr(), col, row, stream);
      break;
    case 21:
      mul_mat_vec_iq3_s_q8_1_cuda((void*)W.data_ptr(),
                                  (void*)quant_X.data_ptr(),
                                  (half*)Y.data_ptr(), col, row, stream);
      break;
    case 22:
      mul_mat_vec_iq2_s_q8_1_cuda((void*)W.data_ptr(),
                                  (void*)quant_X.data_ptr(),
                                  (half*)Y.data_ptr(), col, row, stream);
      break;
    case 23:
      mul_mat_vec_iq4_xs_q8_1_cuda((void*)W.data_ptr(),
                                   (void*)quant_X.data_ptr(),
                                   (half*)Y.data_ptr(), col, row, stream);
      break;
  }
  return Y;
}

torch::Tensor ggml_mul_mat_a8(torch::Tensor W,  // quant weight
                              torch::Tensor X,  // input
                              int8_t type, int64_t row) {
  int col = X.sizes()[1];
  int padded = (col + 512 - 1) / 512 * 512;
  int batch = X.sizes()[0];
  const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
  auto options =
      torch::TensorOptions().dtype(torch::kFloat16).device(W.device());
  at::Tensor Y = torch::empty({batch, row}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  options = torch::TensorOptions().dtype(torch::kInt32).device(W.device());
  at::Tensor quant_X = torch::empty({batch, padded / 32 * 9}, options);
  quantize_row_q8_1_cuda((half*)X.data_ptr(), (void*)quant_X.data_ptr(), col,
                         batch, stream);

  switch (type) {
    case 2:
      ggml_mul_mat_q4_0_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 3:
      ggml_mul_mat_q4_1_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 6:
      ggml_mul_mat_q5_0_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 7:
      ggml_mul_mat_q5_1_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 8:
      ggml_mul_mat_q8_0_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 10:
      ggml_mul_mat_q2_K_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 11:
      ggml_mul_mat_q3_K_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 12:
      ggml_mul_mat_q4_K_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 13:
      ggml_mul_mat_q5_K_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
    case 14:
      ggml_mul_mat_q6_K_q8_1_cuda(
          (void*)W.data_ptr(), (void*)quant_X.data_ptr(), (half*)Y.data_ptr(),
          col, row, batch, padded, row, stream);
      break;
  }
  return Y;
}