#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>
#include "core/math.hpp"
#include "../cuda_compat.h"
#include "dispatch_utils.h"

#include "quantization/fp8/common.cuh"

#include <c10/util/Float8_e4m3fn.h>

#ifndef USE_ROCM
  #include <hip/hip_bf16.h>
  #include <hip/hip_fp16.h>
  #include <hip/hip_fp8.h>
#else
  #include <hip/hip_bf16.h>
  #include <hip/hip_fp16.h>
  #include <hip/hip_fp8.h>

typedef __hip_bfloat162 __hip_bfloat162;
typedef __hip_bfloat16 __hip_bfloat16;
typedef __hip_bfloat16_raw __hip_bfloat16_raw;

typedef __hip_fp8_e4m3 __hip_fp8_e4m3_fnuz;
typedef __hip_fp8x4_e4m3 __hip_fp8x4_e4m3_fnuz;
#endif

#include "core/registration.h"
namespace vllm {

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

// Activation and gating kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          typename fp8_type>
__global__ void act_and_mul_quant_kernel(
    fp8_type* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const float* scale, const int d) {
  const int32_t blocks_per_token = gridDim.y;

  const int32_t elems_per_128bit_load = (128 / 8) / sizeof(scalar_t);

  // We don't expect the hidden dimension to exceed 32 bits so int32 should
  // be safe here.
  const int32_t tgt_elems_per_block = div_ceil(d, blocks_per_token);
  const int32_t elems_per_block =
      round_to_next_multiple_of(tgt_elems_per_block, elems_per_128bit_load);
  const int32_t block_start = blockIdx.y * elems_per_block;
  int32_t block_end = block_start + elems_per_block;
  block_end = block_end > d ? d : block_end;

  // token_idx is 64 bit to prevent 32 bit overflow when the number of tokens
  // is very large
  const int64_t token_idx = blockIdx.x;
  const scalar_t* __restrict__ x_ptr = input + token_idx * 2 * d;
  const scalar_t* __restrict__ y_ptr = input + token_idx * 2 * d + d;
  fp8_type* __restrict__ out_ptr = out + token_idx * d;

  // 128-bit vectorized code
  const int32_t vec_loop_end =
      round_to_previous_multiple_of(elems_per_128bit_load, block_end);
  const int32_t vec_end_idx = vec_loop_end / elems_per_128bit_load;
  const int32_t vec_start_idx = block_start / elems_per_128bit_load;

  const int4* __restrict__ x_128bit_ptr = reinterpret_cast<const int4*>(x_ptr);
  const int4* __restrict__ y_128bit_ptr = reinterpret_cast<const int4*>(y_ptr);
  int2* __restrict__ out_128bit_ptr = reinterpret_cast<int2*>(out_ptr);

  float inverted_scale = 1 / *scale;
#pragma unroll
  for (int32_t vec_idx = vec_start_idx + threadIdx.x; vec_idx < vec_end_idx;
       vec_idx += blockDim.x) {
    const int4 x_128bit = VLLM_LDG(&x_128bit_ptr[vec_idx]);
    const int4 y_128bit = VLLM_LDG(&y_128bit_ptr[vec_idx]);
    using scalar_128bit_vec_t = std::array<scalar_t, elems_per_128bit_load>;
    using scalar_64bit_vec_t = std::array<fp8_type, elems_per_128bit_load>;

    scalar_64bit_vec_t out_vec;
    const auto x_vec = reinterpret_cast<scalar_128bit_vec_t const&>(x_128bit);
    const auto y_vec = reinterpret_cast<scalar_128bit_vec_t const&>(y_128bit);

#pragma unroll
    for (int i = 0; i < elems_per_128bit_load; i++) {
      out_vec[i] = scaled_fp8_conversion<true, fp8_type>(
          ACT_FN(x_vec[i]) * y_vec[i], inverted_scale);
    }

    out_128bit_ptr[vec_idx] = reinterpret_cast<const int2&>(out_vec);
  }

  // Scalar cleanup code
  if (block_end > vec_loop_end) {
    for (int64_t idx = vec_loop_end + threadIdx.x; idx < block_end;
         idx += blockDim.x) {
      const scalar_t x = VLLM_LDG(&x_ptr[idx]);
      const scalar_t y = VLLM_LDG(&y_ptr[idx]);
      out_ptr[idx] =
          scaled_fp8_conversion<true, fp8_type>(ACT_FN(x) * y, inverted_scale);
    }
  }
}

__device__ __forceinline__ float silu(float x) {
  return (__fdividef(x, (1.f + expf(-x))));
}

__device__ __forceinline__ float2 silu2(float2 x) {
  return make_float2(silu(x.x), silu(x.y));
}

#ifndef USE_ROCM
__device__ __forceinline__ float warp_max(float v) {
  static constexpr unsigned FULL_MASK = 0xffffffffu;
  for (int offset = 1; offset < WARP_SIZE; offset *= 2) {
    v = fmaxf(v, __shfl_xor_sync(FULL_MASK, v, offset));
  }
  return v;
}

__device__ __forceinline__ __hip_bfloat16 warp_max(__hip_bfloat16 v) {
  static constexpr unsigned FULL_MASK = 0xffffffffu;
  for (int offset = 1; offset < WARP_SIZE; offset *= 2) {
    v = __hmax(v, __shfl_xor_sync(FULL_MASK, v, offset));
  }
  return v;
}
#endif

template <typename T, typename U>
__device__ __forceinline__ void cp_async4(T* _smem_ptr, const U* _glob_ptr) {
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDA_ARCH__ >= 800
  auto smem_ptr = reinterpret_cast<void*>(_smem_ptr);
  auto glob_ptr = reinterpret_cast<const void*>(_glob_ptr);
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
      "{\n"
      "   cp.async.cg.shared.global [%0], [%1], %2;\n"
      "}\n" ::"r"(smem),
      "l"(glob_ptr), "n"(BYTES));
#else
  _smem_ptr[0] = _glob_ptr[0];
#endif
}

__device__ __forceinline__ void cp_async_fence() {
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDA_ARCH__ >= 800
  asm volatile("cp.async.commit_group;\n" ::);
#else
#endif
}

template <int N>
__device__ __forceinline__ void cp_async_wait() {
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDA_ARCH__ >= 800
  asm volatile("cp.async.wait_group %0;\n" ::"n"(N));
#else
#endif
}

template <>
__device__ __forceinline__ void cp_async_wait<0>() {
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDA_ARCH__ >= 800
  asm volatile("cp.async.wait_all;\n" ::);
#else
#endif
}

__device__ __forceinline__ float clip(float v, float mmin, float mmax) {
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDA_ARCH__ >= 800
  return fminf(mmax, fmaxf(v, mmin));
#else
#endif
}

__device__ __forceinline__ __hip_bfloat16 clip(__hip_bfloat16 v,
                                              __hip_bfloat16 mmin,
                                              __hip_bfloat16 mmax) {
  return __hmin(mmax, __hmax(v, mmin));
}

__device__ __forceinline__ __hip_bfloat162 clip(__hip_bfloat162 v,
                                               __hip_bfloat162 mmin,
                                               __hip_bfloat162 mmax) {
  return __hmin2(mmax, __hmax2(v, mmin));
}

// We use the following values for fp8 min/max:
//  __hip_fp8_e4m3_fnuz = (-448, +448)
//  __nv_fp8_e4m3uz = (-240.0, +240.0)
// It is currently assumed that only
template <class T>
constexpr __hip_bfloat16 get_fp8_max() {
  static_assert(std::is_same_v<T, c10::Float8_e4m3fn> ||
                std::is_same_v<T, c10::Float8_e4m3fnuz>);
  if constexpr (std::is_same_v<T, c10::Float8_e4m3fn>) {
    return __hip_bfloat16(__hip_bfloat16_raw{.x = 17376});
  } else {
    return __hip_bfloat16(__hip_bfloat16_raw{.x = 17264});
  }
}

template <class T>
constexpr __hip_bfloat16 get_fp8_min() {
  static_assert(std::is_same_v<T, c10::Float8_e4m3fn> ||
                std::is_same_v<T, c10::Float8_e4m3fnuz>);
  if constexpr (std::is_same_v<T, c10::Float8_e4m3fn>) {
    return __hip_bfloat16(__hip_bfloat16_raw{.x = 50144});
  } else {
    return __hip_bfloat16(__hip_bfloat16_raw{.x = 50032});
  }
}
#ifndef USE_ROCM
template <typename fp8_type, int32_t NUM_WARPS, typename Idx_t,
          int NUM_PARALLEL_TOKENS, bool USE_UE8M0, int GROUP_SIZE = 128,
          int NUM_STAGES = 3>
__global__ void silu_mul_fp8_quant_deep_gemm_kernel(
    const __hip_bfloat16* __restrict__ _input, fp8_type* __restrict__ _y_q,
    float* __restrict__ _y_s, const int32_t* __restrict__ counts,

    // sizes
    int H, int G,

    // strides (in elements)
    Idx_t stride_i_e, Idx_t stride_i_t, Idx_t stride_i_h, Idx_t stride_yq_e,
    Idx_t stride_yq_t, Idx_t stride_yq_h, Idx_t stride_ys_e, Idx_t stride_ys_t,
    Idx_t stride_ys_g, Idx_t stride_counts_e) {
  static constexpr __hip_bfloat16 fp8_min = get_fp8_min<fp8_type>();
  static constexpr __hip_bfloat16 fp8_max = get_fp8_max<fp8_type>();
  // We assign EPS with its 16-bit unsigned counterpart to allow constexpr.
  static constexpr __hip_bfloat16 EPS = (__hip_bfloat16_raw{.x = 11996});

  // We pack 8 16-bit bfloat16 values into a 128-bit __int128_t.
  static constexpr int32_t BFLOAT16_PER_GROUP = 8;

  // We split the shared memory in half, corresponding to gate and up matrices:
  // [...gate_i, ...up_i]  where 0 <= i < stages.
  static constexpr int32_t S_NUM_128 =
      2u * (GROUP_SIZE / BFLOAT16_PER_GROUP) * NUM_WARPS * NUM_STAGES;
  static constexpr auto THREAD_COUNT = NUM_WARPS * WARP_SIZE;
  static constexpr int HALF_THREAD_COUNT = THREAD_COUNT / 2;
  static constexpr int32_t S_NUM_64 = S_NUM_128 * 2;
  __shared__ __int128_t __align__(16) s_buff_128[S_NUM_128];

  const int32_t tid = threadIdx.x;
  const int32_t warp_id = tid / WARP_SIZE;
  const int32_t lane_id = tid % WARP_SIZE;

  auto s_buff_compute_32 = reinterpret_cast<__hip_bfloat162*>(s_buff_128);

  // block handles one (expert e, group g)
  int32_t pid = blockIdx.x;
  int32_t e = pid / G;
  int32_t g = pid % G;

  const int32_t n_tokens = counts[e * stride_counts_e];

  if (!n_tokens) {
    return;  // Exit ASAP.
  }

  const Idx_t stride_i_t_128 = stride_i_t / 8u;

  int32_t n_tokens_lower, n_tokens_upper;

  // Each block i iterates over tokens of a slice of n_tokens =
  // expert_counts[i], with the size of chunk being
  // (n_tokens / NUM_PARALLEL_TOKENS) + residual, instead of
  // updiv(n_tokens, NUM_PARALLEL_TOKENS) for better scheduling.
  if (n_tokens < NUM_PARALLEL_TOKENS && blockIdx.y < n_tokens) {
    // Specialize this, but can be likely fused.
    if (blockIdx.y >= NUM_PARALLEL_TOKENS) {
      return;
    }
    n_tokens_lower = blockIdx.y;
    n_tokens_upper = blockIdx.y + 1;
  } else {
    auto chunk_size = n_tokens / NUM_PARALLEL_TOKENS;
    auto residual = n_tokens - chunk_size * NUM_PARALLEL_TOKENS;
    auto calc_id = [&](int32_t id) {
      if (id < residual) {
        return min(n_tokens, id * (chunk_size + 1));
      } else {
        return min(n_tokens, id * chunk_size + residual);
      }
    };
    n_tokens_lower = calc_id(blockIdx.y);
    n_tokens_upper = calc_id(blockIdx.y + 1);
  }

  if (n_tokens_lower >= n_tokens_upper) {
    return;
  }

  // We do calculations here, using constexpr wherever possible.
  const Idx_t base_i = e * stride_i_e + NUM_WARPS * g * GROUP_SIZE * stride_i_h;
  const Idx_t base_ys = e * stride_ys_e + NUM_WARPS * g * stride_ys_g;
  const Idx_t base_yq =
      e * stride_yq_e + NUM_WARPS * g * GROUP_SIZE * stride_yq_h;
  Idx_t gate_off_128 = (base_i / static_cast<Idx_t>(8u));
  auto input_128_ptr = reinterpret_cast<const __int128_t*>(_input);
  auto gate_128_ptr = input_128_ptr + gate_off_128 + (tid % HALF_THREAD_COUNT) +
                      stride_i_t_128 * n_tokens_lower;
  auto up_128_ptr = gate_128_ptr + (H * stride_i_h) / 8u;
  auto y_s_ptr =
      _y_s + base_ys + warp_id * stride_ys_g + n_tokens_lower * stride_ys_t;
  auto y_q_ptr = _y_q + base_yq + warp_id * GROUP_SIZE +
                 stride_yq_t * n_tokens_lower + 4 * lane_id;
  int32_t t_load = n_tokens_lower, load_stage_id = 0;
  auto s_buff_gate_load_128 = s_buff_128 + (tid % HALF_THREAD_COUNT);
  auto s_buff_up_load_128 = s_buff_gate_load_128 + S_NUM_128 / 2u;
  int32_t stage_offset{};

  static constexpr int32_t LOAD_STAGE_SIZE = (NUM_WARPS * WARP_SIZE / 2);
  static constexpr int32_t LOAD_STAGE_MOD =
      NUM_STAGES * (NUM_WARPS * WARP_SIZE / 2);

  // Two halves of all threads in a block conduct global loads for gate and up,
  // repsectively.
  auto load_and_advance_y_pred = [&] {
    if (t_load < n_tokens_upper) {
      auto s_gate_stage_128_staged_ptr = s_buff_gate_load_128 + stage_offset;
      auto s_up_stage_128_staged_ptr = s_buff_up_load_128 + stage_offset;

      // It is very important that LOAD_STAGE_SIZE is constexpr to avoid
      // unnecessary ALU ops.
      stage_offset += LOAD_STAGE_SIZE;
      stage_offset %= LOAD_STAGE_MOD;

      if (tid < HALF_THREAD_COUNT) {
        cp_async4(s_gate_stage_128_staged_ptr, gate_128_ptr);
        gate_128_ptr += stride_i_t_128;
      } else {
        cp_async4(s_up_stage_128_staged_ptr, up_128_ptr);
        up_128_ptr += stride_i_t_128;
      }
      ++t_load;
      ++load_stage_id;
    }
    // We fence even if there is nothing to load to simplify pipelining.
    cp_async_fence();
  };

  #pragma unroll
  for (int i = 0; i < NUM_STAGES - 1; i++) {
    load_and_advance_y_pred();
  }

  __int64_t* s_gate_ptr = reinterpret_cast<__int64_t*>(
                              s_buff_compute_32 + warp_id * (GROUP_SIZE / 2)) +
                          lane_id;
  __int64_t* s_up_ptr = s_gate_ptr + S_NUM_64 / 2;

  static constexpr int32_t STAGE_SIZE = (GROUP_SIZE * NUM_WARPS) / 4u;
  static constexpr int32_t STAGE_MOD = STAGE_SIZE * NUM_STAGES;

  int32_t compute_pipeline_offset_64 = 0;

  for (int32_t t = n_tokens_lower; t < n_tokens_upper; ++t) {
    __hip_bfloat162 results_bf162[2];

    cp_async_wait<NUM_STAGES - 2>();
    __syncthreads();

    // We double-buffer pipelined loads so that the next load will
    // concurrently run with compute without overwrites.
    load_and_advance_y_pred();

    auto s_gate_compute_64 = s_gate_ptr + compute_pipeline_offset_64;
    auto s_up_compute_64 = s_up_ptr + compute_pipeline_offset_64;

    // STAGE_SIZE must also be constexpr!
    compute_pipeline_offset_64 += STAGE_SIZE;
    compute_pipeline_offset_64 %= STAGE_MOD;

    // Each thread loads (gate/up) 2X 4X bfloat16 values into registers.
    __int64_t gate64 = *s_gate_compute_64;
    __hip_bfloat162* s_gate_compute_32 =
        reinterpret_cast<__hip_bfloat162*>(&gate64);

    __int64_t up64 = *s_up_compute_64;
    __hip_bfloat162* s_up_compute_32 = reinterpret_cast<__hip_bfloat162*>(&up64);

  #pragma unroll
    for (int i = 0; i < 2; i++) {
      // For silu, we make sure that div is emitted.
      float2 gate = silu2(__bfloat1622float2(s_gate_compute_32[i]));
      results_bf162[i] = __float22bfloat162_rn(gate);
    }

  #pragma unroll
    for (int i = 0; i < 2; i++) {
      results_bf162[i] = __hmul2(results_bf162[i], s_up_compute_32[i]);
    }

    auto _y_max2 =
        __hmax2(__habs2(results_bf162[0]), __habs2(results_bf162[1]));

    __hip_bfloat16 y_max_bf16 = __hmax(EPS, __hmax(_y_max2.x, _y_max2.y));

    // An entire group is assigned to a single warp, so a simple warp reduce
    // is used.
    __hip_bfloat16 y_s = warp_max(y_max_bf16) / fp8_max;

    if constexpr (USE_UE8M0) {
      y_s = hexp2(hceil(hlog2(y_s)));
    }

    auto inv_y = __float2bfloat16_rn(1.f) / y_s;

    auto y_s2 = make_bfloat162(inv_y, inv_y);

  #pragma unroll
    for (int32_t i = 0; i < 2; ++i) {
      results_bf162[i] =
          clip(__hmul2(results_bf162[i], y_s2), __bfloat162bfloat162(fp8_min),
               __bfloat162bfloat162(fp8_max));
    }

    auto fp8x4 = __hip_fp8x4_e4m3_fnuz(results_bf162[0], results_bf162[1]);
    *reinterpret_cast<__hip_fp8x4_e4m3_fnuz*>(y_q_ptr) = fp8x4;
    y_q_ptr += stride_yq_t;

    if (lane_id == 0) {
      *y_s_ptr = y_s;
      y_s_ptr += stride_ys_t;
    }
  }
}
#endif

}  // namespace vllm

// Launch activation, gating, and quantize kernel.
#define LAUNCH_ACTIVATION_GATE_KERNEL(KERNEL)                               \
  int d = input.size(-1) / 2;                                               \
  int64_t num_tokens = input.numel() / input.size(-1);                      \
  dim3 grid(num_tokens, num_tokens > 16 ? num_tokens > 32 ? 1 : 2 : 4);     \
  dim3 block(std::min(d, 512));                                             \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));         \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();             \
  VLLM_DISPATCH_FLOATING_TYPES(                                             \
      input.scalar_type(), "act_and_mul_kernel", [&] {                      \
        VLLM_DISPATCH_FP8_TYPES(                                            \
            out.scalar_type(), "fused_add_rms_norm_kernel_fp8_type", [&] {  \
              vllm::act_and_mul_quant_kernel<scalar_t, KERNEL<scalar_t>,    \
                                             fp8_t>                         \
                  <<<grid, block, 0, stream>>>(out.data_ptr<fp8_t>(),       \
                                               input.data_ptr<scalar_t>(),  \
                                               scale.data_ptr<float>(), d); \
            });                                                             \
      });

void silu_and_mul_quant(torch::Tensor& out,    // [..., d]
                        torch::Tensor& input,  // [..., 2 * d]
                        torch::Tensor& scale) {
  TORCH_CHECK(out.dtype() == torch::kFloat8_e4m3fn ||
              out.dtype() == torch::kFloat8_e4m3fnuz);
  TORCH_CHECK(input.dtype() == torch::kFloat16 ||
              input.dtype() == torch::kBFloat16);
  TORCH_CHECK(input.size(-1) % 2 == 0);
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::silu_kernel);
}

void silu_mul_fp8_quant_deep_gemm_cuda(
    const at::Tensor& input,   // (E, T, 2*H)
    const at::Tensor& counts,  // (E)
    at::Tensor& y_q,           // (E, T, H) [OUT]
    at::Tensor& y_s,           // (E, T, H//group_size) [OUT]
    int64_t group_size, bool use_ue8m0, int64_t num_parallel_tokens) {
#ifndef USE_ROCM
  // This kernel relies heavily on cp.async and fp8 support.
  // This kernel currently only supports H % 128 == 0 and assumes a
  // fixed GROUP_SIZE of 128.
  TORCH_CHECK(input.dtype() == torch::kBFloat16);
  TORCH_CHECK(y_q.dtype() == torch::kFloat8_e4m3fn ||
              y_q.dtype() == torch::kFloat8_e4m3fnuz);
  TORCH_CHECK(y_s.dtype() == torch::kFloat32);
  TORCH_CHECK(input.size(-1) % 256 == 0);

  // Check that num_parallel_tokens is of power of 2 and between 1 and 64.
  TORCH_CHECK(1 <= num_parallel_tokens && num_parallel_tokens <= 64);
  TORCH_CHECK(!(num_parallel_tokens & (num_parallel_tokens - 1)));

  using Idx_t = int64_t;

  Idx_t E = input.size(0);
  Idx_t T = input.size(1);
  Idx_t H = input.size(2) / 2;
  Idx_t stride_i_e = input.stride(0);
  Idx_t stride_i_t = input.stride(1);
  Idx_t stride_i_h = input.stride(2);
  Idx_t stride_yq_e = y_q.stride(0);
  Idx_t stride_yq_t = y_q.stride(1);
  Idx_t stride_yq_h = y_q.stride(2);
  Idx_t stride_ys_e = y_s.stride(0);
  Idx_t stride_ys_t = y_s.stride(1);
  Idx_t stride_ys_g = y_s.stride(2);

  Idx_t stride_counts_e = counts.stride(0);

  static constexpr int GROUP_SIZE = 128;

  #define KERNEL_FN                                                         \
    if (use_ue8m0) {                                                        \
      vllm::silu_mul_fp8_quant_deep_gemm_kernel<fp8_t, NUM_WARPS, Idx_t,    \
                                                NUM_PARALLEL_TOKENS, true>  \
          <<<grid, block, 0, stream>>>(                                     \
              reinterpret_cast<__hip_bfloat16*>(input.data_ptr()),           \
              (fp8_t*)y_q.data_ptr(), y_s.data_ptr<float>(),                \
              reinterpret_cast<int32_t*>(counts.data_ptr<int>()), H, G,     \
              stride_i_e, stride_i_t, stride_i_h, stride_yq_e, stride_yq_t, \
              stride_yq_h, stride_ys_e, stride_ys_t, stride_ys_g,           \
              stride_counts_e);                                             \
    } else {                                                                \
      vllm::silu_mul_fp8_quant_deep_gemm_kernel<fp8_t, NUM_WARPS, Idx_t,    \
                                                NUM_PARALLEL_TOKENS, false> \
          <<<grid, block, 0, stream>>>(                                     \
              reinterpret_cast<__hip_bfloat16*>(input.data_ptr()),           \
              (fp8_t*)y_q.data_ptr(), y_s.data_ptr<float>(),                \
              reinterpret_cast<int32_t*>(counts.data_ptr<int>()), H, G,     \
              stride_i_e, stride_i_t, stride_i_h, stride_yq_e, stride_yq_t, \
              stride_yq_h, stride_ys_e, stride_ys_t, stride_ys_g,           \
              stride_counts_e);                                             \
    }

  #define KERNEL_CALL_H                                       \
    if (H % (4 * GROUP_SIZE) == 0) {                          \
      static constexpr int NUM_WARPS = 4;                     \
      populate_launch_params(NUM_WARPS, NUM_PARALLEL_TOKENS); \
      KERNEL_FN                                               \
    } else {                                                  \
      static constexpr int NUM_WARPS = 1;                     \
      populate_launch_params(NUM_WARPS, NUM_PARALLEL_TOKENS); \
      KERNEL_FN                                               \
    }

  #define KERNEL_CALL_TOP_LEVEL                      \
    if (num_parallel_tokens == 1) {                  \
      static constexpr int NUM_PARALLEL_TOKENS = 1;  \
      KERNEL_CALL_H                                  \
    } else if (num_parallel_tokens == 2) {           \
      static constexpr int NUM_PARALLEL_TOKENS = 2;  \
      KERNEL_CALL_H                                  \
    } else if (num_parallel_tokens == 4) {           \
      static constexpr int NUM_PARALLEL_TOKENS = 4;  \
      KERNEL_CALL_H                                  \
    } else if (num_parallel_tokens == 8) {           \
      static constexpr int NUM_PARALLEL_TOKENS = 8;  \
      KERNEL_CALL_H                                  \
    } else if (num_parallel_tokens == 16) {          \
      static constexpr int NUM_PARALLEL_TOKENS = 16; \
      KERNEL_CALL_H                                  \
    } else if (num_parallel_tokens == 32) {          \
      static constexpr int NUM_PARALLEL_TOKENS = 32; \
      KERNEL_CALL_H                                  \
    } else if (num_parallel_tokens == 64) {          \
      static constexpr int NUM_PARALLEL_TOKENS = 64; \
      KERNEL_CALL_H                                  \
    }

  Idx_t G;
  dim3 block, grid;
  auto populate_launch_params = [&](int num_warps, int _num_parallel_tokens) {
    G = H / Idx_t(group_size * num_warps);
    grid = dim3(E * G, _num_parallel_tokens);
    block = dim3(num_warps * WARP_SIZE);
  };

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  VLLM_DISPATCH_FP8_TYPES(y_q.scalar_type(),
                          "silu_mul_fp8_quant_deep_gemm_kernel",
                          [&] { KERNEL_CALL_TOP_LEVEL });

#endif
}
