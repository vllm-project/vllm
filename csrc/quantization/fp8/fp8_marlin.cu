#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Adapted from https://github.com/IST-DASLab/marlin
 */

#include "../gptq_marlin/marlin.cuh"
#include "../gptq_marlin/marlin_dtypes.cuh"

#include "core/registration.h"

using namespace marlin;

#define STATIC_ASSERT_SCALAR_TYPE_VALID(scalar_t)               \
  static_assert(std::is_same<scalar_t, half>::value ||          \
                    std::is_same<scalar_t, hip_bfloat16>::value, \
                "only float16 and bfloat16 is supported");

template <typename T>
inline std::string str(T x) {
  return std::to_string(x);
}

namespace fp8_marlin {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

template <typename scalar_t,          // compute dtype, half or nv_float16
          const int num_bits,         // number of bits used for weights
          const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1  // number of consecutive 16x16 blocks
                                       // with a separate quantization scale
          >
__global__ void Marlin(
    const int4* __restrict__ A,  // fp16 input matrix of shape mxk
    const int4* __restrict__ B,  // 4bit quantized weight matrix of shape kxn
    int4* __restrict__ C,        // fp16 output buffer of shape mxn
    const int4* __restrict__ scales_ptr,  // fp16 quantization scales of shape
                                          // (k/groupsize)xn
    int num_groups,  // number of scale groups per output channel
    int prob_m,      // batch dimension m
    int prob_n,      // output dimension n
    int prob_k,      // reduction dimension k
    int* locks       // extra global storage for barrier synchronization
) {}

}  // namespace fp8_marlin

torch::Tensor fp8_marlin_gemm(torch::Tensor& a, torch::Tensor& b_q_weight,
                              torch::Tensor& b_scales, torch::Tensor& workspace,
                              int64_t num_bits, int64_t size_m, int64_t size_n,
                              int64_t size_k) {
  TORCH_CHECK_NOT_IMPLEMENTED(false,
                              "marlin_gemm(..) requires CUDA_ARCH >= 8.0");
  return torch::empty({1, 1});
}

#else

// m16n8k16 tensor core mma instruction with fp16 inputs and fp32
// output/accumulation.
template <typename scalar_t>
__device__ inline void mma(const typename ScalarType<scalar_t>::FragA& a_frag,
                           const typename ScalarType<scalar_t>::FragB& frag_b,
                           typename ScalarType<scalar_t>::FragC& frag_c) {
  const uint32_t* a = reinterpret_cast<const uint32_t*>(&a_frag);
  const uint32_t* b = reinterpret_cast<const uint32_t*>(&frag_b);
  float* c = reinterpret_cast<float*>(&frag_c);
  if constexpr (std::is_same<scalar_t, half>::value) {
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
        : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]), "r"(b[1]),
          "f"(c[0]), "f"(c[1]), "f"(c[2]), "f"(c[3]));
  } else if constexpr (std::is_same<scalar_t, hip_bfloat16>::value) {
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
        : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]), "r"(b[1]),
          "f"(c[0]), "f"(c[1]), "f"(c[2]), "f"(c[3]));
  } else {
    STATIC_ASSERT_SCALAR_TYPE_VALID(scalar_t);
  }
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared
// memory, directly in tensor core layout.
template <typename scalar_t>
__device__ inline void ldsm4(typename ScalarType<scalar_t>::FragA& frag_a,
                             const void* smem_ptr) {
  uint32_t* a = reinterpret_cast<uint32_t*>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
               : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3])
               : "r"(smem));
}

// Fast FP8ToFp16/FP8ToBf16: Efficiently dequantize 8bit fp8_e4m3 values to fp16
// bf16 Reference:
// - FP16:
// https://github.com/NVIDIA/FasterTransformer/blob/release/v5.3_tag/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h#L53-L85
// - BF16:
// https://github.com/NVIDIA/FasterTransformer/blob/release/v5.3_tag/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h#L125-L175
template <typename scalar_t>
__device__ inline typename ScalarType<scalar_t>::FragB dequant_8bit(int q) {
  STATIC_ASSERT_SCALAR_TYPE_VALID(scalar_t);
}

template <>
__device__ inline typename ScalarType<half>::FragB dequant_8bit<half>(int q) {
  // Constants for FP8 (E4M3) and FP16 formats
  constexpr int FP8_EXPONENT = 4, FP8_MANTISSA = 3, FP16_EXPONENT = 5;
  constexpr int RIGHT_SHIFT = FP16_EXPONENT - FP8_EXPONENT;

  // Calculate MASK for extracting mantissa and exponent
  constexpr int MASK1 = 0x80000000;
  constexpr int MASK2 = MASK1 >> (FP8_EXPONENT + FP8_MANTISSA);
  constexpr int MASK3 = MASK2 & 0x7fffffff;
  constexpr int MASK = MASK3 | (MASK3 >> 16);
  // Final MASK value: 0x7F007F00

  // Extract and shift FP8 values to FP16 format
  int Out1 = (q & 0x80008000) | ((q & MASK) >> RIGHT_SHIFT);
  int Out2 = ((q << 8) & 0x80008000) | (((q << 8) & MASK) >> RIGHT_SHIFT);

  // Construct and apply exponent bias
  constexpr int BIAS_OFFSET =
      (1 << (FP16_EXPONENT - 1)) - (1 << (FP8_EXPONENT - 1));
  const half2 bias_reg = __float2half2_rn(float(1 << BIAS_OFFSET));

  // Convert to half2 and apply bias
  typename ScalarType<half>::FragB frag_b;
  // Note: reverse indexing is intentional because weights are permuted
  frag_b[1] = __hmul2(*reinterpret_cast<const half2*>(&Out1), bias_reg);
  frag_b[0] = __hmul2(*reinterpret_cast<const half2*>(&Out2), bias_reg);
  return frag_b;
}

template <>
__device__ inline typename ScalarType<hip_bfloat16>::FragB
dequant_8bit<hip_bfloat16>(int q) {
  // Constants for FP8 (E4M3) and BF16 formats
  constexpr int FP8_EXPONENT = 4, FP8_MANTISSA = 3, BF16_EXPONENT = 8;
  constexpr int RIGHT_SHIFT = BF16_EXPONENT - FP8_EXPONENT;

  // Calculate MASK for extracting mantissa and exponent
  constexpr int MASK1 = 0x80000000;
  constexpr int MASK2 = MASK1 >> (FP8_EXPONENT + FP8_MANTISSA);
  constexpr int MASK3 = MASK2 & 0x7fffffff;
  constexpr int MASK = MASK3 | (MASK3 >> 16);
  // Final MASK value: 0x7F007F00

  // Extract and shift FP8 values to BF16 format
  int Out1 = (q & 0x80008000) | ((q & MASK) >> RIGHT_SHIFT);
  int Out2 = ((q << 8) & 0x80008000) | (((q << 8) & MASK) >> RIGHT_SHIFT);

  // Construct and apply exponent bias
  constexpr int BIAS_OFFSET =
      (1 << (BF16_EXPONENT - 1)) - (1 << (FP8_EXPONENT - 1));
  // Add 127 (float exponent bias) to BIAS_OFFSET and shift to float exponent
  // position
  constexpr uint32_t BIAS = (BIAS_OFFSET + 127) << 23;
  const nv_bfloat162 bias_reg =
      __float2bfloat162_rn(*reinterpret_cast<const float*>(&BIAS));

  // Convert to bfloat162 and apply bias
  typename ScalarType<hip_bfloat16>::FragB frag_b;
  // Note: reverse indexing is intentional because weights are permuted
  frag_b[1] = __hmul2(*reinterpret_cast<const nv_bfloat162*>(&Out1), bias_reg);
  frag_b[0] = __hmul2(*reinterpret_cast<const nv_bfloat162*>(&Out2), bias_reg);
  return frag_b;
}

// Multiply dequantized values by the corresponding quantization scale; used
// only for grouped quantization.
template <typename scalar_t>
__device__ inline void scale(typename ScalarType<scalar_t>::FragB& frag_b,
                             typename ScalarType<scalar_t>::FragS& frag_s,
                             int i) {
  using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;
  scalar_t2 s =
      ScalarType<scalar_t>::num2num2(reinterpret_cast<scalar_t*>(&frag_s)[i]);
  frag_b[0] = __hmul2(frag_b[0], s);
  frag_b[1] = __hmul2(frag_b[1], s);
}

// Given 2 floats multiply by 2 scales (halves)
template <typename scalar_t>
__device__ inline void scale_float(float* c,
                                   typename ScalarType<scalar_t>::FragS& s) {
  scalar_t* s_ptr = reinterpret_cast<scalar_t*>(&s);
  c[0] = __fmul_rn(c[0], ScalarType<scalar_t>::num2float(s_ptr[0]));
  c[1] = __fmul_rn(c[1], ScalarType<scalar_t>::num2float(s_ptr[1]));
}

// Wait until barrier reaches `count`, then lock for current threadblock.
__device__ inline void barrier_acquire(int* lock, int count) {
  if (threadIdx.x == 0) {
    int state = -1;
    do
      // Guarantee that subsequent writes by this threadblock will be visible
      // globally.
      asm volatile("ld.global.acquire.gpu.b32 %0, [%1];\n"
                   : "=r"(state)
                   : "l"(lock));
    while (state != count);
  }
  __syncthreads();
}

// Release barrier and increment visitation count.
__device__ inline void barrier_release(int* lock, bool reset = false) {
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    // Make sure that all writes since acquiring this barrier are visible
    // globally, while releasing the barrier.
    asm volatile("fence.acq_rel.gpu;\n");
    asm volatile("red.relaxed.gpu.global.add.s32 [%0], %1;\n"
                 :
                 : "l"(lock), "r"(val));
  }
}

template <typename scalar_t,          // compute dtype, half or nv_float16
          const int num_bits,         // number of bits used for weights
          const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1  // number of consecutive 16x16 blocks
                                       // with a separate quantization scale
          >
__global__ void Marlin(
    const int4* __restrict__ A,  // fp16 input matrix of shape mxk
    const int4* __restrict__ B,  // 4bit quantized weight matrix of shape kxn
    int4* __restrict__ C,        // fp16 output buffer of shape mxn
    const int4* __restrict__ scales_ptr,  // fp16 quantization scales of shape
                                          // (k/groupsize)xn
    int num_groups,  // number of scale groups per output channel
    int prob_m,      // batch dimension m
    int prob_n,      // output dimension n
    int prob_k,      // reduction dimension k
    int* locks       // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the
  // same size, which might involve multiple column "slices" (of width 16 *
  // `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM
  // example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it
  // ensures good utilization of all SMs for many kinds of shape and GPU
  // configurations, while requiring as few slow global cross-threadblock
  // reductions as possible.
  using Dtype = ScalarType<scalar_t>;
  using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;
  using FragA = typename ScalarType<scalar_t>::FragA;
  using FragB = typename ScalarType<scalar_t>::FragB;
  using FragC = typename ScalarType<scalar_t>::FragC;
  using FragS = typename ScalarType<scalar_t>::FragS;

  constexpr int pack_factor = 32 / num_bits;

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a
  // better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = div_ceil(k_tiles * n_tiles * parallel, gridDim.x);

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters;  // number of threadblock tiles in the current slice
  int slice_count =
      0;          // total number of active threadblocks in the current slice
  int slice_idx;  // index of threadblock in current slice; numbered bottom to
                  // top

  // We can easily implement parallel problem execution by just remapping
  // indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for
  // synchronization.
  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel) slice_iters = 0;
    if (slice_iters == 0) return;
    if (slice_row + slice_iters > k_tiles) slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * div_ceil(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = div_ceil(k_tiles - col_off, iters);
      if (col_off > 0) slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0) slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  // A sizes/strides

  // stride of the A matrix in global memory
  int a_gl_stride = prob_k / 8;
  // stride of an A matrix tile in shared memory
  constexpr int a_sh_stride = 16 * thread_k_blocks / 8;
  // delta between subsequent A tiles in global memory
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 8;
  // between subsequent accesses within a tile
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o);
  // between shared memory writes
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o);
  // between shared memory tile reads
  constexpr int a_sh_rd_delta_o = 2 * ((threads / 32) / (thread_n_blocks / 4));
  // within a shared memory tile
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16;
  // overall size of a tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks);
  // number of shared write iterations for a tile
  constexpr int a_sh_wr_iters = div_ceil(a_sh_stage, a_sh_wr_delta);

  // B sizes/strides
  int b_gl_stride = 16 * prob_n / (pack_factor * 4);
  constexpr int b_sh_stride = ((thread_n_blocks * 16) * 16 / pack_factor) / 4;
  constexpr int b_thread_vecs = num_bits == 4 ? 1 : 2;
  constexpr int b_sh_stride_threads = b_sh_stride / b_thread_vecs;

  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride_threads);
  constexpr int b_sh_wr_delta = threads * b_thread_vecs;
  constexpr int b_sh_rd_delta = threads * b_thread_vecs;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  // Scale sizes/strides without act_order
  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;

  // Scale size/strides with act_order
  constexpr int tb_k = 16 * thread_k_blocks;
  constexpr int g_idx_stage = 0;
  // constexpr int act_s_row_stride      = 1;
  // int           act_s_col_stride      = act_s_row_stride * num_groups;
  int act_s_col_stride = 1;
  int act_s_col_warp_stride = act_s_col_stride * 8;
  int tb_n_warps = thread_n_blocks / 4;
  int act_s_col_tb_stride = act_s_col_warp_stride * tb_n_warps;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  int a_sh_rd =
      a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride_threads) +
                (threadIdx.x % b_sh_stride_threads) * b_thread_vecs;
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x * b_thread_vecs;
  int b_sh_rd = threadIdx.x * b_thread_vecs;

  // For act_order
  int slice_k_start = tb_k * slice_row;
  int slice_k_start_shared_fetch = slice_k_start;
  int slice_n_offset = act_s_col_tb_stride * slice_col;

  // No act_order
  int s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
  int s_sh_wr = threadIdx.x;
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  // We scale a `half2` tile in row-major layout for column-wise quantization.
  int s_sh_rd =
      8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) % 4;

  // Precompute which thread should not read memory in which iterations; this is
  // needed if there are more threads than required for a certain tilesize or
  // when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;

  // To ensure that writing and reading A tiles to/from shared memory, the
  // latter in fragment format, is fully bank conflict free, we need to use a
  // rather fancy XOR-based layout. The key here is that neither reads nor
  // writes of the 16-byte `int4` blocks of 8 consecutive threads involve the
  // same shared memory banks. Further, it seems (based on NSight-Compute) that
  // each warp must also write a consecutive memory segment?
  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main
  // loop unrolls, all shared memory accesses are static, we simply precompute
  // both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at
  // runtime; we break dependencies between subsequent accesses with a tile by
  // maintining multiple pointers (we have enough registers), a tiny
  // optimization.
  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_g_idx = sh_b + (stages * b_sh_stage);
  int4* sh_s = sh_g_idx + (stages * g_idx_stage);

  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2][b_thread_vecs];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];

  // Zero accumulators.
  auto zero_accums = [&]() {
  #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  int sh_first_group_id = -1;
  int sh_num_groups = -1;
  constexpr int sh_max_num_groups = 32;

  auto fetch_scales_to_shared = [&](bool is_async, int first_group_id,
                                    int last_group_id) {
    sh_first_group_id = first_group_id;
    sh_num_groups = last_group_id - first_group_id + 1;

    if (sh_num_groups < sh_max_num_groups) {
      sh_num_groups = sh_max_num_groups;
    }

    if (sh_first_group_id + sh_num_groups > num_groups) {
      sh_num_groups = num_groups - sh_first_group_id;
    }

    int row_offset = first_group_id * s_gl_stride;

    if (is_async) {
      for (int i = 0; i < sh_num_groups; i++) {
        if (threadIdx.x < s_sh_stride) {
          cp_async4_pred(&sh_s[(i * s_sh_stride) + threadIdx.x],
                         &scales_ptr[row_offset + (i * s_gl_stride) +
                                     slice_n_offset + threadIdx.x]);
        }
      }
    } else {
      for (int i = 0; i < sh_num_groups; i++) {
        if (threadIdx.x < s_sh_stride) {
          sh_s[(i * s_sh_stride) + threadIdx.x] =
              scales_ptr[row_offset + (i * s_gl_stride) + slice_n_offset +
                         threadIdx.x];
        }
      }
    }
  };
  // Asynchronously fetch the next A, B and s tile from global to the next
  // shared memory pipeline location.
  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
        for (int j = 0; j < b_thread_vecs; j++) {
          cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr + j], B_ptr[i] + j);
        }

        B_ptr[i] += b_gl_rd_delta_o;
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that
    // waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe
  // into the current register buffer.
  auto fetch_to_registers = [&](int k, int pipe) {
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4<scalar_t>(frag_a[k % 2][i],
                      &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4* sh_b_stage = sh_b + b_sh_stage * pipe;

  #pragma unroll
    for (int i = 0; i < b_thread_vecs; i++) {
      frag_b_quant[k % 2][i] = *reinterpret_cast<I4*>(
          &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd + i]);
    }
  };

  bool is_same_group[stages];
  int same_group_id[stages];

  auto init_same_group = [&](int pipe) {
    is_same_group[pipe] = false;
    same_group_id[pipe] = 0;
    return;
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&](int k) {
  // We have the m dimension as the inner loop in order to encourage overlapping
  // dequantization and matmul operations.
  #pragma unroll
    for (int j = 0; j < 4; j++) {
      FragB frag_b0;
      FragB frag_b1;

      int* frag_b_quant_ptr = reinterpret_cast<int*>(frag_b_quant[k % 2]);
      int b_quant_0 = frag_b_quant_ptr[j * 2 + 0];
      int b_quant_1 = frag_b_quant_ptr[j * 2 + 1];

      frag_b0 = dequant_8bit<scalar_t>(b_quant_0);
      frag_b1 = dequant_8bit<scalar_t>(b_quant_1);

  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma<scalar_t>(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma<scalar_t>(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the
  // number of warps while keeping the n dimension of a tile reasonable, we have
  // multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride_threads / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride_threads;
      constexpr int red_sh_stride = b_sh_stride_threads * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride_threads;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride_threads) +
                      (threadIdx.x % b_sh_stride_threads);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any
      // unnecessary read or write iterations, e.g., for two warps we write only
      // once by warp 1 and read only once by warp 0.

  #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
  #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
  #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd =
                    reinterpret_cast<float*>(&sh[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh[red_sh_wr]);
  #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] =
                  reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
  #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd =
                reinterpret_cast<float*>(&sh[red_sh_delta * i + red_sh_rd]);
  #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we
  // finally have to globally reduce over the results. As the striped
  // partitioning minimizes the number of such reductions and our outputs are
  // usually rather small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&](bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to
    // maximize L2 cache utilization in this step. To do this, we write out
    // results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) +
                    4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
  // Interestingly, doing direct global accesses here really seems to mess up
  // the compiler and lead to slowdowns, hence we also use async-copies even
  // though these fetches are not actually asynchronous.
  #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
              &sh[c_sh_wr + c_sh_wr_delta * i],
              &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                 c_gl_wr_delta_i * (i % 2)],
              i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
  #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float*>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] +=
                  Dtype::num2float(reinterpret_cast<scalar_t*>(&c_red)[j]);
            }
          }
          if (!last) {
            int4 c;
  #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<scalar_t*>(&c)[j] =
                  Dtype::float2num(reinterpret_cast<float*>(
                      &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]);
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually
  // reshuffle matrix fragments in this step, the reduction above is performed
  // in fragment layout.
  auto write_result = [&]() {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta =
        c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr =
        (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final
    // global write patterns
    auto write = [&](int idx, float c0, float c1, FragS& s) {
      scalar_t2 res =
          Dtype::nums2num2(Dtype::float2num(c0), Dtype::float2num(c1));

      ((scalar_t2*)sh)[idx] = res;
    };

    if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
  #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0],
                frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2],
                frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0],
                frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2],
                frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

  #pragma unroll
    for (int i = 0;
         i < div_ceil(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&]() {

  #pragma unroll
    for (int i = 0; i < stages - 1; i++) {
      fetch_to_shared(i, i, i < slice_iters);
    }

    zero_accums();
    wait_for_stage();
    init_same_group(0);
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
    slice_k_start_shared_fetch += tb_k * (stages - 1);
  };
  if (slice_iters) {
    start_pipes();
  }

  // Main loop.
  while (slice_iters) {
    // We unroll over both the global fetch and the register load pipeline to
    // ensure all shared memory accesses are static. Note that both pipelines
    // have even length meaning that the next iteration will always start at
    // index 0.

  #pragma unroll
    for (int pipe = 0; pipe < stages;) {
  #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe,
                          slice_iters >= stages);
          pipe++;
          wait_for_stage();
          init_same_group(pipe % stages);
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0) {
        break;
      }
    }

    a_gl_rd += a_gl_rd_delta_o * stages;
    slice_k_start += tb_k * stages;
    slice_k_start_shared_fetch += tb_k * stages;

    // Process results and, if necessary, proceed to the next column slice.
    // While this pattern may not be the most readable, other ways of writing
    // the loop seemed to noticeably worse performance after compilation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before
      // write-out
      if (s_sh_wr_pred) {
        cp_async4(&sh_s[s_sh_wr], &scales_ptr[s_gl_rd]);
      }
      cp_async_fence();

      thread_block_reduce();

      cp_async_wait<0>();
      __syncthreads();
      if (threadIdx.x / 32 < thread_n_blocks / 4) {
        reinterpret_cast<int4*>(&frag_s)[0] = sh_s[s_sh_rd + 0];
        reinterpret_cast<int4*>(&frag_s)[1] = sh_s[s_sh_rd + 4];
      }

      // For 8-bit channelwise, we apply the scale before the global reduction
      // that converts the fp32 results to fp16 (so that we avoid possible
      // overflow in fp16)
      if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
        for (int i = 0; i < thread_m_blocks; i++) {
  #pragma unroll
          for (int j = 0; j < 4; j++) {
            scale_float<scalar_t>(reinterpret_cast<float*>(&frag_c[i][j][0][0]),
                                  frag_s[j / 2][2 * (j % 2) + 0]);
            scale_float<scalar_t>(reinterpret_cast<float*>(&frag_c[i][j][0][2]),
                                  frag_s[j / 2][2 * (j % 2) + 0]);

            scale_float<scalar_t>(reinterpret_cast<float*>(&frag_c[i][j][1][0]),
                                  frag_s[j / 2][2 * (j % 2) + 1]);
            scale_float<scalar_t>(reinterpret_cast<float*>(&frag_c[i][j][1][2]),
                                  frag_s[j / 2][2 * (j % 2) + 1]);
          }
        }
      }

      if (slice_count > 1) {  // only globally reduce if there is more than one
                              // block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last)  // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
  #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
  #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++) B_ptr[i] -= b_gl_stride;
        }

        // Update slice k/n for scales loading
        s_gl_rd = s_sh_stride * slice_col + threadIdx.x;

        start_pipes();
      }
    }
  }
}

  #define __CALL_IF(NUM_BITS, THREAD_M_BLOCKS, THREAD_N_BLOCKS,                \
                    THREAD_K_BLOCKS, GROUP_BLOCKS, NUM_THREADS)                \
    else if (num_bits == NUM_BITS && thread_m_blocks == THREAD_M_BLOCKS &&     \
             thread_n_blocks == THREAD_N_BLOCKS &&                             \
             thread_k_blocks == THREAD_K_BLOCKS &&                             \
             group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {     \
      hipFuncSetAttribute(reinterpret_cast<const void*>(                                                    \
          Marlin<scalar_t), NUM_BITS, NUM_THREADS, THREAD_M_BLOCKS,             \
                 THREAD_N_BLOCKS, THREAD_K_BLOCKS, pipe_stages, GROUP_BLOCKS>, \
          hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);        \
      Marlin<scalar_t, NUM_BITS, NUM_THREADS, THREAD_M_BLOCKS,                 \
             THREAD_N_BLOCKS, THREAD_K_BLOCKS, pipe_stages, GROUP_BLOCKS>      \
          <<<blocks, NUM_THREADS, max_shared_mem, stream>>>(                   \
              A_ptr, B_ptr, C_ptr, s_ptr, num_groups, prob_m, prob_n, prob_k,  \
              locks);                                                          \
    }

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

typedef struct {
  int max_m_blocks;
  thread_config_t tb_cfg;
} exec_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},
    {64, 128, 128},
    {128, 64, 128},
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},
    {64, 128, 128},
    {128, 64, 128},

};

int get_scales_cache_size(thread_config_t const& th_config, int prob_m,
                          int prob_n, int prob_k, int num_bits,
                          int group_size) {
  int tb_n = th_config.thread_n;

  // Get max scale groups per thread-block
  // Fixed for channelwise
  int tb_groups = 1;
  int tb_scales = tb_groups * tb_n * 2;

  return tb_scales * pipe_stages;
}

bool is_valid_cache_size(thread_config_t const& th_config, int max_m_blocks,
                         int prob_m, int prob_n, int prob_k, int num_bits,
                         int scales_cache_size, int max_shared_mem) {
  int pack_factor = 32 / num_bits;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;

  int b_size = (tb_k * tb_n / pack_factor) * 4;

  // Get A size
  int m_blocks = div_ceil(prob_m, 16);
  int tb_max_m = 16;

  while (true) {
    if (m_blocks >= max_m_blocks) {
      tb_max_m *= max_m_blocks;
      break;
    }

    max_m_blocks--;
    if (max_m_blocks == 0) {
      TORCH_CHECK(false, "Unexpected m_blocks = ", m_blocks);
    }
  }

  int a_size = (tb_max_m * tb_k) * 2;

  float pipe_size = (a_size + b_size) * pipe_stages;

  TORCH_CHECK(max_shared_mem / 2 > scales_cache_size);  // Sanity

  return pipe_size < 0.95f * (max_shared_mem - scales_cache_size);
}

bool is_valid_config(thread_config_t const& th_config, int max_m_blocks,
                     int prob_m, int prob_n, int prob_k, int num_bits,
                     int group_size, int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  //  Determine cache for scales
  int scales_cache_size = get_scales_cache_size(th_config, prob_m, prob_n,
                                                prob_k, num_bits, group_size);

  // Check that pipeline fits into cache
  if (!is_valid_cache_size(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                           num_bits, scales_cache_size, max_shared_mem)) {
    return false;
  }

  return true;
}

exec_config_t determine_thread_config(int prob_m, int prob_n, int prob_k,
                                      int num_bits, int group_size,
                                      int max_shared_mem) {
  int max_m_blocks = 4;
  while (max_m_blocks > 0) {
    if (prob_m <= 16) {
      for (auto th_config : small_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    } else {
      for (auto th_config : large_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    }

    max_m_blocks--;  // Process less M blocks per invocation to reduce cache
                     // usage
  }

  return exec_config_t{0, {-1, -1, -1}};
}

  #define CALL_IF(NUM_BITS, N_BLOCKS, K_BLOCKS, NUM_THREADS)    \
    __CALL_IF(NUM_BITS, 1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
    __CALL_IF(NUM_BITS, 2, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
    __CALL_IF(NUM_BITS, 3, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
    __CALL_IF(NUM_BITS, 4, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS)

template <typename scalar_t>
void marlin_mm_f16i4(const void* A, const void* B, void* C, void* s, int prob_m,
                     int prob_n, int prob_k, void* workspace, int num_bits,
                     int num_groups, int group_size, int dev,
                     hipStream_t stream, int thread_k, int thread_n, int sms,
                     int max_par) {
  TORCH_CHECK(num_bits == 8, "num_bits must be 8. Got = ", num_bits);
  TORCH_CHECK(prob_m > 0 && prob_n > 0 && prob_k > 0, "Invalid MNK = [", prob_m,
              ", ", prob_n, ", ", prob_k, "]");

  int tot_m = prob_m;
  int tot_m_blocks = div_ceil(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1) {
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  }

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  // Set thread config
  exec_config_t exec_cfg;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    exec_cfg =
        exec_config_t{4, thread_config_t{thread_k, thread_n, default_threads}};
  } else {
    // Auto config
    exec_cfg = determine_thread_config(prob_m, prob_n, prob_k, num_bits,
                                       group_size, max_shared_mem);
  }

  TORCH_CHECK(
      exec_cfg.max_m_blocks > 0 &&
          is_valid_config(exec_cfg.tb_cfg, exec_cfg.max_m_blocks, prob_m,
                          prob_n, prob_k, num_bits, group_size, max_shared_mem),
      "Invalid thread config: max_m_blocks = ", exec_cfg.max_m_blocks,
      ", thread_k = ", exec_cfg.tb_cfg.thread_k,
      ", thread_n = ", exec_cfg.tb_cfg.thread_n,
      ", num_threads = ", exec_cfg.tb_cfg.num_threads, " for MKN = [", prob_m,
      ", ", prob_k, ", ", prob_n, "] and num_bits = ", num_bits,
      ", group_size = ", group_size, ", max_shared_mem = ", max_shared_mem);

  int num_threads = exec_cfg.tb_cfg.num_threads;
  thread_k = exec_cfg.tb_cfg.thread_k;
  thread_n = exec_cfg.tb_cfg.thread_n;

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;

  int blocks = sms;

  TORCH_CHECK(prob_n % thread_n == 0, "prob_n = ", prob_n,
              " is not divisible by thread_n = ", thread_n);
  TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
              " is not divisible by thread_k = ", thread_k);

  int group_blocks = -1;

  const int4* A_ptr = (const int4*)A;
  const int4* B_ptr = (const int4*)B;
  int4* C_ptr = (int4*)C;
  const int4* s_ptr = (const int4*)s;

  int* locks = (int*)workspace;

  // Main loop
  for (int i = 0; i < tot_m_blocks; i += exec_cfg.max_m_blocks) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > exec_cfg.max_m_blocks) {
      // Note that parallel > 1 currently only works for inputs without any
      // padding
      par = (16 * thread_m_blocks - pad) / (16 * exec_cfg.max_m_blocks);
      if (par > max_par) par = max_par;
      prob_m = (16 * exec_cfg.max_m_blocks) * par;
      i += exec_cfg.max_m_blocks * (par - 1);
      thread_m_blocks = exec_cfg.max_m_blocks;
    }

    // Define kernel configurations
    if (false) {
    }
    CALL_IF(8, 32, 2, 256)
    CALL_IF(8, 16, 4, 256)
    CALL_IF(8, 8, 8, 256)
    CALL_IF(8, 8, 4, 128)
    CALL_IF(8, 4, 8, 128)
    else {
      TORCH_CHECK(false, "Unsupported shapes: MNK = [" + str(prob_m) + ", " +
                             str(prob_n) + ", " + str(prob_k) + "]" +
                             ", num_groups = " + str(num_groups) +
                             ", group_size = " + str(group_size) +
                             ", thread_m_blocks = " + str(thread_m_blocks) +
                             ", thread_n_blocks = " + str(thread_n_blocks) +
                             ", thread_k_blocks = " + str(thread_k_blocks));
    }

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }
}

}  // namespace fp8_marlin

torch::Tensor fp8_marlin_gemm(torch::Tensor& a, torch::Tensor& b_q_weight,
                              torch::Tensor& b_scales, torch::Tensor& workspace,
                              int64_t num_bits, int64_t size_m, int64_t size_n,
                              int64_t size_k) {
  // Verify num_bits
  TORCH_CHECK(num_bits == 8, "num_bits must be 8. Got = ", num_bits);
  int pack_factor = 32 / num_bits;

  // Verify A
  TORCH_CHECK(a.size(0) == size_m, "Shape mismatch: a.size(0) = ", a.size(0),
              ", size_m = ", size_m);
  TORCH_CHECK(a.size(1) == size_k, "Shape mismatch: a.size(1) = ", a.size(1),
              ", size_k = ", size_k);

  // Verify B
  TORCH_CHECK(size_k % marlin::tile_size == 0, "size_k = ", size_k,
              " is not divisible by tile_size = ", marlin::tile_size);
  TORCH_CHECK((size_k / marlin::tile_size) == b_q_weight.size(0),
              "Shape mismatch: b_q_weight.size(0) = ", b_q_weight.size(0),
              ", size_k = ", size_k, ", tile_size = ", marlin::tile_size);
  TORCH_CHECK(b_q_weight.size(1) % marlin::tile_size == 0,
              "b_q_weight.size(1) = ", b_q_weight.size(1),
              " is not divisible by tile_size = ", marlin::tile_size);
  int actual_size_n = (b_q_weight.size(1) / marlin::tile_size) * pack_factor;
  TORCH_CHECK(size_n == actual_size_n, "size_n = ", size_n,
              ", actual_size_n = ", actual_size_n);

  // Verify device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  // Alloc buffers
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c = torch::empty({size_m, size_n}, options);

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel (can usually be left as auto -1)
  int sms = -1;

  // Detect groupsize and act_order
  int num_groups = -1;
  int group_size = -1;

  int b_rank = b_scales.sizes().size();
  TORCH_CHECK(b_rank == 2, "b_scales rank = ", b_rank, " is not 2");
  TORCH_CHECK(b_scales.size(1) == size_n, "b_scales dim 1 = ", b_scales.size(1),
              " is not size_n = ", size_n);
  // Channelwise only for FP8
  TORCH_CHECK(b_scales.size(0) == 1)
  num_groups = b_scales.size(0);

  // Verify workspace size
  TORCH_CHECK(size_n % marlin::min_thread_n == 0, "size_n = ", size_n,
              ", is not divisible by min_thread_n = ", marlin::min_thread_n);
  int min_workspace_size = (size_n / marlin::min_thread_n) * marlin::max_par;
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = ", workspace.numel(),
              " is below min_workspace_size = ", min_workspace_size);

  int dev = a.get_device();
  if (a.scalar_type() == at::ScalarType::Half) {
    fp8_marlin::marlin_mm_f16i4<half>(
        a.data_ptr<at::Half>(), b_q_weight.data_ptr(), c.data_ptr<at::Half>(),
        b_scales.data_ptr<at::Half>(), size_m, size_n, size_k,
        workspace.data_ptr(), num_bits, num_groups, group_size, dev,
        at::cuda::getCurrentCUDAStream(dev), thread_k, thread_n, sms,
        marlin::max_par);
  } else if (a.scalar_type() == at::ScalarType::BFloat16) {
    fp8_marlin::marlin_mm_f16i4<hip_bfloat16>(
        a.data_ptr<at::BFloat16>(), b_q_weight.data_ptr(),
        c.data_ptr<at::BFloat16>(), b_scales.data_ptr<at::BFloat16>(), size_m,
        size_n, size_k, workspace.data_ptr(), num_bits, num_groups, group_size,
        dev, at::cuda::getCurrentCUDAStream(dev), thread_k, thread_n, sms,
        marlin::max_par);
  } else {
    TORCH_CHECK(false, "fp8_marlin_gemm only supports bfloat16 and float16");
  }

  return c;
}

#endif

TORCH_LIBRARY_IMPL_EXPAND(TORCH_EXTENSION_NAME, CUDA, m) {
  m.impl("fp8_marlin_gemm", &fp8_marlin_gemm);
}