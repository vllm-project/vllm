#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#ifndef USE_ROCM
  #include <cub/util_type.cuh>
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/util_type.hpp>
  #include <hipcub/hipcub.hpp>
#endif

#ifndef USE_ROCM
using FP8_TYPE = c10::Float8_e4m3fn;
C10_HOST_DEVICE constexpr auto FP8_E4M3_MAX =
    std::numeric_limits<FP8_TYPE>::max();
#else
  #include "amd/hip_float8.h"
using FP8_TYPE = c10::Float8_e4m3fnuz;
// Using the default max value from pytorch (240.0) will cause accuracy
// issue when running dynamic quantization. Here use 224.0f for rocm.
constexpr auto FP8_E4M3_MAX = 224.0f;
#endif

namespace vllm {

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
  float old;
  old = (value >= 0)
            ? __int_as_float(atomicMax((int*)addr, __float_as_int(value)))
            : __uint_as_float(
                  atomicMin((unsigned int*)addr, __float_as_uint(value)));

  return old;
}

template <bool is_scale_inverted>
__device__ __forceinline__ FP8_TYPE scaled_fp8_conversion(float const val,
                                                          float const scale) {
  float x = 0.0f;
  if constexpr (is_scale_inverted) {
    x = val * scale;
  } else {
    x = val / scale;
  }

  float r = fmax(-FP8_E4M3_MAX, fmin(x, FP8_E4M3_MAX));
#ifndef USE_ROCM
  return static_cast<c10::Float8_e4m3fn>(r);
#else
  // Use hardware cvt instruction for fp8 on rocm
  return c10::Float8_e4m3fnuz(hip_fp8(r).data,
                              c10::Float8_e4m3fnuz::from_bits());
#endif
}

// Compute the absolute maximum m of the input tensor and store
// m / float8_e4m3::max() in *scale. Each thread block performs a
// reduction tree and the memory in scale is atomically updated.
// So to get the right answer, *scale needs to be initialized to
// a value <= 0.0 and we need to wait for all thread blocks to
// finish before consuming *scale.
template <typename scalar_t>
__global__ void segmented_max_reduction(float* __restrict__ scale,
                                        const scalar_t* __restrict__ input,
                                        int64_t num_elems) {
  __shared__ float cache[1024];
  int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

  // First store maximum for all values processes by
  // the current thread in cache[threadIdx.x]
  scalar_t tmp = 0.0;
  while (i < num_elems) {
    float x = static_cast<float>(input[i]);
    tmp = max(tmp, fabs(x));
    i += blockDim.x * gridDim.x;
  }
  cache[threadIdx.x] = tmp;

  __syncthreads();

  // Now perform parallel reduction within the thread block
  int ib = blockDim.x / 2;
  while (ib != 0) {
    if (threadIdx.x < ib && cache[threadIdx.x + ib] > cache[threadIdx.x]) {
      cache[threadIdx.x] = cache[threadIdx.x + ib];
    }
    __syncthreads();
    ib /= 2;
  }
  // Finally, since cache[0] contains the maximum for this thread block,
  // atomically write the max to the target location
  if (threadIdx.x == 0) {
    atomicMaxFloat(scale, cache[0] / FP8_E4M3_MAX);
  }
}

template <typename scalar_t>
struct __align__(8) vec4_t {
  scalar_t x;
  scalar_t y;
  scalar_t z;
  scalar_t w;
};

typedef struct __align__(4) {
  FP8_TYPE x;
  FP8_TYPE y;
  FP8_TYPE z;
  FP8_TYPE w;
}
float8x4_t;

template <typename scalar_t>
__device__ float thread_max_vec(scalar_t const* __restrict__ input,
                                int64_t const num_elems, int const tid,
                                int const step) {
  // Vectorized input/output to better utilize memory bandwidth.
  vec4_t<scalar_t> const* vectorized_in =
      reinterpret_cast<vec4_t<scalar_t> const*>(input);

  int64_t const num_vec_elems = num_elems >> 2;
  float absmax_val = 0.0f;

#pragma unroll 4
  for (int64_t i = tid; i < num_vec_elems; i += step) {
    vec4_t<scalar_t> in_vec = vectorized_in[i];
    absmax_val = max(absmax_val, fabs(in_vec.x));
    absmax_val = max(absmax_val, fabs(in_vec.y));
    absmax_val = max(absmax_val, fabs(in_vec.z));
    absmax_val = max(absmax_val, fabs(in_vec.w));
  }

  // Handle the remaining elements if num_elems is not divisible by 4
  for (int64_t i = num_vec_elems * 4 + tid; i < num_elems; i += step) {
    absmax_val = max(absmax_val, fabs(input[i]));
  }

  return absmax_val;
}

template <typename scalar_t, bool is_scale_inverted>
__device__ void scaled_fp8_conversion_vec(FP8_TYPE* __restrict__ out,
                                          scalar_t const* __restrict__ input,
                                          float const scale,
                                          int64_t const num_elems,
                                          int const tid, int const step) {
  // Vectorized input/output to better utilize memory bandwidth.
  vec4_t<scalar_t> const* vectorized_in =
      reinterpret_cast<vec4_t<scalar_t> const*>(input);
  float8x4_t* vectorized_out = reinterpret_cast<float8x4_t*>(out);

  int64_t const num_vec_elems = num_elems >> 2;

#pragma unroll 4
  for (int64_t i = tid; i < num_vec_elems; i += step) {
    vec4_t<scalar_t> in_vec = vectorized_in[i];
    float8x4_t out_vec;

    out_vec.x = scaled_fp8_conversion<is_scale_inverted>(
        static_cast<float>(in_vec.x), scale);
    out_vec.y = scaled_fp8_conversion<is_scale_inverted>(
        static_cast<float>(in_vec.y), scale);
    out_vec.z = scaled_fp8_conversion<is_scale_inverted>(
        static_cast<float>(in_vec.z), scale);
    out_vec.w = scaled_fp8_conversion<is_scale_inverted>(
        static_cast<float>(in_vec.w), scale);
    vectorized_out[i] = out_vec;
  }

  // Handle the remaining elements if num_elems is not divisible by 4
  for (int64_t i = num_vec_elems * 4 + tid; i < num_elems; i += step) {
    out[i] = scaled_fp8_conversion<is_scale_inverted>(
        static_cast<float>(input[i]), scale);
  }
}

template <typename scalar_t>
__global__ void scaled_fp8_quant_kernel(FP8_TYPE* __restrict__ out,
                                        const scalar_t* __restrict__ input,
                                        const float* __restrict__ scale,
                                        int64_t num_elems) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Invert the scale so that we can use multiplications to avoid expensive
  // division.
  const float inverted_scale = 1.0f / (*scale);
  scaled_fp8_conversion_vec<scalar_t, true>(
      out, input, inverted_scale, num_elems, tid, blockDim.x * gridDim.x);
}

template <typename scalar_t>
__global__ void dynamic_per_token_scaled_fp8_quant_kernel(
    FP8_TYPE* __restrict__ out, float* __restrict__ scale,
    scalar_t const* __restrict__ input, float const* __restrict__ scale_ub,
    const int hidden_size) {
  float const min_scaling_factor = 1.0f / (FP8_E4M3_MAX * 512.f);

  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;

  // Use int64 to avoid overflowing an int32 when calculating this offset
  int64_t offset = static_cast<int64_t>(token_idx) * hidden_size;
  scalar_t const* __restrict__ token_input = &input[offset];
  FP8_TYPE* __restrict__ token_output = &out[offset];

  // For vectorization, token_input and token_output pointers need to be
  // aligned at 8-byte and 4-byte addresses respectively.
  bool const can_vectorize = hidden_size % 4 == 0;

  float absmax_val = 0.0f;
  if (can_vectorize) {
    absmax_val = thread_max_vec(token_input, hidden_size, tid, blockDim.x);
  } else {
    for (int i = tid; i < hidden_size; i += blockDim.x) {
      float const x = static_cast<float>(token_input[i]);
      absmax_val = max(absmax_val, fabs(x));
    }
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float token_scale;
  if (tid == 0) {
    if (scale_ub) {
      token_scale = min(block_absmax_val_maybe, *scale_ub);
    } else {
      token_scale = block_absmax_val_maybe;
    }
    // token scale computation
    token_scale = max(token_scale / FP8_E4M3_MAX, min_scaling_factor);
    scale[token_idx] = token_scale;
  }
  __syncthreads();

  // Note that we don't use inverted scales so we can match FBGemm impl.
  if (can_vectorize) {
    scaled_fp8_conversion_vec<scalar_t, false>(
        token_output, token_input, token_scale, hidden_size, tid, blockDim.x);
  } else {
    for (int i = tid; i < hidden_size; i += blockDim.x) {
      token_output[i] = scaled_fp8_conversion<false>(
          static_cast<float>(token_input[i]), token_scale);
    }
  }
}

}  // namespace vllm

void static_scaled_fp8_quant(torch::Tensor& out,          // [..., d]
                             torch::Tensor const& input,  // [..., d]
                             torch::Tensor const& scale)  // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<FP8_TYPE>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

void dynamic_scaled_fp8_quant(torch::Tensor& out,          // [..., d]
                              torch::Tensor const& input,  // [..., d]
                              torch::Tensor& scale)        // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::segmented_max_reduction<scalar_t><<<grid, block, 0, stream>>>(
            scale.data_ptr<float>(), input.data_ptr<scalar_t>(), num_elems);
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<FP8_TYPE>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

void dynamic_per_token_scaled_fp8_quant(
    torch::Tensor& out,          // [..., d]
    torch::Tensor const& input,  // [..., d]
    torch::Tensor& scales, std::optional<at::Tensor> const& scale_ub) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));

  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_per_token_scaled_fp8_quant_kernel", [&] {
        vllm::dynamic_per_token_scaled_fp8_quant_kernel<scalar_t>
            <<<grid, block, 0, stream>>>(
                out.data_ptr<FP8_TYPE>(), scales.data_ptr<float>(),
                input.data_ptr<scalar_t>(),
                scale_ub.has_value() ? scale_ub->data_ptr<float>() : nullptr,
                hidden_size);
      });
}
