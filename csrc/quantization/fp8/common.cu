#include "hip/hip_runtime.h"
#include "common.cuh"
#include "dispatch_utils.h"

#include <c10/cuda/CUDAGuard.h>

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/hipcub.hpp>
#endif

namespace vllm {

template <typename scalar_t>
__global__ void scaled_fp8_quant_kernel(FP8_TYPE* __restrict__ out,
                                        const scalar_t* __restrict__ input,
                                        const float* __restrict__ scale,
                                        int64_t num_elems) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Invert the scale so that we can use multiplications to avoid expensive
  // division.
  const float inverted_scale = 1.0f / (*scale);
  scaled_fp8_conversion_vec<scalar_t, true>(
      out, input, inverted_scale, num_elems, tid, blockDim.x * gridDim.x);
}

template <typename scalar_t>
__global__ void dynamic_per_token_scaled_fp8_quant_kernel(
    FP8_TYPE* __restrict__ out, float* __restrict__ scale,
    scalar_t const* __restrict__ input, float const* __restrict__ scale_ub,
    const int hidden_size) {
  float const min_scaling_factor = 1.0f / (FP8_E4M3_MAX * 512.f);

  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;

  // Use int64 to avoid overflowing an int32 when calculating this offset
  int64_t offset = static_cast<int64_t>(token_idx) * hidden_size;
  scalar_t const* __restrict__ token_input = &input[offset];
  FP8_TYPE* __restrict__ token_output = &out[offset];

  // For vectorization, token_input and token_output pointers need to be
  // aligned at 8-byte and 4-byte addresses respectively.
  bool const can_vectorize = hidden_size % 4 == 0;

  float absmax_val = 0.0f;
  if (can_vectorize) {
    absmax_val = thread_max_vec(token_input, hidden_size, tid, blockDim.x);
  } else {
    for (int i = tid; i < hidden_size; i += blockDim.x) {
      float const x = static_cast<float>(token_input[i]);
      absmax_val = max(absmax_val, fabs(x));
    }
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float token_scale;
  if (tid == 0) {
    if (scale_ub) {
      token_scale = min(block_absmax_val_maybe, *scale_ub);
    } else {
      token_scale = block_absmax_val_maybe;
    }
    // token scale computation
    token_scale = max(token_scale / FP8_E4M3_MAX, min_scaling_factor);
    scale[token_idx] = token_scale;
  }
  __syncthreads();

  // Note that we don't use inverted scales so we can match FBGemm impl.
  if (can_vectorize) {
    scaled_fp8_conversion_vec<scalar_t, false>(
        token_output, token_input, token_scale, hidden_size, tid, blockDim.x);
  } else {
    for (int i = tid; i < hidden_size; i += blockDim.x) {
      token_output[i] = scaled_fp8_conversion<false>(
          static_cast<float>(token_input[i]), token_scale);
    }
  }
}

}  // namespace vllm

void static_scaled_fp8_quant(torch::Tensor& out,          // [..., d]
                             torch::Tensor const& input,  // [..., d]
                             torch::Tensor const& scale)  // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<FP8_TYPE>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

void dynamic_scaled_fp8_quant(torch::Tensor& out,          // [..., d]
                              torch::Tensor const& input,  // [..., d]
                              torch::Tensor& scale)        // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::segmented_max_reduction<scalar_t><<<grid, block, 0, stream>>>(
            scale.data_ptr<float>(), input.data_ptr<scalar_t>(), num_elems);
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<FP8_TYPE>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

void dynamic_per_token_scaled_fp8_quant(
    torch::Tensor& out,          // [..., d]
    torch::Tensor const& input,  // [..., d]
    torch::Tensor& scales, std::optional<at::Tensor> const& scale_ub) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));

  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_per_token_scaled_fp8_quant_kernel", [&] {
        vllm::dynamic_per_token_scaled_fp8_quant_kernel<scalar_t>
            <<<grid, block, 0, stream>>>(
                out.data_ptr<FP8_TYPE>(), scales.data_ptr<float>(),
                input.data_ptr<scalar_t>(),
                scale_ub.has_value() ? scale_ub->data_ptr<float>() : nullptr,
                hidden_size);
      });
}
