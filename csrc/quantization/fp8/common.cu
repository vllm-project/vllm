#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
  float old;
  old = (value >= 0)
            ? __int_as_float(atomicMax((int*)addr, __float_as_int(value)))
            : __uint_as_float(
                  atomicMin((unsigned int*)addr, __float_as_uint(value)));

  return old;
}

#define FP8_E4M3_MAX std::numeric_limits<c10::Float8_e4m3fn>::max()

template <typename scalar_t>
__device__ __forceinline__ c10::Float8_e4m3fn scaled_fp8_conversion(
    const scalar_t val, const float inverted_scale) {
  float x = static_cast<float>(val) * inverted_scale;
  float r = fmax(-FP8_E4M3_MAX, fmin(x, FP8_E4M3_MAX));
  return static_cast<c10::Float8_e4m3fn>(r);
}

// Compute the absolute maximum m of the input tensor and store
// m / float8_e4m3::max() in *scale. Each thread block performs a
// reduction tree and the memory in scale is atomically updated.
// So to get the right answer, *scale needs to be initialized to
// a value <= 0.0 and we need to wait for all thread blocks to
// finish before consuming *scale.
template <typename scalar_t>
__global__ void segmented_max_reduction(float* __restrict__ scale,
                                        const scalar_t* __restrict__ input,
                                        int64_t num_elems) {
  __shared__ float cache[1024];
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  // First store maximum for all values processes by
  // the current thread in cache[threadIdx.x]
  scalar_t tmp = 0.0;
  while (i < num_elems) {
    float x = static_cast<float>(input[i]);
    tmp = max(tmp, fabs(x));
    i += blockDim.x * gridDim.x;
  }
  cache[threadIdx.x] = tmp;

  __syncthreads();

  // Now perform parallel reduction within the thread block
  int ib = blockDim.x / 2;
  while (ib != 0) {
    if (threadIdx.x < ib && cache[threadIdx.x + ib] > cache[threadIdx.x]) {
      cache[threadIdx.x] = cache[threadIdx.x + ib];
    }
    __syncthreads();
    ib /= 2;
  }
  // Finally, since cache[0] contains the maximum for this thread block,
  // atomically write the max to the target location
  if (threadIdx.x == 0) {
    atomicMaxFloat(scale,
                   cache[0] / std::numeric_limits<c10::Float8_e4m3fn>::max());
  }
}

template <typename scalar_t>
struct __align__(8) vec4_t {
  scalar_t x;
  scalar_t y;
  scalar_t z;
  scalar_t w;
};

typedef struct __align__(4) {
  c10::Float8_e4m3fn x;
  c10::Float8_e4m3fn y;
  c10::Float8_e4m3fn z;
  c10::Float8_e4m3fn w;
}
float8x4_t;

template <typename scalar_t>
__global__ void scaled_fp8_quant_kernel(c10::Float8_e4m3fn* __restrict__ out,
                                        const scalar_t* __restrict__ input,
                                        const float* __restrict__ scale,
                                        int64_t num_elems) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Invert the scale so that we can use multiplications to avoid expensive
  // division.
  const float inverted_scale = 1.0f / (*scale);

  // Vectorized input/output to better utilize memory bandwidth.
  const vec4_t<scalar_t>* vectorized_in =
      reinterpret_cast<const vec4_t<scalar_t>*>(input);
  float8x4_t* vectorized_out = reinterpret_cast<float8x4_t*>(out);

  int num_vec_elems = num_elems >> 2;

#pragma unroll 4
  for (int i = tid; i < num_vec_elems; i += blockDim.x * gridDim.x) {
    vec4_t<scalar_t> in_vec = vectorized_in[i];
    float8x4_t out_vec;

    out_vec.x = scaled_fp8_conversion(in_vec.x, inverted_scale);
    out_vec.y = scaled_fp8_conversion(in_vec.y, inverted_scale);
    out_vec.z = scaled_fp8_conversion(in_vec.z, inverted_scale);
    out_vec.w = scaled_fp8_conversion(in_vec.w, inverted_scale);
    vectorized_out[i] = out_vec;
  }

  // Handle the remaining elements if num_elems is not divisible by 4
  for (int i = num_vec_elems * 4 + tid; i < num_elems;
       i += blockDim.x * gridDim.x) {
    out[i] = scaled_fp8_conversion(input[i], inverted_scale);
  }
}

}  // namespace vllm

void static_scaled_fp8_quant(torch::Tensor& out,    // [..., d]
                             torch::Tensor& input,  // [..., d]
                             torch::Tensor& scale)  // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<c10::Float8_e4m3fn>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

void dynamic_scaled_fp8_quant(torch::Tensor& out,    // [..., d]
                              torch::Tensor& input,  // [..., d]
                              torch::Tensor& scale)  // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::segmented_max_reduction<scalar_t><<<grid, block, 0, stream>>>(
            scale.data_ptr<float>(), input.data_ptr<scalar_t>(), num_elems);
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<c10::Float8_e4m3fn>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}
