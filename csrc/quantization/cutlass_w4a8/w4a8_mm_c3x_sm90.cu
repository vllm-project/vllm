
#include <hip/hip_runtime.h>
/*
   This file defines quantized GEMM operations using the CUTLASS 3.x API, for
   NVIDIA GPUs with sm90a (Hopper).
*/

#if defined ENABLE_W4A8_MM_SM90 && ENABLE_W4A8_MM_SM90
void cutlass_scaled_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            std::optional<torch::Tensor> const& bias) {
  printf("hello czhu! \n");
  return;
}
#endif