#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>
#include <cutlass/arch/arch.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_runtime.h>
#include <unordered_map>
#include <algorithm>
#include <climits>
#include <cstdio>
#include <cstdlib>
#include <cassert>

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/packed_stride.hpp"

using namespace cute;

#if defined ENABLE_NVFP4_SM120 && ENABLE_NVFP4_SM120

#define CHECK_TYPE(x, st, m) \
  TORCH_CHECK(x.scalar_type() == st, ": Inconsistency of Tensor type:", m)
#define CHECK_TH_CUDA(x, m) \
  TORCH_CHECK(x.is_cuda(), m, ": must be a CUDA tensor.")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, ": must be contiguous.")
#define CHECK_INPUT(x, st, m) \
  CHECK_TH_CUDA(x, m);        \
  CHECK_CONTIGUOUS(x, m);     \
  CHECK_TYPE(x, st, m)

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;
constexpr auto SF_DTYPE = at::ScalarType::Float8_e4m3fn;

template <typename ElementAB, typename ElementC, typename ElementSF,
          typename ElementAccumulator, typename LayoutSFA, typename LayoutSFB,
          typename ScaleConfig>
__global__ void __get_group_gemm_starts_sm120(
    ElementAB** a_offsets, ElementAB** b_offsets, ElementC** out_offsets,
    ElementSF** a_scales_offsets, ElementSF** b_scales_offsets,
    ElementAccumulator** alpha_offsets, LayoutSFA* layout_sfa_base_as_int,
    LayoutSFB* layout_sfb_base_as_int, ElementAB* a_base_as_int,
    ElementAB* b_base_as_int, ElementC* out_base_as_int,
    ElementSF* a_scales_base_as_int, ElementSF* b_scales_base_as_int,
    ElementAccumulator* alphas_base_as_int, const int32_t* expert_offsets,
    const int32_t* sf_offsets, const int32_t* problem_sizes_as_shapes,
    const int K, const int N) {
  int64_t expert_id = threadIdx.x;
  if (expert_id >= gridDim.x * blockDim.x) {
    return;
  }
  int64_t expert_offset = static_cast<int64_t>(expert_offsets[expert_id]);
  int64_t sf_offset = static_cast<int64_t>(sf_offsets[expert_id]);
  int64_t group_size = 16;
  int64_t m = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3]);
  int64_t n = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3 + 1]);
  int64_t k = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3 + 2]);

  assert((m >= 0 && n == N && k == K && k % 2 == 0) &&
         "unexpected problem sizes");

  int64_t half_k = static_cast<int64_t>(k / 2);
  int64_t group_k = static_cast<int64_t>(k / group_size);

  a_offsets[expert_id] = a_base_as_int + expert_offset * half_k;
  b_offsets[expert_id] = b_base_as_int + expert_id * n * half_k;
  out_offsets[expert_id] = out_base_as_int + expert_offset * n;

  a_scales_offsets[expert_id] = a_scales_base_as_int + sf_offset * group_k;
  b_scales_offsets[expert_id] = b_scales_base_as_int + expert_id * n * group_k;
  alpha_offsets[expert_id] = alphas_base_as_int + expert_id;

  assert((reinterpret_cast<uintptr_t>(a_scales_offsets[expert_id]) % 128) == 0 &&
         "TMA requires 128-byte alignment");
  assert((reinterpret_cast<uintptr_t>(b_scales_offsets[expert_id]) % 128) == 0 &&
         "TMA requires 128-byte alignment");

  LayoutSFA* layout_sfa_ptr = layout_sfa_base_as_int + expert_id;
  LayoutSFB* layout_sfb_ptr = layout_sfb_base_as_int + expert_id;

  *layout_sfa_ptr = ScaleConfig::tile_atom_to_shape_SFA(
      cute::make_shape(static_cast<int>(m), static_cast<int>(n),
                       static_cast<int>(k), 1));
  *layout_sfb_ptr = ScaleConfig::tile_atom_to_shape_SFB(
      cute::make_shape(static_cast<int>(m), static_cast<int>(n),
                       static_cast<int>(k), 1));
}

#define __CALL_GET_STARTS_KERNEL_BLOCKSCALE_SM120(ELEMENT_AB_TYPE, SF_TYPE,    \
                                                  TENSOR_C_TYPE, C_TYPE,      \
                                                  LayoutSFA, LayoutSFB,       \
                                                  ScaleConfig)                \
  else if (out_tensors.dtype() == TENSOR_C_TYPE) {                            \
    __get_group_gemm_starts_sm120<ELEMENT_AB_TYPE, C_TYPE, SF_TYPE, float,     \
                                  LayoutSFA, LayoutSFB, ScaleConfig>          \
        <<<1, num_experts, 0, stream>>>(                                      \
            static_cast<ELEMENT_AB_TYPE**>(a_starts.data_ptr()),              \
            static_cast<ELEMENT_AB_TYPE**>(b_starts.data_ptr()),              \
            static_cast<C_TYPE**>(out_starts.data_ptr()),                     \
            static_cast<SF_TYPE**>(a_scales_starts.data_ptr()),               \
            static_cast<SF_TYPE**>(b_scales_starts.data_ptr()),               \
            static_cast<float**>(alpha_starts.data_ptr()),                    \
            reinterpret_cast<LayoutSFA*>(layout_sfa.data_ptr()),              \
            reinterpret_cast<LayoutSFB*>(layout_sfb.data_ptr()),              \
            static_cast<ELEMENT_AB_TYPE*>(a_tensors.data_ptr()),              \
            static_cast<ELEMENT_AB_TYPE*>(b_tensors.data_ptr()),              \
            static_cast<C_TYPE*>(out_tensors.data_ptr()),                     \
            static_cast<SF_TYPE*>(a_scales.data_ptr()),                       \
            static_cast<SF_TYPE*>(b_scales.data_ptr()),                       \
            static_cast<float*>(alphas.data_ptr()),                           \
            static_cast<int32_t*>(expert_offsets.data_ptr()),                 \
            static_cast<int32_t*>(sf_offsets.data_ptr()),                     \
            static_cast<int32_t*>(problem_sizes.data_ptr()), K, N);           \
  }

template <typename LayoutSFA, typename LayoutSFB, typename ScaleConfig>
static inline void run_get_group_gemm_starts_sm120(
    const torch::Tensor& a_starts, const torch::Tensor& b_starts,
    const torch::Tensor& out_starts, const torch::Tensor& a_scales_starts,
    const torch::Tensor& b_scales_starts, const torch::Tensor& alpha_starts,
    const torch::Tensor& layout_sfa, const torch::Tensor& layout_sfb,
    torch::Tensor const& a_tensors, torch::Tensor const& b_tensors,
    torch::Tensor const& out_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& alphas,
    torch::Tensor const& expert_offsets, torch::Tensor const& sf_offsets,
    torch::Tensor const& problem_sizes, int M, int N, int K) {
  int num_experts = (int)expert_offsets.size(0);
  auto stream = at::cuda::getCurrentCUDAStream(a_tensors.device().index());

  TORCH_CHECK(out_tensors.size(1) == N,
              "Output tensor shape doesn't match expected shape");
  TORCH_CHECK(K / 2 == b_tensors.size(2),
              "b_tensors(dim = 2) and a_tensors(dim = 1) trailing"
              " dimension must match");
  if (false) {
  }
  __CALL_GET_STARTS_KERNEL_BLOCKSCALE_SM120(
      cutlass::float_e2m1_t, cutlass::float_ue4m3_t, torch::kBFloat16,
      cutlass::bfloat16_t, LayoutSFA, LayoutSFB, ScaleConfig)
  __CALL_GET_STARTS_KERNEL_BLOCKSCALE_SM120(cutlass::float_e2m1_t,
                                            cutlass::float_ue4m3_t,
                                            torch::kFloat16, half, LayoutSFA,
                                            LayoutSFB, ScaleConfig)
  else {
    TORCH_CHECK(false, "Invalid output type (must be float16 or bfloat16)");
  }
}

static inline bool nvfp4_sm120_debug_enabled() {
  static int inited = 0;
  static bool enabled = false;
  if (!inited) {
    enabled = std::getenv("VLLM_DEBUG_NVFP4_MOE_SM120") != nullptr;
    inited = 1;
  }
  return enabled;
}

template <typename OutType>
static inline void run_fp4_blockwise_scaled_group_mm_sm120(
    torch::Tensor& output, const torch::Tensor& a, const torch::Tensor& b,
    const torch::Tensor& a_blockscale, const torch::Tensor& b_blockscales,
    const torch::Tensor& alphas, const torch::Tensor& problem_sizes,
    const torch::Tensor& expert_offsets, const torch::Tensor& sf_offsets, int M,
    int N, int K) {
  using ProblemShape =
      cutlass::gemm::GroupProblemShape<Shape<int32_t, int32_t, int32_t>>;
  using ElementType = cutlass::float_e2m1_t;
  using ElementSFType = cutlass::float_ue4m3_t;
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;

  using ElementC = OutType;
  using ElementD = ElementC;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using LayoutD = LayoutC;

  static constexpr int AlignmentA = 32;
  static constexpr int AlignmentB = 32;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  using ArchTag = cutlass::arch::Sm120;
  using EpilogueOperatorClass = cutlass::arch::OpClassTensorOp;
  using MainloopOperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;
  using ClusterShape = Shape<_1, _1, _1>;

  struct MMA1SMConfig {
    using MmaTileShape = Shape<_128, _128, _128>;
    using KernelSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpong;
    using EpilogueSchedule = cutlass::epilogue::collective::EpilogueScheduleAuto;
  };

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, EpilogueOperatorClass, typename MMA1SMConfig::MmaTileShape,
          ClusterShape, cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator, ElementAccumulator, ElementC, LayoutC*, AlignmentC,
          ElementD, LayoutD, AlignmentD,
          typename MMA1SMConfig::EpilogueSchedule>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, MainloopOperatorClass, ElementA, LayoutA*, AlignmentA,
          ElementB, LayoutB*, AlignmentB, ElementAccumulator,
          typename MMA1SMConfig::MmaTileShape, ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          typename MMA1SMConfig::KernelSchedule>::CollectiveOp;

  using GemmKernel =
      cutlass::gemm::kernel::GemmUniversal<ProblemShape, CollectiveMainloop,
                                           CollectiveEpilogue>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideA = typename Gemm::GemmKernel::InternalStrideA;
  using StrideB = typename Gemm::GemmKernel::InternalStrideB;
  using StrideC = typename Gemm::GemmKernel::InternalStrideC;
  using StrideD = typename Gemm::GemmKernel::InternalStrideD;

  using LayoutSFA =
      typename Gemm::GemmKernel::CollectiveMainloop::InternalLayoutSFA;
  using LayoutSFB =
      typename Gemm::GemmKernel::CollectiveMainloop::InternalLayoutSFB;
  using ScaleConfig =
      typename Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  using UnderlyingProblemShape = ProblemShape::UnderlyingProblemShape;
  int num_experts = static_cast<int>(expert_offsets.size(0));
  auto options_int =
      torch::TensorOptions().dtype(torch::kInt64).device(a.device());

  torch::Tensor a_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor b_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor out_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor a_scales_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor b_scales_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor alpha_ptrs = torch::empty(num_experts, options_int);
  torch::Tensor layout_sfa = torch::empty({num_experts, 5}, options_int);
  torch::Tensor layout_sfb = torch::empty({num_experts, 5}, options_int);
  torch::Tensor c_strides1 =
      torch::full({num_experts}, output.stride(0), options_int);
  torch::Tensor a_strides1 =
      torch::full({num_experts}, a.stride(0) * 2, options_int);
  torch::Tensor b_strides1 =
      torch::full({num_experts}, b.stride(1) * 2, options_int);

  run_get_group_gemm_starts_sm120<LayoutSFA, LayoutSFB, ScaleConfig>(
      a_ptrs, b_ptrs, out_ptrs, a_scales_ptrs, b_scales_ptrs, alpha_ptrs,
      layout_sfa, layout_sfb, a, b, output, a_blockscale, b_blockscales, alphas,
      expert_offsets, sf_offsets, problem_sizes, M, N, K);

  Gemm gemm_op;
  UnderlyingProblemShape* problem_sizes_as_shapes =
      static_cast<UnderlyingProblemShape*>(problem_sizes.data_ptr());

  if (nvfp4_sm120_debug_enabled()) {
    std::fprintf(stderr,
                 "[nvfp4-sm120] preparing grouped GEMM: num_experts=%d M=%d N=%d K=%d out_dtype=%d\n",
                 num_experts, M, N, K, static_cast<int>(output.scalar_type()));
  }

  cutlass::KernelHardwareInfo hw_info;
  typename Gemm::GemmKernel::TileSchedulerArguments scheduler{};
  hw_info.device_id = a.get_device();
  static std::unordered_map<int, int> cached_sm_counts;
  if (cached_sm_counts.find(hw_info.device_id) == cached_sm_counts.end()) {
    cached_sm_counts[hw_info.device_id] =
        cutlass::KernelHardwareInfo::query_device_multiprocessor_count(
            hw_info.device_id);
  }
  hw_info.sm_count = std::min(cached_sm_counts[hw_info.device_id], INT_MAX);

  typename GemmKernel::MainloopArguments mainloop_args{
      static_cast<const ElementType**>(a_ptrs.data_ptr()),
      static_cast<StrideA*>(a_strides1.data_ptr()),
      static_cast<const ElementType**>(b_ptrs.data_ptr()),
      static_cast<StrideB*>(b_strides1.data_ptr()),
      static_cast<const ElementSFType**>(a_scales_ptrs.data_ptr()),
      reinterpret_cast<LayoutSFA*>(layout_sfa.data_ptr()),
      static_cast<const ElementSFType**>(b_scales_ptrs.data_ptr()),
      reinterpret_cast<LayoutSFB*>(layout_sfb.data_ptr())};

  typename GemmKernel::EpilogueArguments epilogue_args{
      {},  // epilogue.thread
      nullptr,
      static_cast<StrideC*>(c_strides1.data_ptr()),
      static_cast<ElementD**>(out_ptrs.data_ptr()),
      static_cast<StrideC*>(c_strides1.data_ptr())};
  auto& fusion_args = epilogue_args.thread;
  fusion_args.alpha_ptr_array =
      reinterpret_cast<float**>(alpha_ptrs.data_ptr());
  fusion_args.dAlpha = {_0{}, _0{}, 1};

  typename GemmKernel::Arguments args{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {num_experts, problem_sizes_as_shapes, nullptr},
      mainloop_args,
      epilogue_args,
      hw_info,
      scheduler};

  size_t workspace_size = Gemm::get_workspace_size(args);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto can_implement_status = gemm_op.can_implement(args);
  if (nvfp4_sm120_debug_enabled()) {
    std::fprintf(stderr,
                 "[nvfp4-sm120] can_implement status=%d (0==success)\n",
                 static_cast<int>(can_implement_status));
  }
  TORCH_CHECK(can_implement_status == cutlass::Status::kSuccess,
              "Failed to implement GEMM (SM120 NVFP4 MoE)");

  auto status = gemm_op.initialize(args, workspace.data_ptr());
  if (nvfp4_sm120_debug_enabled()) {
    std::fprintf(stderr,
                 "[nvfp4-sm120] initialize status=%d (0==success)\n",
                 static_cast<int>(status));
  }
  TORCH_CHECK(status == cutlass::Status::kSuccess,
              "Failed to initialize GEMM (SM120 NVFP4 MoE)");

  status = gemm_op.run(args, workspace.data_ptr(), stream);
  if (nvfp4_sm120_debug_enabled()) {
    std::fprintf(stderr, "[nvfp4-sm120] run status=%d (0==success)\n",
                 static_cast<int>(status));
  }
  TORCH_CHECK(status == cutlass::Status::kSuccess,
              "Failed to run GEMM (SM120 NVFP4 MoE)");

  if (nvfp4_sm120_debug_enabled()) {
    hipError_t errSync = hipDeviceSynchronize();
    if (errSync != hipSuccess) {
      std::fprintf(stderr, "[nvfp4-sm120] hipDeviceSynchronize error: %s\n",
                   hipGetErrorString(errSync));
    }
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      std::fprintf(stderr, "[nvfp4-sm120] CUDA last error after run: %s\n",
                   hipGetErrorString(err));
    }
  }
}

void cutlass_fp4_group_mm_sm120(
    torch::Tensor& output, const torch::Tensor& a, const torch::Tensor& b,
    const torch::Tensor& a_blockscale, const torch::Tensor& b_blockscales,
    const torch::Tensor& alphas, const torch::Tensor& problem_sizes,
    const torch::Tensor& expert_offsets, const torch::Tensor& sf_offsets) {
  CHECK_INPUT(a, FLOAT4_E2M1X2, "a");
  CHECK_INPUT(b, FLOAT4_E2M1X2, "b");
  CHECK_INPUT(a_blockscale, SF_DTYPE, "a_blockscale");
  CHECK_INPUT(b_blockscales, SF_DTYPE, "b_blockscales");
  CHECK_INPUT(alphas, at::ScalarType::Float, "alphas");

  TORCH_CHECK(a_blockscale.dim() == 2,
              "expected a_blockscale to be of shape [num_experts, rounded_m,"
              " k // group_size], observed rank: ",
              a_blockscale.dim())
  TORCH_CHECK(b_blockscales.dim() == 3,
              "expected b_blockscale to be of shape: "
              " [num_experts, n, k // group_size], observed rank: ",
              b_blockscales.dim())
  TORCH_CHECK(problem_sizes.dim() == 2, "problem_sizes must be  a 2D tensor");
  TORCH_CHECK(problem_sizes.size(1) == 3,
              "problem_sizes must have the shape (num_experts, 3)");
  TORCH_CHECK(problem_sizes.size(0) == expert_offsets.size(0),
              "Number of experts in problem_sizes must match expert_offsets");
  TORCH_CHECK(problem_sizes.dtype() == torch::kInt32,
              "problem_sizes must be int32.");

  int M = static_cast<int>(a.size(0));
  int N = static_cast<int>(b.size(1));
  int E = static_cast<int>(b.size(0));
  int K = static_cast<int>(2 * b.size(2));

  if (nvfp4_sm120_debug_enabled()) {
    std::fprintf(stderr,
                 "[nvfp4-sm120] dispatch: E=%d M=%d N=%d K=%d out_dtype=%d\n",
                 E, M, N, K, static_cast<int>(output.scalar_type()));
  }

  if (output.scalar_type() == torch::kBFloat16) {
    run_fp4_blockwise_scaled_group_mm_sm120<cutlass::bfloat16_t>(
        output, a, b, a_blockscale, b_blockscales, alphas, problem_sizes,
        expert_offsets, sf_offsets, M, N, K);
  } else {
    run_fp4_blockwise_scaled_group_mm_sm120<cutlass::half_t>(
        output, a, b, a_blockscale, b_blockscales, alphas, problem_sizes,
        expert_offsets, sf_offsets, M, N, K);
  }
}

#endif // ENABLE_NVFP4_SM120
