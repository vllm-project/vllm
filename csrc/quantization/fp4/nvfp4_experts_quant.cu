#include "hip/hip_runtime.h"
#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>

template <typename T>
struct TypeConverter {
  using Type = half2;
};  // keep for generality

template <>
struct TypeConverter<half2> {
  using Type = half;
};

template <>
struct TypeConverter<half> {
  using Type = half2;
};

template <>
struct TypeConverter<__hip_bfloat162> {
  using Type = __hip_bfloat16;
};

template <>
struct TypeConverter<__hip_bfloat16> {
  using Type = __hip_bfloat162;
};

#define ELTS_PER_THREAD 8

constexpr int CVT_FP4_ELTS_PER_THREAD = 8;
constexpr int CVT_FP4_SF_VEC_SIZE = 16;

// Convert 8 float32 values into 8 e2m1 values (represented as one uint32_t).
inline __device__ uint32_t fp32_vec_to_e2m1(float (&array)[8]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{\n"
      ".reg .b8 byte0;\n"
      ".reg .b8 byte1;\n"
      ".reg .b8 byte2;\n"
      ".reg .b8 byte3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte0, %2, %1;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte1, %4, %3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte2, %6, %5;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte3, %8, %7;\n"
      "mov.b32 %0, {byte0, byte1, byte2, byte3};\n"
      "}"
      : "=r"(val)
      : "f"(array[0]), "f"(array[1]), "f"(array[2]), "f"(array[3]),
        "f"(array[4]), "f"(array[5]), "f"(array[6]), "f"(array[7]));
  return val;
#else
  return 0;
#endif
}

// Convert 4 float2 values into 8 e2m1 values (represented as one uint32_t).
inline __device__ uint32_t fp32_vec_to_e2m1(float2 (&array)[4]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{\n"
      ".reg .b8 byte0;\n"
      ".reg .b8 byte1;\n"
      ".reg .b8 byte2;\n"
      ".reg .b8 byte3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte0, %2, %1;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte1, %4, %3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte2, %6, %5;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte3, %8, %7;\n"
      "mov.b32 %0, {byte0, byte1, byte2, byte3};\n"
      "}"
      : "=r"(val)
      : "f"(array[0].x), "f"(array[0].y), "f"(array[1].x), "f"(array[1].y),
        "f"(array[2].x), "f"(array[2].y), "f"(array[3].x), "f"(array[3].y));
  return val;
#else
  return 0;
#endif
}

// Fast reciprocal.
inline __device__ float reciprocal_approximate_ftz(float a) {
  float b;
  asm volatile("rcp.approx.ftz.f32 %0, %1;\n" : "=f"(b) : "f"(a));
  return b;
}

template <class SFType, int CVT_FP4_NUM_THREADS_PER_SF>
__device__ uint8_t* cvt_quant_to_fp4_get_sf_out_offset(int rowIdx, int colIdx,
                                                       int numCols,
                                                       SFType* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  static_assert(CVT_FP4_NUM_THREADS_PER_SF == 1 ||
                CVT_FP4_NUM_THREADS_PER_SF == 2);

  // One pair of threads write one SF to global memory.
  // TODO: stage through smem for packed STG.32
  // is it better than STG.8 from 4 threads ?
  if (threadIdx.x % CVT_FP4_NUM_THREADS_PER_SF == 0) {
    // SF vector index (16 elements share one SF in the K dimension).
    int32_t kIdx = colIdx / CVT_FP4_NUM_THREADS_PER_SF;
    int32_t mIdx = rowIdx;

    // SF layout [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
    // --> index [mTileIdx, kTileIdx, outerMIdx, innerMIdx, innerKIdx]

    int32_t mTileIdx = mIdx / (32 * 4);
    // SF vector size 16.
    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    int32_t numKTiles = (numCols + factor - 1) / factor;
    int64_t mTileStride = numKTiles * 32 * 4 * 4;

    int32_t kTileIdx = (kIdx / 4);
    int64_t kTileStride = 32 * 4 * 4;

    // M tile layout [32, 4] is column-major.
    int32_t outerMIdx = (mIdx % 32);
    int64_t outerMStride = 4 * 4;

    int32_t innerMIdx = (mIdx % (32 * 4)) / 32;
    int64_t innerMStride = 4;

    int32_t innerKIdx = (kIdx % 4);
    int64_t innerKStride = 1;

    // Compute the global offset.
    int64_t SFOffset = mTileIdx * mTileStride + kTileIdx * kTileStride +
                       outerMIdx * outerMStride + innerMIdx * innerMStride +
                       innerKIdx * innerKStride;

    return reinterpret_cast<uint8_t*>(SFout) + SFOffset;
  }
#endif
  return nullptr;
}

// Define a 16 bytes packed data type.
template <class Type>
struct PackedVec {
  typename TypeConverter<Type>::Type elts[4];
};

template <>
struct PackedVec<__hip_fp8_e4m3_fnuz> {
  __hip_fp8x2_e4m3_fnuz elts[8];
};

// Quantizes the provided PackedVec into the uint32_t output
template <class Type, bool UE8M0_SF = false>
__device__ uint32_t cvt_warp_fp16_to_fp4(PackedVec<Type>& vec, float SFScaleVal,
                                         uint8_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

  // Local maximum value.
  #pragma unroll
  for (int i = 1; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 16 values (two threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e2m1).
  // maximum value of e2m1 = 6.0.
  // TODO: use half as compute data type.
  float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  if constexpr (UE8M0_SF) {
    // Extract the 8 exponent bits from float32.
    // float 32bits = 1 sign bit + 8 exponent bits + 23 mantissa bits.
    uint32_t tmp = reinterpret_cast<uint32_t&>(SFValue) >> 23;
    fp8SFVal = tmp & 0xff;
    // Convert back to fp32.
    reinterpret_cast<uint32_t&>(SFValue) = tmp << 23;
  } else {
    // Here SFValue is always positive, so E4M3 is the same as UE4M3.
    __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(SFValue);
    reinterpret_cast<__hip_fp8_e4m3_fnuz&>(fp8SFVal) = tmp;
    // Convert back to fp32.
    SFValue = float(tmp);
  }
  // Get the output scale.
  // Recipe: final_scale = reciprocal(fp32(fp8(SFValue * SFScaleVal))) *
  //                       reciprocal(SFScaleVal))
  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(
                         SFValue * reciprocal_approximate_ftz(SFScaleVal))
                   : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP4_ELTS_PER_THREAD / 2];

  #pragma unroll
  for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e2m1 values.
  uint32_t e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

  // Write the e2m1 values to global memory.
  return e2m1Vec;
#else
  return 0;
#endif
}

// Use UE4M3 by default.
template <class Type, bool UE8M0_SF = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(512, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows, int32_t numCols, Type const* in, float const* SFScale,
    uint32_t* out, uint32_t* SFout, uint32_t* input_offset_by_experts,
    uint32_t* output_scale_offset_by_experts, int n_experts) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF =
      (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD,
                "Vec size is not matched.");

  // Input tensor row/col loops.
  for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x) {
    for (int colIdx = threadIdx.x; colIdx < numCols / CVT_FP4_ELTS_PER_THREAD;
         colIdx += blockDim.x) {
      int64_t inOffset = rowIdx * (numCols / CVT_FP4_ELTS_PER_THREAD) + colIdx;
      PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
      // Get the output tensor offset.
      // Same as inOffset because 8 elements are packed into one uint32_t.
      int64_t outOffset = inOffset;
      auto& out_pos = out[outOffset];

      // Find index within the experts.
      int rowIdx_in_expert = 0;
      int expert_idx = 0;
      for (int i = 0; i < n_experts; i++) {
        if (rowIdx >= input_offset_by_experts[i] &&
            rowIdx < input_offset_by_experts[i + 1]) {
          rowIdx_in_expert = rowIdx - input_offset_by_experts[i];
          expert_idx = i;
          break;
        }
      }

      // Get the global scaling factor, which will be applied to the SF.
      // Note SFScale is the same as next GEMM's alpha, which is
      // (448.f / (Alpha_A / 6.f)).
      float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

      int factor = CVT_FP4_SF_VEC_SIZE * 4;
      // The actual output_scales dim is computed from the padded numCols.
      int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
      int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
      uint32_t* SFout_in_expert =
          SFout + output_scale_offset_by_experts[expert_idx] * numCols_SFout;

      auto sf_out =
          cvt_quant_to_fp4_get_sf_out_offset<uint32_t,
                                             CVT_FP4_NUM_THREADS_PER_SF>(
              rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

      out_pos =
          cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
    }
  }
#endif
}

template <typename T>
void quant_impl(void* output, void* output_scale, void* input,
                void* input_global_scale, void* input_offset_by_experts,
                void* output_scale_offset_by_experts, int m_topk, int k,
                int n_experts, hipStream_t stream) {
  // TODO: this multiProcessorCount should be cached.
  int device;
  hipGetDevice(&device);
  int multiProcessorCount;
  hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount,
                         device);

  // Grid, Block size.
  // Each thread converts 8 values.
  dim3 block(std::min(int(k / ELTS_PER_THREAD), 512));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 2048 / block.x;
  dim3 grid(std::min(int(m_topk), multiProcessorCount * numBlocksPerSM));

  cvt_fp16_to_fp4<T, false><<<grid, block, 0, stream>>>(
      m_topk, k, reinterpret_cast<T*>(input),
      reinterpret_cast<float*>(input_global_scale),
      reinterpret_cast<uint32_t*>(output),
      reinterpret_cast<uint32_t*>(output_scale),
      reinterpret_cast<uint32_t*>(input_offset_by_experts),
      reinterpret_cast<uint32_t*>(output_scale_offset_by_experts), n_experts);
}

/*Quantization entry for fp4 experts quantization*/
#define CHECK_TH_CUDA(x, m) TORCH_CHECK(x.is_cuda(), m, "must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, "must be contiguous")
#define CHECK_INPUT(x, m) \
  CHECK_TH_CUDA(x, m);    \
  CHECK_CONTIGUOUS(x, m);

constexpr auto HALF = at::ScalarType::Half;
constexpr auto BF16 = at::ScalarType::BFloat16;
constexpr auto FLOAT = at::ScalarType::Float;
constexpr auto INT = at::ScalarType::Int;
constexpr auto UINT8 = at::ScalarType::Byte;

void scaled_fp4_experts_quant_sm100a(
    torch::Tensor& output, torch::Tensor& output_scale,
    torch::Tensor const& input, torch::Tensor const& input_global_scale,
    torch::Tensor const& input_offset_by_experts,
    torch::Tensor const& output_scale_offset_by_experts) {
  CHECK_INPUT(output, "output must be a CUDA tensor");
  CHECK_INPUT(output_scale, "output_scale must be a CUDA tensor");
  CHECK_INPUT(input, "input must be a CUDA tensor");
  CHECK_INPUT(input_global_scale, "input_global_scale must be a CUDA tensor");
  CHECK_INPUT(input_offset_by_experts,
              "input_offset_by_experts must be a CUDA tensor");
  CHECK_INPUT(output_scale_offset_by_experts,
              "output_scale_offset_by_experts must be a CUDA tensor");

  TORCH_CHECK(output.dim() == 2);
  TORCH_CHECK(output_scale.dim() == 2);
  TORCH_CHECK(input.dim() == 2);
  TORCH_CHECK(input_global_scale.dim() == 1);
  TORCH_CHECK(input_offset_by_experts.dim() == 1);
  TORCH_CHECK(output_scale_offset_by_experts.dim() == 1);

  TORCH_CHECK(input.scalar_type() == HALF || input.scalar_type() == BF16);
  TORCH_CHECK(input_global_scale.scalar_type() == FLOAT);
  TORCH_CHECK(input_offset_by_experts.scalar_type() == INT);
  TORCH_CHECK(output_scale_offset_by_experts.scalar_type() == INT);
  // output is uint8 (two nvfp4 values are packed into one uint8)
  // output_scale is int32 (four fp8 values are packed into one int32)
  TORCH_CHECK(output.scalar_type() == UINT8);
  TORCH_CHECK(output_scale.scalar_type() == INT);

  const int BLOCK_SIZE = 16;
  auto m_topk = input.size(0);
  auto k = input.size(1);
  TORCH_CHECK(k % BLOCK_SIZE == 0, "k must be a multiple of 16");
  auto n_experts = input_global_scale.size(0);
  TORCH_CHECK(input_offset_by_experts.size(0) == n_experts + 1);
  TORCH_CHECK(output_scale_offset_by_experts.size(0) == n_experts + 1);
  TORCH_CHECK(output.size(0) == m_topk);
  TORCH_CHECK(output.size(1) == k / 2);
  int scales_k = k / BLOCK_SIZE;
  // 4 means the swizzle requirement by nvidia nvfp4.
  int padded_k = (scales_k + (4 - 1)) / 4 * 4;
  // 4 means 4 fp8 values are packed into one int32
  TORCH_CHECK(output_scale.size(1) * 4 == padded_k);

  auto in_dtype = input.dtype();
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream =
      at::cuda::getCurrentCUDAStream(input.get_device());
  if (in_dtype == at::ScalarType::Half) {
    quant_impl<half>(output.data_ptr(), output_scale.data_ptr(),
                     input.data_ptr(), input_global_scale.data_ptr(),
                     input_offset_by_experts.data_ptr(),
                     output_scale_offset_by_experts.data_ptr(), m_topk, k,
                     n_experts, stream);
  } else if (in_dtype == at::ScalarType::BFloat16) {
    quant_impl<__hip_bfloat16>(output.data_ptr(), output_scale.data_ptr(),
                              input.data_ptr(), input_global_scale.data_ptr(),
                              input_offset_by_experts.data_ptr(),
                              output_scale_offset_by_experts.data_ptr(), m_topk,
                              k, n_experts, stream);
  } else {
    TORCH_CHECK(false, "Expected input data type to be half or bfloat16");
  }
}