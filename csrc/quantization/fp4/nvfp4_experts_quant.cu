#include "hip/hip_runtime.h"
#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>

template <typename T>
struct TypeConverter {
  using Type = half2;
};  // keep for generality

template <>
struct TypeConverter<half2> {
  using Type = half;
};

template <>
struct TypeConverter<half> {
  using Type = half2;
};

template <>
struct TypeConverter<__hip_bfloat162> {
  using Type = __hip_bfloat16;
};

template <>
struct TypeConverter<__hip_bfloat16> {
  using Type = __hip_bfloat162;
};

#define ELTS_PER_THREAD 8

constexpr int CVT_FP4_ELTS_PER_THREAD = 8;
constexpr int CVT_FP4_SF_VEC_SIZE = 16;

// Convert 8 float32 values into 8 e2m1 values (represented as one uint32_t).
inline __device__ uint32_t fp32_vec_to_e2m1(float (&array)[8]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{\n"
      ".reg .b8 byte0;\n"
      ".reg .b8 byte1;\n"
      ".reg .b8 byte2;\n"
      ".reg .b8 byte3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte0, %2, %1;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte1, %4, %3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte2, %6, %5;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte3, %8, %7;\n"
      "mov.b32 %0, {byte0, byte1, byte2, byte3};\n"
      "}"
      : "=r"(val)
      : "f"(array[0]), "f"(array[1]), "f"(array[2]), "f"(array[3]),
        "f"(array[4]), "f"(array[5]), "f"(array[6]), "f"(array[7]));
  return val;
#else
  return 0;
#endif
}

// Convert 4 float2 values into 8 e2m1 values (represented as one uint32_t).
inline __device__ uint32_t fp32_vec_to_e2m1(float2 (&array)[4]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{\n"
      ".reg .b8 byte0;\n"
      ".reg .b8 byte1;\n"
      ".reg .b8 byte2;\n"
      ".reg .b8 byte3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte0, %2, %1;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte1, %4, %3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte2, %6, %5;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte3, %8, %7;\n"
      "mov.b32 %0, {byte0, byte1, byte2, byte3};\n"
      "}"
      : "=r"(val)
      : "f"(array[0].x), "f"(array[0].y), "f"(array[1].x), "f"(array[1].y),
        "f"(array[2].x), "f"(array[2].y), "f"(array[3].x), "f"(array[3].y));
  return val;
#else
  return 0;
#endif
}

// Fast reciprocal.
inline __device__ float reciprocal_approximate_ftz(float a) {
  float b;
  asm volatile("rcp.approx.ftz.f32 %0, %1;\n" : "=f"(b) : "f"(a));
  return b;
}

template <class SFType, int CVT_FP4_NUM_THREADS_PER_SF>
__device__ uint8_t* cvt_quant_to_fp4_get_sf_out_offset(int rowIdx, int colIdx,
                                                       int numCols,
                                                       SFType* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  static_assert(CVT_FP4_NUM_THREADS_PER_SF == 1 ||
                CVT_FP4_NUM_THREADS_PER_SF == 2);

  // One pair of threads write one SF to global memory.
  // TODO: stage through smem for packed STG.32
  // is it better than STG.8 from 4 threads ?
  if (threadIdx.x % CVT_FP4_NUM_THREADS_PER_SF == 0) {
    // SF vector index (16 elements share one SF in the K dimension).
    int32_t kIdx = colIdx / CVT_FP4_NUM_THREADS_PER_SF;
    int32_t mIdx = rowIdx;

    // SF layout [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
    // --> index [mTileIdx, kTileIdx, outerMIdx, innerMIdx, innerKIdx]

    int32_t mTileIdx = mIdx / (32 * 4);
    // SF vector size 16.
    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    int32_t numKTiles = (numCols + factor - 1) / factor;
    int64_t mTileStride = numKTiles * 32 * 4 * 4;

    int32_t kTileIdx = (kIdx / 4);
    int64_t kTileStride = 32 * 4 * 4;

    // M tile layout [32, 4] is column-major.
    int32_t outerMIdx = (mIdx % 32);
    int64_t outerMStride = 4 * 4;

    int32_t innerMIdx = (mIdx % (32 * 4)) / 32;
    int64_t innerMStride = 4;

    int32_t innerKIdx = (kIdx % 4);
    int64_t innerKStride = 1;

    // Compute the global offset.
    int64_t SFOffset = mTileIdx * mTileStride + kTileIdx * kTileStride +
                       outerMIdx * outerMStride + innerMIdx * innerMStride +
                       innerKIdx * innerKStride;

    return reinterpret_cast<uint8_t*>(SFout) + SFOffset;
  }
#endif
  return nullptr;
}

// Define a 16 bytes packed data type.
template <class Type>
struct PackedVec {
  typename TypeConverter<Type>::Type elts[4];
};

template <>
struct PackedVec<__hip_fp8_e4m3_fnuz> {
  __hip_fp8x2_e4m3_fnuz elts[8];
};

// Quantizes the provided PackedVec into the uint32_t output
template <class Type, bool UE8M0_SF = false>
__device__ uint32_t cvt_warp_fp16_to_fp4(PackedVec<Type>& vec, float SFScaleVal,
                                         uint8_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

  // Local maximum value.
  #pragma unroll
  for (int i = 1; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 16 values (two threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e2m1).
  // maximum value of e2m1 = 6.0.
  // TODO: use half as compute data type.
  float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  if constexpr (UE8M0_SF) {
    // Extract the 8 exponent bits from float32.
    // float 32bits = 1 sign bit + 8 exponent bits + 23 mantissa bits.
    uint32_t tmp = reinterpret_cast<uint32_t&>(SFValue) >> 23;
    fp8SFVal = tmp & 0xff;
    // Convert back to fp32.
    reinterpret_cast<uint32_t&>(SFValue) = tmp << 23;
  } else {
    // Here SFValue is always positive, so E4M3 is the same as UE4M3.
    __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(SFValue);
    reinterpret_cast<__hip_fp8_e4m3_fnuz&>(fp8SFVal) = tmp;
    // Convert back to fp32.
    SFValue = float(tmp);
  }
  // Get the output scale.
  // Recipe: final_scale = reciprocal(fp32(fp8(SFValue * SFScaleVal))) *
  //                       reciprocal(SFScaleVal))
  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(
                         SFValue * reciprocal_approximate_ftz(SFScaleVal))
                   : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP4_ELTS_PER_THREAD / 2];

  #pragma unroll
  for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e2m1 values.
  uint32_t e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

  // Write the e2m1 values to global memory.
  return e2m1Vec;
#else
  return 0;
#endif
}

// Use UE4M3 by default.
template <class Type, bool UE8M0_SF = false, bool SMALL_NUM_EXPERTS = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(512, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows, int32_t numCols, Type const* in, float const* SFScale,
    uint32_t* out, uint32_t* SFout, uint32_t* input_offset_by_experts,
    uint32_t* output_scale_offset_by_experts, int n_experts, bool low_latency) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF =
      (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD,
                "Vec size is not matched.");

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int colsPerRow = numCols / CVT_FP4_ELTS_PER_THREAD;

  // Each global thread processes one element
  for (int globalIdx = tid; globalIdx < numRows * colsPerRow;
       globalIdx += gridDim.x * blockDim.x) {
    // Calculate which row and column this global thread should process
    int rowIdx = globalIdx / colsPerRow;
    int colIdx = globalIdx % colsPerRow;

    int64_t inOffset = rowIdx * colsPerRow + colIdx;
    PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
    // Get the output tensor offset.
    // Same as inOffset because 8 elements are packed into one uint32_t.
    int64_t outOffset = inOffset;
    auto& out_pos = out[outOffset];

    // Find index within the experts using different strategies based on expert
    // count
    int rowIdx_in_expert = 0;
    int expert_idx = 0;

    if constexpr (SMALL_NUM_EXPERTS) {
      for (int i = 0; i < n_experts; i++) {
        uint32_t current_offset = __ldca(&input_offset_by_experts[i]);
        uint32_t next_offset = __ldca(&input_offset_by_experts[i + 1]);
        if (rowIdx >= current_offset && rowIdx < next_offset) {
          rowIdx_in_expert = rowIdx - current_offset;
          expert_idx = i;
          break;
        }
      }
    } else {
      // Load input offsets into registers first, then do the computation.
      // Local array size set to 17 because of register limit.
      uint32_t local_offsets[17];
      for (int chunk_start = 0; chunk_start < n_experts; chunk_start += 16) {
        *reinterpret_cast<int4*>(local_offsets) =
            __ldca(reinterpret_cast<const int4*>(
                &input_offset_by_experts[chunk_start]));
        *reinterpret_cast<int4*>(local_offsets + 4) =
            __ldca(reinterpret_cast<const int4*>(
                &input_offset_by_experts[chunk_start + 4]));
        *reinterpret_cast<int4*>(local_offsets + 8) =
            __ldca(reinterpret_cast<const int4*>(
                &input_offset_by_experts[chunk_start + 8]));
        *reinterpret_cast<int4*>(local_offsets + 12) =
            __ldca(reinterpret_cast<const int4*>(
                &input_offset_by_experts[chunk_start + 12]));
        local_offsets[16] = __ldca(&input_offset_by_experts[chunk_start + 16]);

  // Check against the 16 loaded offsets
  #pragma unroll
        for (int i = 0; i < 16; i++) {
          if (rowIdx >= local_offsets[i] && rowIdx < local_offsets[i + 1]) {
            rowIdx_in_expert = rowIdx - local_offsets[i];
            expert_idx = chunk_start + i;
            break;
          }
        }
      }
    }

    // Get the global scaling factor, which will be applied to the SF.
    // Note SFScale is the same as next GEMM's alpha, which is
    // (448.f / (Alpha_A / 6.f)).
    float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    // The actual output_scales dim is computed from the padded numCols.
    int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
    int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
    uint32_t* SFout_in_expert =
        SFout + output_scale_offset_by_experts[expert_idx] * numCols_SFout;

    auto sf_out =
        cvt_quant_to_fp4_get_sf_out_offset<uint32_t,
                                           CVT_FP4_NUM_THREADS_PER_SF>(
            rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

    out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
  }
#endif
}

// Kernel for LARGE_M_TOPK = true (large m_topk optimized version)
template <class Type, bool UE8M0_SF = false, bool SMALL_NUM_EXPERTS = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(1024, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows, int32_t numCols, Type const* in, float const* SFScale,
    uint32_t* out, uint32_t* SFout, uint32_t* input_offset_by_experts,
    uint32_t* output_scale_offset_by_experts, int n_experts) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF =
      (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD,
                "Vec size is not matched.");
  extern __shared__ uint32_t shared_input_offsets[];

  // Load input offsets into shared memory.
  // If n_experts is larger than 4, use vectorized int4 to save instructions.
  // If n_experts is smaller than 4, read directly.
  if constexpr (SMALL_NUM_EXPERTS) {
    for (int i = threadIdx.x; i < n_experts + 1; i += blockDim.x) {
      shared_input_offsets[i] = input_offset_by_experts[i];
    }
  } else {
    for (int i = threadIdx.x * 4; i < n_experts; i += blockDim.x * 4) {
      *reinterpret_cast<int4*>(&shared_input_offsets[i]) =
          *reinterpret_cast<const int4*>(&input_offset_by_experts[i]);
    }
    if (threadIdx.x == 0) {
      shared_input_offsets[n_experts] = input_offset_by_experts[n_experts];
    }
  }

  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int colsPerRow = numCols / CVT_FP4_ELTS_PER_THREAD;

  // Each global thread processes one element
  for (int globalIdx = tid; globalIdx < numRows * colsPerRow;
       globalIdx += gridDim.x * blockDim.x) {
    // Calculate which row and column this global thread should process
    int rowIdx = globalIdx / colsPerRow;
    int colIdx = globalIdx % colsPerRow;

    int64_t inOffset = rowIdx * colsPerRow + colIdx;
    PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
    int64_t outOffset = inOffset;
    auto& out_pos = out[outOffset];

    // Find expert using binary search for better performance with large m_topk
    int rowIdx_in_expert = 0;
    int expert_idx = 0;

    // Binary search through experts using shared memory
    int left = 0, right = n_experts - 1;
    while (left <= right) {
      int mid = (left + right) / 2;
      // Get offsets: shared_input_offsets[i] corresponds to
      // input_offset_by_experts[i]
      uint32_t mid_offset = shared_input_offsets[mid];
      uint32_t next_offset = shared_input_offsets[mid + 1];

      if (rowIdx >= mid_offset && rowIdx < next_offset) {
        rowIdx_in_expert = rowIdx - mid_offset;
        expert_idx = mid;
        break;
      } else if (rowIdx < mid_offset) {
        right = mid - 1;
      } else {
        left = mid + 1;
      }
    }

    float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
    int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
    uint32_t* SFout_in_expert =
        SFout + output_scale_offset_by_experts[expert_idx] * numCols_SFout;

    auto sf_out =
        cvt_quant_to_fp4_get_sf_out_offset<uint32_t,
                                           CVT_FP4_NUM_THREADS_PER_SF>(
            rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

    out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
  }
#endif
}

template <typename T>
void quant_impl(void* output, void* output_scale, void* input,
                void* input_global_scale, void* input_offset_by_experts,
                void* output_scale_offset_by_experts, int m_topk, int k,
                int n_experts, hipStream_t stream) {
  // TODO: this multiProcessorCount should be cached.
  int device;
  hipGetDevice(&device);
  int multiProcessorCount;
  hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount,
                         device);

  // Grid, Block size.
  // Each thread converts 8 values.
  int const workSizePerRow = k / ELTS_PER_THREAD;
  int const totalWorkSize = m_topk * workSizePerRow;
  dim3 block(std::min(workSizePerRow, 512));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 2048 / block.x;
  dim3 grid(std::min(static_cast<int>((totalWorkSize + block.x - 1) / block.x),
                     multiProcessorCount * numBlocksPerSM));
  while (grid.x <= multiProcessorCount && block.x > 64) {
    grid.x *= 2;
    block.x = (block.x + 1) / 2;
  }

  int const blockRepeat =
      (totalWorkSize + block.x * grid.x - 1) / (block.x * grid.x);
  if (blockRepeat > 1) {
    size_t shared_mem_size = (n_experts + 1) * sizeof(uint32_t);
    if (n_experts >= 4) {
      cvt_fp16_to_fp4<T, false, false>
          <<<grid, block, shared_mem_size, stream>>>(
              m_topk, k, reinterpret_cast<T*>(input),
              reinterpret_cast<float*>(input_global_scale),
              reinterpret_cast<uint32_t*>(output),
              reinterpret_cast<uint32_t*>(output_scale),
              reinterpret_cast<uint32_t*>(input_offset_by_experts),
              reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
              n_experts);
    } else {
      cvt_fp16_to_fp4<T, false, true><<<grid, block, shared_mem_size, stream>>>(
          m_topk, k, reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          n_experts);
    }
  } else {
    if (n_experts >= 16) {
      cvt_fp16_to_fp4<T, false, false><<<grid, block, 0, stream>>>(
          m_topk, k, reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          n_experts, /* bool low_latency */ true);
    } else {
      cvt_fp16_to_fp4<T, false, true><<<grid, block, 0, stream>>>(
          m_topk, k, reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          n_experts, /* bool low_latency */ true);
    }
  }
}

/*Quantization entry for fp4 experts quantization*/
#define CHECK_TH_CUDA(x, m) TORCH_CHECK(x.is_cuda(), m, "must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, "must be contiguous")
#define CHECK_INPUT(x, m) \
  CHECK_TH_CUDA(x, m);    \
  CHECK_CONTIGUOUS(x, m);

constexpr auto HALF = at::ScalarType::Half;
constexpr auto BF16 = at::ScalarType::BFloat16;
constexpr auto FLOAT = at::ScalarType::Float;
constexpr auto INT = at::ScalarType::Int;
constexpr auto UINT8 = at::ScalarType::Byte;

void scaled_fp4_experts_quant_sm100a(
    torch::Tensor& output, torch::Tensor& output_scale,
    torch::Tensor const& input, torch::Tensor const& input_global_scale,
    torch::Tensor const& input_offset_by_experts,
    torch::Tensor const& output_scale_offset_by_experts) {
  CHECK_INPUT(output, "output must be a CUDA tensor");
  CHECK_INPUT(output_scale, "output_scale must be a CUDA tensor");
  CHECK_INPUT(input, "input must be a CUDA tensor");
  CHECK_INPUT(input_global_scale, "input_global_scale must be a CUDA tensor");
  CHECK_INPUT(input_offset_by_experts,
              "input_offset_by_experts must be a CUDA tensor");
  CHECK_INPUT(output_scale_offset_by_experts,
              "output_scale_offset_by_experts must be a CUDA tensor");

  TORCH_CHECK(output.dim() == 2);
  TORCH_CHECK(output_scale.dim() == 2);
  TORCH_CHECK(input.dim() == 2);
  TORCH_CHECK(input_global_scale.dim() == 1);
  TORCH_CHECK(input_offset_by_experts.dim() == 1);
  TORCH_CHECK(output_scale_offset_by_experts.dim() == 1);

  TORCH_CHECK(input.scalar_type() == HALF || input.scalar_type() == BF16);
  TORCH_CHECK(input_global_scale.scalar_type() == FLOAT);
  TORCH_CHECK(input_offset_by_experts.scalar_type() == INT);
  TORCH_CHECK(output_scale_offset_by_experts.scalar_type() == INT);
  // output is uint8 (two nvfp4 values are packed into one uint8)
  // output_scale is int32 (four fp8 values are packed into one int32)
  TORCH_CHECK(output.scalar_type() == UINT8);
  TORCH_CHECK(output_scale.scalar_type() == INT);

  const int BLOCK_SIZE = 16;
  auto m_topk = input.size(0);
  auto k = input.size(1);
  TORCH_CHECK(k % BLOCK_SIZE == 0, "k must be a multiple of 16");
  auto n_experts = input_global_scale.size(0);
  TORCH_CHECK(input_offset_by_experts.size(0) == n_experts + 1);
  TORCH_CHECK(output_scale_offset_by_experts.size(0) == n_experts + 1);
  TORCH_CHECK(output.size(0) == m_topk);
  TORCH_CHECK(output.size(1) == k / 2);
  int scales_k = k / BLOCK_SIZE;
  // 4 means the swizzle requirement by nvidia nvfp4.
  int padded_k = (scales_k + (4 - 1)) / 4 * 4;
  // 4 means 4 fp8 values are packed into one int32
  TORCH_CHECK(output_scale.size(1) * 4 == padded_k);

  auto in_dtype = input.dtype();
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream =
      at::cuda::getCurrentCUDAStream(input.get_device());
  if (in_dtype == at::ScalarType::Half) {
    quant_impl<half>(output.data_ptr(), output_scale.data_ptr(),
                     input.data_ptr(), input_global_scale.data_ptr(),
                     input_offset_by_experts.data_ptr(),
                     output_scale_offset_by_experts.data_ptr(), m_topk, k,
                     n_experts, stream);
  } else if (in_dtype == at::ScalarType::BFloat16) {
    quant_impl<__hip_bfloat16>(output.data_ptr(), output_scale.data_ptr(),
                              input.data_ptr(), input_global_scale.data_ptr(),
                              input_offset_by_experts.data_ptr(),
                              output_scale_offset_by_experts.data_ptr(), m_topk,
                              k, n_experts, stream);
  } else {
    TORCH_CHECK(false, "Expected input data type to be half or bfloat16");
  }
}