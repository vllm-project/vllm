#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cutlass_extensions/common.hpp"

#include "cutlass/cutlass.h"

#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/packed_stride.hpp"

#include "core/math.hpp"

using namespace cute;

#define CHECK_TYPE(x, st, m) \
  TORCH_CHECK(x.scalar_type() == st, ": Inconsistency of Tensor type:", m)
#define CHECK_TH_CUDA(x, m) \
  TORCH_CHECK(x.is_cuda(), m, ": must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, ": must be contiguous")
#define CHECK_INPUT(x, st, m) \
  CHECK_TH_CUDA(x, m);        \
  CHECK_CONTIGUOUS(x, m);     \
  CHECK_TYPE(x, st, m)

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;
constexpr auto SF_DTYPE = at::ScalarType::Float8_e4m3fn;

struct sm120_fp4_config_M256 {
  using ClusterShape = Shape<_1, _1, _1>;
  using MmaTileShape = Shape<_128, _128, _128>;
  using PerSmTileShape_MNK = Shape<_128, _128, _128>;
};

struct sm120_fp4_config_default {
  using ClusterShape = Shape<_1, _1, _1>;
  using MmaTileShape = Shape<_256, _128, _128>;
  using PerSmTileShape_MNK = Shape<_256, _128, _128>;
};

template <typename Config, typename OutType>
struct Fp4GemmSm120 {
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  using ElementD = OutType;
  using ElementC = OutType;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm120;
  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  using MmaTileShape = typename Config::MmaTileShape;
  using ClusterShape = typename Config::ClusterShape;
  using PerSmTileShape_MNK = typename Config::PerSmTileShape_MNK;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, PerSmTileShape_MNK, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, MmaTileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
};

template <typename Gemm>
typename Gemm::Arguments args_from_options(at::Tensor& D, at::Tensor const& A,
                                           at::Tensor const& B,
                                           at::Tensor const& A_sf,
                                           at::Tensor const& B_sf,
                                           torch::Tensor const& alpha, int M,
                                           int N, int K) {
  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementD = typename Gemm::ElementD;
  using ElementSFA = cutlass::float_ue4m3_t;
  using ElementSFB = cutlass::float_ue4m3_t;
  using ElementCompute = float;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using Sm1xxBlkScaledConfig =
      typename Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
  auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {M, N, 1});

  auto layout_SFA = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(
      cute::make_shape(M, N, K, 1));
  auto layout_SFB = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(
      cute::make_shape(M, N, K, 1));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K, 1},
      {static_cast<ElementA const*>(A.data_ptr()), stride_A,
       static_cast<ElementB const*>(B.data_ptr()), stride_B,
       static_cast<ElementSFA const*>(A_sf.data_ptr()), layout_SFA,
       static_cast<ElementSFB const*>(B_sf.data_ptr()), layout_SFB},
      {{},
       static_cast<ElementD const*>(D.data_ptr()),
       stride_D,
       static_cast<ElementD*>(D.data_ptr()),
       stride_D}};
  auto& fusion_args = arguments.epilogue.thread;
  fusion_args.alpha_ptr = static_cast<ElementCompute const*>(alpha.data_ptr());

  return arguments;
}

template <typename Gemm>
void runGemm(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
             at::Tensor const& A_sf, at::Tensor const& B_sf,
             torch::Tensor const& alpha, int M, int N, int K,
             hipStream_t stream) {
  Gemm gemm;

  auto arguments = args_from_options<Gemm>(D, A, B, A_sf, B_sf, alpha, M, N, K);

  size_t workspace_size = Gemm::get_workspace_size(arguments);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(A.device());
  auto workspace = torch::empty(workspace_size, workspace_options);

  CUTLASS_CHECK(gemm.can_implement(arguments));

  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));

  CUTLASS_CHECK(gemm.run(arguments, workspace.data_ptr(), stream));
}

void cutlass_fp4_bf16_gemm_dispatch(torch::Tensor& D, torch::Tensor const& A,
                                    torch::Tensor const& B,
                                    torch::Tensor const& A_sf,
                                    torch::Tensor const& B_sf,
                                    torch::Tensor const& alpha, int m, int n,
                                    int k, hipStream_t stream) {
  uint32_t const mp2 = std::max(static_cast<uint32_t>(16), next_pow_2(m));
  if (mp2 <= 256) {
    runGemm<Fp4GemmSm120<sm120_fp4_config_M256, cutlass::bfloat16_t>::Gemm>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else {
    runGemm<Fp4GemmSm120<sm120_fp4_config_default, cutlass::bfloat16_t>::Gemm>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  }
}

void cutlass_fp4_f16_gemm_dispatch(torch::Tensor& D, torch::Tensor const& A,
                                   torch::Tensor const& B,
                                   torch::Tensor const& A_sf,
                                   torch::Tensor const& B_sf,
                                   torch::Tensor const& alpha, int m, int n,
                                   int k, hipStream_t stream) {
  uint32_t const mp2 = std::max(static_cast<uint32_t>(16), next_pow_2(m));
  if (mp2 <= 256) {
    runGemm<Fp4GemmSm120<sm120_fp4_config_M256, cutlass::half_t>::Gemm>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else {
    runGemm<Fp4GemmSm120<sm120_fp4_config_default, cutlass::half_t>::Gemm>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  }
}

void cutlass_scaled_fp4_mm_sm120a(torch::Tensor& D, torch::Tensor const& A,
                                  torch::Tensor const& B,
                                  torch::Tensor const& A_sf,
                                  torch::Tensor const& B_sf,
                                  torch::Tensor const& alpha) {
#if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)
  CHECK_INPUT(A, FLOAT4_E2M1X2, "a");
  CHECK_INPUT(B, FLOAT4_E2M1X2, "b");

  CHECK_INPUT(A_sf, SF_DTYPE, "scale_a");
  CHECK_INPUT(B_sf, SF_DTYPE, "scale_b");

  CHECK_INPUT(alpha, at::ScalarType::Float, "alpha");

  TORCH_CHECK(A.dim() == 2, "a must be a matrix");
  TORCH_CHECK(B.dim() == 2, "b must be a matrix");
  TORCH_CHECK(A.sizes()[1] == B.sizes()[1],
              "a and b shapes cannot be multiplied (", A.sizes()[0], "x",
              A.sizes()[1], " and ", B.sizes()[0], "x", B.sizes()[1], ")");

  auto const m = A.sizes()[0];
  auto const n = B.sizes()[0];
  auto const k = A.sizes()[1] * 2;

  constexpr int alignment = 32;
  TORCH_CHECK(k % alignment == 0, "Expected k to be divisible by ", alignment,
              ", but got a shape: (", A.sizes()[0], "x", A.sizes()[1],
              "), k: ", k, ".");
  TORCH_CHECK(n % alignment == 0, "Expected n to be divisible by ", alignment,
              ", but got b shape: (", B.sizes()[0], "x", B.sizes()[1], ").");

  auto round_up = [](int x, int y) { return (x + y - 1) / y * y; };
  int rounded_m = round_up(m, 128);
  int rounded_n = round_up(n, 128);
  // Since k is divisible by 32 (alignment), k / 16 is guaranteed to be an
  // integer.
  int rounded_k = round_up(k / 16, 4);

  TORCH_CHECK(A_sf.dim() == 2, "scale_a must be a matrix");
  TORCH_CHECK(B_sf.dim() == 2, "scale_b must be a matrix");
  TORCH_CHECK(A_sf.sizes()[1] == B_sf.sizes()[1],
              "scale_a and scale_b shapes cannot be multiplied (",
              A_sf.sizes()[0], "x", A_sf.sizes()[1], " and ", B_sf.sizes()[0],
              "x", B_sf.sizes()[1], ")");
  TORCH_CHECK(A_sf.sizes()[0] == rounded_m && A_sf.sizes()[1] == rounded_k,
              "scale_a must be padded and swizzled to a shape (", rounded_m,
              "x", rounded_k, "), but got a shape (", A_sf.sizes()[0], "x",
              A_sf.sizes()[1], ")");
  TORCH_CHECK(B_sf.sizes()[0] == rounded_n && B_sf.sizes()[1] == rounded_k,
              "scale_b must be padded and swizzled to a shape (", rounded_n,
              "x", rounded_k, "), but got a shape (", B_sf.sizes()[0], "x",
              B_sf.sizes()[1], ")");

  auto out_dtype = D.dtype();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(A.get_device());

  if (out_dtype == at::ScalarType::BFloat16) {
    return cutlass_fp4_bf16_gemm_dispatch(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                          stream);
  } else if (out_dtype == at::ScalarType::Half) {
    return cutlass_fp4_f16_gemm_dispatch(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                         stream);
  } else {
    TORCH_CHECK(false, "Unsupported output data type of nvfp4 mm sm120 (",
                out_dtype, ")");
  }
#else
  TORCH_CHECK(false,
              "Unsupported CUTLASS version. Set VLLM_CUTLASS_SRC_DIR to "
              "a CUTLASS 3.8 source directory to enable support.");
#endif  // defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)
}