#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cutlass_extensions/common.hpp"

#include "cutlass/cutlass.h"

#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/packed_stride.hpp"

#include "core/math.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

// Configuration for M in (256, inf)
struct sm100_fp4_config_default {
  using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;
  using EpilogueSchedule = cutlass::epilogue::collective::EpilogueScheduleAuto;
  using TileShape = Shape<_256, _256, _256>;
  using ClusterShape = Shape<_2, _1, _1>;
  using PerSmTileShape_MNK = Shape<_128, _256, _256>;
};

// Configuration for M in (16, 256]
struct sm100_fp4_config_M256 {
  using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;
  using EpilogueSchedule = cutlass::epilogue::collective::EpilogueScheduleAuto;
  using TileShape = Shape<_256, _128, _256>;
  using ClusterShape = Shape<_2, _1, _1>;
  using PerSmTileShape_MNK = Shape<_128, _128, _256>;
};

// Configuration for M in [1, 16]
struct sm100_fp4_config_M16 {
  using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;
  using EpilogueSchedule = cutlass::epilogue::collective::EpilogueScheduleAuto;
  using TileShape = Shape<_128, _128, _256>;
  using ClusterShape = Shape<_1, _1, _1>;
  using PerSmTileShape_MNK = Shape<_128, _128, _256>;
};

template <typename Config, typename OutType>
struct Fp4GemmSm100 {
  // A matrix configuration
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  // B matrix configuration
  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  // C/D matrix configuration
  using ElementD = OutType;
  using ElementC = OutType;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

  // Kernel functional config
  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm100;
  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  // Use config's tile shapes
  using MmaTileShape = typename Config::TileShape;
  using ClusterShape = typename Config::ClusterShape;
  using PerSmTileShape_MNK = typename Config::PerSmTileShape_MNK;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, PerSmTileShape_MNK, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, MmaTileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using StrideA = typename Gemm::GemmKernel::StrideA;
  using LayoutA = decltype(cute::make_layout(make_shape(0, 0, 0), StrideA{}));
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using LayoutB = decltype(cute::make_layout(make_shape(0, 0, 0), StrideB{}));
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using LayoutC = decltype(cute::make_layout(make_shape(0, 0, 0), StrideC{}));
  using StrideD = typename Gemm::GemmKernel::StrideD;
  using LayoutD = decltype(cute::make_layout(make_shape(0, 0, 0), StrideD{}));
};

template <typename Config>
typename Config::Gemm::Arguments args_from_options(
    at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
    at::Tensor const& A_sf, at::Tensor const& B_sf, at::Tensor const& alpha,
    int64_t M, int64_t N, int64_t K) {
  using ElementA = typename Config::Gemm::ElementA;
  using ElementB = typename Config::Gemm::ElementB;
  using ElementSFA = cutlass::float_ue4m3_t;
  using ElementSFB = cutlass::float_ue4m3_t;
  using ElementD = typename Config::Gemm::ElementD;
  using ElementCompute = float;
  using StrideA = typename Config::StrideA;
  using StrideB = typename Config::StrideB;
  using StrideD = typename Config::StrideD;
  using Sm100BlkScaledConfig = typename Config::Gemm::GemmKernel::
      CollectiveMainloop::Sm1xxBlkScaledConfig;

  int m = static_cast<int>(M);
  int n = static_cast<int>(N);
  int k = static_cast<int>(K);
  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {m, k, 1});
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {n, k, 1});
  auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {m, n, 1});

  auto layout_SFA = Sm100BlkScaledConfig::tile_atom_to_shape_SFA(
      cute::make_shape(m, n, k, 1));
  auto layout_SFB = Sm100BlkScaledConfig::tile_atom_to_shape_SFB(
      cute::make_shape(m, n, k, 1));

  typename Config::Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      {// Mainloop arguments
       static_cast<ElementA const*>(A.data_ptr()), stride_A,
       static_cast<ElementB const*>(B.data_ptr()), stride_B,
       static_cast<ElementSFA const*>(A_sf.data_ptr()), layout_SFA,
       static_cast<ElementSFB const*>(B_sf.data_ptr()), layout_SFB},
      {     // Epilogue arguments
       {},  // epilogue.thread
       static_cast<ElementD const*>(D.data_ptr()),
       stride_D,
       static_cast<ElementD*>(D.data_ptr()),
       stride_D}};
  auto& fusion_args = arguments.epilogue.thread;
  fusion_args.alpha_ptr = static_cast<ElementCompute const*>(alpha.data_ptr());
  return arguments;
}

template <typename Config>
void runGemm(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
             at::Tensor const& A_sf, at::Tensor const& B_sf,
             at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
             hipStream_t stream) {
  typename Config::Gemm gemm;

  auto arguments =
      args_from_options<Config>(D, A, B, A_sf, B_sf, alpha, m, n, k);

  size_t workspace_size = Config::Gemm::get_workspace_size(arguments);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(A.device());
  auto workspace = torch::empty(workspace_size, workspace_options);

  CUTLASS_CHECK(gemm.can_implement(arguments));

  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));

  CUTLASS_CHECK(gemm.run(arguments, workspace.data_ptr(), stream));
}

// Dispatch function to select appropriate config based on M
template <typename OutType>
void cutlass_fp4_gemm_dispatch(torch::Tensor& D, torch::Tensor const& A,
                               torch::Tensor const& B,
                               torch::Tensor const& A_sf,
                               torch::Tensor const& B_sf,
                               torch::Tensor const& alpha, int64_t m, int64_t n,
                               int64_t k, hipStream_t stream) {
  uint32_t const mp2 = std::max(static_cast<uint32_t>(16), next_pow_2(m));

  if (mp2 <= 16) {
    // m in [1, 16]
    runGemm<Fp4GemmSm100<sm100_fp4_config_M16, OutType>>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else if (mp2 <= 256) {
    // m in (16, 256]
    runGemm<Fp4GemmSm100<sm100_fp4_config_M256, OutType>>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else {
    // m in (256, inf)
    runGemm<Fp4GemmSm100<sm100_fp4_config_default, OutType>>(
        D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  }
}

#else
template <typename OutType>
void cutlass_fp4_gemm_dispatch(torch::Tensor& D, torch::Tensor const& A,
                               torch::Tensor const& B,
                               torch::Tensor const& A_sf,
                               torch::Tensor const& B_sf,
                               torch::Tensor const& alpha, int64_t m, int64_t n,
                               int64_t k, hipStream_t stream) {
  TORCH_CHECK(false,
              "Unsupported CUTLASS version. Set VLLM_CUTLASS_SRC_DIR to "
              "a CUTLASS 3.8 source directory to enable support.");
}
#endif  // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

#define CHECK_TYPE(x, st, m) \
  TORCH_CHECK(x.scalar_type() == st, ": Inconsistency of Tensor type:", m)
#define CHECK_TH_CUDA(x, m) \
  TORCH_CHECK(x.is_cuda(), m, ": must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, ": must be contiguous")
#define CHECK_INPUT(x, st, m) \
  CHECK_TH_CUDA(x, m);        \
  CHECK_CONTIGUOUS(x, m);     \
  CHECK_TYPE(x, st, m)

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;
constexpr auto SF_DTYPE = at::ScalarType::Float8_e4m3fn;

void cutlass_scaled_fp4_mm_sm100a(torch::Tensor& D, torch::Tensor const& A,
                                  torch::Tensor const& B,
                                  torch::Tensor const& A_sf,
                                  torch::Tensor const& B_sf,
                                  torch::Tensor const& alpha) {
  CHECK_INPUT(A, FLOAT4_E2M1X2, "a");
  CHECK_INPUT(B, FLOAT4_E2M1X2, "b");

  CHECK_INPUT(A_sf, SF_DTYPE, "scale_a");
  CHECK_INPUT(B_sf, SF_DTYPE, "scale_b");

  CHECK_INPUT(alpha, at::ScalarType::Float, "alpha");

  TORCH_CHECK(A.dim() == 2, "a must be a matrix");
  TORCH_CHECK(B.dim() == 2, "b must be a matrix");
  TORCH_CHECK(A.sizes()[1] == B.sizes()[1],
              "a and b shapes cannot be multiplied (", A.sizes()[0], "x",
              A.sizes()[1], " and ", B.sizes()[0], "x", B.sizes()[1], ")");

  auto const m = A.sizes()[0];
  auto const n = B.sizes()[0];
  auto const k = A.sizes()[1] * 2;

  constexpr int alignment = 32;
  TORCH_CHECK(k % alignment == 0, "Expected k to be divisible by ", alignment,
              ", but got a shape: (", A.sizes()[0], "x", A.sizes()[1],
              "), k: ", k, ".");
  TORCH_CHECK(n % alignment == 0, "Expected n to be divisible by ", alignment,
              ", but got b shape: (", B.sizes()[0], "x", B.sizes()[1], ").");

  auto round_up = [](int x, int y) { return (x + y - 1) / y * y; };
  int rounded_m = round_up(m, 128);
  int rounded_n = round_up(n, 128);
  // Since k is divisible by 32 (alignment), k / 16 is guaranteed to be an
  // integer.
  int rounded_k = round_up(k / 16, 4);

  TORCH_CHECK(A_sf.dim() == 2, "scale_a must be a matrix");
  TORCH_CHECK(B_sf.dim() == 2, "scale_b must be a matrix");
  TORCH_CHECK(A_sf.sizes()[1] == B_sf.sizes()[1],
              "scale_a and scale_b shapes cannot be multiplied (",
              A_sf.sizes()[0], "x", A_sf.sizes()[1], " and ", B_sf.sizes()[0],
              "x", B_sf.sizes()[1], ")");
  TORCH_CHECK(A_sf.sizes()[0] == rounded_m && A_sf.sizes()[1] == rounded_k,
              "scale_a must be padded and swizzled to a shape (", rounded_m,
              "x", rounded_k, "), but got a shape (", A_sf.sizes()[0], "x",
              A_sf.sizes()[1], ")");
  TORCH_CHECK(B_sf.sizes()[0] == rounded_n && B_sf.sizes()[1] == rounded_k,
              "scale_b must be padded and swizzled to a shape (", rounded_n,
              "x", rounded_k, "), but got a shape (", B_sf.sizes()[0], "x",
              B_sf.sizes()[1], ")");

  auto out_dtype = D.dtype();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(A.get_device());

  if (out_dtype == at::ScalarType::Half) {
    cutlass_fp4_gemm_dispatch<cutlass::half_t>(D, A, B, A_sf, B_sf, alpha, m, n,
                                               k, stream);
  } else if (out_dtype == at::ScalarType::BFloat16) {
    cutlass_fp4_gemm_dispatch<cutlass::bfloat16_t>(D, A, B, A_sf, B_sf, alpha,
                                                   m, n, k, stream);
  } else {
    TORCH_CHECK(false, "Unsupported output data type of nvfp4 mm (", out_dtype,
                ")");
  }
}
