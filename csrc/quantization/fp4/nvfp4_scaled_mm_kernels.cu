/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cutlass_extensions/common.hpp"

#include "cutlass/cutlass.h"

#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/packed_stride.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
// Kernel Perf config
template <typename T>
struct KernelTraits;

template <>
struct KernelTraits<float> {
  using MmaTileShape = Shape<_128, _128, _256>;
  using ClusterShape = Shape<_1, _1, _1>;
  using PerSmTileShape_MNK = Shape<_128, _128, _256>;
};

template <>
struct KernelTraits<cutlass::half_t> {
  using MmaTileShape = Shape<_256, _256, _256>;
  using ClusterShape = Shape<_4, _4, _1>;
  using PerSmTileShape_MNK = Shape<_128, _256, _256>;
};

template <>
struct KernelTraits<cutlass::bfloat16_t> {
  using MmaTileShape = Shape<_256, _256, _256>;
  using ClusterShape = Shape<_4, _4, _1>;
  using PerSmTileShape_MNK = Shape<_128, _256, _256>;
};

template <typename T>
struct Fp4GemmSm100 {
  // A matrix configuration
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  // B matrix configuration
  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  // C/D matrix configuration
  using ElementD = T;
  using ElementC = T;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  // Kernel functional config
  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm100;
  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  // Kernel Perf config
  using MmaTileShape = typename KernelTraits<T>::MmaTileShape;
  using ClusterShape = typename KernelTraits<T>::ClusterShape;
  using PerSmTileShape_MNK = typename KernelTraits<T>::PerSmTileShape_MNK;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, PerSmTileShape_MNK, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, MmaTileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using StrideA = typename Gemm::GemmKernel::StrideA;
  using LayoutA = decltype(cute::make_layout(make_shape(0, 0, 0), StrideA{}));
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using LayoutB = decltype(cute::make_layout(make_shape(0, 0, 0), StrideB{}));
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using LayoutC = decltype(cute::make_layout(make_shape(0, 0, 0), StrideC{}));
  using StrideD = typename Gemm::GemmKernel::StrideD;
  using LayoutD = decltype(cute::make_layout(make_shape(0, 0, 0), StrideD{}));
};

template <typename T>
typename T::Gemm::Arguments args_from_options(
    at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
    at::Tensor const& A_sf, at::Tensor const& B_sf, at::Tensor const& alpha,
    int64_t M, int64_t N, int64_t K) {
  using ElementA = typename T::Gemm::ElementA;
  using ElementB = typename T::Gemm::ElementB;
  using ElementSFA = cutlass::float_ue4m3_t;
  using ElementSFB = cutlass::float_ue4m3_t;
  using ElementD = typename T::Gemm::ElementD;
  using ElementCompute = float;
  using StrideA = typename T::StrideA;
  using StrideB = typename T::StrideB;
  using StrideD = typename T::StrideD;
  using Sm100BlkScaledConfig =
      typename T::Gemm::GemmKernel::CollectiveMainloop::Sm100BlkScaledConfig;

  int m = static_cast<int>(M);
  int n = static_cast<int>(N);
  int k = static_cast<int>(K);
  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {m, k, 1});
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {n, k, 1});
  auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {m, n, 1});

  auto layout_SFA = Sm100BlkScaledConfig::tile_atom_to_shape_SFA(
      cute::make_shape(m, n, k, 1));
  auto layout_SFB = Sm100BlkScaledConfig::tile_atom_to_shape_SFB(
      cute::make_shape(m, n, k, 1));

  typename T::Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      {// Mainloop arguments
       static_cast<ElementA const*>(A.data_ptr()), stride_A,
       static_cast<ElementB const*>(B.data_ptr()), stride_B,
       static_cast<ElementSFA const*>(A_sf.data_ptr()), layout_SFA,
       static_cast<ElementSFB const*>(B_sf.data_ptr()), layout_SFB},
      {     // Epilogue arguments
       {},  // epilogue.thread
       static_cast<ElementD const*>(D.data_ptr()),
       stride_D,
       static_cast<ElementD*>(D.data_ptr()),
       stride_D}};
  auto& fusion_args = arguments.epilogue.thread;
  fusion_args.alpha_ptr = static_cast<ElementCompute const*>(alpha.data_ptr());
  return arguments;
}

template <typename T>
void runGemm(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
             at::Tensor const& A_sf, at::Tensor const& B_sf,
             at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
             hipStream_t stream) {
  typename Fp4GemmSm100<T>::Gemm gemm;

  auto arguments =
      args_from_options<Fp4GemmSm100<T>>(D, A, B, A_sf, B_sf, alpha, m, n, k);

  size_t workspace_size = Fp4GemmSm100<T>::Gemm::get_workspace_size(arguments);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(A.device());
  auto workspace = torch::empty(workspace_size, workspace_options);

  CUTLASS_CHECK(gemm.can_implement(arguments));

  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));

  CUTLASS_CHECK(gemm.run(arguments, workspace.data_ptr(), stream));
}
#else
template <typename T>
void runGemm(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
             at::Tensor const& A_sf, at::Tensor const& B_sf,
             at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
             hipStream_t stream) {
  TORCH_CHECK(false,
              "Unsupported CUTLASS version. Set VLLM_CUTLASS_SRC_DIR to "
              "a CUTLASS 3.8 source directory to enable support.");
}
#endif  // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

#define CHECK_TYPE(x, st, m) \
  TORCH_CHECK(x.scalar_type() == st, "Inconsistency of Tensor type:", m)
#define CHECK_TH_CUDA(x, m) TORCH_CHECK(x.is_cuda(), m, "must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, "must be contiguous")
#define CHECK_INPUT(x, st, m) \
  CHECK_TH_CUDA(x, m);        \
  CHECK_CONTIGUOUS(x, m);     \
  CHECK_TYPE(x, st, m)

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;
constexpr auto SF_DTYPE = at::ScalarType::Float8_e4m3fn;

void cutlass_scaled_fp4_mm_sm100a(torch::Tensor& D, torch::Tensor const& A,
                                  torch::Tensor const& B,
                                  torch::Tensor const& A_sf,
                                  torch::Tensor const& B_sf,
                                  torch::Tensor const& alpha) {
  CHECK_INPUT(A, FLOAT4_E2M1X2, "a");
  CHECK_INPUT(B, FLOAT4_E2M1X2, "b");

  CHECK_INPUT(A_sf, SF_DTYPE, "scale_a");
  CHECK_INPUT(B_sf, SF_DTYPE, "scale_b");

  CHECK_INPUT(alpha, at::ScalarType::Float, "alpha");

  TORCH_CHECK(A.dim() == 2, "a must be a matrix");
  TORCH_CHECK(B.dim() == 2, "b must be a matrix");
  TORCH_CHECK(A.sizes()[1] == B.sizes()[1],
              "a and b shapes cannot be multiplied (", A.sizes()[0], "x",
              A.sizes()[1], " and ", B.sizes()[0], "x", B.sizes()[1], ")");

  auto const m = A.sizes()[0];
  auto const n = B.sizes()[0];
  auto const k = A.sizes()[1] * 2;

  constexpr int alignment = 32;
  TORCH_CHECK(k % alignment == 0, "Expected k to be divisible by ", alignment,
              ", but got a shape: (", A.sizes()[0], "x", A.sizes()[1],
              "), k: ", k, ".");
  TORCH_CHECK(n % alignment == 0, "Expected n to be divisible by ", alignment,
              ", but got b shape: (", B.sizes()[0], "x", B.sizes()[1], ").");

  auto round_up = [](int x, int y) { return (x + y - 1) / y * y; };
  int rounded_m = round_up(m, 128);
  int rounded_n = round_up(n, 128);
  // Since k is divisible by 32 (alignment), k / 16 is guaranteed to be an
  // integer.
  int rounded_k = round_up(k / 16, 4);

  TORCH_CHECK(A_sf.dim() == 2, "scale_a must be a matrix");
  TORCH_CHECK(B_sf.dim() == 2, "scale_b must be a matrix");
  TORCH_CHECK(A_sf.sizes()[1] == B_sf.sizes()[1],
              "scale_a and scale_b shapes cannot be multiplied (",
              A_sf.sizes()[0], "x", A_sf.sizes()[1], " and ", B_sf.sizes()[0],
              "x", B_sf.sizes()[1], ")");
  TORCH_CHECK(A_sf.sizes()[0] == rounded_m && A_sf.sizes()[1] == rounded_k,
              "scale_a must be padded and swizzled to a shape (", rounded_m,
              "x", rounded_k, "), but got a shape (", A_sf.sizes()[0], "x",
              A_sf.sizes()[1], ")");
  TORCH_CHECK(B_sf.sizes()[0] == rounded_n && B_sf.sizes()[1] == rounded_k,
              "scale_b must be padded and swizzled to a shape (", rounded_n,
              "x", rounded_k, "), but got a shape (", B_sf.sizes()[0], "x",
              B_sf.sizes()[1], ")");

  auto out_dtype = D.dtype();
  at::cuda::CUDAGuard device_guard{(char)A.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(A.get_device());

  if (out_dtype == at::ScalarType::Half) {
    runGemm<cutlass::half_t>(D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else if (out_dtype == at::ScalarType::BFloat16) {
    runGemm<cutlass::bfloat16_t>(D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else if (out_dtype == at::ScalarType::Float) {
    runGemm<float>(D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
  } else {
    TORCH_CHECK(false, "Unsupported output data type of nvfp4 mm");
  }
}
