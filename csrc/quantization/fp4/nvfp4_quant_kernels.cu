#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_fp8.h>

#include "cuda_utils.h"

// Get type2 from type or vice versa (applied to half and bfloat16)
template <typename T>
struct TypeConverter {
  using Type = half2;
};  // keep for generality

template <>
struct TypeConverter<half2> {
  using Type = half;
};

template <>
struct TypeConverter<half> {
  using Type = half2;
};

template <>
struct TypeConverter<__hip_bfloat162> {
  using Type = __hip_bfloat16;
};

template <>
struct TypeConverter<__hip_bfloat16> {
  using Type = __hip_bfloat162;
};

#define ELTS_PER_THREAD 8

constexpr int CVT_FP4_ELTS_PER_THREAD = 8;
constexpr int CVT_FP4_SF_VEC_SIZE = 16;

// Convert 8 float32 values into 8 e2m1 values (represented as one uint32_t).
inline __device__ uint32_t fp32_vec_to_e2m1(float (&array)[8]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{\n"
      ".reg .b8 byte0;\n"
      ".reg .b8 byte1;\n"
      ".reg .b8 byte2;\n"
      ".reg .b8 byte3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte0, %2, %1;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte1, %4, %3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte2, %6, %5;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte3, %8, %7;\n"
      "mov.b32 %0, {byte0, byte1, byte2, byte3};\n"
      "}"
      : "=r"(val)
      : "f"(array[0]), "f"(array[1]), "f"(array[2]), "f"(array[3]),
        "f"(array[4]), "f"(array[5]), "f"(array[6]), "f"(array[7]));
  return val;
#else
  return 0;
#endif
}

// Convert 4 float2 values into 8 e2m1 values (represented as one uint32_t).
inline __device__ uint32_t fp32_vec_to_e2m1(float2 (&array)[4]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{\n"
      ".reg .b8 byte0;\n"
      ".reg .b8 byte1;\n"
      ".reg .b8 byte2;\n"
      ".reg .b8 byte3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte0, %2, %1;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte1, %4, %3;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte2, %6, %5;\n"
      "cvt.rn.satfinite.e2m1x2.f32   byte3, %8, %7;\n"
      "mov.b32 %0, {byte0, byte1, byte2, byte3};\n"
      "}"
      : "=r"(val)
      : "f"(array[0].x), "f"(array[0].y), "f"(array[1].x), "f"(array[1].y),
        "f"(array[2].x), "f"(array[2].y), "f"(array[3].x), "f"(array[3].y));
  return val;
#else
  return 0;
#endif
}

// Fast reciprocal.
inline __device__ float reciprocal_approximate_ftz(float a) {
  float b;
  asm volatile("rcp.approx.ftz.f32 %0, %1;\n" : "=f"(b) : "f"(a));
  return b;
}

template <class SFType, int CVT_FP4_NUM_THREADS_PER_SF>
__device__ uint8_t* cvt_quant_to_fp4_get_sf_out_offset(int rowIdx, int colIdx,
                                                       int numCols,
                                                       SFType* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  static_assert(CVT_FP4_NUM_THREADS_PER_SF == 1 ||
                CVT_FP4_NUM_THREADS_PER_SF == 2);

  // One pair of threads write one SF to global memory.
  // TODO: stage through smem for packed STG.32
  // is it better than STG.8 from 4 threads ?
  if (threadIdx.x % CVT_FP4_NUM_THREADS_PER_SF == 0) {
    // SF vector index (16 elements share one SF in the K dimension).
    int32_t kIdx = colIdx / CVT_FP4_NUM_THREADS_PER_SF;
    int32_t mIdx = rowIdx;

    // SF layout [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
    // --> index [mTileIdx, kTileIdx, outerMIdx, innerMIdx, innerKIdx]

    int32_t mTileIdx = mIdx / (32 * 4);
    // SF vector size 16.
    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    int32_t numKTiles = (numCols + factor - 1) / factor;
    int64_t mTileStride = numKTiles * 32 * 4 * 4;

    int32_t kTileIdx = (kIdx / 4);
    int64_t kTileStride = 32 * 4 * 4;

    // M tile layout [32, 4] is column-major.
    int32_t outerMIdx = (mIdx % 32);
    int64_t outerMStride = 4 * 4;

    int32_t innerMIdx = (mIdx % (32 * 4)) / 32;
    int64_t innerMStride = 4;

    int32_t innerKIdx = (kIdx % 4);
    int64_t innerKStride = 1;

    // Compute the global offset.
    int64_t SFOffset = mTileIdx * mTileStride + kTileIdx * kTileStride +
                       outerMIdx * outerMStride + innerMIdx * innerMStride +
                       innerKIdx * innerKStride;

    return reinterpret_cast<uint8_t*>(SFout) + SFOffset;
  }
#endif
  return nullptr;
}

// Define a 16 bytes packed data type.
template <class Type>
struct PackedVec {
  typename TypeConverter<Type>::Type elts[4];
};

template <>
struct PackedVec<__hip_fp8_e4m3_fnuz> {
  __hip_fp8x2_e4m3_fnuz elts[8];
};

// Quantizes the provided PackedVec into the uint32_t output
template <class Type, bool UE8M0_SF = false>
__device__ uint32_t cvt_warp_fp16_to_fp4(PackedVec<Type>& vec, float SFScaleVal,
                                         uint8_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

  // Local maximum value.
  #pragma unroll
  for (int i = 1; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 16 values (two threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e2m1).
  // maximum value of e2m1 = 6.0.
  // TODO: use half as compute data type.
  float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  if constexpr (UE8M0_SF) {
    // Extract the 8 exponent bits from float32.
    // float 32bits = 1 sign bit + 8 exponent bits + 23 mantissa bits.
    uint32_t tmp = reinterpret_cast<uint32_t&>(SFValue) >> 23;
    fp8SFVal = tmp & 0xff;
    // Convert back to fp32.
    reinterpret_cast<uint32_t&>(SFValue) = tmp << 23;
  } else {
    // Here SFValue is always positive, so E4M3 is the same as UE4M3.
    __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(SFValue);
    reinterpret_cast<__hip_fp8_e4m3_fnuz&>(fp8SFVal) = tmp;
    // Convert back to fp32.
    SFValue = float(tmp);
  }
  // Get the output scale.
  // Recipe: final_scale = reciprocal(fp32(fp8(SFValue * SFScaleVal))) *
  //                       reciprocal(SFScaleVal))
  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(
                         SFValue * reciprocal_approximate_ftz(SFScaleVal))
                   : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP4_ELTS_PER_THREAD / 2];

  #pragma unroll
  for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e2m1 values.
  uint32_t e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

  // Write the e2m1 values to global memory.
  return e2m1Vec;
#else
  return 0;
#endif
}

// Use UE4M3 by default.
template <class Type, bool UE8M0_SF = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(512, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows, int32_t numCols, Type const* in, float const* SFScale,
    uint32_t* out, uint32_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF =
      (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD,
                "Vec size is not matched.");

  // Get the global scaling factor, which will be applied to the SF.
  // Note SFScale is the same as next GEMM's alpha, which is
  // (448.f / (Alpha_A / 6.f)).
  float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[0];

  // Input tensor row/col loops.
  for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x) {
    for (int colIdx = threadIdx.x; colIdx < numCols / CVT_FP4_ELTS_PER_THREAD;
         colIdx += blockDim.x) {
      int64_t inOffset = rowIdx * (numCols / CVT_FP4_ELTS_PER_THREAD) + colIdx;
      PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
      // Get the output tensor offset.
      // Same as inOffset because 8 elements are packed into one uint32_t.
      int64_t outOffset = inOffset;
      auto& out_pos = out[outOffset];

      auto sf_out =
          cvt_quant_to_fp4_get_sf_out_offset<uint32_t,
                                             CVT_FP4_NUM_THREADS_PER_SF>(
              rowIdx, colIdx, numCols, SFout);

      out_pos =
          cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
    }
  }
#endif
}

template <typename T>
void invokeFP4Quantization(int m, int n, T const* input, float const* SFScale,
                           int64_t* output, int32_t* SFOuput, bool useUE8M0,
                           int multiProcessorCount, hipStream_t stream) {
  // Grid, Block size.
  // Each thread converts 8 values.
  dim3 block(std::min(int(n / ELTS_PER_THREAD), 512));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 2048 / block.x;
  dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

  // Launch the cvt kernel.
  if (useUE8M0) {
    cvt_fp16_to_fp4<T, true><<<grid, block, 0, stream>>>(
        m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
        reinterpret_cast<uint32_t*>(SFOuput));
  } else {
    cvt_fp16_to_fp4<T, false><<<grid, block, 0, stream>>>(
        m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
        reinterpret_cast<uint32_t*>(SFOuput));
  }
}

// Instantiate the function.
template void invokeFP4Quantization(int m, int n, half const* input,
                                    float const* SFScale, int64_t* output,
                                    int32_t* SFOuput, bool useUE8M0,
                                    int multiProcessorCount,
                                    hipStream_t stream);

template void invokeFP4Quantization(int m, int n, __hip_bfloat16 const* input,
                                    float const* SFScale, int64_t* output,
                                    int32_t* SFOuput, bool useUE8M0,
                                    int multiProcessorCount,
                                    hipStream_t stream);

void scaled_fp4_quant_sm100a(torch::Tensor const& output,
                             torch::Tensor const& input,
                             torch::Tensor const& output_sf,
                             torch::Tensor const& input_sf) {
  int32_t m = input.size(0);
  int32_t n = input.size(1);

  TORCH_CHECK(n % 16 == 0, "The N dimension must be multiple of 16.");

  int multiProcessorCount =
      get_device_attribute(hipDeviceAttributeMultiprocessorCount, -1);

  auto input_sf_ptr = static_cast<float const*>(input_sf.data_ptr());
  auto sf_out = static_cast<int32_t*>(output_sf.data_ptr());
  auto output_ptr = static_cast<int64_t*>(output.data_ptr());
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  auto stream = at::cuda::getStreamFromPool(false, input.get_device());
  if (stream == nullptr) {
    std::cerr << "Warning: Null CUDA stream" << std::endl;
  }

  // We don't support e8m0 scales at this moment.
  bool useUE8M0 = false;

  switch (input.scalar_type()) {
    case torch::kHalf: {
      auto input_ptr = reinterpret_cast<half const*>(input.data_ptr());
      invokeFP4Quantization(m, n, input_ptr, input_sf_ptr, output_ptr, sf_out,
                            useUE8M0, multiProcessorCount, stream);
      break;
    }
    case torch::kBFloat16: {
      auto input_ptr = reinterpret_cast<__hip_bfloat16 const*>(input.data_ptr());
      invokeFP4Quantization(m, n, input_ptr, input_sf_ptr, output_ptr, sf_out,
                            useUE8M0, multiProcessorCount, stream);
      break;
    }
    default: {
      std::cerr << "Observing: " << input.scalar_type()
                << " for the input datatype which is invalid";
      throw std::runtime_error(
          "Unsupported input data type for quantize_to_fp4.");
    }
  }
}
