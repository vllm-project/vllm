#include "hip/hip_runtime.h"
/*
Adapted from https://github.com/mit-han-lab/llm-awq
@article{lin2023awq,
  title={AWQ: Activation-aware Weight Quantization for LLM Compression and Acceleration},
  author={Lin, Ji and Tang, Jiaming and Tang, Haotian and Yang, Shang and Dang, Xingyu and Han, Song},
  journal={arXiv},
  year={2023}
}
 */


#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include "dequantize.cuh"

#include <hip/hip_fp16.h>

namespace vllm {
namespace awq {

// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

__global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n128k32(int G, int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 750
  assert(false);
#else
  static constexpr uint32_t ZERO = 0x0;
  float C_warp[32];
  __shared__ half A_shared[16 * (32 + 8)];
  __shared__ half B_shared[32 * (128 + 8)];
  
  __shared__ half scaling_factors_shared[128];
  __shared__ half zeros_shared[128];

  int j_factors1 = ((OC + 128 - 1) / 128);
  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
  int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

  half A_shared_warp[8];
  half B_shared_warp[32];
  for (int j_0_4_init = 0; j_0_4_init < 4; ++j_0_4_init) {
    for (int i = 0; i < 8; ++i) {
      C_warp[(j_0_4_init * 8) + i] = 0.0;
    }
  }

  static constexpr int row_stride_warp = 32 * 8 / 32;
  static constexpr int row_stride = 2 * 32 * 8 / 128;
  bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 128;
  // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
  bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
  // bool wb_C_flag = (threadIdx.x / 4) < M;

  half* A_ptr = A 
                + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
                + (((int)threadIdx.x) % (32 / 8)) * 8;
  
  int* B_ptr = B
            + ((int)threadIdx.y) * (OC / 8) * 2
            + (((int)threadIdx.x) / (128 / 8)) * (OC / 8)
            + (((int)blockIdx_y) % j_factors1) * (128 / 8)
            + (((int)threadIdx.x) % (128 / 8)) * 1;
// Why * 1 in the above line?
                        
  half* A_shared_ptr = A_shared 
                    + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
                    + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
                    + (((int)threadIdx.x) % (32 / 8) ) * 8;

  half* B_shared_ptr = B_shared
                    + ((int)threadIdx.y) * (row_stride / 2) * (128 + 8)
                    + (((int)threadIdx.x) / (128 / 8)) * (128 + 8)
                    + (((int)threadIdx.x) % (128 / 8)) * 8;
  
  int* zeros_ptr = zeros
                + (((int)blockIdx_y) % j_factors1) * (128 / 8)
                + ((int)threadIdx.x) % (128 / 8);
  
  half* scaling_factors_ptr = scaling_factors
                            + (((int)blockIdx_y) % j_factors1) * (128) 
                            + (((int)threadIdx.x) % (128 / 8)) * 8;

  half* C_ptr = C 
              + static_cast<long long>(blockIdx_z) * M * OC        // blockIdz.x -> split_k dim
              + (((int)blockIdx_y) % j_factors1) * 128
              + ((int)threadIdx.y) * 64
              + (((int)threadIdx.x) % 4) * 2;

  // preload s.f. and zeros
  int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
  if ((k_bound - 1) * split_k_iters * 32 + blockIdx_z * 32 >= IC) k_bound -= 1;
  for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
    int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
    __syncthreads();
    // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
    if (ld_A_flag)
    {
      *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
    }
    else
    {
      *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
    }

    // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
    uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / G * (OC / 8));
    uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
    uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / G * (OC));
    /*
    if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
      printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
    }
    */
    // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
    int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 8; ++ax0_ax1_fused_0) {

      // B: 32 x 136 (128+8) float16
      // each warp: 32 x 4
      // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
      // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
      // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
      uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
      uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
      //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

      // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
      // - zero and * scale
      // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
      /*
      if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
        printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
      }
      */

      // write back
      *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (128 + 8)) = B_loaded_fp16;
    }
    __syncthreads();

    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      {
        unsigned int addr;
        __asm__ __volatile__(
          "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
          : "=r"(addr)
          : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
        );


        __asm__ __volatile__(
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
          "{%0, %1, %2, %3}, [%4];\n"
          : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
          : "r"(addr)
        );
      }

      for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
        {
          unsigned int addr;
          __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)((&(B_shared[(((k_0_1 * 2176) + (((int)threadIdx.y) * 64)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 136) + ((((int)threadIdx.x) >> 4) * 8))))
          );
          __asm__ __volatile__(
            "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
            "{%0, %1, %2, %3}, [%4];\n"
            : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
            : "r"(addr)
          );
        }
      }
      for (int j_0_4 = 0; j_0_4 < 4; ++j_0_4) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ == 750
        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }
#else
        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }

#endif
      }
    }
  }

// TODO: Shang: Hoist loop invariance.
  for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {
    for (int local_id = 0; local_id < 8; ++local_id) {
      int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
      if (row_offset < M)
      {
        *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
      }
    }
  }
#endif
}


__global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n64k32(int G, int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 750
  assert(false);
#else
  static constexpr uint32_t ZERO = 0x0;
  float C_warp[32];
  __shared__ half A_shared[16 * (32 + 8)];
  __shared__ half B_shared[32 * (64 + 8)];
  
  __shared__ half scaling_factors_shared[64];
  __shared__ half zeros_shared[64];

  int j_factors1 = ((OC + 64 - 1) / 64);

  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
  int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

  half A_shared_warp[8];
  half B_shared_warp[16];
  for (int j_0_4_init = 0; j_0_4_init < 2; ++j_0_4_init) {
    for (int i = 0; i < 8; ++i) {
      C_warp[(j_0_4_init * 8) + i] = 0.0;
    }
  }

  static constexpr int row_stride_warp = 32 * 8 / 32;
  static constexpr int row_stride = 2 * 32 * 8 / 64;
  bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 64;
  // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
  bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
  // bool wb_C_flag = (threadIdx.x / 4) < M;

  half* A_ptr = A 
                + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
                + (((int)threadIdx.x) % (32 / 8)) * 8;
  
  int* B_ptr = B
            + ((int)threadIdx.y) * (OC / 8) * 4
            + (((int)threadIdx.x) / (64 / 8)) * (OC / 8)
            + (((int)blockIdx_y) % j_factors1) * (64 / 8)
            + (((int)threadIdx.x) % (64 / 8)) * 1;
// Why * 1 in the above line?
                        
  half* A_shared_ptr = A_shared 
                    + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
                    + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
                    + (((int)threadIdx.x) % (32 / 8) ) * 8;

  half* B_shared_ptr = B_shared
                    + ((int)threadIdx.y) * (row_stride / 2) * (64 + 8)
                    + (((int)threadIdx.x) / (64 / 8)) * (64 + 8)
                    + (((int)threadIdx.x) % (64 / 8)) * 8;
  
  int* zeros_ptr = zeros
                + (((int)blockIdx_y) % j_factors1) * (64 / 8)
                + ((int)threadIdx.x) % (64 / 8);
  
  half* scaling_factors_ptr = scaling_factors
                            + (((int)blockIdx_y) % j_factors1) * (64) 
                            + (((int)threadIdx.x) % (64 / 8)) * 8;

  half* C_ptr = C 
              + static_cast<long long>(blockIdx_z) * M * OC        // blockIdz.x -> split_k dim
              + (((int)blockIdx_y) % j_factors1) * 64
              + ((int)threadIdx.y) * 32
              + (((int)threadIdx.x) % 4) * 2;

  // preload s.f. and zeros
  int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
  if ((k_bound - 1) * split_k_iters * 32 + blockIdx_z * 32 >= IC) k_bound -= 1;
  for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
    int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
    __syncthreads();
    // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
    if (ld_A_flag)
    {
      *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
    }
    else
    {
      *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
    }

    // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
    uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / G * (OC / 8));
    uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
    uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / G * (OC));
    /*
    if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
      printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
    }
    */
    // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
    int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 4; ++ax0_ax1_fused_0) {

      // B: 32 x 136 (128+8) float16
      // each warp: 32 x 4
      // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
      // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
      // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
      uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
      uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
      //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

      // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
      // - zero and * scale
      // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
      /*
      if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
        printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
      }
      */

      // write back
      *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (64 + 8)) = B_loaded_fp16;
    }
    __syncthreads();

    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) 
    {
      {
        unsigned int addr;
        __asm__ __volatile__(
          "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
          : "=r"(addr)
          : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
        );
        __asm__ __volatile__(
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
          "{%0, %1, %2, %3}, [%4];\n"
          : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
          : "r"(addr)
        );
      }
        

      for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) 
      {
        {
          unsigned int addr;
          __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)((&(B_shared[(((k_0_1 * 1152) + (((int)threadIdx.y) * 32)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 72) + ((((int)threadIdx.x) >> 4) * 8))))
          );
          __asm__ __volatile__(
            "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
            "{%0, %1, %2, %3}, [%4];\n"
            : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
            : "r"(addr)
          );
        }
      }
      
      for (int j_0_4 = 0; j_0_4 < 2; ++j_0_4) 
      {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ == 750
        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }
#else
        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }
#endif
      }
    }
  }

// TODO: Shang: Hoist loop invariance.
  for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
    for (int local_id = 0; local_id < 8; ++local_id) {
      int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
      if (row_offset < M)
      {
        *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
      }
    }
  }
#endif
}

} // namespace awq
} // namespace vllm

// in_feats: M, IC [float16]
// kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// scaling_factors: IC // G, OC [float16]
// zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// assume that batch_size < 16 for now

torch::Tensor awq_gemm(
    torch::Tensor _in_feats,
    torch::Tensor _kernel,
    torch::Tensor _scaling_factors,
    torch::Tensor _zeros,
    int split_k_iters)
{
    int num_in_feats = _in_feats.size(0);
    int num_in_channels = _in_feats.size(1);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(_in_feats));

    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    at::Tensor _out_feats = torch::empty({split_k_iters, num_in_feats, _kernel.size(1) * 8}, options);
    int num_out_feats = _out_feats.size(-2);
    int num_out_channels = _out_feats.size(-1);

    auto in_feats = reinterpret_cast<half*>(_in_feats.data_ptr<at::Half>());
    auto kernel = reinterpret_cast<int*>(_kernel.data_ptr<int>());
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());
    auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<at::Half>());
    auto zeros = reinterpret_cast<int*>(_zeros.data_ptr<int>());
    int group_size = num_in_channels / _scaling_factors.size(0);

    if (num_out_channels % 64 != 0)
        throw std::invalid_argument("OC is not multiple of cta_N = 64");
    if (num_out_channels % 8 != 0)
        throw std::invalid_argument("OC is not multiple of pack_num = 8");
    if (group_size % 32 != 0)
	      throw std::invalid_argument("Group size should be a multiple of 32");
    if (num_out_channels % group_size != 0)
        throw std::invalid_argument("OC is not multiple of Group size");

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    if (num_out_channels % 128 == 0)
    {
        int j_factors1 = num_out_channels / 128 / 1;
        dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
        // threadIdx.x: 32
        // threadIdx.y: i_factors[2] * j_factors[2]
        dim3 threads_per_block(32, 2);
        vllm::awq::gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block, 0, stream>>>(
            group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
    }
    else if (num_out_channels % 64 == 0)
    {
        int j_factors1 = num_out_channels / 64 / 1;
        dim3 num_blocks(1 * (num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
    
        // threadIdx.x: 32
        // threadIdx.y: i_factors[2] * j_factors[2]
        dim3 threads_per_block(32, 2);
        vllm::awq::gemm_forward_4bit_cuda_m16n64k32<<<num_blocks, threads_per_block, 0, stream>>>(
            group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
    }
    return _out_feats.sum(0);
}
