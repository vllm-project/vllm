#include "hip/hip_runtime.h"
/*
 * Adapted from
 * https://github.com/IST-DASLab/marlin/blob/master/marlin/marlin_cuda_kernel.cu
 * https://github.com/IST-DASLab/marlin/blob/master/marlin/marlin_cuda.cpp
 * Modified by HandH1998
 * Copyright (C) 2024 HandH1998
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "../dense/common/base.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  #include "../dense/common/mem.h"
#endif

template <typename T>
inline std::string str(T x) {
  return std::to_string(x);
}

namespace {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800

using I4 = Vec<int, 4>;
// Matrix fragments for tensor core instructions; their precise layout is
// documented here:
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-integer-type
using FragA = Vec<uint32_t, 2>;
using FragB = Vec<uint32_t, 1>;
using FragC = Vec<int, 4>;
using FragS_GROUP = Vec<half2, 1>;  // weight per-group quantization scales
using FragS_CHANNEL =
    Vec<float, 2>;  // weight per-channel quantization scales or activaton
                    // per-token quantization scales

// NOTE(HandH1998): cp.async.cg only support BYTES = 16, however,
// cp.async.ca can support BYTES = 4, 8, 16;
// as s_tok's shape is equal to prob_m, we need set s_tok to float type,
// and cp_size = 1 float, i.e., 4 BYTES
// Asynchronous global->shared copy for activation quantizaton scales s_tok
__device__ inline void cp_async1(void* smem_ptr, const void* glob_ptr) {
  const int BYTES = 4;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
      "{\n"
      "   cp.async.ca.shared.global [%0], [%1], %2;\n"
      "}\n" ::"r"(smem),
      "l"(glob_ptr), "n"(BYTES));
}

// m16n8k16 tensor core mma instruction with int8 inputs and int32
// output/accumulation.
__device__ inline void mma(const FragA& a_frag, const FragB& frag_b,
                           FragC& frag_c) {
  const uint32_t* a = reinterpret_cast<const uint32_t*>(&a_frag);
  const uint32_t* b = reinterpret_cast<const uint32_t*>(&frag_b);
  int* c = reinterpret_cast<int*>(&frag_c);
  asm volatile(
      "mma.sync.aligned.m16n8k16.row.col.satfinite.s32.s8.s8.s32 "
      "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=r"(c[0]), "=r"(c[1]), "=r"(c[2]), "=r"(c[3])
      : "r"(a[0]), "r"(a[1]), "r"(b[0]), "r"(c[0]), "r"(c[1]), "r"(c[2]),
        "r"(c[3]));
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared
// memory, directly in int8 tensor core layout.
__device__ inline void ldsm4(FragA& frag_a, const void* smem_ptr) {
  uint32_t* a = reinterpret_cast<uint32_t*>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0,%1}, [%2];\n"
               : "=r"(a[0]), "=r"(a[1])
               : "r"(smem));
}

inline __device__ half2 float2_to_half2(float2 f) {
  uint32_t res;
  // NOTE(HandH1998): h0,h1 should be uint16_t, not half
  uint16_t h0, h1;
  asm volatile("cvt.rn.f16.f32 %0, %1;\n" : "=h"(h0) : "f"(f.x));
  asm volatile("cvt.rn.f16.f32 %0, %1;\n" : "=h"(h1) : "f"(f.y));
  asm volatile("mov.b32 %0, {%1, %2};\n" : "=r"(res) : "h"(h0), "h"(h1));
  return reinterpret_cast<half2&>(res);
}

inline __device__ float int32_to_float(int h) {
  float res;
  asm volatile("cvt.rn.f32.s32 %0, %1;\n" : "=f"(res) : "r"(h));
  return res;
}

// Lookup-table based 3-input logical operation; explicitly used for
// dequantization as the compiler does not seem to automatically recognize it in
// all cases.
template <int lut>
__device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(res)
               : "r"(a), "r"(b), "r"(c), "n"(lut));
  return res;
}

// Efficiently dequantize an int32 value into a full B-fragment of 4 int8 values
// for weight per channel dequant.
__device__ inline FragB dequant_per_channel(int q) {
  static constexpr int MASK = 0xf0f0f0f0;
  FragB frag_b;
  frag_b[0] = (q & MASK);
  return frag_b;
}

// Efficiently dequantize an int32 value into a full B-fragment of 4 int8 values
// for weight per group dequant.
__device__ inline FragB dequant_per_group(int q, FragS_GROUP& frag_s, int i) {
  static constexpr uint32_t LO = 0x000f000f;
  static constexpr uint32_t HI = 0x00f000f0;
  static constexpr uint32_t EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  uint32_t t0 = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  uint32_t t1 = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  // We want signed int4 outputs, hence we fuse the `-8` symmetric zero point
  // directly into `SUB` and `ADD`.
  static constexpr uint32_t SUB = 0x64086408;
  static constexpr uint32_t MUL = 0x2c002c00;
  static constexpr uint32_t ADD = 0xd480d480;
  *reinterpret_cast<half2*>(&t0) = __hsub2(
      *reinterpret_cast<half2*>(&t0), *reinterpret_cast<const half2*>(&SUB));
  *reinterpret_cast<half2*>(&t1) = __hfma2(
      *reinterpret_cast<half2*>(&t1), *reinterpret_cast<const half2*>(&MUL),
      *reinterpret_cast<const half2*>(&ADD));

  uint16_t s = reinterpret_cast<uint16_t*>(&frag_s)[i];
  uint32_t double_s;
  // pack 2xfp16 to half2
  asm volatile("mov.b32 %0, {%1, %2};\n" : "=r"(double_s) : "h"(s), "h"(s));
  // dequant and convert 4 half to 4 uint8 (be placed at the low 8 bits of 4
  // half, respectively)
  static constexpr uint32_t MAGIC_NUM = 0x64806480;
  *reinterpret_cast<half2*>(&t0) = __hfma2(
      *reinterpret_cast<half2*>(&t0), *reinterpret_cast<half2*>(&double_s),
      *reinterpret_cast<const half2*>(&MAGIC_NUM));
  *reinterpret_cast<half2*>(&t1) = __hfma2(
      *reinterpret_cast<half2*>(&t1), *reinterpret_cast<half2*>(&double_s),
      *reinterpret_cast<const half2*>(&MAGIC_NUM));
  // take out the 4 uint8 from 4 half, then convert them to 4 int8 and pack 4
  // int8 into 1 uint32
  FragB frag_b;
  uint32_t uint8s;
  static constexpr uint32_t MASK_0246 = 0x6420;
  static constexpr uint32_t UINT8s_TO_INT8s_MASK = 0x80808080;
  asm volatile("prmt.b32 %0,%1,%2,%3;\n"
               : "=r"(uint8s)
               : "r"(t0), "r"(t1), "n"(MASK_0246));
  frag_b[0] = (uint8s ^ UINT8s_TO_INT8s_MASK);
  return frag_b;
}

template <const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1  // number of consecutive 16x16 blocks
                                       // with a separate quantization scale
          >
__global__ void Marlin(
    const int4* __restrict__ A,  // int8 input matrix of shape mxk
    const int4* __restrict__ B,  // 4bit quantized weight matrix of shape kxn
    int4* __restrict__ C,        // int32 global_reduce buffer of shape
                           // (max_par*16*4)xn, as int8 tensor core's output is
                           // int32 dtype
    int4* __restrict__ D,              // fp16 output buffer of shape mxn
    const float* __restrict__ s_tok,   // fp32 activation per-token quantization
                                       // scales of shape mx1
    const int4* __restrict__ s_ch,     // fp32 weight per-channel quantization
                                       // scales of shape 1xn
    const int4* __restrict__ s_group,  // fp16 weight per-group quantization
                                       // scales of shape (k/groupsize)xn, when
                                       // group_blocks=-1, it should be nullptr
    int prob_m,                        // batch dimension m
    int prob_n,                        // output dimension n
    int prob_k,                        // reduction dimension k
    int* locks  // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the
  // same size, which might involve multiple column "slices" (of width 16 *
  // `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM
  // example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it
  // ensures good utilization of all SMs for many kinds of shape and GPU
  // configurations, while requiring as few slow global cross-threadblock
  // reductions as possible.

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a
  // better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);
  // Ensure that the number of tiles in each stripe is a multiple of the
  // groupsize; this avoids an annoying special case where a stripe starts in
  // the middle of group.
  if constexpr (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) *
            ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters;  // number of threadblock tiles in the current slice
  int slice_count =
      0;          // total number of active threadblocks in the current slice
  int slice_idx;  // index of threadblock in current slice; numbered bottom to
                  // top

  // We can easily implement parallel problem execution by just remapping
  // indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 16;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 4;
    D += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    s_tok += (slice_col_par / n_tiles) * 16 * thread_m_blocks;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for
  // synchronization.
  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel) slice_iters = 0;
    if (slice_iters == 0) return;
    if (slice_row + slice_iters > k_tiles) slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0) slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0) slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 16;
      C += 16 * thread_m_blocks * prob_n / 4;
      D += 16 * thread_m_blocks * prob_n / 8;
      s_tok += 16 * thread_m_blocks;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 16;  // stride of the A matrix in global memory
  // We typically use `constexpr` to indicate that this value is a compile-time
  // constant
  constexpr int a_sh_stride =
      16 * thread_k_blocks / 16;  // stride of an A matrix tile in shared memory
  constexpr int a_gl_rd_delta_o =
      16 * thread_k_blocks /
      16;  // delta between subsequent A tiles in global memory
  int a_gl_rd_delta_i =
      a_gl_stride *
      (threads / a_gl_rd_delta_o);  // between subsequent accesses within a tile
  constexpr int a_sh_wr_delta =
      a_sh_stride *
      (threads / a_gl_rd_delta_o);  // between shared memory writes
  constexpr int a_sh_rd_delta_o =
      1 * ((threads / 32) /
           (thread_n_blocks / 4));  // between shared memory tile reads
  constexpr int a_sh_rd_delta_i =
      a_sh_stride * 16;  // within a shared memory tile
  constexpr int a_sh_stage =
      a_sh_stride * (16 * thread_m_blocks);  // overall size of a tile
  constexpr int a_sh_wr_iters =
      ceildiv(a_sh_stage,
              a_sh_wr_delta);  // number of shared write iterations for a tile

  int b_gl_stride = 16 * prob_n / 32;
  constexpr int b_sh_stride = 32 * thread_n_blocks / 4;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride);
  constexpr int b_sh_wr_delta = threads;
  constexpr int b_sh_rd_delta = threads;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  constexpr int s_tok_sh_stride = 16 * thread_m_blocks;

  constexpr int s_ch_sh_stride = 16 * thread_n_blocks / 4;

  int s_group_gl_stride = prob_n / 8;
  constexpr int s_group_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_group_sh_stage = s_group_sh_stride;
  int s_group_gl_rd_delta = s_group_gl_stride;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  // NOTE(HandH1998): int8 input a only need 16 threads to load 16x16 matrix
  int a_sh_rd = a_sh_stride * ((threadIdx.x % 32) % 16);
  a_sh_rd += 1 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd =
      b_gl_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x;
  int b_sh_rd = threadIdx.x;

  int s_tok_gl_rd = threadIdx.x;
  // NOTE(HandH1998): activation scale s_tok need shuffle to [0, 8, 1, 9, 2, 10,
  // 3, 11, 4, 12, 5, 13, 6, 14, 7, 15] for example, 0, 8 row scales serve for
  // thread 0, 1, 2, 3. For more details, refer to mma operand A layout as
  // s_tok's size is not fixed, we can not shuffle before inference we shuffle
  // it when fetching s_tok from global memory to shared memory, that's why
  // s_tok_sh_wr is like this
  int s_tok_sh_wr =
      (threadIdx.x / 16) * 16 + (threadIdx.x % 8) * 2 + (threadIdx.x % 16) / 8;
  int s_tok_sh_rd = (threadIdx.x % 32) / 4;
  bool s_tok_sh_wr_pred = threadIdx.x < prob_m;

  int s_ch_gl_rd = s_ch_sh_stride * slice_col + threadIdx.x;
  int s_ch_sh_wr = threadIdx.x;
  int s_ch_sh_rd = 16 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
                   2 * ((threadIdx.x % 32) % 4);
  bool s_ch_sh_wr_pred = threadIdx.x < s_ch_sh_stride;

  int s_group_gl_rd, s_group_sh_wr, s_group_sh_rd;
  bool s_group_sh_wr_pred;
  if constexpr (group_blocks != -1) {
    s_group_gl_rd =
        s_group_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
        s_group_sh_stride * slice_col + threadIdx.x;
    s_group_sh_wr = threadIdx.x;
    // NOTE(HandH1998): s_group_sh_rd is related to mma output C
    s_group_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
                    (threadIdx.x % 32) / 4;
    s_group_sh_wr_pred = threadIdx.x < s_group_sh_stride;
  }

  // Precompute which thread should not read memory in which iterations; this is
  // needed if there are more threads than required for a certain tilesize or
  // when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;

  // To ensure that writing and reading A tiles to/from shared memory, the
  // latter in fragment format, is fully bank conflict free, we need to use a
  // rather fancy XOR-based layout. The key here is that neither reads nor
  // writes of the 16-byte `int4` blocks of 8 consecutive threads involve the
  // same shared memory banks. Further, it seems (based on NSight-Compute) that
  // each warp must also write a consecutive memory segment?
  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main
  // loop unrolls, all shared memory accesses are static, we simply precompute
  // both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at
  // runtime; we break dependencies between subsequent accesses with a tile by
  // maintining multiple pointers (we have enough registers), a tiny
  // optimization.
  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  // NOTE(HandH1998): stages need >= 4, otherwise, sh_s_tok = sh + max(stages *
  // a_sh_stage + stages * b_sh_stage, 4 * stages * a_sh_stage)
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_s_tok = sh_b + (stages * b_sh_stage);
  int4* sh_s_ch = sh_s_tok + s_tok_sh_stride;
  int4* sh_s_group = sh_s_ch + s_ch_sh_stride;

  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS_GROUP frag_s_group[2][4];
  FragS_CHANNEL frag_s_tok[thread_m_blocks];
  FragS_CHANNEL frag_s_ch[2][4];

  // Zero accumulators.
  auto zero_accums = [&]() {
  #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<int*>(frag_c)[i] = 0;
  };

  // Asynchronously fetch the next A, B and s tile from global to the next
  // shared memory pipeline location.
  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
        cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr], B_ptr[i]);
        B_ptr[i] += b_gl_rd_delta_o;
      }
      // Only fetch scales if this tile starts a new group
      if constexpr (group_blocks != -1) {
        if (pipe % (group_blocks / thread_k_blocks) == 0) {
          int4* sh_s_group_stage = sh_s_group + s_group_sh_stage * pipe;
          if (s_group_sh_wr_pred)
            cp_async4(&sh_s_group_stage[s_group_sh_wr],
                      &s_group[s_group_gl_rd]);
          s_group_gl_rd += s_group_gl_rd_delta;
        }
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that
    // waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe
  // into the current register buffer.
  auto fetch_to_registers = [&](int k, int pipe) {
    // It may seem inefficient that we reload the groups for every sub-tile;
    // however, this does not seem to be a significant bottleneck, while some
    // theoretically better attempts have lead to bad instruction ordering by
    // the compiler and correspondingly a noticeable drop in performance.
    if constexpr (group_blocks != -1) {
      int4* sh_s_group_stage =
          sh_s_group +
          s_group_sh_stage * ((group_blocks / thread_k_blocks) *
                              (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4*>(&frag_s_group[k % 2])[0] =
          sh_s_group_stage[s_group_sh_rd];
    }
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4* sh_b_stage = sh_b + b_sh_stage * pipe;
    frag_b_quant[k % 2] = *reinterpret_cast<I4*>(
        &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd]);
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&](int k) {
  // We have the m dimension as the inner loop in order to encourage overlapping
  // dequantization and matmul operations.
  #pragma unroll
    for (int j = 0; j < 4; j++) {
      int b_quant = frag_b_quant[k % 2][j];
      // int b_quant_shift = b_quant << 4;
      FragB frag_b0, frag_b1;
      // If there are no groups, we can just scale the final output once and can
      // avoid doing so for each weight.
      if constexpr (group_blocks != -1) {
        int b_quant_shift = b_quant >> 8;
        frag_b0 = dequant_per_group(b_quant, frag_s_group[k % 2][j], 0);
        frag_b1 = dequant_per_group(b_quant_shift, frag_s_group[k % 2][j], 1);
      } else {
        int b_quant_shift = b_quant << 4;
        frag_b0 = dequant_per_channel(b_quant);
        frag_b1 = dequant_per_channel(b_quant_shift);
      }
  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the
  // number of warps while keeping the n dimension of a tile reasonable, we have
  // multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride;
      constexpr int red_sh_stride = b_sh_stride * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride) +
                      (threadIdx.x % b_sh_stride);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any
      // unnecessary read or write iterations, e.g., for two warps we write only
      // once by warp 1 and read only once by warp 0.

  #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
  #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
  #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                int* c_rd =
                    reinterpret_cast<int*>(&sh[red_sh_delta * j + red_sh_rd]);
                int* c_wr = reinterpret_cast<int*>(&sh[red_sh_wr]);
  #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] =
                  reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
  #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            int* c_rd =
                reinterpret_cast<int*>(&sh[red_sh_delta * i + red_sh_rd]);
  #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we
  // finally have to globally reduce over the results. As the striped
  // partitioning minimizes the number of such reductions and our outputs are
  // usually rather small, we perform this reduction serially in L2 cache.
  // global_reduce works on INT32 elements, which are the results of INT8 GEMM.
  // This is why we need another INT32 maxtrix `C` to reduce instead of the
  // original half matrix `D`.
  auto global_reduce = [&](bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to
    // maximize L2 cache utilization in this step. To do this, we write out
    // results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 4;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 8 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) +
                    8 * (threadIdx.x / 32) + (threadIdx.x % 4) * 2;
      c_gl_wr += (4 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads * 2;
      int c_sh_wr = 2 * threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
  // Interestingly, doing direct global accesses here really seems to mess up
  // the compiler and lead to slowdowns, hence we also use async-copies even
  // though these fetches are not actually asynchronous.
  #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
              &sh[c_sh_wr + c_sh_wr_delta * i],
              &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                 c_gl_wr_delta_i * (i % 2)],
              i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m);
          cp_async4_pred(
              &sh[c_sh_wr + c_sh_wr_delta * i + 1],
              &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                 c_gl_wr_delta_i * (i % 2) + 1],
              i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 d_red1 = sh[c_sh_wr + i * c_sh_wr_delta];
            int4 d_red2 = sh[c_sh_wr + i * c_sh_wr_delta + 1];
  #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] +=
                  reinterpret_cast<int*>(&d_red1)[j];
            }
  #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * (j + 4) + (i % 4)] +=
                  reinterpret_cast<int*>(&d_red2)[j];
            }
          }
          if (!last) {
            int4 d1, d2;
  #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(&d1)[j] = reinterpret_cast<int*>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)];
            }
  #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(&d2)[j] = reinterpret_cast<int*>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * (j + 4) + (i % 4)];
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                d1;
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2) +
              1] = d2;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually
  // reshuffle matrix fragments in this step, the reduction above is performed
  // in fragment layout.
  auto write_result = [&]() {
    int d_gl_stride = prob_n / 8;
    constexpr int d_sh_stride = 2 * thread_n_blocks + 1;
    int d_gl_wr_delta = d_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int d_sh_rd_delta =
        d_sh_stride * (threads / (2 * thread_n_blocks));

    int d_gl_wr = d_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    d_gl_wr += (2 * thread_n_blocks) * slice_col;
    int d_sh_wr =
        (4 * d_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    d_sh_wr += 32 * (threadIdx.x / 32);
    int d_sh_rd = d_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));

    int d_gl_wr_end = d_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final
    // global write patterns
    auto write = [&](int idx, int c0, int c1, float a_s, FragS_CHANNEL& w_s) {
      float2 deq_res;
      deq_res.x = int32_to_float(c0) * w_s[0] * a_s;
      deq_res.y = int32_to_float(c1) * w_s[1] * a_s;
      ((half2*)sh)[idx] = float2_to_half2(deq_res);
    };

    if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
  #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = d_sh_wr + 8 * j;
          write(wr + (4 * d_sh_stride) * 0 + 0, frag_c[i][j][0][0],
                frag_c[i][j][0][1], frag_s_tok[i][0],
                frag_s_ch[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * d_sh_stride) * 8 + 0, frag_c[i][j][0][2],
                frag_c[i][j][0][3], frag_s_tok[i][1],
                frag_s_ch[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * d_sh_stride) * 0 + 4, frag_c[i][j][1][0],
                frag_c[i][j][1][1], frag_s_tok[i][0],
                frag_s_ch[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * d_sh_stride) * 8 + 4, frag_c[i][j][1][2],
                frag_c[i][j][1][3], frag_s_tok[i][1],
                frag_s_ch[j / 2][2 * (j % 2) + 1]);
        }
        d_sh_wr += 16 * (4 * d_sh_stride);
      }
    }
    __syncthreads();

  #pragma unroll
    for (int i = 0;
         i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (d_gl_wr < d_gl_wr_end) {
        D[d_gl_wr] = sh[d_sh_rd];
        d_gl_wr += d_gl_wr_delta;
        d_sh_rd += d_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&]() {
  #pragma unroll
    for (int i = 0; i < stages - 1; i++) fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  // Main loop.
  while (slice_iters) {
  // We unroll over both the global fetch and the register load pipeline to
  // ensure all shared memory accesses are static. Note that both pipelines have
  // even length meaning that the next iteration will always start at index 0.
  #pragma unroll
    for (int pipe = 0; pipe < stages;) {
  #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe,
                          slice_iters >= stages);
          pipe++;
          wait_for_stage();
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0) break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    // Process results and, if necessary, proceed to the next column slice.
    // While this pattern may not be the most readable, other ways of writing
    // the loop seemed to noticeably worse performance after compilation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before
      // write-out
      if (last) {
        if (s_tok_sh_wr_pred) {
          cp_async1(&sh_s_tok[s_tok_sh_wr], &s_tok[s_tok_gl_rd]);
        }
        if (s_ch_sh_wr_pred) {
          cp_async4(&sh_s_ch[s_ch_sh_wr], &s_ch[s_ch_gl_rd]);
        }
        cp_async_fence();
      }
      thread_block_reduce();
      if (last) {
        cp_async_wait<0>();
        __syncthreads();
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
          for (int i = 0; i < thread_m_blocks; i++) {
            frag_s_tok[i][0] =
                *reinterpret_cast<float*>(&sh_s_tok[16 * i + 2 * s_tok_sh_rd]);
            frag_s_tok[i][1] = *reinterpret_cast<float*>(
                &sh_s_tok[16 * i + 2 * s_tok_sh_rd + 1]);
          }
          reinterpret_cast<int4*>(&frag_s_ch)[0] = sh_s_ch[s_ch_sh_rd + 0];
          reinterpret_cast<int4*>(&frag_s_ch)[1] = sh_s_ch[s_ch_sh_rd + 1];
          reinterpret_cast<int4*>(&frag_s_ch)[2] = sh_s_ch[s_ch_sh_rd + 8];
          reinterpret_cast<int4*>(&frag_s_ch)[3] = sh_s_ch[s_ch_sh_rd + 9];
        }
      }
      if (slice_count > 1) {  // only globally reduce if there is more than one
                              // block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last)  // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
  #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
  #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++) B_ptr[i] -= b_gl_stride;
        }
        s_group_gl_rd = s_group_sh_stride * slice_col + threadIdx.x;
        s_ch_gl_rd = s_ch_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}

#else

template <const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1  // number of consecutive 16x16 blocks
                                       // with a separate quantization scale
          >
__global__ void Marlin(
    const int4* __restrict__ A,  // int8 input matrix of shape mxk
    const int4* __restrict__ B,  // 4bit quantized weight matrix of shape kxn
    int4* __restrict__ C,        // int32 global_reduce buffer of shape
                           // (max_par*16*4)xn, as int8 tensor core's output is
                           // int32 dtype
    int4* __restrict__ D,              // fp16 output buffer of shape mxn
    const float* __restrict__ s_tok,   // fp32 activation per-token quantization
                                       // scales of shape mx1
    const int4* __restrict__ s_ch,     // fp32 weight per-channel quantization
                                       // scales of shape 1xn
    const int4* __restrict__ s_group,  // fp16 weight per-group quantization
                                       // scales of shape (k/groupsize)xn, when
                                       // group_blocks=-1, it should be nullptr
    int prob_m,                        // batch dimension m
    int prob_n,                        // output dimension n
    int prob_k,                        // reduction dimension k
    int* locks  // extra global storage for barrier synchronization
) {
  // Marlin is not implemented yet for SM < 8.0
  assert(false);
  return;
}

#endif

// 8 warps are a good choice since every SM has 4 schedulers and having more
// than 1 warp per schedule allows some more latency hiding. At the same time,
// we want relatively few warps to have many registers per warp and small tiles.
const int USER_THREADS =
    256;               // Note: This is only used with user-provided thread_k/n
const int STAGES = 4;  // 4 pipeline stages fit into shared memory

static constexpr int min_thread_n = 64;
static constexpr int min_thread_k = 64;

static constexpr int tile_size = 16;
static constexpr int max_par = 16;

static constexpr int pack_factor_4bit =
    8;  // We have 8 4-bit vals inside a 32 bit

#define __CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,           \
                  GROUP_BLOCKS, NUM_THREADS)                                   \
  else if (thread_m_blocks == THREAD_M_BLOCKS &&                               \
           thread_n_blocks == THREAD_N_BLOCKS &&                               \
           thread_k_blocks == THREAD_K_BLOCKS &&                               \
           group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {       \
    hipFuncSetAttribute(reinterpret_cast<const void*>(Marlin<NUM_THREADS), THREAD_M_BLOCKS, THREAD_N_BLOCKS, \
                                THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>,        \
                         hipFuncAttributeMaxDynamicSharedMemorySize,          \
                         max_shared_mem);                                      \
    Marlin<NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,     \
           STAGES, GROUP_BLOCKS>                                               \
        <<<blocks, NUM_THREADS, max_shared_mem, stream>>>(                     \
            A_ptr, B_ptr, C_ptr, D_ptr, s_tok_ptr, s_ch_ptr, s_group_ptr,      \
            prob_m, prob_n, prob_k, locks);                                    \
  }

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},  // Default
    {128, 64, 128},   // Reduce N 2X, same K
    {64, 256, 256},   // Reduce K 2X, increase N 2X
    {64, 128, 128},   // Reduce K 2X, same N
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},   // Default
    {128, 128, 256},  // Reduce N 2X, increase K 2X
    {64, 128, 128},   // Reduce N 2X, same K
    {128, 64, 128},   // Reduce N 4X, increase K 2X
};

bool is_valid_config(thread_config_t const& th_config, int prob_m, int prob_n,
                     int prob_k) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // thread_k can be only 128 or 64 (because it must be less than groupsize
  // which is 128)
  if (th_config.thread_k != 128 && th_config.thread_k != 64) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  return true;
}

thread_config_t determine_thread_config(int prob_m, int prob_n, int prob_k) {
  if (prob_m <= 16) {
    for (auto th_config : small_batch_thread_configs) {
      if (is_valid_config(th_config, prob_m, prob_n, prob_k)) {
        return th_config;
      }
    }

  } else {
    for (auto th_config : large_batch_thread_configs) {
      if (is_valid_config(th_config, prob_m, prob_n, prob_k)) {
        return th_config;
      }
    }
  }

  return thread_config_t{-1, -1, -1};
}

#define CALL_IF(N_BLOCKS, K_BLOCKS, NUM_THREADS)    \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)

void marlin_qqq_cuda(const void* A, const void* B, void* C, void* D,
                     void* s_tok, void* s_ch, void* s_group, int prob_m,
                     int prob_n, int prob_k, void* workspace,
                     int groupsize = -1, int dev = 0, hipStream_t stream = 0,
                     int thread_k = -1, int thread_n = -1, int sms = -1,
                     int max_par = 16) {
  int tot_m = prob_m;
  int tot_m_blocks = ceildiv(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  // Set thread config
  thread_config_t th_config;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    th_config = thread_config_t{thread_k, thread_n, USER_THREADS};
  } else {
    // Auto config
    th_config = determine_thread_config(prob_m, prob_n, prob_k);
  }

  if (!is_valid_config(th_config, prob_m, prob_n, prob_k)) {
    throw std::runtime_error(
        "Invalid thread config: thread_k = " + str(th_config.thread_k) +
        ", thread_n = " + str(th_config.thread_n) +
        ", num_threads = " + str(th_config.num_threads) + " for MKN = [" +
        str(prob_m) + ", " + str(prob_k) + ", " + str(prob_n) + "]");
  }

  int num_threads = th_config.num_threads;
  thread_k = th_config.thread_k;
  thread_n = th_config.thread_n;

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  int blocks = sms;

  if (prob_m == 0 || prob_n == 0 || prob_k == 0) {
    return;
  }

  TORCH_CHECK(prob_n % thread_n == 0, "prob_n = ", prob_n,
              " is not divisible by thread_n = ", thread_n);
  TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
              " is not divisible by thread_k = ", thread_k);
  if (group_blocks != -1) {
    TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                " is not divisible by group_blocks = ", group_blocks);
  }

  const int4* A_ptr = (const int4*)A;
  const int4* B_ptr = (const int4*)B;
  int4* C_ptr = (int4*)C;
  int4* D_ptr = (int4*)D;
  const float* s_tok_ptr = (const float*)s_tok;
  const int4* s_ch_ptr = (const int4*)s_ch;
  const int4* s_group_ptr = (const int4*)s_group;

  int* locks = (int*)workspace;

  for (int i = 0; i < tot_m_blocks; i += 4) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > 4) {
      // Note that parallel > 1 currently only works for inputs without any
      // padding
      par = (16 * thread_m_blocks - pad) / 64;
      if (par > max_par) par = max_par;
      prob_m = 64 * par;
      i += 4 * (par - 1);
      thread_m_blocks = 4;
    }

    // For compilation speed, we only define the kernel configurations that have
    // seemed useful (in terms of performance) in our testing, however many more
    // are, in principle, possible.
    if (false) {
    }
    CALL_IF(8, 8, 256)
    CALL_IF(16, 4, 256)
    CALL_IF(8, 4, 128)
    CALL_IF(4, 8, 128)
    else {
      throw std::runtime_error("Unsupported shapes: MKN = [" + str(prob_m) +
                               ", " + str(prob_k) + ", " + str(prob_n) + "]" +
                               ", groupsize = " + str(groupsize) +
                               ", thread_m_blocks = " + str(thread_m_blocks) +
                               ", thread_n_blocks = " + str(thread_n_blocks) +
                               ", thread_k_blocks = " + str(thread_k_blocks));
    }

    A_ptr += 16 * thread_m_blocks * (prob_k / 16) * par;
    D_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
    s_tok_ptr += 16 * thread_m_blocks * par;
  }
}
}  // anonymous namespace

torch::Tensor marlin_qqq_gemm(torch::Tensor const& a,
                              torch::Tensor const& b_q_weight,
                              torch::Tensor const& s_tok,
                              torch::Tensor const& s_ch,
                              torch::Tensor const& s_group,
                              torch::Tensor& workspace, int64_t size_m,
                              int64_t size_n, int64_t size_k) {
  // Verify M
  TORCH_CHECK(size_m == a.size(0),
              "Shape mismatch: a.size(0) = " + str(a.size(0)) +
                  ", size_m = " + str(size_m));
  TORCH_CHECK(size_m == s_tok.numel(),
              "Shape mismatch: s_tok.numel() = " + str(s_tok.numel()) +
                  ", size_m = " + str(size_m));

  // Verify K
  TORCH_CHECK(size_k == a.size(1),
              "Shape mismatch: a.size(1) = " + str(a.size(1)) +
                  ", size_k = " + str(size_k));
  TORCH_CHECK(size_k % tile_size == 0,
              "size_k = " + str(size_k) +
                  " is not divisible by tile_size = " + str(tile_size));
  TORCH_CHECK(
      (size_k / tile_size) == b_q_weight.size(0),
      "Shape mismatch: b_q_weight.size(0) = " + str(b_q_weight.size(0)) +
          ", size_k = " + str(size_k) + ", tile_size = " + str(tile_size));

  int groupsize = (s_group.numel() == 0) ? -1 : size_k / s_group.size(0);
  // Verify groupsize
  TORCH_CHECK(groupsize == -1 || groupsize == 128,
              "Unexpected groupsize = " + str(groupsize));

  // Verify N
  TORCH_CHECK(s_ch.numel() == size_n,
              "Shape mismatch: s_ch.numel() = " + str(s_ch.numel()) +
                  ", size_n = " + str(size_n));
  TORCH_CHECK(b_q_weight.size(1) % tile_size == 0,
              "b_q_weight.size(1) = " + str(b_q_weight.size(1)) +
                  " is not divisible by tile_size = " + str(tile_size));
  if (groupsize != -1) {
    TORCH_CHECK(s_group.size(1) == size_n,
                "Shape mismatch: s_group.size(1) = " + str(s_group.size(1)) +
                    ", size_n = " + str(size_n));
    TORCH_CHECK(
        size_k % s_group.size(0) == 0,
        "size_k = " + str(size_k) +
            ", is not divisible by s_group.size(0) = " + str(s_group.size(0)));
  }

  int actual_size_n = (b_q_weight.size(1) / tile_size) * pack_factor_4bit;
  TORCH_CHECK(size_n == actual_size_n,
              "Shape mismatch: size_n = " + str(size_n) +
                  ", actual_size_n = " + str(actual_size_n));

  // Verify A device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  // Verify B device and strides
  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  // Verify s_tok device, strides and dtype
  TORCH_CHECK(s_tok.device().is_cuda(), "s_tok is not on GPU");
  TORCH_CHECK(s_tok.is_contiguous(), "s_tok is not contiguous");
  TORCH_CHECK(s_tok.dtype() == torch::kFloat32, "s_tok's dtype is not float32");

  // Verify s_ch device, strides and dtype
  TORCH_CHECK(s_ch.device().is_cuda(), "s_ch is not on GPU");
  TORCH_CHECK(s_ch.is_contiguous(), "s_ch is not contiguous");
  TORCH_CHECK(s_ch.dtype() == torch::kFloat32, "s_ch's dtype is not float32");

  // Verify s_group device, strides and dtype
  TORCH_CHECK(s_group.device().is_cuda(), "s_group is not on GPU");
  TORCH_CHECK(s_group.is_contiguous(), "s_group is not contiguous");
  TORCH_CHECK(s_group.dtype() == torch::kFloat16,
              "s_group's dtype is not float16");

  // Verify workspace size
  TORCH_CHECK(size_n % min_thread_n == 0,
              "size_n = " + str(size_n) +
                  ", is not divisible by min_thread_n = " + str(min_thread_n));
  int min_workspace_size = (size_n / min_thread_n) * max_par;
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = " + str(workspace.numel()) +
                  " is below min_workspace_size = " + str(min_workspace_size));

  // Alloc C matrix
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options_c = torch::TensorOptions().dtype(torch::kInt).device(a.device());
  torch::Tensor c = torch::empty({max_par * 64, size_n}, options_c);

  // Alloc D matrix
  auto options_d =
      torch::TensorOptions().dtype(torch::kFloat16).device(a.device());
  torch::Tensor d = torch::empty({size_m, size_n}, options_d);

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel (can usually be left as auto -1)
  int sms = -1;

  int dev = a.get_device();
  marlin_qqq_cuda(
      a.data_ptr(), b_q_weight.data_ptr(), c.data_ptr(), d.data_ptr(),
      s_tok.data_ptr(), s_ch.data_ptr(), s_group.data_ptr(), size_m, size_n,
      size_k, workspace.data_ptr(), groupsize, dev,
      at::cuda::getCurrentCUDAStream(dev), thread_k, thread_n, sms, max_par);

  return d;
}
