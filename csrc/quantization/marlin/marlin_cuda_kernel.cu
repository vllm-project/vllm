#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>

template <typename T> inline std::string str(T x) { return std::to_string(x); }

namespace marlin {

constexpr int ceildiv(int a, int b) { return (a + b - 1) / b; }

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800

// Instances of `Vec` are used to organize groups of >>registers<<, as needed
// for instance as inputs to tensor core operations. Consequently, all
// corresponding index accesses must be compile-time constants, which is why we
// extensively use `#pragma unroll` throughout the kernel code to guarantee
// this.
template <typename T, int n> struct Vec {
  T elems[n];
  __device__ T &operator[](int i) { return elems[i]; }
};

using I4 = Vec<int, 4>;

// Matrix fragments for tensor core instructions; their precise layout is
// documented here:
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-floating-point-type
using FragA = Vec<half2, 4>;
using FragB = Vec<half2, 2>;
using FragC = Vec<float, 4>;
using FragS = Vec<half2, 1>; // quantization scales

// Predicated asynchronous global->shared copy; used for inputs A where we apply
// predication to handle batchsizes that are not multiples of 16.
__device__ inline void cp_async4_pred(void *smem_ptr, const void *glob_ptr,
                                      bool pred = true) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile("{\n"
               "   .reg .pred p;\n"
               "   setp.ne.b32 p, %0, 0;\n"
               "   @p cp.async.cg.shared.global [%1], [%2], %3;\n"
               "}\n" ::"r"((int)pred),
               "r"(smem), "l"(glob_ptr), "n"(BYTES));
}

// Asynchronous global->shared copy with a cache hint indicating that the values
// may be evicted immediately; used for quantized weights B, which are only
// accessed precisely once and should thus not pollute the L2 cache which we
// need for inputs A and outputs C.
__device__ inline void cp_async4_stream(void *smem_ptr, const void *glob_ptr) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
      "{\n"
      "   .reg .b64 p;\n"
      "   createpolicy.fractional.L2::evict_first.b64 p, 1.0;"
      "   cp.async.cg.shared.global.L2::cache_hint [%0], [%1], %2, p;\n"
      "}\n" ::"r"(smem),
      "l"(glob_ptr), "n"(BYTES));
}

// Async copy fence.
__device__ inline void cp_async_fence() {
  asm volatile("cp.async.commit_group;\n" ::);
}

// Wait until at most `n` async copy stages are still pending.
template <int n> __device__ inline void cp_async_wait() {
  asm volatile("cp.async.wait_group %0;\n" ::"n"(n));
}

// m16n8k16 tensor core mma instruction with fp16 inputs and fp32
// output/accumulation.
__device__ inline void mma(const FragA &a_frag, const FragB &frag_b,
                           FragC &frag_c) {
  const uint32_t *a = reinterpret_cast<const uint32_t *>(&a_frag);
  const uint32_t *b = reinterpret_cast<const uint32_t *>(&frag_b);
  float *c = reinterpret_cast<float *>(&frag_c);
  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
               : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
               : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]),
                 "r"(b[1]), "f"(c[0]), "f"(c[1]), "f"(c[2]), "f"(c[3]));
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared
// memory, directly in tensor core layout.
__device__ inline void ldsm4(FragA &frag_a, const void *smem_ptr) {
  uint32_t *a = reinterpret_cast<uint32_t *>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
               : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3])
               : "r"(smem));
}

// Lookup-table based 3-input logical operation; explicitly used for
// dequantization as the compiler does not seem to automatically recognize it in
// all cases.
template <int lut> __device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(res)
               : "r"(a), "r"(b), "r"(c), "n"(lut));
  return res;
}

// Efficiently dequantize an int32 value into a full B-fragment of 4 fp16
// values. We mostly follow the strategy in the link below, with some small
// changes:
// https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
__device__ inline FragB dequant(int q) {
  const int LO = 0x000f000f;
  const int HI = 0x00f000f0;
  const int EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  int lo = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  int hi = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  // We want signed int4 outputs, hence we fuse the `-8` symmetric zero point
  // directly into `SUB` and `ADD`.
  const int SUB = 0x64086408;
  const int MUL = 0x2c002c00;
  const int ADD = 0xd480d480;
  FragB frag_b;
  frag_b[0] = __hsub2(*reinterpret_cast<half2 *>(&lo),
                      *reinterpret_cast<const half2 *>(&SUB));
  frag_b[1] = __hfma2(*reinterpret_cast<half2 *>(&hi),
                      *reinterpret_cast<const half2 *>(&MUL),
                      *reinterpret_cast<const half2 *>(&ADD));
  return frag_b;
}

// Multiply dequantized values by the corresponding quantization scale; used
// only for grouped quantization.
__device__ inline void scale(FragB &frag_b, FragS &frag_s, int i) {
  half2 s = __half2half2(reinterpret_cast<__half *>(&frag_s)[i]);
  frag_b[0] = __hmul2(frag_b[0], s);
  frag_b[1] = __hmul2(frag_b[1], s);
}

// Wait until barrier reaches `count`, then lock for current threadblock.
__device__ inline void barrier_acquire(int *lock, int count) {
  if (threadIdx.x == 0) {
    int state = -1;
    do
      // Guarantee that subsequent writes by this threadblock will be visible
      // globally.
      asm volatile("ld.global.acquire.gpu.b32 %0, [%1];\n"
                   : "=r"(state)
                   : "l"(lock));
    while (state != count);
  }
  __syncthreads();
}

// Release barrier and increment visitation count.
__device__ inline void barrier_release(int *lock, bool reset = false) {
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    // Make sure that all writes since acquiring this barrier are visible
    // globally, while releasing the barrier.
    asm volatile("fence.acq_rel.gpu;\n");
    asm volatile("red.relaxed.gpu.global.add.s32 [%0], %1;\n"
                 :
                 : "l"(lock), "r"(val));
  }
}

template <const int threads,         // number of threads in a threadblock
          const int thread_m_blocks, // number of 16x16 blocks in the m
                                     // dimension (batchsize) of the threadblock
          const int thread_n_blocks, // same for n dimension (output)
          const int thread_k_blocks, // same for k dimension (reduction)
          const int stages, // number of stages for the async global->shared
                            // fetch pipeline
          const int group_blocks = -1 // number of consecutive 16x16 blocks with
                                      // a separate quantization scale
          >
__global__ void
Marlin(const int4 *__restrict__ A, // fp16 input matrix of shape mxk
       const int4 *__restrict__ B, // 4bit quantized weight matrix of shape kxn
       int4 *__restrict__ C,       // fp16 output buffer of shape mxn
       const int4
           *__restrict__ s, // fp16 quantization scales of shape (k/groupsize)xn
       int prob_m,          // batch dimension m
       int prob_n,          // output dimension n
       int prob_k,          // reduction dimension k
       int *locks           // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the
  // same size, which might involve multiple column "slices" (of width 16 *
  // `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM
  // example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it
  // ensures good utilization of all SMs for many kinds of shape and GPU
  // configurations, while requiring as few slow global cross-threadblock
  // reductions as possible.

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a
  // better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);
  // Ensure that the number of tiles in each stripe is a multiple of the
  // groupsize; this avoids an annoying special case where a stripe starts in
  // the middle of group.
  if (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) *
            ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters; // number of threadblock tiles in the current slice
  int slice_count =
      0;         // total number of active threadblocks in the current slice
  int slice_idx; // index of threadblock in current slice; numbered bottom to
                 // top

  // We can easily implement parallel problem execution by just remapping
  // indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for
  // synchronization.
  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel)
      slice_iters = 0;
    if (slice_iters == 0)
      return;
    if (slice_row + slice_iters > k_tiles)
      slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0)
        slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0)
          slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 8; // stride of the A matrix in global memory
  // We typically use `constexpr` to indicate that this value is a compile-time
  // constant
  constexpr int a_sh_stride =
      16 * thread_k_blocks / 8; // stride of an A matrix tile in shared memory
  constexpr int a_gl_rd_delta_o =
      16 * thread_k_blocks /
      8; // delta between subsequent A tiles in global memory
  int a_gl_rd_delta_i =
      a_gl_stride *
      (threads / a_gl_rd_delta_o); // between subsequent accesses within a tile
  constexpr int a_sh_wr_delta =
      a_sh_stride * (threads / a_gl_rd_delta_o); // between shared memory writes
  constexpr int a_sh_rd_delta_o =
      2 * ((threads / 32) /
           (thread_n_blocks / 4)); // between shared memory tile reads
  constexpr int a_sh_rd_delta_i =
      a_sh_stride * 16; // within a shared memory tile
  constexpr int a_sh_stage =
      a_sh_stride * (16 * thread_m_blocks); // overall size of a tile
  constexpr int a_sh_wr_iters =
      ceildiv(a_sh_stage,
              a_sh_wr_delta); // number of shared write iterations for a tile

  int b_gl_stride = 16 * prob_n / 32;
  constexpr int b_sh_stride = 32 * thread_n_blocks / 4;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride);
  constexpr int b_sh_wr_delta = threads;
  constexpr int b_sh_rd_delta = threads;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_sh_stage = s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  int a_sh_rd =
      a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd =
      b_gl_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x;
  int b_sh_rd = threadIdx.x;

  int s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
                s_sh_stride * slice_col + threadIdx.x;
  int s_sh_wr = threadIdx.x;
  int s_sh_rd;
  // We use a different scale layout for grouped and column-wise quantization as
  // we scale a `half2` tile in column-major layout in the former and in
  // row-major in the latter case.
  if (group_blocks != -1)
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) / 4;
  else
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) % 4;

  // Precompute which thread should not read memory in which iterations; this is
  // needed if there are more threads than required for a certain tilesize or
  // when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
#pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  // To ensure that writing and reading A tiles to/from shared memory, the
  // latter in fragment format, is fully bank conflict free, we need to use a
  // rather fancy XOR-based layout. The key here is that neither reads nor
  // writes of the 16-byte `int4` blocks of 8 consecutive threads involve the
  // same shared memory banks. Further, it seems (based on NSight-Compute) that
  // each warp must also write a consecutive memory segment?
  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main
  // loop unrolls, all shared memory accesses are static, we simply precompute
  // both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
#pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
#pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
#pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at
  // runtime; we break dependencies between subsequent accesses with a tile by
  // maintining multiple pointers (we have enough registers), a tiny
  // optimization.
  const int4 *B_ptr[b_sh_wr_iters];
#pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  int4 *sh_a = sh;
  int4 *sh_b = sh_a + (stages * a_sh_stage);
  int4 *sh_s = sh_b + (stages * b_sh_stage);
  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];

  // Zero accumulators.
  auto zero_accums = [&]() {
#pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float *>(frag_c)[i] = 0;
  };

  // Asynchronously fetch the next A, B and s tile from global to the next
  // shared memory pipeline location.
  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4 *sh_a_stage = sh_a + a_sh_stage * pipe;
#pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4 *sh_b_stage = sh_b + b_sh_stage * pipe;
#pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
        cp_async4_stream(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr], B_ptr[i]);
        B_ptr[i] += b_gl_rd_delta_o;
      }
      // Only fetch scales if this tile starts a new group
      if (group_blocks != -1 && pipe % (group_blocks / thread_k_blocks) == 0) {
        int4 *sh_s_stage = sh_s + s_sh_stage * pipe;
        if (s_sh_wr_pred)
          cp_async4_stream(&sh_s_stage[s_sh_wr], &s[s_gl_rd]);
        s_gl_rd += s_gl_rd_delta;
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that
    // waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe
  // into the current register buffer.
  auto fetch_to_registers = [&](int k, int pipe) {
    // It may seem inefficient that we reload the groups for every sub-tile;
    // however, this does not seem to be a significant bottleneck, while some
    // theoretically better attempts have lead to bad instruction ordering by
    // the compiler and correspondingly a noticeable drop in performance.
    if (group_blocks != -1) {
      int4 *sh_s_stage =
          sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) *
                               (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4 *>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
    }
    int4 *sh_a_stage = sh_a + a_sh_stage * pipe;
#pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4 *sh_b_stage = sh_b + b_sh_stage * pipe;
    frag_b_quant[k % 2] = *reinterpret_cast<I4 *>(
        &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd]);
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&](int k) {
// We have the m dimension as the inner loop in order to encourage overlapping
// dequantization and matmul operations.
#pragma unroll
    for (int j = 0; j < 4; j++) {
      int b_quant = frag_b_quant[k % 2][j];
      int b_quant_shift = b_quant >> 8;
      FragB frag_b0 = dequant(b_quant);
      // If there are no groups, we can just scale the final output once and can
      // avoid doing so for each weight.
      if (group_blocks != -1)
        scale(frag_b0, frag_s[k % 2][j], 0);
      FragB frag_b1 = dequant(b_quant_shift);
      if (group_blocks != -1)
        scale(frag_b1, frag_s[k % 2][j], 1);
#pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the
  // number of warps while keeping the n dimension of a tile reasonable, we have
  // multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride;
      constexpr int red_sh_stride = b_sh_stride * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride) +
                      (threadIdx.x % b_sh_stride);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any
      // unnecessary read or write iterations, e.g., for two warps we write only
      // once by warp 1 and read only once by warp 0.

#pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
#pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
#pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float *c_rd = reinterpret_cast<float *>(
                    &sh[red_sh_delta * j + red_sh_rd]);
                float *c_wr = reinterpret_cast<float *>(&sh[red_sh_wr]);
#pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC *>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] =
                  reinterpret_cast<int4 *>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
#pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float *c_rd =
                reinterpret_cast<float *>(&sh[red_sh_delta * i + red_sh_rd]);
#pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC *>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we
  // finally have to globally reduce over the results. As the striped partitioning
  // minimizes the number of such reductions and our outputs are usually rather
  // small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&](bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to
    // maximize L2 cache utilization in this step. To do this, we write out
    // results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) +
                    4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
// Interestingly, doing direct global accesses here really seems to mess up the
// compiler and lead to slowdowns, hence we also use async-copies even though
// these fetches are not actually asynchronous.
#pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(&sh[c_sh_wr + c_sh_wr_delta * i],
                         &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                            c_gl_wr_delta_i * (i % 2)],
                         i < (thread_m_blocks - 1) * 4 ||
                             8 * (i / 2) + row < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

#pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
#pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float *>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] +=
                  __half2float(reinterpret_cast<__half *>(&c_red)[j]);
            }
          }
          if (!last) {
            int4 c;
#pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<__half *>(&c)[j] =
                  __float2half(reinterpret_cast<float *>(
                      &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]);
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually
  // reshuffle matrix fragments in this step, the reduction above is performed
  // in fragment layout.
  auto write_result = [&]() {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta =
        c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr =
        (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final
    // global write patterns
    auto write = [&](int idx, float c0, float c1, FragS &s) {
      half2 res = __halves2half2(__float2half(c0), __float2half(c1));
      if (group_blocks ==
          -1) // for per-column quantization we finally apply the scale here
        res = __hmul2(res, s[0]);
      ((half2 *)sh)[idx] = res;
    };
    if (threadIdx.x / 32 < thread_n_blocks / 4) {
#pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
#pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0],
                frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2],
                frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0],
                frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2],
                frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

#pragma unroll
    for (int i = 0;
         i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&]() {
#pragma unroll
    for (int i = 0; i < stages - 1; i++)
      fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  // Main loop.
  while (slice_iters) {
// We unroll over both the global fetch and the register load pipeline to ensure
// all shared memory accesses are static. Note that both pipelines have even
// length meaning that the next iteration will always start at index 0.
#pragma unroll
    for (int pipe = 0; pipe < stages;) {
#pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe,
                          slice_iters >= stages);
          pipe++;
          wait_for_stage();
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0)
        break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    // Process results and, if necessary, proceed to the next column slice.
    // While this pattern may not be the most readable, other ways of writing
    // the loop seemed to noticeably worse performance after compilation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before
      // write-out
      if (group_blocks == -1 && last) {
        if (s_sh_wr_pred)
          cp_async4_stream(&sh_s[s_sh_wr], &s[s_gl_rd]);
        cp_async_fence();
      }
      thread_block_reduce();
      if (group_blocks == -1 && last) {
        cp_async_wait<0>();
        __syncthreads();
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
          reinterpret_cast<int4 *>(&frag_s)[0] = sh_s[s_sh_rd + 0];
          reinterpret_cast<int4 *>(&frag_s)[1] = sh_s[s_sh_rd + 4];
        }
      }
      if (slice_count > 1) { // only globally reduce if there is more than one
                             // block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last) // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
#pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
#pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++)
            B_ptr[i] -= b_gl_stride;
        }
        s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}

#else

template <const int threads,         // number of threads in a threadblock
          const int thread_m_blocks, // number of 16x16 blocks in the m
                                     // dimension (batchsize) of the threadblock
          const int thread_n_blocks, // same for n dimension (output)
          const int thread_k_blocks, // same for k dimension (reduction)
          const int stages, // number of stages for the async global->shared
                            // fetch pipeline
          const int group_blocks = -1 // number of consecutive 16x16 blocks with
                                      // a separate quantization scale
          >
__global__ void
Marlin(const int4 *__restrict__ A, // fp16 input matrix of shape mxk
       const int4 *__restrict__ B, // 4bit quantized weight matrix of shape kxn
       int4 *__restrict__ C,       // fp16 output buffer of shape mxn
       const int4
           *__restrict__ s, // fp16 quantization scales of shape (k/groupsize)xn
       int prob_m,          // batch dimension m
       int prob_n,          // output dimension n
       int prob_k,          // reduction dimension k
       int *locks           // extra global storage for barrier synchronization
) {
  // Marlin is not implemented yet for SM < 8.0
  assert(false);
  return;
}

#endif

// 8 warps are a good choice since every SM has 4 schedulers and having more
// than 1 warp per schedule allows some more latency hiding. At the same time,
// we want relatively few warps to have many registers per warp and small tiles.
const int USER_THREADS =
    256;              // Note: This is only used with user-provided thread_k/n
const int STAGES = 4; // 4 pipeline stages fit into shared memory
const int SHARED_MEM =
    96 * 1024; // max shared memory on compute capability 8.6 (< 8.0)

static constexpr int min_thread_n = 64;
static constexpr int min_thread_k = 64;

static constexpr int tile_size = 16;
static constexpr int max_par = 16;

static constexpr int pack_factor_4bit =
    8; // We have 8 4-bit vals inside a 32 bit

#define __CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,           \
                  GROUP_BLOCKS, NUM_THREADS)                                   \
  else if (thread_m_blocks == THREAD_M_BLOCKS &&                               \
           thread_n_blocks == THREAD_N_BLOCKS &&                               \
           thread_k_blocks == THREAD_K_BLOCKS &&                               \
           group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {       \
    hipFuncSetAttribute(reinterpret_cast<const void*>(Marlin<NUM_THREADS), THREAD_M_BLOCKS, THREAD_N_BLOCKS, \
                                THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>,        \
                         hipFuncAttributeMaxDynamicSharedMemorySize,          \
                         SHARED_MEM);                                          \
    Marlin<NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,     \
           STAGES, GROUP_BLOCKS><<<blocks, NUM_THREADS, SHARED_MEM, stream>>>( \
        A_ptr, B_ptr, C_ptr, s_ptr, prob_m, prob_n, prob_k, locks);            \
  }

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256}, // Default
    {128, 64, 128},  // Reduce N 2X, same K
    {64, 256, 256},  // Reduce K 2X, increase N 2X
    {64, 128, 128},  // Reduce K 2X, same N
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},  // Default
    {128, 128, 256}, // Reduce N 2X, increase K 2X
    {64, 128, 128},  // Reduce N 2X, same K
    {128, 64, 128},  // Reduce N 4X, increase K 2X
};

bool is_valid_config(thread_config_t const &th_config, int prob_m, int prob_n,
                     int prob_k) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // thread_k can be only 128 or 64 (because it must be less than groupsize
  // which is 128)
  if (th_config.thread_k != 128 && th_config.thread_k != 64) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  return true;
}

thread_config_t determine_thread_config(int prob_m, int prob_n, int prob_k) {

  if (prob_m <= 16) {
    for (auto th_config : small_batch_thread_configs) {
      if (is_valid_config(th_config, prob_m, prob_n, prob_k)) {
        return th_config;
      }
    }

  } else {
    for (auto th_config : large_batch_thread_configs) {
      if (is_valid_config(th_config, prob_m, prob_n, prob_k)) {
        return th_config;
      }
    }
  }

  return thread_config_t{-1, -1, -1};
}

#define CALL_IF(N_BLOCKS, K_BLOCKS, NUM_THREADS)                               \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS)                            \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)                             \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS)                            \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)                             \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS)                            \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)                             \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS)                            \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)                             \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS)                            \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)

void marlin_cuda(const void *A, const void *B, void *C, void *s, int prob_m,
                 int prob_n, int prob_k, void *workspace, int groupsize = -1,
                 int dev = 0, hipStream_t stream = 0, int thread_k = -1,
                 int thread_n = -1, int sms = -1, int max_par = 16) {
  int tot_m = prob_m;
  int tot_m_blocks = ceildiv(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);

  // Set thread config
  thread_config_t th_config;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    th_config = thread_config_t{thread_k, thread_n, USER_THREADS};
  } else {
    // Auto config
    th_config = determine_thread_config(prob_m, prob_n, prob_k);
  }

  if (!is_valid_config(th_config, prob_m, prob_n, prob_k)) {
    throw std::runtime_error(
        "Invalid thread config: thread_k = " + str(th_config.thread_k) +
        ", thread_n = " + str(th_config.thread_n) +
        ", num_threads = " + str(th_config.num_threads) + " for MKN = [" +
        str(prob_m) + ", " + str(prob_k) + ", " + str(prob_n) + "]");
  }

  // Uncomment for debug
  // std::cout << "Using thread_config: thread_k = " + str(th_config.thread_k) +
  //                  ", thread_n = " + str(th_config.thread_n) +
  //                  ", num_threads = " + str(th_config.num_threads) + " for
  //                  MKN = [" + str(prob_m) +
  //                  ", " + str(prob_k) + ", " + str(prob_n) + "]\n";

  int num_threads = th_config.num_threads;
  thread_k = th_config.thread_k;
  thread_n = th_config.thread_n;

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  int blocks = sms;

  if (prob_m == 0 || prob_n == 0 || prob_k == 0) {
    return;
  }

  TORCH_CHECK(prob_n % thread_n == 0, "prob_n = ", prob_n,
              " is not divisible by thread_n = ", thread_n);
  TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
              " is not divisible by thread_k = ", thread_k);
  if (group_blocks != -1) {
    TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                " is not divisible by group_blocks = ", group_blocks);
  }

  const int4 *A_ptr = (const int4 *)A;
  const int4 *B_ptr = (const int4 *)B;
  int4 *C_ptr = (int4 *)C;
  const int4 *s_ptr = (const int4 *)s;

  int *locks = (int *)workspace;

  for (int i = 0; i < tot_m_blocks; i += 4) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > 4) {
      // Note that parallel > 1 currently only works for inputs without any
      // padding
      par = (16 * thread_m_blocks - pad) / 64;
      if (par > max_par)
        par = max_par;
      prob_m = 64 * par;
      i += 4 * (par - 1);
      thread_m_blocks = 4;
    }

    // For compilation speed, we only define the kernel configurations that have
    // seemed useful (in terms of performance) in our testing, however many more
    // are, in principle, possible.
    if (false) {
    }
    CALL_IF(8, 8, 256)
    CALL_IF(16, 4, 256)
    CALL_IF(8, 4, 128)
    CALL_IF(4, 8, 128)
    else {
      throw std::runtime_error("Unsupported shapes: MKN = [" + str(prob_m) +
                               ", " + str(prob_k) + ", " + str(prob_n) + "]" +
                               ", groupsize = " + str(groupsize) +
                               ", thread_m_blocks = " + str(thread_m_blocks) +
                               ", thread_n_blocks = " + str(thread_n_blocks) +
                               ", thread_k_blocks = " + str(thread_k_blocks));
    }

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }
}

} // namespace marlin

torch::Tensor marlin_gemm(torch::Tensor &a, torch::Tensor &b_q_weight,
                          torch::Tensor &b_scales, torch::Tensor &workspace,
                          int64_t size_m, int64_t size_n, int64_t size_k) {

  // Verify M
  TORCH_CHECK(size_m == a.size(0),
              "Shape mismatch: a.size(0) = " + str(a.size(0)) +
                  ", size_m = " + str(size_m));

  // Verify K
  TORCH_CHECK(size_k == a.size(1),
              "Shape mismatch: a.size(1) = " + str(a.size(1)) +
                  ", size_k = " + str(size_k));
  TORCH_CHECK(size_k % marlin::tile_size == 0,
              "size_k = " + str(size_k) +
                  " is not divisible by tile_size = " + str(marlin::tile_size));
  TORCH_CHECK((size_k / marlin::tile_size) == b_q_weight.size(0),
              "Shape mismatch: b_q_weight.size(0) = " +
                  str(b_q_weight.size(0)) + ", size_k = " + str(size_k) +
                  ", tile_size = " + str(marlin::tile_size));

  // Verify N
  TORCH_CHECK(b_scales.size(1) == size_n,
              "b_scales.size(1) = " + str(b_scales.size(1)) +
                  ", size_n = " + str(size_n));
  TORCH_CHECK(b_q_weight.size(1) % marlin::tile_size == 0,
              "b_q_weight.size(1) = " + str(b_q_weight.size(1)) +
                  " is not divisible by tile_size = " + str(marlin::tile_size));

  int actual_size_n =
      (b_q_weight.size(1) / marlin::tile_size) * marlin::pack_factor_4bit;
  TORCH_CHECK(size_n == actual_size_n,
              "size_n = " + str(size_n) +
                  ", actual_size_n = " + str(actual_size_n));

  // Verify A device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  // Verify B device and strides
  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  // Verify scales device and strides
  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  // Alloc C matrix
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c = torch::empty({size_m, size_n}, options);

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel (can usually be left as auto -1)
  int sms = -1;

  // Detect groupsize
  if (b_scales.size(0) != 1) {
    TORCH_CHECK(size_k % b_scales.size(0) == 0,
                "size_k = " + str(size_k) +
                    ", is not divisible by b_scales.size(0) = " +
                    str(b_scales.size(0)));
  }
  int groupsize = b_scales.size(0) == 1 ? -1 : size_k / b_scales.size(0);

  // Verify groupsize
  TORCH_CHECK(groupsize == -1 || groupsize == 128,
              "Unexpected groupsize = " + str(groupsize));

  // Verify workspace size
  TORCH_CHECK(
      size_n % marlin::min_thread_n == 0,
      "size_n = " + str(size_n) +
          ", is not divisible by min_thread_n = " + str(marlin::min_thread_n));
  int min_workspace_size = (size_n / marlin::min_thread_n) * marlin::max_par;
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = " + str(workspace.numel()) +
                  " is below min_workspace_size = " + str(min_workspace_size));

  int dev = a.get_device();
  marlin::marlin_cuda(a.data_ptr(), b_q_weight.data_ptr(), c.data_ptr(),
                      b_scales.data_ptr(), size_m, size_n, size_k,
                      workspace.data_ptr(), groupsize, dev,
                      at::cuda::getCurrentCUDAStream(dev), thread_k, thread_n,
                      sms, marlin::max_par);

  return c;
}
