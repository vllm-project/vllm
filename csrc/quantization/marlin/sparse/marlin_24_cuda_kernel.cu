#include "hip/hip_runtime.h"
/*
 * Notice: This file was modified by Neuralmagic inc to include 8-bit support
 *
 * Copyright (C) 2024 Roberto Lopez Castro (roberto.lopez.castro@udc.es). All
 * Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "common/base.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

#else

  #include "common/mem.h"
  #include "common/mma.h"

#endif

template <typename T>
inline std::string str(T x) {
  return std::to_string(x);
}

namespace marlin_24 {

// 8 warps are a good choice since every SM has 4 schedulers and having more
// than 1 warp per schedule allows some more latency hiding. At the same time,
// we want relatively few warps to have many registers per warp and small tiles.
static constexpr int THREADS = 256;
static constexpr int STAGES = 4;  // 4 pipeline stages fit into shared memory

static constexpr int min_thread_n = 128;

static constexpr int tile_size = 16;
static constexpr int max_par = 16;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

template <const int num_bits,         // weight bits
          const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1  // number of consecutive 16x16 blocks
                                       // with a separate quantization scale
          >
__global__ void Marlin_24(
    const int4* __restrict__ A,     // fp16 input matrix of shape mxk
    const int4* __restrict__ B,     // 4bit quantized weight matrix of shape kxn
    const int4* __restrict__ meta,  // 2bit metadata information about 2:4
                                    // format on B
    int4* __restrict__ C,           // fp16 output buffer of shape mxn
    const int4* __restrict__ s,     // fp16 quantization scales of shape
                                    // (k/groupsize)xn
    int prob_m,                     // batch dimension m
    int prob_n,                     // output dimension n
    int prob_k,                     // reduction dimension k
    int* locks  // extra global storage for barrier synchronization
) {}

torch::Tensor gptq_marlin_24_gemm(torch::Tensor& a, torch::Tensor& b_q_weight,
                                  torch::Tensor& b_meta,
                                  torch::Tensor& b_scales,
                                  torch::Tensor& workspace, int64_t num_bits,
                                  int64_t size_m, int64_t size_n,
                                  int64_t size_k) {
  TORCH_CHECK_NOT_IMPLEMENTED(
      false, "gptq_marlin_24_gemm(..) requires CUDA_ARCH >= 8.0");
  return torch::empty({1, 1});
}

#else

template <const int num_bits,         // weight bits
          const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1  // number of consecutive 16x16 blocks
                                       // with a separate quantization scale
          >
__global__ void Marlin_24(
    const int4* __restrict__ A,     // fp16 input matrix of shape mxk
    const int4* __restrict__ B,     // 4bit quantized weight matrix of shape kxn
    const int4* __restrict__ meta,  // 2bit metadata information about 2:4
                                    // format on B
    int4* __restrict__ C,           // fp16 output buffer of shape mxn
    const int4* __restrict__ s,     // fp16 quantization scales of shape
                                    // (k/groupsize)xn
    int prob_m,                     // batch dimension m
    int prob_n,                     // output dimension n
    int prob_k,                     // reduction dimension k
    int* locks  // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the
  // same size, which might involve multiple column "slices" (of width 16 *
  // `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM
  // example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it
  // ensures good utilization of all SMs for many kinds of shape and GPU
  // configurations, while requiring as few slow global cross-threadblock
  // reductions as possible.

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a
  // better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  // number of thread_k_blocks in k-dim
  int k_tiles = prob_k / 32 / thread_k_blocks;
  // number of thread_n_blocks in n-dim
  int n_tiles = prob_n / 16 / thread_n_blocks;
  // iters needed to cover all slices
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);

  // Ensure that the number of tiles in each stripe is a multiple of the
  // groupsize; this avoids an annoying special case where a stripe starts in
  // the middle of group.
  if (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) *
            ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  // number of threadblock tiles in the current slice
  int slice_iters;
  // total number of active threadblocks in the current slice
  int slice_count = 0;
  // index of threadblock in current slice; numbered bottom to top
  int slice_idx;

  // We can easily implement parallel problem execution by just remapping
  // indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for
  // synchronization.
  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel) slice_iters = 0;
    if (slice_iters == 0) return;
    if (slice_row + slice_iters > k_tiles) slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0) slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0) slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  // RLC: 8 is vec_size -> 128-bit instructions, 8 fp16 elements
  int a_gl_stride = prob_k / 8;  // stride of the A matrix in global memory

  // stride of an A matrix tile in shared memory
  constexpr int a_sh_stride = 32 * thread_k_blocks / 8;
  // delta between subsequent A tiles in global memory
  constexpr int a_gl_rd_delta_o = 32 * thread_k_blocks / 8;
  // between subsequent accesses within a tile
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o);
  // between shared memory writes
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o);
  // between shared memory tile reads //RLC: 2 * #warps k-dim
  constexpr int a_sh_rd_delta_o = 4 * ((threads / 32) / (thread_n_blocks / 4));
  // within a shared memory tile
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16;
  // overall size of a tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks);
  // number of shared write iterations for a tile
  constexpr int a_sh_wr_iters = ceildiv(a_sh_stage, a_sh_wr_delta);

  constexpr int pack_factor = 32 / num_bits;

  int b_gl_stride = 16 * prob_n / (pack_factor * 4);
  constexpr int b_sh_stride = ((thread_n_blocks * 16) * 16 / pack_factor) / 4;
  constexpr int b_thread_vecs = num_bits == 4 ? 1 : 2;
  constexpr int b_sh_stride_threads = b_sh_stride / b_thread_vecs;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride_threads);
  constexpr int b_sh_wr_delta = threads * b_thread_vecs;
  constexpr int b_sh_rd_delta = threads * b_thread_vecs;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  int m_gl_stride = 2 * prob_n / 8;  // (16*2*4 / 8) = 16
  constexpr int m_sh_stride =
      (16 * thread_n_blocks) / 4;  // #warps n-dim * threads/warp
  int m_gl_rd_delta_o = m_gl_stride * thread_k_blocks;
  int m_gl_rd_delta_i = m_gl_stride * (threads / m_sh_stride);
  constexpr int m_sh_wr_delta = threads / 2;
  constexpr int m_sh_rd_delta = threads / 2;
  constexpr int m_sh_stage = m_sh_stride * thread_k_blocks;
  constexpr int m_sh_iters = ceildiv(m_sh_stage, m_sh_wr_delta);

  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_sh_stage = s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  int a_sh_rd =
      a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 4 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride_threads) +
                (threadIdx.x % b_sh_stride_threads) * b_thread_vecs;
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x * b_thread_vecs;
  int b_sh_rd = threadIdx.x * b_thread_vecs;

  int m_gl_rd = m_gl_stride * (threadIdx.x / (m_sh_stride)) +
                (threadIdx.x % (m_sh_stride));
  m_gl_rd += (m_sh_stride)*slice_col;
  m_gl_rd += m_gl_rd_delta_o * slice_row;
  int m_sh_wr = threadIdx.x;
  int m_sh_rd = threadIdx.x % 16 + (threadIdx.x / 32) * 16;

  int s_gl_rd;
  if constexpr (group_blocks == -1) {
    s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
  } else {
    s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
              s_sh_stride * slice_col + threadIdx.x;
  }

  int s_sh_wr = threadIdx.x;
  int s_sh_rd;
  // We use a different scale layout for grouped and column-wise quantization as
  // we scale a `half2` tile in column-major layout in the former and in
  // row-major in the latter case.
  if (group_blocks != -1) {
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) / 4;
  } else {
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) / 4;
  }

  // Precompute which thread should not read memory in which iterations; this is
  // needed if there are more threads than required for a certain tilesize or
  // when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++) {
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;
  }
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  // To ensure that writing and reading A tiles to/from shared memory, the
  // latter in fragment format, is fully bank conflict free, we need to use a
  // rather fancy XOR-based layout. The key here is that neither reads nor
  // writes of the 16-byte `int4` blocks of 8 consecutive threads involve the
  // same shared memory banks. Further, it seems (based on NSight-Compute) that
  // each warp must also write a consecutive memory segment?
  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main
  // loop unrolls, all shared memory accesses are static, we simply precompute
  // both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[2][b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++) {
      a_sh_rd_trans[0][i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
      a_sh_rd_trans[1][i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd + 2);
    }
  }

  // Since B-accesses have non-constant stride they have to be computed at
  // runtime; we break dependencies between subsequent accesses with a tile by
  // maintining multiple pointers (we have enough registers), a tiny
  // optimization.
  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  bool m_sh_wr_pred = threadIdx.x < m_sh_wr_delta;
  const int4* meta_ptr[m_sh_iters];
  #pragma unroll
  for (int i = 0; i < m_sh_iters; i++)
    meta_ptr[i] = meta + m_gl_rd_delta_i * i + m_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_s = sh_b + (stages * b_sh_stage);
  int4* sh_m = sh_s + (stages * s_sh_stage);
  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks][2];
  I4 frag_b_quant[2][b_thread_vecs];
  FragM frag_m[2][2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];

  // Zero accumulators.
  auto zero_accums = [&]() {
  #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  // Asynchronously fetch the next A, B and s tile from global to the next
  // shared memory pipeline location.
  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
        for (int j = 0; j < b_thread_vecs; j++) {
          cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr + j], B_ptr[i] + j);
        }
        B_ptr[i] += b_gl_rd_delta_o;
      }
      int4* sh_meta_stage = sh_m + m_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < m_sh_iters; i++) {
        if (m_sh_wr_pred)
          cp_async4(&sh_meta_stage[m_sh_wr_delta * i + m_sh_wr], meta_ptr[i]);
        meta_ptr[i] += m_gl_rd_delta_o;
      }
      // Only fetch scales if this tile starts a new group
      if (group_blocks != -1 && pipe % (group_blocks / thread_k_blocks) == 0) {
        int4* sh_s_stage = sh_s + s_sh_stage * pipe;
        if (s_sh_wr_pred) cp_async4(&sh_s_stage[s_sh_wr], &s[s_gl_rd]);
        s_gl_rd += s_gl_rd_delta;
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that
    // waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe
  // into the current register buffer.
  auto fetch_to_registers = [&](int k, int pipe) {
    // It may seem inefficient that we reload the groups for every sub-tile;
    // however, this does not seem to be a significant bottleneck, while some
    // theoretically better attempts have lead to bad instruction ordering by
    // the compiler and correspondingly a noticeable drop in performance.
    if (group_blocks != -1) {
      int4* sh_s_stage =
          sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) *
                               (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4*>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
    }
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++) {
      ldsm4(frag_a[k % 2][i][0],
            &sh_a_stage[a_sh_rd_trans[0][k % b_sh_wr_iters][i]]);
      ldsm4(frag_a[k % 2][i][1],
            &sh_a_stage[a_sh_rd_trans[1][k % b_sh_wr_iters][i]]);
    }

    int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < b_thread_vecs; i++) {
      frag_b_quant[k % 2][i] = *reinterpret_cast<I4*>(
          &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd + i]);
    }

    // Load meta with ldsm4
    int4* sh_m_stage = sh_m + m_sh_stage * pipe;
    ldsm4_m(frag_m[k % 2][0],
            &sh_m_stage[m_sh_rd_delta * (k % m_sh_iters) + m_sh_rd]);
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&](int k) {
  // We have the m dimension as the inner loop in order to encourage overlapping
  // dequantization and matmul operations.
  #pragma unroll
    for (int j = 0; j < 4; j++) {
      FragB frag_b0;
      FragB frag_b1;

      if constexpr (num_bits == 4) {
        int b_quant = frag_b_quant[k % 2][0][j];
        int b_quant_shift = b_quant >> 8;

        frag_b0 = dequant_4bit(b_quant);
        frag_b1 = dequant_4bit(b_quant_shift);

      } else {
        int* frag_b_quant_ptr = reinterpret_cast<int*>(frag_b_quant[k % 2]);
        int b_quant_0 = frag_b_quant_ptr[j * 2 + 0];
        int b_quant_1 = frag_b_quant_ptr[j * 2 + 1];

        frag_b0 = dequant_8bit(b_quant_0);
        frag_b1 = dequant_8bit(b_quant_1);
      }

      // If there are no groups, we can just scale the final output once and can
      // avoid doing so for each weight.
      if constexpr (group_blocks != -1) {
        scale(frag_b0, frag_s[k % 2][j], 0);
      }
      if constexpr (group_blocks != -1) {
        scale(frag_b1, frag_s[k % 2][j], 1);
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma_sp(frag_b0, frag_b1, frag_a[k % 2][i][0], frag_c[i][j][0],
               frag_m[k % 2][j / 2], j % 2);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the
  // number of warps while keeping the n dimension of a tile reasonable, we have
  // multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride_threads / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride_threads;
      constexpr int red_sh_stride = b_sh_stride_threads * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride_threads;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride_threads) +
                      (threadIdx.x % b_sh_stride_threads);

  // Parallel logarithmic shared memory reduction. We make sure to avoid any
  // unnecessary read or write iterations, e.g., for two warps we write only
  // once by warp 1 and read only once by warp 0.
  #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
  #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
  #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd =
                    reinterpret_cast<float*>(&sh[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh[red_sh_wr]);
  #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] =
                  reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
  #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd =
                reinterpret_cast<float*>(&sh[red_sh_delta * i + red_sh_rd]);
  #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we
  // finally have to globally reduce over the results. As the striped
  // partitioning minimizes the number of such reductions and our outputs are
  // usually rather small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&](bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to
    // maximize L2 cache utilization in this step. To do this, we write out
    // results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 2 * 4 * c_gl_stride;
      int c_gl_wr_delta_i =
          c_gl_stride;  // 8 threads (e.g., 0,4,8,12,16,20,24,28)
      int c_gl_wr = 2 * c_gl_stride * (threadIdx.x % 4) +
                    8 * (threadIdx.x / 32) + (threadIdx.x % 32) / 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;

      int col = 2 * ((threadIdx.x % 32) % 4);

      if (!first) {
  // Interestingly, doing direct global accesses here really seems to mess up
  // the compiler and lead to slowdowns, hence we also use async-copies even
  // though these fetches are not actually asynchronous.
  #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(&sh[c_sh_wr + c_sh_wr_delta * i],
                         &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                            c_gl_wr_delta_i * (i % 2)],
                         i < (thread_m_blocks - 1) * 4 ||
                             8 * (i / 2) + col + (i % 2) < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 ||
            8 * (i / 2) + col + (i % 2) < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
  #pragma unroll
            for (int j2 = 0; j2 < 2; j2++) {
  #pragma unroll
              for (int j1 = 0; j1 < 4; j1++) {
                reinterpret_cast<float*>(
                    &frag_c)[4 * 2 * 4 * (i / 4) + 8 * j1 + 2 * j2 +
                             4 * ((i % 4) / 2) + i % 2] +=
                    __half2float(
                        reinterpret_cast<__half*>(&c_red)[(j2 * 4 + j1)]);
              }
            }
          }
          if (!last) {
            int4 c;
  #pragma unroll
            for (int j2 = 0; j2 < 2; j2++) {
  #pragma unroll
              for (int j1 = 0; j1 < 4; j1++) {
                reinterpret_cast<__half*>(&c)[(j2 * 4 + j1)] =
                    __float2half(reinterpret_cast<float*>(
                        &frag_c)[4 * 2 * 4 * (i / 4) + 8 * j1 + 2 * j2 +
                                 4 * ((i % 4) / 2) + i % 2]);
              }
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually
  // reshuffle matrix fragments in this step, the reduction above is performed
  // in fragment layout.
  auto write_result = [&]() {
    int c_gl_stride = prob_n / 8;

    constexpr int c_sh_stride = 2 * thread_n_blocks;              // RLC:
    constexpr int c_sh_stride_2 = 2 * c_sh_stride + 2;            // RLC:
    constexpr int c_sh_stride_3 = 2 * (2 * thread_n_blocks) + 2;  // RLC:

    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;

    int c_sh_wr = c_sh_stride_2 * ((threadIdx.x % 32) % 4) +
                  ((threadIdx.x % 32) / 4);  // RLC:
    c_sh_wr += 8 * (threadIdx.x / 32);       // 128/4(half4)

    constexpr int c_sh_rd_delta =
        c_sh_stride_3 * (threads / (2 * 2 * thread_n_blocks));  // RLC:
    int c_sh_rd = c_sh_stride_3 * (threadIdx.x / (2 * 2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * 2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    auto write = [&](int idx, float c0, float c1, float c2, float c3, FragS& s0,
                     float c4, float c5, float c6, float c7, FragS& s1) {
      uint2 res[2];
      res[0] = to_half4(c0, c1, c2, c3);
      res[1] = to_half4(c4, c5, c6, c7);
      half2* tmp = (half2*)&res;
      // for per-column quantization we finally apply the scale here
      if constexpr (group_blocks == -1 && num_bits == 4) {
        tmp[0] = __hmul2(tmp[0], s0[0]);
        tmp[1] = __hmul2(tmp[1], s0[1]);
        tmp[2] = __hmul2(tmp[2], s1[0]);
        tmp[3] = __hmul2(tmp[3], s1[1]);
      }
      ((int4*)sh)[idx] = *((int4*)&res[0]);
    };

    // RLC:  only warp 0 and 1 baseline example
    if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        int wr = c_sh_wr;
        write(wr, frag_c[i][0][0][0], frag_c[i][1][0][0], frag_c[i][2][0][0],
              frag_c[i][3][0][0], frag_s[0][0], frag_c[i][0][0][2],
              frag_c[i][1][0][2], frag_c[i][2][0][2], frag_c[i][3][0][2],
              frag_s[0][2]);
        write(wr + c_sh_stride, frag_c[i][0][0][1], frag_c[i][1][0][1],
              frag_c[i][2][0][1], frag_c[i][3][0][1], frag_s[0][0],
              frag_c[i][0][0][3], frag_c[i][1][0][3], frag_c[i][2][0][3],
              frag_c[i][3][0][3], frag_s[0][2]);
        write(wr + 4 * c_sh_stride_2, frag_c[i][0][1][0], frag_c[i][1][1][0],
              frag_c[i][2][1][0], frag_c[i][3][1][0], frag_s[0][0],
              frag_c[i][0][1][2], frag_c[i][1][1][2], frag_c[i][2][1][2],
              frag_c[i][3][1][2], frag_s[0][2]);
        write(wr + 4 * c_sh_stride_2 + c_sh_stride, frag_c[i][0][1][1],
              frag_c[i][1][1][1], frag_c[i][2][1][1], frag_c[i][3][1][1],
              frag_s[0][0], frag_c[i][0][1][3], frag_c[i][1][1][3],
              frag_c[i][2][1][3], frag_c[i][3][1][3], frag_s[0][2]);

        c_sh_wr += 8 * c_sh_stride_2;
      }
    }
    __syncthreads();

  #pragma unroll
    for (int i = 0;
         i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&]() {
  #pragma unroll
    for (int i = 0; i < stages - 1; i++) fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  // Main loop.
  while (slice_iters) {
  // We unroll over both the global fetch and the register load pipeline to
  // ensure all shared memory accesses are static. Note that both pipelines have
  // even length meaning that the next iteration will always start at index 0.
  #pragma unroll
    for (int pipe = 0; pipe < stages;) {
      fetch_to_shared((pipe + stages - 1) % stages, pipe,
                      slice_iters >= stages);
      wait_for_stage();

      fetch_to_registers(pipe + 1, (pipe + 1) % stages);
      matmul(pipe);

      pipe++;
      slice_iters--;
      if (slice_iters == 0) break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    // Process results and, if necessary, proceed to the next column slice.
    // While this pattern may not be the most readable, other ways of writing
    // the loop seemed to noticeably worse performance after compilation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before
      // write-out
      if constexpr (group_blocks == -1) {
        if constexpr (num_bits == 8) {
          if (s_sh_wr_pred) cp_async4(&sh_s[s_sh_wr], &s[s_gl_rd]);
          cp_async_fence();
        } else {
          if (last) {
            if (s_sh_wr_pred) cp_async4(&sh_s[s_sh_wr], &s[s_gl_rd]);
            cp_async_fence();
          }
        }
      }
      thread_block_reduce();

      if constexpr (group_blocks == -1) {
        if constexpr (num_bits == 8) {
          cp_async_wait<0>();
          __syncthreads();
          if (threadIdx.x / 32 < thread_n_blocks / 4) {
            *(float4*)(frag_s) = *(float4*)(&sh_s[s_sh_rd]);
          }
        } else {
          if (last) {
            cp_async_wait<0>();
            __syncthreads();
            if (threadIdx.x / 32 < thread_n_blocks / 4) {
              *(float4*)(frag_s) = *(float4*)(&sh_s[s_sh_rd]);
            }
          }
        }
      }

      // For 8-bit channelwise, we apply the scale before the global reduction
      // that converts the fp32 results to fp16 (so that we avoid possible
      // overflow in fp16)
      if constexpr (group_blocks == -1 && num_bits == 8) {
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
          for (int i = 0; i < thread_m_blocks; i++) {
            scale_floats(&frag_c[i][0][0][0], &frag_c[i][1][0][0],
                         &frag_c[i][2][0][0], &frag_c[i][3][0][0], frag_s[0][0],
                         &frag_c[i][0][0][2], &frag_c[i][1][0][2],
                         &frag_c[i][2][0][2], &frag_c[i][3][0][2],
                         frag_s[0][2]);

            scale_floats(&frag_c[i][0][0][1], &frag_c[i][1][0][1],
                         &frag_c[i][2][0][1], &frag_c[i][3][0][1], frag_s[0][0],
                         &frag_c[i][0][0][3], &frag_c[i][1][0][3],
                         &frag_c[i][2][0][3], &frag_c[i][3][0][3],
                         frag_s[0][2]);

            scale_floats(&frag_c[i][0][1][0], &frag_c[i][1][1][0],
                         &frag_c[i][2][1][0], &frag_c[i][3][1][0], frag_s[0][0],
                         &frag_c[i][0][1][2], &frag_c[i][1][1][2],
                         &frag_c[i][2][1][2], &frag_c[i][3][1][2],
                         frag_s[0][2]);

            scale_floats(&frag_c[i][0][1][1], &frag_c[i][1][1][1],
                         &frag_c[i][2][1][1], &frag_c[i][3][1][1], frag_s[0][0],
                         &frag_c[i][0][1][3], &frag_c[i][1][1][3],
                         &frag_c[i][2][1][3], &frag_c[i][3][1][3],
                         frag_s[0][2]);
          }
        }
      }

      if (slice_count > 1) {  // only globally reduce if there is more than one
                              // block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last)  // only the last block in a slice actually writes the result
        write_result();

      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
  #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
  #pragma unroll
        for (int i = 0; i < m_sh_iters; i++)
          meta_ptr[i] += (m_sh_stride)-m_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
  #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++) B_ptr[i] -= b_gl_stride;
  #pragma unroll
          for (int i = 0; i < m_sh_iters; i++) meta_ptr[i] -= m_gl_stride;
        }
        s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}

#endif

#define CALL_IF_2_4(NUM_BITS, THREAD_M_BLOCKS, THREAD_N_BLOCKS,               \
                    THREAD_K_BLOCKS, GROUP_BLOCKS)                            \
  else if (num_bits == NUM_BITS && thread_m_blocks == THREAD_M_BLOCKS &&      \
           thread_n_blocks == THREAD_N_BLOCKS &&                              \
           thread_k_blocks == THREAD_K_BLOCKS &&                              \
           group_blocks == GROUP_BLOCKS) {                                    \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                     \
        Marlin_24<NUM_BITS), THREADS, THREAD_N_BLOCKS, THREAD_M_BLOCKS,        \
                  THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>,                     \
        hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);         \
    Marlin_24<NUM_BITS, THREADS, THREAD_N_BLOCKS, THREAD_M_BLOCKS,            \
              THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>                          \
        <<<blocks, THREADS, max_shared_mem, stream>>>(A_ptr, B_ptr, meta_ptr, \
                                                      C_ptr, s_ptr, prob_n,   \
                                                      prob_m, prob_k, locks); \
  }

void marlin_cuda_2_4(const void* A, const void* B, const void* meta, void* C,
                     void* s, int prob_m, int prob_n, int prob_k,
                     void* workspace, int num_bits, int groupsize = -1,
                     int dev = 0, hipStream_t stream = 0, int thread_k = -1,
                     int thread_m = -1, int sms = -1, int max_par = 16) {
  int tot_n = prob_n;
  int tot_n_blocks = ceildiv(tot_n, 16);
  int pad = 16 * tot_n_blocks - tot_n;

  if (sms == -1) {
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  }
  TORCH_CHECK(sms > 0);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  if (thread_k == -1 || thread_m == -1) {
    if (prob_n <= 16) {
      // For small batchizes, better partitioningif is slightly more important
      // than better compute utilization
      thread_k = 128;
      thread_m = 128;
    } else {
      thread_k = 64;
      thread_m = 256;
    }
  }

  int thread_k_blocks = thread_k / 32;  // 2:4 version with m16n8k32 instruction
  int thread_m_blocks = thread_m / 16;
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  int blocks = sms;

  TORCH_CHECK(prob_m % thread_m == 0, "prob_m = ", prob_m,
              " is not divisible by thread_m = ", thread_m);
  TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
              " is not divisible by thread_k = ", thread_k);
  if (group_blocks != -1) {
    TORCH_CHECK((prob_k / 2) % group_blocks == 0, "prob_k/2 = ", prob_k / 2,
                " is not divisible by group_blocks = ", group_blocks);
  }

  TORCH_CHECK(prob_m > 0 && prob_n > 0 && prob_k > 0, "Invalid MNK = [", prob_m,
              ", ", prob_n, ", ", prob_k, "]");

  const int4* A_ptr = (const int4*)A;
  const int4* B_ptr = (const int4*)B;
  const int4* meta_ptr = (const int4*)meta;
  int4* C_ptr = (int4*)C;
  const int4* s_ptr = (const int4*)s;

  int* locks = (int*)workspace;
  for (int i = 0; i < tot_n_blocks; i += 4) {
    int thread_n_blocks = tot_n_blocks - i;
    prob_n = tot_n - 16 * i;
    int par = 1;
    if (thread_n_blocks > 4) {
      // Note that parallel > 1 currently only works for inputs without any
      // padding
      par = (16 * thread_n_blocks - pad) / 64;
      if (par > max_par) par = max_par;
      prob_n = 64 * par;
      i += 4 * (par - 1);
      thread_n_blocks = 4;
    }

    // For compilation speed, we only define the kernel configurations that have
    // seemed useful (in terms of performance) in our testing, however many more
    // are, in principle, possible.

    // the false is start of the CALL_IF macros
    if (false) {
    }  //         BMxBNxBK,   group
    // 4-bit
    CALL_IF_2_4(4, 8, 1, 4, -1)   // e.g., 16x128x128
    CALL_IF_2_4(4, 8, 1, 4, 4)    // e.g., 16x128x128, 64
    CALL_IF_2_4(4, 16, 1, 2, -1)  // e.g., 16x256x64
    CALL_IF_2_4(4, 16, 1, 2, 4)   // e.g., 16x256x64,  64
    CALL_IF_2_4(4, 16, 2, 2, -1)  // e.g.. 32x256x64
    CALL_IF_2_4(4, 16, 2, 2, 4)
    CALL_IF_2_4(4, 16, 3, 2, -1)
    CALL_IF_2_4(4, 16, 3, 2, 4)
    CALL_IF_2_4(4, 16, 4, 2, -1)
    CALL_IF_2_4(4, 16, 4, 2, 4)

    // 8-bit
    CALL_IF_2_4(8, 8, 1, 4, -1)   // e.g., 16x128x128
    CALL_IF_2_4(8, 8, 1, 4, 4)    // e.g., 16x128x128, 64
    CALL_IF_2_4(8, 16, 1, 2, -1)  // e.g., 16x256x64
    CALL_IF_2_4(8, 16, 1, 2, 4)   // e.g., 16x256x64,  64
    CALL_IF_2_4(8, 16, 2, 2, -1)  // e.g.. 32x256x64
    CALL_IF_2_4(8, 16, 2, 2, 4)
    CALL_IF_2_4(8, 16, 3, 2, -1)
    CALL_IF_2_4(8, 16, 3, 2, 4)
    CALL_IF_2_4(8, 16, 4, 2, -1)
    CALL_IF_2_4(8, 16, 4, 2, 4)
    else {
      throw std::runtime_error("Unsupported shapes: MKN = [" + str(prob_m) +
                               ", " + str(prob_k) + ", " + str(prob_n) + "]" +
                               ", groupsize = " + str(groupsize) +
                               ", thread_m_blocks = " + str(thread_m_blocks) +
                               ", thread_n_blocks = " + str(thread_n_blocks) +
                               ", thread_k_blocks = " + str(thread_k_blocks));
    }

    A_ptr += 16 * thread_n_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_n_blocks * (prob_m / 8) * par;
  }
}

}  // namespace marlin_24

torch::Tensor gptq_marlin_24_gemm(torch::Tensor& a, torch::Tensor& b_q_weight,
                                  torch::Tensor& b_meta,
                                  torch::Tensor& b_scales,
                                  torch::Tensor& workspace, int64_t num_bits,
                                  int64_t size_m, int64_t size_n,
                                  int64_t size_k) {
  // Verify num_bits
  TORCH_CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  int pack_factor = 32 / num_bits;

  // Verify M
  TORCH_CHECK(size_m == a.size(0),
              "Shape mismatch: a.size(0) = " + str(a.size(0)) +
                  ", size_m = " + str(size_m));

  // Verify K
  TORCH_CHECK(size_k == a.size(1),
              "Shape mismatch: a.size(1) = " + str(a.size(1)) +
                  ", size_k = " + str(size_k));
  TORCH_CHECK(size_k % marlin_24::tile_size == 0,
              "size_k = " + str(size_k) + " is not divisible by tile_size = " +
                  str(marlin_24::tile_size));
  TORCH_CHECK((size_k / marlin_24::tile_size / 2) == b_q_weight.size(0),
              "Shape mismatch: b_q_weight.size(0) = " +
                  str(b_q_weight.size(0)) + ", size_k = " + str(size_k) +
                  ", tile_size = " + str(marlin_24::tile_size));

  // Verify N
  TORCH_CHECK(b_scales.size(1) == size_n,
              "b_scales.size(1) = " + str(b_scales.size(1)) +
                  ", size_n = " + str(size_n));
  TORCH_CHECK(
      b_q_weight.size(1) % marlin_24::tile_size == 0,
      "b_q_weight.size(1) = " + str(b_q_weight.size(1)) +
          " is not divisible by tile_size = " + str(marlin_24::tile_size));

  int actual_size_n = (b_q_weight.size(1) / marlin_24::tile_size) * pack_factor;
  TORCH_CHECK(
      size_n == actual_size_n,
      "size_n = " + str(size_n) + ", actual_size_n = " + str(actual_size_n));

  // Verify meta
  TORCH_CHECK(b_meta.size(0) == size_k / 8 / 2 / 2,
              "b_meta.size(0) = ", b_meta.size(0),
              " is not size_k / 8 / 2 / 2 = ", size_k / 8 / 2 / 2);
  TORCH_CHECK(b_meta.size(1) == size_n * 2, "b_meta.size(1) = ", b_meta.size(1),
              " is not size_n * 2 = ", size_n * 2);

  // Verify A device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  // Verify B device and strides
  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  // Verify b_meta device and strides
  TORCH_CHECK(b_meta.device().is_cuda(), "b_meta is not on GPU");
  TORCH_CHECK(b_meta.is_contiguous(), "b_meta is not contiguous");

  // Verify scales device and strides
  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  // Alloc C matrix
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c = torch::empty({size_m, size_n}, options);

  int thread_k = -1;
  int thread_m = -1;
  int sms = -1;
  int max_par = 16;

  int groupsize = -1;
  if (b_scales.size(0) > 1) {
    TORCH_CHECK(size_k % b_scales.size(0) == 0,
                "size_k = " + str(size_k) +
                    ", is not divisible by b_scales.size(0) = " +
                    str(b_scales.size(0)));
    groupsize = size_k / b_scales.size(0);
    groupsize /= 2;  // Because of 24
  }

  // Verify groupsize
  TORCH_CHECK(groupsize == -1 || groupsize == 64,
              "Unexpected groupsize = " + str(groupsize));

  // Verify workspace size
  TORCH_CHECK(size_n % marlin_24::min_thread_n == 0,
              "size_n = " + str(size_n) +
                  ", is not divisible by min_thread_n = " +
                  str(marlin_24::min_thread_n));
  int min_workspace_size =
      (size_n / marlin_24::min_thread_n) * marlin_24::max_par;
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = " + str(workspace.numel()) +
                  " is below min_workspace_size = " + str(min_workspace_size));

  int dev = a.get_device();
  marlin_24::marlin_cuda_2_4(
      a.data_ptr(), b_q_weight.data_ptr(), b_meta.data_ptr(), c.data_ptr(),
      b_scales.data_ptr(), size_n, size_m, size_k, workspace.data_ptr(),
      num_bits, groupsize, dev, at::cuda::getCurrentCUDAStream(dev), thread_k,
      thread_m, sms, max_par);

  return c;
}
