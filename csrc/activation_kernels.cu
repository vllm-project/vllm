#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

template<typename T>
__device__ __forceinline__ T silu(const T& x) {
  // x * sigmoid(x)
  return (T) (((float) x) / (1.0f + expf((float) -x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
  scalar_t* __restrict__ out,               // [..., d]
  const scalar_t* __restrict__ input,       // [..., 2, d]
  const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = silu(x) * y;
  }
}

} // namespace vllm

void silu_and_mul(
  torch::Tensor& out,      // [..., d]
  torch::Tensor& input)    // [..., 2 * d]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int d = input.size(-1) / 2;

  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "silu_and_mul_kernel",
    [&] {
      vllm::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        d);
    });
}

namespace vllm {

// Element-wise activation kernel template.
template<typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void activation_kernel(
  scalar_t* __restrict__ out,               // [..., d]
  const scalar_t* __restrict__ input,       // [..., d]
  const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * d + idx]);
    out[token_idx * d + idx] = ACT_FN(x);
  }
}

} // namespace vllm

// Launch element-wise activation kernel.
#define LAUNCH_ACTIVATION_KERNEL(KERNEL)                                                  \
  int d = input.size(-1);                                                                 \
  int64_t num_tokens = input.numel() / d;                                                 \
  dim3 grid(num_tokens);                                                                  \
  dim3 block(std::min(d, 1024));                                                          \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();                           \
  VLLM_DISPATCH_FLOATING_TYPES(                                                           \
    input.scalar_type(),                                                                  \
    "activation_kernel",                                                                  \
    [&] {                                                                                 \
      vllm::activation_kernel<scalar_t, KERNEL<scalar_t>><<<grid, block, 0, stream>>>(    \
        out.data_ptr<scalar_t>(),                                                         \
        input.data_ptr<scalar_t>(),                                                       \
        d);                                                                               \
    });

namespace vllm {

template<typename T>
__device__ __forceinline__ T gelu_new_kernel(const T& x) {
  const float x3 = (float) (x * x * x);
  const T t = (T) tanhf((T) (0.79788456f * (float) (x + (T) (0.044715f * x3))));
  return ((T) 0.5) * x * (((T) 1.0) + t);
}

template<typename T>
__device__ __forceinline__ T gelu_fast_kernel(const T& x) {
  const float f = (float) x;
  const T t = (T) tanhf(((T) (f * 0.79788456f)) * (((T) 1.0) + (T) (0.044715f * f) * x));
  return ((T) 0.5) * x * (((T) 1.0) + t);
}

} // namespace vllm

void gelu_new(
  torch::Tensor& out,     // [..., d]
  torch::Tensor& input)   // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_new_kernel);
}

void gelu_fast(
  torch::Tensor& out,     // [..., d]
  torch::Tensor& input)   // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_fast_kernel);
}
