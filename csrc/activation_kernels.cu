#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

namespace vllm {

template<typename T>
__device__ __forceinline__ T silu(const T& x) {
  // x * sigmoid(x)
  return (T) (((float) x) / (1.0f + expf((float) -x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
  scalar_t* __restrict__ out,               // [num_tokens, d]
  const scalar_t* __restrict__ input,       // [num_tokens, 2, d]
  const int d) {
  const int token_idx = blockIdx.x;
  for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = __ldg(&input[token_idx * 2 * d + idx]);
    const scalar_t y = __ldg(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = silu(x) * y;
  }
}

} // namespace vllm

void silu_and_mul(
  torch::Tensor& out,      // [num_tokens, d]
  torch::Tensor& input)    // [num_tokens, 2 * d]
{
  int num_tokens = input.size(0);
  int d = input.size(1) / 2;

  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    input.scalar_type(),
    "silu_and_mul_kernel",
    [&] {
      vllm::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        d);
    });
}
