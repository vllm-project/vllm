#include "hip/hip_runtime.h"
/*
 * Adapted from
 * https://github.com/NVIDIA/FasterTransformer/blob/release/v5.3_tag/src/fastertransformer/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_template.hpp
 * Copyright (c) 2023, The vLLM team.
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "attention_kernels.cuh"

#ifndef USE_ROCM
  #define WARP_SIZE 32
#else
  #define WARP_SIZE warpSize
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#define LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE)                                \
  VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(                     \
      ((void*)vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE,        \
                                              BLOCK_SIZE, NUM_THREADS,      \
                                              KV_DTYPE, IS_BLOCK_SPARSE>),  \
      shared_mem_size);                                                     \
  vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE>   \
      <<<grid, block, shared_mem_size, stream>>>(                           \
          out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, \
          scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
          k_scale, v_scale, tp_rank, blocksparse_local_blocks,              \
          blocksparse_vert_stride, blocksparse_block_size,                  \
          blocksparse_head_sliding_step);

// TODO(woosuk): Tune NUM_THREADS.
template <typename T, typename CACHE_T, int BLOCK_SIZE,
          vllm::Fp8KVCacheDataType KV_DTYPE, bool IS_BLOCK_SPARSE,
          int NUM_THREADS = 128>
void paged_attention_v1_launcher(
    torch::Tensor& out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& seq_lens, int max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes, float k_scale,
    float v_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  [[maybe_unused]] int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* seq_lens_ptr = seq_lens.data_ptr<int>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int padded_max_seq_len =
      DIVIDE_ROUND_UP(max_seq_len, BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_seq_len * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);
  // Python-side check in vllm.worker.worker._check_if_can_support_max_seq_len
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size);

  dim3 grid(num_heads, num_seqs, 1);
  dim3 block(NUM_THREADS);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model. However, we can easily extend this
    // to support any head size which is a multiple of 16.
    case 64:
      LAUNCH_PAGED_ATTENTION_V1(64);
      break;
    case 80:
      LAUNCH_PAGED_ATTENTION_V1(80);
      break;
    case 96:
      LAUNCH_PAGED_ATTENTION_V1(96);
      break;
    case 112:
      LAUNCH_PAGED_ATTENTION_V1(112);
      break;
    case 120:
      LAUNCH_PAGED_ATTENTION_V1(120);
      break;
    case 128:
      LAUNCH_PAGED_ATTENTION_V1(128);
      break;
    case 192:
      LAUNCH_PAGED_ATTENTION_V1(192);
      break;
    case 256:
      LAUNCH_PAGED_ATTENTION_V1(256);
      break;
    default:
      TORCH_CHECK(false, "Unsupported head size: ", head_size);
      break;
  }
}

#define CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, KV_DTYPE, IS_BLOCK_SPARSE)  \
  paged_attention_v1_launcher<T, CACHE_T, BLOCK_SIZE, KV_DTYPE,              \
                              IS_BLOCK_SPARSE>(                              \
      out, query, key_cache, value_cache, num_kv_heads, scale, block_tables, \
      seq_lens, max_seq_len, alibi_slopes, k_scale, v_scale, tp_rank,        \
      blocksparse_local_blocks, blocksparse_vert_stride,                     \
      blocksparse_block_size, blocksparse_head_sliding_step);

#define CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE) \
  switch (is_block_sparse) {                                               \
    case true:                                                             \
      CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, true);     \
      break;                                                               \
    case false:                                                            \
      CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, false);    \
      break;                                                               \
  }

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V1_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
  switch (block_size) {                                           \
    case 8:                                                       \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 8, KV_DTYPE);         \
      break;                                                      \
    case 16:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
      break;                                                      \
    case 32:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 32, KV_DTYPE);        \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

void paged_attention_v1(
    torch::Tensor& out,    // [num_seqs, num_heads, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,       // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads,  // [num_heads]
    double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& seq_lens,      // [num_seqs]
    int64_t block_size, int64_t max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, double k_scale, double v_scale,
    const int64_t tp_rank, const int64_t blocksparse_local_blocks,
    const int64_t blocksparse_vert_stride, const int64_t blocksparse_block_size,
    const int64_t blocksparse_head_sliding_step) {
  const bool is_block_sparse = (blocksparse_vert_stride > 1);

  DISPATCH_BY_KV_CACHE_DTYPE(query.dtype(), kv_cache_dtype,
                             CALL_V1_LAUNCHER_BLOCK_SIZE)
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP