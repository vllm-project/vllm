#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <assert.h>

#include <hip/hip_runtime.h>

#include <torch/all.h>

__device__ int64_t save_blocks(int* block_offset, int64_t range_start,
                               int64_t range_end, int64_t block_size,
                               int64_t input_block_count, int64_t kv_seqlen) {
  if (range_start >= kv_seqlen) {
    return input_block_count;
  }
  if (range_end > kv_seqlen) {
    range_end = kv_seqlen;
  }
  int64_t current_block_count = input_block_count;
  for (int idx = range_start; idx < range_end; idx += block_size) {
    block_offset[current_block_count++] = idx;
  }
  return current_block_count;
}

__global__ void convert_vertical_slash_indexes_kernel(
    const int* q_seqlens,         // [BATCH, ]
    const int* kv_seqlens,        // [BATCH, ]
    const int* vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    const int* slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int* block_count,             // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* block_offset,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_S]
    int* column_count,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* column_index,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_V]
    int64_t N_HEADS, int64_t N_ROWS, int64_t BLOCK_SIZE_M, int64_t BLOCK_SIZE_N,
    int64_t NNZ_V, int64_t NNZ_S,
    bool causal  // True for intra, False for succ
) {
  const int batch_idx = blockIdx.y;
  const int head_idx = blockIdx.x;
  const int group_idx = blockIdx.z;

  int64_t q_seqlen = q_seqlens[batch_idx];
  int64_t kv_seqlen = kv_seqlens[batch_idx];
  int64_t block_idx_m = group_idx * blockDim.x + threadIdx.x;
  int64_t start_m = block_idx_m * BLOCK_SIZE_M;
  if (start_m >= q_seqlen) {
    return;
  }
  int64_t end_m = start_m + BLOCK_SIZE_M;
  vertical_indexes += (batch_idx * N_HEADS + head_idx) * NNZ_V;
  slash_indexes += (batch_idx * N_HEADS + head_idx) * NNZ_S;
  int64_t row_offset = (batch_idx * N_HEADS + head_idx) * N_ROWS + block_idx_m;
  block_count += row_offset;
  block_offset += row_offset * NNZ_S;
  column_count += row_offset;
  column_index += row_offset * NNZ_V;

  bool has_slash = true;
  int64_t tmp_col_cnt = 0, tmp_blk_cnt = 0;
  int64_t s = 0, v = 0;
  int64_t v_idx = vertical_indexes[v++];
  int64_t s_idx = slash_indexes[s++];
  if (causal) {
    while (s_idx >= end_m + (kv_seqlen - q_seqlen) && s < NNZ_S) {
      s_idx = slash_indexes[s++];
    }
    if (s_idx > end_m + (kv_seqlen - q_seqlen)) has_slash = false;
    s_idx = max((kv_seqlen - q_seqlen) + end_m - s_idx, BLOCK_SIZE_M);
  } else {
    while (s_idx >= end_m + kv_seqlen && s < NNZ_S) {
      s_idx = slash_indexes[s++];
    }
    if (s_idx > end_m + kv_seqlen) has_slash = false;
    s_idx = max(kv_seqlen + end_m - s_idx, BLOCK_SIZE_M);
  }

  int64_t range_start = s_idx - BLOCK_SIZE_M, range_end = s_idx;
  if (!has_slash) {
    if (causal) {
      range_start = (kv_seqlen - q_seqlen) + end_m;
      range_end = (kv_seqlen - q_seqlen) + end_m + BLOCK_SIZE_N;
    } else {
      range_start = kv_seqlen;
      range_end = kv_seqlen + BLOCK_SIZE_N;
    }
  }

  bool slash_finished = false;
  while (1) {
    if (v_idx < range_end) {
      if (v_idx < range_start) {
        column_index[tmp_col_cnt++] = v_idx;
      }
      if (v < NNZ_V) {
        v_idx = vertical_indexes[v++];
      } else {
        if (causal)
          v_idx = end_m + BLOCK_SIZE_N + (kv_seqlen - q_seqlen);
        else
          v_idx = end_m + BLOCK_SIZE_N + kv_seqlen;
      }
    } else {
      if ((s < NNZ_S && causal) ||
          (s < NNZ_S && !causal && slash_indexes[s] >= start_m)) {
        if (causal)
          s_idx = max((kv_seqlen - q_seqlen) + end_m - slash_indexes[s++],
                      BLOCK_SIZE_M);
        else
          s_idx = max(kv_seqlen + end_m - slash_indexes[s++], BLOCK_SIZE_M);
      } else {
        if (v == NNZ_V || (v_idx > range_start && causal)) {
          // add the last vertical if no more slash
          if (v == NNZ_V && !causal && v_idx < kv_seqlen) {
            column_index[tmp_col_cnt++] = v_idx;
          }
          tmp_blk_cnt = save_blocks(block_offset, range_start, range_end,
                                    BLOCK_SIZE_N, tmp_blk_cnt, kv_seqlen);
          break;
        } else {
          if (causal) {
            range_start = (kv_seqlen - q_seqlen) + end_m;
            range_end = (kv_seqlen - q_seqlen) + end_m + BLOCK_SIZE_N;
          } else {
            // if slash_finished but there are vertical left, save current
            // blocks
            tmp_blk_cnt = save_blocks(block_offset, range_start, range_end,
                                      BLOCK_SIZE_N, tmp_blk_cnt, kv_seqlen);
            range_start = kv_seqlen;
            range_end = kv_seqlen + BLOCK_SIZE_N;
          }
          slash_finished = true;
        }
      }
      if (!slash_finished) {
        if (s_idx > range_end + BLOCK_SIZE_M) {
          tmp_blk_cnt = save_blocks(block_offset, range_start, range_end,
                                    BLOCK_SIZE_N, tmp_blk_cnt, kv_seqlen);
          range_start = s_idx - BLOCK_SIZE_M;
          range_end = s_idx;
        } else if (s_idx > range_end) {
          range_end += BLOCK_SIZE_M;
        }
      }
    }
  }

  block_count[0] = tmp_blk_cnt;
  column_count[0] = tmp_col_cnt;
}

void convert_vertical_slash_indexes_64x64(
    const int* q_seqlens,         // [BATCH, ]
    const int* kv_seqlens,        // [BATCH, ]
    const int* vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    const int* slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int* block_count,             // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* block_offset,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_S]
    int* column_count,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* column_index,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_V]
    int64_t BATCH_SIZE, int64_t N_HEADS, int64_t N_ROWS, int64_t BLOCK_SIZE_M,
    int64_t BLOCK_SIZE_N, int64_t NNZ_V, int64_t NNZ_S, bool causal) {
  const int N_THREADS = 64;
  const dim3 dimBlock(N_THREADS);
  const dim3 dimGrid(N_HEADS, BATCH_SIZE, (N_ROWS + N_THREADS - 1) / N_THREADS);
  convert_vertical_slash_indexes_kernel<<<dimGrid, dimBlock>>>(
      q_seqlens, kv_seqlens, vertical_indexes, slash_indexes, block_count,
      block_offset, column_count, column_index, N_HEADS, N_ROWS, BLOCK_SIZE_M,
      BLOCK_SIZE_N, NNZ_V, NNZ_S, causal);
}

/**
 * Implements the Algorithm 4 in paper https://arxiv.org/abs/2407.02490.
 *
 * This function builds the index of each row of blocks from vertical indices
 * and slash indices. The vertical indices are treated as points, while the
 * slash indices are converted as ranges. The output consists of the merged
 * ranges and separate column indices, where the ranges are represented by
 * block indices.
 *
 * The implementation is referenced from the original MInference repo:
 * https://github.com/microsoft/MInference/blob/main/csrc/vertical_slash_index.cu.
 */
void convert_vertical_slash_indexes(
    torch::Tensor& block_count,      // [BATCH, N_HEADS, NUM_ROWS]
    torch::Tensor& block_offset,     // [BATCH, N_HEADS, NUM_ROWS, NNZ_S]
    torch::Tensor& column_count,     // [BATCH, N_HEADS, NUM_ROWS]
    torch::Tensor& column_index,     // [BATCH, N_HEADS, NUM_ROWS, NNZ_V]
    torch::Tensor q_seqlens,         // [BATCH, ]
    torch::Tensor kv_seqlens,        // [BATCH, ]
    torch::Tensor vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    torch::Tensor slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int64_t context_size, int64_t block_size_M, int64_t block_size_N,
    bool causal) {
  hipSetDevice(q_seqlens.get_device());

  int batch_size = slash_indexes.size(0);
  int num_heads = slash_indexes.size(1);
  int nnz_slash = slash_indexes.size(2);
  int nnz_vertical = vertical_indexes.size(2);
  int num_rows = (context_size + block_size_M - 1) / block_size_M;

  convert_vertical_slash_indexes_64x64(
      q_seqlens.data_ptr<int>(), kv_seqlens.data_ptr<int>(),
      vertical_indexes.data_ptr<int>(), slash_indexes.data_ptr<int>(),
      block_count.data_ptr<int>(), block_offset.data_ptr<int>(),
      column_count.data_ptr<int>(), column_index.data_ptr<int>(), batch_size,
      num_heads, num_rows, block_size_M, block_size_N, nnz_vertical, nnz_slash,
      causal);
}

__global__ void convert_vertical_slash_indexes_kernel_mergehead(
    const int* q_seqlens,         // [BATCH, ]
    const int* kv_seqlens,        // [BATCH, ]
    const int* vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    const int* slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    const int* per_head_vertical_topkv, const int* per_head_slash_topkv,
    int* block_count,   // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* block_offset,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_S]
    int* column_count,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* column_index,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_V]
    int64_t N_HEADS, int64_t N_ROWS, int64_t BLOCK_SIZE_M, int64_t BLOCK_SIZE_N,
    int64_t NNZ_V, int64_t NNZ_S,
    bool causal  // True for intra, False for succ
) {
  const int batch_idx = blockIdx.y;
  const int head_idx = blockIdx.x;
  const int group_idx = blockIdx.z;

  int64_t q_seqlen = q_seqlens[batch_idx];
  int64_t kv_seqlen = kv_seqlens[batch_idx];
  int64_t block_idx_m = group_idx * blockDim.x + threadIdx.x;
  int64_t start_m = block_idx_m * BLOCK_SIZE_M;
  if (start_m >= q_seqlen) {
    return;
  }
  int64_t end_m = start_m + BLOCK_SIZE_M;
  vertical_indexes += (batch_idx * N_HEADS + head_idx) * NNZ_V;
  slash_indexes += (batch_idx * N_HEADS + head_idx) * NNZ_S;
  int64_t row_offset = (batch_idx * N_HEADS + head_idx) * N_ROWS + block_idx_m;
  block_count += row_offset;
  block_offset += row_offset * NNZ_S;
  column_count += row_offset;
  column_index += row_offset * NNZ_V;

  // MergeHead: each head has it's unique max topk NNZ_V，NNZ_S. (NNZ_V，NNZ_S
  // above is buffer size, use to compute offset)
  NNZ_S = per_head_slash_topkv[head_idx];
  NNZ_V = per_head_vertical_topkv[head_idx];

  bool has_slash = true;
  int64_t tmp_col_cnt = 0, tmp_blk_cnt = 0;
  int64_t s = 0, v = 0;
  int64_t v_idx = vertical_indexes[v++];
  int64_t s_idx = slash_indexes[s++];
  if (causal) {
    while (s_idx >= end_m + (kv_seqlen - q_seqlen) && s < NNZ_S) {
      s_idx = slash_indexes[s++];
    }
    if (s_idx > end_m + (kv_seqlen - q_seqlen)) has_slash = false;
    s_idx = max((kv_seqlen - q_seqlen) + end_m - s_idx, BLOCK_SIZE_M);
  } else {
    while (s_idx >= end_m + kv_seqlen && s < NNZ_S) {
      s_idx = slash_indexes[s++];
    }
    if (s_idx > end_m + kv_seqlen) has_slash = false;
    s_idx = max(kv_seqlen + end_m - s_idx, BLOCK_SIZE_M);
  }

  int64_t range_start = s_idx - BLOCK_SIZE_M, range_end = s_idx;
  if (!has_slash) {
    if (causal) {
      range_start = (kv_seqlen - q_seqlen) + end_m;
      range_end = (kv_seqlen - q_seqlen) + end_m + BLOCK_SIZE_N;
    } else {
      range_start = kv_seqlen;
      range_end = kv_seqlen + BLOCK_SIZE_N;
    }
  }

  bool slash_finished = false;
  while (1) {
    if (v_idx < range_end) {
      if (v_idx < range_start) {
        column_index[tmp_col_cnt++] = v_idx;
      }
      if (v < NNZ_V) {
        v_idx = vertical_indexes[v++];
      } else {
        if (causal)
          v_idx = end_m + BLOCK_SIZE_N + (kv_seqlen - q_seqlen);
        else
          v_idx = end_m + BLOCK_SIZE_N + kv_seqlen;
      }
    } else {
      if ((s < NNZ_S && causal) ||
          (s < NNZ_S && !causal && slash_indexes[s] >= start_m)) {
        if (causal)
          s_idx = max((kv_seqlen - q_seqlen) + end_m - slash_indexes[s++],
                      BLOCK_SIZE_M);
        else
          s_idx = max(kv_seqlen + end_m - slash_indexes[s++], BLOCK_SIZE_M);
      } else {
        if (v == NNZ_V || (v_idx > range_start && causal)) {
          // add the last vertical if no more slash
          if (v == NNZ_V && !causal && v_idx < kv_seqlen) {
            column_index[tmp_col_cnt++] = v_idx;
          }
          tmp_blk_cnt = save_blocks(block_offset, range_start, range_end,
                                    BLOCK_SIZE_N, tmp_blk_cnt, kv_seqlen);
          break;
        } else {
          if (causal) {
            range_start = (kv_seqlen - q_seqlen) + end_m;
            range_end = (kv_seqlen - q_seqlen) + end_m + BLOCK_SIZE_N;
          } else {
            // if slash_finished but there are vertical left, save current
            // blocks
            tmp_blk_cnt = save_blocks(block_offset, range_start, range_end,
                                      BLOCK_SIZE_N, tmp_blk_cnt, kv_seqlen);
            range_start = kv_seqlen;
            range_end = kv_seqlen + BLOCK_SIZE_N;
          }
          slash_finished = true;
        }
      }
      if (!slash_finished) {
        if (s_idx > range_end + BLOCK_SIZE_M) {
          tmp_blk_cnt = save_blocks(block_offset, range_start, range_end,
                                    BLOCK_SIZE_N, tmp_blk_cnt, kv_seqlen);
          range_start = s_idx - BLOCK_SIZE_M;
          range_end = s_idx;
        } else if (s_idx > range_end) {
          range_end += BLOCK_SIZE_M;
        }
      }
    }
  }

  block_count[0] = tmp_blk_cnt;
  column_count[0] = tmp_col_cnt;
}

void convert_vertical_slash_indexes_64x64_mergehead(
    const int* q_seqlens,         // [BATCH, ]
    const int* kv_seqlens,        // [BATCH, ]
    const int* vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    const int* slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int* per_head_vertical_topkv, int* per_head_slash_topkv,
    int* block_count,   // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* block_offset,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_S]
    int* column_count,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* column_index,  // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_V]
    int64_t BATCH_SIZE, int64_t N_HEADS, int64_t N_ROWS, int64_t BLOCK_SIZE_M,
    int64_t BLOCK_SIZE_N, int64_t NNZ_V, int64_t NNZ_S, bool causal) {
  const int N_THREADS = 64;
  const dim3 dimBlock(N_THREADS);
  const dim3 dimGrid(N_HEADS, BATCH_SIZE, (N_ROWS + N_THREADS - 1) / N_THREADS);
  convert_vertical_slash_indexes_kernel_mergehead<<<dimGrid, dimBlock>>>(
      q_seqlens, kv_seqlens, vertical_indexes, slash_indexes,
      per_head_vertical_topkv, per_head_slash_topkv, block_count, block_offset,
      column_count, column_index, N_HEADS, N_ROWS, BLOCK_SIZE_M, BLOCK_SIZE_N,
      NNZ_V, NNZ_S, causal);
}

/**
 * Implements the Algorithm 4 in paper https://arxiv.org/abs/2407.02490.
 *
 * Like the above convert_vertical_slash_indexes, but with
 * pre-computed vertical and slash counts.
 */
void convert_vertical_slash_indexes_mergehead(
    torch::Tensor& block_count,            // [BATCH, N_HEADS, NUM_ROWS]
    torch::Tensor& block_offset,           // [BATCH, N_HEADS, NUM_ROWS, NNZ_S]
    torch::Tensor& column_count,           // [BATCH, N_HEADS, NUM_ROWS]
    torch::Tensor& column_index,           // [BATCH, N_HEADS, NUM_ROWS, NNZ_V]
    torch::Tensor q_seqlens,               // [BATCH, ]
    torch::Tensor kv_seqlens,              // [BATCH, ]
    torch::Tensor vertical_indexes,        // [BATCH, N_HEADS, NNZ_V]
    torch::Tensor slash_indexes,           // [BATCH, N_HEADS, NNZ_S]
    torch::Tensor vertical_indices_count,  // [N_HEADS, ]
    torch::Tensor slash_indices_count,     // [N_HEADS, ]
    int64_t context_size, int64_t block_size_M, int64_t block_size_N,
    bool causal) {
  hipSetDevice(q_seqlens.get_device());

  int batch_size = slash_indexes.size(0);
  int num_heads = slash_indexes.size(1);
  int nnz_slash = slash_indexes.size(2);
  int nnz_vertical = vertical_indexes.size(2);
  int num_rows = (context_size + block_size_M - 1) / block_size_M;

  convert_vertical_slash_indexes_64x64_mergehead(
      q_seqlens.data_ptr<int>(), kv_seqlens.data_ptr<int>(),
      vertical_indexes.data_ptr<int>(), slash_indexes.data_ptr<int>(),
      vertical_indices_count.data_ptr<int>(),
      slash_indices_count.data_ptr<int>(), block_count.data_ptr<int>(),
      block_offset.data_ptr<int>(), column_count.data_ptr<int>(),
      column_index.data_ptr<int>(), batch_size, num_heads, num_rows,
      block_size_M, block_size_N, nnz_vertical, nnz_slash, causal);
}
