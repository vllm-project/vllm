#include "hip/hip_runtime.h"
// clang-format will break include orders
// clang-format off
#include <cudaTypedefs.h>

#if defined CUDA_VERSION && CUDA_VERSION >= 12020
#include "sparse_scaled_mm_c3x.cuh"
// clang-format on

using namespace cute;
using namespace vllm;

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm90_fp8_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                    torch::Tensor const& bt_nzs,
                                    torch::Tensor const& bt_meta,
                                    EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(bt_meta.dtype() == torch::kUInt8);
  TORCH_CHECK(bt_nzs.dtype() == torch::kFloat8_e4m3fn);

  using Cutlass3xGemmDefault =
      typename sm90_config_default<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_fp8_config_M64<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_fp8_config_M128<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM256 =
      typename sm90_fp8_config_M256<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM512 =
      typename sm90_fp8_config_M512<InType, OutType, Epilogue>::Cutlass3xGemm;

  using Cutlass3xGemm1 =
      typename sm90_fp8_config_1<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm2 =
      typename sm90_fp8_config_2<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm3 =
      typename sm90_fp8_config_3<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm4 =
      typename sm90_fp8_config_4<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm5 =
      typename sm90_fp8_config_5<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm6 =
      typename sm90_fp8_config_6<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm7 =
      typename sm90_fp8_config_7<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemm8 =
      typename sm90_fp8_config_8<InType, OutType, Epilogue>::Cutlass3xGemm;

  uint32_t const n = bt_nzs.size(0);
  uint32_t const m = a.size(0);  // Batch size
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(64), next_pow_2(m));  // next power of 2

  if (mp2 <= 64) {
    if (n == 28672) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm2>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else if (n == 4096 || n == 6144) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm1>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    }
  } else if (mp2 <= 128) {
    if (n == 4096) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm3>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else if (n == 28672) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm5>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else if (n == 6144) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm4>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    }
  } else if (mp2 <= 256) {
    if (n == 4096) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm6>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else if (n == 28672) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm8>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else if (n == 6144) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm7>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    }
  } else {
    if (n == 6144 || n == 28672) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm8>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else if (n == 4096) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemm7>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    }
  }

  // Otherwise the default heuristic
  if (mp2 <= 64) {
    // n in [1, 64]
    return cutlass_sparse_gemm_caller<Cutlass3xGemmM64>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 128) {
    // n in (64, 128]
    return cutlass_sparse_gemm_caller<Cutlass3xGemmM128>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 256) {
    // n in (128, 256]
    return cutlass_sparse_gemm_caller<Cutlass3xGemmM256>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  } else {
    // n in (256, inf)
    return cutlass_sparse_gemm_caller<Cutlass3xGemmM512>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  }
}

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm90_fp16_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& bt_nzs,
                                     torch::Tensor const& bt_meta,
                                     EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, cutlass::half_t>());
  TORCH_CHECK(a.dtype() == torch::kFloat16);
  TORCH_CHECK(bt_meta.dtype() == torch::kUInt8);
  TORCH_CHECK(bt_nzs.dtype() == torch::kFloat16);

  using Cutlass3xGemmDefault =
      typename sm90_config_default<InType, OutType, Epilogue>::Cutlass3xGemm;

  // m in (128, inf)
  return cutlass_sparse_gemm_caller<Cutlass3xGemmDefault>(
      out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
}

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm90_bf16_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& bt_nzs,
                                     torch::Tensor const& bt_meta,
                                     EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, cutlass::bfloat16_t>());
  TORCH_CHECK(a.dtype() == torch::kBFloat16);
  TORCH_CHECK(bt_meta.dtype() == torch::kUInt8);
  TORCH_CHECK(bt_nzs.dtype() == torch::kBFloat16);

  using Cutlass3xGemmDefault =
      typename sm90_config_default<InType, OutType, Epilogue>::Cutlass3xGemm;

  // m in (128, inf)
  return cutlass_sparse_gemm_caller<Cutlass3xGemmDefault>(
      out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
}

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm90_int8_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& bt_nzs,
                                     torch::Tensor const& bt_meta,
                                     EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, int8_t>());
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(bt_meta.dtype() == torch::kUInt8);
  TORCH_CHECK(bt_nzs.dtype() == torch::kInt8);

  using Cutlass3xGemmDefault =
      typename sm90_config_default<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_int8_config_M128<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_int8_config_M64<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM32NBig =
      typename sm90_int8_config_M32_NBig<InType, OutType,
                                         Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM32NSmall =
      typename sm90_int8_config_M32_NSmall<InType, OutType,
                                           Epilogue>::Cutlass3xGemm;

  uint32_t const n = out.size(1);
  bool const is_small_n = n < 8192;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(32), next_pow_2(m));  // next power of 2

  if (mp2 <= 32) {
    // m in [1, 32]
    if (is_small_n) {
      return cutlass_sparse_gemm_caller<Cutlass3xGemmM32NSmall>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    } else {
      return cutlass_sparse_gemm_caller<Cutlass3xGemmM32NBig>(
          out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
    }
  } else if (mp2 <= 64) {
    // m in (32, 64]
    return cutlass_sparse_gemm_caller<Cutlass3xGemmM64>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 128) {
    // m in (64, 128]
    return cutlass_sparse_gemm_caller<Cutlass3xGemmM128>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  } else {
    // m in (128, inf)
    return cutlass_sparse_gemm_caller<Cutlass3xGemmDefault>(
        out, a, bt_nzs, bt_meta, std::forward<EpilogueArgs>(args)...);
  }
}

template <template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_sparse_mm_sm90_epilogue(torch::Tensor& out,
                                            torch::Tensor const& a,
                                            torch::Tensor const& bt_nzs,
                                            torch::Tensor const& bt_meta,
                                            EpilogueArgs&&... epilogue_args) {
  TORCH_CHECK(bt_meta.dtype() == torch::kUInt8);
  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(bt_nzs.dtype() == torch::kInt8);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_sm90_int8_dispatch<int8_t, cutlass::bfloat16_t,
                                             Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_sm90_int8_dispatch<int8_t, cutlass::half_t, Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    }
  } else if (a.dtype() == torch::kFloat8_e4m3fn) {
    TORCH_CHECK(bt_nzs.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                            cutlass::bfloat16_t, Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                            cutlass::half_t, Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    }
  } else if (a.dtype() == torch::kFloat16) {
    TORCH_CHECK(bt_nzs.dtype() == torch::kFloat16);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_sm90_fp16_dispatch<cutlass::half_t,
                                             cutlass::bfloat16_t, Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_sm90_fp16_dispatch<cutlass::half_t, cutlass::half_t,
                                             Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    }
  } else {  // a.dtype() == torch::kBFloat16
    TORCH_CHECK(a.dtype() == torch::kBFloat16);
    TORCH_CHECK(bt_nzs.dtype() == torch::kBFloat16);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_sm90_bf16_dispatch<cutlass::bfloat16_t,
                                             cutlass::bfloat16_t, Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_sm90_bf16_dispatch<cutlass::bfloat16_t,
                                             cutlass::half_t, Epilogue>(
          out, a, bt_nzs, bt_meta,
          std::forward<EpilogueArgs>(epilogue_args)...);
    }
  }
}

void cutlass_scaled_sparse_mm_sm90(torch::Tensor& out, torch::Tensor const& a,
                                   torch::Tensor const& bt_nzs,
                                   torch::Tensor const& bt_meta,
                                   torch::Tensor const& a_scales,
                                   torch::Tensor const& b_scales,
                                   c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == out.dtype(),
                "currently bias dtype must match output dtype ", out.dtype());
    return cutlass_scaled_sparse_mm_sm90_epilogue<c3x::ScaledEpilogueBias>(
        out, a, bt_nzs, bt_meta, b_scales, a_scales, *bias);
  } else {
    return cutlass_scaled_sparse_mm_sm90_epilogue<c3x::ScaledEpilogue>(
        out, a, bt_nzs, bt_meta, b_scales, a_scales);
  }
}

#endif
