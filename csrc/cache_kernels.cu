#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"
#ifdef ENABLE_FP8_E5M2
#include "quantization/fp8_e5m2_kvcache/quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
  typedef __hip_bfloat16 __hip_bfloat16;
#endif

void swap_blocks(
  torch::Tensor& src,
  torch::Tensor& dst,
  const std::map<int64_t, int64_t>& block_mapping) {
  torch::Device src_device = src.device();
  torch::Device dst_device = dst.device();
  hipMemcpyKind memcpy_type;
  if (src_device.is_cuda() && dst_device.is_cuda()) {
    TORCH_CHECK(
      src_device.index() == dst_device.index(),
      "src and dst must be on the same GPU");
    memcpy_type = hipMemcpyDeviceToDevice;
  } else if (src_device.is_cuda() && dst_device.is_cpu()) {
    memcpy_type = hipMemcpyDeviceToHost;
  } else if (src_device.is_cpu() && dst_device.is_cuda()) {
    memcpy_type = hipMemcpyHostToDevice;
  } else {
    TORCH_CHECK(false, "Invalid device combination");
  }

  char *src_ptr = static_cast<char*>(src.data_ptr());
  char *dst_ptr = static_cast<char*>(dst.data_ptr());

  const int64_t block_size_in_bytes = src.element_size() * src[0].numel();
  const at::cuda::OptionalCUDAGuard device_guard(src_device.is_cuda() ? src_device : dst_device);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // NOTE(woosuk): This can be slow if the number of blocks is large.
  for (const auto& pair : block_mapping) {
    int64_t src_block_number = pair.first;
    int64_t dst_block_number = pair.second;
    int64_t src_offset = src_block_number * block_size_in_bytes;
    int64_t dst_offset = dst_block_number * block_size_in_bytes;
    hipMemcpyAsync(
      dst_ptr + dst_offset,
      src_ptr + src_offset,
      block_size_in_bytes,
      memcpy_type,
      stream);
  }
}

namespace vllm {

// Grid: (num_layers, num_pairs)
template<typename scalar_t>
__global__ void copy_blocks_kernel(
  int64_t* key_cache_ptrs,
  int64_t* value_cache_ptrs,
  const int64_t* __restrict__ block_mapping,
  const int numel_per_block) {
  const int layer_idx = blockIdx.x;
  const int pair_idx = blockIdx.y;

  scalar_t* key_cache = reinterpret_cast<scalar_t*>(key_cache_ptrs[layer_idx]);
  scalar_t* value_cache = reinterpret_cast<scalar_t*>(value_cache_ptrs[layer_idx]);
  int64_t src_block_number = block_mapping[2 * pair_idx];
  int64_t dst_block_number = block_mapping[2 * pair_idx + 1];

  const int64_t src_block_offset = src_block_number * numel_per_block;
  const int64_t dst_block_offset = dst_block_number * numel_per_block;
  for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
    int64_t src_offset = src_block_offset + i;
    int64_t dst_offset = dst_block_offset + i;
    key_cache[dst_offset] = key_cache[src_offset];
  }
  for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
    int64_t src_offset = src_block_offset + i;
    int64_t dst_offset = dst_block_offset + i;
    value_cache[dst_offset] = value_cache[src_offset];
  }
}

} // namespace vllm

void copy_blocks(
  std::vector<torch::Tensor>& key_caches,
  std::vector<torch::Tensor>& value_caches,
  const std::map<int64_t, std::vector<int64_t>>& block_mapping) {
  int num_layers = key_caches.size();
  TORCH_CHECK(num_layers == value_caches.size());
  if (num_layers == 0) {
    return;
  }
  torch::Device cache_device = key_caches[0].device();
  TORCH_CHECK(cache_device.is_cuda());

  // Create data structures for the kernel.
  // Create an array of pointers to the key and value caches.
  int64_t key_cache_ptrs[num_layers];
  int64_t value_cache_ptrs[num_layers];
  for (int layer_idx = 0; layer_idx < num_layers; ++layer_idx) {
    key_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(key_caches[layer_idx].data_ptr());
    value_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(value_caches[layer_idx].data_ptr());
  }
  // Create block mapping array.
  std::vector<int64_t> block_mapping_vec;
  for (const auto& pair : block_mapping) {
    int64_t src_block_number = pair.first;
    for (int64_t dst_block_number : pair.second) {
      block_mapping_vec.push_back(src_block_number);
      block_mapping_vec.push_back(dst_block_number);
    }
  }
  int64_t* block_mapping_array = block_mapping_vec.data();
  int num_pairs = block_mapping_vec.size() / 2;

  // Move the data structures to the GPU.
  // NOTE: This synchronizes the CPU and GPU.
  torch::Tensor key_cache_ptrs_tensor = torch::from_blob(
    key_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);
  torch::Tensor value_cache_ptrs_tensor = torch::from_blob(
    value_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);
  torch::Tensor block_mapping_tensor = torch::from_blob(
    block_mapping_array, {2 * num_pairs}, torch::kInt64).to(cache_device);

  // Launch the kernel.
  const int numel_per_block = key_caches[0][0].numel();
  dim3 grid(num_layers, num_pairs);
  dim3 block(std::min(1024, numel_per_block));
  const at::cuda::OptionalCUDAGuard device_guard(cache_device);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_AND_BYTE_TYPES(
    key_caches[0].scalar_type(), "copy_blocks_kernel", ([&] {
      vllm::copy_blocks_kernel<scalar_t><<<grid, block, 0, stream>>>(
        key_cache_ptrs_tensor.data_ptr<int64_t>(),
        value_cache_ptrs_tensor.data_ptr<int64_t>(),
        block_mapping_tensor.data_ptr<int64_t>(),
        numel_per_block);
    }));
}

namespace vllm {

template<typename scalar_t, typename cache_t, bool is_fp8_e5m2_kv_cache>
__global__ void reshape_and_cache_kernel(
  const scalar_t* __restrict__ key,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,         // [num_tokens, num_heads, head_size]
  cache_t* __restrict__ key_cache,            // [num_blocks, num_heads, head_size/x, block_size, x]
  cache_t* __restrict__ value_cache,          // [num_blocks, num_heads, head_size, block_size]
  const int64_t* __restrict__ slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }

  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int x_idx = head_offset / x;
    const int x_offset = head_offset % x;

    const int64_t tgt_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                                + head_idx * (head_size / x) * block_size * x
                                + x_idx * block_size * x
                                + block_offset * x
                                + x_offset;
    const int64_t tgt_value_idx = block_idx * num_heads * head_size * block_size
                                  + head_idx * head_size * block_size
                                  + head_offset * block_size
                                  + block_offset;
    scalar_t tgt_key = key[src_key_idx];
    scalar_t tgt_value = value[src_value_idx];
    if constexpr (is_fp8_e5m2_kv_cache) {
#ifdef ENABLE_FP8_E5M2
      key_cache[tgt_key_idx] = fp8_e5m2_unscaled::vec_conversion<uint8_t, scalar_t>(tgt_key);
      value_cache[tgt_value_idx] = fp8_e5m2_unscaled::vec_conversion<uint8_t, scalar_t>(tgt_value);
#else
      assert(false);
#endif
    } else {
      key_cache[tgt_key_idx] = tgt_key;
      value_cache[tgt_value_idx] = tgt_value;
    }
  }
}

} // namespace vllm

#define CALL_RESHAPE_AND_CACHE(KV_T, CACHE_T, IS_FP8_E5M2_KV_CACHE)                                \
  vllm::reshape_and_cache_kernel<KV_T, CACHE_T, IS_FP8_E5M2_KV_CACHE><<<grid, block, 0, stream>>>( \
    reinterpret_cast<KV_T*>(key.data_ptr()),                                                       \
    reinterpret_cast<KV_T*>(value.data_ptr()),                                                     \
    reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),                                              \
    reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),                                            \
    slot_mapping.data_ptr<int64_t>(),                                                              \
    key_stride,                                                                                    \
    value_stride,                                                                                  \
    num_heads,                                                                                     \
    head_size,                                                                                     \
    block_size,                                                                                    \
    x);

void reshape_and_cache(
  torch::Tensor& key,           // [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,   // [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& slot_mapping,  // [num_tokens]
  const std::string& kv_cache_dtype)
{
  int num_tokens = key.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(3);
  int x = key_cache.size(4);

  int key_stride = key.stride(0);
  int value_stride = value.stride(0);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (kv_cache_dtype == "auto") {
    if (key.dtype() == at::ScalarType::Float) {
      CALL_RESHAPE_AND_CACHE(float, float, false);
    } else if (key.dtype() == at::ScalarType::Half) {
      CALL_RESHAPE_AND_CACHE(uint16_t, uint16_t, false);
    } else if (key.dtype() == at::ScalarType::BFloat16) {
      CALL_RESHAPE_AND_CACHE(__hip_bfloat16, __hip_bfloat16, false);
    }
  } else if (kv_cache_dtype == "fp8_e5m2") {
    if (key.dtype() == at::ScalarType::Float) {
      CALL_RESHAPE_AND_CACHE(float, uint8_t, true);
    } else if (key.dtype() == at::ScalarType::Half) {
      CALL_RESHAPE_AND_CACHE(uint16_t, uint8_t, true);
    } else if (key.dtype() == at::ScalarType::BFloat16) {
      CALL_RESHAPE_AND_CACHE(__hip_bfloat16, uint8_t, true);
    }
  } else {
    TORCH_CHECK(false, "Unsupported data type of kv cache: ", kv_cache_dtype);
  }
}

namespace vllm {

// Grid: (num_blocks, block_size).
template<typename scalar_t>
__global__ void gather_cached_kv_kernel(
  scalar_t* __restrict__ key,             // [num_tokens, [stride], num_heads, head_size]
  scalar_t* __restrict__ value,           // [num_tokens, [stride], num_heads, head_size]
  const scalar_t* __restrict__ key_cache,   // [num_blocks, num_heads, head_size/x, block_size, x]
  const scalar_t* __restrict__ value_cache,   // [num_blocks, num_heads, head_size, block_size]
  const int* __restrict__ slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x) {
    const int token_idx = blockIdx.x;
    const int slot_idx = slot_mapping[token_idx];
    const int block_idx = slot_idx / block_size;
    const int block_offset = slot_idx % block_size;

    const int num_tokens = num_heads * head_size;
    for (int i = threadIdx.x; i < num_tokens; i += blockDim.x) {
      const int tgt_key_idx = token_idx * key_stride + i;
      const int tgt_value_idx = token_idx * value_stride + i;

      const int head_idx = i / head_size;
      const int head_offset = i % head_size;
      const int x_idx = head_offset / x;  // the offset of the [head_size/x] dimension
      const int x_offset = head_offset % x;

      const int src_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                              + head_idx * (head_size / x) * block_size * x
                              + x_idx * block_size * x
                              + block_offset * x
                              + x_offset;
      const int src_value_idx = block_idx * num_heads * head_size * block_size
                                + head_idx * head_size * block_size
                                + head_offset * block_size
                                + block_offset;

      key[tgt_key_idx] = VLLM_LDG(&key_cache[src_key_idx]);
      value[tgt_value_idx] = VLLM_LDG(&value_cache[src_value_idx]);
    }
}

template <typename scalar_t>
__global__ void gather_cached_kv_kernel_optimized(
    scalar_t *__restrict__ key,             // [num_tokens, [stride], num_heads, head_size]
    scalar_t *__restrict__ value,           // [num_tokens, [stride], num_heads, head_size]
    const scalar_t *__restrict__ key_cache, // [num_blocks, num_heads, head_size/x, block_size, x]
    const scalar_t *__restrict__ value_cache, // [num_blocks, num_heads, head_size, block_size]
    const int *__restrict__ slot_mapping,   // [num_tokens]
    const int key_stride,
    const int value_stride,
    const int num_heads,
    const int head_size,
    const int block_size,
    const int x)
{
    const int token_idx = blockIdx.x;
    const int slot_idx = slot_mapping[token_idx];
    const int block_idx = slot_idx / block_size;
    const int block_offset = slot_idx % block_size;

    const int dim = num_heads * head_size;
    assert(dim % 4 == 0);  // this is true for known use cases
    const int unroll_factor = 4;
    const int unrolled_dim = dim / unroll_factor;

    for (int i = threadIdx.x; i < unrolled_dim; i += blockDim.x)
    {
        int tgt_key_indices[unroll_factor];
        int tgt_value_indices[unroll_factor];
        int src_key_indices[unroll_factor];
        int src_value_indices[unroll_factor];
        scalar_t keys_to_store[unroll_factor];
        scalar_t values_to_store[unroll_factor];

        #pragma unroll
        for (int j = 0; j < unroll_factor; ++j)
        {
            int index = i + j * unrolled_dim;

            const int tgt_key_idx = token_idx * key_stride + index;
            const int tgt_value_idx = token_idx * value_stride + index;

            const int head_idx = index / head_size;
            const int head_offset = index % head_size;
            const int x_idx = head_offset / x;
            const int x_offset = head_offset % x;

            const int src_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                                    + head_idx * (head_size / x) * block_size * x
                                    + x_idx * block_size * x
                                    + block_offset * x
                                    + x_offset;
            const int src_value_idx = block_idx * num_heads * head_size * block_size
                                      + head_idx * head_size * block_size
                                      + head_offset * block_size
                                      + block_offset;

            tgt_key_indices[j] = tgt_key_idx;
            tgt_value_indices[j] = tgt_value_idx;
            src_key_indices[j] = src_key_idx;
            src_value_indices[j] = src_value_idx;

            keys_to_store[j] = VLLM_LDG(&key_cache[src_key_idx]);
            values_to_store[j] = VLLM_LDG(&value_cache[src_value_idx]);
        }

        #pragma unroll
        for (int j = 0; j < unroll_factor; ++j)
        {
            key[tgt_key_indices[j]] = keys_to_store[j];
            value[tgt_value_indices[j]] = values_to_store[j];
        }
    }
}

} // namespace vllm

void gather_cached_kv(
  torch::Tensor& key,           // [out] [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [out] [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,     // [in]  [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,   // [in]  [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& slot_mapping)  // [in]  [num_tokens]
{
  int num_tokens = key.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(3);
  int x = key_cache.size(4);

  int key_stride = key.stride(0);
  int value_stride = value.stride(0);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_AND_BYTE_TYPES(
    key.scalar_type(),
    "gather_cached_kv_kernel_optimized",
    [&] {
      vllm::gather_cached_kv_kernel_optimized<scalar_t><<<grid, block, 0, stream>>>(
        key.data_ptr<scalar_t>(),
        value.data_ptr<scalar_t>(),
        key_cache.data_ptr<scalar_t>(),
        value_cache.data_ptr<scalar_t>(),
        slot_mapping.data_ptr<int>(),
        key_stride,
        value_stride,
        num_heads,
        head_size,
        block_size,
        x);
    });
}

namespace vllm {

template<typename Tout, typename Tin>
__global__ void convert_fp8_e5m2_kernel(
  const Tin* __restrict__ src_cache,
  Tout* __restrict__ dst_cache,
  const int64_t block_stride) {
  const int64_t block_idx = blockIdx.x;
  for (int i = threadIdx.x; i < block_stride; i += blockDim.x) {
    int64_t idx = block_idx * block_stride + i;
#ifdef ENABLE_FP8_E5M2
    dst_cache[idx] = fp8_e5m2_unscaled::vec_conversion<Tout, Tin>(src_cache[idx]);
#else
    assert(false);
#endif
  }
}

} // namespace vllm

#define CALL_CONVERT_FP8_E5M2(Tout, Tin)                                 \
  vllm::convert_fp8_e5m2_kernel<Tout, Tin><<<grid, block, 0, stream>>>(  \
    reinterpret_cast<Tin*>(src_cache.data_ptr()),                        \
    reinterpret_cast<Tout*>(dst_cache.data_ptr()),                       \
    block_stride);

void convert_fp8_e5m2(
  torch::Tensor& src_cache,
  torch::Tensor& dst_cache)
{
  int64_t num_blocks = src_cache.size(0);
  int64_t block_stride = src_cache.stride(0);

  dim3 grid(num_blocks);
  dim3 block(std::min(block_stride, int64_t(512)));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (src_cache.dtype() == at::ScalarType::Float) {
    CALL_CONVERT_FP8_E5M2(uint8_t, float);
  } else if (src_cache.dtype() == at::ScalarType::Half) {
    CALL_CONVERT_FP8_E5M2(uint8_t, uint16_t);
  } else if (src_cache.dtype() == at::ScalarType::BFloat16) {
    CALL_CONVERT_FP8_E5M2(uint8_t, __hip_bfloat16);
  } else if (dst_cache.dtype() == at::ScalarType::Float) {
    CALL_CONVERT_FP8_E5M2(float, uint8_t);
  } else if (dst_cache.dtype() == at::ScalarType::Half) {
    CALL_CONVERT_FP8_E5M2(uint16_t, uint8_t);
  } else if (dst_cache.dtype() == at::ScalarType::BFloat16) {
    CALL_CONVERT_FP8_E5M2(__hip_bfloat16, uint8_t);
  }
}
