#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>

#include <algorithm>
#include <cassert>
#include <map>

void copy_blocks(
  torch::Tensor& src,
  torch::Tensor& dst,
  const std::map<int64_t, int64_t>& block_mapping) {
  torch::Device src_device = src.device();
  torch::Device dst_device = dst.device();
  hipMemcpyKind memcpy_type;
  if (src_device.is_cuda() && dst_device.is_cuda()) {
    assert(src_device.index() == dst_device.index());
    memcpy_type = hipMemcpyDeviceToDevice;
  } else if (src_device.is_cuda() && dst_device.is_cpu()) {
    memcpy_type = hipMemcpyDeviceToHost;
  } else if (src_device.is_cpu() && dst_device.is_cuda()) {
    memcpy_type = hipMemcpyHostToDevice;
  } else {
    assert(false);
  }

  void *src_ptr = src.data_ptr();
  void *dst_ptr = dst.data_ptr();

  const int64_t block_size_in_bytes = src.element_size() * src[0].numel();
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  for (const auto& pair : block_mapping) {
    int64_t src_block_number = pair.first;
    int64_t dst_block_number = pair.second;
    int64_t src_offset = src_block_number * block_size_in_bytes;
    int64_t dst_offset = dst_block_number * block_size_in_bytes;
    hipMemcpyAsync(
      dst_ptr + dst_offset,
      src_ptr + src_offset,
      block_size_in_bytes,
      memcpy_type,
      stream);
  }
}

template<typename scalar_t>
__global__ void reshape_and_cache_kernel(
  const scalar_t* __restrict__ key,     // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,   // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
  scalar_t* __restrict__ value_cache,   // [num_blocks, num_heads, block_size, head_size]
  const int* __restrict__ slot_mapping, // [num_tokens]
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x) {
  const int token_idx = blockIdx.x;
  const int slot_idx = slot_mapping[token_idx];
  const int block_idx = slot_idx / block_size;
  const int block_offset = slot_idx % block_size;

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int src_idx = token_idx * n + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int x_idx = head_offset / x;
    const int x_offset = head_offset % x;

    const int tgt_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                            + head_idx * (head_size / x) * block_size * x
                            + x_idx * block_size * x
                            + block_offset * x
                            + x_offset;
    const int tgt_value_idx = block_idx * num_heads * block_size * head_size
                              + head_idx * block_size * head_size
                              + block_offset * head_size
                              + head_offset;
    key_cache[tgt_key_idx] = __ldg(&key[src_idx]);
    value_cache[tgt_value_idx] = __ldg(&value[src_idx]);
  }
}

void reshape_and_cache(
  torch::Tensor& key,
  torch::Tensor& value,
  torch::Tensor& key_cache,
  torch::Tensor& value_cache,
  torch::Tensor& slot_mapping) {
  int num_tokens = key.size(0);
  int head_num = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(3);
  int x = key_cache.size(4);

  dim3 grid(num_tokens);
  dim3 block(std::min(head_num * head_size, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    key.scalar_type(),
    "reshape_and_cache_kernel",
    [&] {
      reshape_and_cache_kernel<scalar_t><<<grid, block, 0, stream>>>(
        key.data_ptr<scalar_t>(),
        value.data_ptr<scalar_t>(),
        key_cache.data_ptr<scalar_t>(),
        value_cache.data_ptr<scalar_t>(),
        slot_mapping.data_ptr<int>(),
        head_num,
        head_size,
        block_size,
        x);
    });
}
