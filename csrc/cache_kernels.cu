#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

void swap_blocks(
  torch::Tensor& src,
  torch::Tensor& dst,
  const std::map<int64_t, int64_t>& block_mapping) {
  torch::Device src_device = src.device();
  torch::Device dst_device = dst.device();
  hipMemcpyKind memcpy_type;
  if (src_device.is_cuda() && dst_device.is_cuda()) {
    assert(src_device.index() == dst_device.index());
    memcpy_type = hipMemcpyDeviceToDevice;
  } else if (src_device.is_cuda() && dst_device.is_cpu()) {
    memcpy_type = hipMemcpyDeviceToHost;
  } else if (src_device.is_cpu() && dst_device.is_cuda()) {
    memcpy_type = hipMemcpyHostToDevice;
  } else {
    assert(false);
  }

  void *src_ptr = src.data_ptr();
  void *dst_ptr = dst.data_ptr();

  const int64_t block_size_in_bytes = src.element_size() * src[0].numel();
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  for (const auto& pair : block_mapping) {
    int64_t src_block_number = pair.first;
    int64_t dst_block_number = pair.second;
    int64_t src_offset = src_block_number * block_size_in_bytes;
    int64_t dst_offset = dst_block_number * block_size_in_bytes;
    hipMemcpyAsync(
      dst_ptr + dst_offset,
      src_ptr + src_offset,
      block_size_in_bytes,
      memcpy_type,
      stream);
  }
}

void copy_blocks(
  torch::Tensor& src,
  torch::Tensor& dst,
  const std::map<int64_t, std::vector<int64_t>>& block_mapping) {
  torch::Device src_device = src.device();
  torch::Device dst_device = dst.device();
  assert(src_device.is_cuda() && dst_device.is_cuda());
  hipMemcpyKind memcpy_type = hipMemcpyDeviceToDevice;

  void *src_ptr = src.data_ptr();
  void *dst_ptr = dst.data_ptr();

  const int64_t block_size_in_bytes = src.element_size() * src[0].numel();
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  for (const auto& pair : block_mapping) {
    int64_t src_block_number = pair.first;
    for (int64_t dst_block_number : pair.second) {
      int64_t src_offset = src_block_number * block_size_in_bytes;
      int64_t dst_offset = dst_block_number * block_size_in_bytes;
      hipMemcpyAsync(
        dst_ptr + dst_offset,
        src_ptr + src_offset,
        block_size_in_bytes,
        memcpy_type,
        stream);
    }
  }
}

namespace cacheflow {

template<typename scalar_t>
__global__ void reshape_and_cache_kernel(
  const scalar_t* __restrict__ key,     // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,   // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
  scalar_t* __restrict__ value_cache,   // [num_blocks, num_heads, head_size, block_size]
  const int* __restrict__ slot_mapping, // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x) {
  const int token_idx = blockIdx.x;
  const int slot_idx = slot_mapping[token_idx];
  const int block_idx = slot_idx / block_size;
  const int block_offset = slot_idx % block_size;

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int src_key_idx = token_idx * key_stride + i;
    const int src_value_idx = token_idx * value_stride + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int x_idx = head_offset / x;
    const int x_offset = head_offset % x;

    const int tgt_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                            + head_idx * (head_size / x) * block_size * x
                            + x_idx * block_size * x
                            + block_offset * x
                            + x_offset;
    const int tgt_value_idx = block_idx * num_heads * head_size * block_size
                              + head_idx * head_size * block_size
                              + head_offset * block_size
                              + block_offset;
    key_cache[tgt_key_idx] = __ldg(&key[src_key_idx]);
    value_cache[tgt_value_idx] = __ldg(&value[src_value_idx]);
  }
}

} // namespace cacheflow

void reshape_and_cache(
  torch::Tensor& key,           // [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,   // [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& slot_mapping)  // [num_tokens]
{
  int num_tokens = key.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(3);
  int x = key_cache.size(4);

  int key_stride = key.stride(0);
  int value_stride = value.stride(0);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    key.scalar_type(),
    "reshape_and_cache_kernel",
    [&] {
      cacheflow::reshape_and_cache_kernel<scalar_t><<<grid, block, 0, stream>>>(
        key.data_ptr<scalar_t>(),
        value.data_ptr<scalar_t>(),
        key_cache.data_ptr<scalar_t>(),
        value_cache.data_ptr<scalar_t>(),
        slot_mapping.data_ptr<int>(),
        key_stride,
        value_stride,
        num_heads,
        head_size,
        block_size,
        x);
    });
}
