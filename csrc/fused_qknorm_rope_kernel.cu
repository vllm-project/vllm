#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#define CHECK_TYPE(x, st)                                              \
  TORCH_CHECK(x.scalar_type() == st, #x " dtype is ", x.scalar_type(), \
              ", while ", st, " is expected")
#define CHECK_TH_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x, st) \
  CHECK_TH_CUDA(x);        \
  CHECK_CONTIGUOUS(x);     \
  CHECK_TYPE(x, st)

#define FINAL_MASK 0xffffffff

namespace tensorrt_llm::common {
template <typename T, int num>
struct packed_as;
// Specialization for packed_as used in this kernel.
template <>
struct packed_as<uint, 1> {
  using type = uint;
};

template <>
struct packed_as<uint, 2> {
  using type = uint2;
};

template <>
struct packed_as<uint, 4> {
  using type = uint4;
};

template <typename T>
__inline__ __device__ T warpReduceSum(T val) {
#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask,
                           32);  //__shfl_sync bf16 return float when sm < 80
  return val;
}

template <typename T>
inline __device__ __host__ T divUp(T m, T n) {
  return (m + n - 1) / n;
}

}  // namespace tensorrt_llm::common

namespace tensorrt_llm::kernels {
// NOTE(zhuhaoran): This kernel is adapted from TensorRT-LLM implementation,
// with added support for passing the cos_sin_cache as an input.
// https://github.com/NVIDIA/TensorRT-LLM/blob/main/cpp/tensorrt_llm/kernels/fusedQKNormRopeKernel.cu

// Perform per-head QK Norm and RoPE in a single kernel.
// head_dim: the dimension of each head
// interleave: interleave=!is_neox.
template <int head_dim, bool interleave>
__global__ void fusedQKNormRopeKernel(
    __hip_bfloat16* qkv,     // Combined QKV tensor [num_tokens,
                            // (num_heads_q+num_heads_k+num_heads_v)*head_dim]
    int const num_heads_q,  // Number of query heads
    int const num_heads_k,  // Number of key heads
    int const num_heads_v,  // Number of value heads
    float const eps,        // Epsilon for RMS normalization
    __hip_bfloat16 const* q_weight,       // RMSNorm weights for query
    __hip_bfloat16 const* k_weight,       // RMSNorm weights for key
    __hip_bfloat16 const* cos_sin_cache,  // Pre-computed cos/sin cache
    int64_t const* position_ids,         // Position IDs for RoPE
    int const num_tokens                 // Number of tokens
) {
  int const warpsPerBlock = blockDim.x / 32;
  int const warpId = threadIdx.x / 32;
  int const laneId = threadIdx.x % 32;

  // Calculate global warp index to determine which head/token this warp
  // processes
  int const globalWarpIdx = blockIdx.x * warpsPerBlock + warpId;

  // Total number of attention heads (Q and K)
  int const total_qk_heads = num_heads_q + num_heads_k;

  // Determine which token and head type (Q or K) this warp processes
  int const tokenIdx = globalWarpIdx / total_qk_heads;
  int const localHeadIdx = globalWarpIdx % total_qk_heads;

  // Skip if this warp is assigned beyond the number of tokens
  if (tokenIdx >= num_tokens) return;

  bool const isQ = localHeadIdx < num_heads_q;
  int const headIdx = isQ ? localHeadIdx : localHeadIdx - num_heads_q;

  int const num_heads = num_heads_q + num_heads_k + num_heads_v;

  static_assert(head_dim % (32 * 2) == 0,
                "head_dim must be divisible by 64 (each warp processes one "
                "head, and each thread gets even number of "
                "elements)");
  constexpr int numElemsPerThread = head_dim / 32;
  float elements[numElemsPerThread];
  constexpr int elemSizeBytes = numElemsPerThread * sizeof(__hip_bfloat16);
  static_assert(elemSizeBytes % 4 == 0, "numSizeBytes must be a multiple of 4");
  constexpr int vecSize =
      elemSizeBytes /
      4;  // Use packed_as<uint, vecSize> to perform loading/saving.
  using vec_T = typename tensorrt_llm::common::packed_as<uint, vecSize>::type;

  int offsetWarp;  // Offset for the warp
  if (isQ) {
    // Q segment: token offset + head offset within Q segment
    offsetWarp = tokenIdx * num_heads * head_dim + headIdx * head_dim;
  } else {
    // K segment: token offset + entire Q segment + head offset within K segment
    offsetWarp = tokenIdx * num_heads * head_dim + num_heads_q * head_dim +
                 headIdx * head_dim;
  }
  int offsetThread = offsetWarp + laneId * numElemsPerThread;

  // Sum of squares for RMSNorm
  float sumOfSquares = 0.0f;

  // Load.
  {
    vec_T vec = *reinterpret_cast<vec_T const*>(&qkv[offsetThread]);
    for (int i = 0; i < vecSize; i++) {
      float2 vals = __bfloat1622float2(*reinterpret_cast<__hip_bfloat162*>(
          reinterpret_cast<uint*>(&vec) + i));
      sumOfSquares += vals.x * vals.x;
      sumOfSquares += vals.y * vals.y;

      elements[2 * i] = vals.x;
      elements[2 * i + 1] = vals.y;
    }
  }

  // Reduce sum across warp using the utility function
  sumOfSquares = tensorrt_llm::common::warpReduceSum(sumOfSquares);

  // Compute RMS normalization factor
  float rms_rcp = rsqrtf(sumOfSquares / static_cast<float>(head_dim) + eps);

  // Normalize elements
  for (int i = 0; i < numElemsPerThread; i++) {
    int dim = laneId * numElemsPerThread + i;
    float weight =
        isQ ? __bfloat162float(q_weight[dim]) : __bfloat162float(k_weight[dim]);
    elements[i] *= rms_rcp * weight;
  }

  // Apply RoPE to normalized elements
  float elements2[numElemsPerThread];  // Additional buffer required for RoPE.

  int64_t pos_id = position_ids[tokenIdx];

  // Calculate cache pointer for this position - similar to
  // pos_encoding_kernels.cu
  __hip_bfloat16 const* cache_ptr = cos_sin_cache + pos_id * head_dim;
  int const embed_dim = head_dim / 2;
  __hip_bfloat16 const* cos_ptr = cache_ptr;
  __hip_bfloat16 const* sin_ptr = cache_ptr + embed_dim;

  if constexpr (interleave) {
    // Perform interleaving. Use pre-computed cos/sin values.
    for (int i = 0; i < numElemsPerThread / 2; ++i) {
      int const idx0 = 2 * i;
      int const idx1 = 2 * i + 1;

      float const val0 = elements[idx0];
      float const val1 = elements[idx1];

      int const dim_idx = laneId * numElemsPerThread + idx0;
      int const half_dim = dim_idx / 2;
      float const cos_val = __bfloat162float(VLLM_LDG(cos_ptr + half_dim));
      float const sin_val = __bfloat162float(VLLM_LDG(sin_ptr + half_dim));

      float const rotated_val0 = val0 * cos_val - val1 * sin_val;
      float const rotated_val1 = val0 * sin_val + val1 * cos_val;
  
      elements[idx0] = rotated_val0;
      elements[idx1] = rotated_val1;
    }
  } else {
    // Before data exchange with in warp, we need to sync.
    __syncwarp();
    // Get the data from the other half of the warp. Use pre-computed cos/sin
    // values.
    for (int i = 0; i < numElemsPerThread; i++) {
      elements2[i] = __shfl_xor_sync(0xffffffff, elements[i], 16);
      if (laneId < 16) {
        elements2[i] = -elements2[i];
      }

      int dim_idx = laneId * numElemsPerThread + i;
      dim_idx = (dim_idx * 2) % head_dim;
      int half_dim = dim_idx / 2;
      // Use pre-computed cos/sin from cache with optimized memory access
      float cos_val = __bfloat162float(VLLM_LDG(cos_ptr + half_dim));
      float sin_val = __bfloat162float(VLLM_LDG(sin_ptr + half_dim));

      elements[i] = elements[i] * cos_val + elements2[i] * sin_val;
    }
    // __shfl_xor_sync does not provide memfence. Need to sync again.
    __syncwarp();
  }

  // Store.
  {
    vec_T vec;
    for (int i = 0; i < vecSize; i++) {
      __hip_bfloat162 vals = __float22bfloat162_rn(
          make_float2(elements[2 * i], elements[2 * i + 1]));
      reinterpret_cast<__hip_bfloat162&>(*(reinterpret_cast<uint*>(&vec) + i)) =
          vals;
    }
    vec_T* outputPtr = reinterpret_cast<vec_T*>(&qkv[offsetThread]);
    *outputPtr = vec;
  }
}

// Borrowed from
// https://github.com/flashinfer-ai/flashinfer/blob/8125d079a43e9a0ba463a4ed1b639cefd084cec9/include/flashinfer/pos_enc.cuh#L568
#define DISPATCH_INTERLEAVE(interleave, INTERLEAVE, ...) \
  if (interleave) {                                      \
    const bool INTERLEAVE = true;                        \
    __VA_ARGS__                                          \
  } else {                                               \
    const bool INTERLEAVE = false;                       \
    __VA_ARGS__                                          \
  }

void launchFusedQKNormRope(void* qkv, int const num_tokens,
                           int const num_heads_q, int const num_heads_k,
                           int const num_heads_v, int const head_dim,
                           float const eps, void const* q_weight,
                           void const* k_weight,
                           __hip_bfloat16 const* cos_sin_cache,
                           bool const interleave, int64_t const* position_ids,
                           hipStream_t stream) {
  constexpr int blockSize = 256;

  int const warpsPerBlock = blockSize / 32;
  int const totalQKHeads = num_heads_q + num_heads_k;
  int const totalWarps = num_tokens * totalQKHeads;

  int const gridSize = common::divUp(totalWarps, warpsPerBlock);
  dim3 gridDim(gridSize);
  dim3 blockDim(blockSize);

  // Head dimensions should be a multiple of 64
  // Add more cases as needed
  switch (head_dim) {
    case 64:
      DISPATCH_INTERLEAVE(interleave, INTERLEAVE, {
        fusedQKNormRopeKernel<64, INTERLEAVE><<<gridDim, blockDim, 0, stream>>>(
            reinterpret_cast<__hip_bfloat16*>(qkv), num_heads_q, num_heads_k,
            num_heads_v, eps, reinterpret_cast<__hip_bfloat16 const*>(q_weight),
            reinterpret_cast<__hip_bfloat16 const*>(k_weight), cos_sin_cache,
            position_ids, num_tokens);
      });
      break;
    case 128:
      DISPATCH_INTERLEAVE(interleave, INTERLEAVE, {
        fusedQKNormRopeKernel<128, INTERLEAVE>
            <<<gridDim, blockDim, 0, stream>>>(
                reinterpret_cast<__hip_bfloat16*>(qkv), num_heads_q, num_heads_k,
                num_heads_v, eps,
                reinterpret_cast<__hip_bfloat16 const*>(q_weight),
                reinterpret_cast<__hip_bfloat16 const*>(k_weight), cos_sin_cache,
                position_ids, num_tokens);
      });
      break;
    case 256:
      DISPATCH_INTERLEAVE(interleave, INTERLEAVE, {
        fusedQKNormRopeKernel<256, INTERLEAVE>
            <<<gridDim, blockDim, 0, stream>>>(
                reinterpret_cast<__hip_bfloat16*>(qkv), num_heads_q, num_heads_k,
                num_heads_v, eps,
                reinterpret_cast<__hip_bfloat16 const*>(q_weight),
                reinterpret_cast<__hip_bfloat16 const*>(k_weight), cos_sin_cache,
                position_ids, num_tokens);
      });
      break;
    default:
      TORCH_CHECK(false,
                  "Unsupported head dimension for fusedQKNormRope: ", head_dim);
  }
}
}  // namespace tensorrt_llm::kernels

void fused_qk_norm_rope(
    torch::Tensor& qkv,       // Combined QKV tensor [num_tokens,
                              // (num_heads_q+num_heads_k+num_heads_v)*head_dim]
    int64_t num_heads_q,      // Number of query heads
    int64_t num_heads_k,      // Number of key heads
    int64_t num_heads_v,      // Number of value heads
    int64_t head_dim,         // Dimension per head
    double eps,               // Epsilon for RMS normalization
    torch::Tensor& q_weight,  // RMSNorm weights for query [head_dim]
    torch::Tensor& k_weight,  // RMSNorm weights for key [head_dim]
    torch::Tensor& cos_sin_cache,  // Cos/sin cache [max_position, head_dim]
    bool is_neox,                  // Whether RoPE is applied in Neox style
    torch::Tensor& position_ids    // Position IDs for RoPE [num_tokens]
) {
  // Input validation
  TORCH_CHECK(qkv.dim() == 2,
              "QKV tensor must be 2D: [num_tokens, "
              "(num_heads_q+num_heads_k+num_heads_v)*head_dim]");
  TORCH_CHECK(position_ids.dim() == 1, "Position IDs must be 1D: [num_tokens]");
  TORCH_CHECK(q_weight.dim() == 1, "Query weights must be 1D: [head_dim]");
  TORCH_CHECK(k_weight.dim() == 1, "Key weights must be 1D: [head_dim]");
  TORCH_CHECK(cos_sin_cache.dim() == 2,
              "Cos/sin cache must be 2D: [max_position, head_dim]");
  TORCH_CHECK(q_weight.size(0) == head_dim,
              "Query weights size must match head dimension");
  TORCH_CHECK(k_weight.size(0) == head_dim,
              "Key weights size must match head dimension");
  TORCH_CHECK(cos_sin_cache.size(1) == head_dim,
              "Cos/sin cache dimension must match head_dim");

  CHECK_INPUT(qkv, torch::kBFloat16);
  CHECK_INPUT(position_ids, torch::kInt64);
  CHECK_INPUT(q_weight, torch::kBFloat16);
  CHECK_INPUT(k_weight, torch::kBFloat16);
  CHECK_INPUT(cos_sin_cache, torch::kBFloat16);

  int64_t num_tokens = qkv.size(0);
  TORCH_CHECK(position_ids.size(0) == num_tokens,
              "Number of tokens in position_ids must match QKV");

  int64_t total_heads = num_heads_q + num_heads_k + num_heads_v;
  TORCH_CHECK(
      qkv.size(1) == total_heads * head_dim,
      "QKV tensor size must match total number of heads and head dimension");

  auto stream = at::cuda::getCurrentCUDAStream(qkv.get_device());

  tensorrt_llm::kernels::launchFusedQKNormRope(
      reinterpret_cast<__hip_bfloat16*>(qkv.data_ptr()),
      static_cast<int>(num_tokens), static_cast<int>(num_heads_q),
      static_cast<int>(num_heads_k), static_cast<int>(num_heads_v),
      static_cast<int>(head_dim), static_cast<float>(eps),
      reinterpret_cast<__hip_bfloat16 const*>(q_weight.data_ptr()),
      reinterpret_cast<__hip_bfloat16 const*>(k_weight.data_ptr()),
      reinterpret_cast<__hip_bfloat16 const*>(cos_sin_cache.data_ptr()),
      !is_neox,  // interleave
      reinterpret_cast<int64_t const*>(position_ids.data_ptr()), stream);
}
