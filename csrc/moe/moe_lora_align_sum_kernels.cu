#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <ATen/ATen.h>
#include <ATen/cuda/Atomic.cuh>

#include "../cuda_compat.h"
#include "../dispatch_utils.h"
#include "core/math.hpp"

namespace {

__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row,
                                         int32_t col) {
  return row * total_col + col;
}

}  // namespace

// TODO: Refactor common parts with moe_align_sum_kernels
template <typename scalar_t, typename token_cnts_t>
__global__ void moe_lora_align_sum_kernel(
    scalar_t* __restrict__ topk_ids, int32_t* token_lora_mapping,
    int64_t block_size, int num_experts, int max_loras, size_t numel,
    int max_num_tokens_padded, int max_num_m_blocks,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int topk_num, int32_t* total_tokens_post_pad) {
  const size_t tokens_per_thread = div_ceil(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  int lora_id = blockIdx.x;
  extern __shared__ int32_t shared_mem[];
  int32_t* cumsum = shared_mem;
  token_cnts_t* tokens_cnts = (token_cnts_t*)(shared_mem + num_experts + 1);

  // Initialize sorted_token_ids with numel
  for (size_t it = threadIdx.x; it < max_num_tokens_padded; it += blockDim.x) {
    sorted_token_ids[lora_id * max_num_tokens_padded + it] = numel;
  }

  // Initialize expert_ids with -1
  for (size_t it = threadIdx.x; it < max_num_m_blocks; it += blockDim.x) {
    expert_ids[lora_id * max_num_m_blocks + it] = -1;
  }

  // Initialize total_tokens_post_pad with 0
  if (threadIdx.x == 0) {
    total_tokens_post_pad[lora_id] = 0;
  }

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[index(num_experts, threadIdx.x + 1, i)] = 0;
  }

  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int mask = token_lora_mapping[i / topk_num] == lora_id;
    int idx = index(num_experts, threadIdx.x + 1, topk_ids[i]);
    tokens_cnts[idx] += mask;
  }

  __syncthreads();

  // For each expert we accumulate the token counts from the different threads.
  if (threadIdx.x < num_experts) {
    tokens_cnts[index(num_experts, 0, threadIdx.x)] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[index(num_experts, i, threadIdx.x)] +=
          tokens_cnts[index(num_experts, i - 1, threadIdx.x)];
    }
  }

  __syncthreads();

  // We accumulate the token counts of all experts in thread 0.
  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] = cumsum[i - 1] +
                  div_ceil(tokens_cnts[index(num_experts, blockDim.x, i - 1)],
                           block_size) *
                      block_size;
    }
    total_tokens_post_pad[lora_id] = static_cast<int32_t>(cumsum[num_experts]);
  }

  __syncthreads();

  /**
   * For each expert, each thread processes the tokens of the corresponding
   * blocks and stores the corresponding expert_id for each block.
   */
  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[index(max_num_m_blocks, lora_id, i / block_size)] =
          threadIdx.x;
    }
  }

  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int32_t expert_id = topk_ids[i];
    /** The cumsum[expert_id] stores the starting index of the tokens that the
     * expert with expert_id needs to process, and
     * tokens_cnts[threadIdx.x][expert_id] stores the indices of the tokens
     * processed by the expert with expert_id within the current thread's token
     * shard.
     */
    int32_t rank_post_pad =
        tokens_cnts[index(num_experts, threadIdx.x, expert_id)] +
        cumsum[expert_id];

    int mask = (int)token_lora_mapping[i / topk_num] == lora_id;
    atomicAdd(
        &sorted_token_ids[index(max_num_tokens_padded, lora_id, rank_post_pad)],
        (i - numel) * mask);
    tokens_cnts[index(num_experts, threadIdx.x, expert_id)] += mask;
  }
}

void moe_lora_align_block_size(torch::Tensor topk_ids,
                               torch::Tensor token_lora_mapping,
                               int64_t num_experts, int64_t block_size,
                               int64_t max_loras, int64_t max_num_tokens_padded,
                               int64_t max_num_m_blocks,
                               torch::Tensor sorted_token_ids,
                               torch::Tensor expert_ids,
                               torch::Tensor num_tokens_post_pad) {
  const int topk_num = topk_ids.size(1);

  TORCH_CHECK(block_size > 0, "block_size should be greater than 0. ");

  int device_max_shared_mem;
  auto dev = topk_ids.get_device();
  hipDeviceGetAttribute(&device_max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int32_t num_thread = max((int32_t)num_experts, 128);  // WARP_SIZE,
  TORCH_CHECK(num_thread <= 1024,
              "num_thread must be less than 1024, "
              "and fallback is not implemented yet.");
  const int32_t shared_mem = (num_thread + 1) * num_experts * sizeof(int32_t) +
                             (num_experts + 1) * sizeof(int32_t);

  if (shared_mem > device_max_shared_mem) {
    TORCH_CHECK(false,
                "Shared memory usage exceeds device limit, and global memory "
                "fallback is not implemented yet.");
  }

  VLLM_DISPATCH_INTEGRAL_TYPES(
      topk_ids.scalar_type(), "moe_lora_align_sum_kernel", [&] {
        dim3 blockDim(num_thread);
        auto kernel = moe_lora_align_sum_kernel<scalar_t, int32_t>;
        AT_CUDA_CHECK(VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(
            (void*)kernel, shared_mem));
        kernel<<<max_loras, blockDim, shared_mem, stream>>>(
            topk_ids.data_ptr<scalar_t>(),
            token_lora_mapping.data_ptr<int32_t>(), block_size, num_experts,
            max_loras, topk_ids.numel(), max_num_tokens_padded,
            max_num_m_blocks, sorted_token_ids.data_ptr<int32_t>(),
            expert_ids.data_ptr<int32_t>(), topk_num,
            num_tokens_post_pad.data_ptr<int32_t>());
      });
}