#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <ATen/ATen.h>
#include <ATen/cuda/Atomic.cuh>

#include "../cuda_compat.h"
#include "../dispatch_utils.h"

#define CEILDIV(x, y) (((x) + (y) - 1) / (y))

namespace vllm {
namespace moe {

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad, int32_t num_experts,
    int32_t padded_num_experts, int32_t experts_per_warp, int32_t block_size,
    size_t numel, int32_t* __restrict__ cumsum) {
  extern __shared__ int32_t shared_counts[];

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int my_expert_start = warp_id * experts_per_warp;

  for (int i = 0; i < experts_per_warp; ++i) {
    if (my_expert_start + i < padded_num_experts) {
      shared_counts[warp_id * experts_per_warp + i] = 0;
    }
  }

  __syncthreads();

  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int expert_id = topk_ids[i];
    int warp_idx = expert_id / experts_per_warp;
    int expert_offset = expert_id % experts_per_warp;
    atomicAdd(&shared_counts[warp_idx * experts_per_warp + expert_offset], 1);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      int expert_count = 0;
      int warp_idx = (i - 1) / experts_per_warp;
      int expert_offset = (i - 1) % experts_per_warp;
      expert_count = shared_counts[warp_idx * experts_per_warp + expert_offset];

      cumsum[i] =
          cumsum[i - 1] + CEILDIV(expert_count, block_size) * block_size;
    }
    *total_tokens_post_pad = cumsum[num_experts];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }
}

template <typename scalar_t>
__global__ void count_and_sort_expert_tokens_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ cumsum_buffer,
    size_t numel) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad = atomicAdd(&cumsum_buffer[expert_id], 1);
    sorted_token_ids[rank_post_pad] = i;
  }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., topk, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    scalar_t x = 0.0;
#pragma unroll
    for (int k = 0; k < TOPK; ++k) {
      x += VLLM_LDG(&input[token_idx * TOPK * d + k * d + idx]);
    }
    out[token_idx * d + idx] = x;
  }
}

template <typename scalar_t>
__global__ void moe_align_block_size_small_batch_expert_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad, int32_t num_experts,
    int32_t block_size, size_t numel) {
  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  extern __shared__ int32_t shared_mem[];
  int32_t* cumsum = shared_mem;
  int32_t* tokens_cnts = (int32_t*)(shared_mem + num_experts + 1);

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[(threadIdx.x + 1) * num_experts + i] = 0;
  }

  for (size_t i = tid; i < numel; i += stride) {
    ++tokens_cnts[(threadIdx.x + 1) * num_experts + topk_ids[i]];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    tokens_cnts[threadIdx.x] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[i * num_experts + threadIdx.x] +=
          tokens_cnts[(i - 1) * num_experts + threadIdx.x];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] =
          cumsum[i - 1] +
          CEILDIV(tokens_cnts[blockDim.x * num_experts + i - 1], block_size) *
              block_size;
    }
    *total_tokens_post_pad = static_cast<int32_t>(cumsum[num_experts]);
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad =
        tokens_cnts[threadIdx.x * num_experts + expert_id] + cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[threadIdx.x * num_experts + expert_id];
  }
}

}  // namespace moe
}  // namespace vllm

// taken from
// https://github.com/sgl-project/sglang/blob/8b5f83ed3b7d2a49ad5c5cd5aa61c5d502f47dbc
void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts,
                          int64_t block_size, torch::Tensor sorted_token_ids,
                          torch::Tensor experts_ids,
                          torch::Tensor num_tokens_post_pad) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t padded_num_experts =
      ((num_experts + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
  int experts_per_warp = WARP_SIZE;
  int threads = 1024;
  threads = ((threads + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

  VLLM_DISPATCH_INTEGRAL_AND_UNSIGNED_TYPES(
      topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
        // calc needed amount of shared mem for `cumsum` tensors
        auto options_int =
            torch::TensorOptions().dtype(torch::kInt).device(topk_ids.device());
        torch::Tensor cumsum_buffer =
            torch::zeros({num_experts + 1}, options_int);
        bool small_batch_expert_mode =
            (topk_ids.numel() < 1024) && (num_experts <= 64);

        if (small_batch_expert_mode) {
          const int32_t threads = max((int32_t)num_experts, WARP_SIZE);
          const int32_t shared_mem_size =
              ((threads + 1) * num_experts + (num_experts + 1)) *
              sizeof(int32_t);

          auto small_batch_expert_kernel =
              vllm::moe::moe_align_block_size_small_batch_expert_kernel<
                  scalar_t>;
          small_batch_expert_kernel<<<1, threads, shared_mem_size, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts, block_size,
              topk_ids.numel());
        } else {
          auto align_kernel = vllm::moe::moe_align_block_size_kernel<scalar_t>;

          size_t num_warps = CEILDIV(padded_num_experts, experts_per_warp);
          size_t shared_mem_size =
              num_warps * experts_per_warp * sizeof(int32_t);

          align_kernel<<<1, threads, shared_mem_size, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts,
              padded_num_experts, experts_per_warp, block_size,
              topk_ids.numel(), cumsum_buffer.data_ptr<int32_t>());

          const int block_threads = std::min(256, (int)threads);
          const int num_blocks =
              (topk_ids.numel() + block_threads - 1) / block_threads;
          const int max_blocks = 65535;
          const int actual_blocks = std::min(num_blocks, max_blocks);

          auto sort_kernel =
              vllm::moe::count_and_sort_expert_tokens_kernel<scalar_t>;
          sort_kernel<<<actual_blocks, block_threads, 0, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              cumsum_buffer.data_ptr<int32_t>(), topk_ids.numel());
        }
      });
}

void moe_sum(torch::Tensor& input,   // [num_tokens, topk, hidden_size]
             torch::Tensor& output)  // [num_tokens, hidden_size]
{
  const int hidden_size = input.size(-1);
  const int num_tokens = output.numel() / hidden_size;
  const int topk = input.size(1);

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (topk) {
    case 2:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 3:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 3><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 4:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    default:
      at::sum_out(output, input, 1);
      break;
  }
}
