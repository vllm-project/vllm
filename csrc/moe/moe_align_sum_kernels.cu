#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include "../cuda_compat.h"
#include "../dispatch_utils.h"

#define CEILDIV(x, y) (((x) + (y) - 1) / (y))

namespace vllm {
namespace moe {

namespace {
__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row,
                                         int32_t col) {
  // don't worry about overflow because num_experts is relatively small
  return row * total_col + col;
}
}  // namespace

template <typename scalar_t, typename token_cnts_t>
__global__ void moe_align_block_size_kernel(scalar_t* __restrict__ topk_ids,
                                            int32_t* sorted_token_ids,
                                            int32_t* expert_ids,
                                            int32_t* total_tokens_post_pad,
                                            int32_t num_experts,
                                            int32_t block_size, size_t numel) {
  const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  extern __shared__ int32_t shared_mem[];
  int32_t* cumsum = shared_mem;  // 1d tensor with shape (num_experts + 1)
  token_cnts_t* tokens_cnts = (token_cnts_t*)(shared_mem + blockDim.x + 1);

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[index(num_experts, threadIdx.x + 1, i)] = 0;
  }

  /**
   * In the first step we compute token_cnts[thread_index + 1][expert_index],
   * which counts how many tokens in the token shard of thread_index are
   * assigned to expert expert_index.
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    ++tokens_cnts[index(num_experts, threadIdx.x + 1, topk_ids[i])];
  }

  __syncthreads();

  // For each expert we accumulate the token counts from the different threads.
  if (threadIdx.x < num_experts) {
    tokens_cnts[index(num_experts, 0, threadIdx.x)] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[index(num_experts, i, threadIdx.x)] +=
          tokens_cnts[index(num_experts, i - 1, threadIdx.x)];
    }
  }

  __syncthreads();

  // We accumulate the token counts of all experts in thread 0.
  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] = cumsum[i - 1] +
                  CEILDIV(tokens_cnts[index(num_experts, blockDim.x, i - 1)],
                          block_size) *
                      block_size;
    }
    *total_tokens_post_pad = static_cast<int32_t>(cumsum[num_experts]);
  }

  __syncthreads();

  /**
   * For each expert, each thread processes the tokens of the corresponding
   * blocks and stores the corresponding expert_id for each block.
   */
  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  /**
   * Each thread processes a token shard, calculating the index of each token
   * after sorting by expert number. Given the example topk_ids =
   * [0,1,2,1,2,3,0,3,4] and block_size = 4, then the output would be [0, 6, *,
   * *, 1, 3, *, *, 2, 4, *, *, 5, 7, *, *, 8, *, *, *], where * represents a
   * padding value(preset in python).
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int32_t expert_id = topk_ids[i];
    /** The cumsum[expert_id] stores the starting index of the tokens that the
     * expert with expert_id needs to process, and
     * tokens_cnts[threadIdx.x][expert_id] stores the indices of the tokens
     * processed by the expert with expert_id within the current thread's token
     * shard.
     */
    int32_t rank_post_pad =
        tokens_cnts[index(num_experts, threadIdx.x, expert_id)] +
        cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[index(num_experts, threadIdx.x, expert_id)];
  }
}

// TODO(simon): this is temporarily adapted from
// https://github.com/sgl-project/sglang/commit/31548116a8dc8c6df7e146e0587335a59fc5b9d7
// we did this to unblock Deepseek V3 but there should be a better
// implementation to manage shared memory.
template <typename scalar_t>
__global__ void moe_align_block_size_global_mem_kernel(
    scalar_t* __restrict__ topk_ids, int32_t* sorted_token_ids,
    int32_t* expert_ids, int32_t* total_tokens_post_pad, int32_t num_experts,
    int32_t block_size, size_t numel, int32_t* tokens_cnts, int32_t* cumsum) {
  const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[index(num_experts, threadIdx.x + 1, i)] = 0;
  }

  /**
   * In the first step we compute token_cnts[thread_index + 1][expert_index],
   * which counts how many tokens in the token shard of thread_index are
   * assigned to expert expert_index.
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    ++tokens_cnts[index(num_experts, threadIdx.x + 1, topk_ids[i])];
  }

  __syncthreads();

  // For each expert we accumulate the token counts from the different threads.
  if (threadIdx.x < num_experts) {
    tokens_cnts[index(num_experts, 0, threadIdx.x)] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[index(num_experts, i, threadIdx.x)] +=
          tokens_cnts[index(num_experts, i - 1, threadIdx.x)];
    }
  }

  __syncthreads();

  // We accumulate the token counts of all experts in thread 0.
  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] = cumsum[i - 1] +
                  CEILDIV(tokens_cnts[index(num_experts, blockDim.x, i - 1)],
                          block_size) *
                      block_size;
    }
    *total_tokens_post_pad = cumsum[num_experts];
  }

  __syncthreads();

  /**
   * For each expert, each thread processes the tokens of the corresponding
   * blocks and stores the corresponding expert_id for each block.
   */
  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  /**
   * Each thread processes a token shard, calculating the index of each token
   * after sorting by expert number. Given the example topk_ids =
   * [0,1,2,1,2,3,0,3,4] and block_size = 4, then the output would be [0, 6, *,
   * *, 1, 3, *, *, 2, 4, *, *, 5, 7, *, *, 8, *, *, *], where * represents a
   * padding value(preset in python).
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int32_t expert_id = topk_ids[i];
    /** The cumsum[expert_id] stores the starting index of the tokens that the
     * expert with expert_id needs to process, and
     * tokens_cnts[threadIdx.x][expert_id] stores the indices of the tokens
     * processed by the expert with expert_id within the current thread's token
     * shard.
     */
    int32_t rank_post_pad =
        tokens_cnts[index(num_experts, threadIdx.x, expert_id)] +
        cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[index(num_experts, threadIdx.x, expert_id)];
  }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., topk, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    scalar_t x = 0.0;
#pragma unroll
    for (int k = 0; k < TOPK; ++k) {
      x += VLLM_LDG(&input[token_idx * TOPK * d + k * d + idx]);
    }
    out[token_idx * d + idx] = x;
  }
}

}  // namespace moe
}  // namespace vllm

void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts,
                          int64_t block_size, torch::Tensor sorted_token_ids,
                          torch::Tensor experts_ids,
                          torch::Tensor num_tokens_post_pad) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int device_max_shared_mem;
  auto dev = topk_ids.get_device();
  hipDeviceGetAttribute(&device_max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);

  const int32_t num_thread = max((int32_t)num_experts, WARP_SIZE);
  const int32_t shared_mem_i32 =
      ((num_thread + 1) * num_experts + (num_experts + 1)) * sizeof(int32_t);
  const int32_t shared_mem_i16 =
      ((num_thread + 1) * num_experts) * sizeof(uint16_t) +
      (num_experts + 1) * sizeof(int32_t);

  bool use_global_memory = false;
  bool use_i16 = false;  // Use uint16_t for shared memory token counts
  if (shared_mem_i32 < device_max_shared_mem) {
    // Do nothing in this case. We're all set to use int32_t token counts
  } else if (shared_mem_i16 < device_max_shared_mem &&
             topk_ids.numel() <= 65535) {
    // when nelements of topk_ids is smaller than 65535 (max value of uint16),
    // element value of token_cnts would also smaller than 65535,
    // so we can use uint16 as dtype of token_cnts
    use_i16 = true;
  } else {
    use_global_memory = true;
  }

  if (use_global_memory) {
    VLLM_DISPATCH_INTEGRAL_TYPES(
        topk_ids.scalar_type(), "moe_align_block_size_global_mem_kernel", [&] {
          // calc needed amount of shared mem for `tokens_cnts` and `cumsum`
          // tensors
          const int32_t num_thread = max((int32_t)num_experts, WARP_SIZE);

          auto options_int = torch::TensorOptions()
                                 .dtype(torch::kInt)
                                 .device(topk_ids.device());
          torch::Tensor token_cnts_buffer =
              torch::empty({(num_experts + 1) * num_experts}, options_int);
          torch::Tensor cumsum_buffer =
              torch::empty({num_experts + 1}, options_int);

          auto kernel =
              vllm::moe::moe_align_block_size_global_mem_kernel<scalar_t>;
          kernel<<<1, num_thread, 0, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts, block_size,
              topk_ids.numel(), token_cnts_buffer.data_ptr<int32_t>(),
              cumsum_buffer.data_ptr<int32_t>());
        });
  } else if (use_i16) {
    VLLM_DISPATCH_INTEGRAL_TYPES(
        topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
          // set dynamic shared mem
          auto kernel =
              vllm::moe::moe_align_block_size_kernel<scalar_t, uint16_t>;
          AT_CUDA_CHECK(VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(
              (void*)kernel, shared_mem_i16));
          kernel<<<1, num_thread, shared_mem_i16, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts, block_size,
              topk_ids.numel());
        });
  } else {
    VLLM_DISPATCH_INTEGRAL_TYPES(
        topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
          auto kernel =
              vllm::moe::moe_align_block_size_kernel<scalar_t, int32_t>;
          AT_CUDA_CHECK(VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(
              (void*)kernel, shared_mem_i32));
          kernel<<<1, num_thread, shared_mem_i32, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts, block_size,
              topk_ids.numel());
        });
  }
}

void moe_sum(torch::Tensor& input,   // [num_tokens, topk, hidden_size]
             torch::Tensor& output)  // [num_tokens, hidden_size]
{
  const int hidden_size = input.size(-1);
  const int num_tokens = output.numel() / hidden_size;
  const int topk = input.size(1);

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (topk) {
    case 2:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 3:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 3><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 4:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    default:
      at::sum_out(output, input, 1);
      break;
  }
}
