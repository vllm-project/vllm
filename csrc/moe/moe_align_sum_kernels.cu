#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include "../cuda_compat.h"
#include "../dispatch_utils.h"

#define CEILDIV(x, y) (((x) + (y) - 1) / (y))

namespace vllm {
namespace moe {

namespace {
__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row,
                                         int32_t col) {
  // don't worry about overflow because num_experts is relatively small
  return row * total_col + col;
}
}  // namespace

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(scalar_t* __restrict__ topk_ids,
                                            int32_t* sorted_token_ids,
                                            int32_t* expert_ids,
                                            int32_t* total_tokens_post_pad,
                                            int32_t num_experts,
                                            int32_t block_size, size_t numel) {
  const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  extern __shared__ int32_t shared_mem[];

  int32_t* tokens_cnts =
      shared_mem;  // 2d tensor with shape (blockDim.x + 1, num_experts)
  int32_t* cumsum =
      shared_mem +
      (blockDim.x + 1) * num_experts;  // 1d tensor with shape (num_experts + 1)

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[index(num_experts, threadIdx.x + 1, i)] = 0;
  }

  /**
   * In the first step we compute token_cnts[thread_index + 1][expert_index],
   * which counts how many tokens in the token shard of thread_index are
   * assigned to expert expert_index.
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    ++tokens_cnts[index(num_experts, threadIdx.x + 1, topk_ids[i])];
  }

  __syncthreads();

  // For each expert we accumulate the token counts from the different threads.
  if (threadIdx.x < num_experts) {
    tokens_cnts[index(num_experts, 0, threadIdx.x)] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[index(num_experts, i, threadIdx.x)] +=
          tokens_cnts[index(num_experts, i - 1, threadIdx.x)];
    }
  }

  __syncthreads();

  // We accumulate the token counts of all experts in thread 0.
  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] = cumsum[i - 1] +
                  CEILDIV(tokens_cnts[index(num_experts, blockDim.x, i - 1)],
                          block_size) *
                      block_size;
    }
    *total_tokens_post_pad = cumsum[num_experts];
  }

  __syncthreads();

  /**
   * For each expert, each thread processes the tokens of the corresponding
   * blocks and stores the corresponding expert_id for each block.
   */
  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  /**
   * Each thread processes a token shard, calculating the index of each token
   * after sorting by expert number. Given the example topk_ids =
   * [0,1,2,1,2,3,0,3,4] and block_size = 4, then the output would be [0, 6, *,
   * *, 1, 3, *, *, 2, 4, *, *, 5, 7, *, *, 8, *, *, *], where * represents a
   * padding value(preset in python).
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int32_t expert_id = topk_ids[i];
    /** The cumsum[expert_id] stores the starting index of the tokens that the
     * expert with expert_id needs to process, and
     * tokens_cnts[threadIdx.x][expert_id] stores the indices of the tokens
     * processed by the expert with expert_id within the current thread's token
     * shard.
     */
    int32_t rank_post_pad =
        tokens_cnts[index(num_experts, threadIdx.x, expert_id)] +
        cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[index(num_experts, threadIdx.x, expert_id)];
  }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., topk, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    scalar_t x = 0.0;
#pragma unroll
    for (int k = 0; k < TOPK; ++k) {
      x += VLLM_LDG(&input[token_idx * TOPK * d + k * d + idx]);
    }
    out[token_idx * d + idx] = x;
  }
}

}  // namespace moe
}  // namespace vllm

void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts,
                          int64_t block_size, torch::Tensor sorted_token_ids,
                          torch::Tensor experts_ids,
                          torch::Tensor num_tokens_post_pad) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_INTEGRAL_TYPES(
      topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
        // calc needed amount of shared mem for `tokens_cnts` and `cumsum`
        // tensors
        const int32_t num_thread = max((int32_t)num_experts, WARP_SIZE);
        const int32_t shared_mem =
            ((num_thread + 1) * num_experts + (num_experts + 1)) *
            sizeof(int32_t);

        // set dynamic shared mem
        auto kernel = vllm::moe::moe_align_block_size_kernel<scalar_t>;
        AT_CUDA_CHECK(VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(
            (void*)kernel, shared_mem));
        kernel<<<1, num_thread, shared_mem, stream>>>(
            topk_ids.data_ptr<scalar_t>(), sorted_token_ids.data_ptr<int32_t>(),
            experts_ids.data_ptr<int32_t>(),
            num_tokens_post_pad.data_ptr<int32_t>(), num_experts, block_size,
            topk_ids.numel());
      });
}

void moe_sum(torch::Tensor& input,   // [num_tokens, topk, hidden_size]
             torch::Tensor& output)  // [num_tokens, hidden_size]
{
  const int hidden_size = input.size(-1);
  const int num_tokens = output.numel() / hidden_size;
  const int topk = input.size(1);

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (topk) {
    case 2:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 3:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 3><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 4:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    default:
      at::sum_out(output, input, 1);
      break;
  }
}
