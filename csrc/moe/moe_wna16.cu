#include "hip/hip_runtime.h"

#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include "moe_wna16_utils.h"

#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

template <typename scalar_t, int bit, int GROUPS>
__global__ void moe_wna16_gemm_kernel(
    const scalar_t* __restrict__ input, scalar_t* __restrict__ output,

    const uint32_t* __restrict__ qweight, const scalar_t* __restrict__ scales,
    const uint32_t* __restrict__ qzeros,

    const float* __restrict__ topk_weights,
    const int32_t* __restrict__ sorted_token_ids,
    const int32_t* __restrict__ expert_ids,
    const int32_t* __restrict__ num_tokens_post_pad,

    uint16_t num_experts, uint16_t group_size, uint16_t top_k, uint32_t size_m,
    uint32_t size_n, uint32_t size_k, uint16_t BLOCK_SIZE_M,
    uint16_t BLOCK_SIZE_N, uint16_t BLOCK_SIZE_K, bool has_zp,
    bool mul_topk_weight) {
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ < 800
  if constexpr (std::is_same<scalar_t, hip_bfloat16>::value) {
    return;
  } else {
#endif

    using Dtype = ScalarType<scalar_t>;
    using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;

    if (blockIdx.x * BLOCK_SIZE_M >= num_tokens_post_pad[0]) return;

    const int32_t offset_n = blockIdx.y * BLOCK_SIZE_N + threadIdx.x;
    const int32_t offset_k = blockIdx.z * BLOCK_SIZE_K;

    const int32_t expert_id = expert_ids[blockIdx.x];

    int32_t num_valid_tokens = 0;
    extern __shared__ uint16_t block_input_tmp[];
    scalar_t* block_input = reinterpret_cast<scalar_t*>(block_input_tmp);
    scalar_t2* block_input_half2 = reinterpret_cast<scalar_t2*>(block_input);

    // load BLOCK_SIZE_M * BLOCK_SIZE_K into shared memory
    for (int m = 0; m < BLOCK_SIZE_M; m++) {
      const int32_t offset_m = blockIdx.x * BLOCK_SIZE_M + m;
      const int32_t token_index = sorted_token_ids[offset_m];
      if (token_index / top_k >= size_m) break;

      num_valid_tokens = m + 1;
      if (blockIdx.z == 0 && offset_n < size_n)
        output[token_index * size_n + offset_n] = Dtype::int2num(0);

      if (expert_id != -1) {
        int k_per_thread = DIVIDE(BLOCK_SIZE_K, BLOCK_SIZE_N);
        for (int i = 0; i < k_per_thread; i++) {
          int k = BLOCK_SIZE_N * i + threadIdx.x;
          if (k >= BLOCK_SIZE_K) break;
          if (offset_k + k >= size_k) break;

          // load input to shared memory
          // use a special layout to fit the layout of dequanted-weight
          int origin_k;
          if constexpr (bit == 4) {
            // [0, 4, 1, 5, 2, 6, 3, 7]
            int8_t order = (threadIdx.x % 2) * 4 + ((threadIdx.x % 8) / 2);
            origin_k = BLOCK_SIZE_N * i + threadIdx.x / 8 * 8 + order;
          } else {
            // [0, 2, 1, 3]
            int8_t order = (threadIdx.x % 2) * 2 + ((threadIdx.x % 4) / 2);
            origin_k = BLOCK_SIZE_N * i + threadIdx.x / 4 * 4 + order;
          }

          origin_k += token_index / top_k * size_k + blockIdx.z * BLOCK_SIZE_K;
          block_input[m * BLOCK_SIZE_K + k] = input[origin_k];
        }
      }
    }

    if (expert_id == -1) return;
    __syncthreads();
    if (threadIdx.x >= BLOCK_SIZE_N || offset_n >= size_n) return;

    float res[64];  // assume BLOCK_SIZE_M <= 64
    scalar_t2 res2;
    scalar_t2 scale_f2;
    scalar_t2 qzero_f2;

    // note that (size_n * size_k * expert_id) may greater than 2 ** 31
    constexpr int8_t pack_factor = 32 / bit;
    const uint64_t expert_offset = ((uint64_t)size_n) * size_k * expert_id;
    const uint32_t* expert_qweight = qweight + expert_offset / pack_factor;
    const scalar_t* expert_scales = scales + expert_offset / group_size;
    const uint32_t* expert_qzeros =
        qzeros + expert_offset / group_size / pack_factor;

    // load 4*int32 one time: 4 int32 = 128 bit = 1 float4
    // weight would be loaded in loop
    uint32_t expert_qweight_tmp[4];
    float4* expert_qweight_tmp_float4 =
        reinterpret_cast<float4*>(expert_qweight_tmp);

    // load all required scales one time
    scalar_t expert_scales_groups[GROUPS];
    int scales_offset_tmp =
        (offset_n * size_k + offset_k) / group_size / GROUPS;
    if constexpr (GROUPS == 1) {
      *expert_scales_groups = expert_scales[scales_offset_tmp];
    } else if constexpr (GROUPS == 2) {
      float* expert_scales_groups_tmp =
          reinterpret_cast<float*>(expert_scales_groups);
      *expert_scales_groups_tmp =
          reinterpret_cast<const float*>(expert_scales)[scales_offset_tmp];
    } else if constexpr (GROUPS == 4) {
      float2* expert_scales_groups_tmp =
          reinterpret_cast<float2*>(expert_scales_groups);
      *expert_scales_groups_tmp =
          reinterpret_cast<const float2*>(expert_scales)[scales_offset_tmp];
    } else if constexpr (GROUPS == 8) {
      float4* expert_scales_groups_tmp =
          reinterpret_cast<float4*>(expert_scales_groups);
      *expert_scales_groups_tmp =
          reinterpret_cast<const float4*>(expert_scales)[scales_offset_tmp];
    }

    // load all required qzeros one time
    uint8_t expert_qzeros_groups[GROUPS];
    if (!has_zp) {
      if constexpr (bit == 4) {
        qzero_f2 = Dtype::num2num2(Dtype::int2num(8));
      } else {
        qzero_f2 = Dtype::num2num2(Dtype::int2num(128));
      }
    } else {
      int qzeros_offset_tmp =
          (offset_n / (8 / bit)) * (size_k / group_size / GROUPS) +
          offset_k / group_size / GROUPS;
      if constexpr (GROUPS == 1) {
        uint8_t* expert_qzeros_groups_tmp =
            reinterpret_cast<uint8_t*>(expert_qzeros_groups);
        *expert_qzeros_groups_tmp =
            reinterpret_cast<const uint8_t*>(expert_qzeros)[qzeros_offset_tmp];
      } else if constexpr (GROUPS == 2) {
        uint16_t* expert_qzeros_groups_tmp =
            reinterpret_cast<uint16_t*>(expert_qzeros_groups);
        *expert_qzeros_groups_tmp =
            reinterpret_cast<const uint16_t*>(expert_qzeros)[qzeros_offset_tmp];
      } else if constexpr (GROUPS == 4) {
        uint32_t* expert_qzeros_groups_tmp =
            reinterpret_cast<uint32_t*>(expert_qzeros_groups);
        *expert_qzeros_groups_tmp =
            reinterpret_cast<const uint32_t*>(expert_qzeros)[qzeros_offset_tmp];
      } else if constexpr (GROUPS == 8) {
        uint64_t* expert_qzeros_groups_tmp =
            reinterpret_cast<uint64_t*>(expert_qzeros_groups);
        *expert_qzeros_groups_tmp =
            reinterpret_cast<const uint64_t*>(expert_qzeros)[qzeros_offset_tmp];
      }
    }

    for (int tmp_k = 0; tmp_k < BLOCK_SIZE_K / pack_factor; tmp_k++) {
      int k = offset_k + tmp_k * pack_factor;
      if (k >= size_k) break;
      const int32_t weight_offset = offset_n * size_k + k;

      if (tmp_k % 4 == 0) {
        *expert_qweight_tmp_float4 = reinterpret_cast<const float4*>(
            expert_qweight)[weight_offset / pack_factor / 4];
      }

      if (tmp_k % (group_size / pack_factor) == 0) {
        scalar_t scale_f =
            expert_scales_groups[tmp_k / (group_size / pack_factor)];
        scale_f2 = Dtype::num2num2(scale_f);

        if (has_zp) {
          uint8_t qzero =
              expert_qzeros_groups[tmp_k / (group_size / pack_factor)];
          if constexpr (bit == 4) {
            qzero = (qzero >> ((threadIdx.x % 2) * 4)) & 0xF;
          }
          qzero_f2 = Dtype::num2num2(Dtype::int2num(qzero));
        }
      }

      scalar_t2 weight_half2[16 / bit];
      dequant<scalar_t2, bit>(expert_qweight_tmp[tmp_k % 4], weight_half2);

      for (int m = 0; m < num_valid_tokens; m++) {
        res2 = {};

#pragma unroll
        for (int i = 0; i < 16 / bit; i++) {
          int32_t offset_input = m * BLOCK_SIZE_K / 2 + tmp_k * (16 / bit) + i;
          res2 = __hfma2(__hmul2(__hsub2(weight_half2[i], qzero_f2), scale_f2),
                         block_input_half2[offset_input], res2);
        }

        if (tmp_k == 0) {
          res[m] = Dtype::num2float(res2.x) + Dtype::num2float(res2.y);
        } else {
          res[m] += Dtype::num2float(res2.x) + Dtype::num2float(res2.y);
        }
      }
    }

    for (int m = 0; m < num_valid_tokens; ++m) {
      const int32_t token_index =
          sorted_token_ids[blockIdx.x * BLOCK_SIZE_M + m];
      if (mul_topk_weight) {
        res[m] *= topk_weights[token_index];
      }
      atomicAdd(&output[token_index * size_n + offset_n],
                Dtype::float2num(res[m]));
    }

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ < 800
  }
#endif
}

template <typename scalar_t>
void run_moe_wna16_gemm(const scalar_t* input, scalar_t* output,
                        const uint32_t* b_qweight, const scalar_t* b_scales,
                        const uint32_t* b_qzeros, const float* topk_weights,
                        const int32_t* sorted_token_ids,
                        const int32_t* expert_ids,
                        const int32_t* num_tokens_post_pad, int num_experts,
                        int group_size, int num_token_blocks, int top_k,
                        int size_m, int size_n, int size_k, int BLOCK_SIZE_M,
                        int BLOCK_SIZE_N, int BLOCK_SIZE_K, int bit,
                        bool has_zp, bool mul_topk_weight) {
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_SIZE_N;
  blockDim.y = 1;
  blockDim.z = 1;
  gridDim.x = num_token_blocks;
  gridDim.y = DIVIDE(size_n, BLOCK_SIZE_N);
  gridDim.z = DIVIDE(size_k, BLOCK_SIZE_K);

  auto kernel = moe_wna16_gemm_kernel<scalar_t, 4, 1>;
  if (bit == 4) {
    if (BLOCK_SIZE_K / group_size == 2) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 4, 2>;
    } else if (BLOCK_SIZE_K / group_size == 4) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 4, 4>;
    } else if (BLOCK_SIZE_K / group_size == 8) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 4, 8>;
    }
  } else {
    if (BLOCK_SIZE_K / group_size == 1) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 8, 1>;
    } else if (BLOCK_SIZE_K / group_size == 2) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 8, 2>;
    } else if (BLOCK_SIZE_K / group_size == 4) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 8, 4>;
    } else if (BLOCK_SIZE_K / group_size == 8) {
      kernel = moe_wna16_gemm_kernel<scalar_t, 8, 8>;
    }
  }

  const int shared_mem_size = BLOCK_SIZE_M * BLOCK_SIZE_K * 2;
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  kernel<<<gridDim, blockDim, shared_mem_size, stream>>>(
      input, output, b_qweight, b_scales, b_qzeros, topk_weights,
      sorted_token_ids, expert_ids, num_tokens_post_pad, num_experts,
      group_size, top_k, size_m, size_n, size_k, BLOCK_SIZE_M, BLOCK_SIZE_N,
      BLOCK_SIZE_K, has_zp, mul_topk_weight);
}

torch::Tensor moe_wna16_gemm(torch::Tensor input, torch::Tensor output,
                             torch::Tensor b_qweight, torch::Tensor b_scales,
                             std::optional<torch::Tensor> b_qzeros,
                             std::optional<torch::Tensor> topk_weights,
                             torch::Tensor sorted_token_ids,
                             torch::Tensor expert_ids,
                             torch::Tensor num_tokens_post_pad, int64_t top_k,
                             int64_t BLOCK_SIZE_M, int64_t BLOCK_SIZE_N,
                             int64_t BLOCK_SIZE_K, int64_t bit) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  auto options =
      torch::TensorOptions().dtype(input.dtype()).device(input.device());

  const int num_experts = b_qweight.size(0);
  const int size_m = input.size(0);
  const int size_n = b_qweight.size(1);
  const int size_k = input.size(1);
  const int group_size = size_k / b_scales.size(2);

  int64_t EM = sorted_token_ids.size(0);
  if (size_m <= BLOCK_SIZE_M) {
    EM = min(EM, size_m * BLOCK_SIZE_M * top_k);
  }
  const int num_token_blocks = (EM + BLOCK_SIZE_M - 1) / BLOCK_SIZE_M;

  const uint32_t* b_qzeros_ptr;
  if (b_qzeros.has_value())
    b_qzeros_ptr = (const uint32_t*)b_qzeros.value().data_ptr<uint8_t>();
  const float* topk_weights_ptr;
  if (topk_weights.has_value())
    topk_weights_ptr = (const float*)topk_weights.value().data_ptr();

  int groups_per_block_row = BLOCK_SIZE_K / group_size;
  TORCH_CHECK(bit == 4 || bit == 8, "bit must be 4 or 8");
  TORCH_CHECK(size_k % BLOCK_SIZE_K == 0,
              "size_k must divisible by BLOCK_SIZE_K");
  TORCH_CHECK(BLOCK_SIZE_K % group_size == 0,
              "BLOCK_SIZE_K must divisible by group_size");
  TORCH_CHECK(BLOCK_SIZE_M <= 64, "BLOCK_SIZE_M must less or equal to 64");
  TORCH_CHECK(groups_per_block_row == 1 || groups_per_block_row == 2 ||
                  groups_per_block_row == 4 || groups_per_block_row == 8,
              "BLOCK_SIZE_K // group_size must be one of [1, 2, 4, 8]");

  if (input.scalar_type() == at::ScalarType::Half) {
    run_moe_wna16_gemm<half>(
        (const half*)input.data_ptr<at::Half>(),
        (half*)output.data_ptr<at::Half>(),
        (const uint32_t*)b_qweight.data_ptr<uint8_t>(),
        (const half*)b_scales.data_ptr<at::Half>(), b_qzeros_ptr,
        topk_weights_ptr, sorted_token_ids.data_ptr<int32_t>(),
        expert_ids.data_ptr<int32_t>(), num_tokens_post_pad.data_ptr<int32_t>(),
        num_experts, group_size, num_token_blocks, top_k, size_m, size_n,
        size_k, BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K, bit,
        b_qzeros.has_value(), topk_weights.has_value());
  } else if (input.scalar_type() == at::ScalarType::BFloat16) {
    run_moe_wna16_gemm<hip_bfloat16>(
        (const hip_bfloat16*)input.data_ptr<at::BFloat16>(),
        (hip_bfloat16*)output.data_ptr<at::BFloat16>(),
        (const uint32_t*)b_qweight.data_ptr<uint8_t>(),
        (const hip_bfloat16*)b_scales.data_ptr<at::BFloat16>(), b_qzeros_ptr,
        topk_weights_ptr, sorted_token_ids.data_ptr<int32_t>(),
        expert_ids.data_ptr<int32_t>(), num_tokens_post_pad.data_ptr<int32_t>(),
        num_experts, group_size, num_token_blocks, top_k, size_m, size_n,
        size_k, BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K, bit,
        b_qzeros.has_value(), topk_weights.has_value());
  } else {
    TORCH_CHECK(false, "moe_wna16_gemm only supports bfloat16 and float16");
  }
  return output;
}
