#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "core/exception.hpp"
#include "core/scalar_type.hpp"
#include "core/registration.h"
#include "marlin_kernels/marlin_moe_kernel_ku4b8.h"
#include "marlin_kernels/marlin_moe_kernel_ku8b128.h"
#include "marlin_kernels/marlin_moe_kernel_ku4.h"

template <typename T>
inline std::string str(T x) {
  return std::to_string(x);
}

namespace marlin_moe {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800

// For a given "a" of size [M,K] performs a permutation of the K columns based
// on the given "perm" indices.
__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr,
                                    int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {
  int start_row = block_rows * blockIdx.x;
  int finish_row = start_row + block_rows;
  if (finish_row > size_m) {
    finish_row = size_m;
  }
  int cur_block_rows = finish_row - start_row;

  int row_stride = size_k * sizeof(half) / 16;

  auto permute_row = [&](int row) {
    int iters = size_k / blockDim.x;
    int rest = size_k % blockDim.x;

    int offset = row * row_stride;

    half const* a_row_half = reinterpret_cast<half const*>(a_int4_ptr + offset);
    half* out_half = reinterpret_cast<half*>(out_int4_ptr + offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      int cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];

      out_half[cur_k] = a_row_half[src_pos];

      base_k += blockDim.x;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        int cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];

        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int i = 0; i < cur_block_rows; i++) {
    int cur_row = start_row + i;
    if (cur_row < size_m) {
      permute_row(cur_row);
    }
  }
}

__global__ void compute_expert_offsets(int const* __restrict__ topk_ids,
                                       int* __restrict__ expert_offsets,
                                       int topk_length, int block_size) {
  int expert_id = threadIdx.x;
  int num_experts = blockDim.x;

  int occurrences = 0;
  for (int i = 0; i < topk_length; ++i) {
    occurrences += (topk_ids[i] == expert_id);
  }
  expert_offsets[expert_id + 1] = occurrences;
  __syncthreads();

  if (threadIdx.x == 0) {
    int tot_offset = 0;
    expert_offsets[0] = 0;
    for (int i = 0; i < num_experts; ++i) {
      tot_offset += ceildiv(expert_offsets[i + 1], block_size) * block_size;
      expert_offsets[i + 1] = tot_offset;
    }
  }
  __syncthreads();
}

#else

__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr,
                                    int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {
  // Marlin is not implemented yet for SM < 8.0
  assert(false);
  return;
}

__global__ void compute_expert_offsets(int const* __restrict__ topk_ids,
                                       int* __restrict__ expert_offsets,
                                       int topk_length, int block_size) {
  // Marlin is not implemented yet for SM < 8.0
  assert(false);
  return;
}

#endif

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

typedef struct {
  int max_m_blocks;
  thread_config_t tb_cfg;
} exec_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},  // Default
    {128, 64, 128},   // Reduce N 2X, same K
    {64, 256, 256},   // Reduce K 2X, increase N 2X
    {64, 128, 128},   // Reduce K 2X, same N
    {64, 64, 128},    // Reduce both 2X
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},   // Default
    {128, 128, 256},  // Reduce N 2X, increase K 2X
    {64, 128, 128},   // Reduce N 2X, same K
    {128, 64, 128},   // Reduce N 4X, increase K 2X
    {64, 64, 128},    // Reduce N 4X, same K
};

int get_scales_cache_size(thread_config_t const& th_config, int prob_m,
                          int prob_n, int prob_k, int num_bits, int group_size,
                          bool has_act_order, bool is_k_full) {
  bool cache_scales_chunk = has_act_order && !is_k_full;

  int tb_n = th_config.thread_n;
  int tb_k = th_config.thread_k;

  // Get max scale groups per thread-block
  int tb_groups;
  if (group_size == -1) {
    tb_groups = 1;
  } else if (group_size == 0) {
    tb_groups = ceildiv(tb_k, 32);  // Worst case is 32 group size
  } else {
    tb_groups = ceildiv(tb_k, group_size);
  }

  if (cache_scales_chunk) {
    int load_groups =
        tb_groups * STAGES * 2;          // Chunk size is 2x pipeline over dim K
    load_groups = max(load_groups, 32);  // We load at least 32 scale groups
    return load_groups * tb_n * 4;

  } else {
    int tb_scales = tb_groups * tb_n * 2;

    return tb_scales * STAGES;
  }
}

bool is_valid_cache_size(thread_config_t const& th_config, int max_m_blocks,
                         int prob_m, int prob_n, int prob_k, int num_bits,
                         int scales_cache_size, int max_shared_mem) {
  int pack_factor = 32 / num_bits;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;

  int b_size = (tb_k * tb_n / pack_factor) * 4;

  // Get A size
  int m_blocks = ceildiv(prob_m, 16);
  int tb_max_m = 16;

  while (true) {
    if (m_blocks >= max_m_blocks) {
      tb_max_m *= max_m_blocks;
      break;
    }

    max_m_blocks--;
    if (max_m_blocks == 0) {
      TORCH_CHECK(false, "Unexpected m_blocks = ", m_blocks);
    }
  }

  int a_size = (tb_max_m * tb_k) * 2;

  float pipe_size = (a_size + b_size) * STAGES;

  TORCH_CHECK(max_shared_mem / 2 > scales_cache_size);  // Sanity

  return pipe_size < 0.95f * (max_shared_mem - scales_cache_size);
}

bool is_valid_config(thread_config_t const& th_config, int max_m_blocks,
                     int prob_m, int prob_n, int prob_k, int num_bits,
                     int group_size, bool has_act_order, bool is_k_full,
                     int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // thread_k can be only 128 or 64 (because it must be less than groupsize
  // which is 128)
  if (th_config.thread_k != 128 && th_config.thread_k != 64) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  //  Determine cache for scales
  int scales_cache_size =
      get_scales_cache_size(th_config, prob_m, prob_n, prob_k, num_bits,
                            group_size, has_act_order, is_k_full);

  // Check that pipeline fits into cache
  if (!is_valid_cache_size(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                           num_bits, scales_cache_size, max_shared_mem)) {
    return false;
  }

  return true;
}

exec_config_t determine_thread_config(int prob_m, int prob_n, int prob_k,
                                      int num_bits, int group_size,
                                      bool has_act_order, bool is_k_full,
                                      int max_shared_mem) {
  int max_m_blocks = 4;
  while (max_m_blocks > 0) {
    if (prob_m <= 16) {
      for (auto th_config : small_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, has_act_order, is_k_full,
                            max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    } else {
      for (auto th_config : large_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, has_act_order, is_k_full,
                            max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    }

    max_m_blocks--;  // Process less M blocks per invocation to reduce cache
                     // usage
  }

  return exec_config_t{0, {-1, -1, -1}};
}

#define CALL_MOE_KERNEL_FUNCTION(KERNEL_FUNCTION)                             \
  else if (KERNEL_FUNCTION(                                                   \
               q_type, thread_n_blocks, thread_k_blocks, has_act_order,       \
               group_blocks, num_threads, blocks, max_shared_mem, stream,     \
               A_ptr, B_ptr, C_ptr, sorted_ids_ptr, topk_weights_ptr, s_ptr,  \
               zp_ptr, g_idx_ptr, expert_offsets_ptr, num_groups, expert_idx, \
               num_experts, topk, prob_m, prob_n, prob_k, tot_m, locks,       \
               replicate_input, apply_weights, m_block, max_par,              \
               exec_cfg.max_m_blocks)) {                                      \
  }

void marlin_mm_moe(const void* A, const void* B, void* C,
                   const void* sorted_ids, const void* topk_weights,
                   const void* topk_ids, const void* s, void* zp,
                   const void* g_idx, const void* perm, void* a_tmp,
                   void* expert_offsets, int prob_m, int prob_n, int prob_k,
                   void* workspace, vllm::ScalarType const& q_type,
                   bool has_act_order, bool is_k_full, bool has_zp,
                   int num_groups, int group_size, int num_experts, int topk,
                   int moe_block_size, int dev, hipStream_t stream,
                   int thread_k, int thread_n, int sms, int max_par,
                   bool replicate_input, bool apply_weights) {
  TORCH_CHECK(prob_m > 0 && prob_n > 0 && prob_k > 0, "Invalid MNK = [", prob_m,
              ", ", prob_n, ", ", prob_k, "]");

  if (sms == -1) {
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  }

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  int num_bits = q_type.size_bits();

  // Set thread config
  exec_config_t exec_cfg;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    exec_cfg =
        exec_config_t{4, thread_config_t{thread_k, thread_n, USER_THREADS}};
  } else {
    // Auto config
    exec_cfg =
        determine_thread_config(prob_m, prob_n, prob_k, num_bits, group_size,
                                has_act_order, is_k_full, max_shared_mem);
  }

  TORCH_CHECK(exec_cfg.max_m_blocks > 0 &&
                  is_valid_config(exec_cfg.tb_cfg, exec_cfg.max_m_blocks,
                                  prob_m, prob_n, prob_k, num_bits, group_size,
                                  has_act_order, is_k_full, max_shared_mem),
              "Invalid thread config: max_m_blocks = ", exec_cfg.max_m_blocks,
              ", thread_k = ", exec_cfg.tb_cfg.thread_k,
              ", thread_n = ", exec_cfg.tb_cfg.thread_n,
              ", num_threads = ", exec_cfg.tb_cfg.num_threads, " for MKN = [",
              prob_m, ", ", prob_k, ", ", prob_n, "] and num_bits = ", num_bits,
              ", group_size = ", group_size,
              ", has_act_order = ", has_act_order, ", is_k_full = ", is_k_full,
              ", max_shared_mem = ", max_shared_mem);

  int num_threads = exec_cfg.tb_cfg.num_threads;
  thread_k = exec_cfg.tb_cfg.thread_k;
  thread_n = exec_cfg.tb_cfg.thread_n;

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;

  int blocks = sms;

  TORCH_CHECK(prob_n % thread_n == 0, "prob_n = ", prob_n,
              " is not divisible by thread_n = ", thread_n);
  TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
              " is not divisible by thread_k = ", thread_k);

  int group_blocks = 0;
  if (has_act_order) {
    if (is_k_full) {
      TORCH_CHECK(group_size != -1);
      group_blocks = group_size / 16;
      TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                  " is not divisible by group_blocks = ", group_blocks);
    } else {
      TORCH_CHECK(group_size == 0);
      group_blocks = 0;
    }

  } else {
    if (group_size == -1) {
      group_blocks = -1;
    } else {
      group_blocks = group_size / 16;
      TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                  " is not divisible by group_blocks = ", group_blocks);
    }
  }

  int tot_m = prob_m;

  const int* topk_ids_ptr = (const int*)topk_ids;
  int* expert_offsets_ptr = (int*)expert_offsets;
  compute_expert_offsets<<<1, num_experts, 0, stream>>>(
      topk_ids_ptr, expert_offsets_ptr, tot_m * topk, moe_block_size);

  bool do_permute_a = has_act_order;

  // If we have a full K, then we can run the non-act-order version of Marlin
  // (since the weight rows are reordered by increasing group ids, and by
  // having a full K, we have full original groups)
  if (is_k_full) {
    has_act_order = false;
  }

  int pack_factor = 32 / q_type.size_bits();

  for (int expert_idx = 0; expert_idx < num_experts; ++expert_idx) {
    const int4* A_ptr = (const int4*)A;
    int4* a_tmp_ptr = (int4*)a_tmp;
    const int4* B_ptr =
        (const int4*)B + (prob_n * prob_k / (pack_factor * 4)) * expert_idx;
    int4* C_ptr = (int4*)C;
    const float* topk_weights_ptr = (const float*)topk_weights;
    const int* sorted_ids_ptr = (const int*)sorted_ids;
    const int4* s_ptr = (const int4*)s + num_groups * prob_n / 8 * expert_idx;
    const int4* zp_ptr =
        (const int4*)zp + num_groups * prob_n / (pack_factor * 4) * expert_idx;
    const int* g_idx_ptr = (const int*)g_idx + prob_k * expert_idx;
    const int* perm_ptr = (const int*)perm + prob_k * expert_idx;
    int* locks = (int*)workspace;

    if (do_permute_a) {
      // Permute A columns
      int topk_rows = replicate_input ? tot_m : tot_m * topk;
      int block_rows = ceildiv(topk_rows, blocks);
      permute_cols_kernel<<<blocks, num_threads, 0, stream>>>(
          A_ptr, perm_ptr, a_tmp_ptr, topk_rows, prob_k, block_rows);
      A_ptr = a_tmp_ptr;
    }

    int tot_m_blocks = ceildiv(tot_m, 16);
    for (int m_block = 0; m_block < tot_m_blocks;
         m_block += 4 * exec_cfg.max_m_blocks) {
      if (false) {
      }
      CALL_MOE_KERNEL_FUNCTION(call_marlin_moe_kernel_ku4b8)
      CALL_MOE_KERNEL_FUNCTION(call_marlin_moe_kernel_ku8b128)
      CALL_MOE_KERNEL_FUNCTION(call_marlin_moe_kernel_ku4)
      else {
        TORCH_CHECK(false, "Unsupported shapes: MNK = [" + str(prob_m) + ", " +
                               str(prob_n) + ", " + str(prob_k) + "]" +
                               ", has_act_order = " + str(has_act_order) +
                               ", num_groups = " + str(num_groups) +
                               ", group_size = " + str(group_size) +
                               ", thread_n_blocks = " + str(thread_n_blocks) +
                               ", thread_k_blocks = " + str(thread_k_blocks));
      }
    }
  }
}

}  // namespace marlin_moe

torch::Tensor marlin_gemm_moe(
    const torch::Tensor& a, const torch::Tensor& b_q_weights,
    const torch::Tensor& sorted_ids, const torch::Tensor& topk_weights,
    const torch::Tensor& topk_ids, const torch::Tensor& b_scales,
    torch::Tensor& b_zeros, const torch::Tensor& g_idx,
    const torch::Tensor& perm, torch::Tensor& workspace,
    vllm::ScalarTypeId const b_q_type_id, int64_t size_m, int64_t size_n,
    int64_t size_k, bool is_k_full, int64_t num_experts, int64_t topk,
    int64_t moe_block_size, bool replicate_input, bool apply_weights) {
  vllm::ScalarType const b_q_type = vllm::ScalarType::from_id(b_q_type_id);
  bool has_zp = b_zeros.size(1) != 0;
  if (has_zp) {
    TORCH_CHECK(
        b_q_type == vllm::kU4,
        "b_q_type must be u4 when has_zp = True. Got = ", b_q_type.str());
  } else {
    TORCH_CHECK(
        b_q_type == vllm::kU4B8 || b_q_type == vllm::kU8B128,
        "b_q_type must be uint4b8 or uint8b128. Got = ", b_q_type.str());
  }

  int pack_factor = 32 / b_q_type.size_bits();

  int max_par = 4;

  int dev = a.get_device();

  auto options_dtype =
      torch::TensorOptions().dtype(a.dtype()).device(a.device());
  auto options_int =
      torch::TensorOptions().dtype(torch::kInt).device(a.device());
  torch::Tensor c = torch::zeros({size_m, topk, size_n}, options_dtype);
  torch::Tensor a_tmp =
      replicate_input ? torch::zeros({size_m, size_k}, options_dtype)
                      : torch::zeros({size_m, topk, size_k}, options_dtype);
  torch::Tensor expert_offsets = torch::empty({num_experts + 1}, options_int);

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel (can usually be left as auto -1)
  int sms = -1;

  // Detect groupsize and act_order
  int num_groups = -1;
  int group_size = -1;
  bool has_act_order = g_idx.size(1) != 0;

  int b_rank = b_scales.sizes().size();
  TORCH_CHECK(b_rank == 3, "b_scales rank = ", b_rank, " is not 3");
  TORCH_CHECK(b_scales.size(2) == size_n, "b_scales dim 2 = ", b_scales.size(2),
              " is not size_n = ", size_n);
  num_groups = b_scales.size(1);

  TORCH_CHECK(VLLM_IMPLIES(!is_k_full, has_act_order),
              "if is_k_full is false, has_act_order must be true");

  if (has_act_order) {
    if (is_k_full) {
      TORCH_CHECK(num_groups > 1, "For act_order, num_groups must be > 1");
      TORCH_CHECK(size_k % num_groups == 0, "size_k = ", size_k,
                  ", is not divisible by num_groups = ", num_groups);
      group_size = size_k / num_groups;
    } else {
      group_size = 0;
    }

  } else {
    if (num_groups > 1) {
      TORCH_CHECK(
          size_k % num_groups == 0, "size_k = ", size_k,
          ", is not divisible by b_scales.size(0) = ", b_scales.size(0));
      group_size = size_k / num_groups;
    } else {
      group_size = -1;
    }
  }

  // Verify b_zeros
  if (has_zp) {
    int rank = b_zeros.sizes().size();
    TORCH_CHECK(rank == 3, "b_zeros rank = ", rank, " is not 3");
    TORCH_CHECK(b_zeros.size(1) == num_groups,
                "b_zeros dim 1 = ", b_zeros.size(1),
                " is not num_groups = ", num_groups);
    TORCH_CHECK(b_zeros.size(2) == size_n / pack_factor,
                "b_zeros dim 2 = ", b_zeros.size(2),
                " is not size_n / pack_factor = ", size_n / pack_factor);
  }

  marlin_moe::marlin_mm_moe(
      a.data_ptr(), b_q_weights.data_ptr(), c.data_ptr(), sorted_ids.data_ptr(),
      topk_weights.data_ptr(), topk_ids.data_ptr(), b_scales.data_ptr(),
      b_zeros.data_ptr(), g_idx.data_ptr(), perm.data_ptr(), a_tmp.data_ptr(),
      expert_offsets.data_ptr(), size_m, size_n, size_k, workspace.data_ptr(),
      b_q_type, has_act_order, is_k_full, has_zp, num_groups, group_size,
      num_experts, topk, moe_block_size, dev,
      at::cuda::getCurrentCUDAStream(dev), thread_k, thread_n, sms, max_par,
      replicate_input, apply_weights);
  return c;
}

TORCH_LIBRARY_IMPL_EXPAND(TORCH_EXTENSION_NAME, CUDA, m) {
  m.impl("marlin_gemm_moe", &marlin_gemm_moe);
}
