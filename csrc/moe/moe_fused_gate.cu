#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <stdio.h>
#include <torch/all.h>

#include <cfloat>
#include <type_traits>
template <typename T, int N>
using AlignedArray = cutlass::AlignedArray<T, N>;
using bfloat16_t = cutlass::bfloat16_t;
using float16_t = cutlass::half_t;
using float32_t = float;

// QQ NOTE: to handle the case for at::Half, error: more than one operator ">"
// matches these operands: built-in operator "arithmetic > arithmetic" function
// "operator>(const __half &, const __half &)"
template <typename T>
__device__ inline bool cmp_gt(const T& a, const T& b) {
  if constexpr (std::is_same<T, at::Half>::value) {
    // at::Half (or float16_t in our native case) causes ambiguity, so we cast
    // to float.
    return static_cast<float>(a) > static_cast<float>(b);
  } else {
    // For types like float, at::BFloat16, or cutlass::half_t /
    // cutlass::bfloat16_t, assume operator> works as expected.
    return a > b;
  }
}

template <typename T>
__device__ inline bool cmp_eq(const T& a, const T& b) {
  if constexpr (std::is_same<T, at::Half>::value) {
    return static_cast<float>(a) == static_cast<float>(b);
  } else {
    return a == b;
  }
}

// Fixed constants common to both dynamic and static template versions:
static constexpr int WARP_SIZE = 32;
static constexpr int WARPS_PER_CTA = 6;
static constexpr int MAX_VPT =
    32;  // maximum VPT we support, > params.VPT = num_expert / num_expert_group

// Create an alias for Array using AlignedArray
template <typename T, int N>
using Array = AlignedArray<T, N>;
// QQ: NOTE expression must have a constant value, this has to be > params.VPT
template <typename T>
using AccessType = AlignedArray<T, MAX_VPT>;

template <typename T, typename Params>
__device__ void moe_fused_gate_impl(void* input, void* bias, float* output_ptr,
                                    int32_t* indices_ptr, int64_t num_rows,
                                    int64_t topk_group, int64_t topk,
                                    int64_t num_fused_shared_experts,
                                    double routed_scaling_factor,
                                    Params params) {
  int tidx = threadIdx.x;
  int64_t thread_row = blockIdx.x * params.ROWS_PER_CTA +
                       threadIdx.y * params.ROWS_PER_WARP +
                       tidx / params.THREADS_PER_ROW;
  if (thread_row >= num_rows) {
    return;
  }

  // Calculate topk_excluding_share_expert_fusion from topk
  int64_t topk_excluding_share_expert_fusion =
      topk - (num_fused_shared_experts > 0 ? 1 : 0);

  // Cast pointers to type T:
  auto* input_ptr = reinterpret_cast<T*>(input);
  auto* bias_ptr = reinterpret_cast<T*>(bias);
  auto* thread_row_ptr = input_ptr + thread_row * params.NUM_EXPERTS;

  int thread_group_idx = tidx % params.THREADS_PER_ROW;
  int first_elt_read_by_thread = thread_group_idx * params.VPT;

  // Create local arrays for the row chunk and bias chunk and then reinterpret
  // the address of row_chunk as a pointer to AccessType.
  T* thread_read_ptr = thread_row_ptr + first_elt_read_by_thread;
  Array<T, MAX_VPT> row_chunk;
  AccessType<T> const* vec_thread_read_ptr =
      reinterpret_cast<AccessType<T> const*>(thread_read_ptr);

  T* bias_thread_read_ptr = bias_ptr + first_elt_read_by_thread;
  Array<T, MAX_VPT> bias_chunk;
  AccessType<T> const* vec_bias_thread_read_ptr =
      reinterpret_cast<AccessType<T> const*>(bias_thread_read_ptr);

// QQ NOTE: doing the follow will be slower than loop assign and more
// importantly have misaligned address issue when params.VPT < 8 and mismatch
// with MAX_VPT AccessType<T>* row_chunk_vec_ptr =
// reinterpret_cast<AccessType<T>*>(&row_chunk); row_chunk_vec_ptr[0] =
// vec_thread_read_ptr[0];
#pragma unroll
  for (int ii = 0; ii < params.VPT; ++ii) {
    row_chunk[ii] = vec_thread_read_ptr[0][ii];
    bias_chunk[ii] = vec_bias_thread_read_ptr[0][ii];
  }

  __syncthreads();

////////////////////// Sigmoid //////////////////////
#pragma unroll
  for (int ii = 0; ii < params.VPT; ++ii) {
    row_chunk[ii] = static_cast<T>(1.0f / (1.0f + expf(-float(row_chunk[ii]))));
  }
  __syncthreads();

////////////////////// Add Bias //////////////////////
#pragma unroll
  for (int ii = 0; ii < params.VPT; ++ii) {
    bias_chunk[ii] = row_chunk[ii] + bias_chunk[ii];
  }

////////////////////// Exclude Groups //////////////////////
#pragma unroll
  for (int k_idx = 0; k_idx < params.THREADS_PER_ROW - topk_group;
       ++k_idx) {  // QQ NOTE Here params.THREADS_PER_ROW = num_expert_group
    int expert = first_elt_read_by_thread;
    // local argmax
    T max_val = static_cast<T>(-FLT_MAX);
    T max_val_second = static_cast<T>(-FLT_MAX);
#pragma unroll
    for (int ii = 0; ii < params.VPT; ++ii) {
      T val = bias_chunk[ii];

      if (cmp_gt(val, max_val)) {
        max_val_second = max_val;
        max_val = val;
      } else if (cmp_gt(val, max_val_second)) {
        max_val_second = val;
      }
    }

    // QQ NOTE: currently fixed to pick top2 sigmoid weight value in each expert
    // group and sum them as the group weight to select expert groups
    T max_sum = max_val + max_val_second;

// argmin reduce
#pragma unroll
    for (int mask = params.THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
      T other_max_sum = static_cast<T>(
          __shfl_xor_sync(0xFFFFFFFF, static_cast<float>(max_sum), mask,
                          params.THREADS_PER_ROW));
      int other_expert =
          __shfl_xor_sync(0xFFFFFFFF, expert, mask, params.THREADS_PER_ROW);

      // higher indices win
      if (cmp_gt(max_sum, other_max_sum) ||
          (cmp_eq(other_max_sum, max_sum) && other_expert > expert)) {
        max_sum = other_max_sum;
        expert = other_expert;
      }
    }

    // clear the max value in the thread
    if (k_idx < params.THREADS_PER_ROW - topk_group) {
      int const thread_to_clear_in_group = expert / params.VPT;

      if (thread_group_idx == thread_to_clear_in_group) {
#pragma unroll
        for (int ii = 0; ii < params.VPT; ++ii) {
          bias_chunk[ii] = static_cast<T>(FLT_MAX);
        }
      }
    }
  }

  __syncthreads();

  ////////////////////// Topk //////////////////////
  float output_sum = 0.0f;
  for (int k_idx = 0; k_idx < topk_excluding_share_expert_fusion; ++k_idx) {
    // local argmax
    T max_val = bias_chunk[0];
    int expert = first_elt_read_by_thread;

    if (!cmp_eq(max_val, static_cast<T>(FLT_MAX))) {
#pragma unroll
      for (int ii = 1; ii < params.VPT; ++ii) {
        T val = bias_chunk[ii];
        if (cmp_gt(val, max_val)) {
          max_val = val;
          expert = first_elt_read_by_thread + ii;
        }
      }
    } else {
      max_val = static_cast<T>(-FLT_MAX);
    }

    // argmax reduce
#pragma unroll
    for (int mask = params.THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
      T other_max = static_cast<T>(
          __shfl_xor_sync(0xFFFFFFFF, static_cast<float>(max_val), mask,
                          params.THREADS_PER_ROW));
      int other_expert =
          __shfl_xor_sync(0xFFFFFFFF, expert, mask, params.THREADS_PER_ROW);

      // lower indices to win
      if (cmp_gt(other_max, max_val) ||
          (cmp_eq(other_max, max_val) && other_expert < expert)) {
        max_val = other_max;
        expert = other_expert;
      }
    }

    int thread_to_clear_in_group = expert / params.VPT;
    int64_t idx = topk * thread_row + k_idx;

    if (thread_group_idx == thread_to_clear_in_group) {
      int expert_to_clear_in_thread = expert % params.VPT;

      // clear the max value in the thread
      bias_chunk[expert_to_clear_in_thread] = static_cast<T>(-FLT_MAX);

      // store output
      output_ptr[idx] =
          static_cast<float>(row_chunk[expert_to_clear_in_thread]);
      indices_ptr[idx] = static_cast<int32_t>(expert);
    }

    // accumulate sum for all elements
    if (thread_group_idx == 0) {
      output_sum += output_ptr[idx];
    }

    __syncthreads();
  }

  if (thread_group_idx == 0 && num_fused_shared_experts > 0) {
    int64_t last_idx = topk * thread_row + topk_excluding_share_expert_fusion;

    // Use round-robin to select expert
    int64_t expert_offset = thread_row % num_fused_shared_experts;

    indices_ptr[last_idx] =
        static_cast<int32_t>(params.NUM_EXPERTS + expert_offset);

    // Set the weight to the sum of all weights divided by routed_scaling_factor
    output_ptr[last_idx] = output_sum / routed_scaling_factor;

    // if (num_fused_shared_experts > 1) {
    //   for (int i = 1; i < num_fused_shared_experts; ++i) {
    //     ++last_idx;
    //     ++expert_offset;
    //     indices_ptr[last_idx] = static_cast<int32_t>(params.NUM_EXPERTS +
    //     expert_offset);
    //     // Set the weight to the sum of all weights divided by
    //     routed_scaling_factor output_ptr[last_idx] = output_sum /
    //     routed_scaling_factor;
    //   }
    // }
  }
  __syncthreads();

  ////////////////////// Rescale Output //////////////////////
  if (thread_group_idx == 0) {
#pragma unroll
    for (int ii = 0; ii < topk; ++ii) {
      int64_t const idx = topk * thread_row + ii;
      output_ptr[idx] = output_ptr[idx] / output_sum;
    }
  }
}

//------------------------------------------------------------------------------
// Templated Kernel Version (using compile-time constants)
//------------------------------------------------------------------------------
template <int VPT_, int NUM_EXPERTS_, int THREADS_PER_ROW_, int ROWS_PER_WARP_,
          int ROWS_PER_CTA_, int WARPS_PER_CTA_>
struct KernelParams {
  static constexpr int VPT = VPT_;
  static constexpr int NUM_EXPERTS = NUM_EXPERTS_;
  static constexpr int THREADS_PER_ROW = THREADS_PER_ROW_;
  static constexpr int ROWS_PER_WARP = ROWS_PER_WARP_;
  static constexpr int ROWS_PER_CTA = ROWS_PER_CTA_;
  static constexpr int WARPS_PER_CTA = WARPS_PER_CTA_;
};

template <typename T, int VPT, int NUM_EXPERTS, int THREADS_PER_ROW,
          int ROWS_PER_WARP, int ROWS_PER_CTA, int WARPS_PER_CTA>
__global__ void moe_fused_gate_kernel(void* input, void* bias,
                                      float* output_ptr, int32_t* indices_ptr,
                                      int64_t num_rows, int64_t topk_group,
                                      int64_t topk,
                                      int64_t num_fused_shared_experts,
                                      double routed_scaling_factor) {
  KernelParams<VPT, NUM_EXPERTS, THREADS_PER_ROW, ROWS_PER_WARP, ROWS_PER_CTA,
               WARPS_PER_CTA>
      params;
  moe_fused_gate_impl<T>(input, bias, output_ptr, indices_ptr, num_rows,
                         topk_group, topk, num_fused_shared_experts,
                         routed_scaling_factor, params);
}

// Macro to compute compile-time constants and launch the kernel.
#define LAUNCH_MOE_GATE_CONFIG(T, EXPERTS, EXPERT_GROUP)                    \
  do {                                                                      \
    constexpr int VPT = (EXPERTS) / (EXPERT_GROUP);                         \
    /* If EXPERT_GROUP > WARP_SIZE, fall back to 1 row per warp */          \
    constexpr int ROWS_PER_WARP =                                           \
        ((EXPERT_GROUP) <= WARP_SIZE) ? (WARP_SIZE / (EXPERT_GROUP)) : 1;   \
    constexpr int ROWS_PER_CTA = WARPS_PER_CTA * ROWS_PER_WARP;             \
    moe_fused_gate_kernel<T, VPT, (EXPERTS), (EXPERT_GROUP), ROWS_PER_WARP, \
                          ROWS_PER_CTA, WARPS_PER_CTA>                      \
        <<<num_blocks, block_dim, 0, stream>>>(                             \
            input.data_ptr(), bias.data_ptr(), output.data_ptr<float>(),    \
            indices.data_ptr<int32_t>(), num_rows, topk_group, topk,        \
            num_fused_shared_experts, routed_scaling_factor);               \
    dispatched = true;                                                      \
  } while (0)

//------------------------------------------------------------------------------
// Dynamic Kernel Version (parameters computed at runtime)
//------------------------------------------------------------------------------
struct KernelParamsDynamic {
  int VPT;
  int NUM_EXPERTS;
  int THREADS_PER_ROW;
  int ROWS_PER_WARP;
  int ROWS_PER_CTA;
  int WARPS_PER_CTA;
};

template <typename T>
__global__ void moe_fused_gate_kernel_dynamic(
    void* input, void* bias, float* output_ptr, int32_t* indices_ptr,
    int64_t num_rows, int64_t num_experts, int64_t num_expert_group,
    int64_t topk_group, int64_t topk, int64_t num_fused_shared_experts,
    double routed_scaling_factor) {
  KernelParamsDynamic params;
  params.NUM_EXPERTS = num_experts;  // e.g, for deepseek v3, this is 256
  params.VPT = num_experts /
               num_expert_group;  // e.g., for deepseek v3, this is 256 / 8 = 32
  params.THREADS_PER_ROW =
      num_expert_group;  // fixed as num_expert_group, e.g., for deepseek v3,
                         // this is 8
  params.WARPS_PER_CTA = WARPS_PER_CTA;  // fixed as 6
  params.ROWS_PER_WARP = std::max<int64_t>(
      1, WARP_SIZE / num_expert_group);  // WARP_SIZE is fixed as 32
  params.ROWS_PER_CTA = params.WARPS_PER_CTA * params.ROWS_PER_WARP;

  moe_fused_gate_impl<T>(input, bias, output_ptr, indices_ptr, num_rows,
                         topk_group, topk, num_fused_shared_experts,
                         routed_scaling_factor, params);
}

//------------------------------------------------------------------------------
// Host Launcher Function
//------------------------------------------------------------------------------
std::vector<at::Tensor> moe_fused_gate(at::Tensor& input, at::Tensor& bias,
                                       int64_t num_expert_group,
                                       int64_t topk_group, int64_t topk,
                                       int64_t num_fused_shared_experts,
                                       double routed_scaling_factor) {
  int64_t num_rows = input.size(0);
  int32_t num_experts = input.size(1);
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
  auto output = torch::empty({num_rows, topk}, options);
  auto indices = torch::empty({num_rows, topk}, options.dtype(torch::kInt32));

  // Compute grid dimensions based on runtime value for num_expert_group.
  int64_t rows_per_warp = std::max<int64_t>(1, WARP_SIZE / num_expert_group);
  int64_t num_warps = (num_rows + rows_per_warp - 1) / rows_per_warp;
  int64_t num_blocks = (num_warps + WARPS_PER_CTA - 1) / WARPS_PER_CTA;
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 block_dim(WARP_SIZE, WARPS_PER_CTA);

  // Check 1: Ensure that num_experts is a power of 2.
  TORCH_CHECK((num_experts & (num_experts - 1)) == 0,
              "num_experts must be a power of 2, but got ", num_experts);

  // Check 2: Ensure that num_experts is divisible by num_expert_group. (this
  // also means num_expert_group is power of 2)
  TORCH_CHECK(num_experts % num_expert_group == 0,
              "num_experts must be divisible by num_expert_group, but got ",
              num_experts, " / ", num_expert_group);

  int computed_vpt = num_experts / num_expert_group;
  // Check 3: Ensure that num_experts/num_expert_group does not exceed
  // MAX_VPT=32. Maximum VPT indicate max value per threads we can process.
  TORCH_CHECK(computed_vpt <= MAX_VPT,
              "Per group experts: num_experts / num_expert_group = (",
              computed_vpt, ") exceeds the maximum supported (", MAX_VPT, ")");

  // Dispatch to templated kernel for known compile-time configurations.
  // We currently only support for:
  //   Case 1: 256 experts, with 8 or 16 groups.
  //   Case 2: 128 experts, with 4 or 8 groups.
  //   Case 3: other cases, require 8 <= num_experts / num_expert_group <= 32
  bool dispatched = false;
  switch (num_experts) {
    case 256:
      if (num_expert_group == 8) {
        // This is deepseek v3 case. Here VPT = 256/8 = 32, ROWS_PER_WARP = 32/8
        // = 4, ROWS_PER_CTA = 6 * 4 = 24.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 256, 8);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 256, 8);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 256, 8);
        } else if (num_expert_group == 16) {
          // Here VPT = 256/16 = 16, ROWS_PER_WARP = 32/16 = 2, ROWS_PER_CTA = 6
          // * 2 = 12.
          if (input.scalar_type() == at::kBFloat16) {
            LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 256, 16);
          } else if (input.scalar_type() == at::kHalf) {
            LAUNCH_MOE_GATE_CONFIG(float16_t, 256, 16);
          } else if (input.scalar_type() == at::kFloat) {
            LAUNCH_MOE_GATE_CONFIG(float32_t, 256, 16);
          }
        }
      }
      break;
    case 128:
      if (num_expert_group == 4) {
        // VPT = 128/4 = 32, ROWS_PER_WARP = 32/16 = 2, ROWS_PER_CTA = 6 * 2
        // = 12.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 128, 4);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 128, 4);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 128, 4);
        } else if (num_expert_group == 8) {
          // VPT = 128/8 = 16, ROWS_PER_WARP = 32/8 = 4, ROWS_PER_CTA = 6 * 4
          // = 24.
          if (input.scalar_type() == at::kBFloat16) {
            LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 128, 8);
          } else if (input.scalar_type() == at::kHalf) {
            LAUNCH_MOE_GATE_CONFIG(float16_t, 128, 8);
          } else if (input.scalar_type() == at::kFloat) {
            LAUNCH_MOE_GATE_CONFIG(float32_t, 128, 8);
          }
        }
      }
      break;
    default:
      break;
  }
  if (!dispatched) {
    // Fallback to the dynamic kernel if none of the supported combinations
    // match. currently only support num_experts / num_expert_group <= 32 for
    // dynamic kernels
    if (input.scalar_type() == at::kBFloat16) {
      moe_fused_gate_kernel_dynamic<bfloat16_t>
          <<<num_blocks, block_dim, 0, stream>>>(
              input.data_ptr(), bias.data_ptr(), output.data_ptr<float>(),
              indices.data_ptr<int32_t>(), num_rows, num_experts,
              num_expert_group, topk_group, topk, num_fused_shared_experts,
              routed_scaling_factor);
    } else if (input.scalar_type() == at::kHalf) {
      moe_fused_gate_kernel_dynamic<float16_t>
          <<<num_blocks, block_dim, 0, stream>>>(
              input.data_ptr(), bias.data_ptr(), output.data_ptr<float>(),
              indices.data_ptr<int32_t>(), num_rows, num_experts,
              num_expert_group, topk_group, topk, num_fused_shared_experts,
              routed_scaling_factor);
    } else if (input.scalar_type() == at::kFloat) {
      moe_fused_gate_kernel_dynamic<float32_t>
          <<<num_blocks, block_dim, 0, stream>>>(
              input.data_ptr(), bias.data_ptr(), output.data_ptr<float>(),
              indices.data_ptr<int32_t>(), num_rows, num_experts,
              num_expert_group, topk_group, topk, num_fused_shared_experts,
              routed_scaling_factor);
    } else {
      TORCH_CHECK(false, "Unsupported data type for moe_fused_gate");
    }
  }
  return {output, indices};
}
