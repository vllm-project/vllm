#include <c10/core/ScalarType.h>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include "permute_unpermute_kernels/moe_permute_unpermute_kernel.h"
#include "permute_unpermute_kernels/dispatch.h"
#include "core/registration.h"

// moe_permute kernels require at least CUDA 12.0
#if defined(CUDA_VERSION) && (CUDA_VERSION >= 12000)

void moe_permute(
    const torch::Tensor& input,                      // [n_token, hidden]
    const torch::Tensor& topk_weights,               //[n_token, topk]
    torch::Tensor& topk_ids,                         // [n_token, topk]
    const torch::Tensor& token_expert_indicies,      // [n_token, topk]
    const std::optional<torch::Tensor>& expert_map,  // [n_expert]
    int64_t n_expert, int64_t n_local_expert, int64_t topk,
    const std::optional<int64_t>& align_block_size,
    torch::Tensor&
        permuted_input,  // [topk * n_token/align_block_size_m, hidden]
    torch::Tensor& expert_first_token_offset,  // [n_local_expert + 1]
    torch::Tensor& src_row_id2dst_row_id_map,  // [n_token, topk]
    torch::Tensor& m_indices) {                // [align_expand_m]
  TORCH_CHECK(topk_weights.scalar_type() == at::ScalarType::Float,
              "topk_weights must be float32");
  TORCH_CHECK(expert_first_token_offset.scalar_type() == at::ScalarType::Long,
              "expert_first_token_offset must be int64");
  TORCH_CHECK(topk_ids.scalar_type() == at::ScalarType::Int,
              "topk_ids must be int32");
  TORCH_CHECK(token_expert_indicies.scalar_type() == at::ScalarType::Int,
              "token_expert_indicies must be int32");
  TORCH_CHECK(src_row_id2dst_row_id_map.scalar_type() == at::ScalarType::Int,
              "src_row_id2dst_row_id_map must be int32");
  TORCH_CHECK(expert_first_token_offset.size(0) == n_local_expert + 1,
              "expert_first_token_offset shape != n_local_expert+1")
  TORCH_CHECK(
      src_row_id2dst_row_id_map.sizes() == token_expert_indicies.sizes(),
      "token_expert_indicies shape must be same as src_row_id2dst_row_id_map");
  auto n_token = input.sizes()[0];
  auto n_hidden = input.sizes()[1];
  auto align_block_size_value =
      align_block_size.has_value() ? align_block_size.value() : -1;
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  const long sorter_size =
      CubKeyValueSorter::getWorkspaceSize(n_token * topk, n_expert);
  auto sort_workspace = torch::empty(
      {sorter_size},
      torch::dtype(torch::kInt8).device(torch::kCUDA).requires_grad(false));
  auto permuted_experts_id = torch::empty_like(topk_ids);
  auto dst_row_id2src_row_id_map = torch::empty_like(src_row_id2dst_row_id_map);
  auto align_expert_first_token_offset =
      torch::zeros_like(expert_first_token_offset);

  CubKeyValueSorter sorter{};
  int64_t* valid_num_ptr = nullptr;
  // pre-process kernel for expert-parallelism:
  // no local expert id plus "n_expert" offset for priority to local expert
  // map local expert id [n, .., n+n_local_expert-1] to [0, n_local_expert -1]
  // For example, 4 expert with ep_size=2. ep_rank=1 owns global expert id
  // [2,3] with expert_map[-1, -1, 0, 1], preprocess_topk_id  process topk_ids
  // and map global expert id [2, 3] to local_expert id [0, 1] and map global
  // expert id [0, 1] ( not in ep rank=1)  to [4, 5] by plus n_expert. This map
  // operation is to make local expert high priority in following sort topk_ids
  // and scan local expert_first_token_offset for each ep rank for next group
  // gemm.
  if (expert_map.has_value()) {
    const int* expert_map_ptr = get_ptr<int>(expert_map.value());
    valid_num_ptr =
        get_ptr<int64_t>(expert_first_token_offset) + n_local_expert;
    preprocessTopkIdLauncher(get_ptr<int>(topk_ids), n_token * topk,
                             expert_map_ptr, n_expert, stream);
  }
  // expert sort topk expert id and scan expert id get expert_first_token_offset
  sortAndScanExpert(get_ptr<int>(topk_ids), get_ptr<int>(token_expert_indicies),
                    get_ptr<int>(permuted_experts_id),
                    get_ptr<int>(dst_row_id2src_row_id_map),
                    get_ptr<int64_t>(expert_first_token_offset), n_token,
                    n_expert, n_local_expert, topk, sorter,
                    get_ptr<int>(sort_workspace), stream);

  // dispatch expandInputRowsKernelLauncher
  MOE_DISPATCH(input.scalar_type(), [&] {
    expandInputRowsKernelLauncher<scalar_t>(
        get_ptr<scalar_t>(input), get_ptr<scalar_t>(permuted_input),
        get_ptr<float>(topk_weights), get_ptr<int>(permuted_experts_id),
        get_ptr<int>(dst_row_id2src_row_id_map),
        get_ptr<int>(src_row_id2dst_row_id_map),
        get_ptr<int64_t>(expert_first_token_offset), n_token, valid_num_ptr,
        n_hidden, topk, n_local_expert, align_block_size_value, stream);
  });

  // get m_indices and update expert_first_token_offset with align block
  getMIndices(get_ptr<int64_t>(expert_first_token_offset),
              get_ptr<int64_t>(align_expert_first_token_offset),
              get_ptr<int>(m_indices), n_local_expert, align_block_size_value,
              stream);
  if (align_block_size.has_value()) {
    // update align_expert_first_token_offset
    expert_first_token_offset.copy_(align_expert_first_token_offset);
  }
}

void moe_unpermute(
    const torch::Tensor& permuted_hidden_states,     // [n_token * topk, hidden]
    const torch::Tensor& topk_weights,               //[n_token, topk]
    const torch::Tensor& topk_ids,                   // [n_token, topk]
    const torch::Tensor& src_row_id2dst_row_id_map,  // [n_token, topk]
    const torch::Tensor& expert_first_token_offset,  // [n_local_expert+1]
    int64_t n_expert, int64_t n_local_expert, int64_t topk,
    torch::Tensor& hidden_states  // [n_token, hidden]
) {
  TORCH_CHECK(src_row_id2dst_row_id_map.sizes() == topk_ids.sizes(),
              "topk_ids shape must be same as src_row_id2dst_row_id_map");
  TORCH_CHECK(topk_ids.scalar_type() == at::ScalarType::Int,
              "topk_ids must be int32");
  TORCH_CHECK(
      permuted_hidden_states.scalar_type() == hidden_states.scalar_type(),
      "topk_ids dtype must be same as src_row_id2dst_row_id_map");
  auto n_token = hidden_states.size(0);
  auto n_hidden = hidden_states.size(1);
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  const int64_t* valid_ptr =
      get_ptr<int64_t>(expert_first_token_offset) + n_local_expert;
  MOE_DISPATCH(hidden_states.scalar_type(), [&] {
    finalizeMoeRoutingKernelLauncher<scalar_t, scalar_t>(
        get_ptr<scalar_t>(permuted_hidden_states),
        get_ptr<scalar_t>(hidden_states), get_ptr<float>(topk_weights),
        get_ptr<int>(src_row_id2dst_row_id_map), get_ptr<int>(topk_ids),
        n_token, n_hidden, topk, valid_ptr, stream);
  });
}

#else

void moe_permute(const torch::Tensor& input, const torch::Tensor& topk_weights,
                 torch::Tensor& topk_ids,
                 const torch::Tensor& token_expert_indicies,
                 const std::optional<torch::Tensor>& expert_map,
                 int64_t n_expert, int64_t n_local_expert, int64_t topk,
                 const std::optional<int64_t>& align_block_size,
                 torch::Tensor& permuted_input,
                 torch::Tensor& expert_first_token_offset,
                 torch::Tensor& src_row_id2dst_row_id_map,
                 torch::Tensor& m_indices) {
  TORCH_CHECK(false, "moe_unpermute is not supported on CUDA < 12.0");
}

void moe_unpermute(const torch::Tensor& input,
                   const torch::Tensor& topk_weights, torch::Tensor& topk_ids,
                   const torch::Tensor& token_expert_indicies,
                   const std::optional<torch::Tensor>& expert_map,
                   int64_t n_expert, int64_t n_local_expert, int64_t topk,
                   const std::optional<int64_t>& align_block_size,
                   torch::Tensor& permuted_input,
                   torch::Tensor& expert_first_token_offset,
                   torch::Tensor& src_row_id2dst_row_id_map,
                   torch::Tensor& m_indices) {
  TORCH_CHECK(false, "moe_unpermute is not supported on CUDA < 12.0");
}

#endif

bool moe_permute_unpermute_supported() {
#if defined(CUDA_VERSION) && (CUDA_VERSION >= 12000)
  return true;
#else
  return false;
#endif
}

TORCH_LIBRARY_IMPL_EXPAND(TORCH_EXTENSION_NAME, CUDA, m) {
  m.impl("moe_permute", &moe_permute);
  m.impl("moe_unpermute", &moe_unpermute);
}
