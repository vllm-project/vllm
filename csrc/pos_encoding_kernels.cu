#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  scalar_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int query_stride,
  const int key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

} // namespace vllm

void rotary_embedding(
  torch::Tensor& positions,         // [batch_size, seq_len] or [num_tokens]
  torch::Tensor& query,             // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
  torch::Tensor& key,               // [batch_size, seq_len, num_kv_heads * head_size] or [num_tokens, num_kv_heads * head_size]
  int head_size,
  torch::Tensor& cos_sin_cache,     // [max_position, rot_dim]
  bool is_neox) {
  int64_t num_tokens = query.numel() / query.size(-1);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(-1) / head_size;
  int num_kv_heads = key.size(-1) / head_size;
  int query_stride = query.stride(-2);
  int key_stride = key.stride(-2);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    query.scalar_type(),
    "rotary_embedding",
    [&] {
      if (is_neox) {
        vllm::rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      } else {
        vllm::rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      }
    });
}
