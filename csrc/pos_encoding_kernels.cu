#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "dispatch_utils.h"

namespace vllm {

template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [num_tokens]
  scalar_t* __restrict__ query,                 // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int query_stride,
  const int key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

} // namespace vllm

void rotary_embedding(
  torch::Tensor& positions,         // [num_tokens]
  torch::Tensor& query,             // [num_tokens, num_heads * head_size]
  torch::Tensor& key,               // [num_tokens, num_kv_heads * head_size]
  int head_size,
  torch::Tensor& cos_sin_cache,     // [max_position, rot_dim]
  bool is_neox) {
  int num_tokens = query.size(0);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(1) / head_size;
  int num_kv_heads = key.size(1) / head_size;
  int query_stride = query.stride(0);
  int key_stride = key.stride(0);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    query.scalar_type(),
    "rotary_embedding",
    [&] {
      if (is_neox) {
        vllm::rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      } else {
        vllm::rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      }
    });
}
