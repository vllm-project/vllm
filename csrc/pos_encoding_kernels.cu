#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_token_rotary_embedding(
    scalar_t* __restrict__ arr, const scalar_t* __restrict__ cos_ptr,
    const scalar_t* __restrict__ sin_ptr, int rot_offset, int embed_dim) {
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
    scalar_t* __restrict__ query,  // [batch_size, seq_len, num_heads,
                                   // head_size] or [num_tokens, num_heads,
                                   // head_size]
    scalar_t* __restrict__ key,    // [batch_size, seq_len, num_kv_heads,
                                   // head_size] or [num_tokens, num_kv_heads,
                                   // head_size]
    const scalar_t* cache_ptr, const int head_size, const int num_heads,
    const int num_kv_heads, const int rot_dim, const int token_idx,
    const int64_t query_stride, const int64_t key_stride) {
  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_token_rotary_embedding<scalar_t, IS_NEOX>(
        query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_token_rotary_embedding<scalar_t, IS_NEOX>(
        key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }
}

template <typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
    const int64_t* __restrict__ positions,  // [batch_size, seq_len] or
                                            // [num_tokens]
    scalar_t* __restrict__ query,           // [batch_size, seq_len, num_heads,
                                   // head_size] or [num_tokens, num_heads,
                                   // head_size]
    scalar_t* __restrict__ key,  // [batch_size, seq_len, num_kv_heads,
                                 // head_size] or [num_tokens, num_kv_heads,
                                 // head_size]
    const scalar_t* __restrict__ cos_sin_cache,  // [max_position, 2, rot_dim //
                                                 // 2]
    const int rot_dim, const int64_t query_stride, const int64_t key_stride,
    const int num_heads, const int num_kv_heads, const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  apply_rotary_embedding<scalar_t, IS_NEOX>(
      query, key, cache_ptr, head_size, num_heads, num_kv_heads, rot_dim,
      token_idx, query_stride, key_stride);
}

template <typename scalar_t, bool IS_NEOX>
__global__ void batched_rotary_embedding_kernel(
    const int64_t* __restrict__ positions,  // [batch_size, seq_len] or
                                            // [num_tokens]
    scalar_t* __restrict__ query,           // [batch_size, seq_len, num_heads,
                                   // head_size] or [num_tokens, num_heads,
                                   // head_size]
    scalar_t* __restrict__ key,  // [batch_size, seq_len, num_kv_heads,
                                 // head_size] or [num_tokens, num_kv_heads,
                                 // head_size]
    const scalar_t* __restrict__ cos_sin_cache,  // [max_position, 2, rot_dim //
                                                 // 2]
    const int64_t* __restrict__ cos_sin_cache_offsets,  // [batch_size, seq_len]
                                                        // or [num_tokens]
    const int rot_dim, const int64_t query_stride, const int64_t key_stride,
    const int num_heads, const int num_kv_heads, const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  int64_t cos_sin_cache_offset = cos_sin_cache_offsets[token_idx];
  const scalar_t* cache_ptr =
      cos_sin_cache + (cos_sin_cache_offset + pos) * rot_dim;

  apply_rotary_embedding<scalar_t, IS_NEOX>(
      query, key, cache_ptr, head_size, num_heads, num_kv_heads, rot_dim,
      token_idx, query_stride, key_stride);
}

}  // namespace vllm

void rotary_embedding(
    torch::Tensor& positions,  // [batch_size, seq_len] or [num_tokens]
    torch::Tensor& query,  // [batch_size, seq_len, num_heads * head_size] or
                           // [num_tokens, num_heads * head_size] or
                           // [batch_size, seq_len, num_heads, head_size] or
                           // [num_tokens, num_heads, head_size]
    torch::Tensor& key,    // [batch_size, seq_len, num_kv_heads * head_size] or
                           // [num_tokens, num_kv_heads * head_size] or
                           // [batch_size, seq_len, num_heads, head_size] or
                           // [num_tokens, num_heads, head_size]
    int64_t head_size,
    torch::Tensor& cos_sin_cache,  // [max_position, rot_dim]
    bool is_neox) {
  // num_tokens = batch_size * seq_len
  int64_t num_tokens = positions.numel();
  int positions_ndim = positions.dim();

  // Make sure num_tokens dim is consistent across positions, query, and key.
  TORCH_CHECK(
      positions_ndim == 1 || positions_ndim == 2,
      "positions must have shape [num_tokens] or [batch_size, seq_len]");
  if (positions_ndim == 1) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) && key.size(0) == positions.size(0),
        "query, key and positions must have the same number of tokens");
  }
  if (positions_ndim == 2) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) &&
            key.size(0) == positions.size(0) &&
            query.size(1) == positions.size(1) &&
            key.size(1) == positions.size(1),
        "query, key and positions must have the same batch_size and seq_len");
  }

  // Make sure head_size is valid for query and key
  // hidden_size = num_heads * head_size
  int query_hidden_size = query.numel() / num_tokens;
  int key_hidden_size = key.numel() / num_tokens;
  TORCH_CHECK(query_hidden_size % head_size == 0);
  TORCH_CHECK(key_hidden_size % head_size == 0);

  // Make sure query and key have consistent number of heads
  int num_heads = query_hidden_size / head_size;
  int num_kv_heads = key_hidden_size / head_size;
  TORCH_CHECK(num_heads % num_kv_heads == 0);

  int rot_dim = cos_sin_cache.size(1);
  int seq_dim_idx = positions_ndim - 1;
  int64_t query_stride = query.stride(seq_dim_idx);
  int64_t key_stride = key.stride(seq_dim_idx);

  dim3 grid(num_tokens);
  dim3 block(std::min<int64_t>(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding", [&] {
    if (is_neox) {
      vllm::rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(), cos_sin_cache.data_ptr<scalar_t>(), rot_dim,
          query_stride, key_stride, num_heads, num_kv_heads, head_size);
    } else {
      vllm::rotary_embedding_kernel<scalar_t, false>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
              key.data_ptr<scalar_t>(), cos_sin_cache.data_ptr<scalar_t>(),
              rot_dim, query_stride, key_stride, num_heads, num_kv_heads,
              head_size);
    }
  });
}

/*
Batched version of rotary embedding, pack multiple LoRAs together
and process in batched manner.
*/
void batched_rotary_embedding(
    torch::Tensor& positions,  // [batch_size, seq_len] or [num_tokens]
    torch::Tensor& query,  // [batch_size, seq_len, num_heads * head_size] or
                           // [num_tokens, num_heads * head_size] or
                           // [batch_size, seq_len, num_heads, head_size] or
                           // [num_tokens, num_heads, head_size]
    torch::Tensor& key,    // [batch_size, seq_len, num_kv_heads * head_size] or
                           // [num_tokens, num_kv_heads * head_size] or
                           // [batch_size, seq_len, num_heads, head_size] or
                           // [num_tokens, num_heads, head_size]
    int64_t head_size,
    torch::Tensor& cos_sin_cache,  // [max_position, rot_dim]
    bool is_neox, int64_t rot_dim,
    torch::Tensor& cos_sin_cache_offsets  // [num_tokens] or [batch_size]
) {
  // num_tokens = batch_size * seq_len
  int64_t num_tokens = cos_sin_cache_offsets.size(0);
  TORCH_CHECK(
      positions.size(0) == num_tokens || positions.numel() == num_tokens,
      "positions must have the same num_tokens or batch_size as "
      "cos_sin_cache_offsets");

  int positions_ndim = positions.dim();
  // Make sure num_tokens dim is consistent across positions, query, and key.
  TORCH_CHECK(
      positions_ndim == 1 || positions_ndim == 2,
      "positions must have shape [num_tokens] or [batch_size, seq_len]");
  if (positions_ndim == 1) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) && key.size(0) == positions.size(0),
        "query, key and positions must have the same number of tokens");
  }
  if (positions_ndim == 2) {
    TORCH_CHECK(
        query.size(0) == positions.size(0) &&
            key.size(0) == positions.size(0) &&
            query.size(1) == positions.size(1) &&
            key.size(1) == positions.size(1),
        "query, key and positions must have the same batch_size and seq_len");
  }

  // Make sure head_size is valid for query and key
  int query_hidden_size = query.numel() / num_tokens;
  int key_hidden_size = key.numel() / num_tokens;
  TORCH_CHECK(query_hidden_size % head_size == 0);
  TORCH_CHECK(key_hidden_size % head_size == 0);

  // Make sure query and key have concistent number of heads
  int num_heads = query_hidden_size / head_size;
  int num_kv_heads = key_hidden_size / head_size;
  TORCH_CHECK(num_heads % num_kv_heads == 0);

  int seq_dim_idx = positions_ndim - 1;
  int64_t query_stride = query.stride(seq_dim_idx);
  int64_t key_stride = key.stride(seq_dim_idx);

  dim3 grid(num_tokens);
  dim3 block(std::min<int64_t>(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding", [&] {
    if (is_neox) {
      vllm::batched_rotary_embedding_kernel<scalar_t, true>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
              key.data_ptr<scalar_t>(), cos_sin_cache.data_ptr<scalar_t>(),
              cos_sin_cache_offsets.data_ptr<int64_t>(), rot_dim, query_stride,
              key_stride, num_heads, num_kv_heads, head_size);
    } else {
      vllm::batched_rotary_embedding_kernel<scalar_t, false>
          <<<grid, block, 0, stream>>>(
              positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
              key.data_ptr<scalar_t>(), cos_sin_cache.data_ptr<scalar_t>(),
              cos_sin_cache_offsets.data_ptr<int64_t>(), rot_dim, query_stride,
              key_stride, num_heads, num_kv_heads, head_size);
    }
  });
}
