#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

namespace cacheflow {

template<typename scalar_t>
__global__ void rotary_embedding_neox_kernel(
  scalar_t* __restrict__ out_query,             // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ out_key,               // [num_tokens, num_heads, head_size]
  const int64_t* __restrict__ positions,        // [num_tokens]
  const scalar_t* __restrict__ query,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ key,             // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, head_size // 2]
  const int num_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * head_size;

  const int embed_dim = head_size / 2;
  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int idx = token_idx * n + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int token_head = token_idx * n + head_idx * head_size;

    const bool is_first_half = head_offset < embed_dim;
    const int rot_offset = head_offset % embed_dim;
    const int x_index = rot_offset;
    const int y_index = embed_dim + rot_offset;

    const scalar_t cos = __ldg(cache_ptr + x_index);
    const scalar_t sin = __ldg(cache_ptr + y_index);

    const scalar_t q_x = __ldg(query + token_head + x_index);
    const scalar_t q_y = __ldg(query + token_head + y_index);
    const scalar_t q_cos = is_first_half ? q_x : q_y;
    const scalar_t q_sin = is_first_half ? -q_y : q_x;
    out_query[idx] = q_cos * cos + q_sin * sin;

    const scalar_t k_x = __ldg(key + token_head + x_index);
    const scalar_t k_y = __ldg(key + token_head + y_index);
    const scalar_t k_cos = is_first_half ? k_x : k_y;
    const scalar_t k_sin = is_first_half ? -k_y : k_x;
    out_key[idx] = k_cos * cos + k_sin * sin;
  }
}

} // namespace cacheflow

void rotary_embedding_neox(
  torch::Tensor& out_query,         // [num_tokens, num_heads * head_size]
  torch::Tensor& out_key,           // [num_tokens, num_heads * head_size]
  torch::Tensor& positions,         // [num_tokens]
  torch::Tensor& query,             // [num_tokens, num_heads * head_size]
  torch::Tensor& key,               // [num_tokens, num_heads * head_size]
  torch::Tensor& cos_sin_cache)     // [max_position, head_size]
{
  int num_tokens = query.size(0);
  int head_size = cos_sin_cache.size(1);
  int num_heads = query.size(1) / head_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    query.scalar_type(),
    "rotary_embedding_neox",
    [&] {
      cacheflow::rotary_embedding_neox_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out_query.data_ptr<scalar_t>(),
        out_key.data_ptr<scalar_t>(),
        positions.data_ptr<int64_t>(),
        query.data_ptr<scalar_t>(),
        key.data_ptr<scalar_t>(),
        cos_sin_cache.data_ptr<scalar_t>(),
        num_heads,
        head_size);
    });
}
