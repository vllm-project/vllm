#include "hip/hip_runtime.h"
// clang-format off
// adapted from https://github.com/Dao-AILab/causal-conv1d/blob/main/csrc/causal_conv1d_fwd.cu
// and https://github.com/Dao-AILab/causal-conv1d/blob/main/csrc/causal_conv1d_update.cu
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "causal_conv1d.h"
#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>

#include "../static_switch.h"

#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")

#define DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(ITYPE, NAME, ...)              \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        using weight_t = at::Half;                                                  \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::BFloat16) {                                 \
        using input_t = at::BFloat16;                                               \
        using weight_t = at::BFloat16;                                              \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }


template<typename input_t, typename weight_t>
void causal_conv1d_fwd_cuda(ConvParamsBase &params, hipStream_t stream);

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream);

// (APC writeback is implemented in Python wrapper for functional equivalence)
void set_conv_params_fwd(ConvParamsBase &params,
                         // sizes
                         const size_t batch,
                         const size_t dim,
                         const size_t seqlen,
                         const size_t width,
                         // device tensors
                         const at::Tensor& x,
                         const at::Tensor& weight,
                         const at::Tensor& out,
                         // optional pointers (can be nullptr)
                         void* bias_ptr,
                         bool silu_activation,
                         int64_t pad_slot_id,
                         void* query_start_loc_ptr,
                         void* cache_indices_ptr,
                         void* has_initial_state_ptr) {

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.batch = batch;
    params.dim = dim;
    params.seqlen = seqlen;
    params.width = width;
    params.pad_slot_id = pad_slot_id;

    params.silu_activation = silu_activation;

    // Set the pointers and strides.
    params.x_ptr = const_cast<void*>(x.const_data_ptr());
    params.weight_ptr = const_cast<void*>(weight.const_data_ptr());
    params.bias_ptr = bias_ptr;
    params.out_ptr = const_cast<void*>(out.const_data_ptr());
    // All stride are in elements, not bytes.
    params.query_start_loc_ptr = query_start_loc_ptr;
    params.cache_indices_ptr = cache_indices_ptr;
    params.has_initial_state_ptr = has_initial_state_ptr;
    const bool varlen = params.query_start_loc_ptr != nullptr;
    params.x_batch_stride = x.stride(varlen ? 1 : 0);
    params.x_c_stride = x.stride(varlen ? 0 : 1);
    params.x_l_stride = x.stride(varlen ? 1 : -1);
    params.weight_c_stride = weight.stride(0);
    params.weight_width_stride = weight.stride(1);
    params.out_batch_stride = out.stride(varlen ? 1 : 0);
    params.out_c_stride = out.stride(varlen ? 0 : 1);
    params.out_l_stride = out.stride(varlen ? 1 : -1);
}





void causal_conv1d_update(const at::Tensor &x,
                     const at::Tensor &conv_state,
                     const at::Tensor &weight,
                     const std::optional<at::Tensor> &bias_,
                     bool silu_activation,
                     const std::optional<at::Tensor> &cache_seqlens_,
                     const std::optional<at::Tensor> &conv_state_indices_,
                     // used to identify padding entries if cache_indices provided
                     // in case of padding, the kernel will return early
                     int64_t pad_slot_id) {
    auto input_type = x.scalar_type();
    auto weight_type = weight.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float || weight_type == at::ScalarType::Half || weight_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == input_type, "weight type must equal to input type, other variations are disabled due to binary size limitations");
    TORCH_CHECK(conv_state.scalar_type() == input_type);

    TORCH_CHECK(x.is_cuda());
    TORCH_CHECK(conv_state.is_cuda());
    TORCH_CHECK(weight.is_cuda());

    const auto sizes = x.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int seqlen = sizes[2];
    const int width = weight.size(-1);
    const int conv_state_len = conv_state.size(2);
    TORCH_CHECK(conv_state_len >= width - 1);

    CHECK_SHAPE(x, batch_size, dim, seqlen);
    CHECK_SHAPE(weight, dim, width);

    TORCH_CHECK(width >= 2 && width <= 4, "causal_conv1d only supports width between 2 and 4");

    if (bias_.has_value()) {
        auto bias = bias_.value();
        TORCH_CHECK(bias.scalar_type() == weight_type);
        TORCH_CHECK(bias.is_cuda());
        TORCH_CHECK(bias.stride(-1) == 1);
        CHECK_SHAPE(bias, dim);
    }

    at::Tensor out = x;

    ConvParamsBase params;
    void* bias_ptr = bias_.has_value() ? bias_.value().data_ptr() : nullptr;
    set_conv_params_fwd(params, batch_size, dim, seqlen, width, x, weight, out,
                        bias_ptr,
                        silu_activation,
                        pad_slot_id,
                        /*query_start_loc_ptr*/ nullptr,
                        /*cache_indices_ptr*/ nullptr,
                        /*has_initial_state_ptr*/ nullptr);
    params.conv_state_ptr = conv_state.data_ptr();
    params.conv_state_len = conv_state_len;
    // All stride are in elements, not bytes.
    params.conv_state_batch_stride = conv_state.stride(0);
    params.conv_state_c_stride = conv_state.stride(1);
    params.conv_state_l_stride = conv_state.stride(2);

    if (cache_seqlens_.has_value()) {
        auto cache_seqlens = cache_seqlens_.value();
        TORCH_CHECK(cache_seqlens.scalar_type() == torch::kInt32);
        TORCH_CHECK(cache_seqlens.is_cuda());
        TORCH_CHECK(cache_seqlens.stride(-1) == 1);
        CHECK_SHAPE(cache_seqlens, batch_size);
        params.cache_seqlens = cache_seqlens.data_ptr<int32_t>();
    } else {
        params.cache_seqlens = nullptr;
    }

    if (conv_state_indices_.has_value()) {
        auto conv_state_indices = conv_state_indices_.value();
        TORCH_CHECK(conv_state_indices.scalar_type() == torch::kInt32);
        TORCH_CHECK(conv_state_indices.is_cuda());
        TORCH_CHECK(conv_state_indices.stride(0) == 1);
        CHECK_SHAPE(conv_state_indices, batch_size);

        int conv_state_entries = conv_state.size(0);
        CHECK_SHAPE(conv_state, conv_state_entries, dim, conv_state_len);

        params.conv_state_indices_ptr = conv_state_indices.data_ptr<int32_t>();
    } else {
        CHECK_SHAPE(conv_state, batch_size, dim, conv_state_len);
        params.conv_state_indices_ptr = nullptr;
    }

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)x.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(x.scalar_type(), "causal_conv1d_update", [&] {
            causal_conv1d_update_cuda<input_t, weight_t>(params, stream);
    });
}


template<int kNThreads_, int kWidth_, typename input_t_, typename weight_t_>
struct Causal_conv1d_update_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
};

template<typename Ktraits, bool kIsCircularBuffer>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_update_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y * kNThreads + tidx;
    if (channel_id >= params.dim) return;

    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + channel_id * params.x_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + channel_id * params.out_c_stride;

    // If params.conv_state_batch_indices is set, then the conv state is gathered from the conv state tensor
    // along the batch axis. Otherwise, the conv state coordinate is the same as the batch id.
    const int conv_state_batch_coord = params.conv_state_indices_ptr == nullptr
        ? batch_id
        : params.conv_state_indices_ptr[batch_id];

    // Skip padding tokens when the selected coord equals pad_slot_id.
    if (conv_state_batch_coord == params.pad_slot_id) {
        #pragma unroll 2
        for (int i = 0; i < params.seqlen; ++i) {
            out[i * params.out_l_stride] = input_t(0.f);
        }
        return;
    }
    input_t *conv_state = reinterpret_cast<input_t *>(params.conv_state_ptr)
        + conv_state_batch_coord * params.conv_state_batch_stride
        + channel_id * params.conv_state_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    int state_len = params.conv_state_len;
    int advance_len = params.seqlen;
    int cache_seqlen = kIsCircularBuffer ? params.cache_seqlens[batch_id] % state_len : 0;
    int update_idx = cache_seqlen - (kWidth - 1);
    update_idx = update_idx < 0 ? update_idx + state_len : update_idx;

    float weight_vals[kWidth] = {0};
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }

    float x_vals[kWidth] = {0};
    if constexpr (!kIsCircularBuffer) {
        #pragma unroll 2
        for (int i = 0; i < state_len - advance_len - (kWidth - 1); ++i) {
            conv_state[i * params.conv_state_l_stride] = conv_state[(i + advance_len) * params.conv_state_l_stride];
        }
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) {
            input_t state_val = conv_state[(state_len - (kWidth - 1) + i) * params.conv_state_l_stride];
            if (i < advance_len + (kWidth - 1) && state_len - advance_len - (kWidth - 1) + i >= 0) {
                conv_state[(state_len - advance_len - (kWidth - 1) + i) * params.conv_state_l_stride] = state_val;
            }
            x_vals[i] = float(state_val);
        }
    } else {
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i, update_idx = update_idx + 1 >= state_len ? update_idx + 1 - state_len : update_idx + 1) {
            input_t state_val = conv_state[update_idx * params.conv_state_l_stride];
            x_vals[i] = float(state_val);
        }
    }
    #pragma unroll 2
    for (int i = 0; i < params.seqlen; ++i) {
        input_t x_val = x[i * params.x_l_stride];
        if constexpr (!kIsCircularBuffer) {
            if (i < advance_len && state_len - advance_len + i >= 0) {
                conv_state[(state_len - advance_len + i) * params.conv_state_l_stride] = x_val;
            }
        } else {
            conv_state[update_idx * params.conv_state_l_stride] = x_val;
            ++update_idx;
            update_idx = update_idx >= state_len ? update_idx - state_len : update_idx;
        }
        x_vals[kWidth - 1] = float(x_val);
        float out_val = bias_val;
        #pragma unroll
        for (int j = 0; j < kWidth; ++j) { out_val += weight_vals[j] * x_vals[j]; }
        if (params.silu_activation) { out_val = out_val / (1 + expf(-out_val)); }
        out[i * params.out_l_stride] = input_t(out_val);
        // Shift the input buffer by 1
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) { x_vals[i] = x_vals[i + 1]; }
    }
}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_update_launch(ConvParamsBase &params, hipStream_t stream) {
    using Ktraits = Causal_conv1d_update_kernel_traits<kNThreads, kWidth, input_t, weight_t>;
    dim3 grid(params.batch, (params.dim + kNThreads - 1) / kNThreads);
    auto kernel = params.cache_seqlens == nullptr
        ? &causal_conv1d_update_kernel<Ktraits, false>
        : &causal_conv1d_update_kernel<Ktraits, true>;
    kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_update_launch<64, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_update_launch<64, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_update_launch<64, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_update_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<at::Half, at::Half>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<at::BFloat16, at::BFloat16>(ConvParamsBase &params, hipStream_t stream);