#include "hip/hip_runtime.h"
// clang-format off
// adapted from https://github.com/Dao-AILab/causal-conv1d/blob/main/csrc/causal_conv1d_fwd.cu 
// and https://github.com/Dao-AILab/causal-conv1d/blob/main/csrc/causal_conv1d_update.cu
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "causal_conv1d.h"
#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>

#include "static_switch.h"



#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")

#define DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(ITYPE, NAME, ...)              \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        using weight_t = at::Half;                                                  \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::BFloat16) {                                 \
        using input_t = at::BFloat16;                                               \
        using weight_t = at::BFloat16;                                              \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }


template<typename input_t, typename weight_t>
void causal_conv1d_fwd_cuda(ConvParamsBase &params, hipStream_t stream);

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream);

void set_conv_params_fwd(ConvParamsBase &params,
                         // sizes
                         const size_t batch,
                         const size_t dim,
                         const size_t seqlen,
                         const size_t width,
                         // device pointers
                         const at::Tensor x,
                         const at::Tensor weight,
                         const at::Tensor out,
                         const c10::optional<at::Tensor>& bias,
                         bool silu_activation,
                         int64_t pad_slot_id,
                         const c10::optional<at::Tensor>& query_start_loc = std::nullopt,
                         const c10::optional<at::Tensor>& cache_indices = std::nullopt,
                         const c10::optional<at::Tensor>& has_initial_state = std::nullopt) {

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.batch = batch;
    params.dim = dim;
    params.seqlen = seqlen;
    params.width = width;
    params.pad_slot_id = pad_slot_id;

    params.silu_activation = silu_activation;

    // Set the pointers and strides.
    params.x_ptr = x.data_ptr();
    params.weight_ptr = weight.data_ptr();
    params.bias_ptr = bias.has_value() ? bias.value().data_ptr() : nullptr;
    params.out_ptr = out.data_ptr();
    // All stride are in elements, not bytes.
    params.query_start_loc_ptr = query_start_loc.has_value() ? query_start_loc.value().data_ptr() : nullptr;
    params.cache_indices_ptr = cache_indices.has_value() ? cache_indices.value().data_ptr() : nullptr;
    params.has_initial_state_ptr = has_initial_state.has_value() ? has_initial_state.value().data_ptr() : nullptr;
    const bool varlen = params.query_start_loc_ptr != nullptr;
    params.x_batch_stride = x.stride(varlen ? 1 : 0);
    params.x_c_stride = x.stride(varlen ? 0 : 1);
    params.x_l_stride = x.stride(varlen ? 1 : -1);
    params.weight_c_stride = weight.stride(0);
    params.weight_width_stride = weight.stride(1);
    params.out_batch_stride = out.stride(varlen ? 1 : 0);
    params.out_c_stride = out.stride(varlen ? 0 : 1);
    params.out_l_stride = out.stride(varlen ? 1 : -1);
}


void causal_conv1d_fwd(const at::Tensor &x, const at::Tensor &weight,
                  const c10::optional<at::Tensor> &bias_,
                  const c10::optional<at::Tensor> &conv_states,
                  const c10::optional<at::Tensor> &query_start_loc,
                  const c10::optional<at::Tensor> &cache_indices,
                  const c10::optional<at::Tensor> &has_initial_state,
                  bool silu_activation,
                 // used to identify padding entries if cache_indices provided
                 // in case of padding, the kernel will return early
                  int64_t pad_slot_id) {
    auto input_type = x.scalar_type();
    auto weight_type = weight.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float || weight_type == at::ScalarType::Half || weight_type == at::ScalarType::BFloat16);

    TORCH_CHECK(x.is_cuda());
    TORCH_CHECK(weight.is_cuda());
    
    const bool varlen = query_start_loc.has_value() ? true : false;
    const auto sizes = x.sizes();
    const int batch_size = varlen ? query_start_loc.value().sizes()[0] - 1 : sizes[0];
    const int dim = varlen ? sizes[0] : sizes[1];
    const int seqlen = varlen ? sizes[1] : sizes[2];
    const int width = weight.size(-1);
    if (varlen){
        CHECK_SHAPE(x, dim, seqlen);
    }
    else {
        CHECK_SHAPE(x, batch_size, dim, seqlen);
    }
    CHECK_SHAPE(weight, dim, width);



    if (bias_.has_value()) {
        auto bias = bias_.value();
        TORCH_CHECK(bias.scalar_type() == weight_type);
        TORCH_CHECK(bias.is_cuda());
        TORCH_CHECK(bias.stride(-1) == 1);
        CHECK_SHAPE(bias, dim);
    }


    if (has_initial_state.has_value()) {
        auto has_initial_state_ = has_initial_state.value();
        TORCH_CHECK(has_initial_state_.scalar_type() == at::ScalarType::Bool);
        TORCH_CHECK(has_initial_state_.is_cuda());
        CHECK_SHAPE(has_initial_state_, batch_size);
    }


    if (query_start_loc.has_value()) {
        auto query_start_loc_ = query_start_loc.value();
        TORCH_CHECK(query_start_loc_.scalar_type() == at::ScalarType::Int);
        TORCH_CHECK(query_start_loc_.is_cuda());
    }


    if (cache_indices.has_value()) {
        auto cache_indices_ = cache_indices.value();
        TORCH_CHECK(cache_indices_.scalar_type() == at::ScalarType::Int);
        TORCH_CHECK(cache_indices_.is_cuda());
        CHECK_SHAPE(cache_indices_, batch_size);
    }

    at::Tensor out = x;

    ConvParamsBase params;
    set_conv_params_fwd(params, batch_size, dim, seqlen, width, x, weight, out,
                        bias_,
                        silu_activation, 
                        pad_slot_id,
                        query_start_loc,
                        cache_indices,
                        has_initial_state
                        );

    if (conv_states.has_value()) {
        auto conv_states_ = conv_states.value();
        TORCH_CHECK(conv_states_.scalar_type() == input_type);
        TORCH_CHECK(conv_states_.is_cuda());
        params.conv_states_ptr = conv_states_.data_ptr();
        params.conv_states_batch_stride = conv_states_.stride(0);
        params.conv_states_c_stride = conv_states_.stride(1);
        params.conv_states_l_stride = conv_states_.stride(2);
    } else {
        params.conv_states_ptr = nullptr;
    }

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)x.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(x.scalar_type(), "causal_conv1d_fwd", [&] {
            causal_conv1d_fwd_cuda<input_t, weight_t>(params, stream);
    });
}


void causal_conv1d_update(const at::Tensor &x,
                     const at::Tensor &conv_state,
                     const at::Tensor &weight,
                     const c10::optional<at::Tensor> &bias_,
                     bool silu_activation,
                     const c10::optional<at::Tensor> &cache_seqlens_,
                     const c10::optional<at::Tensor> &conv_state_indices_,
                     // used to identify padding entries if cache_indices provided
                     // in case of padding, the kernel will return early
                     int64_t pad_slot_id) {
    auto input_type = x.scalar_type();
    auto weight_type = weight.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float || weight_type == at::ScalarType::Half || weight_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == input_type, "weight type must equal to input type, other variations are disabled due to binary size limitations");
    TORCH_CHECK(conv_state.scalar_type() == input_type);

    TORCH_CHECK(x.is_cuda());
    TORCH_CHECK(conv_state.is_cuda());
    TORCH_CHECK(weight.is_cuda());

    const auto sizes = x.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int seqlen = sizes[2];
    const int width = weight.size(-1);
    const int conv_state_len = conv_state.size(2);
    TORCH_CHECK(conv_state_len >= width - 1);

    CHECK_SHAPE(x, batch_size, dim, seqlen);
    CHECK_SHAPE(weight, dim, width);

    TORCH_CHECK(width >= 2 && width <= 4, "causal_conv1d only supports width between 2 and 4");

    if (bias_.has_value()) {
        auto bias = bias_.value();
        TORCH_CHECK(bias.scalar_type() == weight_type);
        TORCH_CHECK(bias.is_cuda());
        TORCH_CHECK(bias.stride(-1) == 1);
        CHECK_SHAPE(bias, dim);
    }

    at::Tensor out = x;

    ConvParamsBase params;
    set_conv_params_fwd(params, batch_size, dim, seqlen, width, x, weight, out,
                        bias_,
                        silu_activation,
                        pad_slot_id);
    params.conv_state_ptr = conv_state.data_ptr();
    params.conv_state_len = conv_state_len;
    // All stride are in elements, not bytes.
    params.conv_state_batch_stride = conv_state.stride(0);
    params.conv_state_c_stride = conv_state.stride(1);
    params.conv_state_l_stride = conv_state.stride(2);

    if (cache_seqlens_.has_value()) {
        auto cache_seqlens = cache_seqlens_.value();
        TORCH_CHECK(cache_seqlens.scalar_type() == torch::kInt32);
        TORCH_CHECK(cache_seqlens.is_cuda());
        TORCH_CHECK(cache_seqlens.stride(-1) == 1);
        CHECK_SHAPE(cache_seqlens, batch_size);
        params.cache_seqlens = cache_seqlens.data_ptr<int32_t>();
    } else {
        params.cache_seqlens = nullptr;
    }

    if (conv_state_indices_.has_value()) {
        auto conv_state_indices = conv_state_indices_.value();
        TORCH_CHECK(conv_state_indices.scalar_type() == torch::kInt32)
        TORCH_CHECK(conv_state_indices.is_cuda());
        TORCH_CHECK(conv_state_indices.stride(0) == 1)
        CHECK_SHAPE(conv_state_indices, batch_size);

        int conv_state_entries = conv_state.size(0);
        CHECK_SHAPE(conv_state, conv_state_entries, dim, conv_state_len);

        params.conv_state_indices_ptr = conv_state_indices.data_ptr<int32_t>();
    } else {
        CHECK_SHAPE(conv_state, batch_size, dim, conv_state_len);
        params.conv_state_indices_ptr = nullptr;
    }

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)x.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(x.scalar_type(), "causal_conv1d_update", [&] {
            causal_conv1d_update_cuda<input_t, weight_t>(params, stream);
    });
}

template<int kNThreads_, int kWidth_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_fwd_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static_assert(kWidth <= kNElts);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNElts, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, 1, hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNElts, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, 1, hipcub::BLOCK_STORE_DIRECT>;
    static constexpr int kSmemIOSize = kIsVecLoad
        ? 0
        : custom_max({sizeof(typename BlockLoadT::TempStorage), sizeof(typename BlockStoreT::TempStorage)});
    static constexpr int kSmemExchangeSize = kNThreads * kNBytes * kNElts;
    static constexpr int kSmemSize = kSmemIOSize + kSmemExchangeSize;
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_fwd_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    constexpr bool kIsVecLoad = Ktraits::kIsVecLoad;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    extern __shared__ char smem_[];
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_vec = reinterpret_cast<typename Ktraits::BlockLoadVecT::TempStorage&>(smem_);
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_store_vec = reinterpret_cast<typename Ktraits::BlockStoreVecT::TempStorage&>(smem_);
    vec_t *smem_exchange = reinterpret_cast<vec_t *>(smem_ + Ktraits::kSmemIOSize);

    const bool kVarlen = params.query_start_loc_ptr != nullptr;
    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y;
    const int *query_start_loc = kVarlen ? reinterpret_cast<int *>(params.query_start_loc_ptr) : nullptr;
    const int sequence_start_index = kVarlen ? query_start_loc[batch_id] : batch_id;
    const int seqlen = kVarlen ? query_start_loc[batch_id + 1] - sequence_start_index : params.seqlen;

    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + sequence_start_index * params.x_batch_stride
        + channel_id * params.x_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + sequence_start_index * params.out_batch_stride
        + channel_id * params.out_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    bool has_initial_state = params.has_initial_state_ptr == nullptr ? false
        : reinterpret_cast<bool *>(params.has_initial_state_ptr)[batch_id];

    int* cache_indices = params.cache_indices_ptr == nullptr ? nullptr
        : reinterpret_cast<int *>(params.cache_indices_ptr);
    int cache_index = cache_indices == nullptr ? batch_id : cache_indices[batch_id];
    // cache_index == params.pad_slot_id is defined as padding, so we exit early
    if (cache_index == params.pad_slot_id){
        return;
    }
    input_t *conv_states = params.conv_states_ptr == nullptr ? nullptr
        : reinterpret_cast<input_t *>(params.conv_states_ptr) + cache_index * params.conv_states_batch_stride + channel_id * params.conv_states_c_stride;

    // Thread 0 will load the last elements of the previous chunk, so we initialize those to 0.
    if (tidx == 0) {
        input_t initial_state[kNElts] = {0};
        if (has_initial_state) {
            #pragma unroll
            for (int w = 0; w < kWidth - 1; ++w){ initial_state[kNElts - 1 - (kWidth - 2) + w ] = conv_states[w]; }
        }
        smem_exchange[kNThreads - 1] = reinterpret_cast<vec_t *>(initial_state)[0];
    }

    float weight_vals[kWidth];
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }

    constexpr int kChunkSize = kNThreads * kNElts;
    const int n_chunks = (seqlen + kChunkSize - 1) / kChunkSize;
    for (int chunk = 0; chunk < n_chunks; ++chunk) {
        input_t x_vals_load[2 * kNElts] = {0};
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockLoadVecT(smem_load_vec).Load(reinterpret_cast<vec_t*>(x), *reinterpret_cast<vec_t (*)[1]>(&x_vals_load[kNElts]), (seqlen - chunk * kChunkSize) / kNElts);
        } else {
            __syncthreads();
            typename Ktraits::BlockLoadT(smem_load).Load(x, *reinterpret_cast<input_t (*)[kNElts]>(&x_vals_load[kNElts]), seqlen - chunk * kChunkSize);
        }
        x += kChunkSize;
        __syncthreads();
        // Thread kNThreads - 1 don't write yet, so that thread 0 can read
        // the last elements of the previous chunk.
        if (tidx < kNThreads - 1) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1]; }
        __syncthreads();
        reinterpret_cast<vec_t *>(x_vals_load)[0] = smem_exchange[tidx > 0 ? tidx - 1 : kNThreads - 1];
        __syncthreads();
        // Now thread kNThreads - 1 can write the last elements of the current chunk.
        if (tidx == kNThreads - 1) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1]; }

        float x_vals[2 * kNElts];
        #pragma unroll
        for (int i = 0; i < 2 * kNElts; ++i) { x_vals[i] = float(x_vals_load[i]); }

        float out_vals[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) {
            out_vals[i] = bias_val;
            #pragma unroll
            for (int w = 0; w < kWidth; ++w) {
                out_vals[i] += weight_vals[w] * x_vals[kNElts + i - (kWidth - w - 1)];
            }
        }

        if (params.silu_activation) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) {
                out_vals[i] = out_vals[i] / (1 + expf(-out_vals[i]));
            }
        }

        input_t out_vals_store[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) { out_vals_store[i] = out_vals[i]; }
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockStoreVecT(smem_store_vec).Store(reinterpret_cast<vec_t*>(out), reinterpret_cast<vec_t (&)[1]>(out_vals_store), (seqlen - chunk * kChunkSize) / kNElts);
        } else {
            typename Ktraits::BlockStoreT(smem_store).Store(out, out_vals_store, seqlen - chunk * kChunkSize);
        }
        out += kChunkSize;
    }
    // Final state is stored in the smem_exchange last token slot,
    // in case seqlen < kWidth, we would need to take the final state from the 
    // initial state which is stored in conv_states
    // in case seqlen > kWidth, we would need to load the last kWidth - 1 data
    // and load it into conv_state accordingly
    int last_thread =  ((seqlen - (kWidth - 1)) - (n_chunks - 1) * kChunkSize) / kNElts;
    if (conv_states != nullptr && tidx == last_thread) { 
        input_t x_vals_load[kNElts * 2] = {0};
        // in case we are on the first kWidth tokens
        if (last_thread == 0 && seqlen < kWidth){
            // Need to take the initial state
            reinterpret_cast<vec_t *>(x_vals_load)[0] = smem_exchange[0];
            const int offset = seqlen - (kWidth - 1);
            #pragma unroll
            for (int w = 0; w < kWidth - 1; ++w){
                // pad the existing state
                if ((w - seqlen) >= 0 && has_initial_state) { conv_states[w - seqlen] = conv_states[w]; }
                else if ((w - seqlen) >= 0 && !has_initial_state) { conv_states[w - seqlen] = input_t(0.0f); }
            }
            #pragma unroll
            for (int w = 0; w < kWidth - 1; ++w){
                if (offset + w >= 0) 
                    conv_states[w] = x_vals_load[offset + w ];
            }
        }
        else {
            // in case the final state is in between the threads data
            reinterpret_cast<vec_t *>(x_vals_load)[1] = smem_exchange[last_thread + 1];
            reinterpret_cast<vec_t *>(x_vals_load)[0] = smem_exchange[last_thread];
            const int offset = ((seqlen - (kWidth - 1)) % (kNElts));
            #pragma unroll
            for (int w = 0; w < kWidth - 1; ++w){
                conv_states[w] = x_vals_load[offset + w ];
            }
        }
        
    }
}


template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_fwd_launch(ConvParamsBase &params, hipStream_t stream) {
    static constexpr int kNElts = sizeof(input_t) == 4 ? 4 : 8;
    const bool kVarlen = params.query_start_loc_ptr != nullptr;
    BOOL_SWITCH(params.seqlen % kNElts == 0 && !kVarlen, kIsVecLoad, [&] {
        using Ktraits = Causal_conv1d_fwd_kernel_traits<kNThreads, kWidth, kIsVecLoad, input_t, weight_t>;
        constexpr int kSmemSize = Ktraits::kSmemSize;
        dim3 grid(params.batch, params.dim);

        auto kernel = &causal_conv1d_fwd_kernel<Ktraits>;

        if (kSmemSize >= 48 * 1024) {
            #ifndef USE_ROCM
            C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
            #else
            // There is a slight signature discrepancy in HIP and CUDA "FuncSetAttribute" function.
            C10_CUDA_CHECK(hipFuncSetAttribute(
                (void *) kernel, hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
            std::cerr << "Warning (causal_conv1d fwd launch): attempting to set maxDynamicSharedMemorySize on an AMD GPU which is currently a non-op (in ROCm versions <= 6.1). This might lead to undefined behavior. \n" << std::endl;
            #endif
        }
        kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);

        C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_fwd_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_fwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_fwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_fwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}


template void causal_conv1d_fwd_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<at::Half, at::Half>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<at::BFloat16, at::BFloat16>(ConvParamsBase &params, hipStream_t stream);




template<int kNThreads_, int kWidth_, typename input_t_, typename weight_t_>
struct Causal_conv1d_update_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
};

template<typename Ktraits, bool kIsCircularBuffer>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_update_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y * kNThreads + tidx;
    if (channel_id >= params.dim) return;

    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + channel_id * params.x_c_stride;

    // If params.conv_state_batch_indices is set, then the conv state is gathered from the conv state tensor
    // along the batch axis. Otherwise, the conv state coordinate is the same as the batch id.
    const int conv_state_batch_coord = params.conv_state_indices_ptr == nullptr
        ? batch_id
        : params.conv_state_indices_ptr[batch_id];
    // conv_state_batch_coord == params.pad_slot_id is defined as padding so we exit early
    if (conv_state_batch_coord == params.pad_slot_id){
        return;
    }
    input_t *conv_state = reinterpret_cast<input_t *>(params.conv_state_ptr) 
        + conv_state_batch_coord * params.conv_state_batch_stride
        + channel_id * params.conv_state_c_stride;

    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + channel_id * params.out_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    int state_len = params.conv_state_len;
    int advance_len = params.seqlen;
    int cache_seqlen = kIsCircularBuffer ? params.cache_seqlens[batch_id] % state_len : 0;
    int update_idx = cache_seqlen - (kWidth - 1);
    update_idx = update_idx < 0 ? update_idx + state_len : update_idx;

    float weight_vals[kWidth] = {0};
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }

    float x_vals[kWidth] = {0};
    if constexpr (!kIsCircularBuffer) {
        #pragma unroll 2
        for (int i = 0; i < state_len - advance_len - (kWidth - 1); ++i) {
            conv_state[i * params.conv_state_l_stride] = conv_state[(i + advance_len) * params.conv_state_l_stride];
        }
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) {
            input_t state_val = conv_state[(state_len - (kWidth - 1) + i) * params.conv_state_l_stride];
            if (i < advance_len + (kWidth - 1) && state_len - advance_len - (kWidth - 1) + i >= 0) {
                conv_state[(state_len - advance_len - (kWidth - 1) + i) * params.conv_state_l_stride] = state_val;
            }
            x_vals[i] = float(state_val);
        }
    } else {
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i, update_idx = update_idx + 1 >= state_len ? update_idx + 1 - state_len : update_idx + 1) {
            input_t state_val = conv_state[update_idx * params.conv_state_l_stride];
            x_vals[i] = float(state_val);
        }
    }
    #pragma unroll 2
    for (int i = 0; i < params.seqlen; ++i) {
        input_t x_val = x[i * params.x_l_stride];
        if constexpr (!kIsCircularBuffer) {
            if (i < advance_len && state_len - advance_len + i >= 0) {
                conv_state[(state_len - advance_len + i) * params.conv_state_l_stride] = x_val;
            }
        } else {
            conv_state[update_idx * params.conv_state_l_stride] = x_val;
            ++update_idx;
            update_idx = update_idx >= state_len ? update_idx - state_len : update_idx;
        }
        x_vals[kWidth - 1] = float(x_val);
        float out_val = bias_val;
        #pragma unroll
        for (int j = 0; j < kWidth; ++j) { out_val += weight_vals[j] * x_vals[j]; }
        if (params.silu_activation) { out_val = out_val / (1 + expf(-out_val)); }
        out[i * params.out_l_stride] = input_t(out_val);
        // Shift the input buffer by 1
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) { x_vals[i] = x_vals[i + 1]; }
    }
}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_update_launch(ConvParamsBase &params, hipStream_t stream) {
    using Ktraits = Causal_conv1d_update_kernel_traits<kNThreads, kWidth, input_t, weight_t>;
    dim3 grid(params.batch, (params.dim + kNThreads - 1) / kNThreads);
    auto kernel = params.cache_seqlens == nullptr
        ? &causal_conv1d_update_kernel<Ktraits, false>
        : &causal_conv1d_update_kernel<Ktraits, true>;
    kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_update_launch<64, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_update_launch<64, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_update_launch<64, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_update_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<at::Half, at::Half>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<at::BFloat16, at::BFloat16>(ConvParamsBase &params, hipStream_t stream);
