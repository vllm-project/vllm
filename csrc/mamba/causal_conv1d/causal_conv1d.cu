#include "hip/hip_runtime.h"
// clang-format off
// adapted from https://github.com/Dao-AILab/causal-conv1d/blob/main/csrc/causal_conv1d_fwd.cu 
// and https://github.com/Dao-AILab/causal-conv1d/blob/main/csrc/causal_conv1d_update.cu
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "causal_conv1d.h"
#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>

#include "static_switch.h"



#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")

#define DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(ITYPE, NAME, ...)              \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        using weight_t = at::Half;                                                  \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::BFloat16) {                                 \
        using input_t = at::BFloat16;                                               \
        using weight_t = at::BFloat16;                                              \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }


template<typename input_t, typename weight_t>
void causal_conv1d_fwd_cuda(ConvParamsBase &params, hipStream_t stream);
template <typename input_t, typename weight_t>
void causal_conv1d_channellast_fwd_cuda(ConvParamsBase &params, hipStream_t stream);

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream);

void set_conv_params_fwd(ConvParamsBase &params,
                         // sizes
                         const size_t batch,
                         const size_t dim,
                         const size_t seqlen,
                         const size_t width,
                         // device pointers
                         const at::Tensor x,
                         const at::Tensor weight,
                         const at::Tensor out,
                         void* bias_ptr,
                         bool silu_activation) {

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.batch = batch;
    params.dim = dim;
    params.seqlen = seqlen;
    params.width = width;

    params.silu_activation = silu_activation;

    // Set the pointers and strides.
    params.x_ptr = x.data_ptr();
    params.weight_ptr = weight.data_ptr();
    params.bias_ptr = bias_ptr;
    params.out_ptr = out.data_ptr();
    // All stride are in elements, not bytes.
    params.x_batch_stride = x.stride(0);
    params.x_c_stride = x.stride(1);
    params.x_l_stride = x.stride(-1);
    params.weight_c_stride = weight.stride(0);
    params.weight_width_stride = weight.stride(1);
    params.out_batch_stride = out.stride(0);
    params.out_c_stride = out.stride(1);
    params.out_l_stride = out.stride(-1);
}


at::Tensor
causal_conv1d_fwd(const at::Tensor &x, const at::Tensor &weight,
                  const c10::optional<at::Tensor> &bias_,
                  const c10::optional<at::Tensor> &seq_idx_,
                  const c10::optional<at::Tensor> &initial_states_,
                  const c10::optional<at::Tensor> &final_states_out_,
                  bool silu_activation) {
    auto input_type = x.scalar_type();
    auto weight_type = weight.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float || weight_type == at::ScalarType::Half || weight_type == at::ScalarType::BFloat16);

    TORCH_CHECK(x.is_cuda());
    TORCH_CHECK(weight.is_cuda());

    const auto sizes = x.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int seqlen = sizes[2];
    const int width = weight.size(-1);

    CHECK_SHAPE(x, batch_size, dim, seqlen);
    CHECK_SHAPE(weight, dim, width);

    TORCH_CHECK(x.stride(2) == 1 || x.stride(1) == 1);
    const bool is_channel_last = x.stride(1) == 1 && x.stride(2) > 1;

    if (is_channel_last) {
        TORCH_CHECK(dim % 8 == 0, "causal_conv1d only supports channel dimension divisible by 8 for now");
        TORCH_CHECK(x.stride(2) % 8 == 0 and x.stride(0) % 8 == 0, "causal_conv1d with channel last layout requires strides (x.stride(0) and x.stride(2)) to be multiples of 8");
    }
    TORCH_CHECK(width >= 2 && width <= 4, "causal_conv1d only supports width between 2 and 4");

    if (bias_.has_value()) {
        auto bias = bias_.value();
        TORCH_CHECK(bias.scalar_type() == weight_type);
        TORCH_CHECK(bias.is_cuda());
        TORCH_CHECK(bias.stride(-1) == 1);
        CHECK_SHAPE(bias, dim);
    }

    if (seq_idx_.has_value()) {
        TORCH_CHECK(is_channel_last, "seq_idx is only supported for channel last layout");
        auto seq_idx = seq_idx_.value();
        TORCH_CHECK(seq_idx.scalar_type() == torch::kInt32);
        TORCH_CHECK(seq_idx.is_cuda());
        TORCH_CHECK(seq_idx.is_contiguous());
        CHECK_SHAPE(seq_idx, batch_size, seqlen);
    }

    at::Tensor out = torch::empty_like(x);

    ConvParamsBase params;
    set_conv_params_fwd(params, batch_size, dim, seqlen, width, x, weight, out,
                        bias_.has_value() ? bias_.value().data_ptr() : nullptr,
                        silu_activation);

    if (seq_idx_.has_value()) {
        params.seq_idx_ptr = seq_idx_.value().data_ptr();
    } else {
        params.seq_idx_ptr = nullptr;
    }

    if (initial_states_.has_value()) {
        TORCH_CHECK(is_channel_last, "initial_states is only supported for channel last layout");
        auto initial_states = initial_states_.value();
        TORCH_CHECK(initial_states.scalar_type() == input_type);
        TORCH_CHECK(initial_states.is_cuda());
        CHECK_SHAPE(initial_states, batch_size, dim, width - 1);
        TORCH_CHECK(initial_states.stride(1) == 1);
        params.initial_states_ptr = initial_states.data_ptr();
        params.initial_states_batch_stride = initial_states.stride(0);
        params.initial_states_c_stride = initial_states.stride(1);
        params.initial_states_l_stride = initial_states.stride(2);
    } else {
        params.initial_states_ptr = nullptr;
    }

    if (final_states_out_.has_value()) {
        TORCH_CHECK(is_channel_last, "final_states is only supported for channel last layout");
        auto final_states = final_states_out_.value();
        TORCH_CHECK(final_states.scalar_type() == input_type);
        TORCH_CHECK(final_states.is_cuda());
        CHECK_SHAPE(final_states, batch_size, dim, width - 1);
        TORCH_CHECK(final_states.stride(1) == 1);
        params.final_states_ptr = final_states.data_ptr();
        params.final_states_batch_stride = final_states.stride(0);
        params.final_states_c_stride = final_states.stride(1);
        params.final_states_l_stride = final_states.stride(2);
    } else {
        params.final_states_ptr = nullptr;
    }

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)x.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(x.scalar_type(), "causal_conv1d_fwd", [&] {
            if (!is_channel_last) {
                causal_conv1d_fwd_cuda<input_t, weight_t>(params, stream);
            } else {
                causal_conv1d_channellast_fwd_cuda<input_t, weight_t>(params, stream);
            }
    });
    return out;
}


at::Tensor
causal_conv1d_update(const at::Tensor &x,
                     const at::Tensor &conv_state,
                     const at::Tensor &weight,
                     const c10::optional<at::Tensor> &bias_,
                     bool silu_activation) {
    auto input_type = x.scalar_type();
    auto weight_type = weight.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float || weight_type == at::ScalarType::Half || weight_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == input_type, "weight type must equal to input type, other variations are disabled due to binary size limitations");
    TORCH_CHECK(conv_state.scalar_type() == input_type);

    TORCH_CHECK(x.is_cuda());
    TORCH_CHECK(conv_state.is_cuda());
    TORCH_CHECK(weight.is_cuda());

    const auto sizes = x.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int width = weight.size(-1);

    CHECK_SHAPE(x, batch_size, dim);
    CHECK_SHAPE(conv_state, batch_size, dim, width);
    CHECK_SHAPE(weight, dim, width);

    TORCH_CHECK(width >= 2 && width <= 4, "causal_conv1d only supports width between 2 and 4");

    if (bias_.has_value()) {
        auto bias = bias_.value();
        TORCH_CHECK(bias.scalar_type() == weight_type);
        TORCH_CHECK(bias.is_cuda());
        TORCH_CHECK(bias.stride(-1) == 1);
        CHECK_SHAPE(bias, dim);
    }

    at::Tensor out = torch::empty_like(x);

    ConvParamsBase params;
    set_conv_params_fwd(params, batch_size, dim, /*seqlen=*/1, width, x, weight, out,
                        bias_.has_value() ? bias_.value().data_ptr() : nullptr,
                        silu_activation);
    params.conv_state_ptr = conv_state.data_ptr();
    // All stride are in elements, not bytes.
    params.conv_state_batch_stride = conv_state.stride(0);
    params.conv_state_c_stride = conv_state.stride(1);
    params.conv_state_l_stride = conv_state.stride(2);

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)x.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(x.scalar_type(), "causal_conv1d_update", [&] {
            causal_conv1d_update_cuda<input_t, weight_t>(params, stream);
    });
    return out;
}

template<int kNThreads_, int kWidth_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_fwd_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static_assert(kWidth <= kNElts);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNElts, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, 1, hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNElts, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, 1, hipcub::BLOCK_STORE_DIRECT>;
    static constexpr int kSmemIOSize = kIsVecLoad
        ? 0
        : custom_max({sizeof(typename BlockLoadT::TempStorage), sizeof(typename BlockStoreT::TempStorage)});
    static constexpr int kSmemExchangeSize = kNThreads * kNBytes * kNElts;
    static constexpr int kSmemSize = kSmemIOSize + kSmemExchangeSize;
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_fwd_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    static constexpr bool kIsVecLoad = Ktraits::kIsVecLoad;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    extern __shared__ char smem_[];
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_vec = reinterpret_cast<typename Ktraits::BlockLoadVecT::TempStorage&>(smem_);
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_store_vec = reinterpret_cast<typename Ktraits::BlockStoreVecT::TempStorage&>(smem_);
    vec_t *smem_exchange = reinterpret_cast<vec_t *>(smem_ + Ktraits::kSmemIOSize);

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + channel_id * params.x_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + channel_id * params.out_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    // Thread 0 will load the last elements of the previous chunk, so we initialize those to 0.
    if (tidx == 0) {
        input_t zeros[kNElts] = {0};
        smem_exchange[kNThreads - 1] = reinterpret_cast<vec_t *>(zeros)[0];
    }

    float weight_vals[kWidth];
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }

    constexpr int kChunkSize = kNThreads * kNElts;
    const int n_chunks = (params.seqlen + kChunkSize - 1) / kChunkSize;
    for (int chunk = 0; chunk < n_chunks; ++chunk) {
        input_t x_vals_load[2 * kNElts] = {0};
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockLoadVecT(smem_load_vec).Load(reinterpret_cast<vec_t*>(x), *reinterpret_cast<vec_t (*)[1]>(&x_vals_load[kNElts]), (params.seqlen - chunk * kChunkSize) / kNElts);
        } else {
            __syncthreads();
            typename Ktraits::BlockLoadT(smem_load).Load(x, *reinterpret_cast<input_t (*)[kNElts]>(&x_vals_load[kNElts]), params.seqlen - chunk * kChunkSize);
        }
        x += kChunkSize;
        __syncthreads();
        // Thread kNThreads - 1 don't write yet, so that thread 0 can read
        // the last elements of the previous chunk.
        if (tidx < kNThreads - 1) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1]; }
        __syncthreads();
        reinterpret_cast<vec_t *>(x_vals_load)[0] = smem_exchange[tidx > 0 ? tidx - 1 : kNThreads - 1];
        __syncthreads();
        // Now thread kNThreads - 1 can write the last elements of the current chunk.
        if (tidx == kNThreads - 1) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1]; }

        float x_vals[2 * kNElts];
        #pragma unroll
        for (int i = 0; i < 2 * kNElts; ++i) { x_vals[i] = float(x_vals_load[i]); }

        float out_vals[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) {
            out_vals[i] = bias_val;
            #pragma unroll
            for (int w = 0; w < kWidth; ++w) {
                out_vals[i] += weight_vals[w] * x_vals[kNElts + i - (kWidth - w - 1)];
            }
        }

        if (params.silu_activation) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) {
                out_vals[i] = out_vals[i] / (1 + expf(-out_vals[i]));
            }
        }

        input_t out_vals_store[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) { out_vals_store[i] = out_vals[i]; }
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockStoreVecT(smem_store_vec).Store(reinterpret_cast<vec_t*>(out), reinterpret_cast<vec_t (&)[1]>(out_vals_store), (params.seqlen - chunk * kChunkSize) / kNElts);
        } else {
            typename Ktraits::BlockStoreT(smem_store).Store(out, out_vals_store, params.seqlen - chunk * kChunkSize);
        }
        out += kChunkSize;
    }
}


template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_fwd_launch(ConvParamsBase &params, hipStream_t stream) {
    static constexpr int kNElts = sizeof(input_t) == 4 ? 4 : 8;
    BOOL_SWITCH(params.seqlen % kNElts == 0, kIsVecLoad, [&] {
        using Ktraits = Causal_conv1d_fwd_kernel_traits<kNThreads, kWidth, kIsVecLoad, input_t, weight_t>;
        constexpr int kSmemSize = Ktraits::kSmemSize;
        dim3 grid(params.batch, params.dim);

        auto kernel = &causal_conv1d_fwd_kernel<Ktraits>;

        if (kSmemSize >= 48 * 1024) {
            #ifndef USE_ROCM
            C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
            #else
            // There is a slight signature discrepancy in HIP and CUDA "FuncSetAttribute" function.
            C10_CUDA_CHECK(hipFuncSetAttribute(
                (void *) kernel, hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
            std::cerr << "Warning (causal_conv1d fwd launch): attempting to set maxDynamicSharedMemorySize on an AMD GPU which is currently a non-op (in ROCm versions <= 6.1). This might lead to undefined behavior. \n" << std::endl;
            #endif
        }
        kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);

        C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_fwd_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_fwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_fwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_fwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}

template<int kNThreads_, int kWidth_, int kChunkSizeL_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_channellast_fwd_kernel_traits {
    // The cache line is 128 bytes, and we try to read 16 bytes per thread.
    // So we have 8 threads per "row", so 32 or 64 elements in the channel dimension.
    // That leaves 4 columns per warp, and so 16 columns per block (assuming each block has 128
    // threads). Each each load is 16 x 32|64 elements in the L x C dimensions.
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static_assert(kNThreads % 32 == 0);
    static constexpr int kNWarps = kNThreads / 32;
    static constexpr int kWidth = kWidth_;
    static constexpr int kChunkSizeL = kChunkSizeL_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static constexpr int kNEltsPerRow = 128 / kNBytes;
    static constexpr int kNThreadsPerRow = kNEltsPerRow / kNElts;  // Always 8 for now
    static_assert(kNThreadsPerRow * kNBytes * kNElts == 128);
    static constexpr int kNColsPerWarp = 32 / kNThreadsPerRow;  // Always 4 for now
    static_assert(kNColsPerWarp * kNThreadsPerRow == 32);
    static constexpr int kNColsPerLoad = kNColsPerWarp * kNWarps;
    static constexpr int kNLoads = kChunkSizeL / kNColsPerLoad;
    static_assert(kNLoads * kNColsPerLoad == kChunkSizeL);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    // using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    // using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    // static constexpr int kSmemSize = std::max({sizeof(typename BlockLoadT::TempStorage),
    //                                            sizeof(typename BlockStoreT::TempStorage)});
    // static constexpr int kSmemSize = kChunkSizeL * kNEltsPerRow * kNBytes;
};

template<typename Ktraits, bool kHasSeqIdx>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_channellast_fwd_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    constexpr int kNThreadsPerC = Ktraits::kNThreadsPerRow;
    constexpr int kLPerLoad = Ktraits::kNColsPerLoad;
    constexpr int kChunkSizeL = Ktraits::kChunkSizeL;
    constexpr int kChunkSizeC = Ktraits::kNEltsPerRow;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    __shared__ input_t x_smem[kWidth - 1 + kChunkSizeL][kChunkSizeC + kNElts];

    const int batch_id = blockIdx.x;
    const int chunk_l_id = blockIdx.y;
    const int chunk_c_id = blockIdx.z;
    const int tid = threadIdx.x;
    const int l_idx = tid / kNThreadsPerC;
    const int c_idx = tid % kNThreadsPerC;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.x_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr)
        + chunk_c_id * kChunkSizeC * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.out_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    int *seq_idx = !kHasSeqIdx ? nullptr : reinterpret_cast<int *>(params.seq_idx_ptr)
        + batch_id * params.seqlen + chunk_l_id * kChunkSizeL;
    input_t *initial_states = params.initial_states_ptr == nullptr || chunk_l_id > 0 ? nullptr
        : reinterpret_cast<input_t *>(params.initial_states_ptr) + batch_id * params.initial_states_batch_stride + l_idx * params.initial_states_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    // The last L-chunk will also have enough info to write to final states, since it also contain a few x values
    // from the previous L-chunk.
    input_t *final_states = params.final_states_ptr == nullptr || chunk_l_id < gridDim.y - 1 ? nullptr
        : reinterpret_cast<input_t *>(params.final_states_ptr) + batch_id * params.final_states_batch_stride + l_idx * params.final_states_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;

    #pragma unroll
    for (int l = 0; l < Ktraits::kNLoads; ++l) {
        input_t x_vals_load[kNElts] = {0};
        if (chunk_l_id * kChunkSizeL + l * kLPerLoad + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x + l * kLPerLoad * params.x_l_stride);
        }
        reinterpret_cast<vec_t *>(x_smem[kWidth - 1 + l * kLPerLoad + l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
    }
    // Load the elements from the previous chunk that are needed for convolution.
    if (l_idx < kWidth - 1) {
        input_t x_vals_load[kNElts] = {0};
        if (chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) >= 0
            && chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x - (kWidth - 1) * params.x_l_stride);
        } else if (initial_states != nullptr
                   && chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) < 0
                   && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(initial_states);
        }
        reinterpret_cast<vec_t *>(x_smem[l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
    }

    __syncthreads();

    if (final_states != nullptr
        && l_idx < kWidth - 1
        && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
        // x_smem[0] contains element at index chunk_l_id * kChunkSizeL - (kWidth - 1)
        // So last few elements (index params.seqlen - kWidth + 1 + l_idx) are stored in x_smem[params.seqlen - kWidth + 1 + l_idx - (chunk_l_id * kChunkSizeL - kWidth + 1)][c_idx]
        *reinterpret_cast<vec_t *>(final_states) = reinterpret_cast<vec_t *>(x_smem[params.seqlen + l_idx - chunk_l_id * kChunkSizeL])[c_idx];
    }

    constexpr int kLPerThread = constexpr_min(kChunkSizeL * kChunkSizeC / kNThreads, kChunkSizeL);
    static_assert(kLPerThread * kNThreads == kChunkSizeL * kChunkSizeC);
    constexpr int kNThreadsPerRow = kChunkSizeL / kLPerThread;
    static_assert(kNThreadsPerRow * kLPerThread == kChunkSizeL);
    // kChunkSizeL, kLPerThread, kNThreadsPerRow should be powers of 2 for simplicity
    static_assert((kChunkSizeL & (kChunkSizeL - 1)) == 0);
    static_assert((kLPerThread & (kLPerThread - 1)) == 0);
    static_assert((kNThreadsPerRow & (kNThreadsPerRow - 1)) == 0);
    static_assert(kNThreadsPerRow <= 32);

    const int row_idx = tid / kNThreadsPerRow;
    const int col_idx = tid % kNThreadsPerRow;

    float bias_val = params.bias_ptr == nullptr || chunk_c_id * kChunkSizeC + row_idx >= params.dim ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[chunk_c_id * kChunkSizeC + row_idx]);
    float weight_vals[kWidth] = {0};
    if (chunk_c_id * kChunkSizeC + row_idx < params.dim) {
        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            weight_vals[w] = weight[row_idx * params.weight_c_stride + w * params.weight_width_stride];
        }
    }
    float x_vals[kWidth - 1 + kLPerThread];
    #pragma unroll
    for (int i = 0; i < kWidth - 1 + kLPerThread; ++i) {
        x_vals[i] = float(x_smem[col_idx * kLPerThread + i][row_idx]);
    }
    int seq_idx_thread[kWidth - 1 + kLPerThread];
    if constexpr (kHasSeqIdx) {
        #pragma unroll
        for (int i = 0; i < kWidth - 1 + kLPerThread; ++i) {
            seq_idx_thread[i] = chunk_l_id * kChunkSizeL + col_idx * kLPerThread + i - (kWidth - 1) >= 0 ? seq_idx[col_idx * kLPerThread + i - (kWidth - 1)] : -1;
        }
    }

    float out_vals[kLPerThread];
    #pragma unroll
    for (int i = 0; i < kLPerThread; ++i) {
        out_vals[i] = bias_val;
        const int seq_idx_cur = !kHasSeqIdx ? 0 : seq_idx_thread[i + kWidth - 1];
        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            if constexpr (!kHasSeqIdx) {
                out_vals[i] += weight_vals[w] * x_vals[i + w];
            } else {
                out_vals[i] += seq_idx_thread[i + w] == seq_idx_cur ? weight_vals[w] * x_vals[i + w] : 0.f;
            }
        }
        if (params.silu_activation) {out_vals[i] = out_vals[i] / (1 + expf(-out_vals[i])); }
    }

    __syncthreads();
    #pragma unroll
    for (int i = 0; i < kLPerThread; ++i) { x_smem[col_idx * kLPerThread + i][row_idx] = out_vals[i]; }
    __syncthreads();

    #pragma unroll
    for (int l = 0; l < Ktraits::kNLoads; ++l) {
        input_t out_vals_store[kNElts];
        reinterpret_cast<vec_t *>(out_vals_store)[0] = reinterpret_cast<vec_t *>(x_smem[l * kLPerLoad + l_idx])[c_idx];
        if (chunk_l_id * kChunkSizeL + l * kLPerLoad + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            *reinterpret_cast<vec_t *>(out + l * kLPerLoad * params.out_l_stride) = reinterpret_cast<vec_t *>(out_vals_store)[0];
        }
    }

}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_channellast_fwd_launch(ConvParamsBase &params, hipStream_t stream) {
    BOOL_SWITCH(params.seq_idx_ptr != nullptr, kHasSeqIdx, [&] {
        using Ktraits = Causal_conv1d_channellast_fwd_kernel_traits<kNThreads, kWidth, 64, true, input_t, weight_t>;
        // constexpr int kSmemSize = Ktraits::kSmemSize;
        constexpr int kChunkSizeL = Ktraits::kChunkSizeL;
        constexpr int kChunkSizeC = Ktraits::kNEltsPerRow;
        const int n_chunks_L = (params.seqlen + kChunkSizeL - 1) / kChunkSizeL;
        const int n_chunks_C = (params.dim + kChunkSizeC - 1) / kChunkSizeC;
        dim3 grid(params.batch, n_chunks_L, n_chunks_C);
        dim3 block(Ktraits::kNThreads);
        auto kernel = &causal_conv1d_channellast_fwd_kernel<Ktraits, kHasSeqIdx>;
        // if (kSmemSize >= 48 * 1024) {
        //     C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        //         kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        //     }
        // kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
        kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_channellast_fwd_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_channellast_fwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_channellast_fwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_channellast_fwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_fwd_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<at::Half, at::Half>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<at::BFloat16, at::BFloat16>(ConvParamsBase &params, hipStream_t stream);

template void causal_conv1d_channellast_fwd_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<at::Half, at::Half>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<at::BFloat16, at::BFloat16>(ConvParamsBase &params, hipStream_t stream);
///////




template<int kNThreads_, int kWidth_, typename input_t_, typename weight_t_>
struct Causal_conv1d_update_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_update_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y * kNThreads + tidx;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + channel_id * params.x_c_stride;
    input_t *conv_state = reinterpret_cast<input_t *>(params.conv_state_ptr) + batch_id * params.conv_state_batch_stride
        + channel_id * params.conv_state_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + channel_id * params.out_c_stride;
    float bias_val = params.bias_ptr == nullptr || channel_id >= params.dim ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    float weight_vals[kWidth] = {0};
    if (channel_id < params.dim) {
        #pragma unroll
        for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }
    }

    float x_vals[kWidth] = {0};
    if (channel_id < params.dim) {
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) { x_vals[i] = float(conv_state[(i + 1) * params.conv_state_l_stride]); }
        x_vals[kWidth - 1] = float(x[0]);
        #pragma unroll
        for (int i = 0; i < kWidth; ++i) { conv_state[i * params.conv_state_l_stride] = input_t(x_vals[i]); }
    }

    float out_val = bias_val;
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { out_val += weight_vals[i] * x_vals[i]; }
    if (params.silu_activation) { out_val = out_val / (1 + expf(-out_val)); }
    if (channel_id < params.dim) { out[0] = input_t(out_val); }
}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_update_launch(ConvParamsBase &params, hipStream_t stream) {
    using Ktraits = Causal_conv1d_update_kernel_traits<kNThreads, kWidth, input_t, weight_t>;
    dim3 grid(params.batch, (params.dim + kNThreads - 1) / kNThreads);
    auto kernel = &causal_conv1d_update_kernel<Ktraits>;
    kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_update_launch<64, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_update_launch<64, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_update_launch<64, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_update_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<at::Half, at::Half>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<at::BFloat16, at::BFloat16>(ConvParamsBase &params, hipStream_t stream);
