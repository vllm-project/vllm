#include "hip/hip_runtime.h"
// clang-format off
// adapted from https://github.com/state-spaces/mamba/blob/main/csrc/selective_scan/selective_scan_fwd_kernel.cuh
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "selective_scan.h"

#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#ifndef USE_ROCM
    #include <cub/block/block_load.cuh>
    #include <cub/block/block_store.cuh>
    #include <cub/block/block_scan.cuh>
#else
    #include <hipcub/hipcub.hpp>
    namespace cub = hipcub;
#endif

#include "selective_scan.h"
#include "static_switch.h"

template<int kNThreads_, int kNItems_, int kNRows_, bool kIsEvenLen_,
         bool kIsVariableB_, bool kIsVariableC_,
         bool kHasZ_, bool kUseIndex_, typename input_t_, typename weight_t_>
struct Selective_Scan_fwd_kernel_traits {
    static_assert(kNItems_ % 4 == 0);
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    // Setting MinBlocksPerMP to be 3 (instead of 2) for 128 threads improves occupancy.
    static constexpr int kMinBlocks = kNThreads < 128 ? 5 : 3;
    static constexpr int kNItems = kNItems_;
    static constexpr int kNRows = kNRows_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : constexpr_min(8, kNItems);
    static_assert(kNItems % kNElts == 0);
    static constexpr int kNLoads = kNItems / kNElts;
    static constexpr bool kIsEvenLen = kIsEvenLen_;
    static constexpr bool kIsVariableB = kIsVariableB_;
    static constexpr bool kIsVariableC = kIsVariableC_;
    static constexpr bool kHasZ = kHasZ_;
    static constexpr bool kUseIndex = kUseIndex_;

    static constexpr bool kDirectIO = kIsEvenLen && kNLoads == 1;
    static constexpr int kNLoadsIndex = kNItems / 4;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using scan_t = float2;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockLoadIndexT = hipcub::BlockLoad<int, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadIndexVecT = hipcub::BlockLoad<uint4, kNThreads, kNLoadsIndex,
        !(kIsEvenLen && kNLoadsIndex == 1) ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockLoadWeightT = hipcub::BlockLoad<input_t, kNThreads, kNItems , hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadWeightVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads ,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE  : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_STORE_WARP_TRANSPOSE : hipcub::BLOCK_STORE_DIRECT>;
    // using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_RAKING_MEMOIZE>;
    // using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_RAKING>;
    using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    static constexpr int kSmemIOSize = custom_max({sizeof(typename BlockLoadT::TempStorage),
                                                 sizeof(typename BlockLoadVecT::TempStorage),
                                                 sizeof(typename BlockLoadIndexT::TempStorage),
                                                 sizeof(typename BlockLoadIndexVecT::TempStorage),
                                                 (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightT::TempStorage),
                                                 (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightVecT::TempStorage),
                                                 sizeof(typename BlockStoreT::TempStorage),
                                                 sizeof(typename BlockStoreVecT::TempStorage)});
    static constexpr int kSmemSize = kSmemIOSize + sizeof(typename BlockScanT::TempStorage);
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads, Ktraits::kMinBlocks)
void selective_scan_fwd_kernel(SSMParamsBase params) {
    constexpr bool kIsVariableB = Ktraits::kIsVariableB;
    constexpr bool kIsVariableC = Ktraits::kIsVariableC;
    constexpr bool kHasZ = Ktraits::kHasZ;
    constexpr bool kUseIndex = Ktraits::kUseIndex;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNItems = Ktraits::kNItems;
    constexpr int kNRows = Ktraits::kNRows;
    constexpr bool kDirectIO = Ktraits::kDirectIO;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;
    using scan_t = typename Ktraits::scan_t;

    // Shared memory.
    extern __shared__ char smem_[];
    // cast to lvalue reference of expected type
    // char *smem_loadstorescan = smem_ + 2 * MAX_DSTATE * sizeof(weight_t);
    // auto& smem_load = reinterpret_cast<typename BlockLoadT::TempStorage&>(smem_ + 2 * MAX_DSTATE * sizeof(weight_t));
    // auto& smem_load = reinterpret_cast<typename BlockLoadT::TempStorage&>(smem_loadstorescan);
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_weight = reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage&>(smem_);
    auto& smem_load_index = reinterpret_cast<typename Ktraits::BlockLoadIndexT::TempStorage&>(smem_);
    auto& smem_load_weight1 = *reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage*>(smem_ + sizeof(typename Ktraits::BlockLoadWeightT::TempStorage));
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_scan = *reinterpret_cast<typename Ktraits::BlockScanT::TempStorage*>(smem_ + Ktraits::kSmemIOSize);
    // weight_t *smem_a = reinterpret_cast<weight_t *>(smem_ + smem_loadstorescan_size);
    // weight_t *smem_bc = reinterpret_cast<weight_t *>(smem_a + MAX_DSTATE);
    scan_t *smem_running_prefix = reinterpret_cast<scan_t *>(smem_ + Ktraits::kSmemSize);

    const int batch_id = blockIdx.x;
    const int dim_id = blockIdx.y;
    const int group_id = dim_id / (params.dim_ngroups_ratio);
    input_t *u = reinterpret_cast<input_t *>(params.u_ptr) + batch_id * params.u_batch_stride
        + dim_id * kNRows * params.u_d_stride;
    input_t *delta = reinterpret_cast<input_t *>(params.delta_ptr) + batch_id * params.delta_batch_stride
        + dim_id * kNRows * params.delta_d_stride;
    weight_t *A = reinterpret_cast<weight_t *>(params.A_ptr) + dim_id * kNRows * params.A_d_stride;
    weight_t *B = reinterpret_cast<weight_t *>(params.B_ptr) + dim_id * kNRows * params.B_d_stride;
    input_t *Bvar = reinterpret_cast<input_t *>(params.B_ptr) + batch_id * params.B_batch_stride + group_id * params.B_group_stride;
    weight_t *C = reinterpret_cast<weight_t *>(params.C_ptr) + dim_id * kNRows * params.C_d_stride;
    input_t *Cvar = reinterpret_cast<input_t *>(params.C_ptr) + batch_id * params.C_batch_stride + group_id * params.C_group_stride;
    scan_t *x = reinterpret_cast<scan_t *>(params.x_ptr) + (batch_id * params.dim + dim_id * kNRows) * params.n_chunks * params.dstate;
    int *index = !kUseIndex ? nullptr :reinterpret_cast<int *>(params.index_ptr) + batch_id * params.seqlen;

    float D_val[kNRows] = {0};
    if (params.D_ptr != nullptr) {
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            D_val[r] = reinterpret_cast<float *>(params.D_ptr)[dim_id * kNRows + r];
        }
    }
    float delta_bias[kNRows] = {0};
    if (params.delta_bias_ptr != nullptr) {
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            delta_bias[r] = reinterpret_cast<float *>(params.delta_bias_ptr)[dim_id * kNRows + r];
        }
    }


    // for (int state_idx = threadIdx.x; state_idx < params.dstate; state_idx += blockDim.x) {
    //     smem_a[state_idx] = A[state_idx * params.A_dstate_stride];
    //     smem_bc[state_idx] = B[state_idx * params.B_dstate_stride] * C[state_idx * params.C_dstate_stride];
    // }

    constexpr int kChunkSize = kNThreads * kNItems;
    for (int chunk = 0; chunk < params.n_chunks; ++chunk) {
        input_t u_vals[kNRows][kNItems], delta_vals_load[kNRows][kNItems];
        int index_vals_load[kNRows][kNItems];

        __syncthreads();
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            if constexpr (!kDirectIO) {
                if (r > 0) { __syncthreads(); }
            }
            load_input<Ktraits>(u + r * params.u_d_stride, u_vals[r], smem_load, params.seqlen - chunk * kChunkSize);
            if constexpr (!kDirectIO) { __syncthreads(); }
            load_input<Ktraits>(delta + r * params.delta_d_stride, delta_vals_load[r], smem_load, params.seqlen - chunk * kChunkSize);
            if constexpr (kUseIndex) {
                load_index<Ktraits>(index + r * params.delta_d_stride, index_vals_load[r], smem_load_index, params.seqlen - chunk * kChunkSize);
            }
        }
        if constexpr (kUseIndex) {
            index += kChunkSize;
        }
        u += kChunkSize;
        delta += kChunkSize;
    
        float delta_vals[kNRows][kNItems], delta_u_vals[kNRows][kNItems], out_vals[kNRows][kNItems];
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            #pragma unroll
            for (int i = 0; i < kNItems; ++i) {
                float u_val = float(u_vals[r][i]);
                delta_vals[r][i] = float(delta_vals_load[r][i]) + delta_bias[r];
                if (params.delta_softplus) {
                    delta_vals[r][i] = delta_vals[r][i] <= 20.f ? log1pf(expf(delta_vals[r][i])) : delta_vals[r][i];
                }
                delta_u_vals[r][i] = delta_vals[r][i] * u_val;
                out_vals[r][i] = D_val[r] * u_val;
            }
        }

        __syncthreads();
        for (int state_idx = 0; state_idx < params.dstate; ++state_idx) {
            weight_t A_val[kNRows];
            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                A_val[r] = A[state_idx * params.A_dstate_stride + r * params.A_d_stride];
                // Multiply the real part of A with LOG2E so we can use exp2f instead of expf.
                constexpr float kLog2e = M_LOG2E;
                A_val[r] *= kLog2e;
            }
            // This variable holds B * C if both B and C are constant across seqlen. If only B varies
            // across seqlen, this holds C. If only C varies across seqlen, this holds B.
            // If both B and C vary, this is unused.
            weight_t BC_val[kNRows];
            weight_t B_vals[kNItems], C_vals[kNItems];
                        if constexpr (kIsVariableB) {
                load_weight<Ktraits>(Bvar + state_idx * params.B_dstate_stride, B_vals,
                    smem_load_weight, (params.seqlen - chunk * kChunkSize) * (1));
                if constexpr (!kIsVariableC) {
                    #pragma unroll
                    for (int r = 0; r < kNRows; ++r) {
                        BC_val[r] = C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                    }
                }
            }
            if constexpr (kIsVariableC) {
                auto &smem_load_weight_C = !kIsVariableB ? smem_load_weight : smem_load_weight1;
                load_weight<Ktraits>(Cvar + state_idx * params.C_dstate_stride, C_vals,
                    smem_load_weight_C, (params.seqlen - chunk * kChunkSize) * (1 ));
                if constexpr (!kIsVariableB) {
                    #pragma unroll
                    for (int r = 0; r < kNRows; ++r) {
                        BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride];
                    }
                }
            }
            if constexpr (!kIsVariableB && !kIsVariableC) {
                #pragma unroll
                for (int r = 0; r < kNRows; ++r) {
                    BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride] * C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                }
            }

            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                if (r > 0) { __syncthreads(); }  // Scan could be using the same smem
                scan_t thread_data[kNItems];
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    thread_data[i] = make_float2(exp2f(delta_vals[r][i] * A_val[r]),
                                                 !kIsVariableB ? delta_u_vals[r][i] : B_vals[i] * delta_u_vals[r][i]);
                    
                    // Reset A bar for cumulative sequences (Real)
                    if constexpr (kUseIndex) {
                        if (index_vals_load[r][i] == 0) {
                            thread_data[i].x = 0.f;
                        }
                    }

                    if constexpr (!Ktraits::kIsEvenLen) {  // So that the last state is correct
                        if (threadIdx.x * kNItems + i >= params.seqlen - chunk * kChunkSize) {
                            thread_data[i] = make_float2(1.f, 0.f);
                        }
                    }
                }
                // Initialize running total
                scan_t running_prefix;
                    // If we use WARP_SCAN then all lane 0 of all warps (not just thread 0) needs to read
                running_prefix = chunk == 0 ? x[(r * params.n_chunks) * params.dstate + state_idx] : ( threadIdx.x % 32 == 0 ? smem_running_prefix[state_idx + r * MAX_DSTATE] : make_float2(1.f, 0.f));
                    // running_prefix = chunk > 0 && threadIdx.x == 0 ? smem_running_prefix[state_idx] : make_float2(1.f, 0.f);
                SSMScanPrefixCallbackOp<weight_t> prefix_op(running_prefix);
                typename Ktraits::BlockScanT(smem_scan).InclusiveScan(
                    thread_data, thread_data, SSMScanOp<weight_t>(), prefix_op
                );
                // There's a syncthreads in the scan op, so we don't need to sync here.
                // Unless there's only 1 warp, but then it's the same thread (0) reading and writing.
                if (threadIdx.x == 0) {
                    smem_running_prefix[state_idx] = prefix_op.running_prefix;
                    x[(r * params.n_chunks + chunk) * params.dstate + state_idx] = prefix_op.running_prefix;
                }
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    const weight_t C_val = !kIsVariableC
                        ? BC_val[r]
                        : (!kIsVariableB ? BC_val[r] * C_vals[i] : C_vals[i]);
                    out_vals[r][i] += thread_data[i].y * C_val;
                }
            }
        }
        
        input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
            + dim_id * kNRows * params.out_d_stride + chunk * kChunkSize;
        __syncthreads();
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            if constexpr (!kDirectIO) {
                if (r > 0) { __syncthreads(); }
            }
            store_output<Ktraits>(out + r * params.out_d_stride, out_vals[r], smem_store, params.seqlen - chunk * kChunkSize);
        }

        if constexpr (kHasZ) {
            input_t *z = reinterpret_cast<input_t *>(params.z_ptr) + batch_id * params.z_batch_stride
                + dim_id * kNRows * params.z_d_stride + chunk * kChunkSize;
            input_t *out_z = reinterpret_cast<input_t *>(params.out_z_ptr) + batch_id * params.out_z_batch_stride
                + dim_id * kNRows * params.out_z_d_stride + chunk * kChunkSize;
            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                input_t z_vals[kNItems];
                __syncthreads();
                load_input<Ktraits>(z + r * params.z_d_stride, z_vals, smem_load, params.seqlen - chunk * kChunkSize);
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    float z_val = z_vals[i];
                    out_vals[r][i] *= z_val / (1 + expf(-z_val));
                }
                __syncthreads();
                store_output<Ktraits>(out_z + r * params.out_z_d_stride, out_vals[r], smem_store, params.seqlen - chunk * kChunkSize);
            }
        }

        Bvar += kChunkSize * 1;
        Cvar += kChunkSize * 1;
    }
}

template<int kNThreads, int kNItems, typename input_t, typename weight_t>
void selective_scan_fwd_launch(SSMParamsBase &params, hipStream_t stream) {
    // Only kNRows == 1 is tested for now, which ofc doesn't differ from previously when we had each block
    // processing 1 row.
    constexpr int kNRows = 1;
    // kIsVariableB, kIsVariableC and kHasZ are all set to True to reduce binary size
    constexpr bool kIsVariableB = true;
    constexpr bool kIsVariableC = true;
    constexpr bool kHasZ = true;
    BOOL_SWITCH(params.seqlen % (kNThreads * kNItems) == 0, kIsEvenLen, [&] {
        BOOL_SWITCH(params.index_ptr != nullptr , kUseIndex, [&] {
            using Ktraits = Selective_Scan_fwd_kernel_traits<kNThreads, kNItems, kNRows, kIsEvenLen, kIsVariableB, kIsVariableC, kHasZ,  kUseIndex, input_t, weight_t>;
            constexpr int kSmemSize = Ktraits::kSmemSize + kNRows * MAX_DSTATE * sizeof(typename Ktraits::scan_t);
            dim3 grid(params.batch, params.dim / kNRows);
            auto kernel = &selective_scan_fwd_kernel<Ktraits>;
            if (kSmemSize >= 48 * 1024) {
                C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
            }
            kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
    });
}

template<typename input_t, typename weight_t>
void selective_scan_fwd_cuda(SSMParamsBase &params, hipStream_t stream) {

    #ifndef USE_ROCM
        if (params.seqlen <= 128) {           
            selective_scan_fwd_launch<32, 4, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 256) {
            selective_scan_fwd_launch<32, 8, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 512) {
            selective_scan_fwd_launch<32, 16, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 1024) {
            selective_scan_fwd_launch<64, 16, input_t, weight_t>(params, stream);
        } else {
            selective_scan_fwd_launch<128, 16, input_t, weight_t>(params, stream);
        }
    #else
        if (params.seqlen <= 256) {
            selective_scan_fwd_launch<64, 4, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 512) {
            selective_scan_fwd_launch<64, 8, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 1024) {
            selective_scan_fwd_launch<64, 16, input_t, weight_t>(params, stream);
        } else {
            selective_scan_fwd_launch<128, 16, input_t, weight_t>(params, stream);
        }
    #endif
}

template void selective_scan_fwd_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);

#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")

#define DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(ITYPE, NAME, ...)              \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::BFloat16) {                                 \
        using input_t = at::BFloat16;                                               \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }


template<typename input_t, typename weight_t>
void selective_scan_fwd_cuda(SSMParamsBase &params, hipStream_t stream);

void set_ssm_params_fwd(SSMParamsBase &params,
                        // sizes
                        const size_t batch,
                        const size_t dim,
                        const size_t seqlen,
                        const size_t dstate,
                        const size_t n_groups,
                        const size_t n_chunks,
                        const bool is_variable_B,
                        const bool is_variable_C,
                        // device pointers
                        const torch::Tensor u,
                        const torch::Tensor delta,
                        const torch::Tensor A,
                        const torch::Tensor B,
                        const torch::Tensor C,
                        const torch::Tensor out,
                        const torch::Tensor z,
                        const torch::Tensor out_z,
                        void* D_ptr,
                        void* delta_bias_ptr,
                        void* x_ptr,
                        bool has_z, 
                        bool delta_softplus,
                        void* index_ptr) {

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.batch = batch;
    params.dim = dim;
    params.seqlen = seqlen;
    params.dstate = dstate;
    params.n_groups = n_groups;
    params.n_chunks = n_chunks;
    params.dim_ngroups_ratio = dim / n_groups;

    params.delta_softplus = delta_softplus;

    params.is_variable_B = is_variable_B;
    params.is_variable_C = is_variable_C;

    // Set the pointers and strides.
    params.u_ptr = u.data_ptr();
    params.delta_ptr = delta.data_ptr();
    params.A_ptr = A.data_ptr();
    params.B_ptr = B.data_ptr();
    params.C_ptr = C.data_ptr();
    params.D_ptr = D_ptr;
    params.delta_bias_ptr = delta_bias_ptr;
    params.out_ptr = out.data_ptr();
    params.x_ptr = x_ptr;
    params.z_ptr = has_z ? z.data_ptr() : nullptr;
    params.out_z_ptr = has_z ? out_z.data_ptr() : nullptr;

    params.index_ptr = index_ptr;

    // All stride are in elements, not bytes.
    params.A_d_stride = A.stride(0);
    params.A_dstate_stride = A.stride(1);
    if (!is_variable_B) {
        params.B_d_stride = B.stride(0);
    } else {
        params.B_batch_stride = B.stride(0);
        params.B_group_stride = B.stride(1);
    }
    params.B_dstate_stride = !is_variable_B ? B.stride(1) : B.stride(2);
    if (!is_variable_C) {
        params.C_d_stride = C.stride(0);
    } else {
        params.C_batch_stride = C.stride(0);
        params.C_group_stride = C.stride(1);
    }
    params.C_dstate_stride = !is_variable_C ? C.stride(1) : C.stride(2);
    params.u_batch_stride = u.stride(0);
    params.u_d_stride = u.stride(1);
    params.delta_batch_stride = delta.stride(0);
    params.delta_d_stride = delta.stride(1);
    if (has_z) {
        params.z_batch_stride = z.stride(0);
        params.z_d_stride = z.stride(1);
        params.out_z_batch_stride = out_z.stride(0);
        params.out_z_d_stride = out_z.stride(1);
    }
    params.out_batch_stride = out.stride(0);
    params.out_d_stride = out.stride(1);
}

std::vector<torch::Tensor>
selective_scan_fwd(const torch::Tensor &u, const torch::Tensor &delta,
                  const torch::Tensor &A, const torch::Tensor &B, const torch::Tensor &C,
                  const c10::optional<torch::Tensor> &D_,
                  const c10::optional<torch::Tensor> &z_,
                  const c10::optional<torch::Tensor> &delta_bias_,
                  bool delta_softplus,
                  const c10::optional<torch::Tensor> &index_,
                  const c10::optional<torch::Tensor> &x) {
    auto input_type = u.scalar_type();
    auto weight_type = A.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float);

    const bool is_variable_B = B.dim() >= 3;
    const bool is_variable_C = C.dim() >= 3;

    TORCH_CHECK(delta.scalar_type() == input_type);
    TORCH_CHECK(B.scalar_type() == (!is_variable_B ? weight_type : input_type));
    TORCH_CHECK(C.scalar_type() == (!is_variable_C ? weight_type : input_type));

    TORCH_CHECK(u.is_cuda());
    TORCH_CHECK(delta.is_cuda());
    TORCH_CHECK(A.is_cuda());
    TORCH_CHECK(B.is_cuda());
    TORCH_CHECK(C.is_cuda());

    TORCH_CHECK(u.stride(-1) == 1 || u.size(-1) == 1);
    TORCH_CHECK(delta.stride(-1) == 1 || delta.size(-1) == 1);

    const auto sizes = u.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int seqlen = sizes[2];
    const int dstate = A.size(1);
    const int n_groups = is_variable_B ? B.size(1) : 1;

    TORCH_CHECK(dstate <= 256, "selective_scan only supports state dimension <= 256");

    CHECK_SHAPE(u, batch_size, dim, seqlen);
    CHECK_SHAPE(delta, batch_size, dim, seqlen);
    CHECK_SHAPE(A, dim, dstate);
    TORCH_CHECK(is_variable_B, "is_variable_B = False is disabled in favor of reduced binary size")
    CHECK_SHAPE(B, batch_size, n_groups, dstate, seqlen );
    TORCH_CHECK(B.stride(-1) == 1 || B.size(-1) == 1);

    TORCH_CHECK(is_variable_C, "is_variable_C = False is disabled in favor of reduced binary size")
    CHECK_SHAPE(C, batch_size, n_groups, dstate, seqlen);
    TORCH_CHECK(C.stride(-1) == 1 || C.size(-1) == 1);

    if (D_.has_value()) {
        auto D = D_.value();
        TORCH_CHECK(D.scalar_type() == at::ScalarType::Float);
        TORCH_CHECK(D.is_cuda());
        TORCH_CHECK(D.stride(-1) == 1 || D.size(-1) == 1);
        CHECK_SHAPE(D, dim);
    }

    if (delta_bias_.has_value()) {
        auto delta_bias = delta_bias_.value();
        TORCH_CHECK(delta_bias.scalar_type() == at::ScalarType::Float);
        TORCH_CHECK(delta_bias.is_cuda());
        TORCH_CHECK(delta_bias.stride(-1) == 1 || delta_bias.size(-1) == 1);
        CHECK_SHAPE(delta_bias, dim);
    }
    if (index_.has_value()) {
        auto index = index_.value();
        TORCH_CHECK(index.scalar_type() == at::ScalarType::Int);
        TORCH_CHECK(index.is_cuda());
        CHECK_SHAPE(index, batch_size, seqlen);
    }

    at::Tensor z, out_z;
    const bool has_z = z_.has_value();
    TORCH_CHECK(has_z, "has_z = False is disabled in favor of reduced binary size")
    z = z_.value();
    TORCH_CHECK(z.scalar_type() == input_type);
    TORCH_CHECK(z.is_cuda());
    TORCH_CHECK(z.stride(-1) == 1 || z.size(-1) == 1);
    CHECK_SHAPE(z, batch_size, dim, seqlen);
    out_z = torch::empty_like(z);

    const int n_chunks = (seqlen + 2048 - 1) / 2048;
    // const int n_chunks = (seqlen + 1024 - 1) / 1024;
    // at::Tensor out = torch::empty_like(u);
    // Right now u has BHL layout and delta has HBL layout, and we want out to have HBL layout
    at::Tensor out = torch::empty_like(delta);
    if (x.has_value()){
        auto _x = x.value();
        TORCH_CHECK(_x.scalar_type() == weight_type);
        TORCH_CHECK(_x.is_cuda());
        TORCH_CHECK(_x.stride(-1) == 1);
        CHECK_SHAPE(_x, batch_size, dim, n_chunks, dstate * 2);
    }

    SSMParamsBase params;
    set_ssm_params_fwd(params, batch_size, dim, seqlen, dstate, n_groups, n_chunks, is_variable_B, is_variable_C,
                       u, delta, A, B, C, out, z, out_z,
                       D_.has_value() ? D_.value().data_ptr() : nullptr,
                       delta_bias_.has_value() ? delta_bias_.value().data_ptr() : nullptr,
                       x.value().data_ptr(),
                       has_z,
                       delta_softplus,
                       index_.has_value() ? index_.value().data_ptr() : nullptr);

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)u.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(u.scalar_type(), "selective_scan_fwd", [&] {
        selective_scan_fwd_cuda<input_t, weight_t>(params, stream);
    });
    std::vector<at::Tensor> result = {out};
    if (has_z) { result.push_back(out_z); }
    return result;
}

