#include "hip/hip_runtime.h"
// clang-format off
// adapted from https://github.com/state-spaces/mamba/blob/main/csrc/selective_scan/selective_scan_fwd_kernel.cuh
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "selective_scan.h"

#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#ifndef USE_ROCM
    #include <cub/block/block_load.cuh>
    #include <cub/block/block_store.cuh>
    #include <cub/block/block_scan.cuh>
#else
    #include <hipcub/hipcub.hpp>
    namespace cub = hipcub;
#endif

#include "selective_scan.h"
#include "static_switch.h"

template<int kNThreads_, int kNItems_, int kNRows_, bool kIsEvenLen_,
         bool kIsVariableB_, bool kIsVariableC_,
         bool kHasZ_, bool kVarlen_, typename input_t_, typename weight_t_>
struct Selective_Scan_fwd_kernel_traits {
    static_assert(kNItems_ % 4 == 0);
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    // Setting MinBlocksPerMP to be 3 (instead of 2) for 128 threads improves occupancy.
    static constexpr int kMinBlocks = kNThreads < 128 ? 5 : 3;
    static constexpr int kNItems = kNItems_;
    static constexpr int kNRows = kNRows_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : constexpr_min(8, kNItems);
    static_assert(kNItems % kNElts == 0);
    static constexpr int kNLoads = kNItems / kNElts;
    static constexpr bool kIsEvenLen = kVarlen_ ? false : kIsEvenLen_;
    static constexpr bool kIsVariableB = kIsVariableB_;
    static constexpr bool kIsVariableC = kIsVariableC_;
    static constexpr bool kHasZ = kHasZ_;
    static constexpr bool kVarlen = kVarlen_;

    static constexpr bool kDirectIO = kVarlen_ ? false : kIsEvenLen && kNLoads == 1;
    static constexpr int kNLoadsIndex = kNItems / 4;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using scan_t = float2;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockLoadWeightT = hipcub::BlockLoad<input_t, kNThreads, kNItems , hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadWeightVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads ,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE  : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_STORE_WARP_TRANSPOSE : hipcub::BLOCK_STORE_DIRECT>;
    // using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_RAKING_MEMOIZE>;
    // using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_RAKING>;
    using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    static constexpr int kSmemIOSize = custom_max({sizeof(typename BlockLoadT::TempStorage),
                                                 sizeof(typename BlockLoadVecT::TempStorage),
                                                 (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightT::TempStorage),
                                                 (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightVecT::TempStorage),
                                                 sizeof(typename BlockStoreT::TempStorage),
                                                 sizeof(typename BlockStoreVecT::TempStorage)});
    static constexpr int kSmemSize = kSmemIOSize + sizeof(typename BlockScanT::TempStorage);
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads, Ktraits::kMinBlocks)
void selective_scan_fwd_kernel(SSMParamsBase params) {
    constexpr bool kIsVariableB = Ktraits::kIsVariableB;
    constexpr bool kIsVariableC = Ktraits::kIsVariableC;
    constexpr bool kHasZ = Ktraits::kHasZ;
    constexpr bool kVarlen = Ktraits::kVarlen;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNItems = Ktraits::kNItems;
    constexpr int kNRows = Ktraits::kNRows;
    constexpr bool kDirectIO = Ktraits::kDirectIO;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;
    using scan_t = typename Ktraits::scan_t;

    // Shared memory.
    extern __shared__ char smem_[];
    // cast to lvalue reference of expected type
    // char *smem_loadstorescan = smem_ + 2 * MAX_DSTATE * sizeof(weight_t);
    // auto& smem_load = reinterpret_cast<typename BlockLoadT::TempStorage&>(smem_ + 2 * MAX_DSTATE * sizeof(weight_t));
    // auto& smem_load = reinterpret_cast<typename BlockLoadT::TempStorage&>(smem_loadstorescan);
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_weight = reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage&>(smem_);
    auto& smem_load_weight1 = *reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage*>(smem_ + sizeof(typename Ktraits::BlockLoadWeightT::TempStorage));
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_scan = *reinterpret_cast<typename Ktraits::BlockScanT::TempStorage*>(smem_ + Ktraits::kSmemIOSize);
    // weight_t *smem_a = reinterpret_cast<weight_t *>(smem_ + smem_loadstorescan_size);
    // weight_t *smem_bc = reinterpret_cast<weight_t *>(smem_a + MAX_DSTATE);
    scan_t *smem_running_prefix = reinterpret_cast<scan_t *>(smem_ + Ktraits::kSmemSize);

    const int batch_id = blockIdx.x;
    const int dim_id = blockIdx.y;
    const int group_id = dim_id / (params.dim_ngroups_ratio);
    int seqlen = params.seqlen;
    int sequence_start_index = batch_id;
    if constexpr (kVarlen){
        int *query_start_loc = reinterpret_cast<int *>(params.query_start_loc_ptr);
        sequence_start_index = query_start_loc[batch_id];
        seqlen = query_start_loc[batch_id + 1] - sequence_start_index;
    }
    const bool has_initial_state = params.has_initial_state_ptr == nullptr ? false
        : reinterpret_cast<bool *>(params.has_initial_state_ptr)[batch_id];

    const int* cache_indices = params.cache_indices_ptr == nullptr ? nullptr
        : reinterpret_cast<int *>(params.cache_indices_ptr);
    const int cache_index = cache_indices == nullptr ? batch_id : cache_indices[batch_id];
    // cache_index == params.pad_slot_id is defined as padding, so we exit early
    if (cache_index == params.pad_slot_id){
        return;
    }
    input_t *u = reinterpret_cast<input_t *>(params.u_ptr) + sequence_start_index * params.u_batch_stride
        + dim_id * kNRows * params.u_d_stride;
    input_t *delta = reinterpret_cast<input_t *>(params.delta_ptr) + sequence_start_index * params.delta_batch_stride
        + dim_id * kNRows * params.delta_d_stride;
    weight_t *A = reinterpret_cast<weight_t *>(params.A_ptr) + dim_id * kNRows * params.A_d_stride;
    weight_t *B = reinterpret_cast<weight_t *>(params.B_ptr) + dim_id * kNRows * params.B_d_stride;
    input_t *Bvar = reinterpret_cast<input_t *>(params.B_ptr) + sequence_start_index * params.B_batch_stride + group_id * params.B_group_stride;
    weight_t *C = reinterpret_cast<weight_t *>(params.C_ptr) + dim_id * kNRows * params.C_d_stride;
    input_t *Cvar = reinterpret_cast<input_t *>(params.C_ptr) + sequence_start_index * params.C_batch_stride + group_id * params.C_group_stride;
    input_t *ssm_states = reinterpret_cast<input_t *>(params.ssm_states_ptr) + (cache_index * params.dim + dim_id * kNRows) * params.dstate;

    float D_val[kNRows] = {0};
    if (params.D_ptr != nullptr) {
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            D_val[r] = reinterpret_cast<float *>(params.D_ptr)[dim_id * kNRows + r];
        }
    }
    float delta_bias[kNRows] = {0};
    if (params.delta_bias_ptr != nullptr) {
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            delta_bias[r] = reinterpret_cast<float *>(params.delta_bias_ptr)[dim_id * kNRows + r];
        }
    }


    // for (int state_idx = threadIdx.x; state_idx < params.dstate; state_idx += blockDim.x) {
    //     smem_a[state_idx] = A[state_idx * params.A_dstate_stride];
    //     smem_bc[state_idx] = B[state_idx * params.B_dstate_stride] * C[state_idx * params.C_dstate_stride];
    // }

    constexpr int kChunkSize = kNThreads * kNItems;
    const int n_chunks = (seqlen + 2048 - 1) / 2048;
    for (int chunk = 0; chunk < n_chunks; ++chunk) {
        input_t u_vals[kNRows][kNItems], delta_vals_load[kNRows][kNItems];

        __syncthreads();
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            if constexpr (!kDirectIO) {
                if (r > 0) { __syncthreads(); }
            }
            load_input<Ktraits>(u + r * params.u_d_stride, u_vals[r], smem_load, seqlen - chunk * kChunkSize);
            if constexpr (!kDirectIO) { __syncthreads(); }
            load_input<Ktraits>(delta + r * params.delta_d_stride, delta_vals_load[r], smem_load, seqlen - chunk * kChunkSize);
        }
        u += kChunkSize;
        delta += kChunkSize;
    
        float delta_vals[kNRows][kNItems], delta_u_vals[kNRows][kNItems], out_vals[kNRows][kNItems];
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            #pragma unroll
            for (int i = 0; i < kNItems; ++i) {
                float u_val = float(u_vals[r][i]);
                delta_vals[r][i] = float(delta_vals_load[r][i]) + delta_bias[r];
                if (params.delta_softplus) {
                    delta_vals[r][i] = delta_vals[r][i] <= 20.f ? log1pf(expf(delta_vals[r][i])) : delta_vals[r][i];
                }
                delta_u_vals[r][i] = delta_vals[r][i] * u_val;
                out_vals[r][i] = D_val[r] * u_val;
            }
        }

        __syncthreads();
        for (int state_idx = 0; state_idx < params.dstate; ++state_idx) {
            weight_t A_val[kNRows];
            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                A_val[r] = A[state_idx * params.A_dstate_stride + r * params.A_d_stride];
                // Multiply the real part of A with LOG2E so we can use exp2f instead of expf.
                constexpr float kLog2e = M_LOG2E;
                A_val[r] *= kLog2e;
            }
            // This variable holds B * C if both B and C are constant across seqlen. If only B varies
            // across seqlen, this holds C. If only C varies across seqlen, this holds B.
            // If both B and C vary, this is unused.
            weight_t BC_val[kNRows];
            weight_t B_vals[kNItems], C_vals[kNItems];
            if constexpr (kIsVariableB) {
                load_weight<Ktraits>(Bvar + state_idx * params.B_dstate_stride, B_vals,
                    smem_load_weight, (seqlen - chunk * kChunkSize) * (1));
                if constexpr (!kIsVariableC) {
                    #pragma unroll
                    for (int r = 0; r < kNRows; ++r) {
                        BC_val[r] = C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                    }
                }
            }
            if constexpr (kIsVariableC) {
                auto &smem_load_weight_C = !kIsVariableB ? smem_load_weight : smem_load_weight1;
                load_weight<Ktraits>(Cvar + state_idx * params.C_dstate_stride, C_vals,
                    smem_load_weight_C, (seqlen - chunk * kChunkSize) * (1 ));
                if constexpr (!kIsVariableB) {
                    #pragma unroll
                    for (int r = 0; r < kNRows; ++r) {
                        BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride];
                    }
                }
            }
            if constexpr (!kIsVariableB && !kIsVariableC) {
                #pragma unroll
                for (int r = 0; r < kNRows; ++r) {
                    BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride] * C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                }
            }

            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                if (r > 0) { __syncthreads(); }  // Scan could be using the same smem
                scan_t thread_data[kNItems];
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    thread_data[i] = make_float2(exp2f(delta_vals[r][i] * A_val[r]),
                                                 !kIsVariableB ? delta_u_vals[r][i] : B_vals[i] * delta_u_vals[r][i]);
                    
                    if (seqlen % (kNItems * kNThreads) != 0) {  // So that the last state is correct
                        if (threadIdx.x * kNItems + i >= seqlen - chunk * kChunkSize) {
                            thread_data[i] = make_float2(1.f, 0.f);
                        }
                    }
                }
                // Initialize running total

                scan_t running_prefix = chunk > 0 ? smem_running_prefix[state_idx + r * MAX_DSTATE] : make_float2(1.0, has_initial_state ? float(ssm_states[state_idx]): 0.0);

                SSMScanPrefixCallbackOp<weight_t> prefix_op(running_prefix);
                typename Ktraits::BlockScanT(smem_scan).InclusiveScan(
                    thread_data, thread_data, SSMScanOp<weight_t>(), prefix_op
                );
                // There's a syncthreads in the scan op, so we don't need to sync here.
                // Unless there's only 1 warp, but then it's the same thread (0) reading and writing.
                if (threadIdx.x == 0) {
                    smem_running_prefix[state_idx] = prefix_op.running_prefix;
                    if (chunk == n_chunks - 1) {
                        ssm_states[state_idx] = input_t(prefix_op.running_prefix.y);
                    }
                }
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    const weight_t C_val = !kIsVariableC
                        ? BC_val[r]
                        : (!kIsVariableB ? BC_val[r] * C_vals[i] : C_vals[i]);
                    out_vals[r][i] += thread_data[i].y * C_val;
                }
            }
        }
        
        input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + sequence_start_index * params.out_batch_stride
            + dim_id * kNRows * params.out_d_stride + chunk * kChunkSize;
        __syncthreads();
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            if constexpr (!kDirectIO) {
                if (r > 0) { __syncthreads(); }
            }
            store_output<Ktraits>(out + r * params.out_d_stride, out_vals[r], smem_store, seqlen - chunk * kChunkSize);
        }

        if constexpr (kHasZ) {
            input_t *z = reinterpret_cast<input_t *>(params.z_ptr) + sequence_start_index * params.z_batch_stride
                + dim_id * kNRows * params.z_d_stride + chunk * kChunkSize;
            input_t *out_z = reinterpret_cast<input_t *>(params.out_z_ptr) + sequence_start_index * params.out_z_batch_stride
                + dim_id * kNRows * params.out_z_d_stride + chunk * kChunkSize;
            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                input_t z_vals[kNItems];
                __syncthreads();
                load_input<Ktraits>(z + r * params.z_d_stride, z_vals, smem_load, seqlen - chunk * kChunkSize);
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    float z_val = z_vals[i];
                    out_vals[r][i] *= z_val / (1 + expf(-z_val));
                }
                __syncthreads();
                store_output<Ktraits>(out_z + r * params.out_z_d_stride, out_vals[r], smem_store, seqlen - chunk * kChunkSize);
            }
        }

        Bvar += kChunkSize * 1;
        Cvar += kChunkSize * 1;
    }
}

template<int kNThreads, int kNItems, typename input_t, typename weight_t>
void selective_scan_fwd_launch(SSMParamsBase &params, hipStream_t stream) {
    // Only kNRows == 1 is tested for now, which ofc doesn't differ from previously when we had each block
    // processing 1 row.
    constexpr int kNRows = 1;
    // kIsVariableB, kIsVariableC and kHasZ are all set to True to reduce binary size
    constexpr bool kIsVariableB = true;
    constexpr bool kIsVariableC = true;
    constexpr bool kHasZ = true;
    BOOL_SWITCH(params.seqlen % (kNThreads * kNItems) == 0, kIsEvenLen, [&] {
        BOOL_SWITCH(params.query_start_loc_ptr != nullptr , kVarlen, [&] {
            using Ktraits = Selective_Scan_fwd_kernel_traits<kNThreads, kNItems, kNRows, kIsEvenLen, kIsVariableB, kIsVariableC, kHasZ,  kVarlen, input_t, weight_t>;
            constexpr int kSmemSize = Ktraits::kSmemSize + kNRows * MAX_DSTATE * sizeof(typename Ktraits::scan_t);
            dim3 grid(params.batch, params.dim / kNRows);
            auto kernel = &selective_scan_fwd_kernel<Ktraits>;
            if (kSmemSize >= 48 * 1024) {
                C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
            }
            kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
    });
}

template<typename input_t, typename weight_t>
void selective_scan_fwd_cuda(SSMParamsBase &params, hipStream_t stream) {

    #ifndef USE_ROCM
        if (params.seqlen <= 128) {           
            selective_scan_fwd_launch<32, 4, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 256) {
            selective_scan_fwd_launch<32, 8, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 512) {
            selective_scan_fwd_launch<32, 16, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 1024) {
            selective_scan_fwd_launch<64, 16, input_t, weight_t>(params, stream);
        } else {
            selective_scan_fwd_launch<128, 16, input_t, weight_t>(params, stream);
        }
    #else
        if (params.seqlen <= 256) {
            selective_scan_fwd_launch<64, 4, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 512) {
            selective_scan_fwd_launch<64, 8, input_t, weight_t>(params, stream);
        } else if (params.seqlen <= 1024) {
            selective_scan_fwd_launch<64, 16, input_t, weight_t>(params, stream);
        } else {
            selective_scan_fwd_launch<128, 16, input_t, weight_t>(params, stream);
        }
    #endif
}

template void selective_scan_fwd_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);

#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")

#define DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(ITYPE, NAME, ...)              \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::BFloat16) {                                 \
        using input_t = at::BFloat16;                                               \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        using weight_t = float;                                                     \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }


template<typename input_t, typename weight_t>
void selective_scan_fwd_cuda(SSMParamsBase &params, hipStream_t stream);

void set_ssm_params_fwd(SSMParamsBase &params,
                        // sizes
                        const size_t batch,
                        const size_t dim,
                        const size_t seqlen,
                        const size_t dstate,
                        const size_t n_groups,
                        const bool is_variable_B,
                        const bool is_variable_C,
                        // device pointers
                        const torch::Tensor u,
                        const torch::Tensor delta,
                        const torch::Tensor A,
                        const torch::Tensor B,
                        const torch::Tensor C,
                        const torch::Tensor out,
                        const torch::Tensor z,
                        const torch::Tensor out_z,
                        const c10::optional<at::Tensor>& D,
                        const c10::optional<at::Tensor>& delta_bias,
                        const torch::Tensor ssm_states,
                        bool has_z, 
                        bool delta_softplus,
                        const c10::optional<at::Tensor>& query_start_loc,
                        const c10::optional<at::Tensor>& cache_indices,
                        const c10::optional<at::Tensor>& has_initial_state,
                        bool varlen,
                        int64_t pad_slot_id) {

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.batch = batch;
    params.dim = dim;
    params.seqlen = seqlen;
    params.dstate = dstate;
    params.n_groups = n_groups;
    params.dim_ngroups_ratio = dim / n_groups;
    params.pad_slot_id = pad_slot_id;

    params.delta_softplus = delta_softplus;

    params.is_variable_B = is_variable_B;
    params.is_variable_C = is_variable_C;

    // Set the pointers and strides.
    params.u_ptr = u.data_ptr();
    params.delta_ptr = delta.data_ptr();
    params.A_ptr = A.data_ptr();
    params.B_ptr = B.data_ptr();
    params.C_ptr = C.data_ptr();
    params.D_ptr = D.has_value() ? D.value().data_ptr() : nullptr;
    params.delta_bias_ptr = delta_bias.has_value() ? delta_bias.value().data_ptr() : nullptr;
    params.out_ptr = out.data_ptr();
    params.ssm_states_ptr = ssm_states.data_ptr();
    params.z_ptr = has_z ? z.data_ptr() : nullptr;
    params.out_z_ptr = has_z ? out_z.data_ptr() : nullptr;
    params.query_start_loc_ptr = query_start_loc.has_value() ? query_start_loc.value().data_ptr() : nullptr;
    params.cache_indices_ptr = cache_indices.has_value() ? cache_indices.value().data_ptr() : nullptr;
    params.has_initial_state_ptr = has_initial_state.has_value() ? has_initial_state.value().data_ptr() : nullptr;


    // All stride are in elements, not bytes.
    params.A_d_stride = A.stride(0);
    params.A_dstate_stride = A.stride(1);

    if (varlen){
        params.B_batch_stride = B.stride(2);
        params.B_group_stride = B.stride(0);
        params.B_dstate_stride = B.stride(1);
        params.C_batch_stride = C.stride(2);
        params.C_group_stride = C.stride(0);
        params.C_dstate_stride = C.stride(1);

        params.u_batch_stride = u.stride(1);
        params.u_d_stride = u.stride(0);
        params.delta_batch_stride = delta.stride(1);
        params.delta_d_stride = delta.stride(0);
        if (has_z) {
            params.z_batch_stride = z.stride(1);
            params.z_d_stride = z.stride(0);
            params.out_z_batch_stride = out_z.stride(1);
            params.out_z_d_stride = out_z.stride(0);
        }
        params.out_batch_stride = out.stride(1);
        params.out_d_stride = out.stride(0);

    }
    else{
        if (!is_variable_B) {
            params.B_d_stride = B.stride(0);
        } else {
            params.B_batch_stride = B.stride(0);
            params.B_group_stride = B.stride(1);
        }
        params.B_dstate_stride = !is_variable_B ? B.stride(1) : B.stride(2);
        if (!is_variable_C) {
            params.C_d_stride = C.stride(0);
        } else {
            params.C_batch_stride = C.stride(0);
            params.C_group_stride = C.stride(1);
        }
        params.C_dstate_stride = !is_variable_C ? C.stride(1) : C.stride(2);
        params.u_batch_stride = u.stride(0);
        params.u_d_stride = u.stride(1);
        params.delta_batch_stride = delta.stride(0);
        params.delta_d_stride = delta.stride(1);
        if (has_z) {
            params.z_batch_stride = z.stride(0);
            params.z_d_stride = z.stride(1);
            params.out_z_batch_stride = out_z.stride(0);
            params.out_z_d_stride = out_z.stride(1);
        }
        params.out_batch_stride = out.stride(0);
        params.out_d_stride = out.stride(1);
    }
}

void selective_scan_fwd(const torch::Tensor &u, const torch::Tensor &delta,
                  const torch::Tensor &A, const torch::Tensor &B, const torch::Tensor &C,
                  const c10::optional<torch::Tensor> &D_,
                  const c10::optional<torch::Tensor> &z_,
                  const c10::optional<torch::Tensor> &delta_bias_,
                  bool delta_softplus,
                  const c10::optional<torch::Tensor> &query_start_loc,
                  const c10::optional<torch::Tensor> &cache_indices,
                  const c10::optional<torch::Tensor> &has_initial_state,
                  const torch::Tensor &ssm_states,
                  // used to identify padding entries if cache_indices provided
                  // in case of padding, the kernel will return early
                  int64_t pad_slot_id) {
    auto input_type = u.scalar_type();
    auto weight_type = A.scalar_type();
    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half || input_type == at::ScalarType::BFloat16);
    TORCH_CHECK(weight_type == at::ScalarType::Float);

    const bool is_variable_B = B.dim() >= 3;
    const bool is_variable_C = C.dim() >= 3;

    TORCH_CHECK(delta.scalar_type() == input_type);
    TORCH_CHECK(B.scalar_type() == (!is_variable_B ? weight_type : input_type));
    TORCH_CHECK(C.scalar_type() == (!is_variable_C ? weight_type : input_type));

    TORCH_CHECK(u.is_cuda());
    TORCH_CHECK(delta.is_cuda());
    TORCH_CHECK(A.is_cuda());
    TORCH_CHECK(B.is_cuda());
    TORCH_CHECK(C.is_cuda());

    TORCH_CHECK(u.stride(-1) == 1 || u.size(-1) == 1);
    TORCH_CHECK(delta.stride(-1) == 1 || delta.size(-1) == 1);

    const auto sizes = u.sizes();
    const bool varlen = query_start_loc.has_value();
    const int batch_size = varlen ? query_start_loc.value().sizes()[0] - 1 : sizes[0];
    const int dim = varlen ? sizes[0] : sizes[1];
    const int seqlen = varlen ? sizes[1] : sizes[2];
    const int dstate = A.size(1);
    const int n_groups = varlen ? B.size(0) : B.size(1);

    TORCH_CHECK(dstate <= 256, "selective_scan only supports state dimension <= 256");

    if (varlen) {
        CHECK_SHAPE(u, dim, seqlen);
        CHECK_SHAPE(delta, dim, seqlen);
    } else {
        CHECK_SHAPE(u, batch_size, dim, seqlen);
        CHECK_SHAPE(delta, batch_size, dim, seqlen);
    }
    CHECK_SHAPE(A, dim, dstate);
    TORCH_CHECK(is_variable_B, "is_variable_B = False is disabled in favor of reduced binary size")
    if (varlen) {
        CHECK_SHAPE(B, n_groups, dstate, seqlen);
    } else {
        CHECK_SHAPE(B, batch_size, n_groups, dstate, seqlen); 
    }
    TORCH_CHECK(B.stride(-1) == 1 || B.size(-1) == 1);

    TORCH_CHECK(is_variable_C, "is_variable_C = False is disabled in favor of reduced binary size")
    if (varlen) {
        CHECK_SHAPE(C, n_groups, dstate, seqlen);
    } else {
        CHECK_SHAPE(C, batch_size, n_groups, dstate, seqlen); 
    }
    TORCH_CHECK(C.stride(-1) == 1 || C.size(-1) == 1);

    if (D_.has_value()) {
        auto D = D_.value();
        TORCH_CHECK(D.scalar_type() == at::ScalarType::Float);
        TORCH_CHECK(D.is_cuda());
        TORCH_CHECK(D.stride(-1) == 1 || D.size(-1) == 1);
        CHECK_SHAPE(D, dim);
    }

    if (delta_bias_.has_value()) {
        auto delta_bias = delta_bias_.value();
        TORCH_CHECK(delta_bias.scalar_type() == at::ScalarType::Float);
        TORCH_CHECK(delta_bias.is_cuda());
        TORCH_CHECK(delta_bias.stride(-1) == 1 || delta_bias.size(-1) == 1);
        CHECK_SHAPE(delta_bias, dim);
    }


    if (has_initial_state.has_value()) {
        auto has_initial_state_ = has_initial_state.value();
        TORCH_CHECK(has_initial_state_.scalar_type() == at::ScalarType::Bool);
        TORCH_CHECK(has_initial_state_.is_cuda());
        CHECK_SHAPE(has_initial_state_, batch_size);
    }


    if (query_start_loc.has_value()) {
        auto query_start_loc_ = query_start_loc.value();
        TORCH_CHECK(query_start_loc_.scalar_type() == at::ScalarType::Int);
        TORCH_CHECK(query_start_loc_.is_cuda());
    }


    if (cache_indices.has_value()) {
        auto cache_indices_ = cache_indices.value();
        TORCH_CHECK(cache_indices_.scalar_type() == at::ScalarType::Int);
        TORCH_CHECK(cache_indices_.is_cuda());
        CHECK_SHAPE(cache_indices_, batch_size);
    }
   

    at::Tensor z, out_z;
    const bool has_z = z_.has_value();
    TORCH_CHECK(has_z, "has_z = False is disabled in favor of reduced binary size")
    z = z_.value();
    TORCH_CHECK(z.scalar_type() == input_type);
    TORCH_CHECK(z.is_cuda());
    TORCH_CHECK(z.stride(-1) == 1 || z.size(-1) == 1);
    if (varlen){
        CHECK_SHAPE(z, dim, seqlen);
    } else {
        CHECK_SHAPE(z, batch_size, dim, seqlen);
    }

    out_z = z;

    // Right now u has BHL layout and delta has HBL layout, and we want out to have HBL layout
    at::Tensor out = delta;
    TORCH_CHECK(ssm_states.scalar_type() == input_type);
    TORCH_CHECK(ssm_states.is_cuda());
    TORCH_CHECK(ssm_states.stride(-1) == 1);

    SSMParamsBase params;
    set_ssm_params_fwd(params, batch_size, dim, seqlen, dstate, n_groups, is_variable_B, is_variable_C,
                       u, delta, A, B, C, out, z, out_z,
                       D_,
                       delta_bias_,
                       ssm_states,
                       has_z,
                       delta_softplus,
                       query_start_loc,
                       cache_indices,
                       has_initial_state,
                       varlen,
                       pad_slot_id
                       );

    
    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)u.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    DISPATCH_WTYPE_ITYPE_FLOAT_AND_HALF_AND_BF16(u.scalar_type(), "selective_scan_fwd", [&] {
        selective_scan_fwd_cuda<input_t, weight_t>(params, stream);
    });
}

