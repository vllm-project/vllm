#include "hip/hip_runtime.h"
#include "dispatch_utils.h"

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/hipcub.hpp>
#endif

namespace vllm {

template <typename scalar_t>
__global__ void apply_repetition_penalties_kernel(
    scalar_t* __restrict__ logits,         // [num_seqs, vocab_size]
    const bool* __restrict__ prompt_mask,  // [num_seqs, vocab_size]
    const bool* __restrict__ output_mask,  // [num_seqs, vocab_size]
    const scalar_t* __restrict__ repetition_penalties,  // [num_seqs]
    const int num_seqs, const int vocab_size, const int tile_size) {
  // Each block handles one sequence and a tile of vocab
  const int seq_idx = blockIdx.x;
  if (seq_idx >= num_seqs) return;

  const int tile_start = blockIdx.y * tile_size;
  const int tile_end = min(tile_start + tile_size, vocab_size);

  // Load repetition penalty for this sequence
  const scalar_t penalty = repetition_penalties[seq_idx];

  // Each thread processes multiple vocab items within the tile
  for (int vocab_idx = tile_start + threadIdx.x; vocab_idx < tile_end;
       vocab_idx += blockDim.x) {
    const int64_t idx = static_cast<int64_t>(seq_idx) * vocab_size + vocab_idx;
    const bool is_repeated = prompt_mask[idx] || output_mask[idx];
    if (is_repeated) {
      scalar_t logit = logits[idx];
      if (logit > 0) {
        logits[idx] = logit / penalty;
      } else {
        logits[idx] = logit * penalty;
      }
    }
  }
}

static inline __device__ uint16_t extractBinIdx(float x) {
  union {
    __half h;
    uint16_t u16;
  } tmp;
  tmp.h = __float2half_rn(x);
  tmp.u16 = (x < 0.f) ? (~tmp.u16 & 0xffff) : (tmp.u16 | 0x8000);
  return 511 - (tmp.u16 >> 7);
}

template <int kNumThreadsPerBlock = 512>
static __global__ void topKPerRow(const float* logits, const int* rowStarts,
                                  const int* rowEnds, int* outIndices,
                                  int stride0, int stride1) {
  // The number of bins in the histogram.
  static constexpr int kNumBins = 512;

  // The top-k width.
  static constexpr int kTopK = 2048;
  // The number of elements per thread for the final top-k sort.
  static constexpr int kNumTopKItemsPerThread = kTopK / kNumThreadsPerBlock;
  // The class to sort the elements during the final top-k sort.
  using TopKSort = hipcub::BlockRadixSort<float, kNumThreadsPerBlock,
                                       kNumTopKItemsPerThread, int>;

  // The number of slots for the final pass.
  static constexpr int kNumFinalItems = 3072;
  // The number of elements per thread for the final sort.
  static constexpr int kNumFinalItemsPerThread =
      kNumFinalItems / kNumThreadsPerBlock;
  // The class to sort the elements during the final pass.
  using FinalSort = hipcub::BlockRadixSort<float, kNumThreadsPerBlock,
                                        kNumFinalItemsPerThread, int>;

  // The class to compute the inclusive prefix-sum over the histogram.
  using Scan = hipcub::BlockScan<int, kNumThreadsPerBlock>;

  // Shared memory to compute the block scan.
  __shared__ typename Scan::TempStorage smemScan;

  // The structure to store the final items (for the final pass).
  struct FinalItems {
    // Shared memory to store the indices for the final pass.
    int indices[kNumFinalItems];
    // Shared memory to store the logits for the final pass.
    float logits[kNumFinalItems];
  };

  // Shared memory to compute the block sort.
  __shared__ union {
    FinalItems items;
    typename FinalSort::TempStorage finalSort;
    typename TopKSort::TempStorage topKSort;
  } smemFinal;

  // Shared memory to store the histogram.
  __shared__ int smemHistogram[kNumBins];
  // Shared memory to store the selected indices.
  __shared__ int smemIndices[kTopK];
  // Shared memory to store the threshold bin.
  __shared__ int smemThresholdBinIdx[1];
  // Shared memory counter to register the candidates for the final phase.
  __shared__ int smemFinalDstIdx[1];

  // The row computed by this block.
  int rowIdx = blockIdx.x;
  // The range of logits within the row.
  int rowStart = rowStarts[rowIdx], rowEnd = rowEnds[rowIdx];
  // The length of the row.
  int rowLen = rowEnd - rowStart;

  // Shortcut if the length of the row is smaller than Top-K. Indices are not
  // sorted by their corresponding logit.
  if (rowLen <= kTopK) {
    for (int rowIt = threadIdx.x; rowIt < rowLen;
         rowIt += kNumThreadsPerBlock) {
      int idx = rowStart + rowIt;
      outIndices[rowIdx * kTopK + rowIt] = idx - rowStart;
    }
    for (int rowIt = rowLen + threadIdx.x; rowIt < kTopK;
         rowIt += kNumThreadsPerBlock) {
      outIndices[rowIdx * kTopK + rowIt] = -1;
    }
    return;
  }

  // Clear the histogram.
  if (threadIdx.x < kNumBins) {
    smemHistogram[threadIdx.x] = 0;
  }

  // Make sure the histogram is ready.
  __syncthreads();

  // Fetch elements one-by-one.
  for (int rowIt = rowStart + threadIdx.x; rowIt < rowEnd;
       rowIt += kNumThreadsPerBlock) {
    uint16_t idx = extractBinIdx(logits[rowIdx * stride0 + rowIt * stride1]);
    atomicAdd(&smemHistogram[idx], 1);
  }

  // Make sure the histogram is ready.
  __syncthreads();

  // Read the values from SMEM.
  int binCount{0};
  if (threadIdx.x < kNumBins) {
    binCount = smemHistogram[threadIdx.x];
  }

  // Make sure each thread has read its value.
  __syncthreads();

  // Compute the prefix sum.
  int prefixSum{0}, totalSum{0};
  Scan(smemScan).ExclusiveSum(binCount, prefixSum, totalSum);

  // Update the histogram with the prefix sums.
  if (threadIdx.x < kNumBins) {
    smemHistogram[threadIdx.x] = prefixSum;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // Find the last valid bin.
  if (threadIdx.x < kNumBins) {
    int nextPrefixSum =
        threadIdx.x == kNumBins - 1 ? totalSum : smemHistogram[threadIdx.x + 1];
    if (prefixSum < kTopK && nextPrefixSum >= kTopK) {
      smemThresholdBinIdx[0] = threadIdx.x;
    }
  }

  // Clear the counter to store the items for the final phase.
  if (threadIdx.x == 0) {
    smemFinalDstIdx[0] = 0;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The threshold bin.
  int thresholdBinIdx = smemThresholdBinIdx[0];

  // Fetch elements one-by-one and populate the shared memory buffers.
  for (int rowIt = rowStart + threadIdx.x; rowIt < rowEnd;
       rowIt += kNumThreadsPerBlock) {
    float logit = logits[rowIdx * stride0 + rowIt * stride1];
    uint16_t idx = extractBinIdx(logit);
    if (idx < thresholdBinIdx) {
      int dstIdx = atomicAdd(&smemHistogram[idx], 1);
      smemIndices[dstIdx] = rowIt;
    } else if (idx == thresholdBinIdx) {
      int dstIdx = atomicAdd(&smemFinalDstIdx[0], 1);
      if (dstIdx < kNumFinalItems) {
        smemFinal.items.logits[dstIdx] = logit;
        smemFinal.items.indices[dstIdx] = rowIt;
      }
    }
  }

  // Make sure the elements are in shared memory.
  __syncthreads();

  // The logits of the elements to be sorted in the final pass.
  float finalLogits[kNumFinalItemsPerThread];
  // The indices of the elements to be sorted in the final pass.
  int finalIndices[kNumFinalItemsPerThread];

// Init.
#pragma unroll
  for (int ii = 0; ii < kNumFinalItemsPerThread; ++ii) {
    finalLogits[ii] = -FLT_MAX;
  }

// Read the elements from SMEM.
#pragma unroll
  for (int ii = 0; ii < kNumFinalItemsPerThread; ++ii) {
    int srcIdx = ii * kNumThreadsPerBlock + threadIdx.x;
    if (srcIdx < smemFinalDstIdx[0]) {
      finalLogits[ii] = smemFinal.items.logits[srcIdx];
      finalIndices[ii] = smemFinal.items.indices[srcIdx];
    }
  }

  // Make sure the shared memory has been read.
  __syncthreads();

  // Sort the elements.
  FinalSort(smemFinal.finalSort)
      .SortDescendingBlockedToStriped(finalLogits, finalIndices);

  // Copy the data back to the shared memory storage.
  int baseIdx = thresholdBinIdx > 0 ? smemHistogram[thresholdBinIdx - 1] : 0;
#pragma unroll
  for (int ii = 0; ii < kNumFinalItemsPerThread; ++ii) {
    int srcIdx = ii * kNumThreadsPerBlock + threadIdx.x;
    int dstIdx = baseIdx + srcIdx;
    if (dstIdx < kTopK) {
      smemIndices[dstIdx] = finalIndices[ii];
    }
  }

  // Make sure the data is in shared memory.
  __syncthreads();

// Store to global memory.
#pragma unroll
  for (int ii = 0; ii < kNumTopKItemsPerThread; ++ii) {
    int offset = rowIdx * kTopK + ii * kNumThreadsPerBlock + threadIdx.x;
    outIndices[offset] =
        smemIndices[ii * kNumThreadsPerBlock + threadIdx.x] - rowStart;
  }
}

}  // namespace vllm

void apply_repetition_penalties_(
    torch::Tensor& logits,             // [num_seqs, vocab_size], in-place
    const torch::Tensor& prompt_mask,  // [num_seqs, vocab_size]
    const torch::Tensor& output_mask,  // [num_seqs, vocab_size]
    const torch::Tensor& repetition_penalties) {  // [num_seqs]
  TORCH_CHECK(logits.is_contiguous());
  TORCH_CHECK(prompt_mask.is_contiguous());
  TORCH_CHECK(output_mask.is_contiguous());
  TORCH_CHECK(repetition_penalties.is_contiguous());

  int vocab_size = logits.size(-1);
  int num_seqs = logits.size(0);

  if (num_seqs == 0) return;

  // Get number of SMs on the current device
  int sms = 0;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount,
                         logits.get_device());

  // Compute tile_num and tile_size
  int tile_num =
      std::min(vocab_size, std::max(1, (sms + num_seqs - 1) / num_seqs));
  int tile_size = (vocab_size + tile_num - 1) / tile_num;

  // Each block handles one sequence and a tile of vocab
  dim3 grid(num_seqs, tile_num);
  dim3 block(std::min(tile_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(logits));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      logits.scalar_type(), "apply_repetition_penalties_kernel", [&] {
        vllm::apply_repetition_penalties_kernel<scalar_t>
            <<<grid, block, 0, stream>>>(
                logits.data_ptr<scalar_t>(), prompt_mask.data_ptr<bool>(),
                output_mask.data_ptr<bool>(),
                repetition_penalties.data_ptr<scalar_t>(), num_seqs, vocab_size,
                tile_size);
      });
}

void top_k_per_row(const torch::Tensor& logits, const torch::Tensor& rowStarts,
                   const torch::Tensor& rowEnds, torch::Tensor& indices,
                   int64_t numRows, int64_t stride0, int64_t stride1) {
  // Compute the results on the device.
  constexpr int kNumThreadsPerBlock = 512;
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  vllm::topKPerRow<kNumThreadsPerBlock>
      <<<numRows, kNumThreadsPerBlock, 0, stream>>>(
          logits.data_ptr<float>(), rowStarts.data_ptr<int>(),
          rowEnds.data_ptr<int>(), indices.data_ptr<int>(),
          static_cast<int>(stride0), static_cast<int>(stride1));
}
