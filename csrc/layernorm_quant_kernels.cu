#include "hip/hip_runtime.h"
/*
 * This file contains the CUDA kernels for the fused quantized layernorm.
 * The kernels correspond to the kernels in layernorm_kernels.cu, except they
 * also produce quantized output directly.
 * Currently, only static fp8 quantization is supported.
 */

#include "type_convert.cuh"
#include "quantization/fp8/common.cuh"
#include "dispatch_utils.h"

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/hipcub.hpp>
#endif

namespace vllm {

// TODO(woosuk): Further optimize this kernel.
template <typename scalar_t>
__global__ void rms_norm_static_fp8_quant_kernel(
    FP8_TYPE* __restrict__ out,           // [..., hidden_size]
    const scalar_t* __restrict__ input,   // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float* __restrict__ scale,      // [1]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  // invert scale to avoid division
  float const scale_inv = 1.0f / *scale;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    float const out_norm = ((scalar_t)(x * s_variance)) * weight[idx];
    out[blockIdx.x * hidden_size + idx] =
        scaled_fp8_conversion<true>(out_norm, scale_inv);
  }
}

/* Function specialization in the case of FP16/BF16 tensors.
   Additional optimizations we can make in this case are
   packed and vectorized operations, which help with the
   memory latency bottleneck. */
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width > 0) && _typeConvert<scalar_t>::exists>
fused_add_rms_norm_static_fp8_quant_kernel(
    FP8_TYPE* __restrict__ out,           // [..., hidden_size]
    scalar_t* __restrict__ input,         // [..., hidden_size]
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float* __restrict__ scale,      // [1]
    const float epsilon, const int num_tokens, const int hidden_size) {
  // Sanity checks on our vector struct and type-punned pointer arithmetic
  static_assert(std::is_pod_v<_f16Vec<scalar_t, width>>);
  static_assert(sizeof(_f16Vec<scalar_t, width>) == sizeof(scalar_t) * width);

  const int vec_hidden_size = hidden_size / width;
  __shared__ float s_variance;
  float variance = 0.0f;
  /* These and the argument pointers are all declared `restrict` as they are
     not aliased in practice. Argument pointers should not be dereferenced
     in this kernel as that would be undefined behavior */
  auto* __restrict__ input_v =
      reinterpret_cast<_f16Vec<scalar_t, width>*>(input);
  auto* __restrict__ residual_v =
      reinterpret_cast<_f16Vec<scalar_t, width>*>(residual);
  auto* __restrict__ weight_v =
      reinterpret_cast<const _f16Vec<scalar_t, width>*>(weight);

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _f16Vec<scalar_t, width> temp = input_v[id];
    temp += residual_v[id];
    variance += temp.sum_squares();
    residual_v[id] = temp;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  // invert scale to avoid division
  float const scale_inv = 1.0f / *scale;

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _f16Vec<scalar_t, width> temp = residual_v[id];
    temp *= s_variance;
    temp *= weight_v[idx];
#pragma unroll
    for (int i = 0; i < width; ++i) {
      out[id * width + i] =
          scaled_fp8_conversion<true>(float(temp.data[i]), scale_inv);
    }
  }
}

/* Generic fused_add_rms_norm_kernel
   The width field is not used here but necessary for other specializations.
 */
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width == 0) || !_typeConvert<scalar_t>::exists>
fused_add_rms_norm_static_fp8_quant_kernel(
    FP8_TYPE* __restrict__ out,           // [..., hidden_size]
    scalar_t* __restrict__ input,         // [..., hidden_size]
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float* __restrict__ scale,      // [1]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    scalar_t z = input[blockIdx.x * hidden_size + idx];
    z += residual[blockIdx.x * hidden_size + idx];
    float x = (float)z;
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = z;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  // invert scale to avoid division
  float const scale_inv = 1.0f / *scale;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)residual[blockIdx.x * hidden_size + idx];
    float const out_norm = ((scalar_t)(x * s_variance)) * weight[idx];
    out[blockIdx.x * hidden_size + idx] =
        scaled_fp8_conversion<true>(out_norm, scale_inv);
  }
}

}  // namespace vllm

void rms_norm_static_fp8_quant(torch::Tensor& out,     // [..., hidden_size]
                               torch::Tensor& input,   // [..., hidden_size]
                               torch::Tensor& weight,  // [hidden_size]
                               torch::Tensor& scale,   // [1]
                               double epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    vllm::rms_norm_static_fp8_quant_kernel<scalar_t>
        <<<grid, block, 0, stream>>>(
            out.data_ptr<FP8_TYPE>(), input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(), scale.data_ptr<float>(), epsilon,
            num_tokens, hidden_size);
  });
}

#define LAUNCH_FUSED_ADD_RMS_NORM(width)                                    \
  VLLM_DISPATCH_FLOATING_TYPES(                                             \
      input.scalar_type(), "fused_add_rms_norm_kernel", [&] {               \
        vllm::fused_add_rms_norm_static_fp8_quant_kernel<scalar_t, width>   \
            <<<grid, block, 0, stream>>>(                                   \
                out.data_ptr<FP8_TYPE>(), input.data_ptr<scalar_t>(),       \
                residual.data_ptr<scalar_t>(), weight.data_ptr<scalar_t>(), \
                scale.data_ptr<float>(), epsilon, num_tokens, hidden_size); \
      });

void fused_add_rms_norm_static_fp8_quant(
    torch::Tensor& out,       // [..., hidden_size],
    torch::Tensor& input,     // [..., hidden_size]
    torch::Tensor& residual,  // [..., hidden_size]
    torch::Tensor& weight,    // [hidden_size]
    torch::Tensor& scale,     // [1]
    double epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  /* This kernel is memory-latency bound in many scenarios.
     When num_tokens is large, a smaller block size allows
     for increased block occupancy on CUs and better latency
     hiding on global mem ops. */
  const int max_block_size = (num_tokens < 256) ? 1024 : 256;
  dim3 block(std::min(hidden_size, max_block_size));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  /*If the tensor types are FP16/BF16, try to use the optimized kernel
    with packed + vectorized ops.
    Max optimization is achieved with a width-8 vector of FP16/BF16s
    since we can load at most 128 bits at once in a global memory op.
    However, this requires each tensor's data to be aligned to 16
    bytes.
   */
  auto inp_ptr = reinterpret_cast<std::uintptr_t>(input.data_ptr());
  auto res_ptr = reinterpret_cast<std::uintptr_t>(residual.data_ptr());
  auto wt_ptr = reinterpret_cast<std::uintptr_t>(weight.data_ptr());
  bool ptrs_are_aligned =
      inp_ptr % 16 == 0 && res_ptr % 16 == 0 && wt_ptr % 16 == 0;
  if (ptrs_are_aligned && hidden_size % 8 == 0) {
    LAUNCH_FUSED_ADD_RMS_NORM(8);
  } else {
    LAUNCH_FUSED_ADD_RMS_NORM(0);
  }
}
