#include "hip/hip_runtime.h"
/**
 * This is a standalone test for custom allreduce.
 * To compile, make sure you have MPI and NCCL installed in your system.
 * export MPI_HOME=XXX
 * nvcc -O2 -arch=native -std=c++17 custom_all_reduce_test.cu -o
 * custom_all_reduce_test -lnccl -I${MPI_HOME}/include -lmpi
 *
 * Warning: this C++ test is not designed to be very readable and was used
 * during the rapid prototyping process.
 *
 * To run:
 * mpirun -np 8 ./custom_all_reduce_test
 */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#include <limits>
#include <vector>

#include "hip/hip_runtime_api.h"
#include "custom_all_reduce.cuh"
#include "mpi.h"
#include "nccl.h"

#define MPICHECK(cmd)                                                  \
  do {                                                                 \
    int e = cmd;                                                       \
    if (e != MPI_SUCCESS) {                                            \
      printf("Failed: MPI error %s:%d '%d'\n", __FILE__, __LINE__, e); \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

#define NCCLCHECK(cmd)                                              \
  do {                                                              \
    ncclResult_t r = cmd;                                           \
    if (r != ncclSuccess) {                                         \
      printf("Failed, NCCL error %s:%d '%s'\n", __FILE__, __LINE__, \
             ncclGetErrorString(r));                                \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

__global__ void dummy_kernel() {
  for (int i = 0; i < 100; i++) __nanosleep(1000000);  // 100ms
}

template <typename T>
__global__ void set_data(T *data, int size, int myRank) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    data[idx] = myRank * 0.11f;
  }
}

template <typename T>
__global__ void convert_data(const T *data1, const T *data2, double *fdata1,
                             double *fdata2, int size) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    fdata1[idx] = data1[idx];
    fdata2[idx] = data2[idx];
  }
}

__global__ void init_rand(hiprandState_t *state, int size, int nRanks) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    for (int i = 0; i < nRanks; i++) {
      hiprand_init(i + 1, idx, 0, &state[idx * nRanks + i]);
    }
  }
}

template <typename T>
__global__ void gen_data(hiprandState_t *state, T *data, double *ground_truth,
                         int myRank, int nRanks, int size) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    double sum = 0.0;
    for (int i = 0; i < nRanks; i++) {
      double val = hiprand_uniform_double(&state[idx * nRanks + i]) * 4;
      T hval = val;  // downcast first
      sum += static_cast<double>(hval);
      if (i == myRank) data[idx] = hval;
    }
    ground_truth[idx] = sum;
  }
}

template <typename T>
void run(int myRank, int nRanks, ncclComm_t &comm, int threads, int block_limit,
         int data_size) {
  T *result;
  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUDACHECK(hipMalloc(&result, data_size * sizeof(T)));
  CUDACHECK(hipMemset(result, 0, data_size * sizeof(T)));

  hipIpcMemHandle_t self_data_handle;
  hipIpcMemHandle_t data_handles[8];
  vllm::Metadata *buffer;
  T *self_data_copy;
  /**
   * Allocate IPC buffer
   *
   * The first section is a temporary buffer for storing intermediate allreduce
   * results, if a particular algorithm requires it. The second section is for
   * the input to the allreduce. The actual API takes the input pointer as an
   * argument (that is, they can and usually should be allocated separately).
   * But since the input pointers and the temporary buffer all require IPC
   * registration, they are allocated and registered together in the test for
   * convenience.
   */
  CUDACHECK(
      hipMalloc(&buffer, 2 * data_size * sizeof(T) + sizeof(vllm::Metadata)));
  CUDACHECK(hipMemset(buffer, 0,
                       2 * data_size * sizeof(T) + sizeof(vllm::Metadata)));
  CUDACHECK(hipMalloc(&self_data_copy, data_size * sizeof(T)));
  CUDACHECK(hipIpcGetMemHandle(&self_data_handle, buffer));

  MPICHECK(MPI_Allgather(&self_data_handle, sizeof(hipIpcMemHandle_t),
                         MPI_BYTE, data_handles, sizeof(hipIpcMemHandle_t),
                         MPI_BYTE, MPI_COMM_WORLD));

  void *rank_data;
  size_t rank_data_sz = 16 * 1024 * 1024;
  CUDACHECK(hipMalloc(&rank_data, rank_data_sz));
  std::vector<int64_t> offsets(nRanks, 0);
  vllm::CustomAllreduce fa(buffer, rank_data, rank_data_sz, data_handles,
                           offsets, myRank);
  auto *self_data =
      reinterpret_cast<T *>(reinterpret_cast<char *>(buffer) +
                            sizeof(vllm::Metadata) + data_size * sizeof(T));
  // hack buffer registration
  {
    std::vector<std::string> handles;
    handles.reserve(nRanks);
    for (int i = 0; i < nRanks; i++) {
      char *begin = (char *)&data_handles[i];
      char *end = (char *)&data_handles[i + 1];
      handles.emplace_back(begin, end);
    }
    std::vector<int64_t> offsets(
        nRanks, sizeof(vllm::Metadata) + data_size * sizeof(T));
    fa.register_buffer(handles, offsets, self_data);
  }

  double *ground_truth;
  CUDACHECK(hipHostMalloc(&ground_truth, data_size * sizeof(double)));
  hiprandState_t *states;
  CUDACHECK(hipMalloc(&states, sizeof(hiprandState_t) * nRanks * data_size));
  init_rand<<<108, 1024, 0, stream>>>(states, data_size, nRanks);
  gen_data<T><<<108, 1024, 0, stream>>>(states, self_data, ground_truth, myRank,
                                        nRanks, data_size);
  CUDACHECK(hipMemcpyAsync(self_data_copy, self_data, data_size * sizeof(T),
                            hipMemcpyDeviceToDevice, stream));
  hipEvent_t start, stop;
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  ncclDataType_t ncclDtype;
  if (std::is_same<T, half>::value) {
    ncclDtype = ncclFloat16;
  } else if (std::is_same<T, hip_bfloat16>::value) {
    ncclDtype = ncclBfloat16;
  } else {
    ncclDtype = ncclFloat;
  }

  dummy_kernel<<<1, 1, 0, stream>>>();
  constexpr int warmup_iters = 5;
  constexpr int num_iters = 25;
  // warmup
  for (int i = 0; i < warmup_iters; i++) {
    NCCLCHECK(ncclAllReduce(result, result, data_size, ncclDtype, ncclSum, comm,
                            stream));
  }
  CUDACHECK(hipEventRecord(start, stream));
  for (int i = 0; i < num_iters; i++) {
    NCCLCHECK(ncclAllReduce(result, result, data_size, ncclDtype, ncclSum, comm,
                            stream));
  }
  CUDACHECK(hipEventRecord(stop, stream));
  CUDACHECK(hipStreamSynchronize(stream));
  float allreduce_ms = 0;
  hipEventElapsedTime(&allreduce_ms, start, stop);

  // if (myRank == 1) dummy_kernel<<<1, 1, 0, stream>>>();
  // set_data<T><<<16, 1024, 0, stream>>>(self_data, data_size, myRank);

  dummy_kernel<<<1, 1, 0, stream>>>();
  // warm up
  for (int i = 0; i < warmup_iters; i++) {
    fa.allreduce<T>(stream, self_data, result, data_size, threads, block_limit);
  }
  CUDACHECK(hipEventRecord(start, stream));
  for (int i = 0; i < num_iters; i++) {
    fa.allreduce<T>(stream, self_data, result, data_size, threads, block_limit);
  }
  CUDACHECK(hipEventRecord(stop, stream));
  CUDACHECK(hipStreamSynchronize(stream));

  float duration_ms = 0;
  hipEventElapsedTime(&duration_ms, start, stop);
  if (myRank == 0)
    printf(
        "Rank %d done, nGPUs:%d, sz (kb): %d, %d, %d, my time:%.2fus, nccl "
        "time:%.2fus\n",
        myRank, nRanks, data_size * sizeof(T) / 1024, threads, block_limit,
        duration_ms * 1e3 / num_iters, allreduce_ms * 1e3 / num_iters);

  // And wait for all the queued up work to complete
  CUDACHECK(hipStreamSynchronize(stream));

  NCCLCHECK(ncclAllReduce(self_data_copy, self_data, data_size, ncclDtype,
                          ncclSum, comm, stream));

  double *nccl_result, *my_result;
  CUDACHECK(hipHostMalloc(&nccl_result, data_size * sizeof(double)));
  CUDACHECK(hipHostMalloc(&my_result, data_size * sizeof(double)));

  convert_data<T><<<108, 1024, 0, stream>>>(self_data, result, nccl_result,
                                            my_result, data_size);
  CUDACHECK(hipStreamSynchronize(stream));

  for (unsigned long j = 0; j < data_size; j++) {
    auto diff = abs(nccl_result[j] - my_result[j]);
    if (diff >= 1e-2) {
      printf("Rank %d: Verification mismatch at %lld: %f != (my) %f, gt=%f\n",
             myRank, j, nccl_result[j], my_result[j], ground_truth[j]);
      break;
    }
  }

  long double nccl_diffs = 0.0;
  long double my_diffs = 0.0;
  for (int j = 0; j < data_size; j++) {
    nccl_diffs += abs(nccl_result[j] - ground_truth[j]);
    my_diffs += abs(my_result[j] - ground_truth[j]);
  }
  if (myRank == 0)
    std::cout << "average abs diffs: nccl: " << nccl_diffs / data_size
              << " me: " << my_diffs / data_size << std::endl;

  CUDACHECK(hipFree(result));
  CUDACHECK(hipFree(self_data_copy));
  CUDACHECK(hipFree(rank_data));
  CUDACHECK(hipFree(buffer));
  CUDACHECK(hipFree(states));
  CUDACHECK(hipHostFree(ground_truth));
  CUDACHECK(hipHostFree(nccl_result));
  CUDACHECK(hipHostFree(my_result));
  CUDACHECK(hipStreamDestroy(stream));
}

int main(int argc, char **argv) {
  int nRanks, myRank;
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
  CUDACHECK(hipSetDevice(myRank));
  ncclUniqueId id;
  ncclComm_t comm;
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast(static_cast<void *>(&id), sizeof(id), MPI_BYTE, 0,
                     MPI_COMM_WORLD));
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  hipProfilerStart();
  // for (int threads : {256, 512}) {
  //   for (int block_limit = 16; block_limit < 112; block_limit += 4) {
  //     run<half>(myRank, nRanks, comm, threads, block_limit, 4096 * 1024);
  //   }
  // }
  for (int sz = 512; sz <= (32 << 20); sz *= 2) {
    run<half>(myRank, nRanks, comm, 512, 36, sz + 8 * 50);
  }

  hipProfilerStop();
  return EXIT_SUCCESS;
}
