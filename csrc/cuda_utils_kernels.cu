
#include <hip/hip_runtime.h>
int get_device_attribute(
    int attribute,
    int device_id)
{
    int device, value;
    if (device_id < 0) {
        hipGetDevice(&device);
    }
    else {
        device = device_id;
    }
    hipDeviceGetAttribute(&value, static_cast<hipDeviceAttribute_t>(attribute), device);
    return value;
}
