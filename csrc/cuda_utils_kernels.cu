
#include <hip/hip_runtime.h>
#ifdef USE_ROCM
  #include <hip/hip_runtime.h>
  #include <hip/hip_runtime_api.h>
#endif
int get_device_attribute(
    int attribute,
    int device_id)
{
    int device, value;
    if (device_id < 0) {
        hipGetDevice(&device);
    }
    else {
        device = device_id;
    }
    hipDeviceGetAttribute(&value, static_cast<hipDeviceAttribute_t>(attribute), device);
    return value;
}


int get_max_shared_memory_per_block_device_attribute(
    int device_id)
{
int attribute;    
// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
// cudaDevAttrMaxSharedMemoryPerBlockOptin = 97 if not is_hip() else 74

#ifdef USE_ROCM
    attribute = hipDeviceAttributeMaxSharedMemoryPerBlock;
#else
    attribute = hipDeviceAttributeSharedMemPerBlockOptin;
#endif

    return get_device_attribute(attribute, device_id);
}
