
#include <hip/hip_runtime.h>
#ifdef USE_ROCM
  #include <hip/hip_runtime.h>
#endif
int get_device_attribute(
    int attribute,
    int device_id)
{
    int device, value;
    if (device_id < 0) {
        hipGetDevice(&device);
    }
    else {
        device = device_id;
    }
    hipDeviceGetAttribute(&value, static_cast<hipDeviceAttribute_t>(attribute), device);
    return value;
}
